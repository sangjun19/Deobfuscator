#include "hip/hip_runtime.h"
// Repository: greenhoody/Fluid-Simulation
// File: GUI/Factory/DeviceCudaSimulation.cu

﻿#include <hip/hip_runtime.h>
#include "DeviceCudaSimulation.cuh"
#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime_api.h> 
#include <hip/hip_runtime.h>


#define IX(i,j) ((i)+(N+2)*(j))

//void CudaSimulation::add_source(int N, float* x, float* s, float dt)
//{
//	int i, size = (N + 2) * (N + 2);
//	for (i = 0; i < size; i++) x[i] += dt * s[i];
//}


// wątek zero rogi, reszta krawędzie, wiem że nie optymalne, ale może późjniej nad tym pomyśle i wywali się przy zamałej ilości wątków, na początku synchronizacja grid. Ta funkcja jest wywołyuwana wiele razy, więc wiele synchronizacji więc trzeba sprawdzić póxniej czy potrzebne
__device__ void set_bnd(int N, int b, float* x)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	
	cooperative_groups::grid_group g = cooperative_groups::this_grid();
	g.sync();

	if (index > 0 && index < N + 2) {
		x[IX(0, index)] = b == 1 ? -x[IX(1, index)] : x[IX(1, index)];
		x[IX(N + 1, index)] = b == 1 ? -x[IX(N, index)] : x[IX(N, index)];
		x[IX(index, 0)] = b == 2 ? -x[IX(index, 1)] : x[IX(index, 1)];
		x[IX(index, N + 1)] = b == 2 ? -x[IX(index, N)] : x[IX(index, N)];
	}

	if (index == 0)
	{
		x[IX(0, 0)] = 0.5f * (x[IX(1, 0)] + x[IX(0, 1)]);
		x[IX(0, N + 1)] = 0.5f * (x[IX(1, N + 1)] + x[IX(0, N)]);
		x[IX(N + 1, 0)] = 0.5f * (x[IX(N, 0)] + x[IX(N + 1, 1)]);
		x[IX(N + 1, N + 1)] = 0.5f * (x[IX(N, N + 1)] + x[IX(N + 1, N)]);
	}
//	g.sync();

}

__device__ void diffuse(int N, int b, float* x, float* x0, float diff, float dt)
{
	int n = N * N;
	float a = dt * diff * n;
	
	
	int index = 0;
	int cores = gridDim.x * blockDim.x;


	for (int k = 0; k < 20; k++) {
		
		// wykonanie przypadających komórek
		index = blockIdx.x * blockDim.x + threadIdx.x;
		while (index < n) {
			// +1 po od 1 zaczynały się pętle, a nie od zera
			int j = (index / N) + 1;
			int i = (index % N) + 1;

			x[IX(i, j)] = (x0[IX(i, j)] + a * (x[IX(i - 1, j)] + x[IX(i + 1, j)] +
				x[IX(i, j - 1)] + x[IX(i, j + 1)])) / (1 + (4 * a));
			index += cores;
		}
		set_bnd(N, b, x);
	}
}

__device__ void advect(int N, int b, float* d, float* d0, float* u, float* v, float dt)
{

	int n = N * N;
	int cores = gridDim.x * blockDim.x;
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	int i0, j0, i1, j1, i, j;
	float x, y, s0, t0, s1, t1, dt0;

	while(index < n)
	{
		// +1 ponieważ od 1 zaczyna się pętla, a nie od zera
		j = (index / N) + 1;
		i = (index % N) + 1;
		dt0 = dt * (float)N;
		//for (int i = 1; i <= N; i++) {
		//	for (int j = 1; j <= N; j++) {

		x = (float)i - dt0 * u[IX(i, j)];
		if (x < 0.5) x = 0.5f;
		if (x > N + 0.5) x = N + 0.5f;
		i0 = (int)x;
		i1 = i0 + 1;

		//proporcje ile gęstości wylądowało z których komórek
		s1 = x - (float)i0;
		s0 = 1.0f - s1;

		y = (float)j - dt0 * v[IX(i, j)];
		if (y < 0.5) y = 0.5f;
		if (y > N + 0.5) y = N + 0.5f;
		j0 = (int)y;
		j1 = j0 + 1;

		//proporcje ile gęstości wylądowało z których komórek
		t1 = y - (float)j0;
		t0 = 1.0f - t1;

		d[IX(i, j)] = s0 * (t0 * d0[IX(i0, j0)] + t1 * d0[IX(i0, j1)]) +
			s1 * (t0 * d0[IX(i1, j0)] + t1 * d0[IX(i1, j1)]);

		//	}
		//}

		index += cores;
	}

	set_bnd(N, b, d);
}

__device__ void project(int N, float* u, float* v, float* p, float* div)
{

	int n = N * N;
	int cores = blockDim.x * gridDim.x;

	int index = blockIdx.x * blockDim.x + threadIdx.x;

	int i, j;
	
	// +1 po od 1 zaczynały się pętle, a nie od zera

	float h = 1.0f / N;

	while (index < n) {
		j = (index / N) + 1;
		i = (index % N) + 1;

		div[IX(i, j)] = -0.5f * h * (u[IX(i + 1, j)] - u[IX(i - 1, j)] +
			v[IX(i, j + 1)] - v[IX(i, j - 1)]);
		p[IX(i, j)] = 0.0f;

		index += cores;
	 }

	set_bnd(N, 0, div);
	set_bnd(N, 0, p);


	for (int k = 0; k < 20; k++) {
		
		//int dlugosc = 1;
		//int zmiana = 1;
		////przekątna zaczyna się od dwóch ponieważ pomijamy ramkę
		//for (int przekotna = 2; przekotna <= N; przekotna++) {
		//	index = blockIdx.x * blockDim.x + threadIdx.x;
		//	// 4*dlugosc - index odpowiada za to, aby były wykorzystywane tylko rdzenie dla których są dane.
		//	while ( (4 * dlugosc - index) > 0) {

		//		switch (index % 4)
		//		{
		//		case 0:
		//			i = przekotna >= N/2 ? N/2 : (przekotna - 1);
		//			i -= (index/4);
		//			j = przekotna - i;
		//			break;
		//		case 1:
		//			i = przekotna >= N / 2 ? N / 2 : (przekotna - 1);
		//			i -= ((index-1) / 4);
		//			j = przekotna - i;
		//			i = N - i + 1;
		//			break;
		//		case 2:
		//			i = przekotna >= N / 2 ? N / 2 : (przekotna - 1);
		//			i -= ((index-2) / 4);
		//			j = przekotna - i;
		//			j = N - j + 1;
		//			break;
		//		case 3:
		//			i = przekotna >= N / 2 ? N / 2 : (przekotna - 1);
		//			i -= ((index - 3) / 4);
		//			j = przekotna - i;
		//			i = N - i + 1;
		//			j = N - j + 1;
		//			break;
		//		}

		//		p[IX(i, j)] = (div[IX(i, j)] + p[IX(i - 1, j)] + p[IX(i + 1, j)] +
		//			p[IX(i, j - 1)] + p[IX(i, j + 1)]) / 4;
		//		index += cores;
		//	}

		//	if (dlugosc == (N/2)) {
		//		zmiana = -1;
		//	}

		//	dlugosc += zmiana;
		//}

		//set_bnd(N, 0, p);





		
		int dlugosc = 1;
		int zmiana = 1;
		//przekątna zaczyna się od dwóch ponieważ pomijamy ramkę
		for (int przekotna = 2; przekotna <= 2 * N; przekotna++) {
			index = blockIdx.x * blockDim.x + threadIdx.x;
			// dlugosc - index odpowiada za to, aby były wykorzystywane tylko rdzenie dla których są dane.
			while (index <= N && (dlugosc - index) > 0) {

				i = przekotna >= N ? N : (przekotna - 1);
				i -= index;
				j = przekotna - i;

				p[IX(i, j)] = (div[IX(i, j)] + p[IX(i - 1, j)] + p[IX(i + 1, j)] +
					p[IX(i, j - 1)] + p[IX(i, j + 1)]) / 4;
				index += cores;
			}

			if (dlugosc == N) {
				zmiana = -1;
			}

			dlugosc += zmiana;
		}

		set_bnd(N, 0, p);
	
	}

	index = blockIdx.x * blockDim.x + threadIdx.x;
	while (index < n)
	{
		j = (index / N) + 1;
		i = (index % N) + 1;

		u[IX(i, j)] -= 0.5f * (p[IX(i + 1, j)] - p[IX(i - 1, j)]) / h;
		v[IX(i, j)] -= 0.5f * (p[IX(i, j + 1)] - p[IX(i, j - 1)]) / h;
		index += cores;
	}

	set_bnd(N, 1, u);
	set_bnd(N, 2, v);
}

__global__ void cuda_NextFrame(int N, float* d_dens, float* d_dens_prev, float* d_u, float* d_v, float* d_u_prev, float* d_v_prev, float visc, float diff , float dt) 
{

	diffuse(N, 1, d_u_prev, d_u, visc, dt);
	diffuse(N, 2, d_v_prev, d_v, visc, dt);
	project(N, d_u_prev, d_v_prev, d_u, d_v);
	advect(N, 1, d_u, d_u_prev, d_u_prev, d_v_prev, dt);
	advect(N, 2, d_v, d_v_prev, d_u_prev, d_v_prev, dt);
	project(N, d_u, d_v, d_u_prev, d_v_prev);

	diffuse(N, 0, d_dens_prev, d_dens, diff, dt);
	advect(N, 0, d_dens, d_dens_prev, d_u, d_v, dt);
}

__global__ void addDensity(int N, float* d_dens, int x1, int x2, int y1, int y2, float dens)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int cores = blockDim.x * gridDim.x;
	int deltay = y2 - y1;
	int deltax = x2 - x1;
	int n = deltay * deltax;

	while (index < n) 
	{
		int j = (index / deltax);
		int i = (index % deltax);
		d_dens[IX(x1 + i, y1 + j)] += dens;
		if (d_dens[IX(x1 + i, y1 + j)] > 1)
		{
			d_dens[IX(x1 + i, y1 + j)] = 1;
		}
		index += cores;
	}
}

__global__ void addVelocity(int N, float* u, float* v, int x, int y, int r, float u_velocity, float v_velocity)
{
	//wyliczanie które pole jest obsługiwane przez dany rdzeń
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int cores = blockDim.x * gridDim.x;
	int j = (index / (r*2));
	int i = (index % (r * 2));
	int r_square = r * r;
	//ograniczenie, aby za daleko nie leciało, a najbliższy int większy od 3,14 to 4
	int n = r_square * 4;
	x -= r;
	y -= r;

	while (index < n) {

		if (N > x + i && x + i > 0 && N > y + j && y + j > 0 && i * i + j * j <= r_square)
		{
			u[IX(x + i, y + j)] += u_velocity;
			v[IX(x + i, y + j)] += v_velocity;
		}

		index += cores;
	}
}
#include "hip/hip_runtime.h"
// Repository: henryfriedlander/CUDA-LBM-simulator
// File: temps.cu

//Curl calcs/rendering:
__device__
int computeCurlMiddleCase(int col, int row, lbm_node * array1) {
	return (array1[getIndex(row, col + 1)].yvel - array1[getIndex(row, col - 1)].yvel) - (array1[getIndex(row + 1, col)].xvel - array1[getIndex(row - 1, col)].xvel);
}
__device__
int computeCurlEdgeCase(int col, int row, lbm_node * array1) {
	return 2 * (array1[getIndex(row, col)].yvel - array1[getIndex(row, col - 1)].yvel) - (array1[getIndex(row, col)].xvel - array1[getIndex(row - 1, col)].xvel);
}
__device__
void updatePictureCurl(int col, int row, uchar4 * picture, lbm_node * array) {
	//printf("updating dat curl!\n");
	int nColors = 255;
	int contrast = 70;
	int colorIndex;
	if (0 < col && col < W - 1) {
		if (0 < row && row < H - 1) {
			//picture[getIndex(x,y)]
			colorIndex = (int)(nColors * (0.5 + computeCurlMiddleCase(col, row, array) * contrast * 0.3));
		}
		//else {
		//	//picture[getIndex(x,y)]
		//	colorIndex = (int)(nColors * (0.5 + computeCurlEdgeCase(col, row, array) * contrast * 0.3));
		//}
	}

	picture[getIndex(row, col)].z = colorIndex;

}

//what to render switch-case
__device__
void computeColor(int col, int row, uchar4 * picture, int display, lbm_node * before, lbm_node * after) {

	switch (display) {
	case 1: // one is curl
		updatePictureCurl(col, row, picture, before);
	}
}

__global__
void test(lbm_node * before, lbm_node * after, unsigned char * barrier, int display, uchar4 * picture) {
	printf("before: %x\nafter: %x\nbarrier: %x\n display: %d\n", before, after, barrier, display); //, picture);
}

__global__
void collide(lbm_node * before, lbm_node * after, unsigned char * barrier, int display, uchar4 * picture) {
	const int col = blockIdx.x * blockDim.x + threadIdx.x;
	const int row = blockIdx.y * blockDim.y + threadIdx.y;
	const int index0 = getIndex(row, col);

	if (col < 0 || col >= W || row < 0 || row >= H)
		return;

	float n, one9thn, one36thn, vx, vy, vx2, vy2, vx3, vy3, vxvy2, v2, v215;

	// particle collision code

	if (index0 < 0) {
		return;
	}

	if (barrier[index0] == 0) {
		n = before[index0].v0 + before[index0].vN + before[index0].vS + before[index0].vE + before[index0].vW + before[index0].vNW + before[index0].vNE + before[index0].vSW + before[index0].vSE;
		before[index0].density = n; // macroscopic density may be needed for plotting
		one9thn = one9th * n;
		one36thn = one36th * n;
		if (n > 0) {
			vx = (before[index0].vE + before[index0].vNE + before[index0].vSE - before[index0].vW - before[index0].vNW - before[index0].vSW) / n;
		}
		else vx = 0;
		before[index0].xvel = vx; // may be needed for plotting
		if (n > 0) {
			vy = (before[index0].vN + before[index0].vNE + before[index0].vNW - before[index0].vS - before[index0].vSE - before[index0].vSW) / n;
		}
		else vy = 0;
		before[index0].yvel = vy; // may be needed for plotting
		vx3 = 3 * vx;
		vy3 = 3 * vy;
		vx2 = vx * vx;
		vy2 = vy * vy;
		vxvy2 = 2 * vx * vy;
		v2 = vx2 + vy2;
		before[index0].speed2 = v2; // may be needed for plotting
		v215 = 1.5 * v2;
		after[index0].v0 = before[index0].v0 + omega * (four9ths * n * (1 - v215) - before[index0].v0);
		after[index0].vE = before[index0].vE + omega * (one9thn * (1 + vx3 + 4.5 * vx2 - v215) - before[index0].vE);
		after[index0].vW = before[index0].vW + omega * (one9thn * (1 - vx3 + 4.5 * vx2 - v215) - before[index0].vW);
		after[index0].vN = before[index0].vN + omega * (one9thn * (1 + vy3 + 4.5 * vy2 - v215) - before[index0].vN);
		after[index0].vS = before[index0].vS + omega * (one9thn * (1 - vy3 + 4.5 * vy2 - v215) - before[index0].vS);
		after[index0].vNE = before[index0].vNE + omega * (one36thn * (1 + vx3 + vy3 + 4.5 * (v2 + vxvy2) - v215) - before[index0].vNE);
		after[index0].vNW = before[index0].vNW + omega * (one36thn * (1 - vx3 + vy3 + 4.5 * (v2 - vxvy2) - v215) - before[index0].vNW);
		after[index0].vSE = before[index0].vSE + omega * (one36thn * (1 + vx3 - vy3 + 4.5 * (v2 - vxvy2) - v215) - before[index0].vSE);
		after[index0].vSW = before[index0].vSW + omega * (one36thn * (1 - vx3 - vy3 + 4.5 * (v2 + vxvy2) - v215) - before[index0].vSW);
	}

	//printf("calling render!\n");
	computeColor(col, row, picture, display, before, after);
}

__global__
void stream(lbm_node * before, lbm_node * after, unsigned char * barrier) {

	const int col = blockIdx.x * blockDim.x + threadIdx.x;
	const int row = blockIdx.y * blockDim.y + threadIdx.y;

	if (col < 1 || col >= W - 1 || row < 1 || row >= H - 1)
		return;

	int index0 = getIndex(row, col);
	// these are the indices which will be used to retrieve the data for the current vectors
	int indexN = getIndex(row - 1, col);
	int indexS = getIndex(row + 1, col);
	int indexE = getIndex(row, col - 1);
	int indexW = getIndex(row, col + 1);
	int indexNE = getIndex(row - 1, col - 1);
	int indexNW = getIndex(row - 1, col + 1);
	int indexSE = getIndex(row + 1, col - 1);
	int indexSW = getIndex(row + 1, col + 1);

	// particle streaming
	if (0 <= row && row < H - 1) {
		if (0 < col < W - 1) {
			after[index0].vN = before[indexN].vN; // move the north-moving particles
			after[index0].vNW = before[indexNW].vNW; // and the northwest-moving particles
			after[index0].vW = before[indexW].vW; // move the west-moving particles
			after[index0].vSW = before[indexSW].vSW; // and the southwest-moving particles
		}
	}
	if (0 < row && row <= H - 1) {
		if (0 < col && col <= W - 1) {
			after[index0].vE = before[indexE].vE; // move the east-moving particles
			after[index0].vNE = before[indexNE].vNE; // and the northeast-moving particles
		}
		if (0 <= col && col < W - 1) {
			after[index0].vS = before[indexS].vS; // move the south-moving particles
			after[index0].vSE = before[indexSE].vSE; // and the southeast-moving particles
		}
	}
	// CHECK THIS CODE IF BARRIERS DO NOT WORK
	if (row == 0) {
		after[index0].vS = before[indexS].vS;
		if (!barrier[index0]) {
			after[index0].vE = one9th * (1 + 3 * v + 3 * v * v);
			after[index0].vNE = one36th * (1 + 3 * v + 3 * v * v);
			after[index0].vSE = one36th * (1 + 3 * v + 3 * v * v);
		}
	}
	if (row == H - 1) {
		after[index0].vN = before[indexN].vN;
		if (!barrier[index0]) {
			after[index0].vW = one9th * (1 - 3 * v + 3 * v * v);
			after[index0].vNW = one36th * (1 - 3 * v + 3 * v * v);
			after[index0].vSW = one36th * (1 - 3 * v + 3 * v * v);
		}
	}
	if (col == 0) {
		after[index0].v0 = four9ths * (1 - 1.5 * v * v);
		after[index0].vE = one9th * (1 + 3 * v + 3 * v * v);
		after[index0].vW = one9th * (1 - 3 * v + 3 * v * v);
		after[index0].vN = one9th * (1 - 1.5 * v * v);
		after[index0].vS = one9th * (1 - 1.5 * v * v);
		after[index0].vNE = one36th * (1 + 3 * v + 3 * v * v);
		after[index0].vSE = one36th * (1 + 3 * v + 3 * v * v);
		after[index0].vNW = one36th * (1 - 3 * v + 3 * v * v);
		after[index0].vSW = one36th * (1 - 3 * v + 3 * v * v);
	}
	else if (col == W - 1) {
		after[index0].v0 = four9ths * (1 - 1.5 * v * v);
		after[index0].vE = one9th * (1 + 3 * v + 3 * v * v);
		after[index0].vW = one9th * (1 - 3 * v + 3 * v * v);
		after[index0].vN = one9th * (1 - 1.5 * v * v);
		after[index0].vS = one9th * (1 - 1.5 * v * v);
		after[index0].vNE = one36th * (1 + 3 * v + 3 * v * v);
		after[index0].vSE = one36th * (1 + 3 * v + 3 * v * v);
		after[index0].vNW = one36th * (1 - 3 * v + 3 * v * v);
		after[index0].vSW = one36th * (1 - 3 * v + 3 * v * v);
	}
}



__global__
void bounce(lbm_node * before, lbm_node * after, unsigned char * barrier) {

	const int col = blockIdx.x * blockDim.x + threadIdx.x;
	const int row = blockIdx.y * blockDim.y + threadIdx.y;

	if (col < 1 || col >= W - 1 || row < 1 || row >= H - 1)
		return;

	int index0 = getIndex(row, col);
	// these are the indices which will be used to retrieve the data for the current vectors
	int indexN = getIndex(row - 1, col);
	int indexS = getIndex(row + 1, col);
	int indexE = getIndex(row, col - 1);
	int indexW = getIndex(row, col + 1);
	int indexNE = getIndex(row - 1, col - 1);
	int indexNW = getIndex(row - 1, col + 1);
	int indexSE = getIndex(row + 1, col - 1);
	int indexSW = getIndex(row + 1, col + 1);

	// BOUNCE STEP
	if (barrier[index0]) {
		if (before[index0].vN > 0) {
			after[indexN].vS = before[indexN].vS + before[index0].vN;
			before[index0].vN = 0;
		}
		if (before[index0].vS > 0) {
			after[indexS].vN = before[indexS].vN + before[index0].vS;
			before[index0].vS = 0;
		}
		if (before[index0].vE > 0) {
			after[indexE].vW = before[indexE].vW + before[index0].vE;
			before[index0].vE = 0;
		}
		if (before[index0].vW > 0) {
			after[indexW].vE = before[indexW].vE + before[index0].vW;
			before[index0].vW = 0;
		}
		if (before[index0].vNW > 0) {
			after[indexNW].vSE = before[indexNW].vSE + before[index0].vNW;
			before[index0].vNW = 0;
		}
		if (before[index0].vNE > 0) {
			after[indexNE].vSW = before[indexNE].vSW + before[index0].vNE;
			before[index0].vNE = 0;
		}
		if (before[index0].vSW > 0) {
			after[indexSW].vNE = before[indexSW].vNE + before[index0].vSW;
			before[index0].vSW = 0;
		}
		if (before[index0].vSE > 0) {
			after[indexSE].vNW = before[indexSE].vNW + before[index0].vSE;
			before[index0].vSE = 0;
		}
	}
}

__device__
unsigned char clip(int n) {
	return n > 255 ? 255 : (n < 0 ? 0 : n);
}

//init stuff:

for (int y = 0; y < H; y++) {
	for (int x = 0; x < W; x++) {
		index0 = getIndex_cpu(y, x);
		if (barrier[index0]) {
			zeroSite(before, index0);
		}
		else {
			before[index0].v0 = four9ths * (1 - 1.5 * v * v);
			before[index0].vE = one9th * (1 + 3 * v + 3 * v * v);
			before[index0].vW = one9th * (1 - 3 * v + 3 * v * v);
			before[index0].vN = one9th * (1 - 1.5 * v * v);
			before[index0].vS = one9th * (1 - 1.5 * v * v);
			before[index0].vNE = one36th * (1 + 3 * v + 3 * v * v);
			before[index0].vSE = one36th * (1 + 3 * v + 3 * v * v);
			before[index0].vNW = one36th * (1 - 3 * v + 3 * v * v);
			before[index0].vSW = one36th * (1 - 3 * v + 3 * v * v);
			before[index0].density = 1;
			before[index0].xvel = v;
			before[index0].yvel = 0;
			before[index0].speed2 = v * v;
		}
	}
}


void zeroSite(lbm_node * array1, int index0) {
	array1[index0].v0 = 0;
	array1[index0].vE = 0;
	array1[index0].vW = 0;
	array1[index0].vN = 0;
	array1[index0].vS = 0;
	array1[index0].vNE = 0;
	array1[index0].vNW = 0;
	array1[index0].vSE = 0;
	array1[index0].vSW = 0;
	array1[index0].xvel = 0;
	array1[index0].yvel = 0;
	array1[index0].speed2 = 0;
}

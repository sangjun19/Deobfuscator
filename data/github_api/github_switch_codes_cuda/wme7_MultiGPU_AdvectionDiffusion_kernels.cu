#include "hip/hip_runtime.h"
// Repository: wme7/MultiGPU_AdvectionDiffusion
// File: SingleGPU/Burgers3d_WENO5_Hybrid2/kernels.cu

//
//  kernels.cu
//  Burgers3d-GPU
//
//  Created by Manuel Diaz on 7/26/16.
//  Copyright © 2016 Manuel Diaz. All rights reserved.
//

extern "C" {
#include "Burgers.h"
}

/*******************************/
/* Define Textures & Constanst */
/*******************************/
__constant__ REAL d_kx;
__constant__ REAL d_ky;
__constant__ REAL d_kz;
texture<int2, 1, hipReadModeElementType> tex_u_1d;
texture<int2, 2, hipReadModeElementType> tex_u_2d;

#define checkCuda(error) __checkCuda(error, __FILE__, __LINE__)

/*********************************************/
/* A method for checking error in CUDA calls */
/*********************************************/
inline void __checkCuda(hipError_t error, const char *file, const int line)
{
  #if defined(DISPL)
    if (error != hipSuccess)
    {
    printf("checkCuda error at %s:%i: %s\n", file, line, hipGetErrorString(hipGetLastError()));
    exit(-1);
    }
  #endif
  return;
}

/*****************/
/* FLUX FUNCTION */
/*****************/
__device__ REAL Flux(
  const REAL u){
  return 0.5*u*u;
}

// *************************************************************************
// Input: v(i) = [v(i-2) v(i-1) v(i) v(i+1) v(i+2) v(i+3)];
// Output: res = df/dx;
//
// Based on:
// C.W. Shu's Lectures notes on: 'ENO and WENO schemes for Hyperbolic
// Conservation Laws'
//
// coded by Manuel Diaz, 02.10.2012, NTU Taiwan.
// *************************************************************************
//
// Domain cells (I{i}) reference:
//
//                |           |   u(i)    |           |
//                |  u(i-1)   |___________|           |
//                |___________|           |   u(i+1)  |
//                |           |           |___________|
//             ...|-----0-----|-----0-----|-----0-----|...
//                |    i-1    |     i     |    i+1    |
//                |-         +|-         +|-         +|
//              i-3/2       i-1/2       i+1/2       i+3/2
//
// ENO stencils (S{r}) reference:
//
//                           |___________S2__________|
//                           |                       |
//                   |___________S1__________|       |
//                   |                       |       |    using only f^{+}
//           |___________S0__________|       |       |
//         ..|---o---|---o---|---o---|---o---|---o---|...
//           | I{i-2}| I{i-1}|  I{i} | I{i+1}| I{i+2}|
//                                  -|
//                                 i+1/2
//
//                   |___________S0__________|
//                   |                       |
//                   |       |___________S1__________|    using only f^{-}
//                   |       |                       |
//                   |       |       |___________S2__________|
//                 ..|---o---|---o---|---o---|---o---|---o---|...
//                   | I{i-1}|  I{i} | I{i+1}| I{i+2}| I{i+3}|
//                                   |+
//                                 i+1/2
//
// WENO stencil: S{i} = [ I{i-2},...,I{i+3} ]
// *************************************************************************

/***********************/
/* WENO RECONSTRUCTION */
/***********************/
__device__ REAL WENO5reconstruction(
  const REAL vmm,
  const REAL vm,
  const REAL v,
  const REAL vp,
  const REAL vpp,
  const REAL umm,
  const REAL um,
  const REAL u,
  const REAL up,
  const REAL upp)
{
  REAL B0, B1, B2, a0, a1, a2, alphasum, dflux;
  
  // Smooth Indicators (Beta factors)
  B0 = C1312*(vmm-2*vm+v  )*(vmm-2*vm+v  ) + C14*(vmm-4*vm+3*v)*(vmm-4*vm+3*v);
  B1 = C1312*(vm -2*v +vp )*(vm -2*v +vp ) + C14*(vm-vp)*(vm-vp);
  B2 = C1312*(v  -2*vp+vpp)*(v  -2*vp+vpp) + C14*(3*v-4*vp+vpp)*(3*v-4*vp+vpp);
  
  // Alpha weights
  a0 = D0N/((EPS + B0)*(EPS + B0));
  a1 = D1N/((EPS + B1)*(EPS + B1));
  a2 = D2N/((EPS + B2)*(EPS + B2));
  alphasum = a0 + a1 + a2;
  
  // Numerical Flux at cell boundary, $v_{i+1/2}^{-}$;
  dflux =(a0*(2*vmm- 7*vm + 11*v) +
          a1*( -vm + 5*v  + 2*vp) +
          a2*( 2*v + 5*vp - vpp ))/(6*alphasum);

  // Smooth Indicators (Beta factors)
  B0 = C1312*(umm-2*um+u  )*(umm-2*um +u  ) + C14*(umm-4*um+3*u)*(umm-4*um+3*u);
  B1 = C1312*(um -2*u +up )*(um -2*u  +up ) + C14*(um-up)*(um-up);
  B2 = C1312*(u  -2*up+upp)*(u  -2*up +upp) + C14*(3*u-4*up+upp)*(3*u-4*up+upp);
  
  // Alpha weights
  a0 = D0P/((EPS + B0)*(EPS + B0));
  a1 = D1P/((EPS + B1)*(EPS + B1));
  a2 = D2P/((EPS + B2)*(EPS + B2));
  alphasum = a0 + a1 + a2;

  // Numerical Flux at cell boundary, $v_{i+1/2}^{+}$;
  dflux+=(a0*( -umm + 5*um + 2*u  ) +
          a1*( 2*um + 5*u  - up   ) +
          a2*(11*u  - 7*up + 2*upp))/(6*alphasum);
  
  // Compute the numerical flux v_{i+1/2}
  return dflux;
}

__device__ REAL WENO5Zreconstruction(
  const REAL vmm,
  const REAL vm,
  const REAL v,
  const REAL vp,
  const REAL vpp,
  const REAL umm,
  const REAL um,
  const REAL u,
  const REAL up,
  const REAL upp)
{
  REAL B0, B1, B2, a0, a1, a2, tau5, alphasum, dflux;
  
  // Smooth Indicators (Beta factors)
  B0 = C1312*(vmm-2*vm+v  )*(vmm-2*vm+v  ) + C14*(vmm-4*vm+3*v)*(vmm-4*vm+3*v);
  B1 = C1312*(vm -2*v +vp )*(vm -2*v +vp ) + C14*(vm-vp)*(vm-vp);
  B2 = C1312*(v  -2*vp+vpp)*(v  -2*vp+vpp) + C14*(3*v-4*vp+vpp)*(3*v-4*vp+vpp);
  
  // Alpha weights
  tau5 = fabs(B0-B2);
  a0 = D0N*(1.+tau5/(B0+EPS));
  a1 = D1N*(1.+tau5/(B1+EPS));
  a2 = D2N*(1.+tau5/(B2+EPS));
  alphasum = a0 + a1 + a2;
  
  // Numerical Flux at cell boundary, $v_{i+1/2}^{-}$;
  dflux =(a0*(2*vmm- 7*vm + 11*v) +
          a1*( -vm + 5*v  + 2*vp) +
          a2*( 2*v + 5*vp - vpp ))/(6*alphasum);

  // Smooth Indicators (Beta factors)
  B0 = C1312*(umm-2*um+u  )*(umm-2*um +u  ) + C14*(umm-4*um+3*u)*(umm-4*um+3*u);
  B1 = C1312*(um -2*u +up )*(um -2*u  +up ) + C14*(um-up)*(um-up);
  B2 = C1312*(u  -2*up+upp)*(u  -2*up +upp) + C14*(3*u-4*up+upp)*(3*u-4*up+upp);
  
  // Alpha weights
  tau5 = fabs(B0-B2);
  a0 = D0P*(1.+tau5/(B0+EPS));
  a1 = D1P*(1.+tau5/(B1+EPS));
  a2 = D2P*(1.+tau5/(B2+EPS));
  alphasum = a0 + a1 + a2;

  // Numerical Flux at cell boundary, $v_{i+1/2}^{+}$;
  dflux+=(a0*( -umm + 5*um + 2*u  ) +
          a1*( 2*um + 5*u  - up   ) +
          a2*(11*u  - 7*up + 2*upp))/(6*alphasum);
  
  // Compute the numerical flux v_{i+1/2}
  return dflux;
}

/*****************/
/* Compute du/dx */ 
/*****************/
__global__ void Compute_dF(
  REAL * __restrict__ Lu, 
  const unsigned int pitch,
  const unsigned int nx, 
  const unsigned int ny, 
  const REAL dx,
  const int k)
{
  // Shared variables
  __shared__ REAL sfu[WIDTH][TILE+1]; // faces = inner nodes + 1

  // Temporary variables
  REAL u0,u1,u2,u3,u4,u5;
  REAL B0,B1,B2,a0,a1,a2;
  REAL umm,um,uo,up,upp,tau5,asum;

  // Indexes
  unsigned int i,j,si,sj,o; int2 Data;
  
  // Global threads indexes
  i =    TILE    * blockIdx.x + threadIdx.x;
  j = blockDim.y * blockIdx.y + threadIdx.y;

  // shared memory indexes
  si = threadIdx.x;
  sj = threadIdx.y;

  // Global index
  o = i+pitch*j+pitch*ny*k;

  if (i < nx){
    // Load data from texture memory
    Data = tex2D(tex_u_2d,i-3,j); u0 = __hiloint2double(Data.y, Data.x);
    Data = tex2D(tex_u_2d,i-2,j); u1 = __hiloint2double(Data.y, Data.x);
    Data = tex2D(tex_u_2d,i-1,j); u2 = __hiloint2double(Data.y, Data.x);
    Data = tex2D(tex_u_2d, i ,j); u3 = __hiloint2double(Data.y, Data.x);
    Data = tex2D(tex_u_2d,i+1,j); u4 = __hiloint2double(Data.y, Data.x);
    Data = tex2D(tex_u_2d,i+2,j); u5 = __hiloint2double(Data.y, Data.x);

    // Reconstruct Face fluxes
    // Split data for f_{i}^{+}
    umm=0.5*(Flux(u0) + fabs(u0)*u0);
    um =0.5*(Flux(u1) + fabs(u1)*u1);
    uo =0.5*(Flux(u2) + fabs(u2)*u2);
    up =0.5*(Flux(u3) + fabs(u3)*u3);
    upp=0.5*(Flux(u4) + fabs(u4)*u4);
    
    // Smooth Indicators (Beta factors)
    B0 = C1312*(umm-2*um+uo )*(umm-2*um +uo ) + 0.25*(umm-4*um+3*uo)*(umm-4*um+3*uo);
    B1 = C1312*(um -2*uo+up )*(um -2*uo +up ) + 0.25*(um-up)*(um-up);
    B2 = C1312*(uo -2*up+upp)*(uo -2*up +upp) + 0.25*(3*uo-4*up+upp)*(3*uo-4*up+upp);
      
    // Alpha weights
    tau5 = fabs(B0-B2);
    a0 = D0N*(1.+tau5/(B0+EPS));
    a1 = D1N*(1.+tau5/(B1+EPS));
    a2 = D2N*(1.+tau5/(B2+EPS));
    asum = a0 + a1 + a2;
    
    // Numerical Flux at cell boundary, $u_{i+1/2}^{-}$;
    sfu[sj][si] = (
      a0*(2*umm- 7*um + 11*uo) +
      a1*( -um + 5*uo + 2*up ) +
      a2*( 2*uo+ 5*up - upp  ))/(6*asum);

    // split data for f_{i}^{-}
    umm=0.5*(Flux(u1) - fabs(u1)*u1);
    um =0.5*(Flux(u2) - fabs(u2)*u2);
    uo =0.5*(Flux(u3) - fabs(u3)*u3);
    up =0.5*(Flux(u4) - fabs(u4)*u4);
    upp=0.5*(Flux(u5) - fabs(u5)*u5);

    // Smooth Indicators (Beta factors)
    B0 = C1312*(umm-2*um+uo )*(umm-2*um +uo ) + 0.25*(umm-4*um+3*uo)*(umm-4*um+3*uo);
    B1 = C1312*(um -2*uo+up )*(um -2*uo +up ) + 0.25*(um-up)*(um-up);
    B2 = C1312*(uo -2*up+upp)*(uo -2*up +upp) + 0.25*(3*uo-4*up+upp)*(3*uo-4*up+upp);
    
    // Alpha weights
    tau5 = fabs(B0-B2);
    a0 = D0P*(1.+tau5/(B0+EPS));
    a1 = D1P*(1.+tau5/(B1+EPS));
    a2 = D2P*(1.+tau5/(B2+EPS));
    asum = a0 + a1 + a2;

    // Numerical Flux at cell boundary, $u_{i+1/2}^{+}$;
    sfu[sj][si] += (
      a0*( -umm + 5*um + 2*uo ) +
      a1*( 2*um + 5*uo - up   ) +
      a2*(11*uo - 7*up + 2*upp))/(6*asum);
    __syncthreads();

    // Compute Lq = (f_{i+1/2}-f_{i-1/2})/dx
    if ( si<TILE ){
      Lu[o] = -(sfu[sj][si+1] - sfu[sj][si])/dx;
    }
  }
}

/*****************/
/* Compute du/dy */ 
/*****************/
__global__ void Compute_dG(
  REAL * __restrict__ Lu, 
  const unsigned int pitch,
  const unsigned int nx, 
  const unsigned int ny, 
  const REAL dy,
  const int k)
{
  // Shared variables
  __shared__ REAL sfu[TILE+1][WIDTH]; // faces = inner nodes + 1

  // Temporary variables
  REAL u0,u1,u2,u3,u4,u5;
  REAL B0,B1,B2,a0,a1,a2;
  REAL umm,um,uo,up,upp,tau5,asum;

  // Indexes
  unsigned int i,j,si,sj,o; int2 Data;
  
  // local threads indexes
  i = blockDim.x * blockIdx.x + threadIdx.x;
  j =    TILE    * blockIdx.y + threadIdx.y;

  // shared memory indexes
  si = threadIdx.x;
  sj = threadIdx.y;

  // Global index
  o = i+pitch*j+pitch*ny*k;

  if (j < ny){
    // Load data from texture memory
    Data = tex2D(tex_u_2d,i,j-3); u0 = __hiloint2double(Data.y, Data.x);
    Data = tex2D(tex_u_2d,i,j-2); u1 = __hiloint2double(Data.y, Data.x);
    Data = tex2D(tex_u_2d,i,j-1); u2 = __hiloint2double(Data.y, Data.x);
    Data = tex2D(tex_u_2d,i, j ); u3 = __hiloint2double(Data.y, Data.x);
    Data = tex2D(tex_u_2d,i,j+1); u4 = __hiloint2double(Data.y, Data.x);
    Data = tex2D(tex_u_2d,i,j+2); u5 = __hiloint2double(Data.y, Data.x);

    // Reconstruct Face fluxes
    // Split data for f_{i}^{+}
    umm=0.5*(Flux(u0) + fabs(u0)*u0);
    um =0.5*(Flux(u1) + fabs(u1)*u1);
    uo =0.5*(Flux(u2) + fabs(u2)*u2);
    up =0.5*(Flux(u3) + fabs(u3)*u3);
    upp=0.5*(Flux(u4) + fabs(u4)*u4);
    
    // Smooth Indicators (Beta factors)
    B0 = C1312*(umm-2*um+uo )*(umm-2*um +uo ) + 0.25*(umm-4*um+3*uo)*(umm-4*um+3*uo);
    B1 = C1312*(um -2*uo+up )*(um -2*uo +up ) + 0.25*(um-up)*(um-up);
    B2 = C1312*(uo -2*up+upp)*(uo -2*up +upp) + 0.25*(3*uo-4*up+upp)*(3*uo-4*up+upp);
      
    // Alpha weights
    tau5 = fabs(B0-B2);
    a0 = D0N*(1.+tau5/(B0+EPS));
    a1 = D1N*(1.+tau5/(B1+EPS));
    a2 = D2N*(1.+tau5/(B2+EPS));
    asum = a0 + a1 + a2;
    
    // Numerical Flux at cell boundary, $u_{i+1/2}^{-}$;
    sfu[sj][si] = (
      a0*(2*umm- 7*um + 11*uo) +
      a1*( -um + 5*uo + 2*up ) +
      a2*( 2*uo+ 5*up - upp  ))/(6*asum);

    // split data for f_{i}^{-}
    umm=0.5*(Flux(u1) - fabs(u1)*u1);
    um =0.5*(Flux(u2) - fabs(u2)*u2);
    uo =0.5*(Flux(u3) - fabs(u3)*u3);
    up =0.5*(Flux(u4) - fabs(u4)*u4);
    upp=0.5*(Flux(u5) - fabs(u5)*u5);

    // Smooth Indicators (Beta factors)
    B0 = C1312*(umm-2*um+uo )*(umm-2*um +uo ) + 0.25*(umm-4*um+3*uo)*(umm-4*um+3*uo);
    B1 = C1312*(um -2*uo+up )*(um -2*uo +up ) + 0.25*(um-up)*(um-up);
    B2 = C1312*(uo -2*up+upp)*(uo -2*up +upp) + 0.25*(3*uo-4*up+upp)*(3*uo-4*up+upp);
    
    // Alpha weights
    tau5 = fabs(B0-B2);
    a0 = D0P*(1.+tau5/(B0+EPS));
    a1 = D1P*(1.+tau5/(B1+EPS));
    a2 = D2P*(1.+tau5/(B2+EPS));
    asum = a0 + a1 + a2;

    // Numerical Flux at cell boundary, $u_{i+1/2}^{+}$;
    sfu[sj][si] += (
      a0*( -umm + 5*um + 2*uo ) +
      a1*( 2*um + 5*uo - up   ) +
      a2*(11*uo - 7*up + 2*upp))/(6*asum);
    __syncthreads();

    // Compute Lq = (f_{i+1/2}-f_{i-1/2})/dx
    if ( sj<TILE ){
      Lu[o] -= (sfu[sj+1][si] - sfu[sj][si])/dy;
    }
  }
}

/*****************/
/* Compute du/dz */ 
/*****************/

__global__ void Compute_dH(
  const REAL * __restrict__ u, 
  REAL * __restrict__ Lu,
  const unsigned int pitch,
  const unsigned int nx, 
  const unsigned int ny, 
  const unsigned int nz, 
  const REAL dz)
{
  // Temporary variables
  REAL fu, fu_old;
  REAL f1mm, f1m, f1, f1p, f1pp;
  REAL g1mm, g1m, g1, g1p, g1pp;

  // Indexes
  unsigned int i, j, k, xy=pitch*ny;

  // local threads indexes
  i = blockDim.x * blockIdx.x + threadIdx.x;
  j = blockDim.y * blockIdx.y + threadIdx.y;

  // Compute only for internal nodes
  if (i>2 && i<nx-3 && j>2 && j<ny-3) {

    // Old resulst arrays
    fu_old=0;
    
    f1mm= 0.5*( Flux(u[i+pitch*j+xy*0]) + fabs(u[i+pitch*j+xy*0])*u[i+pitch*j+xy*0]); // node(i-2)
    f1m = 0.5*( Flux(u[i+pitch*j+xy*1]) + fabs(u[i+pitch*j+xy*1])*u[i+pitch*j+xy*1]); // node(i-1)
    f1  = 0.5*( Flux(u[i+pitch*j+xy*2]) + fabs(u[i+pitch*j+xy*2])*u[i+pitch*j+xy*2]); // node( i )     imm--im--i--ip--ipp--ippp
    f1p = 0.5*( Flux(u[i+pitch*j+xy*3]) + fabs(u[i+pitch*j+xy*3])*u[i+pitch*j+xy*3]); // node(i+1)
       
    g1mm= 0.5*( Flux(u[i+pitch*j+xy*1]) - fabs(u[i+pitch*j+xy*1])*u[i+pitch*j+xy*1]); // node(i-1)
    g1m = 0.5*( Flux(u[i+pitch*j+xy*2]) - fabs(u[i+pitch*j+xy*2])*u[i+pitch*j+xy*2]); // node( i )     imm--im--i--ip--ipp--ippp
    g1  = 0.5*( Flux(u[i+pitch*j+xy*3]) - fabs(u[i+pitch*j+xy*3])*u[i+pitch*j+xy*3]); // node(i+1)
    g1p = 0.5*( Flux(u[i+pitch*j+xy*4]) - fabs(u[i+pitch*j+xy*4])*u[i+pitch*j+xy*4]); // node(i+2)
        
    for (k = 2; k < nz-3; k++) {
        
      // Compute and split fluxes
      f1pp= 0.5*( Flux(u[i+pitch*j+xy*(k+2)]) + fabs(u[i+pitch*j+xy*(k+2)])*u[i+pitch*j+xy*(k+2)]); // node(i+2)
      g1pp= 0.5*( Flux(u[i+pitch*j+xy*(k+3)]) - fabs(u[i+pitch*j+xy*(k+3)])*u[i+pitch*j+xy*(k+3)]); // node(i+3)
      
      // Reconstruct
      fu = WENO5Zreconstruction(f1mm,f1m,f1,f1p,f1pp,g1mm,g1m,g1,g1p,g1pp);
      
      // Compute Lq = dH/dz
      Lu[i+pitch*j+xy*k]-=(fu-fu_old)/dz; // dudz
      
      // Save old results
      fu_old=fu;
      
      f1mm= f1m;   // node(i-2)
      f1m = f1;    // node(i-1)
      f1  = f1p;   // node( i )    imm--im--i--ip--ipp--ippp
      f1p = f1pp;  // node(i+1)
      
      g1mm= g1m;   // node(i-1)
      g1m = g1;    // node( i )    imm--im--i--ip--ipp--ippp
      g1  = g1p;   // node(i+1)
      g1p = g1pp;  // node(i+2)
    }
  }
}

__global__ void Compute_dH_v2(
  REAL * __restrict__ Lu,
  const unsigned int pitch,
  const unsigned int ny, 
  const unsigned int kstart, 
  const unsigned int kstop,
  const REAL dz)
{
  // Shared variables
  __shared__ REAL sfu[TILE+1]; // faces = inner nodes + 1

  // Temporary variables
  REAL u0,u1,u2,u3,u4,u5;
  REAL B0,B1,B2,a0,a1,a2;
  REAL umm,um,uo,up,upp,tau5,asum;

  // Indexes
  unsigned int i,j,k,sk,o,layer=pitch*ny; int2 Data;
  
  // local threads indexes
  k =    TILE    * blockIdx.x + threadIdx.x;
  i = blockIdx.y;
  j = blockIdx.z;

  // shared memory indexes
  sk = threadIdx.x;

  k = MAX(kstart,k);

  // Global index
  o = i+pitch*j+layer*k;

  if (k < kstop){
    // Load data from texture memory
    Data = tex1Dfetch(tex_u_1d,o-3*layer); u0 = __hiloint2double(Data.y, Data.x);
    Data = tex1Dfetch(tex_u_1d,o-2*layer); u1 = __hiloint2double(Data.y, Data.x);
    Data = tex1Dfetch(tex_u_1d, o-layer ); u2 = __hiloint2double(Data.y, Data.x);
    Data = tex1Dfetch(tex_u_1d,   o     ); u3 = __hiloint2double(Data.y, Data.x);
    Data = tex1Dfetch(tex_u_1d, o+layer ); u4 = __hiloint2double(Data.y, Data.x);
    Data = tex1Dfetch(tex_u_1d,o+2*layer); u5 = __hiloint2double(Data.y, Data.x);

    // Reconstruct Face fluxes
    // Split data for f_{i}^{+}
    umm=0.5*(Flux(u0) + fabs(u0)*u0);
    um =0.5*(Flux(u1) + fabs(u1)*u1);
    uo =0.5*(Flux(u2) + fabs(u2)*u2);
    up =0.5*(Flux(u3) + fabs(u3)*u3);
    upp=0.5*(Flux(u4) + fabs(u4)*u4);
    
    // Smooth Indicators (Beta factors)
    B0 = C1312*(umm-2*um+uo )*(umm-2*um +uo ) + 0.25*(umm-4*um+3*uo)*(umm-4*um+3*uo);
    B1 = C1312*(um -2*uo+up )*(um -2*uo +up ) + 0.25*(um-up)*(um-up);
    B2 = C1312*(uo -2*up+upp)*(uo -2*up +upp) + 0.25*(3*uo-4*up+upp)*(3*uo-4*up+upp);
      
    // Alpha weights
    tau5 = fabs(B0-B2);
    a0 = D0N*(1.+tau5/(B0+EPS));
    a1 = D1N*(1.+tau5/(B1+EPS));
    a2 = D2N*(1.+tau5/(B2+EPS));
    asum = a0 + a1 + a2;
    
    // Numerical Flux at cell boundary, $u_{i+1/2}^{-}$;
    sfu[sk] = (
      a0*(2*umm- 7*um + 11*uo) +
      a1*( -um + 5*uo + 2*up ) +
      a2*( 2*uo+ 5*up - upp  ))/(6*asum);

    // split data for f_{i}^{-}
    umm=0.5*(Flux(u1) - fabs(u1)*u1);
    um =0.5*(Flux(u2) - fabs(u2)*u2);
    uo =0.5*(Flux(u3) - fabs(u3)*u3);
    up =0.5*(Flux(u4) - fabs(u4)*u4);
    upp=0.5*(Flux(u5) - fabs(u5)*u5);

    // Smooth Indicators (Beta factors)
    B0 = C1312*(umm-2*um+uo )*(umm-2*um +uo ) + 0.25*(umm-4*um+3*uo)*(umm-4*um+3*uo);
    B1 = C1312*(um -2*uo+up )*(um -2*uo +up ) + 0.25*(um-up)*(um-up);
    B2 = C1312*(uo -2*up+upp)*(uo -2*up +upp) + 0.25*(3*uo-4*up+upp)*(3*uo-4*up+upp);
    
    // Alpha weights
    tau5 = fabs(B0-B2);
    a0 = D0P*(1.+tau5/(B0+EPS));
    a1 = D1P*(1.+tau5/(B1+EPS));
    a2 = D2P*(1.+tau5/(B2+EPS));
    asum = a0 + a1 + a2;

    // Numerical Flux at cell boundary, $u_{i+1/2}^{+}$;
    sfu[sk] += (
      a0*( -umm + 5*um + 2*uo ) +
      a1*( 2*um + 5*uo - up   ) +
      a2*(11*uo - 7*up + 2*upp))/(6*asum);
    __syncthreads();

    // Compute Lq = (f_{i+1/2}-f_{i-1/2})/dx
    if ( sk<TILE ){
      Lu[o] -= (sfu[sk+1] - sfu[sk])/dz;
    }
  }
}

/********************/
/* Laplace Operator */
/********************/
__global__ void Compute_Laplace(
  const REAL * __restrict__ u, 
  REAL * __restrict__ Lu,  
  const unsigned int px, // pitch in the x-direction
  const unsigned int nx, 
  const unsigned int ny, 
  const unsigned int nz)
{
  REAL above2;
  REAL above;
  REAL center;
  REAL below;
  REAL below2;
  unsigned int i, j, k, o, xy, px2, xy2;
  xy = px*ny; px2 = 2*px; xy2 = 2*xy; 

    i = threadIdx.x + blockIdx.x * blockDim.x;
    j = threadIdx.y + blockIdx.y * blockDim.y;

    // For initial slice
    k=3; o=i+px*j+xy*k;

    if (i>2 && i<nx-3 && j>2 && j<ny-3)
    {
      below2=u[o-xy2]; below=u[o-xy]; center=u[o]; above=u[o+xy]; above2=u[o+xy2];

      Lu[o]+= d_kx * (- u[o-2] +16*u[o-1] - 30*center + 16*u[o+1] - u[o+2] ) +
              d_ky * (-u[o-px2]+16*u[o-px]- 30*center + 16*u[o+px]- u[o+px2])+ 
              d_kz * (- below2 +16* below - 30*center + 16* above - above2 );

      // For the rest of the slide
      for(k = 4; k < nz-3; k++)
      {
        o=o+xy; below2=below; below=center; center=above; above=above2; above2=u[o+xy2];

        Lu[o]+= d_kx * (- u[o-2] +16*u[o-1] - 30*center + 16*u[o+1] - u[o+2] ) +
                d_ky * (-u[o-px2]+16*u[o-px]- 30*center + 16*u[o+px]- u[o+px2])+ 
                d_kz * (- below2 +16* below - 30*center + 16* above - above2 );
      }
    }
    // else : do nothing!
}

/**************************/
/* Async Laplace Operator */
/**************************/
__global__ void Compute_Laplace_Async(
  const REAL * __restrict__ u, 
  REAL * __restrict__ Lu, 
  unsigned int px, // pitch in the x-direction
  unsigned int Nx, 
  unsigned int Ny, 
  unsigned int _Nz, 
  unsigned int kstart, 
  unsigned int kstop, 
  unsigned int loop_z)
{
  register REAL above2;
  register REAL above;
  register REAL center;
  register REAL below;
  register REAL below2;
  unsigned int i, j, k, o, z, XY, px2, XY2;

  i = blockIdx.x * blockDim.x + threadIdx.x;
  j = blockIdx.y * blockDim.y + threadIdx.y;
  k = blockIdx.z * loop_z;

  k = MAX(kstart,k);

  XY=px*Ny; px2=px+px; XY2=XY+XY; o=i+px*j+XY*k;

  if (i>2 && i<Nx-3 && j>2 && j<Ny-3)
  {
    below2=u[o-XY2]; below=u[o-XY]; center=u[o]; above=u[o+XY]; above2=u[o+XY2];

    Lu[o]+= d_kx*(- u[o-2] +16* u[o-1]-30*center+16*u[o+1] - u[o+2] ) +
            d_ky*(-u[o-px2]+16*u[o-px]-30*center+16*u[o+px]-u[o+px2]) +
            d_kz*(- below2 +16* below -30*center+16* above - above2 );
      

    for(z = 1; z < loop_z; z++)
    {
      k += 1;

      if (k < MIN(kstop,_Nz+1))
      {
        o=o+XY; below2=below; below=center; center=above; above=above2; above2=u[o+XY2];

        Lu[o]+= d_kx*(- u[o-2] +16*u[o-1] -30*center+16*u[o+1] - u[o+2] ) + 
                d_ky*(-u[o-px2]+16*u[o-px]-30*center+16*u[o+px]-u[o+px2]) + 
                d_kz*(- below2 +16* below -30*center+16* above - above2 );
      }
    }
  }
  // else : do nothing!
}

/***********************/
/* Runge Kutta Methods */  // <==== this is perfectly parallel!
/***********************/
__global__ void Compute_RK( 
  REAL * __restrict__ u, 
  const REAL * __restrict__ uo, 
  const REAL * __restrict__ Lu, 
  const unsigned int step, 
  const unsigned int pitch, 
  const unsigned int nx, 
  const unsigned int ny, 
  const unsigned int nz, 
  const REAL dt)
{
  // local threads indexes
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  int j = blockDim.y * blockIdx.y + threadIdx.y;
  int k = blockDim.z * blockIdx.z + threadIdx.z;
    
  // compute single index
  unsigned int o=i+pitch*j+pitch*ny*k;

  // Compute Runge-Kutta step, update only internal cells
  if (i>2 && i<nx-3 && j>2 && j<ny-3 && k>2 && k<nz-3)
  {
    switch (step) {
      case 1: // step 1
          u[o] = uo[o]+dt*(Lu[o]); break;
      case 2: // step 2
          u[o] = 0.75*uo[o]+0.25*(u[o]+dt*(Lu[o])); break;
      case 3: // step 3
          u[o] = (uo[o]+2*(u[o]+dt*(Lu[o])))/3; break;
    }
  }
  // else : do nothing!
}

/*********************/
/* Function Wrappers */
/*********************/
extern "C" void CopyToConstantMemory(const REAL kx, const REAL ky, const REAL kz)
{
  checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(d_kx), &kx, sizeof(REAL), 0, hipMemcpyHostToDevice));
  checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(d_ky), &ky, sizeof(REAL), 0, hipMemcpyHostToDevice));
  checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(d_kz), &kz, sizeof(REAL), 0, hipMemcpyHostToDevice));
}

extern "C" void InitializeTextures()
{
  // Texture indexing depends on the address Mode
  // index   -7  -6  -5  -4  -3  -2  -1  0  1  2  3  4  5  6  7  8  9  10  11
  // Clamp    0   0   0   0   0   0   0  0  1  2  3  3  3  3  3  3  3   3   3
  // Border   0   0   0   0   0   0   0  0  1  2  3  0  0  0  0  0  0   0   0
  // Wrap     1   2   3   0   1   2   3  0  1  2  3  0  1  2  3  0  1   2   3
  // Mirror   1   2   3   3   2   1   0  0  1  2  3  3  2  1  0  0  1   2   3

  // Texture basic setup 1D
  tex_u_1d.addressMode[0] = hipAddressModeClamp;
  tex_u_1d.filterMode = hipFilterModePoint;
  tex_u_1d.normalized = false;

  // Texture basic setup 2D
  tex_u_2d.addressMode[0] = hipAddressModeClamp;
  tex_u_2d.addressMode[1] = hipAddressModeClamp;
  tex_u_2d.filterMode = hipFilterModePoint;
  tex_u_2d.normalized = false;
}

extern "C" void Call_Adv_x(dim3 numBlocks, dim3 threadsPerBlock, hipStream_t aStream, 
  size_t pitch_bytes, unsigned int pitch, hipChannelFormatDesc *desc, unsigned int nx, unsigned int ny, unsigned int nz, REAL dx, REAL *u, REAL *Lu)
{
  for(int k=3; k<=nz-3; k++){
    // Bind texture and Run
    checkCuda(hipBindTexture2D(0,&tex_u_2d,&u[k*ny*pitch],desc,nx,ny,pitch_bytes));
    Compute_dF<<<numBlocks,threadsPerBlock,0,aStream>>>(Lu,pitch,nx,ny,dx,k);
  }
}

extern "C" void Call_Adv_y(dim3 numBlocks, dim3 threadsPerBlock, hipStream_t aStream, 
  size_t pitch_bytes, unsigned int pitch, hipChannelFormatDesc *desc, unsigned int nx, unsigned int ny, unsigned int nz, REAL dy, REAL *u, REAL *Lu)
{
  for(int k=3; k<=nz-3; k++){
    // Bind Texture and Run
    checkCuda(hipBindTexture2D(0,&tex_u_2d,&u[k*ny*pitch],desc,nx,ny,pitch_bytes));
    Compute_dG<<<numBlocks,threadsPerBlock,0,aStream>>>(Lu,pitch,nx,ny,dy,k);
  }
}

extern "C" void Call_Adv_z(dim3 numBlocks, dim3 threadsPerBlock, hipStream_t aStream, 
  unsigned int pitch, unsigned int nx, unsigned int ny, unsigned int nz, REAL dz, REAL *u, REAL *Lu)
{
  Compute_dH<<<numBlocks,threadsPerBlock,0,aStream>>>(u,Lu,pitch,nx,ny,nz,dz);
}

extern "C" void Call_Diff_(dim3 numBlocks, dim3 threadsPerBlock, hipStream_t aStream,
  unsigned int pitch, unsigned int nx, unsigned int ny, unsigned int nz, REAL *u, REAL *Lu)
{
  // Compute_Laplace<<<numBlocks,threadsPerBlock,0,aStream>>>(u,Lu,pitch,nx,ny,nz);
  Compute_Laplace_Async<<<numBlocks,threadsPerBlock,0,aStream>>>(u,Lu,pitch,nx,ny,nz,3,nz-2,LOOP);
}

extern "C" void Call_sspRK(dim3 numBlocks, dim3 threadsPerBlock, hipStream_t aStream, 
  unsigned int pitch, unsigned int nx, unsigned int ny, unsigned int nz, unsigned int step, REAL dt, REAL *u, REAL *uo, REAL *Lu)
{
  Compute_RK<<<numBlocks,threadsPerBlock,0,aStream>>>(u,uo,Lu,step,pitch,nx,ny,nz,dt);
}

/***********************/
/* Puting all together */
/***********************/
extern "C" void Call_Adv_L(
  dim3 numBlocks_x, dim3 threadsPerBlock_x, dim3 numBlocks_y, dim3 threadsPerBlock_y, dim3 numBlocks_z, dim3 threadsPerBlock_z, 
  hipStream_t aStream, size_t pitch_bytes, unsigned int pitch, hipChannelFormatDesc *desc, unsigned int nx, unsigned int ny, unsigned int nz, 
  REAL dx, REAL dy, REAL dz, REAL *u, REAL *Lu)
{
  // Compute Layer
  for(int k=3; k<=nz-3; k++){
    checkCuda(hipBindTexture2D(0,&tex_u_2d,&u[k*ny*pitch],desc,nx,ny,pitch_bytes));
    Compute_dF<<<numBlocks_x,threadsPerBlock_x,0,aStream>>>(Lu,pitch,nx,ny,dx,k);
    Compute_dG<<<numBlocks_y,threadsPerBlock_y,0,aStream>>>(Lu,pitch,nx,ny,dy,k);
  }

  // Compute Perpendicular Layers
  checkCuda(hipBindTexture(0,&tex_u_1d,&u[0],desc,pitch_bytes*ny*nz));
  Compute_dH_v2<<<numBlocks_z,threadsPerBlock_z,0,aStream>>>(Lu,pitch,ny,3,nz-2,dz);
}
#include "hip/hip_runtime.h"
// Repository: kit-cml/ord_land_gpu
// File: modules/gpu.cu

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "../cellmodels/Land_2016.hpp"
#include "../cellmodels/Ohara_Rudy_2011.hpp"
#include "../utils/constants.hpp"
#include "glob_funct.hpp"
#include "glob_type.hpp"
#include "gpu.cuh"
#include "param.hpp"

/**
 * @brief Main kernel function to run drug simulation for all samples in parallel.
 *
 * @param d_ic50 Array of IC50 values.
 * @param d_cvar Array of conductance variability values.
 * @param d_conc Array of drug concentrations.
 * @param d_CONSTANTS Array of constants.
 * @param d_STATES Array of states.
 * @param d_RATES Array of rates.
 * @param d_ALGEBRAIC Array of algebraic values.
 * @param d_STATES_RESULT Array to store the result states.
 * @param sample_size Sample size.
 * @param temp_result Temporary result array.
 * @param cipa_result CIPA result array.
 * @param p_param Parameters.
 */
__global__ void kernel_DrugSimulation(double *d_ic50, double *d_cvar, double *d_conc, double *d_CONSTANTS,
                                      double *d_STATES, double *d_STATES_init, double *d_RATES, double *d_ALGEBRAIC,
                                      double *d_mec_CONSTANTS, double *d_mec_RATES, double *d_mec_STATES,
                                      double *d_mec_ALGEBRAIC, double *d_STATES_RESULT, double *time, double *states,
                                      double *out_dt, double *cai_result, double *ina, double *inal, double *ical,
                                      double *ito, double *ikr, double *iks, double *ik1, unsigned int sample_size,
                                      cipa_t *temp_result, cipa_t *cipa_result, param_t *p_param) {
    unsigned short thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_id >= sample_size) return;

    // Local arrays for each sample
    double time_for_each_sample[10000];
    double dt_for_each_sample[10000];

    // printf("Calculating %d\n",thread_id);
     // Run the drug simulation for each sample
    kernel_DoDrugSim_init(d_ic50, d_cvar, d_conc[thread_id], d_CONSTANTS, d_STATES, d_RATES, d_ALGEBRAIC,
                          d_mec_CONSTANTS, d_mec_RATES, d_mec_STATES, d_mec_ALGEBRAIC, d_STATES_RESULT,
                          time_for_each_sample, dt_for_each_sample, thread_id, sample_size, temp_result, cipa_result,
                          p_param);
    
}

__global__ void kernel_DrugSimulation_postpro(double *d_ic50, double *d_cvar, double *d_conc, double *d_CONSTANTS,
                                      double *d_STATES, double *d_STATES_cache, double *d_RATES, double *d_ALGEBRAIC,
                                      double *d_mec_CONSTANTS, double *d_mec_STATES, double *d_mec_RATES,
                                      double *d_mec_ALGEBRAIC, double *d_STATES_RESULT, double *d_all_states,
                                      double *time, double *states, double *out_dt, double *cai_result, double *ina,
                                      double *inal, double *ical, double *ito, double *ikr, double *iks, double *ik1, double *tension,
                                      unsigned int sample_size, cipa_t *temp_result, cipa_t *cipa_result,
                                      param_t *p_param) {
    unsigned short thread_id;
    thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_id >= sample_size) return;
    double time_for_each_sample[10000];
    double dt_for_each_sample[10000];

    if(thread_id==0) printf("%lf %lf\n",d_STATES_cache[0],d_STATES_cache[1]);

    kernel_DoDrugSim_post(d_ic50, d_cvar, d_conc[thread_id], d_CONSTANTS, d_STATES, d_STATES_cache, d_RATES,
                            d_ALGEBRAIC, d_mec_CONSTANTS, d_mec_STATES, d_mec_RATES, d_mec_ALGEBRAIC, time, states,
                            out_dt, cai_result, ina, inal, ical, ito, ikr, iks, ik1, tension, time_for_each_sample,
                            dt_for_each_sample, thread_id, sample_size, temp_result, cipa_result, p_param);
}

/**
 * @brief Runs a single drug simulation on the GPU for a given sample.
 *
 * @param d_ic50 Array of IC50 values.
 * @param d_cvar Array of conductance variability values.
 * @param d_conc Drug concentration.
 * @param d_CONSTANTS Array of constants.
 * @param d_STATES Array of states.
 * @param d_RATES Array of rates.
 * @param d_ALGEBRAIC Array of algebraic values.
 * @param d_STATES_RESULT Array to store the result states.
 * @param tcurr Current time array.
 * @param dt Time step array.
 * @param sample_id Sample ID.
 * @param sample_size Sample size.
 * @param temp_result Temporary result array.
 * @param cipa_result CIPA result array.
 * @param p_param Parameters.
 */
__device__ void kernel_DoDrugSim_init(double *d_ic50, double *d_cvar, double d_conc, double *d_CONSTANTS,
                                      double *d_STATES, double *d_RATES, double *d_ALGEBRAIC, double *d_STATES_RESULT,
                                      double *d_mec_CONSTANTS, double *d_mec_RATES, double *d_mec_STATES,
                                      double *d_mec_ALGEBRAIC, double *tcurr, double *dt, unsigned short sample_id,
                                      unsigned int sample_size, cipa_t *temp_result, cipa_t *cipa_result,
                                      param_t *p_param) {
    unsigned int input_counter = 0;

    // Initialize temporary result and CiPA result structures
    auto init_result = [](cipa_t &result, const double *STATES, unsigned int sample_id) {
        result.qnet = 0.;
        result.inal_auc = 0.;
        result.ical_auc = 0.;
        result.dvmdt_repol = -999;
        result.dvmdt_max = -999;
        result.vm_peak = -999;
        result.vm_valley = STATES[(sample_id * ORd_num_of_states) + V];
        result.vm_dia = -999;
        result.apd90 = 0.;
        result.apd50 = 0.;
        result.ca_peak = -999;
        result.ca_valley = STATES[(sample_id * ORd_num_of_states) + cai];
        result.ca_dia = -999;
        result.cad90 = 0.;
        result.cad50 = 0.;
    };

    // Initialize results for this sample
    init_result(temp_result[sample_id], d_STATES, sample_id);
    init_result(cipa_result[sample_id], d_STATES, sample_id);

    // Simulation variables
    bool is_peak = false;
    tcurr[sample_id] = 0.0;
    dt[sample_id] = p_param->dt;
    double max_time_step = 0.1, time_point = 25.0;
    double dt_set;
    int cipa_datapoint = 0;
    unsigned short pace_count = 0;
    double t_peak_capture = 0.0;
    unsigned short pace_steepest = 0;
    bool init_states_captured = false;
    bool is_eligible_AP;
    const double bcl = p_param->bcl;
    const unsigned short pace_max = p_param->pace_max;
    const unsigned short last_drug_check_pace = p_param->find_steepest_start;
    double tmax = pace_max * bcl;
    double conc = d_conc;
    double type = p_param->celltype;
    double y[7] = {0.0, 0.0, 0.0, 1.0, 0.0, 0.0, 0.0};
    double epsilon = 10E-14;
    double vm_repol30, vm_repol90;

    // Initialize constants and apply drug effects
    initConsts(d_CONSTANTS, d_STATES, type, conc, d_ic50, d_cvar, p_param->is_dutta, p_param->is_cvar, bcl, sample_id);
    applyDrugEffect(d_CONSTANTS, conc, d_ic50, epsilon, sample_id);
    land_initConsts(false, false, y, d_mec_CONSTANTS, d_mec_RATES, d_mec_STATES, d_mec_ALGEBRAIC, sample_id);

    d_CONSTANTS[BCL + (sample_id * ORd_num_of_constants)] = bcl;

    // Main simulation loop
    // dt_set = 0.001;
    while (tcurr[sample_id] < tmax) {
        // Compute rates
        // switch for new algo
        land_computeRates(tcurr[sample_id], d_mec_CONSTANTS, d_mec_RATES, d_mec_STATES, d_mec_ALGEBRAIC, y, sample_id);
        coupledComputeRates(tcurr[sample_id], d_CONSTANTS, d_RATES, d_STATES, d_ALGEBRAIC, sample_id, d_mec_RATES[TRPN + (sample_id * Land_num_of_rates)]);
        
        // Set time step (adaptive dt)
        //NOTE: Disabled in Margara
        dt_set = set_time_step(tcurr[sample_id], time_point, max_time_step, d_CONSTANTS, d_RATES, d_STATES, d_ALGEBRAIC, sample_id);
        // dt_set = 0.005;
        // Check if within the same cycle
        if (floor((tcurr[sample_id] + dt_set) / bcl) == floor(tcurr[sample_id] / bcl)) {
            dt[sample_id] = dt_set;
        } else {
            // Handle end of pacing cycle
            dt[sample_id] = (floor(tcurr[sample_id] / bcl) + 1) * bcl - tcurr[sample_id];

            // Update temporary results if this is the steepest pace
            if (temp_result[sample_id].dvmdt_repol > cipa_result[sample_id].dvmdt_repol) {
                pace_steepest = pace_count;
                cipa_result[sample_id] = temp_result[sample_id];
                cipa_result[sample_id].ca_valley = d_STATES[(sample_id * ORd_num_of_states) + cai];
                cipa_result[sample_id].vm_valley = d_STATES[(sample_id * ORd_num_of_states) + V];
                is_peak = true;
                init_states_captured = false;
            } else {
                is_peak = false;
            }

            // Reset variables for next pacing cycle
            t_peak_capture = 0.0;
            init_result(temp_result[sample_id], d_STATES, sample_id);
            pace_count++;
            input_counter = 0;
            cipa_datapoint = 0;
            is_eligible_AP = false;

            // Debug output
            if (sample_id == 0) {
                printf("core: %d pace count: %d t: %lf, steepest: %d, dvmdt_repol: %lf, conc: %lf\n", sample_id,
                       pace_count, tcurr[sample_id], pace_steepest, cipa_result[sample_id].dvmdt_repol, conc);
            }
        }

        // Solve ODEs analytically
        solveAnalytical(d_CONSTANTS, d_STATES, d_ALGEBRAIC, d_RATES, dt[sample_id], sample_id);
        land_solveEuler(dt[sample_id], tcurr[sample_id], d_STATES[cai + (sample_id * ORd_num_of_states)] * 1000., d_mec_CONSTANTS, d_mec_RATES, d_mec_STATES, sample_id);

        // Perform checks in the last few pacing cycles
        if (pace_count >= pace_max - last_drug_check_pace) {
            if (tcurr[sample_id] > ((d_CONSTANTS[(sample_id * ORd_num_of_constants) + BCL] * pace_count) +
                                    (d_CONSTANTS[(sample_id * ORd_num_of_constants) + stim_start] + 2)) &&
                tcurr[sample_id] < ((d_CONSTANTS[(sample_id * ORd_num_of_constants) + BCL] * pace_count) +
                                    (d_CONSTANTS[(sample_id * ORd_num_of_constants) + stim_start] + 10)) &&
                abs(d_ALGEBRAIC[(sample_id * ORd_num_of_algebraic) + INa]) < 1) {
                if (d_STATES[(sample_id * ORd_num_of_states) + V] > temp_result[sample_id].vm_peak) {
                    temp_result[sample_id].vm_peak = d_STATES[(sample_id * ORd_num_of_states) + V];
                    if (temp_result[sample_id].vm_peak > 0) {
                        vm_repol30 = temp_result[sample_id].vm_peak -
                                     (0.3 * (temp_result[sample_id].vm_peak - temp_result[sample_id].vm_valley));
                        vm_repol90 = temp_result[sample_id].vm_peak -
                                     (0.9 * (temp_result[sample_id].vm_peak - temp_result[sample_id].vm_valley));
                        is_eligible_AP = true;
                        t_peak_capture = tcurr[sample_id];
                    } else {
                        is_eligible_AP = false;
                    }
                }
            } else if (tcurr[sample_id] > ((d_CONSTANTS[(sample_id * ORd_num_of_constants) + BCL] * pace_count) +
                                           (d_CONSTANTS[(sample_id * ORd_num_of_constants) + stim_start] + 10)) &&
                       is_eligible_AP) {
                if (d_RATES[(sample_id * ORd_num_of_rates) + V] > temp_result[sample_id].dvmdt_repol &&
                    d_STATES[(sample_id * ORd_num_of_states) + V] <= vm_repol30 &&
                    d_STATES[(sample_id * ORd_num_of_states) + V] >= vm_repol90) {
                    temp_result[sample_id].dvmdt_repol = d_RATES[(sample_id * ORd_num_of_rates) + V];
                }
            }

            // Capture initial states and data points if in the last few paces
            if ((pace_count >= pace_max - last_drug_check_pace) && (is_peak == true) && (pace_count < pace_max)) {
                if (!init_states_captured) {
                    for (int counter = 0; counter < ORd_num_of_states; counter++) {
                        d_STATES_RESULT[(sample_id * ORd_num_of_states) + counter] =
                            d_STATES[(sample_id * ORd_num_of_states) + counter];
                    }
                    init_states_captured = true;
                }

                input_counter += sample_size;
                cipa_datapoint++;
            }
        }
        tcurr[sample_id] += dt[sample_id];
    }
}

__device__ void kernel_DoDrugSim_post(double *d_ic50, double *d_cvar, double d_conc, double *d_CONSTANTS,
                                        double *d_STATES, double *d_STATES_cache, double *d_RATES, double *d_ALGEBRAIC,
                                        double *d_mec_CONSTANTS, double *d_mec_STATES, double *d_mec_RATES,
                                        double *d_mec_ALGEBRAIC, double *time, double *states, double *out_dt,
                                        double *cai_result, double *ina, double *inal, double *ical, double *ito,
                                        double *ikr, double *iks, double *ik1, double *tension, double *tcurr, double *dt,
                                        unsigned short sample_id, unsigned int sample_size, cipa_t *temp_result,
                                        cipa_t *cipa_result, param_t *p_param) {
    unsigned long long input_counter = 0;
   if(sample_id==0) printf("%lf %lf\n",d_STATES_cache[0],d_STATES_cache[1]);


    // INIT STARTS
    
    temp_result[sample_id].qnet = 0.;
    temp_result[sample_id].inal_auc = 0.;
    temp_result[sample_id].ical_auc = 0.;
    temp_result[sample_id].dvmdt_repol = -999;
    temp_result[sample_id].dvmdt_max = -999;
    temp_result[sample_id].vm_peak = -999;
    // temp_result[sample_id].vm_valley = d_STATES[(sample_id * ORd_num_of_states) +V];
    temp_result[sample_id].vm_dia = -999;
    temp_result[sample_id].apd90 = 0.;
    temp_result[sample_id].apd50 = 0.;
    temp_result[sample_id].ca_peak = -999;
    // temp_result[sample_id].ca_valley = d_STATES[(sample_id * ORd_num_of_states) +cai];
    temp_result[sample_id].ca_dia = -999;
    temp_result[sample_id].cad90 = 0.;
    temp_result[sample_id].cad50 = 0.;

    cipa_result[sample_id].qnet = 0.;
    cipa_result[sample_id].inal_auc = 0.;
    cipa_result[sample_id].ical_auc = 0.;
    cipa_result[sample_id].dvmdt_repol = -999;
    cipa_result[sample_id].dvmdt_max = -999;
    cipa_result[sample_id].vm_peak = -999;
    // cipa_result[sample_id].vm_valley = d_STATES[(sample_id * ORd_num_of_states) +V];
    cipa_result[sample_id].vm_dia = -999;
    cipa_result[sample_id].apd90 = 0.;
    cipa_result[sample_id].apd50 = 0.;
    cipa_result[sample_id].ca_peak = -999;
    // cipa_result[sample_id].ca_valley = d_STATES[(sample_id * ORd_num_of_states) +cai];
    cipa_result[sample_id].ca_dia = -999;
    cipa_result[sample_id].cad90 = 0.;
    cipa_result[sample_id].cad50 = 0.;
    // INIT ENDS
    bool is_peak = false;
    // to search max dvmdt repol

    tcurr[sample_id] = 0.0;
    dt[sample_id] = p_param->dt;
    double tmax;
    double max_time_step = 1.0, time_point = 25.0;
    double dt_set;

    int cipa_datapoint = 0;

    // bool writen = false;

    // files for storing results
    // time-series result
    // FILE *fp_vm, *fp_inet, *fp_gate;

    // features
    // double inet, qnet;

    // looping counter
    // unsigned short idx;
  
    // simulation parameters
    // double dtw = 2.0;
    // const char *drug_name = "bepridil";
    // const double bcl = 2000; // bcl is basic cycle length
    const double bcl = p_param->bcl;
    
    const double inet_vm_threshold = p_param->inet_vm_threshold;
    // const unsigned short pace_max = 300;
    // const unsigned short pace_max = 1000;
    const unsigned short pace_max = 2;
    // const unsigned short celltype = 0.;
    // const unsigned short last_pace_print = 3;
    // const unsigned short last_drug_check_pace = 250;
    // const unsigned int print_freq = (1./dt) * dtw;
    // unsigned short pace_count = 0;
    // unsigned short pace_steepest = 0;
    double conc = d_conc; //mmol
    double type = p_param->celltype;
    bool dutta = p_param->is_dutta;
    double epsilon = 10E-14;
    double y[7] = {0.0, 0.0, 0.0, 1.0, 0.0, 0.0, 0.0};
    // double top_dvmdt = -999.0;

    // eligible AP shape means the Vm_peak > 0.
    bool is_eligible_AP = true;
    // Vm value at 30% repol, 50% repol, and 90% repol, respectively.
    // double vm_repol30, vm_repol50, vm_repol90;
    double t_peak_capture = 0.0;
    unsigned short pace_steepest = 0;

    // qnet_ap/inet_ap values
	  // double inet_ap, qnet_ap, inet4_ap, qnet4_ap, inet_cl, qnet_cl, inet4_cl, qnet4_cl;
	  // double inal_auc_ap, ical_auc_ap,inal_auc_cl, ical_auc_cl;
    // qinward_cl;
     double inet,qinward;
     double inal_auc, ical_auc;
     double vm_repol30, vm_repol50, vm_repol90;
     double t_depol;
     double t_ca_peak, ca_amp50, ca_amp90;
     double cad50_prev, cad50_curr, cad90_prev, cad90_curr;

     //inits
     inal_auc = 0.0; ical_auc = 0.0; inet = 0.0;
     t_ca_peak = 0.0; ca_amp50 = 0.0; ca_amp90 = 0.0;
     vm_repol30 = 999.0; vm_repol50 = 999.0; vm_repol90= 999.0;

    // char buffer[255];

    // static const int CALCIUM_SCALING = 1000000;
	  // static const int CURRENT_SCALING = 1000;

    // printf("Core %d:\n",sample_id);
    initConsts(d_CONSTANTS, d_STATES, type, conc, d_ic50, d_cvar, dutta, p_param->is_cvar, bcl, sample_id);
    land_initConsts(false, false, y, d_mec_CONSTANTS, d_mec_RATES, d_mec_STATES, d_mec_ALGEBRAIC, sample_id);

    // starting from initial value, to make things simpler for now, we're just going to replace what initConst has done 
    // to the d_STATES and bring them back to cached initial values:
    for (int temp = 0; temp < ORd_num_of_states; temp++) {
        d_STATES[(sample_id * ORd_num_of_states) + temp] = d_STATES_cache[(sample_id * ORd_num_of_states) + temp];
    }
    
    // these values will follow cache file (instead of regular init)
    temp_result[sample_id].vm_valley = d_STATES[(sample_id * ORd_num_of_states) +V];
    temp_result[sample_id].ca_valley = d_STATES[(sample_id * ORd_num_of_states) +cai];

    cipa_result[sample_id].vm_valley = d_STATES[(sample_id * ORd_num_of_states) +V];
    cipa_result[sample_id].ca_valley = d_STATES[(sample_id * ORd_num_of_states) +cai];

    // temp_result[sample_id].vm_valley = 9.;
    // temp_result[sample_id].ca_valley = 9.;

    // cipa_result[sample_id].vm_valley = 9.;
    // cipa_result[sample_id].ca_valley = 9.;


    // printf("%d: %lf, %d\n", sample_id,d_STATES[V + (sample_id * ORd_num_of_states)], cnt);
    applyDrugEffect(d_CONSTANTS, conc, d_ic50, epsilon, sample_id);

    d_CONSTANTS[BCL + (sample_id * ORd_num_of_constants)] = bcl;

    // generate file for time-series output

    tmax = pace_max * bcl;
    int pace_count = 0;
    
  
    // printf("%d,%lf,%lf,%lf,%lf\n", sample_id, dt[sample_id], tcurr[sample_id], d_STATES[V + (sample_id * ORd_num_of_states)],d_RATES[V + (sample_id * ORd_num_of_rates)]);
    // printf("%lf,%lf,%lf,%lf,%lf\n", d_ic50[0 + (14*sample_id)], d_ic50[1+ (14*sample_id)], d_ic50[2+ (14*sample_id)], d_ic50[3+ (14*sample_id)], d_ic50[4+ (14*sample_id)]);
    while (tcurr[sample_id]<tmax)
    {
        // updated coupling
        land_computeRates(tcurr[sample_id], d_mec_CONSTANTS, d_mec_RATES, d_mec_STATES, d_mec_ALGEBRAIC, y, sample_id);
        coupledComputeRates(tcurr[sample_id], d_CONSTANTS, d_RATES, d_STATES, d_ALGEBRAIC, sample_id, d_mec_RATES[TRPN + (sample_id * Land_num_of_rates)]);
        
        // dt_set = set_time_step( tcurr[sample_id], time_point, max_time_step, 
        // d_CONSTANTS, 
        // d_RATES, 
        // d_STATES, 
        // d_ALGEBRAIC, 
        // sample_id); 
        dt_set = 0.001;
        if(d_STATES[(sample_id * ORd_num_of_states)+V] > inet_vm_threshold){
          inet += (d_ALGEBRAIC[(sample_id * ORd_num_of_algebraic) +INaL]+d_ALGEBRAIC[(sample_id * ORd_num_of_algebraic) +ICaL]+d_ALGEBRAIC[(sample_id * ORd_num_of_algebraic) +Ito]+d_ALGEBRAIC[(sample_id * ORd_num_of_algebraic) +IKr]+d_ALGEBRAIC[(sample_id * ORd_num_of_algebraic) +IKs]+d_ALGEBRAIC[(sample_id * ORd_num_of_algebraic) +IK1])*dt[sample_id];
          inal_auc += d_ALGEBRAIC[(sample_id * ORd_num_of_algebraic) +INaL]*dt[sample_id];
          ical_auc += d_ALGEBRAIC[(sample_id * ORd_num_of_algebraic) +ICaL]*dt[sample_id];
          
          // if (sample_id == 1){
          // printf("%lf %lf %lf %lf %lf %lf\n", 
          // (d_ALGEBRAIC[(sample_id * ORd_num_of_algebraic) +INaL]+d_ALGEBRAIC[(sample_id * ORd_num_of_algebraic) +ICaL]+d_ALGEBRAIC[(sample_id * ORd_num_of_algebraic) +Ito]+d_ALGEBRAIC[(sample_id * ORd_num_of_algebraic) +IKr]+d_ALGEBRAIC[(sample_id * ORd_num_of_algebraic) +IKs]+d_ALGEBRAIC[(sample_id * ORd_num_of_algebraic) +IK1])*dt[sample_id],
          // d_ALGEBRAIC[(sample_id * ORd_num_of_algebraic) +INaL]*dt[sample_id], 
          // d_ALGEBRAIC[(sample_id * ORd_num_of_algebraic) +ICaL]*dt[sample_id],
          // inet,
          // inal_auc,
          // ical_auc
          // );
          // }
          
          } 
          // how can we properly update this value?
          // temp_result[sample_id].ca_valley = temp_result[sample_id].cai_data[0];
        
        // printf("tcurr at core %d: %lf\n",sample_id,tcurr[sample_id]);
        if (floor((tcurr[sample_id] + dt_set) / bcl) == floor(tcurr[sample_id] / bcl)) { 
          dt[sample_id] = dt_set;
        }
        else{
          dt[sample_id] = (floor(tcurr[sample_id] / bcl) + 1) * bcl - tcurr[sample_id];

          // new part starts
              // execute at the beginning of a pace
              // temp_result[sample_id].cad50 = cad50_curr - cad50_prev;
              // temp_result[sample_id].cad90 = cad90_curr - cad90_prev; // cad50 and 90 cur not calculcated yet! use outer loop instead
              temp_result[sample_id].qnet = inet/1000.0;
              temp_result[sample_id].inal_auc = inal_auc;
              temp_result[sample_id].ical_auc = ical_auc;
              temp_result[sample_id].vm_dia = d_STATES[(sample_id * ORd_num_of_states)+V];
              temp_result[sample_id].ca_dia = d_STATES[(sample_id * ORd_num_of_states)+cai];

              // cipa_result = temp_result;
              // if(sample_id == 0) printf(" %.2f percent, cipa_result updates!\n", tcurr[sample_id]/tmax);

              cipa_result[sample_id].qnet = temp_result[sample_id].qnet;
              cipa_result[sample_id].inal_auc = temp_result[sample_id].inal_auc;
              cipa_result[sample_id].ical_auc = temp_result[sample_id].ical_auc;
              cipa_result[sample_id].dvmdt_repol = temp_result[sample_id].dvmdt_repol;
              cipa_result[sample_id].dvmdt_max = temp_result[sample_id].dvmdt_max;
              
              cipa_result[sample_id].vm_dia = temp_result[sample_id].vm_dia;
              cipa_result[sample_id].apd90 = temp_result[sample_id].apd90;
              cipa_result[sample_id].apd50 = temp_result[sample_id].apd50;
              cipa_result[sample_id].ca_peak = temp_result[sample_id].ca_peak;
              cipa_result[sample_id].ca_valley = d_STATES[(sample_id * ORd_num_of_states) +cai];
              cipa_result[sample_id].ca_dia = temp_result[sample_id].ca_dia;
              cipa_result[sample_id].cad90 = temp_result[sample_id].cad90;
              cipa_result[sample_id].cad50 = temp_result[sample_id].cad50;
              
              cipa_result[sample_id].dvmdt_repol = temp_result[sample_id].dvmdt_repol;
              cipa_result[sample_id].vm_peak = temp_result[sample_id].vm_peak;
              cipa_result[sample_id].vm_valley = d_STATES[(sample_id * ORd_num_of_states) +V];

              // temp_result[sample_id].ca_valley = d_STATES[(sample_id * ORd_num_of_states) +cai];
              // temp_valley = d_STATES[(sample_id * ORd_num_of_states) +cai];

              // cipa_result[sample_id].qnet_ap = qnet_ap;
              // cipa_result[sample_id].qnet4_ap = qnet4_ap;
              // cipa_result[sample_id].inal_auc_ap = inal_auc_ap;
              // cipa_result[sample_id].ical_auc_ap = ical_auc_ap;
              
              // cipa_result[sample_id].qnet_cl = qnet_cl;
              // cipa_result[sample_id].qnet4_cl = qnet4_cl;
              // cipa_result[sample_id].inal_auc_cl = inal_auc_cl;
              // cipa_result[sample_id].ical_auc_cl = ical_auc_cl;
              
              cipa_result[sample_id].dvmdt_repol = temp_result[sample_id].dvmdt_repol;
              cipa_result[sample_id].vm_peak = temp_result[sample_id].vm_peak;
              cipa_result[sample_id].vm_valley = d_STATES[(sample_id * ORd_num_of_states) +V];
              is_peak = true;
            
          // resetting inet and AUC values
          // and increase the pace count. UPDATE: Disabled since it is very obvious we are using only one pacing here

          // pace_count++;


          input_counter = 0; // at first, we reset the input counter since we re gonna only take one, but I remember we don't have this kind of thing previously, so do we need this still?
          cipa_datapoint = 0; // new pace? reset variables related to saving the values,
          inet = 0.;
          inal_auc = 0.;
          ical_auc = 0.;
          // if(pace_count >= pace_max-last_drug_check_pace){
            // temp_result.init( p_cell->STATES[V], p_cell->STATES[cai] );

            // t_ca_peak = tcurr[sample_id];

            t_depol = (d_CONSTANTS[BCL + (sample_id * ORd_num_of_constants)]*pace_count) + d_CONSTANTS[stim_start + (sample_id * ORd_num_of_constants)];
            // if (sample_id == 1) printf("t_depol: %lf\n",t_depol);
            // is_eligible_AP = false;
            is_eligible_AP = true;
          // }
              
          // new part ends
		
          // printf("core: %d pace count: %d t: %lf, steepest: %d, dvmdt_repol: %lf, t_peak: %lf\n",sample_id,pace_count, tcurr[sample_id], pace_steepest, cipa_result[sample_id].dvmdt_repol,t_peak_capture);
          // writen = false;
        }
        // verified new coupling algorithm
        solveAnalytical(d_CONSTANTS, d_STATES, d_ALGEBRAIC, d_RATES,  dt[sample_id], sample_id);
        land_solveEuler(dt[sample_id], tcurr[sample_id], d_STATES[cai + (sample_id * ORd_num_of_states)] * 1000., d_mec_CONSTANTS, d_mec_RATES, d_mec_STATES, sample_id);

        if( temp_result[sample_id].dvmdt_max < d_RATES[(sample_id * ORd_num_of_states)+V] )temp_result[sample_id].dvmdt_max = d_RATES[(sample_id * ORd_num_of_states)+V];
          
          // this part should be
          // "get the peak Vm 6 secs after depolarization (when Na channel just closed after bursting)" 
          //now it has a different if
			    if( tcurr[sample_id] > ((d_CONSTANTS[(sample_id * ORd_num_of_constants) +BCL]*pace_count)+(d_CONSTANTS[(sample_id * ORd_num_of_constants) +stim_start]+2.)) && 
				      tcurr[sample_id] < ((d_CONSTANTS[(sample_id * ORd_num_of_constants) +BCL]*pace_count)+(d_CONSTANTS[(sample_id * ORd_num_of_constants) +stim_start]+10.)) && 
				      abs(d_ALGEBRAIC[(sample_id * ORd_num_of_algebraic) +INa]) < 1)
          {
            // printf("check 1\n");
            if( d_STATES[(sample_id * ORd_num_of_states) +V] > temp_result[sample_id].vm_peak )
            {
              temp_result[sample_id].vm_peak = d_STATES[(sample_id * ORd_num_of_states) +V];

              if(temp_result[sample_id].vm_peak > 0)
              {
                vm_repol30 = temp_result[sample_id].vm_peak - (0.3 * (temp_result[sample_id].vm_peak - temp_result[sample_id].vm_valley));
                vm_repol50 = temp_result[sample_id].vm_peak - (0.5 * (temp_result[sample_id].vm_peak - temp_result[sample_id].vm_valley));
                vm_repol90 = temp_result[sample_id].vm_peak - (0.9 * (temp_result[sample_id].vm_peak - temp_result[sample_id].vm_valley));
                is_eligible_AP = true;
                t_peak_capture = tcurr[sample_id];
                // printf("check 2\n");
               
              }
              // else is_eligible_AP = false;
            }
			    }
           // these operations will be executed if it's eligible AP and executed at the beginning of repolarization
			    else if( tcurr[sample_id] > ((d_CONSTANTS[(sample_id * ORd_num_of_constants) +BCL]*pace_count)+(d_CONSTANTS[(sample_id * ORd_num_of_constants) +stim_start]+10)) && is_eligible_AP )
          {
            // printf("check 3\n");
            // printf("rates: %lf, dvmdt_repol: %lf\n states: %lf vm30: %lf, vm90: %lf\n",
            // d_RATES[(sample_id * ORd_num_of_rates) +V],
            // temp_result->dvmdt_repol, 
            // d_STATES[(sample_id * ORd_num_of_states) +V],
            // vm_repol30,
            // vm_repol90
            // );
            // check for valley update
            if( d_STATES[(sample_id * ORd_num_of_states) +cai] < temp_result[sample_id].ca_valley ){
              temp_result[sample_id].ca_valley = d_STATES[(sample_id * ORd_num_of_states) +cai] ;
              // printf("ca valley update\n");
            }


				    if( d_RATES[(sample_id * ORd_num_of_rates) +V] > temp_result[sample_id].dvmdt_repol &&
					      d_STATES[(sample_id * ORd_num_of_states) +V] <= vm_repol30 &&
					      d_STATES[(sample_id * ORd_num_of_states) +V] >= vm_repol90 )
              {
					      temp_result[sample_id].dvmdt_repol = d_RATES[(sample_id * ORd_num_of_rates) +V];
                // printf("check 4\n");
				      }
              // get the APD90, APD50, peak calcium, 50% and 90% of amplitude of Calcium, and time of peak calcium
                // if (sample_id == 1) printf("tcurr[1] : %lf\n",tcurr[sample_id]);

                if( vm_repol50 > d_STATES[(sample_id * ORd_num_of_states) +V] && d_STATES[(sample_id * ORd_num_of_states) +V] > vm_repol50-2 ){
                  temp_result[sample_id].apd50 = tcurr[sample_id] - t_depol;
                  //printf("tcurr: %lf t_depol : %lf\n", tcurr[sample_id], t_depol);  
                } 
                if( vm_repol90 > d_STATES[(sample_id * ORd_num_of_states) +V] && d_STATES[(sample_id * ORd_num_of_states) +V] > vm_repol90-2 ){
                  temp_result[sample_id].apd90 = tcurr[sample_id] - t_depol;
                  } 

                if( temp_result[sample_id].ca_peak < d_STATES[(sample_id * ORd_num_of_states)+cai] ){
                  temp_result[sample_id].ca_peak = d_STATES[(sample_id * ORd_num_of_states) +cai];
                  ca_amp50 = temp_result[sample_id].ca_peak - (0.5 * (temp_result[sample_id].ca_peak - temp_result[sample_id].ca_valley));
                  ca_amp90 = temp_result[sample_id].ca_peak - (0.9 * (temp_result[sample_id].ca_peak - temp_result[sample_id].ca_valley));
                  t_ca_peak = tcurr[sample_id];
                  // printf("ca_amp50 = %lf - (0.5 * (%lf - %lf)) = %lf\n",temp_result[sample_id].ca_peak, temp_result[sample_id].ca_peak,temp_result[sample_id].ca_valley, ca_amp50);
                  // printf("ca_amp90 = %lf - (0.9 * (%lf - %lf)) = %lf\n",temp_result[sample_id].ca_peak, temp_result[sample_id].ca_peak,temp_result[sample_id].ca_valley, ca_amp90);
                  }
          }
          

			    // calculate AP shape
			    // if(is_eligible_AP && d_STATES[(sample_id * ORd_num_of_states) +V] > vm_repol90)
          // {
          //   // printf("check 5 (eligible)\n");
          // // inet_ap/qnet_ap under APD.
          // // inet_ap = (d_ALGEBRAIC[(sample_id * ORd_num_of_algebraic) +INaL]+d_ALGEBRAIC[(sample_id * ORd_num_of_algebraic) +ICaL]+d_ALGEBRAIC[(sample_id * ORd_num_of_algebraic) +Ito]+d_ALGEBRAIC[(sample_id * ORd_num_of_algebraic) +IKr]+d_ALGEBRAIC[(sample_id * ORd_num_of_algebraic) +IKs]+d_ALGEBRAIC[(sample_id * ORd_num_of_algebraic) +IK1]);
          // // inet4_ap = (d_ALGEBRAIC[(sample_id * ORd_num_of_algebraic) +INaL]+d_ALGEBRAIC[(sample_id * ORd_num_of_algebraic) +ICaL]+d_ALGEBRAIC[(sample_id * ORd_num_of_algebraic) +IKr]+d_ALGEBRAIC[(sample_id * ORd_num_of_algebraic) +INa]);
          // // qnet_ap += (inet_ap * dt[sample_id])/1000.;
          // // qnet4_ap += (inet4_ap * dt[sample_id])/1000.;
          // // inal_auc_ap += (d_ALGEBRAIC[(sample_id * ORd_num_of_algebraic) +INaL]*dt[sample_id]);
          // // ical_auc_ap += (d_ALGEBRAIC[(sample_id * ORd_num_of_algebraic) +ICaL]*dt[sample_id]);
			    // }
          // inet_ap/qnet_ap under Cycle Length
          // inet_cl = (d_ALGEBRAIC[(sample_id * ORd_num_of_algebraic) +INaL]+d_ALGEBRAIC[(sample_id * ORd_num_of_algebraic) +ICaL]+d_ALGEBRAIC[(sample_id * ORd_num_of_algebraic) +Ito]+d_ALGEBRAIC[(sample_id * ORd_num_of_algebraic) +IKr]+d_ALGEBRAIC[(sample_id * ORd_num_of_algebraic) +IKs]+d_ALGEBRAIC[(sample_id * ORd_num_of_algebraic) +IK1]);
          // inet4_cl = (d_ALGEBRAIC[(sample_id * ORd_num_of_algebraic) +INaL]+d_ALGEBRAIC[(sample_id * ORd_num_of_algebraic) +ICaL]+d_ALGEBRAIC[(sample_id * ORd_num_of_algebraic) +IKr]+d_ALGEBRAIC[(sample_id * ORd_num_of_algebraic) +INa]);
          // qnet_cl += (inet_cl * dt[sample_id])/1000.;
          // qnet4_cl += (inet4_cl * dt[sample_id])/1000.;
          // inal_auc_cl += (d_ALGEBRAIC[(sample_id * ORd_num_of_algebraic) +INaL]*dt[sample_id]);
          // ical_auc_cl += (d_ALGEBRAIC[(sample_id * ORd_num_of_algebraic) +ICaL]*dt[sample_id]);

          // save temporary result -> ALL TEMP RESULTS IN, TEMP RESULT != WRITTEN RESULT
          float tolerance = 0.001f;
          if(cipa_datapoint<p_param->sampling_limit && fmodf(tcurr[sample_id], 1.0f) < tolerance){ // temporary solution to limit the datapoint :(
            if(sample_id==0) {printf("%lf\n", tcurr[sample_id]);}
            temp_result[sample_id].cai_data[cipa_datapoint] =  d_STATES[(sample_id * ORd_num_of_states) +cai] ;
            temp_result[sample_id].cai_time[cipa_datapoint] =  tcurr[sample_id];
            // printf("core: %d, cai_data and time:  %lf %lf datapoint: %d\n",
            // sample_id,
            // temp_result[sample_id].cai_data[cipa_datapoint],
            // temp_result[sample_id].cai_time[cipa_datapoint],
            // cipa_datapoint  );

            temp_result[sample_id].vm_data[cipa_datapoint] = d_STATES[(sample_id * ORd_num_of_states) +V];
            temp_result[sample_id].vm_time[cipa_datapoint] = tcurr[sample_id];

            temp_result[sample_id].dvmdt_data[cipa_datapoint] = d_RATES[(sample_id * ORd_num_of_rates) +V];
            temp_result[sample_id].dvmdt_time[cipa_datapoint] = tcurr[sample_id];

            // time series result

            time[input_counter + sample_id] = tcurr[sample_id];
            states[input_counter + sample_id] = d_STATES[V + (sample_id * ORd_num_of_states)];
            
            out_dt[input_counter + sample_id] = d_RATES[V + (sample_id * ORd_num_of_states)];
            
            cai_result[input_counter + sample_id] = d_STATES[(sample_id * ORd_num_of_states) +cai];

            ina[input_counter + sample_id] = d_ALGEBRAIC[INa + (sample_id * ORd_num_of_algebraic)] ;
            inal[input_counter + sample_id] = d_ALGEBRAIC[INaL + (sample_id * ORd_num_of_algebraic)] ;

            ical[input_counter + sample_id] = d_ALGEBRAIC[ICaL + (sample_id * ORd_num_of_algebraic)] ;
            ito[input_counter + sample_id] = d_ALGEBRAIC[Ito + (sample_id * ORd_num_of_algebraic)] ;

            ikr[input_counter + sample_id] = d_ALGEBRAIC[IKr + (sample_id * ORd_num_of_algebraic)] ;
            iks[input_counter + sample_id] = d_ALGEBRAIC[IKs + (sample_id * ORd_num_of_algebraic)] ;

            ik1[input_counter + sample_id] = d_ALGEBRAIC[IK1 + (sample_id * ORd_num_of_algebraic)] ;

            tension[input_counter + sample_id] = d_mec_ALGEBRAIC[land_T + (sample_id * 24)] * 480.0;

            input_counter = input_counter + sample_size;
            cipa_datapoint = cipa_datapoint + 1; // this causes the resource usage got so mega and crashed in running
          }

          // cipa result update
              cipa_result[sample_id].qnet = temp_result[sample_id].qnet;
              cipa_result[sample_id].inal_auc = temp_result[sample_id].inal_auc;
              cipa_result[sample_id].ical_auc = temp_result[sample_id].ical_auc;
              cipa_result[sample_id].dvmdt_repol = temp_result[sample_id].dvmdt_repol;
              cipa_result[sample_id].dvmdt_max = temp_result[sample_id].dvmdt_max;
              
              cipa_result[sample_id].vm_dia = temp_result[sample_id].vm_dia;
              cipa_result[sample_id].apd90 = temp_result[sample_id].apd90;
              cipa_result[sample_id].apd50 = temp_result[sample_id].apd50;
              cipa_result[sample_id].ca_peak = temp_result[sample_id].ca_peak;
              cipa_result[sample_id].ca_valley = d_STATES[(sample_id * ORd_num_of_states) +cai];
              cipa_result[sample_id].ca_dia = temp_result[sample_id].ca_dia;
              
              
              cipa_result[sample_id].dvmdt_repol = temp_result[sample_id].dvmdt_repol;
              cipa_result[sample_id].vm_peak = temp_result[sample_id].vm_peak;
              cipa_result[sample_id].vm_valley = d_STATES[(sample_id * ORd_num_of_states) +V];

	
        tcurr[sample_id] = tcurr[sample_id] + dt[sample_id];
        //printf("t after addition: %lf\n", tcurr[sample_id]);
       
  }
    // __syncthreads();

    // // looking for cad50 and 90
    for(int ca_looper = 0; ca_looper < p_param->sampling_limit; ca_looper++){
          // before the peak calcium
          
          if( temp_result[sample_id].cai_time[ca_looper] < t_ca_peak ){
            // printf("cai_data %lf \n",temp_result[sample_id].cai_data[ca_looper]);
            if( temp_result[sample_id].cai_data[ca_looper] < ca_amp50 ){
              cad50_prev = temp_result[sample_id].cai_time[ca_looper];
              // printf("cad50 prev update\n");
            } 
            if( temp_result[sample_id].cai_data[ca_looper] < ca_amp90 ){
              cad90_prev = temp_result[sample_id].cai_time[ca_looper];
              // printf("cad90 prev update\n");
            } 
          }
          // after the peak calcium
          else{
            if( temp_result[sample_id].cai_data[ca_looper] > ca_amp50 ) cad50_curr = temp_result[sample_id].cai_time[ca_looper];
            if( temp_result[sample_id].cai_data[ca_looper] > ca_amp90 ) cad90_curr = temp_result[sample_id].cai_time[ca_looper];
          }
        }
      // printf("core: %d ca_peak %lf | : 50: %lf - %lf 90: %lf - %lf\n",sample_id, t_ca_peak, cad50_curr, cad50_prev, cad90_curr, cad90_prev);
      // printf("ca_peak: %lf ca_valley %lf\n", temp_result[sample_id].ca_peak, temp_result[sample_id].ca_valley);
      // printf("cai_data[0] %lf \n",temp_result[sample_id].cai_data[0]);
      temp_result[sample_id].cad50 = cad50_curr - cad50_prev;// the curr is lower than the prev, like waaay lower, its a negative (it shouldnt be, since its in time)
      temp_result[sample_id].cad90 = cad90_curr - cad90_prev;
      cipa_result[sample_id].cad90 = temp_result[sample_id].cad90;
      cipa_result[sample_id].cad50 = temp_result[sample_id].cad50;
}

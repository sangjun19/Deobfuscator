#include "hip/hip_runtime.h"
// Repository: ctuning/ctuning-programs
// File: program/polybench-cuda-corr/correlation.cu

/**
 * correlation.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 *
 * Updated by Grigori Fursin (http://cTuning.org/lab/people/gfursin)
 * to work with Collective Mind, OpenME plugin interface and 
 * Collective Knowledge Frameworks for automatic, machine-learning based
 * and collective tuning and data mining: http://cTuning.org
 *
 */

#ifndef WINDOWS
 #include <unistd.h>
#endif

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>

#include <hip/hip_runtime.h>

#include "polybench.h"

#ifdef OPENME
#include <openme.h>
#endif
#ifdef XOPENME
#include <xopenme.h>
#endif

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 1.05

#define GPU_DEVICE 0

/* Problem size */
#ifndef NI
#define NI 512 //2048
#endif
#ifndef NJ
#define NJ 512 //2048
#endif

/* Thread block dimensions for kernel 1*/
#ifndef DIM_THREAD_BLOCK_KERNEL_1_X
#define DIM_THREAD_BLOCK_KERNEL_1_X 256
#endif
#ifndef DIM_THREAD_BLOCK_KERNEL_1_Y
#define DIM_THREAD_BLOCK_KERNEL_1_Y 1
#endif

/* Thread block dimensions for kernel 2*/
#ifndef DIM_THREAD_BLOCK_KERNEL_2_X
#define DIM_THREAD_BLOCK_KERNEL_2_X 256
#endif
#ifndef DIM_THREAD_BLOCK_KERNEL_2_Y
#define DIM_THREAD_BLOCK_KERNEL_2_Y 1
#endif

/* Thread block dimensions for kernel 3*/
#ifndef DIM_THREAD_BLOCK_KERNEL_3_X
#define DIM_THREAD_BLOCK_KERNEL_3_X 32
#endif
#ifndef DIM_THREAD_BLOCK_KERNEL_3_Y
#define DIM_THREAD_BLOCK_KERNEL_3_Y 8
#endif

/* Thread block dimensions for kernel 4*/
#ifndef DIM_THREAD_BLOCK_KERNEL_4_X
#define DIM_THREAD_BLOCK_KERNEL_4_X 256
#endif
#ifndef DIM_THREAD_BLOCK_KERNEL_4_Y
#define DIM_THREAD_BLOCK_KERNEL_4_Y 1
#endif

#define sqrt_of_array_cell(x,j) sqrt(x[j])

#define FLOAT_N 3214212.01f
#define EPS 0.005f

/* Can switch DATA_TYPE between float and double */
# ifndef DATA_TYPE
#  define DATA_TYPE float
# endif

void init_arrays(DATA_TYPE* data)
{
	int i, j;
	
	for (i=0; i < (NI+1); i++) 
	{
    		for (j=0; j< (NJ+1); j++) 
		{
       			data[i*(NJ+1) + j] = ((DATA_TYPE) i*j)/ (NI+1);	
       		}
    	}
}


void correlation(DATA_TYPE* data, DATA_TYPE* mean, DATA_TYPE* stddev, DATA_TYPE* symmat)
{
	int i, j, j1, j2;	
	
	// Determine mean of column vectors of input data matrix 
  	for (j = 1; j < (NI+1); j++)
   	{
  		mean[j] = 0.0;

   		for (i = 1; i < (NJ+1); i++)
		{
			mean[j] += data[i*(NI+1) + j];
   		}
		
		mean[j] /= (DATA_TYPE)FLOAT_N;
   	}

	// Determine standard deviations of column vectors of data matrix. 
  	for (j = 1; j < (NI+1); j++)
   	{
   		stddev[j] = 0.0;
      
		for (i = 1; i < (NJ+1); i++)
		{
			stddev[j] += (data[i*(NI+1) + j] - mean[j]) * (data[i*(NI+1) + j] - mean[j]);
		}
		
		stddev[j] /= FLOAT_N;
		stddev[j] = sqrt_of_array_cell(stddev, j);
		stddev[j] = stddev[j] <= EPS ? 1.0 : stddev[j];
	}

 	// Center and reduce the column vectors. 
  	for (i = 1; i < (NJ+1); i++)
	{
		for (j = 1; j < (NI+1); j++)
		{
			data[i*(NI+1) + j] -= mean[j];
			data[i*(NI+1) + j] /= (sqrt(FLOAT_N)*stddev[j]) ;
		}
	}

	// Calculate the m * m correlation matrix. 
  	for (j1 = 1; j1 < NI; j1++)
	{	
		symmat[j1*(NI+1) + j1] = 1.0;
    
		for (j2 = j1+1; j2 < (NI+1); j2++)
		{
	  		symmat[j1*(NI+1) + j2] = 0.0;

	  		for (i = 1; i < (NJ+1); i++)
			{
	   			symmat[j1*(NI+1) + j2] += (data[i*(NI+1) + j1] * data[i*(NI+1) + j2]);
			}

	  		symmat[j2*(NI+1) + j1] = symmat[j1*(NI+1) + j2];
		}
	}
 
	symmat[NI*(NI+1) + NI] = 1.0;
}


void compareResults(DATA_TYPE* symmat, DATA_TYPE* symmat_outputFromGpu)
{
	int i,j,fail;
	fail = 0;

	for (i=1; i < (NI+1); i++)
	{
		for (j=1; j < (NJ+1); j++)
		{
			if (percentDiff(symmat[i*(NJ+1) + j], symmat_outputFromGpu[i*(NJ+1) + j]) > PERCENT_DIFF_ERROR_THRESHOLD)
			{
				fail++;
//				printf("i: %d j: %d\n1: %f 2: %f\n", i, j, symmat[i*N + j], symmat_outputFromGpu[i*N + j]);
		
			}
		}
	}
	
	// print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}


void GPU_argv_init()
{
  /* Grigori Fursin added support for CK widgets */
  int gpgpu_device_id=GPU_DEVICE;

  int devID = 0;
  hipError_t error;
  hipDeviceProp_t deviceProp;
  error = hipGetDevice(&devID);

  if (getenv("CK_COMPUTE_DEVICE_ID")!=NULL) gpgpu_device_id=atol(getenv("CK_COMPUTE_DEVICE_ID"));

  hipGetDeviceProperties(&deviceProp, gpgpu_device_id);

  if (deviceProp.computeMode == hipComputeModeProhibited)
  {
    printf("Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
    exit(EXIT_SUCCESS);
  }

  if (error != hipSuccess)
    printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
  else
    printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);

  hipSetDevice( gpgpu_device_id );
}

	
__global__ void mean_kernel(DATA_TYPE *mean, DATA_TYPE *data)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x + 1;

	if ((j >= 1) && (j < (NI+1)))
	{
		mean[j] = 0.0;

		int i;
		for(i=1; i < (NJ+1); i++)
		{
			mean[j] += data[i*(NI+1) + j];
		}
		
		mean[j] /= (DATA_TYPE)FLOAT_N;
	}
}


__global__ void std_kernel(DATA_TYPE *mean, DATA_TYPE *std, DATA_TYPE *data)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x + 1;
	
	if ((j >= 1) && (j < (NI+1)))
	{
		std[j] = 0.0;

		int i;
		for(i = 1; i < (NJ+1); i++)
		{
			std[j] += (data[i*(NI+1) + j] - mean[j]) * (data[i*(NI+1) + j] - mean[j]);
		}
		std[j] /= (FLOAT_N);
		std[j] = sqrt(std[j]);
		if(std[j] <= EPS) 
		{
			std[j] = 1.0;
		}
	}
}


__global__ void reduce_kernel(DATA_TYPE *mean, DATA_TYPE *std, DATA_TYPE *data)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x + 1;
	int i = blockIdx.y * blockDim.y + threadIdx.y + 1;
	
	if ((i >= 1) && (i < (NJ+1)) && (j >= 1) && (j < (NI+1)))
	{
		data[i*(NI+1) + j] -= mean[j];
		data[i*(NI+1) + j] /= (sqrt(FLOAT_N) * std[j]);
	}
}


__global__ void corr_kernel(DATA_TYPE *symmat, DATA_TYPE *data)
{
	int j1 = blockIdx.x * blockDim.x + threadIdx.x + 1;

	int i, j2;
	if ((j1 >= 1) && (j1 < NI))
	{
		symmat[j1*(NI+1) + j1] = 1.0;

		for (j2 = (j1 + 1); j2 < (NI+1); j2++)
		{
			symmat[j1*(NI+1) + j2] = 0.0;

			for(i = 1; i < (NJ+1); i++)
			{
				symmat[j1*(NI+1) + j2] += data[i*(NI+1) + j1] * data[i*(NI+1) + j2];
			}
			symmat[j2*(NI+1) + j1] = symmat[j1*(NI+1) + j2];
		}
	}
}


void correlationCuda(DATA_TYPE* data, DATA_TYPE* mean, DATA_TYPE* stddev, DATA_TYPE* symmat,
			DATA_TYPE* symmat_outputFromGpu)
{
        hipError_t error;
	double t_start, t_end;

	DATA_TYPE *data_gpu;
	DATA_TYPE *stddev_gpu;
	DATA_TYPE *mean_gpu;
	DATA_TYPE *symmat_gpu;

	error=hipMalloc((void **)&data_gpu, sizeof(DATA_TYPE) * (NI+1) * (NJ+1));
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

	error=hipMalloc((void **)&symmat_gpu, sizeof(DATA_TYPE) * (NI+1) * (NJ+1));
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

	error=hipMalloc((void **)&stddev_gpu, sizeof(DATA_TYPE) * (NI+1));
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

	error=hipMalloc((void **)&mean_gpu, sizeof(DATA_TYPE) * (NI+1));
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

	error=hipMemcpy(data_gpu, data, sizeof(DATA_TYPE) * (NI+1) * (NJ+1), hipMemcpyHostToDevice);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

	error=hipMemcpy(symmat_gpu, symmat, sizeof(DATA_TYPE) * (NI+1) * (NJ+1), hipMemcpyHostToDevice);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

	error=hipMemcpy(stddev_gpu, stddev, sizeof(DATA_TYPE) * (NI+1), hipMemcpyHostToDevice);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

	error=hipMemcpy(mean_gpu, mean, sizeof(DATA_TYPE) * (NI+1), hipMemcpyHostToDevice);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }
		
	dim3 block1(DIM_THREAD_BLOCK_KERNEL_1_X, DIM_THREAD_BLOCK_KERNEL_1_Y);
	dim3 grid1((size_t)(ceil((float)(NI)) / ((float)DIM_THREAD_BLOCK_KERNEL_1_X)), 1);
	
	dim3 block2(DIM_THREAD_BLOCK_KERNEL_2_X, DIM_THREAD_BLOCK_KERNEL_2_Y);
	dim3 grid2((size_t)(ceil((float)(NI)) / ((float)DIM_THREAD_BLOCK_KERNEL_2_X)), 1);
	
	dim3 block3(DIM_THREAD_BLOCK_KERNEL_3_X, DIM_THREAD_BLOCK_KERNEL_3_Y);
	dim3 grid3((size_t)(ceil((float)(NI)) / ((float)DIM_THREAD_BLOCK_KERNEL_3_X)), (size_t)(ceil((float)(NJ)) / ((float)DIM_THREAD_BLOCK_KERNEL_3_Y)));
	
	dim3 block4(DIM_THREAD_BLOCK_KERNEL_4_X, DIM_THREAD_BLOCK_KERNEL_4_Y);
	dim3 grid4((size_t)(ceil((float)(NI)) / ((float)DIM_THREAD_BLOCK_KERNEL_4_X)), 1);

//	t_start = rtclock();
	mean_kernel<<< grid1, block1 >>>(mean_gpu,data_gpu);
	hipDeviceSynchronize();
	std_kernel<<< grid2, block2 >>>(mean_gpu,stddev_gpu,data_gpu);
	hipDeviceSynchronize();
	reduce_kernel<<< grid3, block3 >>>(mean_gpu,stddev_gpu,data_gpu);
	hipDeviceSynchronize();
	corr_kernel<<< grid4, block4 >>>(symmat_gpu,data_gpu);
	hipDeviceSynchronize();
//	t_end = rtclock();
//	fprintf(stdout, "GPU Runtime: %0.6lfs\n", t_end - t_start);	

	DATA_TYPE valueAtSymmatIndexMTimesMPlus1PlusMPoint = 1.0;
	error=hipMemcpy(&(symmat_gpu[(NI)*(NI+1) + (NI)]), &valueAtSymmatIndexMTimesMPlus1PlusMPoint, sizeof(DATA_TYPE), hipMemcpyHostToDevice);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

	error=hipMemcpy(symmat_outputFromGpu, symmat_gpu, sizeof(DATA_TYPE) * (NI+1) * (NJ+1), hipMemcpyDeviceToHost);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }
	
	hipFree(data_gpu);
	hipFree(symmat_gpu);
	hipFree(stddev_gpu);
	hipFree(mean_gpu);
}


int main()
{
  /* Prepare ctuning vars */
  long ct_repeat=0;
  long ct_repeat_max=1;

  double t_start, t_end;

  DATA_TYPE* data;
  DATA_TYPE* mean;
  DATA_TYPE* stddev;
  DATA_TYPE* symmat;
  DATA_TYPE* symmat_outputFromGpu;

#ifdef XOPENME
  xopenme_init(2,0);
#endif

#ifdef OPENME
  openme_init(NULL,NULL,NULL,0);
  openme_callback("PROGRAM_START", NULL);
#endif

  /* Run kernel. */
  if (getenv("CT_REPEAT_MAIN")!=NULL) ct_repeat_max=atol(getenv("CT_REPEAT_MAIN"));

  data = (DATA_TYPE*)malloc((NI+1)*(NJ+1)*sizeof(DATA_TYPE));
  mean = (DATA_TYPE*)malloc((NI+1)*sizeof(DATA_TYPE));
  stddev = (DATA_TYPE*)malloc((NI+1)*sizeof(DATA_TYPE));
  symmat = (DATA_TYPE*)malloc((NI+1)*(NJ+1)*sizeof(DATA_TYPE));
  symmat_outputFromGpu = (DATA_TYPE*)malloc((NI+1)*(NJ+1)*sizeof(DATA_TYPE));

  srand(1);
  init_arrays(data);
  GPU_argv_init();

#ifdef OPENME
  openme_callback("ACC_KERNEL_START", NULL);
#endif
#ifdef XOPENME
  xopenme_clock_start(0);
#endif
  for (ct_repeat=0; ct_repeat<ct_repeat_max; ct_repeat++)
  {
    correlationCuda(data, mean, stddev, symmat, symmat_outputFromGpu);
  }
#ifdef XOPENME
  xopenme_clock_end(0);
#endif
#ifdef OPENME
  openme_callback("ACC_KERNEL_END", NULL);
#endif

/*
  srand(1);
  init_arrays(data);

#ifdef OPENME
  openme_callback("KERNEL_START", NULL);
#endif
#ifdef XOPENME
  xopenme_clock_start(1);
#endif
  for (ct_repeat=0; ct_repeat<ct_repeat_max; ct_repeat++)
  {
    correlation(data, mean, stddev, symmat);
  }
#ifdef XOPENME
  xopenme_clock_end(1);
#endif
#ifdef OPENME
  openme_callback("KERNEL_END", NULL);
#endif
*/

  compareResults(symmat, symmat_outputFromGpu);

  free(data);
  free(mean);
  free(stddev);
  free(symmat);
  free(symmat_outputFromGpu);

#ifdef XOPENME
  xopenme_dump_state();
  xopenme_finish();
#endif

#ifdef OPENME
  openme_callback("PROGRAM_END", NULL);
#endif

  return 0;
}


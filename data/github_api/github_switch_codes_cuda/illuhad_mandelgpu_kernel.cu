#include "hip/hip_runtime.h"
// Repository: illuhad/mandelgpu
// File: kernel.cu

/*
 * This file is part of mandelgpu, a free GPU accelerated fractal viewer,
 * Copyright (C) 2016  Aksel Alpay
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <cassert>

#include "kernel.hpp"
#include "cuda_error.hpp"

#define WITHOUT_HALF //half precision is not yet supported, as it would likely (?)
                     //require a different kernel
#ifndef WITHOUT_HALF
#include <hip/hip_fp16.h>
#endif


const int maxiterations = 2048;
const double limit = 1000.;

__device__ 
uchar3 hsv_to_rgb(const float h,
                  const float s,
                  const float v)
{
  int h_i = h / 60;
  
  float f = h / 60.f - h_i;
  
  float p = v * (1.f - s);
  float q = v * (1.f - s * f);
  float t = v * (1.f - s * (1.f - f));
  
  float3 rgb_temp;
  
  switch(h_i)
  {
  case 0:
    rgb_temp = make_float3(v,t,p);
    break;
  case 1:
    rgb_temp = make_float3(q,v,p);
    break;
  case 2:
    rgb_temp = make_float3(p,v,t);
    break;
  case 3:
    rgb_temp = make_float3(p,q,v);
    break;
  case 4:
    rgb_temp = make_float3(t,p,v);
    break;
  case 5:
    rgb_temp = make_float3(v,p,q);
    break;
  case 6:
    rgb_temp = make_float3(v,t,p);
    break;
  }
  
  float r = rgb_temp.x * 255.f;
  float g = rgb_temp.y * 255.f;
  float b = rgb_temp.z * 255.f;
  uchar3 result = make_uchar3((unsigned char)r,
                              (unsigned char)g,
                              (unsigned char)b);
  return result;
}


__device__
uchar3 color_scheme(int num_iterations, int max_iterations)
{
  uchar3 color;
  
  if(num_iterations == max_iterations)
    color = make_uchar3(0,0,0);
  else
  {
    float h = 360.f * (0.5f * sin(num_iterations * 2.f * M_PI / (float)max_iterations + 0.2f) + 0.5f);
    float s = 0.4f * sin(num_iterations * 0.5f * M_PI / 13.f) + 0.5f;
    float v = 0.3f * sin(num_iterations * 0.5f * M_PI / 200.f) + 0.7f;
    return hsv_to_rgb(h,s,v);
  }
  
  return color;
}


template<typename Arithmetic_type, typename Arithmetic_type2>
__global__
void mandelgpu(Arithmetic_type dx, Arithmetic_type center_x, Arithmetic_type center_y,
               uchar3* pixels, int npx_x, int npx_y)
{
  int gid_x = threadIdx.x + blockIdx.x * blockDim.x;
  int gid_y = threadIdx.y + blockIdx.y * blockDim.y;
  
  
  for(int px_x = gid_x; px_x < npx_x; px_x += blockDim.x * gridDim.x)
  {
    for(int px_y = gid_y; px_y < npx_y; px_y += blockDim.y * gridDim.y)
    {
      int iter_counter = 0;
      
      Arithmetic_type2 coord;
      coord.x = center_x + (px_x - npx_x / 2) * dx;
      coord.y = center_y + (px_y - npx_y / 2) * dx;
      
      Arithmetic_type2 z = coord;
      
#pragma unroll 128
      for(int i = 0; i < maxiterations; ++i)
      {         
        Arithmetic_type2 old_z = z;
        
        z = coord;
        
        Arithmetic_type a2 = 2.f * old_z.x;
        z.x += old_z.x * old_z.x;
        z.x -= old_z.y * old_z.y;
        z.y += a2 * old_z.y;
        
        Arithmetic_type norm2 = z.x * z.x;
        norm2 += z.y * z.y;

        if(norm2 > limit)
          break;
        ++iter_counter;
      }
      
      uchar3 color = color_scheme(iter_counter, maxiterations);
      pixels[px_y * npx_x + px_x] = color;
    }
  }
}


template<typename Arithmetic_type, typename Arithmetic_type2>
__global__
void juliagpu(Arithmetic_type dx, Arithmetic_type center_x, Arithmetic_type center_y,
               uchar3* pixels, int npx_x, int npx_y,
               const Arithmetic_type c0_x,
               const Arithmetic_type c0_y)
{
  int gid_x = threadIdx.x + blockIdx.x * blockDim.x;
  int gid_y = threadIdx.y + blockIdx.y * blockDim.y;
  
  
  for(int px_x = gid_x; px_x < npx_x; px_x += blockDim.x * gridDim.x)
  {
    for(int px_y = gid_y; px_y < npx_y; px_y += blockDim.y * gridDim.y)
    {
      int iter_counter = 0;
      
      Arithmetic_type2 coord;
      coord.x = center_x + (px_x - npx_x / 2) * dx;
      coord.y = center_y + (px_y - npx_y / 2) * dx;
      
      Arithmetic_type2 z = coord;
      
#pragma unroll 128
      for(int i = 0; i < maxiterations; ++i)
      { 
        Arithmetic_type2 old_z = z;
        
        z.x = c0_x;
        z.y = c0_y;

        Arithmetic_type a2 = 2.f * old_z.x;
        z.x += old_z.x * old_z.x;
        z.x -= old_z.y * old_z.y;
        z.y += a2 * old_z.y;
        
        Arithmetic_type norm2 = z.x * z.x;
        norm2 += z.y * z.y;

        if(norm2 > limit)
          break;
        
        ++iter_counter;
      }

      uchar3 color = color_scheme(iter_counter, maxiterations);
      pixels[px_y * npx_x + px_x] = color;
    }
  }
  
}

template<typename Arithmetic_type, typename Arithmetic_type2>
__device__ __forceinline__
void complex_mult(Arithmetic_type2 a, Arithmetic_type2 b, Arithmetic_type2* out)
{
  out->x  = a.x * b.x;
  out->x -= a.y * b.y;
  
  out->y  = a.x * b.y;
  out->y += a.y * b.x;
}

template<typename Arithmetic_type, typename Arithmetic_type2>
__device__ __forceinline__
void complex_scale_add(Arithmetic_type2* out, Arithmetic_type2 s, Arithmetic_type2 b)
{
  out->x += s.x * b.x;
  out->x -= s.y * b.y;
  
  out->y += s.x * b.y;
  out->y += s.y * b.x;
}


template<typename Arithmetic_type, typename Arithmetic_type2, int Polynomial_degree>
__global__
void polynomial_kernel(Arithmetic_type dx, 
                   Arithmetic_type center_x, Arithmetic_type center_y,
                   uchar3* pixels, int npx_x, int npx_y,
                   Arithmetic_type2* coefficients,
                   Arithmetic_type2* coordinate_contrib_to_coefficients,
                   Arithmetic_type2 initial_value,
                   Arithmetic_type2  coordinate_contrib_to_initial_value)
{
  int gid_x = threadIdx.x + blockIdx.x * blockDim.x;
  int gid_y = threadIdx.y + blockIdx.y * blockDim.y;
  
  Arithmetic_type2 effective_coefficients [Polynomial_degree + 1];
  Arithmetic_type2 z_powers[Polynomial_degree];
  
  for(int px_x = gid_x; px_x < npx_x; px_x += blockDim.x * gridDim.x)
  {
    for(int px_y = gid_y; px_y < npx_y; px_y += blockDim.y * gridDim.y)
    {
      int iter_counter = 0;
      
      Arithmetic_type2 coord;
      coord.x = center_x + (px_x - npx_x / 2) * dx;
      coord.y = center_y + (px_y - npx_y / 2) * dx;
      
      Arithmetic_type2 z = initial_value;
      complex_scale_add(&z, coordinate_contrib_to_initial_value, coord);
      
      // Initialise coefficients for pixel
      for(int i = 0; i < Polynomial_degree + 1; ++i)
      {
        effective_coefficients[i] = coefficients[i];
        complex_scale_add(&effective_coefficients[i], 
                          coordinate_contrib_to_coefficients[i], coord);
      }
      
      Arithmetic_type new_z;
#pragma unroll 128
      for(int i = 0; i < maxiterations; ++i)
      {
        new_z = effective_coefficients[0];
        
        // Generate powers
        z_powers[0] = z;
        complex_scale_add(&new_z, z, effective_coefficients[1]);
        
        for(int i = 1; i < Polynomial_degree; ++i)
        {
          complex_mult(z_powers[i-1], z, &z_powers[i]);
          complex_scale_add(&new_z, z_powers[i], effective_coefficients[i + 1]);
        }
        
        z = new_z;
        
        Arithmetic_type norm2 = z.x * z.x;
        norm2 += z.y * z.y;

        if(norm2 > limit)
          break;
        
        ++iter_counter;
      }

      uchar3 color = color_scheme(iter_counter, maxiterations);
      pixels[px_y * npx_x + px_x] = color;
    }
  }
}

cuda_polynomial_coefficients::cuda_polynomial_coefficients(int degree)
: _coefficients(degree + 1), _float_coefficients(degree + 1),
  _device_float_coefficients(nullptr),
  _device_double_coefficients(nullptr)
{
  hipMalloc(&_device_float_coefficients, 2 * (degree + 1) * sizeof(float));
  hipMalloc(&_device_double_coefficients, 2 * (degree + 1) * sizeof(double));
  
  check_cuda_error("Could not allocate device memory!");
}

cuda_polynomial_coefficients::~cuda_polynomial_coefficients()
{
  if(_device_float_coefficients)
    hipFree(_device_float_coefficients);
  
  if(_device_double_coefficients)
    hipFree(_device_double_coefficients);
}

void cuda_polynomial_coefficients::commit()
{
  assert(_device_float_coefficients);
  assert(_device_double_coefficients);
  
  for(std::size_t i = 0; i < _coefficients.size(); ++i)
  {
    _float_coefficients[i].real = static_cast<float>(_coefficients[i].real);
    _float_coefficients[i].imag = static_cast<float>(_coefficients[i].imag);
  }
  
  hipMemcpy(_device_float_coefficients, _float_coefficients.data(), 
              2 * _coefficients.size() * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(_device_double_coefficients, _coefficients.data(), 
              2 * _coefficients.size() * sizeof(double), hipMemcpyHostToDevice);
  
  check_cuda_error("Could not transfer coefficients to device!");
}



template<typename Arithmetic_type, typename Arithmetic_type2>
performance_estimator::result run_kernel(unsigned char* pixels, 
                std::size_t width, std::size_t height,
                double size_x, 
                double center_x, double center_y,
                double c0_x, double c0_y, // for julia
                kernel_type kernel)
{
  std::size_t npx_x = width;
  std::size_t npx_y = height;

  Arithmetic_type dx = size_x / static_cast<Arithmetic_type>(npx_x);
  
  std::size_t num_bytes = npx_x * npx_y * sizeof(uchar3);

  std::size_t nthreads = 8;
  std::size_t nblocks_x = npx_x / nthreads;
  std::size_t nblocks_y = npx_y / nthreads;
  
  if(npx_x % nthreads != 0)
    ++nblocks_x;
  if(npx_y % nthreads != 0)
    ++nblocks_y;
  
  dim3 threads = dim3(nthreads, nthreads, 1);
  dim3 blocks = dim3(nblocks_x, nblocks_y, 1);

  performance_estimator perf;
  perf.start();
  
  std::size_t num_bytes_transferred = num_bytes;
  std::size_t flops = npx_x * npx_y * (maxiterations * 10);
  switch(kernel)
  {
  case MANDELBROT:
    mandelgpu<Arithmetic_type, Arithmetic_type2><<<blocks, threads>>>(
                                   static_cast<Arithmetic_type>(dx), 
                                   static_cast<Arithmetic_type>(center_x), 
                                   static_cast<Arithmetic_type>(center_y), 
                                   reinterpret_cast<uchar3*>(pixels), 
                                   static_cast<int>(npx_x), 
                                   static_cast<int>(npx_y));
    break;
  case JULIA:
    juliagpu<Arithmetic_type, Arithmetic_type2><<<blocks, threads>>>(
                                  static_cast<Arithmetic_type>(dx), 
                                  static_cast<Arithmetic_type>(center_x), 
                                  static_cast<Arithmetic_type>(center_y), 
                                  reinterpret_cast<uchar3*>(pixels), 
                                  static_cast<int>(npx_x), 
                                  static_cast<int>(npx_y),
                                  c0_x, c0_y);

    break;
  }

  check_cuda_error("Kernel execution failed!");
  hipDeviceSynchronize();
  
  performance_estimator::result res = perf.stop(num_bytes_transferred, flops);
  
  check_cuda_error("Device synchronization failed!");
  
  return res;
}


performance_estimator::result run_kernel(unsigned char* pixels, 
                std::size_t width, std::size_t height,
                double size_x, 
                double center_x, double center_y,
                double c0_x, double c0_y, // for julia
                kernel_type kernel,
                precision p)
{
  switch(p)
  {
#ifndef WITHOUT_HALF
  case HALF:
    return run_kernel<half, half2>(pixels, 
                            width, height, 
                            size_x, center_x, center_y, 
                            c0_x, c0_y, kernel);
#endif
  case SINGLE:
    return run_kernel<float, float2>(pixels, 
                            width, height, 
                            size_x, center_x, center_y, 
                            c0_x, c0_y, kernel);
  case DOUBLE:
    return run_kernel<double, double2>(pixels, 
                            width, height, 
                            size_x, center_x, center_y, 
                            c0_x, c0_y, kernel);
  default:
    return run_kernel<double, double2>(pixels, 
                            width, height, 
                            size_x, center_x, center_y, 
                            c0_x, c0_y, kernel);
  }
  
}

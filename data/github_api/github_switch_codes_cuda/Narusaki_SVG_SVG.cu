#include "hip/hip_runtime.h"
// Repository: Narusaki/SVG
// File: SVG.cu

#include "SVG.cuh"
#include "book.cuh"
#include <iostream>
#include <fstream>
#include <ctime>

using namespace std;

__global__ void constructSVG(Mesh mesh, int K,
	SVG::PQWinItem *d_winPQs, SVG::PQPseudoWinItem *d_pseudoWinPQs,
	ICHDevice::SplitItem *d_splitInfoBuf, unsigned splitInfoCoef,
	ICHDevice::VertItem *d_vertInfoBuf, unsigned vertInfoCoef,
	ICHDevice::Window *d_storedWindowsBuf, unsigned *d_keptFacesBuf,
	SVG::SVGNode *d_svg, int *d_svg_tails)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int totalThreadNum = blockDim.x * gridDim.x;

	PriorityQueues<ICHDevice::Window> winPQ(WINPQ_SIZE - 1);
	PriorityQueues<ICHDevice::PseudoWindow> pseudoWinPQ(PSEUDOWINPQ_SIZE - 1);
	winPQ.AssignMemory(d_winPQs + idx * WINPQ_SIZE, WINPQ_SIZE - 1);
	pseudoWinPQ.AssignMemory(d_pseudoWinPQs + idx * PSEUDOWINPQ_SIZE, PSEUDOWINPQ_SIZE - 1);

	ICHDevice ich;
	ich.AssignMesh(&mesh);
	ich.AssignBuffers(d_splitInfoBuf + idx * (K * splitInfoCoef + 1), K * splitInfoCoef,
		d_vertInfoBuf + idx * (K * vertInfoCoef + 1), K * vertInfoCoef,
		winPQ, pseudoWinPQ,
		d_storedWindowsBuf + idx * STORED_WIN_BUF_SIZE, d_keptFacesBuf + idx * KEPT_FACE_SIZE);

// 	InitialValueGeodesic initGeodesic;
// 	initGeodesic.AssignMesh(&mesh);

	for (int i = idx; i < mesh.vertNum; i += totalThreadNum)
	{
		// TODO: run ICH
		ich.Clear();
		ich.AddSource(i);
		ich.Execute(K);

		d_svg_tails[i] = 0;
		SVG::SVGNode *d_cur_svg = d_svg + K * i;

		for (int j = 0; j < mesh.vertNum; ++j)
		{
			if (ich.GetDistanceTo(j) == DBL_MAX) continue;
			if (j == i) continue;

			unsigned srcId;
			unsigned nextToSrcEdge, nextToDstEdge;
			double nextSrcX, nextDstX;
			ich.BuildGeodesicPathTo(j, srcId,
				nextToSrcEdge, nextSrcX, nextToDstEdge, nextDstX);

			// if path passes a (saddle) vertex, then continue
			if (ich.pathPassVert) continue;

			// organize nextToSrcEdge & nextSrcX; and if they don't exist, it means src&dst are the same
			// if nextToDstEdge & nextDstX does not exist either (this should not happen when constructing svg (vert-vert-pair))
			if (nextToSrcEdge == -1)
			{
				nextToSrcEdge = nextToDstEdge;
				nextSrcX = nextDstX;
			}
			d_cur_svg[d_svg_tails[i]].adjNode = j;
			d_cur_svg[d_svg_tails[i]].geodDist = ich.GetDistanceTo(j);
			d_cur_svg[d_svg_tails[i]].nextToSrcX = nextSrcX;
			d_cur_svg[d_svg_tails[i]].nextToSrcEdge = nextToSrcEdge;
			d_cur_svg[d_svg_tails[i]].nextToDstX = nextDstX;
			d_cur_svg[d_svg_tails[i]].nextToDstEdge = nextToDstEdge;

			++d_svg_tails[i];

			// in case of overflow
			if (d_svg_tails[i] == K) break;
		}

		// 		InitialValueGeodesic::GeodesicKeyPoint dstPoint;
		// 		if (nextToSrcEdge != -1 && !ich.pathPassVert)
		// 		{
		// 			initGeodesic.AssignLength(ich.GetDistanceTo(100));
		// 			initGeodesic.AssignStartPoint(i);
		// 			initGeodesic.AssignFirstKeyPoint(
		// 				mesh.edges[nextToSrcEdge].twinEdge,
		// 				mesh.edges[nextToSrcEdge].edgeLen - nextSrcX);
		// 			dstPoint = initGeodesic.BuildGeodesicPath();
		// 		}
		// 		else
		// 		{
		// 			dstPoint.isInterior = true;
		// 			dstPoint.faceIndex = -1;
		// 			dstPoint.facePos3D = Vector3D();
		// 		}
	}

}

SVG::SVG()
{
	d_svg = NULL; svg = NULL;
	d_svg_tails = NULL; svg_tails = NULL;
}

SVG::~SVG()
{

}

void SVG::AssignMesh(Mesh *mesh_, Mesh *d_mesh_)
{
	mesh = mesh_; d_mesh = d_mesh_;
}

void SVG::SetParameters(int K_, unsigned splitInfoCoef_, unsigned vertInfoCoef_)
{
	K = K_; splitInfoCoef = splitInfoCoef_; vertInfoCoef = vertInfoCoef_;
}

bool SVG::Allocation()
{
	// allocation memories for PriorityQueues

	int totalThreadNum = SVG_THREAD_NUM * SVG_BLOCK_NUM;

	HANDLE_ERROR(hipMalloc((void**)&d_winPQs, totalThreadNum * WINPQ_SIZE * sizeof(PQWinItem)));
	HANDLE_ERROR(hipMalloc((void**)&d_pseudoWinPQs, totalThreadNum * PSEUDOWINPQ_SIZE * sizeof(PQPseudoWinItem)));

	// allocation info buffers for ICH
	// for splitInfo and vertInfo, allocate one more buffer for each thread in the case of hash-table getting full
	HANDLE_ERROR(hipMalloc((void**)&d_splitInfoBuf, totalThreadNum * (splitInfoCoef * K + 1) * sizeof(ICHDevice::SplitItem)));
	HANDLE_ERROR(hipMalloc((void**)&d_vertInfoBuf, totalThreadNum * (vertInfoCoef * K + 1) * sizeof(ICHDevice::VertItem)));
	HANDLE_ERROR(hipMalloc((void**)&d_storedWindowsBuf, totalThreadNum * STORED_WIN_BUF_SIZE * sizeof(ICHDevice::Window)));
	HANDLE_ERROR(hipMalloc((void**)&d_keptFacesBuf, totalThreadNum * KEPT_FACE_SIZE * sizeof(unsigned)));

	// allocation for SVG structure
	HANDLE_ERROR(hipMalloc((void**)&d_svg, mesh->vertNum * K * sizeof(SVGNode)));
	HANDLE_ERROR(hipMalloc((void**)&d_svg_tails, mesh->vertNum * sizeof(int)));

	return true;
}

void SVG::Free()
{
	HANDLE_ERROR(hipFree(d_winPQs));
	HANDLE_ERROR(hipFree(d_pseudoWinPQs));
	HANDLE_ERROR(hipFree(d_splitInfoBuf));
	HANDLE_ERROR(hipFree(d_vertInfoBuf));
	HANDLE_ERROR(hipFree(d_storedWindowsBuf));
	HANDLE_ERROR(hipFree(d_keptFacesBuf));
}

void SVG::FreeSVGStructure()
{
	HANDLE_ERROR(hipFree(d_svg));
	HANDLE_ERROR(hipFree(d_svg_tails));
	if (svg) delete[] svg;
	if (svg_tails) delete[] svg_tails;
}

void SVG::ConstructSVG()
{
	clock_t start = clock();
	constructSVG << <SVG_BLOCK_NUM, SVG_THREAD_NUM >> >(*d_mesh, K,
		d_winPQs, d_pseudoWinPQs,
		d_splitInfoBuf, splitInfoCoef,
		d_vertInfoBuf, vertInfoCoef,
		d_storedWindowsBuf, d_keptFacesBuf,
		d_svg, d_svg_tails);
	// TODO: organize the constructed SVG
	HANDLE_ERROR(hipGetLastError());
	HANDLE_ERROR(hipDeviceSynchronize());
	clock_t end = clock();
	cout << "Time consumed: " << (double)(end - start) / (double)CLOCKS_PER_SEC << endl;

	int *svg_tails = new int[mesh->vertNum];
	HANDLE_ERROR(hipMemcpy(svg_tails, d_svg_tails, mesh->vertNum * sizeof(int), hipMemcpyDeviceToHost));
	double degree = 0;
	for (int i = 0; i < mesh->vertNum; ++i)
		degree += svg_tails[i];
	degree /= mesh->vertNum;
	cout << "Average degree of node in SVG: " << degree << endl;
}

void SVG::CopySVGToHost()
{
	if (svg) delete[] svg;
	if (svg_tails) delete[] svg_tails;
	svg = new SVGNode[mesh->vertNum * K];
	svg_tails = new int[mesh->vertNum];

	HANDLE_ERROR(hipMemcpy(svg, d_svg, mesh->vertNum * K * sizeof(SVGNode), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(svg_tails, d_svg_tails, mesh->vertNum * sizeof(int), hipMemcpyDeviceToHost));
	// 	cout << mesh->vertNum << endl;
	// 	for (int j = 0; j < mesh->vertNum; ++j)
	// 	{
	// 		cout << "Node " << j << " degree: " << svg_tails[j] << endl;
	// 		SVGNode *curSVGList = svg + K * j;
	// 		for (int i = 0; i < svg_tails[j]; ++i)
	// 		{
	// 			printf("%d %f %d %f %d %f\n", curSVGList[i].adjNode, curSVGList[i].geodDist, 
	// 				curSVGList[i].nextToSrcEdge, curSVGList[i].nextToSrcX, 
	// 				curSVGList[i].nextToDstEdge, curSVGList[i].nextToDstX);
	// 			system("pause");
	// 		}
	// 	}
}

void SVG::SaveSVGToFile(const char *fileName)
{
	ofstream output(fileName, ios::binary);
	for (int i = 0; i < mesh->vertNum; ++i)
	{
		output.write((char*)(svg_tails + i), sizeof(int));
		output.write((char*)(svg + i * K), sizeof(SVGNode) * svg_tails[i]);
	}
	output.close();
}

void SVG::LoadSVGFromFile(const char *fileName)
{
	if (svg) delete[] svg;
	if (svg_tails) delete[] svg_tails;
	svg = new SVGNode[mesh->vertNum * K];
	svg_tails = new int[mesh->vertNum];

	ifstream input(fileName, ios::binary);
	for (int i = 0; i < mesh->vertNum; ++i)
	{
		input.read((char*)(svg_tails + i), sizeof(int));
		input.read((char*)(svg + i * K), sizeof(SVGNode) * svg_tails[i]);
	}
	input.close();

	double degree = 0;
	for (int i = 0; i < mesh->vertNum; ++i)
		degree += svg_tails[i];
	degree /= mesh->vertNum;
	cout << "Average degree of node in SVG: " << degree << endl;
}

__host__ __device__ void SVG::SolveSSSD(int s, int t, Mesh mesh, GraphDistInfo * graphDistInfos, PriorityQueuesWithHandle<int> pq)
{
	// TODO: use Astar algorithm to search dist&path to t; use Euclidean dist as heuristic prediction
	// TODO: initialize graphDistInfos & pq
	searchType = ASTAR;
	graphDistInfos[s].dist = 0.0;
	graphDistInfos[s].pathParentIndex = -1;
	graphDistInfos[s].srcId = s;
	pq.push(s, &graphDistInfos[s].indexInPQ, 0.0 + (mesh.verts[s].pos - mesh.verts[t].pos).length());
	Astar(&mesh, t, graphDistInfos, pq);
}

__host__ __device__ void SVG::SolveSSSD(int f0, Vector3D p0, int f1, Vector3D p1, Mesh mesh,
	ICHDevice::SplitItem *d_splitInfos, unsigned splitInfoSize,
	ICHDevice::VertItem *d_vertInfos, unsigned vertInfoSize,
	PQWinItem *winPQBuf, PQPseudoWinItem *pseudoWinPQBuf,
	ICHDevice::Window *storedWindows, unsigned int *keptFaces,
	GraphDistInfo * graphDistInfos, PriorityQueuesWithHandle<int> pq,
	SVGNode *res, int *saddleVertNearDst)
{
	// TODO: run ICH on the two arbitrary points separately, and run a special single-source-single-destination algorithm
	searchType = DIJKSTRA;
	ICHDevice ich;

	PriorityQueues<ICHDevice::Window> local_winPQ(WINPQ_SIZE - 1);
	PriorityQueues<ICHDevice::PseudoWindow> local_pseudoWinPQ(PSEUDOWINPQ_SIZE - 1);
	local_winPQ.AssignMemory(winPQBuf, WINPQ_SIZE - 1);
	local_pseudoWinPQ.AssignMemory(pseudoWinPQBuf, PSEUDOWINPQ_SIZE - 1);

	ich.AssignMesh(&mesh);
	ich.AssignBuffers(d_splitInfos, splitInfoSize,
		d_vertInfos, vertInfoSize,
		local_winPQ, local_pseudoWinPQ, storedWindows, keptFaces);

	ich.Clear();
	ich.AddSource(f0, p0);
	ich.AddFacesKeptWindow(f1);
	ich.Execute(K);

	unsigned int srcId;
	res->geodDist = ich.BuildGeodesicPathTo(f1, p1, srcId, res->nextToSrcEdge, res->nextToSrcX, res->nextToDstEdge, res->nextToDstX);
	if (res->nextToSrcEdge != -1) return;

	for (int i = 0; i < vertInfoSize; ++i)
	{
		unsigned idx = d_vertInfos[i].index;
		if (d_vertInfos[i].item.dist == DBL_MAX) continue;
		graphDistInfos[idx].dist = d_vertInfos[i].item.dist;
		graphDistInfos[idx].pathParentIndex = -1;
		graphDistInfos[idx].srcId = idx;
		pq.push(idx, &graphDistInfos[idx].indexInPQ, d_vertInfos[i].item.dist);
	}
	Astar(&mesh, -1, graphDistInfos, pq);
	// TODO: how to virtually link the dst-surface-point into the SVG?
	ich.Clear();
	ich.AddSource(f1, p1);
	ich.Execute(K);

	// find the minimal dist and corresponding last-passed vertex
	double minDist = DBL_MAX;
	unsigned int lastVertId = -1;
	for (int i = 0; i < vertInfoSize; ++i)
	{
		unsigned idx = d_vertInfos[i].index;
		if (d_vertInfos[i].item.dist == DBL_MAX) continue;
		if (graphDistInfos[idx].dist + d_vertInfos[i].item.dist >= minDist) continue;
		minDist = graphDistInfos[idx].dist + d_vertInfos[i].item.dist;
		lastVertId = idx;
	}
	res->geodDist = minDist;
	*saddleVertNearDst = lastVertId;

	unsigned int nextToSrcEdge, nextToDstEdge;
	double nextToSrcX, nextToDstX;
	// set the nextToDstEdge and nextToDstX value (convert to the twin edge)
	ich.BuildGeodesicPathTo(lastVertId, srcId, nextToDstEdge, nextToDstX, res->nextToSrcEdge, res->nextToSrcX);
	if (nextToDstEdge == -1)
	{
		nextToDstEdge = res->nextToSrcEdge;
		nextToDstX = res->nextToDstX;
	}
	nextToDstEdge = mesh.edges[nextToDstEdge].twinEdge;
	nextToDstX = mesh.edges[nextToDstEdge].edgeLen - nextToDstX;

	// find the first-passed vertex
	int parent = graphDistInfos[lastVertId].pathParentIndex;
	while (parent != -1)
	{
		lastVertId = parent;
		parent = graphDistInfos[lastVertId].pathParentIndex;
	}

	ich.Clear();
	ich.AddSource(f0, p0);
	ich.Execute(K);

	// this IF seems to be not necessary ...
	if (d_vertInfos[ich.vertInfos.getKey(lastVertId)].item.dist != DBL_MAX)
	{
		ich.BuildGeodesicPathTo(lastVertId, srcId, nextToSrcEdge, nextToSrcX, res->nextToDstEdge, res->nextToDstX);
		if (nextToSrcEdge == -1)
		{
			nextToSrcEdge = res->nextToDstEdge;
			nextToSrcX = res->nextToDstX;
		}

		res->nextToSrcEdge = nextToSrcEdge;
		res->nextToSrcX = nextToSrcX;
		res->nextToDstEdge = nextToDstEdge;
		res->nextToDstX = nextToDstX;
	}
}

__host__ __device__ void SVG::SolveMSAD(int *sources, double *sourceWeights, int Ns, Mesh mesh, GraphDistInfo * graphDistInfos, PriorityQueuesWithHandle<int> pq)
{
	// TODO: use Dijkstra algorithm to search min-dist&path to destinations
	// TODO: initialize graphDistInfos & pq
	searchType = DIJKSTRA;
	for (int i = 0; i < Ns; ++i)
	{
		graphDistInfos[sources[i]].dist = sourceWeights[i];
		graphDistInfos[sources[i]].pathParentIndex = -1;
		graphDistInfos[sources[i]].srcId = sources[i];
		pq.push(sources[i], &graphDistInfos[sources[i]].indexInPQ, sourceWeights[i]);
	}
	Astar(&mesh, -1, graphDistInfos, pq);
}

__host__ __device__ void SVG::Astar(Mesh *mesh, int t, GraphDistInfo * graphDistInfos, PriorityQueuesWithHandle<int> pq)
{
	SVGNode *local_svg = NULL;
	int *local_svg_tails = NULL;

#ifdef __CUDA_ARCH__
	local_svg = d_svg; local_svg_tails = d_svg_tails;
#else
	local_svg = svg; local_svg_tails = svg_tails;
#endif

	while (!pq.empty())
	{
		int curNodeIndex = pq.pop();
		if (searchType == ASTAR && curNodeIndex == t)
		{
			pq.clear();
			break;
		}
		SVGNode *curNodeList = local_svg + K * curNodeIndex;
		for (int i = 0; i < local_svg_tails[curNodeIndex]; ++i)
		{
			int adjNodeIndex = curNodeList[i].adjNode;
			double newDist = graphDistInfos[curNodeIndex].dist + curNodeList[i].geodDist;
			if (newDist >= graphDistInfos[adjNodeIndex].dist) continue;

			graphDistInfos[adjNodeIndex].dist = newDist;
			graphDistInfos[adjNodeIndex].pathParentIndex = curNodeIndex;
			graphDistInfos[adjNodeIndex].srcId = graphDistInfos[curNodeIndex].srcId;

			double priority = 0.0;
			switch (searchType)
			{
			case SVG::ASTAR: priority = newDist + (mesh->verts[adjNodeIndex].pos - mesh->verts[t].pos).length(); break;
			case SVG::DIJKSTRA: priority = newDist; break;
			default: priority = newDist; break;
			}

			if (graphDistInfos[adjNodeIndex].indexInPQ == -1)
				pq.push(adjNodeIndex, &graphDistInfos[adjNodeIndex].indexInPQ, priority);
			else
				pq.decrease(graphDistInfos[adjNodeIndex].indexInPQ, priority);
		}
	}
}
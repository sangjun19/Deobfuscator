#include "hip/hip_runtime.h"
// Repository: kuronekodaisuki/StitchCam
// File: Blender.cu

//#include <>
#include <opencv2/gpu/device/saturate_cast.hpp>

#include "MyBlender.h"

using namespace std;

namespace cv {
namespace gpu {
namespace device {

template<typename T>
__global__ void kernelFeed(int height, int width, T *dst, const T *src, const uchar *mask, int dStep, int sStep, int mStep)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
	//int offset = x + y * mStep; // offset for mask
	if (x < width && y < height && mask[x + y * mStep])
	{
		int dOffset = x * 3 + y * dStep; // offset for dst
		int offset = x * 3 + y * sStep; // offset for src
		dst[dOffset] = src[offset];
		dst[dOffset + 1] = src[offset + 1];
		dst[dOffset + 2] = src[offset + 2];
	}
}

void cudaFeed(const gpu::GpuMat &image, const gpu::GpuMat &mask, gpu::GpuMat &dst, int dx, int dy)
{
	dim3 threads(16, 16);	// 256 threads yealds better performance
	dim3 blocks(image.cols / threads.x, image.rows / threads.y);

	switch (image.type())
	{
	case CV_8UC3:
		kernelFeed<<<blocks, threads>>>(image.rows, image.cols, 
			dst.ptr<uchar>(dy) + dx * 3, image.ptr<uchar>(), mask.ptr<uchar>(),
			dst.step, image.step, mask.step);
		hipDeviceSynchronize();
		break;
	case CV_16SC3:
		kernelFeed<<<blocks, threads>>>(image.rows, image.cols, 
			dst.ptr<short>(dy) + dx * 3, image.ptr<short>(), mask.ptr<uchar>(),
			dst.step, image.step, mask.step);
		hipDeviceSynchronize();
		break;
	}
}

void cudaFeed(const Mat &image, const Mat &mask, gpu::GpuMat &dst, int dx, int dy)
{
	gpu::GpuMat gpuImg;
	gpu::GpuMat gpuMask;
	gpuImg.upload(image);
	gpuMask.upload(mask);
	cudaFeed(gpuImg, gpuMask, dst, dx, dy);
}

void cudaFeed(const Mat &image, const Mat &mask, Mat &dst, int dx, int dy)
{
	dim3 threads(8, 8);	// 64 threads yealds better performance
	dim3 blocks(image.cols / threads.x, image.rows / threads.y);

	switch (image.type())
	{
	case CV_8UC3:
		kernelFeed<<<blocks, threads>>>(image.rows, image.cols, 
			dst.ptr<uchar>(dy) + dx * 3, image.ptr<uchar>(), mask.ptr<uchar>(),
			dst.step, image.step, mask.step);
		hipDeviceSynchronize();
		break;
	case CV_16SC3:
		kernelFeed<<<blocks, threads>>>(image.rows, image.cols, 
			dst.ptr<short>(dy) + dx * 3, image.ptr<short>(), mask.ptr<uchar>(),
			dst.step, image.step, mask.step);
		hipDeviceSynchronize();
		break;
	}
}

}	// namespace device
}	// namespace gpu
}	// namespace cv;

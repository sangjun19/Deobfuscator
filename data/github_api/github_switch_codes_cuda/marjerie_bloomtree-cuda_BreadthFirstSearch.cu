#include "hip/hip_runtime.h"
// Repository: marjerie/bloomtree-cuda
// File: BreadthFirstSearch.cu

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdint.h>
#include <inttypes.h>

#define FORCE_INLINE

__device__ int Parent(int node);
__device__ int LeftChild(int node);
__device__ int RightChild(int node);
__device__ int Sibling(int node);
__device__ int calculate_lca(int u, int v);
__device__ void traversal(long int prev, long int lca, long int src, long int dest, int n, uint64_t *hash, bool *bit, int h, int m);
__device__ bool check_traversal_up(int prev, int lca, int src, int dest, int n, bool *mask);
__device__ bool check_traversal_down(int prev, int lca, int src, int dest, int n, bool *mask);
__device__ bool CheckBloom(long tid, uint64_t *hash_value, bool *bit, int h, int m);
__device__ void SetBloom(long tid, uint64_t *hash_value, bool *bit, int h, int m);

__device__ static inline FORCE_INLINE uint64_t rotl64 ( uint64_t x, int8_t r )
{
  return (x << r) | (x >> (64 - r));
}

#define ROTL64(x,y)	rotl64(x,y)
#define BIG_CONSTANT(x) (x##LLU)

#define getblock(p, i) (p[i])

__device__ static inline FORCE_INLINE uint64_t fmix64 ( uint64_t k )
{
  k ^= k >> 33;
  k *= BIG_CONSTANT(0xff51afd7ed558ccd);
  k ^= k >> 33;
  k *= BIG_CONSTANT(0xc4ceb9fe1a85ec53);
  k ^= k >> 33;

  return k;
}

__device__ void MurmurHash3_x64_128 ( int tid, char key[], const int len,
                           const uint32_t seed, void * out1 , void * out2)
{


  uint8_t data[16];
  for (int i=0; i<len; i++)
    data[i] = (uint8_t) key[i];
  const int nblocks = len / 16;
  int i;

  uint64_t h1 = seed;
  uint64_t h2 = seed;

  uint64_t c1 = BIG_CONSTANT(0x87c37b91114253d5);
  uint64_t c2 = BIG_CONSTANT(0x4cf5ad432745937f);

  //----------
  // body

  uint64_t blocks[16];
  for (int i=0; i<len; i++)
    blocks[i] = (uint64_t) data[i];

  for(i = 0; i < nblocks; i++)
  {
    uint64_t k1 = blocks[i*2+0];
    uint64_t k2 = blocks[i*2+1];

    k1 *= c1; k1  = ROTL64(k1,31); k1 *= c2; h1 ^= k1;

    h1 = ROTL64(h1,27); h1 += h2; h1 = h1*5+0x52dce729;

    k2 *= c2; k2  = ROTL64(k2,33); k2 *= c1; h2 ^= k2;

    h2 = ROTL64(h2,31); h2 += h1; h2 = h2*5+0x38495ab5;
  }

  //----------
  // tail

  uint8_t tail[16];
  for (int i=0; i<len-nblocks*16; i++)
    tail[i] = (uint64_t) data[i+nblocks*16];

  uint64_t k1 = 0;
  uint64_t k2 = 0;

  switch(len & 15)
  {
  case 15: k2 ^= (uint64_t)(tail[14]) << 48;
  case 14: k2 ^= (uint64_t)(tail[13]) << 40;
  case 13: k2 ^= (uint64_t)(tail[12]) << 32;
  case 12: k2 ^= (uint64_t)(tail[11]) << 24;
  case 11: k2 ^= (uint64_t)(tail[10]) << 16;
  case 10: k2 ^= (uint64_t)(tail[ 9]) << 8;
  case  9: k2 ^= (uint64_t)(tail[ 8]) << 0;
           k2 *= c2; k2  = ROTL64(k2,33); k2 *= c1; h2 ^= k2;

  case  8: k1 ^= (uint64_t)(tail[ 7]) << 56;
  case  7: k1 ^= (uint64_t)(tail[ 6]) << 48;
  case  6: k1 ^= (uint64_t)(tail[ 5]) << 40;
  case  5: k1 ^= (uint64_t)(tail[ 4]) << 32;
  case  4: k1 ^= (uint64_t)(tail[ 3]) << 24;
  case  3: k1 ^= (uint64_t)(tail[ 2]) << 16;
  case  2: k1 ^= (uint64_t)(tail[ 1]) << 8;
  case  1: k1 ^= (uint64_t)(tail[ 0]) << 0;
           k1 *= c1; k1  = ROTL64(k1,31); k1 *= c2; h1 ^= k1;
  };

  //----------
  // finalization

  h1 ^= len; h2 ^= len;

  h1 += h2;
  h2 += h1;

  h1 = fmix64(h1);
  h2 = fmix64(h2);

  h1 += h2;
  h2 += h1;

  ((uint64_t*)out1)[0] = h1;
  ((uint64_t*)out2)[0] = h2;

}

__device__ inline uint64_t NthHash(uint8_t n, uint64_t hashA, uint64_t hashB, uint64_t filter_size) {
	return ((hashA + n * hashB) % filter_size);
}


__global__ void insert_edge(int *u, int *v, bool *bit, int n, int e, long int ful_vertices,long int valsperloop,int ii,int h, int m, uint64_t *hash)
{
	int blockNum = blockIdx.z*(gridDim.x*gridDim.y)+blockIdx.y*gridDim.x+blockIdx.x;
	int threadNum = threadIdx.z*(blockDim.x*blockDim.y)+threadIdx.y*blockDim.x+threadIdx.x;
	int tid = blockNum*(blockDim.x*blockDim.y*blockDim.z)+threadNum;

	long int tid1 = ii*valsperloop + tid;

	if (tid1 < e){
		u[tid] = u[tid] + n - 1;
		v[tid] = v[tid] + n - 1;
		int src = u[tid];
		int dest = v[tid];

		if (!((u[tid] < ful_vertices && v[tid] < ful_vertices) || (u[tid] >= ful_vertices && v[tid] >= ful_vertices))) {
			if (u[tid] > v[tid]){
	 			int cur = Parent(u[tid]);
				SetBloom((cur*n+u[tid]-n+1) << 1,hash+2*tid*sizeof(uint64_t),bit,h,m);
				if (u[tid] == LeftChild(cur)){
					SetBloom((cur*n+v[tid]-n+1) << 1,hash+2*tid*sizeof(uint64_t),bit,h,m); 
				}
				else{
					SetBloom(((cur*n+v[tid]-n+1) << 1) + 1,hash+2*tid*sizeof(uint64_t),bit,h,m); 
				}
				u[tid] = cur;
			}	
		 	else{
	 			int cur = Parent(v[tid]);
				SetBloom((cur*n+v[tid]-n+1) << 1,hash+2*tid*sizeof(uint64_t),bit,h,m);
				if (v[tid] == LeftChild(cur)){
					SetBloom((cur*n+u[tid]-n+1) << 1,hash+2*tid*sizeof(uint64_t),bit,h,m); 
				}
				else{
					SetBloom(((cur*n+u[tid]-n+1) << 1) + 1,hash+2*tid*sizeof(uint64_t),bit,h,m); 
				}
				v[tid] = cur;
			}
		}
		
		__syncthreads();

		int lca = calculate_lca(u[tid], v[tid]);

		traversal(u[tid], lca, src, dest, n, hash+2*tid*sizeof(uint64_t),bit,h,m);
		traversal(v[tid], lca, dest, src, n, hash+2*tid*sizeof(uint64_t),bit,h,m);
	}
}

__global__ void SetBloom(bool *mask, uint64_t *hash_value, bool *bit, int h, int m, long int valsperloop, int ii)
{
	int blockNum = blockIdx.z*(gridDim.x*gridDim.y)+blockIdx.y*gridDim.x+blockIdx.x;
	int threadNum = threadIdx.z*(blockDim.x*blockDim.y)+threadIdx.y*blockDim.x+threadIdx.x;
	int tid = blockNum*(blockDim.x*blockDim.y*blockDim.z)+threadNum;

	if (mask[tid] == 1){
		long tid1 = valsperloop*ii + tid;
		int i =0;
		int count=0; 	
		long num = tid1;
		char str[10];
		do{	
			count++;
			num /= 10;
		}while(num != 0); 
		num = tid1;
		do{	
			str[count-i-1] = num%10 + '0';
			num/=10;
			i++;
		}while(num !=0);	
		str[i] = 48+'\0';
		uint64_t len1 = (uint64_t) count;
		size_t len = (size_t) len1;
		MurmurHash3_x64_128(tid1, str, len, 0, (hash_value)+tid*2*sizeof(uint64_t), (hash_value)+(tid*2+1)*sizeof(uint64_t));	
	
		for (int i=0; i<h; i++){
			bit[NthHash(i,*((hash_value)+tid*2*sizeof(uint64_t)),*((hash_value)+(tid*2+1)*sizeof(uint64_t)),m)] = 1;
		}
	}
}

__global__ void set_mask(bool *mask, uint64_t *hash_value, bool *bit, int h, int m, int n, long int valsperloop, int ii)
{
	int blockNum = blockIdx.z*(gridDim.x*gridDim.y)+blockIdx.y*gridDim.x+blockIdx.x;
	int threadNum = threadIdx.z*(blockDim.x*blockDim.y)+threadIdx.y*blockDim.x+threadIdx.x;
	int tid = blockNum*(blockDim.x*blockDim.y*blockDim.z)+threadNum;

	if ((valsperloop*ii + tid) < n){	
		long tid1 = valsperloop*ii + tid;
		mask[tid] = CheckBloom(tid1, (hash_value)+2*tid*sizeof(uint64_t), bit, h, m);
	}
}

__global__ void check_bloom(int *found, bool *bit, bool *mask, uint64_t *hash_value, int m, int h, int n)
{
	int val = blockIdx.x;
	int hash = threadIdx.x;
	uint64_t filter_size = (uint64_t) m;
	uint8_t hash_no = (uint8_t) hash;
	
	if (val < n && *found == 0){
		if (mask[val] == 1){
			 if (hash < h){
				if (bit[NthHash(hash_no,*(hash_value+2*val*sizeof(uint64_t)),*(hash_value+(2*val+1)*sizeof(uint64_t)), filter_size)] == 0){ 		
					atomicAdd(found, 1);
				}
			}
		}
	}
}


__global__ void get_neighbours(int u, bool *neighs, bool *mask, int n, long int ful_vertices)
{
	int blockNum = blockIdx.z*(gridDim.x*gridDim.y)+blockIdx.y*gridDim.x+blockIdx.x;
	int threadNum = threadIdx.z*(blockDim.x*blockDim.y)+threadIdx.y*blockDim.x+threadIdx.x;
	int tid = blockNum*(blockDim.x*blockDim.y*blockDim.z)+threadNum;

	long v=tid;

	*(neighs+(tid)*sizeof(bool)) = 0;

	if (tid < n && !(u==tid) ){

		u = u + n - 1;
		v = v + n - 1;
		long src = u;
		long dest = v;

		if (!((u < ful_vertices && v < ful_vertices) || (u >= ful_vertices && v >= ful_vertices))) {
			if (u > v){
	 			long cur = Parent(u);
				if ((mask[(cur*n+u-n+1) << 1] == 0)) return;
				u = cur;
			}	
		 	else{
	 			long cur = Parent(v);
				if (v == LeftChild(cur))
				{
					if ((mask[(cur*n+u-n+1) << 1] == 0)) 
						return;
				}
				else
				{
					if ((mask[((cur*n+u-n+1) << 1) + 1] == 0)) 
						return;
				} 
				v = cur;
			}
		}
	
		//__syncthreads();

		int lca = calculate_lca(u, v);
		if (check_traversal_up(u, lca, src, dest, n, mask)) 
		{
			if(check_traversal_down(v, lca, dest, src, n, mask))
			{
				//if ((src-n+1) == 6) printf("%d ",dest-n+1);
				*(neighs+(dest-n+1)*sizeof(bool)) = 1;
			}
		}
	}
}


__device__ int Parent(int node)
{
	return (((node + 1) >> 1) - 1); 
}

__device__ int LeftChild(int node)
{
	return (((node + 1) << 1) - 1); 
}

__device__ int RightChild(int node)
{
	return ((node + 1) << 1); 
}

__device__ int Sibling(int node)
{
	return (((node + 1) ^ 1) - 1); 
}

__device__ int calculate_lca(int u, int v)
{	
	int val1 = 0;
	int val2 = 0;	
	int i = 1;

	do{
		float pow_val = 1 << i;
		val1 = floor((u+1)/pow_val);
		val2 = floor((v+1)/pow_val);
		i++;
	} while(val1 != val2);
	return (val1 - 1);
}

__device__ void traversal(long int prev, long int lca, long int src, long int dest, int n, uint64_t *hash, bool *bit, int h, int m)
{
	int cur = Parent(prev);
	while (cur != lca){
		SetBloom((cur*n+src-n+1) << 1,hash,bit,h,m);
		if (prev == LeftChild(cur)){
			SetBloom((cur*n+dest-n+1) << 1,hash,bit,h,m); 
		}
		else{
			SetBloom(((cur*n+dest-n+1) << 1) + 1,hash,bit,h,m); 
		} 
		prev = cur;
		cur = Parent(cur);
	}
	SetBloom(((cur*n+src-n+1) << 1) + 1,hash,bit,h,m);	
}

__device__ bool check_traversal_up(int prev, int lca, int src, int dest, int n, bool *mask)  
{
	int cur = Parent(prev);
	while (cur != lca){
		if ((mask[(cur*n+src-n+1) << 1]) == 0)  return false;
		prev = cur;
		cur = Parent(cur);
	}
	if ((mask[((cur*n+src-n+1) << 1) + 1]) == 0)  return false;
	return true;	
}

__device__ bool check_traversal_down(int prev, int lca, int src, int dest, int n, bool *mask) 
{
	int cur = Parent(prev);
	while (cur != lca){
		if (prev == LeftChild(cur))
		{
			if ((mask[(cur*n+dest-n+1) << 1]) == 0)  
				return false;
		}
		else
		{
			if ((mask[((cur*n+dest-n+1) << 1) + 1]) == 0) 
				return false;
		}
		prev = cur;
		cur = Parent(cur);
	}
	return true;	
}

__device__ void SetBloom(long tid, uint64_t *hash_value, bool *bit, int h, int m)
{
	int i =0;
	int count=0; 	
	long num = tid;
	char str[11];
	do{	
		count++;
		num /= 10;
	}while(num != 0); 
	num = tid;
	do{	
		str[count-i-1] = num%10 + '0';
		num/=10;
		i++;
	}while(num !=0);	
	str[i] = 48+'\0';
	uint64_t len1 = (uint64_t) count;
	size_t len = (size_t) len1;
	MurmurHash3_x64_128(tid, str, len, 0, (hash_value), (hash_value)+sizeof(uint64_t));

	for (int i=0; i<h; i++){
		bit[NthHash(i,*(hash_value),*((hash_value)+sizeof(uint64_t)),m)] = 1;
	}
}

__device__ bool CheckBloom(long tid, uint64_t *hash_value, bool *bit, int h, int m)
{
	int i =0;
	int count=0; 	
	long num = tid;
	char str[10];
	do{	
		count++;
		num /= 10;
	}while(num != 0); 
	num = tid;
	do{	
		str[count-i-1] = num%10 + '0';
		num/=10;
		i++;
	}while(num !=0);	
	str[i] = 48+'\0';
	uint64_t len1 = (uint64_t) count;
	size_t len = (size_t) len1;
	MurmurHash3_x64_128(tid, str, len, 0, (hash_value), (hash_value)+sizeof(uint64_t));

	for (int i=0; i<h; i++){
		if (bit[NthHash(i,*(hash_value),*((hash_value)+sizeof(uint64_t)),m)] == 0){
			return false;
		}
	}
	return true;
}

void InsertEdge(int num_vertices, int num_edges, int num_hashes, int num_bits, int *h_u, int *h_v, bool *h_bits)
{

	hipError_t err = hipSuccess;
	hipEvent_t start1, stop1;
	hipEventCreate(&start1);
	hipEventCreate(&stop1);

	size_t size = num_edges * sizeof(int);

	int num_ful_levels = floor( log2((double) (2*num_vertices - 1)));
	long int ful_vertices = pow((int) 2,(int) num_ful_levels) - 1;

	size_t size_bits = num_bits * sizeof(bool);
	bool *d_bits = NULL;
    err=hipMalloc((void **)&d_bits, size_bits);
	hipDeviceSynchronize();
	if (err != hipSuccess)
	{
	    fprintf(stderr, "Failed to allocate vector bits (error code %s)!\n", hipGetErrorString(err));
	    exit(EXIT_FAILURE);
	}

	hipMemset(d_bits, 0, size_bits);

	int *d_u = NULL, *d_v = NULL; 
	err=hipMalloc((void **)&d_u, size);
	hipDeviceSynchronize();
	if (err != hipSuccess)
	{
	    fprintf(stderr, "Failed to allocate vector u (error code %s)!\n", hipGetErrorString(err));
	    exit(EXIT_FAILURE);
	}
	err=hipMalloc((void **)&d_v, size);
	hipDeviceSynchronize();
	if (err != hipSuccess)
	{
	    fprintf(stderr, "Failed to allocate vector v (error code %s)!\n", hipGetErrorString(err));
	    exit(EXIT_FAILURE);
	}
	err=hipMemcpy(d_u, h_u, size, hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	if (err != hipSuccess)
	{
	    fprintf(stderr, "Failed to copy u (error code %s)!\n", hipGetErrorString(err));
	    exit(EXIT_FAILURE);
	}
	err=hipMemcpy(d_v, h_v, size, hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	if (err != hipSuccess)
	{
	    fprintf(stderr, "Failed to copy v (error code %s)!\n", hipGetErrorString(err));
	    exit(EXIT_FAILURE);
	}

	clock_t ti;

	hipEventRecord(start1);
	ti=clock();

	dim3 tpb1(32,32,1);
    	dim3 bpg1(8,2,1);

	long int valsperloop = 1 << 14;

	int N = ceil(num_edges/valsperloop)+1;

	uint64_t *d_hash_value = NULL;
	size_t size_hash = 2*num_edges*sizeof(uint64_t);
	hipMalloc((void **)&d_hash_value, size_hash);

	for (int i=0; i<N; i++){
		insert_edge<<<bpg1,tpb1>>>(d_u+i*valsperloop, d_v+i*valsperloop, d_bits, num_vertices, num_edges, ful_vertices, valsperloop, i, num_hashes, num_bits, d_hash_value);

		/*hipDeviceSynchronize();
	
		err = hipGetLastError();

		if (err != hipSuccess)
		{
		    fprintf(stderr, "Failed to launch insert_edge kernel %d (error code %s)!\n",i, hipGetErrorString(err));
		    exit(EXIT_FAILURE);
		}*/
	}

	hipMemcpy(h_bits, d_bits, size_bits, hipMemcpyDeviceToHost);

	ti=clock() -ti;
	hipEventRecord(stop1);

	hipEventSynchronize(stop1);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start1, stop1);

	printf("%.5f\n", float(ti) / CLOCKS_PER_SEC);

	printf("time taken is %.5f\n", milliseconds);

	int value=0;

	for (long i =0; i<num_bits; i++){
		if (h_bits[i] == 1) value++;
	}

	//printf("value is %d\n",value);

	hipFree(d_u);
	hipFree(d_v);
	hipFree(d_hash_value);
	hipFree(d_bits);
}

#define MAX 4039

int q[4039];
int front = 0;
int rear = -1;
int itemCount = 0;

int peek() {
   return q[front];
}

bool isEmpty() {
   return itemCount == 0;
}

bool isFull() {
   return itemCount == MAX;
}

int size() {
   return itemCount;
}  

void push(int data) {

   if(!isFull()) {
	
      if(rear == MAX-1) {
         rear = -1;            
      }       

      q[++rear] = data;
      itemCount++;
   }
}

int removeData() {
   int data = q[front++];
	
   if(front == MAX) {
      front = 0;
   }
	
   itemCount--;
   return data;  
}


int main ()
{

	hipError_t err = hipSuccess;
	const int INF = 1e8;
	int num_vertices, num_edges, num_hashes, num_bits;
	scanf("%d",&num_vertices);
	scanf("%d",&num_edges);
	scanf("%d",&num_bits);
	scanf("%d",&num_hashes);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	size_t size = num_edges * sizeof(int);
	int num_vals = 2*num_vertices*(num_vertices-1);

	int *h_u = (int *)malloc(size);
	int *h_v = (int *)malloc(size);
	
	for (int i =0; i<num_edges; i++)
	{
		scanf("%d",&h_u[i]);
		scanf("%d",&h_v[i]);
	}

	size_t size_bits = num_bits * sizeof(bool);
	bool *h_bits = (bool *)malloc(size_bits);

	InsertEdge(num_vertices, num_edges, num_hashes, num_bits, h_u, h_v, h_bits);

	uint64_t *d_hash_value = NULL;
	size_t size_hash = 2*num_vals*sizeof(uint64_t);
	hipMalloc((void **)&d_hash_value, size_hash);

	size_t size_mask = num_vals * sizeof(bool);
	bool *h_mask = (bool *)malloc(size_mask);
	bool *d_mask = NULL;
    	hipMalloc((void **)&d_mask, size_mask);

	bool *d_bits = NULL;
    	hipMalloc((void **)&d_bits, size_bits);
	hipMemcpy(d_bits, h_bits, size_bits, hipMemcpyHostToDevice);

	bool *d_neighs = NULL;
	size_t size_neighs = (num_vertices)*sizeof(bool);
	hipMalloc((void **)&d_neighs, size_neighs);
	bool *h_neighs = (bool *)malloc(size_neighs);
	
	int num_ful_levels = floor( log2((double) (2*num_vertices - 1)));
	long int ful_vertices = pow((int) 2,(int) num_ful_levels) - 1;

	dim3 tpb2(32,32,1);
    	dim3 bpg2(2,2,1);

	dim3 tpb3(32,32,1);
    	dim3 bpg3(4,4,1);
	long int valsperloop = 1 << 14;

	int N = ceil(num_vals/valsperloop)+1;

	clock_t ti;

	hipEventRecord(start);

	ti=clock();

	for (int i=0; i<N; i++){
		set_mask<<<bpg3,tpb3>>>(d_mask+i*valsperloop,d_hash_value+i*2*valsperloop,d_bits,num_hashes,num_bits,num_vals,valsperloop,i);

        /*hipDeviceSynchronize();
	
		err = hipGetLastError();

		if (err != hipSuccess)
		{
		    fprintf(stderr, "Failed to launch set_mask kernel %d (error code %s)!\n",i, hipGetErrorString(err));
		    exit(EXIT_FAILURE);
		}*/
	}

	int bfs_dist[4039];

	for (int i = 0; i < num_vertices; ++i) {
		bfs_dist[i] = INF;
	}

	push(0);
	bfs_dist[0] = 0;

	while(!isEmpty()){
		int u = removeData();
		get_neighbours<<<bpg2,tpb2>>>(u, d_neighs, d_mask, num_vertices, ful_vertices);
		hipMemcpy(h_neighs, d_neighs, size_neighs, hipMemcpyDeviceToHost);

		for (int i = 0; i < num_vertices; ++i) {
			if (h_neighs[i] == 1){
				if (bfs_dist[i] == INF){
					bfs_dist[i] = bfs_dist[u] + 1;
					push(i);
				}
			}
		}
	}

	ti=clock()-ti;

	hipEventRecord(stop);

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	printf("%.5f\n", float(ti) / CLOCKS_PER_SEC);

	printf("time taken is %.5f\n", milliseconds);

	hipFree(d_neighs);
	hipFree(d_hash_value);
	hipFree(d_bits);
	hipFree(d_mask);

	free(h_u);
	free(h_v);
	free(h_bits);
	
	hipDeviceReset();
	return 0;
}

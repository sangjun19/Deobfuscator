#include "hip/hip_runtime.h"
// Repository: Joey574/MachineLearningCpp
// File: SingleBlockCudaNetwork/CudaNet.cu

#include "SingleBlockCudaNetwork.h"

void CudaNetwork::initialize_train_data(float** d_x_train, float** d_y_train, float** d_x_test, float** d_y_test, matrix h_x_train, matrix h_y_train, matrix h_x_test, matrix h_y_test) {
	// initialize training data on the gpu
	hipMalloc(d_x_train, h_x_train.rows * h_x_train.cols * sizeof(float));
	hipMalloc(d_y_train, h_y_train.rows * h_y_train.cols * sizeof(float));

	hipMalloc(d_x_test, h_x_test.rows * h_x_test.cols * sizeof(float));
	hipMalloc(d_y_test, h_y_test.rows * h_y_test.cols * sizeof(float));


	hipMemcpy(*d_x_train, h_x_train.mat.data(), h_x_train.rows * h_x_train.cols * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(*d_y_train, h_y_train.mat.data(), h_y_train.rows * h_y_train.cols * sizeof(float), hipMemcpyHostToDevice);

	hipMemcpy(*d_x_test, h_x_test.mat.data(), h_x_test.rows * h_x_test.cols * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(*d_y_test, h_y_test.mat.data(), h_y_test.rows * h_y_test.cols * sizeof(float), hipMemcpyHostToDevice);
}
void CudaNetwork::initialize_batch_data(size_t batch_size) {
	m_batch_activation_size = 0;

	m_batch_data_size = m_network_size;

	for (size_t i = 1; i < m_dimensions.size(); i++) {
		m_batch_data_size += 3 * (m_dimensions[i] * batch_size);
		m_batch_activation_size += m_dimensions[i] * batch_size;
	}

	hipMalloc(&m_batch_data, m_batch_data_size * sizeof(float));

	m_activation = &m_batch_data[m_batch_activation_size];

	m_d_total = &m_activation[m_batch_activation_size];
	m_d_weights = &m_d_total[m_batch_activation_size];
	m_d_bias = &m_d_weights[m_weights_size];

}
void CudaNetwork::initialize_test_data(size_t test_size) {
	m_test_activation_size = 0;

	for (size_t i = 1; i < m_dimensions.size(); i++) {
		m_test_activation_size += m_dimensions[i] * test_size;
	}

	hipMalloc(&m_test_data, 2 * m_test_activation_size * sizeof(float));
	m_test_activation = &m_test_data[m_test_activation_size];
}

void CudaNetwork::define(std::vector<size_t> dimensions) {
	this->m_dimensions = dimensions;

	m_weights_size = 0;
	m_bias_size = 0;
	for (size_t i = 0; i < dimensions.size() - 1; i++) {
		m_weights_size += dimensions[i] * dimensions[i + 1];
		m_bias_size += dimensions[i + 1];
	}
	m_network_size = m_weights_size + m_bias_size;
}
void CudaNetwork::compile(CudaNetwork::weight_init init) {

	float* net = (float*)calloc(m_network_size, sizeof(float));

	std::random_device rd;
	std::default_random_engine gen(rd());

	size_t idx = 0;
	switch (init) {
	case weight_init::xavier: {
		for (size_t i = 0; i < m_dimensions.size() - 1; i++) {
			float lower_rand = -(1.0f / std::sqrt(m_dimensions[i + 1]));
			float upper_rand = 1.0f / std::sqrt(m_dimensions[i + 1]);

			std::uniform_real_distribution<float> dist_x(lower_rand, upper_rand);

			for (size_t j = 0; j < m_dimensions[i] * m_dimensions[i + 1]; j++, idx++) {
				net[idx] = dist_x(gen);
			}
		}
		break;
	}
	case weight_init::he: {
		float lower_rand = 0.0f;

		for (size_t i = 0; i < m_dimensions.size() - 1; i++) {
			float upper_rand = std::sqrt(2.0f / m_dimensions[i + 1]);

			std::normal_distribution<float> dist_h(lower_rand, upper_rand);

			for (size_t j = 0; j < m_dimensions[i] * m_dimensions[i + 1]; j++, idx++) {
				net[idx] = dist_h(gen);
			}
		}
		break;
	}
	case weight_init::normalize: {
		float lower_rand = -0.5f;
		float upper_rand = 0.5f;

		for (size_t i = 0; i < m_dimensions.size() - 1; i++) {
			std::uniform_real_distribution<float> dist_n(lower_rand, upper_rand);

			for (size_t j = 0; j < m_dimensions[i] * m_dimensions[i + 1]; j++, idx++) {
				net[idx] = dist_n(gen) * std::sqrt(1.0f / m_dimensions[i + 1]);
			}
		}
		break;
	}
	}

	hipMalloc(&m_network, m_network_size * sizeof(float));
	hipMemcpy(m_network, net, m_network_size * sizeof(float), hipMemcpyHostToDevice);

	m_bias = m_network + m_weights_size;

	free(net);


	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		std::cout << "Status: network failed to compile: " << hipGetErrorString(err) << "\n";
	} else {
		std::cout << "Status: network compiled\n";
	}
}
 
void CudaNetwork::fit(matrix x_train, matrix y_train, matrix x_test, matrix y_test, size_t batch_size, size_t epochs, float learning_rate, bool shuffle, int validation_freq) {
	std::cout << "Status: network_training\n";

	auto start_time = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double, std::milli> time;

	float *d_x_train, *d_y_train, *d_x_test, *d_y_test;
	const size_t iterations = x_train.rows / batch_size;

	initialize_train_data(&d_x_train, &d_y_train, &d_x_test, &d_y_test, x_train, y_train, x_test, y_test);
	initialize_batch_data(batch_size);
	initialize_test_data(x_test.rows);

	std::cout << "\nPointers:\nd_x_train:\t" << d_x_train << "\nd_y_train:\t" << d_y_train << "\nd_x_test:\t" << d_x_test << "\nd_y_test:\t" << d_y_test << "\n";
	std::cout << "m_batch_data:\t" << m_batch_data << "\nm_test_data:\t" << m_test_data << "\n";

	for (size_t e = 0; e < epochs; e++) {
		auto epoch_start_time = std::chrono::high_resolution_clock::now();

		for (size_t i = 0; i < iterations; i++) {

			float* d_x = d_x_train + (i * batch_size * x_train.cols);
			float* d_y = d_y_train + (i * batch_size * y_train.cols);

			forward_prop(d_x, m_batch_data, m_batch_activation_size, batch_size);
			back_prop(d_x, d_y, learning_rate, batch_size);
		}
		std::cout << verbose(d_x_test, d_y_test, x_test.rows, e, validation_freq, epoch_start_time);
	}
	time = std::chrono::high_resolution_clock::now() - start_time;


	hipFree(m_batch_data); m_batch_data = nullptr;
	hipFree(m_test_data); m_test_data = nullptr;

	hipFree(d_x_train);
	hipFree(d_y_train);
	hipFree(d_x_test);
	hipFree(d_y_test);

	std::cout << "Status: training_complete\n";
}

void CudaNetwork::forward_prop(float* x_data, float* result_data, size_t activation_size, size_t num_elements) {

	if (num_elements == 10000) {
		std::cout << "FP Input:\n\trecieved x_data: " << x_data << "\n\trecieved result_data: " << result_data << "\n\trecieved activation_size: " << activation_size << "\n\trecieved num_elements: " << num_elements << "\n";
	}

	size_t weight_idx = 0;
	size_t bias_idx = 0;

	size_t input_idx = activation_size;
	size_t output_idx = 0;

	for (size_t i = 0; i < m_dimensions.size() - 1; i++) {

		dim3 grid(ceil((num_elements * m_dimensions[i + 1]) / 8));

		float* weights = m_network + weight_idx;
		float* bias = m_bias + bias_idx;

		float* input = i == 0 ? x_data : (result_data + input_idx);
		float* output = result_data + output_idx;

		float* activation = output + activation_size;

		// arguments
		void* dp_args[7] = { &weights, &input, &output, &m_dimensions[i + 1], &m_dimensions[i], (i == 0 ? &num_elements : &m_dimensions[i]), (i == 0 ? &m_dimensions[i] : &num_elements) };
		void* ba_args[4] = { &output, &bias, &m_dimensions[i + 1], &num_elements };
		void* af_args[4] = { &output, &activation, &m_dimensions[i + 1], &num_elements };

		// compute total
		i == 0 ? hipLaunchKernel(reinterpret_cast<const void*>(dot_prod_t_b), grid, 8, dp_args, 0, nullptr) :
				 hipLaunchKernel(reinterpret_cast<const void*>(dot_prod), grid, 8, dp_args, 0, nullptr);
		hipDeviceSynchronize();
		
		// add bias
		hipLaunchKernel(reinterpret_cast<const void*>(horizontal_add), ceil(m_dimensions[i + 1] / 8), 8, ba_args, 0, nullptr);
		hipDeviceSynchronize();

		// activation funciton
		hipLaunchKernel(reinterpret_cast<const void*>(leaky_relu), grid, 8, af_args, 0, nullptr);
		hipDeviceSynchronize();

		weight_idx += m_dimensions[i] * m_dimensions[i + 1];
		bias_idx += m_dimensions[i + 1];

		input_idx += i == 0 ? 0 : (m_dimensions[i] * num_elements);
		output_idx += m_dimensions[i + 1] * num_elements;
	}
}
void CudaNetwork::back_prop(float* x_data, float* y_data, float learning_rate, size_t num_elements) {

	float factor = learning_rate / (float)num_elements;

	// -> compute loss
	{
		float* last_d_total = &m_d_total[m_batch_activation_size - (m_dimensions.back() * num_elements)];
		float* last_activation = &m_activation[m_batch_activation_size - (m_dimensions.back() * num_elements)];

		void* args[5] = { &last_activation, &last_d_total, &y_data, &m_dimensions.back(), &num_elements };
		
		hipLaunchKernel(reinterpret_cast<const void*>(one_hot_loss), ceil(num_elements / 8), 8, args, 0, nullptr);
		hipDeviceSynchronize();
	}


	// -> compute d_total
	{
		size_t weight_idx = m_weights_size - (m_dimensions.back() * m_dimensions[m_dimensions.size() - 2]);
		size_t d_total_idx = m_batch_activation_size - (m_dimensions.back() * num_elements);

		for (size_t i = m_dimensions.size() - 2; i > 0; i--) {

			float* weight = &m_network[weight_idx];
			float* prev_total = &m_batch_data[d_total_idx - (m_dimensions[i] * num_elements)];

			float* cur_d_total = &m_d_total[d_total_idx];
			float* prev_d_total = &m_d_total[d_total_idx - (m_dimensions[i] * num_elements)];

			void* dp_args[7] = { &weight, &cur_d_total, &prev_d_total, &m_dimensions[i + 1], &m_dimensions[i], &m_dimensions[i + 1], &num_elements };
			void* af_args[4] = { &prev_total, &prev_d_total, &m_dimensions[i], &num_elements };

			hipLaunchKernel(reinterpret_cast<const void*>(dot_prod_t_a), ceil(m_dimensions[i + 1]), 8, dp_args, 0, nullptr);
			hipDeviceSynchronize();

			// multiply by activation function derivative
			hipLaunchKernel(reinterpret_cast<const void*>(leaky_relu_derivative), ceil(m_dimensions[i + 1] / 8), 8, af_args, 0, nullptr);
			hipDeviceSynchronize();

			d_total_idx -= m_dimensions[i] * num_elements;
			weight_idx -= m_dimensions[i] * m_dimensions[i - 1];
		}
	}

	
	// -> compute d_weights and d_biases
	{
		size_t activation_idx = 0;

		size_t d_total_idx = 0;
		size_t d_weights_idx = 0;
		size_t d_bias_idx = 0;

		for (size_t i = 0; i < m_dimensions.size() - 1; i++) {

			dim3 grid(ceil((m_dimensions[i + 1] * m_dimensions[i]) / 8));

			float* prev_activation = i == 0 ? &x_data[0] : &m_activation[activation_idx];

			float* d_total = &m_d_total[d_total_idx];
			float* d_weights = &m_d_weights[d_weights_idx];
			float* d_bias = &m_d_bias[d_bias_idx];

			void* dw_args[7] = { &d_total, &prev_activation, &d_weights, &m_dimensions[i + 1], &num_elements, (i == 0 ? &num_elements : &m_dimensions[i]), (i == 0 ? &m_dimensions[i] : &num_elements) };
			void* db_args[4] = { &d_total, &d_bias, &m_dimensions[i + 1], &num_elements };

			// d_weights
			i == 0 ?
				hipLaunchKernel(reinterpret_cast<const void*>(dot_prod), grid, 8, dw_args, 0, nullptr) :
				hipLaunchKernel(reinterpret_cast<const void*>(dot_prod_t_b), grid, 8, dw_args, 0, nullptr);
			hipDeviceSynchronize();

			// d_biases
			hipLaunchKernel(reinterpret_cast<const void*>(horizontal_sum), ceil(m_dimensions[i + 1] / 8), 8, db_args, 0, nullptr);
			hipDeviceSynchronize();

			d_bias_idx += m_dimensions[i + 1];
			d_total_idx += m_dimensions[i + 1] * num_elements;
			d_weights_idx += m_dimensions[i] * m_dimensions[i + 1];
			activation_idx += i == 0 ? 0 : (m_dimensions[i] * num_elements);
		}
	}

	// update weights and biases
	{
		void* uw_args[4] = { &m_network, &m_d_weights, &factor, &m_weights_size };
		void* ub_args[4] = { &m_bias, &m_d_bias, &factor, &m_bias_size };

		hipLaunchKernel(reinterpret_cast<const void*>(update_weights), ceil(m_weights_size / 8), 8, uw_args, 0, nullptr);
		hipLaunchKernel(reinterpret_cast<const void*>(update_bias), ceil(m_bias_size / 8), 8, ub_args, 0, nullptr);
		hipDeviceSynchronize();
	}
}

std::string CudaNetwork::test_network(float* x, float* y, size_t test_size) {

	std::cout << "\nTest input:\n\tx: " << x << "\n\ty: " << y << "\n\ttest_size: " << test_size << "\n\txy dif: " << (x - y) << "\n";

	std::cout << "before test: " << hipGetErrorString(hipGetLastError()) << "\n";
	forward_prop(x, m_test_data, m_test_activation_size, test_size);
	std::cout << "after test: " << hipGetErrorString(hipGetLastError()) << "\n";

	int* d_correct;
	int correct = -1;

	hipMalloc(&d_correct, sizeof(int));
	hipMemset(d_correct, 0, sizeof(int));

	float* prediction = m_test_activation + m_test_activation_size - (m_dimensions.back() * test_size);

	void* args[5] = { &prediction, &y, &d_correct, &m_dimensions.back(), &test_size };

	hipLaunchKernel(reinterpret_cast<const void*>(accuracy_score), ceil(test_size / 8), 7, args, 0, nullptr);

	hipMemcpy(&correct, d_correct, sizeof(int), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	return "score: " + std::to_string(correct).append(" :: ").append(std::to_string(test_size)).append(" :: ").append(std::to_string((float)correct / (float)test_size));
	//return "score: " + std::to_string(((float)correct / (float)test_size) * 100.0f);
}
std::string CudaNetwork::verbose(float* d_x_test, float* d_y_test, size_t test_samples, size_t epoch, int validation_freq, std::chrono::steady_clock::time_point start_time) {
	std::string tmp = "Epoch: " + std::to_string(epoch).append(" Time: "); int tmp_len = tmp.length();
	if (epoch % validation_freq == 0) {
		tmp.append(test_network(d_x_test, d_y_test, test_samples));
	}
	const std::chrono::duration<double, std::milli> time = std::chrono::high_resolution_clock::now() - start_time;

	return tmp.insert(tmp_len, clean_time(time.count()).append(" ")).append("\n");
}

std::string CudaNetwork::clean_time(double time) {
	const double hour = 3600000.00;
	const double minute = 60000.00;
	const double second = 1000.00;

	if (time / hour > 1.00) {
		return std::to_string(time / hour).append(" hours");
	} else if (time / minute > 1.00) {
		return std::to_string(time / minute).append(" minutes");
	} else if (time / second > 1.00) {
		return std::to_string(time / second).append(" seconds");
	} else {
		return std::to_string(time).append("(ms)");
	}
}
// Repository: efposadac/openLOWDIN
// File: src/ints/Cudint.cu

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

const int numberOfThreads = 256;
const double pi = 3.14159265358979323846;

__device__ float kroneckerDelta(int i, int j)
{
  double delta;

  delta = 1.0;
  if(i != j)
    delta = 0.0;

  return delta;
}

__device__ float errorFunction(int order, double tFunc)
{

  double tFuncsqrt;
  double errorF; 

  tFuncsqrt = sqrt(tFunc);

  if(round(tFunc) == 0.0)
    errorF = 1.0/(2*order + 1);
  else
    {
      switch(order)
	{
	case 0:
	  errorF = 0.5*erf(tFuncsqrt)*sqrt(pi/tFunc);
	  break;
	case 1:
	  errorF = 0.25*(-2*tFuncsqrt*exp(-tFunc) + sqrt(pi)*erf(tFuncsqrt))/(tFuncsqrt*tFunc);
	  break;
	case 2:
	  errorF = -0.125*(exp(-tFunc)*(6*tFuncsqrt + 4*tFuncsqrt*tFunc) - 3*sqrt(pi)*erf(tFuncsqrt))/(tFuncsqrt*tFunc*tFunc);
	  break;
	case 3:
	  errorF = -0.0625*(exp(-tFunc)*(30*tFuncsqrt + 20*tFunc*tFuncsqrt + 8*tFunc*tFunc*tFuncsqrt) - 15*sqrt(pi)*erf(tFuncsqrt))/(tFuncsqrt*tFunc*tFunc*tFunc);
	  break;
	case 4:
	  errorF = -0.03125*(exp(-tFunc)*(16*tFuncsqrt*tFunc*tFunc*tFunc + 56*tFunc*tFunc*tFuncsqrt + 140*tFunc*tFuncsqrt + 210*tFuncsqrt) - (105*sqrt(pi)*erf(tFuncsqrt)))/(tFuncsqrt*tFunc*tFunc*tFunc*tFunc);
	  break;
	}
    }

  return errorF;
}

__global__ void analyticInts(int N, 
			int *primIndices_d,
			int *contIndices_d,
			double *exponents_d,
			double *primNormalization_d,
			double *coefficients_d,
			int *contCounter_d,
			int *contLength_d,
			double *origin_d,
			int *angularMoments_d,
			double *integralValues_d,
			int control,
			int kernelIter)
{
  int threadIndex = threadIdx.x + threadIdx.y*blockDim.x;
  int blockIndex = blockIdx.x;
  int global1 = threadIndex + blockIndex*blockDim.x*blockDim.y;
  int global = global1 + kernelIter; 
  
  int aa, bb, rr, ss, ii, jj, kk, ll;
  int contractionID;
  double exponentII, exponentJJ, exponentKK, exponentLL;
  double coefficientsII, coefficientsJJ, coefficientsKK, coefficientsLL;
  double primNormII, primNormJJ, primNormKK, primNormLL;
  int exponentIterII, exponentIterJJ, exponentIterKK, exponentIterLL;
  double IIx, IIy, IIz, JJx, JJy, JJz, KKx, KKy, KKz, LLx, LLy, LLz;
  double preIntegral, normIntegral; 
  double etha;
  int lAA, lBB, lRR, lSS; // Angular moments of contractions
  int integralCase;

  double A, B, C, D, KIJ, KKL, rPx, rPy, rPz, rQx, rQy, rQz, rPQ, rIJ, rKL, tFunc, prefact;
  double FA, FB, FC, FD, FE;
  double rPQx, rPQy, rPQz;
  int alpha, beta, kappa, lambda, selectCart;
  double dij, dik, djk, dkl, djl, dil;

  if(global1< control)
    {
      // ID of unic integrals
      contractionID = primIndices_d[global*9];

      // Contraction Indices
      aa = contIndices_d[contractionID*4];
      bb = contIndices_d[contractionID*4+1];
      rr = contIndices_d[contractionID*4+2];
      ss = contIndices_d[contractionID*4+3];
      
      // Primitive indices
      ii = primIndices_d[global*9+1];
      jj = primIndices_d[global*9+2];
      kk = primIndices_d[global*9+3];
      ll = primIndices_d[global*9+4];
      
      // Label of cartesian
      alpha = primIndices_d[global*9+5];
      beta = primIndices_d[global*9+6];
      kappa = primIndices_d[global*9+7];
      lambda = primIndices_d[global*9+8];

      lAA = angularMoments_d[aa-1];
      lBB = angularMoments_d[bb-1];
      lRR = angularMoments_d[rr-1];
      lSS = angularMoments_d[ss-1];
      
      exponentIterII = contCounter_d[aa-1] + ii - 1;
      exponentIterJJ = contCounter_d[bb-1] + jj - 1;
      exponentIterKK = contCounter_d[rr-1] + kk - 1;
      exponentIterLL = contCounter_d[ss-1] + ll - 1;

      exponentII = exponents_d[exponentIterII];
      exponentJJ = exponents_d[exponentIterJJ];
      exponentKK = exponents_d[exponentIterKK];
      exponentLL = exponents_d[exponentIterLL];

      coefficientsII = coefficients_d[exponentIterII];
      coefficientsJJ = coefficients_d[exponentIterJJ];
      coefficientsKK = coefficients_d[exponentIterKK];
      coefficientsLL = coefficients_d[exponentIterLL];

      primNormII = primNormalization_d[exponentIterII];
      primNormJJ = primNormalization_d[exponentIterJJ];
      primNormKK = primNormalization_d[exponentIterKK];
      primNormLL = primNormalization_d[exponentIterLL];

      IIx = origin_d[(aa*3)-3];
      IIy = origin_d[(aa*3)-2];
      IIz = origin_d[(aa*3)-1];
      JJx = origin_d[(bb*3)-3];
      JJy = origin_d[(bb*3)-2];
      JJz = origin_d[(bb*3)-1];
      KKx = origin_d[(rr*3)-3];
      KKy = origin_d[(rr*3)-2];
      KKz = origin_d[(rr*3)-1];
      LLx = origin_d[(ss*3)-3];
      LLy = origin_d[(ss*3)-2];
      LLz = origin_d[(ss*3)-1];
    
      A = exponentII + exponentJJ;
      B = exponentKK + exponentLL;
      C = exponentII*exponentJJ;
      D = exponentKK*exponentLL;

      etha = (A*B)/(A+B);

      rIJ = (IIx-JJx)*(IIx-JJx) + (IIy-JJy)*(IIy-JJy) + (IIz-JJz)*(IIz-JJz);
      rKL = (KKx-LLx)*(KKx-LLx) + (KKy-LLy)*(KKy-LLy) + (KKz-LLz)*(KKz-LLz);

      KIJ = exp(-(C/A)*rIJ);
      KKL = exp(-(D/B)*rKL);

      prefact = sqrt(etha/pi)*sqrt(pi/A)*(pi/A)*sqrt(pi/B)*(pi/B)*KIJ*KKL;
      // if(aa==1 && bb==1 && rr==1 && ss==2)
      // 	printf("etha: %f KIJ: %f KKL: %f prefact: %f %f %f %f\n", etha, KIJ, KKL, prefact, D, B, rKL);
      
      rPx =(exponentII*IIx+exponentJJ*JJx)/A;
      rPy =(exponentII*IIy+exponentJJ*JJy)/A;
      rPz =(exponentII*IIz+exponentJJ*JJz)/A;
      rQx = (exponentKK*KKx+exponentLL*LLx)/B;
      rQy = (exponentKK*KKy+exponentLL*LLy)/B;
      rQz = (exponentKK*KKz+exponentLL*LLz)/B;
      
      rPQx = (rPx*A + rQx*B)/(A+B);
      rPQy = (rPy*A + rQy*B)/(A+B);
      rPQz = (rPz*A + rQz*B)/(A+B);

      rPQ = (rPx-rQx)*(rPx-rQx) + (rPy-rQy)*(rPy-rQy) + (rPz-rQz)*(rPz-rQz);

      tFunc = 0.0;
      tFunc = etha*rPQ;

      FA = 0.0;
      FB = 0.0;
      FC = 0.0;
      FD = 0.0;
      FE = 0.0;
      
      integralCase = 64*lAA + 16*lBB + 4*lRR + lSS;

      switch(integralCase)
	{
	case 0: // Integral (s,s|s,s)
	  FA = errorFunction(0, tFunc);
	  preIntegral = 2*FA*prefact;
	  break;
	case 64:
	  FA = errorFunction(0, tFunc);
	  FB = errorFunction(1, tFunc);
	  switch(alpha)
	    {
	    case 1: // Integral (px,s|s,s)
	      preIntegral = 2*(FB*(rPQx-rPx)+FA*(rPx-IIx))*prefact;
	      break;
	    case 2: // Integral (py,s|s,s)
	      preIntegral = 2*(FB*(rPQy-rPy)+FA*(rPy-IIy))*prefact;
	      break;
	    case 3: // Integral (pz,s|s,s)
	      preIntegral = 2*(FB*(rPQz-rPz)+FA*(rPz-IIz))*prefact;
	      break;
	    }
	  break;
	case 68:
	  FA = errorFunction(0, tFunc);
	  FB = errorFunction(1, tFunc);
	  FC = errorFunction(2, tFunc);
	  selectCart = 64*alpha + 4*kappa;
	  dij = kroneckerDelta(alpha, kappa);
	  switch(selectCart)
	    {
	    case 68:  // Integral (px,s|px,s)
	      preIntegral = prefact*((FB*dij)/(B+A) + 2*(rQx-KKx)*(FB*(rPQx-rPx) + FA*(rPx-IIx)) + 2*(rPQx-rQx)*(FC*(rPQx-rPx) + FB*(rPx-IIx)));
	      break;
	    case 72:  // Integral (px,s|py,s)
	      preIntegral = prefact*((FB*dij)/(B+A) + 2*(rQy-KKy)*(FB*(rPQx-rPx) + FA*(rPx-IIx)) + 2*(rPQy-rQy)*(FC*(rPQx-rPx) + FB*(rPx-IIx)));
	      break;
	    case 76:  // Integral (px,s|pz,s)
	      preIntegral = prefact*((FB*dij)/(B+A) + 2*(rQz-KKz)*(FB*(rPQx-rPx) + FA*(rPx-IIx)) + 2*(rPQz-rQz)*(FC*(rPQx-rPx) + FB*(rPx-IIx)));
	      break;
	    case 132: // Integral (py,s|px,s)
	      preIntegral = prefact*((FB*dij)/(B+A) + 2*(rQx-KKx)*(FB*(rPQy-rPy) + FA*(rPy-IIy)) + 2*(rPQx-rQx)*(FC*(rPQy-rPy) + FB*(rPy-IIy)));
	      break;
	    case 136: // Integral (py,s|py,s)
	      preIntegral = prefact*((FB*dij)/(B+A) + 2*(rQy-KKy)*(FB*(rPQy-rPy) + FA*(rPy-IIy)) + 2*(rPQy-rQy)*(FC*(rPQy-rPy) + FB*(rPy-IIy)));
	      break;
	    case 140: // Integral (py,s|pz,s)
	      preIntegral = prefact*((FB*dij)/(B+A) + 2*(rQz-KKz)*(FB*(rPQy-rPy) + FA*(rPy-IIy)) + 2*(rPQz-rQz)*(FC*(rPQy-rPy) + FB*(rPy-IIy)));
	      break;
	    case 196: // Integral (pz,s|px,s)
	      preIntegral = prefact*((FB*dij)/(B+A) + 2*(rQx-KKx)*(FB*(rPQz-rPz) + FA*(rPz-IIz)) + 2*(rPQx-rQx)*(FC*(rPQz-rPz) + FB*(rPz-IIz)));
	      break;
	    case 200: // Integral (pz,s|py,s)
	      preIntegral = prefact*((FB*dij)/(B+A) + 2*(rQy-KKy)*(FB*(rPQz-rPz) + FA*(rPz-IIz)) + 2*(rPQy-rQy)*(FC*(rPQz-rPz) + FB*(rPz-IIz)));
	      break;
	    case 204: // Integral (pz,s|pz,s)
	      preIntegral = prefact*((FB*dij)/(B+A) + 2*(rQz-KKz)*(FB*(rPQz-rPz) + FA*(rPz-IIz)) + 2*(rPQz-rQz)*(FC*(rPQz-rPz) + FB*(rPz-IIz)));
	      break;
	    }
	  break;
	case 80:
	  FA = errorFunction(0, tFunc);
	  FB = errorFunction(1, tFunc);
	  FC = errorFunction(2, tFunc);
	  selectCart = 64*alpha + 16*beta;
	  dij = kroneckerDelta(alpha, beta);
	  switch(selectCart)
	    {
	    case 80: // Integral (px,px|s,s)
	      preIntegral = prefact*(((A*FA-etha*FB)*dij)/(A*A) + 2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPQx-rPx) + 2*(FB*(rPQx-rPx) + FA*(rPx-IIx))*(rPx-JJx));
	      break;
	    case 96: // Integral (px,py|s,s)
	      preIntegral = prefact*(((A*FA-etha*FB)*dij)/(A*A) + 2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPQy-rPy) + 2*(FB*(rPQx-rPx) + FA*(rPx-IIx))*(rPy-JJy));
	      break;
	    case 112: // Integral (px,pz|s,s)
	      preIntegral = prefact*(((A*FA-etha*FB)*dij)/(A*A) + 2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPQz-rPz) + 2*(FB*(rPQx-rPx) + FA*(rPx-IIx))*(rPz-JJz));
	      break;
	    case 160: // Integral (py,py|s,s)
	      preIntegral = prefact*(((A*FA-etha*FB)*dij)/(A*A) + 2*(FC*(rPQy-rPy) + FB*(rPy-IIy))*(rPQy-rPy) + 2*(FB*(rPQy-rPy) + FA*(rPy-IIy))*(rPy-JJy));
	      break;
	    case 176: // Integral (py,pz|s,s)
	      preIntegral = prefact*(((A*FA-etha*FB)*dij)/(A*A) + 2*(FC*(rPQy-rPy) + FB*(rPy-IIy))*(rPQz-rPz) + 2*(FB*(rPQy-rPy) + FA*(rPy-IIy))*(rPz-JJz));
	      break;
	    case 240: // Integral (pz,pz|s,s)
	      preIntegral = prefact*(((A*FA-etha*FB)*dij)/(A*A) + 2*(FC*(rPQz-rPz) + FB*(rPz-IIz))*(rPQz-rPz) + 2*(FB*(rPQz-rPz) + FA*(rPz-IIz))*(rPz-JJz));
	      break;
	    }
	  break;
	case 84:
	  FA = errorFunction(0, tFunc);
	  FB = errorFunction(1, tFunc);
	  FC = errorFunction(2, tFunc);
	  FD = errorFunction(3, tFunc);
	  selectCart = 64*alpha + 16*beta + 4*kappa;
	  dij = kroneckerDelta(alpha, beta);
	  dik = kroneckerDelta(alpha, kappa);
	  djk = kroneckerDelta(beta, kappa);
	  switch(selectCart)
	    {
	    case 84:  // Integral (px,px|px,s)
	      preIntegral = prefact*((rQx-KKx)*(((A*FA-etha*FB)*dij)/(A*A) +
						2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPQx-rPx) +
						2*(FB*(rPQx-rPx) + FA*(rPx-IIx))*(rPx-JJx)) +
				     (rPQx-rQx)*(((A*FB-etha*FC)*dij)/(A*A) +
						 2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPQx-rPx) +
						 2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPx-JJx)) +
				     (FC*(djk*(rPQx-rPx) + dik*(rPQx-rPx)) +
				      FB*(djk*(rPx-IIx) + dik*(rPx-JJx)))/(B+A));
	      break;
	    case 88:  // Integral (px,px|py,s)
	      preIntegral = prefact*((rQy-KKy)*(((A*FA-etha*FB)*dij)/(A*A) +
						2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPQx-rPx) +
						2*(FB*(rPQx-rPx) + FA*(rPx-IIx))*(rPx-JJx)) +
				     (rPQy-rQy)*(((A*FB-etha*FC)*dij)/(A*A) +
						 2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPQx-rPx) +
						 2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPx-JJx)) +
				     (FC*(djk*(rPQx-rPx) + dik*(rPQx-rPx)) +
				      FB*(djk*(rPx-IIx) + dik*(rPx-JJx)))/(B+A));	      
	      break;
	    case 92:  // Integral (px,px|pz,s)
	      preIntegral = prefact*((rQz-KKz)*(((A*FA-etha*FB)*dij)/(A*A) +
						2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPQx-rPx) +
						2*(FB*(rPQx-rPx) + FA*(rPx-IIx))*(rPx-JJx)) +
				     (rPQz-rQz)*(((A*FB-etha*FC)*dij)/(A*A) +
						 2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPQx-rPx) +
						 2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPx-JJx)) +
				     (FC*(djk*(rPQx-rPx) + dik*(rPQx-rPx)) +
				      FB*(djk*(rPx-IIx) + dik*(rPx-JJx)))/(B+A));	      
	      break;
	    case 100: // Integral (px,py|px,s)
	      preIntegral = prefact*((rQx-KKx)*(((A*FA-etha*FB)*dij)/(A*A) +
						2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPQy-rPy) +
						2*(FB*(rPQx-rPx) + FA*(rPx-IIx))*(rPy-JJy)) +
				     (rPQx-rQx)*(((A*FB-etha*FC)*dij)/(A*A) +
						 2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPQy-rPy) +
						 2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPy-JJy)) +
				     (FC*(djk*(rPQx-rPx) + dik*(rPQy-rPy)) +
				      FB*(djk*(rPx-IIx) + dik*(rPy-JJy)))/(B+A));
	      break;
	    case 104: // Integral (px,py|py,s)
	      preIntegral = prefact*((rQy-KKy)*(((A*FA-etha*FB)*dij)/(A*A) +
						2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPQy-rPy) +
						2*(FB*(rPQx-rPx) + FA*(rPx-IIx))*(rPy-JJy)) +
				     (rPQy-rQy)*(((A*FB-etha*FC)*dij)/(A*A) +
						 2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPQy-rPy) +
						 2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPy-JJy)) +
				     (FC*(djk*(rPQx-rPx) + dik*(rPQy-rPy)) +
				      FB*(djk*(rPx-IIx) + dik*(rPy-JJy)))/(B+A));
	      break;
	    case 108: // Integral (px,py|pz,s)
	      preIntegral = prefact*((rQz-KKz)*(((A*FA-etha*FB)*dij)/(A*A) +
						2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPQy-rPy) +
						2*(FB*(rPQx-rPx) + FA*(rPx-IIx))*(rPy-JJy)) +
				     (rPQz-rQz)*(((A*FB-etha*FC)*dij)/(A*A) +
						 2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPQy-rPy) +
						 2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPy-JJy)) +
				     (FC*(djk*(rPQx-rPx) + dik*(rPQy-rPy)) +
				      FB*(djk*(rPx-IIx) + dik*(rPy-JJy)))/(B+A));
	      break;
	    case 116: // Integral (px,pz|px,s)
	      preIntegral = prefact*((rQx-KKx)*(((A*FA-etha*FB)*dij)/(A*A) +
						2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPQz-rPz) +
						2*(FB*(rPQx-rPx) + FA*(rPx-IIx))*(rPz-JJz)) +
				     (rPQx-rQx)*(((A*FB-etha*FC)*dij)/(A*A) +
						 2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPQz-rPz) +
						 2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPz-JJz)) +
				     (FC*(djk*(rPQx-rPx) + dik*(rPQz-rPz)) +
				      FB*(djk*(rPx-IIx) + dik*(rPz-JJz)))/(B+A));
	      break;
	    case 120: // Integral (px,pz|py,s)
	      preIntegral = prefact*((rQy-KKy)*(((A*FA-etha*FB)*dij)/(A*A) +
						2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPQz-rPz) +
						2*(FB*(rPQx-rPx) + FA*(rPx-IIx))*(rPz-JJz)) +
				     (rPQy-rQy)*(((A*FB-etha*FC)*dij)/(A*A) +
						 2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPQz-rPz) +
						 2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPz-JJz)) +
				     (FC*(djk*(rPQx-rPx) + dik*(rPQz-rPz)) +
				      FB*(djk*(rPx-IIx) + dik*(rPz-JJz)))/(B+A));
	      break;
	    case 124: // Integral (px,pz|pz,s)
	      preIntegral = prefact*((rQz-KKz)*(((A*FA-etha*FB)*dij)/(A*A) +
						2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPQz-rPz) +
						2*(FB*(rPQx-rPx) + FA*(rPx-IIx))*(rPz-JJz)) +
				     (rPQz-rQz)*(((A*FB-etha*FC)*dij)/(A*A) +
						 2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPQz-rPz) +
						 2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPz-JJz)) +
				     (FC*(djk*(rPQx-rPx) + dik*(rPQz-rPz)) +
				      FB*(djk*(rPx-IIx) + dik*(rPz-JJz)))/(B+A));
	      break;
	    case 164: // Integral (py,py|px,s)
	      preIntegral = prefact*((rQx-KKx)*(((A*FA-etha*FB)*dij)/(A*A) +
						2*(FC*(rPQy-rPy) + FB*(rPy-IIy))*(rPQy-rPy) +
						2*(FB*(rPQy-rPy) + FA*(rPy-IIy))*(rPy-JJy)) +
				     (rPQx-rQx)*(((A*FB-etha*FC)*dij)/(A*A) +
						 2*(FD*(rPQy-rPy) + FC*(rPy-IIy))*(rPQy-rPy) +
						 2*(FC*(rPQy-rPy) + FB*(rPy-IIy))*(rPy-JJy)) +
				     (FC*(djk*(rPQy-rPy) + dik*(rPQy-rPy)) +
				      FB*(djk*(rPy-IIy) + dik*(rPy-JJy)))/(B+A));
	      break;
	    case 168: // Integral (py,py|py,s)
	      preIntegral = prefact*((rQy-KKy)*(((A*FA-etha*FB)*dij)/(A*A) +
						2*(FC*(rPQy-rPy) + FB*(rPy-IIy))*(rPQy-rPy) +
						2*(FB*(rPQy-rPy) + FA*(rPy-IIy))*(rPy-JJy)) +
				     (rPQy-rQy)*(((A*FB-etha*FC)*dij)/(A*A) +
						 2*(FD*(rPQy-rPy) + FC*(rPy-IIy))*(rPQy-rPy) +
						 2*(FC*(rPQy-rPy) + FB*(rPy-IIy))*(rPy-JJy)) +
				     (FC*(djk*(rPQy-rPy) + dik*(rPQy-rPy)) +
				      FB*(djk*(rPy-IIy) + dik*(rPy-JJy)))/(B+A));
	      break;
	    case 172: // Integral (py,py|pz,s)
	      preIntegral = prefact*((rQz-KKz)*(((A*FA-etha*FB)*dij)/(A*A) +
						2*(FC*(rPQy-rPy) + FB*(rPy-IIy))*(rPQy-rPy) +
						2*(FB*(rPQy-rPy) + FA*(rPy-IIy))*(rPy-JJy)) +
				     (rPQz-rQz)*(((A*FB-etha*FC)*dij)/(A*A) +
						 2*(FD*(rPQy-rPy) + FC*(rPy-IIy))*(rPQy-rPy) +
						 2*(FC*(rPQy-rPy) + FB*(rPy-IIy))*(rPy-JJy)) +
				     (FC*(djk*(rPQy-rPy) + dik*(rPQy-rPy)) +
				      FB*(djk*(rPy-IIy) + dik*(rPy-JJy)))/(B+A));
	      break;
	    case 180: // Integral (py,pz|px,s)
	      preIntegral = prefact*((rQx-KKx)*(((A*FA-etha*FB)*dij)/(A*A) +
						2*(FC*(rPQy-rPy) + FB*(rPy-IIy))*(rPQz-rPz) +
						2*(FB*(rPQy-rPy) + FA*(rPy-IIy))*(rPz-JJz)) +
				     (rPQx-rQx)*(((A*FB-etha*FC)*dij)/(A*A) +
						 2*(FD*(rPQy-rPy) + FC*(rPy-IIy))*(rPQz-rPz) +
						 2*(FC*(rPQy-rPy) + FB*(rPy-IIy))*(rPz-JJz)) +
				     (FC*(djk*(rPQy-rPy) + dik*(rPQz-rPz)) +
				      FB*(djk*(rPy-IIy) + dik*(rPz-JJz)))/(B+A));
	      break;
	    case 184: // Integral (py,pz|py,s)
	      preIntegral = prefact*((rQy-KKy)*(((A*FA-etha*FB)*dij)/(A*A) +
						2*(FC*(rPQy-rPy) + FB*(rPy-IIy))*(rPQz-rPz) +
						2*(FB*(rPQy-rPy) + FA*(rPy-IIy))*(rPz-JJz)) +
				     (rPQy-rQy)*(((A*FB-etha*FC)*dij)/(A*A) +
						 2*(FD*(rPQy-rPy) + FC*(rPy-IIy))*(rPQz-rPz) +
						 2*(FC*(rPQy-rPy) + FB*(rPy-IIy))*(rPz-JJz)) +
				     (FC*(djk*(rPQy-rPy) + dik*(rPQz-rPz)) +
				      FB*(djk*(rPy-IIy) + dik*(rPz-JJz)))/(B+A));
	      break;
	    case 188: // Integral (py,pz|pz,s)
	      preIntegral = prefact*((rQz-KKz)*(((A*FA-etha*FB)*dij)/(A*A) +
						2*(FC*(rPQy-rPy) + FB*(rPy-IIy))*(rPQz-rPz) +
						2*(FB*(rPQy-rPy) + FA*(rPy-IIy))*(rPz-JJz)) +
				     (rPQz-rQz)*(((A*FB-etha*FC)*dij)/(A*A) +
						 2*(FD*(rPQy-rPy) + FC*(rPy-IIy))*(rPQz-rPz) +
						 2*(FC*(rPQy-rPy) + FB*(rPy-IIy))*(rPz-JJz)) +
				     (FC*(djk*(rPQy-rPy) + dik*(rPQz-rPz)) +
				      FB*(djk*(rPy-IIy) + dik*(rPz-JJz)))/(B+A));
	      break;
	    case 244: // Integral (pz,pz|px,s)
	      preIntegral = prefact*((rQx-KKx)*(((A*FA-etha*FB)*dij)/(A*A) +
						2*(FC*(rPQz-rPz) + FB*(rPz-IIz))*(rPQz-rPz) +
						2*(FB*(rPQz-rPz) + FA*(rPz-IIz))*(rPz-JJz)) +
				     (rPQx-rQx)*(((A*FB-etha*FC)*dij)/(A*A) +
						 2*(FD*(rPQz-rPz) + FC*(rPz-IIz))*(rPQz-rPz) +
						 2*(FC*(rPQz-rPz) + FB*(rPz-IIz))*(rPz-JJz)) +
				     (FC*(djk*(rPQz-rPz) + dik*(rPQz-rPz)) +
				      FB*(djk*(rPz-IIz) + dik*(rPz-JJz)))/(B+A));
	      break;
	    case 248: // Integral (pz,pz|py,s)
	      preIntegral = prefact*((rQy-KKy)*(((A*FA-etha*FB)*dij)/(A*A) +
						2*(FC*(rPQz-rPz) + FB*(rPz-IIz))*(rPQz-rPz) +
						2*(FB*(rPQz-rPz) + FA*(rPz-IIz))*(rPz-JJz)) +
				     (rPQy-rQy)*(((A*FB-etha*FC)*dij)/(A*A) +
						 2*(FD*(rPQz-rPz) + FC*(rPz-IIz))*(rPQz-rPz) +
						 2*(FC*(rPQz-rPz) + FB*(rPz-IIz))*(rPz-JJz)) +
				     (FC*(djk*(rPQz-rPz) + dik*(rPQz-rPz)) +
				      FB*(djk*(rPz-IIz) + dik*(rPz-JJz)))/(B+A));
	      break;
	    case 252: // Integral (pz,pz|pz,s)
	      preIntegral = prefact*((rQz-KKz)*(((A*FA-etha*FB)*dij)/(A*A) +
						2*(FC*(rPQz-rPz) + FB*(rPz-IIz))*(rPQz-rPz) +
						2*(FB*(rPQz-rPz) + FA*(rPz-IIz))*(rPz-JJz)) +
				     (rPQz-rQz)*(((A*FB-etha*FC)*dij)/(A*A) +
						 2*(FD*(rPQz-rPz) + FC*(rPz-IIz))*(rPQz-rPz) +
						 2*(FC*(rPQz-rPz) + FB*(rPz-IIz))*(rPz-JJz)) +
				     (FC*(djk*(rPQz-rPz) + dik*(rPQz-rPz)) +
				      FB*(djk*(rPz-IIz) + dik*(rPz-JJz)))/(B+A));
	      break;	      
	    }
	  break;
	case 85:
	  FA = errorFunction(0, tFunc);
	  FB = errorFunction(1, tFunc);
	  FC = errorFunction(2, tFunc);
	  FD = errorFunction(3, tFunc);
	  FE = errorFunction(4, tFunc);
	  selectCart = 64*alpha + 16*beta + 4*kappa + lambda;
	  dij = kroneckerDelta(alpha, beta);
	  dik = kroneckerDelta(alpha, kappa);
	  dil = kroneckerDelta(alpha, lambda);
	  djk = kroneckerDelta(beta, kappa);
	  djl = kroneckerDelta(beta, lambda);
	  dkl = kroneckerDelta(kappa, lambda);
	  switch(selectCart)
	    {
	    case 85:   // Integral (px,px|px,px)
	      preIntegral = prefact*((dkl*(((A*FA-etha*FB)*dij)/(A*A) +
					   2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPQx-rPx) +
					   2*(FB*(rPQx-rPx) + FA*(rPx-IIx))*(rPx-JJx) -
					   (etha*(((A*FB-etha*FC)*dij)/(A*A) +
						  2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPQx-rPx) +
						  2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPx-JJx)))/B))/(2*B) +
				     (djl*((FC*dik)/(B+A) +
					   2*(rQx-KKx)*(FC*(rPQx-rPx) + FB*(rPx-IIx)) +
					   2*(rPQx-rQx)*(FD*(rPQx-rPx) + FC*(rPx-IIx))) +
				      dil*((FC*dik)/(B+A) +
					   2*(rQx-KKx)*(FC*(rPQx-rPx) + FB*(rPx-JJx)) +
					   2*(rPQx-rQx)*(FD*(rPQx-rPx) + FC*(rPx-JJx))))/(2*(B+A)) +
				     (rQx-LLx)*((rQx-KKx)*(((A*FA-etha*FB)*dij)/(A*A) +
							   2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPQx-rPx) +
							   2*(FB*(rPQx-rPx) + FA*(rPx-IIx))*(rPx-JJx)) +
						(rPQx-rQx)*(((A*FB-etha*FC)*dij)/(A*A) +
							    2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPQx-rPx) +
							    2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPx-JJx)) +
						(FC*(djk*(rPQx-rPx) + dik*(rPQx-rPx)) +
						 FB*(djk*(rPx-IIx) + dik*(rPx-JJx)))/(B+A)) +
				     (rPQx-rQx)*((rQx-KKx)*(((A*FB-etha*FC)*dij)/(A*A) +
							    2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPQx-rPx) +
							    2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPx-JJx)) +
						 (rPQx-rQx)*(((A*FC-etha*FD)*dij)/(A*A) +
							     2*(FE*(rPQx-rPx) + FD*(rPx-IIx))*(rPQx-rPx) +
							     2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPx-JJx)) +
						 (FD*(djk*(rPQx-rPx) + dik*(rPQx-rPx)) +
						  FC*(djk*(rPx-IIx) + dik*(rPx-JJx)))/(B+A)));
	      break;
	    case 86:   // Integral (px,px|px,py)
	      preIntegral = prefact*((dkl*(((A*FA-etha*FB)*dij)/(A*A) +
					   2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPQx-rPx) +
					   2*(FB*(rPQx-rPx) + FA*(rPx-IIx))*(rPx-JJx) -
					   (etha*(((A*FB-etha*FC)*dij)/(A*A) +
						  2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPQx-rPx) +
						  2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPx-JJx)))/B))/(2*B) +
				     (djl*((FC*dik)/(B+A) +
					   2*(rQx-KKx)*(FC*(rPQx-rPx) + FB*(rPx-IIx)) +
					   2*(rPQx-rQx)*(FD*(rPQx-rPx) + FC*(rPx-IIx))) +
				      dil*((FC*dik)/(B+A) +
					   2*(rQx-KKx)*(FC*(rPQx-rPx) + FB*(rPx-JJx)) +
					   2*(rPQx-rQx)*(FD*(rPQx-rPx) + FC*(rPx-JJx))))/(2*(B+A)) +
				     (rQy-LLy)*((rQx-KKx)*(((A*FA-etha*FB)*dij)/(A*A) +
							   2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPQx-rPx) +
							   2*(FB*(rPQx-rPx) + FA*(rPx-IIx))*(rPx-JJx)) +
						(rPQx-rQx)*(((A*FB-etha*FC)*dij)/(A*A) +
							    2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPQx-rPx) +
							    2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPx-JJx)) +
						(FC*(djk*(rPQx-rPx) + dik*(rPQx-rPx)) +
						 FB*(djk*(rPx-IIx) + dik*(rPx-JJx)))/(B+A)) +
				     (rPQy-rQy)*((rQx-KKx)*(((A*FB-etha*FC)*dij)/(A*A) +
							    2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPQx-rPx) +
							    2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPx-JJx)) +
						 (rPQx-rQx)*(((A*FC-etha*FD)*dij)/(A*A) +
							     2*(FE*(rPQx-rPx) + FD*(rPx-IIx))*(rPQx-rPx) +
							     2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPx-JJx)) +
						 (FD*(djk*(rPQx-rPx) + dik*(rPQx-rPx)) +
						  FC*(djk*(rPx-IIx) + dik*(rPx-JJx)))/(B+A)));
	      break;
	    case 87:   // Integral (px,px|px,pz)
	      preIntegral = prefact*((dkl*(((A*FA-etha*FB)*dij)/(A*A) +
					   2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPQx-rPx) +
					   2*(FB*(rPQx-rPx) + FA*(rPx-IIx))*(rPx-JJx) -
					   (etha*(((A*FB-etha*FC)*dij)/(A*A) +
						  2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPQx-rPx) +
						  2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPx-JJx)))/B))/(2*B) +
				     (djl*((FC*dik)/(B+A) +
					   2*(rQx-KKx)*(FC*(rPQx-rPx) + FB*(rPx-IIx)) +
					   2*(rPQx-rQx)*(FD*(rPQx-rPx) + FC*(rPx-IIx))) +
				      dil*((FC*dik)/(B+A) +
					   2*(rQx-KKx)*(FC*(rPQx-rPx) + FB*(rPx-JJx)) +
					   2*(rPQx-rQx)*(FD*(rPQx-rPx) + FC*(rPx-JJx))))/(2*(B+A)) +
				     (rQz-LLz)*((rQx-KKx)*(((A*FA-etha*FB)*dij)/(A*A) +
							   2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPQx-rPx) +
							   2*(FB*(rPQx-rPx) + FA*(rPx-IIx))*(rPx-JJx)) +
						(rPQx-rQx)*(((A*FB-etha*FC)*dij)/(A*A) +
							    2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPQx-rPx) +
							    2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPx-JJx)) +
						(FC*(djk*(rPQx-rPx) + dik*(rPQx-rPx)) +
						 FB*(djk*(rPx-IIx) + dik*(rPx-JJx)))/(B+A)) +
				     (rPQz-rQz)*((rQx-KKx)*(((A*FB-etha*FC)*dij)/(A*A) +
							    2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPQx-rPx) +
							    2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPx-JJx)) +
						 (rPQx-rQx)*(((A*FC-etha*FD)*dij)/(A*A) +
							     2*(FE*(rPQx-rPx) + FD*(rPx-IIx))*(rPQx-rPx) +
							     2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPx-JJx)) +
						 (FD*(djk*(rPQx-rPx) + dik*(rPQx-rPx)) +
						  FC*(djk*(rPx-IIx) + dik*(rPx-JJx)))/(B+A)));
	      break;
	    case 90:   // Integral (px,px|py,py)
	      preIntegral = prefact*((dkl*(((A*FA-etha*FB)*dij)/(A*A) +
					   2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPQx-rPx) +
					   2*(FB*(rPQx-rPx) + FA*(rPx-IIx))*(rPx-JJx) -
					   (etha*(((A*FB-etha*FC)*dij)/(A*A) +
						  2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPQx-rPx) +
						  2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPx-JJx)))/B))/(2*B) +
				     (djl*((FC*dik)/(B+A) +
					   2*(rQy-KKy)*(FC*(rPQx-rPx) + FB*(rPx-IIx)) +
					   2*(rPQy-rQy)*(FD*(rPQx-rPx) + FC*(rPx-IIx))) +
				      dil*((FC*dik)/(B+A) +
					   2*(rQy-KKy)*(FC*(rPQx-rPx) + FB*(rPx-JJx)) +
					   2*(rPQy-rQy)*(FD*(rPQx-rPx) + FC*(rPx-JJx))))/(2*(B+A)) +
				     (rQy-LLy)*((rQy-KKy)*(((A*FA-etha*FB)*dij)/(A*A) +
							   2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPQx-rPx) +
							   2*(FB*(rPQx-rPx) + FA*(rPx-IIx))*(rPx-JJx)) +
						(rPQy-rQy)*(((A*FB-etha*FC)*dij)/(A*A) +
							    2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPQx-rPx) +
							    2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPx-JJx)) +
						(FC*(djk*(rPQx-rPx) + dik*(rPQx-rPx)) +
						 FB*(djk*(rPx-IIx) + dik*(rPx-JJx)))/(B+A)) +
				     (rPQy-rQy)*((rQy-KKy)*(((A*FB-etha*FC)*dij)/(A*A) +
							    2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPQx-rPx) +
							    2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPx-JJx)) +
						 (rPQy-rQy)*(((A*FC-etha*FD)*dij)/(A*A) +
							     2*(FE*(rPQx-rPx) + FD*(rPx-IIx))*(rPQx-rPx) +
							     2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPx-JJx)) +
						 (FD*(djk*(rPQx-rPx) + dik*(rPQx-rPx)) +
						  FC*(djk*(rPx-IIx) + dik*(rPx-JJx)))/(B+A)));
	      break;
	    case 91:   // Integral (px,px|py,pz)
	      preIntegral = prefact*((dkl*(((A*FA-etha*FB)*dij)/(A*A) +
					   2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPQx-rPx) +
					   2*(FB*(rPQx-rPx) + FA*(rPx-IIx))*(rPx-JJx) -
					   (etha*(((A*FB-etha*FC)*dij)/(A*A) +
						  2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPQx-rPx) +
						  2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPx-JJx)))/B))/(2*B) +
				     (djl*((FC*dik)/(B+A) +
					   2*(rQy-KKy)*(FC*(rPQx-rPx) + FB*(rPx-IIx)) +
					   2*(rPQy-rQy)*(FD*(rPQx-rPx) + FC*(rPx-IIx))) +
				      dil*((FC*dik)/(B+A) +
					   2*(rQy-KKy)*(FC*(rPQx-rPx) + FB*(rPx-JJx)) +
					   2*(rPQy-rQy)*(FD*(rPQx-rPx) + FC*(rPx-JJx))))/(2*(B+A)) +
				     (rQz-LLz)*((rQy-KKy)*(((A*FA-etha*FB)*dij)/(A*A) +
							   2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPQx-rPx) +
							   2*(FB*(rPQx-rPx) + FA*(rPx-IIx))*(rPx-JJx)) +
						(rPQy-rQy)*(((A*FB-etha*FC)*dij)/(A*A) +
							    2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPQx-rPx) +
							    2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPx-JJx)) +
						(FC*(djk*(rPQx-rPx) + dik*(rPQx-rPx)) +
						 FB*(djk*(rPx-IIx) + dik*(rPx-JJx)))/(B+A)) +
				     (rPQz-rQz)*((rQy-KKy)*(((A*FB-etha*FC)*dij)/(A*A) +
							    2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPQx-rPx) +
							    2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPx-JJx)) +
						 (rPQy-rQy)*(((A*FC-etha*FD)*dij)/(A*A) +
							     2*(FE*(rPQx-rPx) + FD*(rPx-IIx))*(rPQx-rPx) +
							     2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPx-JJx)) +
						 (FD*(djk*(rPQx-rPx) + dik*(rPQx-rPx)) +
						  FC*(djk*(rPx-IIx) + dik*(rPx-JJx)))/(B+A)));
	      break;
	    case 95:   // Integral (px,px|pz,pz)
	      preIntegral = prefact*((dkl*(((A*FA-etha*FB)*dij)/(A*A) +
					   2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPQx-rPx) +
					   2*(FB*(rPQx-rPx) + FA*(rPx-IIx))*(rPx-JJx) -
					   (etha*(((A*FB-etha*FC)*dij)/(A*A) +
						  2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPQx-rPx) +
						  2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPx-JJx)))/B))/(2*B) +
				     (djl*((FC*dik)/(B+A) +
					   2*(rQz-KKz)*(FC*(rPQx-rPx) + FB*(rPx-IIx)) +
					   2*(rPQz-rQz)*(FD*(rPQx-rPx) + FC*(rPx-IIx))) +
				      dil*((FC*dik)/(B+A) +
					   2*(rQz-KKz)*(FC*(rPQx-rPx) + FB*(rPx-JJx)) +
					   2*(rPQz-rQz)*(FD*(rPQx-rPx) + FC*(rPx-JJx))))/(2*(B+A)) +
				     (rQz-LLz)*((rQz-KKz)*(((A*FA-etha*FB)*dij)/(A*A) +
							   2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPQx-rPx) +
							   2*(FB*(rPQx-rPx) + FA*(rPx-IIx))*(rPx-JJx)) +
						(rPQz-rQz)*(((A*FB-etha*FC)*dij)/(A*A) +
							    2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPQx-rPx) +
							    2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPx-JJx)) +
						(FC*(djk*(rPQx-rPx) + dik*(rPQx-rPx)) +
						 FB*(djk*(rPx-IIx) + dik*(rPx-JJx)))/(B+A)) +
				     (rPQz-rQz)*((rQz-KKz)*(((A*FB-etha*FC)*dij)/(A*A) +
							    2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPQx-rPx) +
							    2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPx-JJx)) +
						 (rPQz-rQz)*(((A*FC-etha*FD)*dij)/(A*A) +
							     2*(FE*(rPQx-rPx) + FD*(rPx-IIx))*(rPQx-rPx) +
							     2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPx-JJx)) +
						 (FD*(djk*(rPQx-rPx) + dik*(rPQx-rPx)) +
						  FC*(djk*(rPx-IIx) + dik*(rPx-JJx)))/(B+A)));
	      break;
	    case 102:  // Integral (px,py|px,py)
	      preIntegral = prefact*((dkl*(((A*FA-etha*FB)*dij)/(A*A) +
					   2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPQy-rPy) +
					   2*(FB*(rPQx-rPx) + FA*(rPx-IIx))*(rPy-JJy) -
					   (etha*(((A*FB-etha*FC)*dij)/(A*A) +
						  2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPQy-rPy) +
						  2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPy-JJy)))/B))/(2*B) +
				     (djl*((FC*dik)/(B+A) +
					   2*(rQx-KKx)*(FC*(rPQx-rPx) + FB*(rPx-IIx)) +
					   2*(rPQx-rQx)*(FD*(rPQx-rPx) + FC*(rPx-IIx))) +
				      dil*((FC*dik)/(B+A) +
					   2*(rQx-KKx)*(FC*(rPQy-rPy) + FB*(rPy-JJy)) +
					   2*(rPQx-rQx)*(FD*(rPQy-rPy) + FC*(rPy-JJy))))/(2*(B+A)) +
				     (rQy-LLy)*((rQx-KKx)*(((A*FA-etha*FB)*dij)/(A*A) +
							   2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPQy-rPy) +
							   2*(FB*(rPQx-rPx) + FA*(rPx-IIx))*(rPy-JJy)) +
						(rPQx-rQx)*(((A*FB-etha*FC)*dij)/(A*A) +
							    2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPQy-rPy) +
							    2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPy-JJy)) +
						(FC*(djk*(rPQx-rPx) + dik*(rPQy-rPy)) +
						 FB*(djk*(rPx-IIx) + dik*(rPy-JJy)))/(B+A)) +
				     (rPQy-rQy)*((rQx-KKx)*(((A*FB-etha*FC)*dij)/(A*A) +
							    2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPQy-rPy) +
							    2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPy-JJy)) +
						 (rPQx-rQx)*(((A*FC-etha*FD)*dij)/(A*A) +
							     2*(FE*(rPQx-rPx) + FD*(rPx-IIx))*(rPQy-rPy) +
							     2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPy-JJy)) +
						 (FD*(djk*(rPQx-rPx) + dik*(rPQy-rPy)) +
						  FC*(djk*(rPx-IIx) + dik*(rPy-JJy)))/(B+A)));
	      break;
	    case 103:  // Integral (px,py|px,pz)
	      preIntegral = prefact*((dkl*(((A*FA-etha*FB)*dij)/(A*A) +
					   2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPQy-rPy) +
					   2*(FB*(rPQx-rPx) + FA*(rPx-IIx))*(rPy-JJy) -
					   (etha*(((A*FB-etha*FC)*dij)/(A*A) +
						  2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPQy-rPy) +
						  2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPy-JJy)))/B))/(2*B) +
				     (djl*((FC*dik)/(B+A) +
					   2*(rQx-KKx)*(FC*(rPQx-rPx) + FB*(rPx-IIx)) +
					   2*(rPQx-rQx)*(FD*(rPQx-rPx) + FC*(rPx-IIx))) +
				      dil*((FC*dik)/(B+A) +
					   2*(rQx-KKx)*(FC*(rPQy-rPy) + FB*(rPy-JJy)) +
					   2*(rPQx-rQx)*(FD*(rPQy-rPy) + FC*(rPy-JJy))))/(2*(B+A)) +
				     (rQz-LLz)*((rQx-KKx)*(((A*FA-etha*FB)*dij)/(A*A) +
							   2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPQy-rPy) +
							   2*(FB*(rPQx-rPx) + FA*(rPx-IIx))*(rPy-JJy)) +
						(rPQx-rQx)*(((A*FB-etha*FC)*dij)/(A*A) +
							    2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPQy-rPy) +
							    2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPy-JJy)) +
						(FC*(djk*(rPQx-rPx) + dik*(rPQy-rPy)) +
						 FB*(djk*(rPx-IIx) + dik*(rPy-JJy)))/(B+A)) +
				     (rPQz-rQz)*((rQx-KKx)*(((A*FB-etha*FC)*dij)/(A*A) +
							    2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPQy-rPy) +
							    2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPy-JJy)) +
						 (rPQx-rQx)*(((A*FC-etha*FD)*dij)/(A*A) +
							     2*(FE*(rPQx-rPx) + FD*(rPx-IIx))*(rPQy-rPy) +
							     2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPy-JJy)) +
						 (FD*(djk*(rPQx-rPx) + dik*(rPQy-rPy)) +
						  FC*(djk*(rPx-IIx) + dik*(rPy-JJy)))/(B+A)));
	      break;
	    case 106:  // Integral (px,py|py,py)
	      preIntegral = prefact*((dkl*(((A*FA-etha*FB)*dij)/(A*A) +
					   2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPQy-rPy) +
					   2*(FB*(rPQx-rPx) + FA*(rPx-IIx))*(rPy-JJy) -
					   (etha*(((A*FB-etha*FC)*dij)/(A*A) +
						  2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPQy-rPy) +
						  2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPy-JJy)))/B))/(2*B) +
				     (djl*((FC*dik)/(B+A) +
					   2*(rQy-KKy)*(FC*(rPQx-rPx) + FB*(rPx-IIx)) +
					   2*(rPQy-rQy)*(FD*(rPQx-rPx) + FC*(rPx-IIx))) +
				      dil*((FC*dik)/(B+A) +
					   2*(rQy-KKy)*(FC*(rPQy-rPy) + FB*(rPy-JJy)) +
					   2*(rPQy-rQy)*(FD*(rPQy-rPy) + FC*(rPy-JJy))))/(2*(B+A)) +
				     (rQy-LLy)*((rQy-KKy)*(((A*FA-etha*FB)*dij)/(A*A) +
							   2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPQy-rPy) +
							   2*(FB*(rPQx-rPx) + FA*(rPx-IIx))*(rPy-JJy)) +
						(rPQy-rQy)*(((A*FB-etha*FC)*dij)/(A*A) +
							    2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPQy-rPy) +
							    2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPy-JJy)) +
						(FC*(djk*(rPQx-rPx) + dik*(rPQy-rPy)) +
						 FB*(djk*(rPx-IIx) + dik*(rPy-JJy)))/(B+A)) +
				     (rPQy-rQy)*((rQy-KKy)*(((A*FB-etha*FC)*dij)/(A*A) +
							    2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPQy-rPy) +
							    2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPy-JJy)) +
						 (rPQy-rQy)*(((A*FC-etha*FD)*dij)/(A*A) +
							     2*(FE*(rPQx-rPx) + FD*(rPx-IIx))*(rPQy-rPy) +
							     2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPy-JJy)) +
						 (FD*(djk*(rPQx-rPx) + dik*(rPQy-rPy)) +
						  FC*(djk*(rPx-IIx) + dik*(rPy-JJy)))/(B+A)));
	      break;
	    case 107:  // Integral (px,py|py,pz)
	      preIntegral = prefact*((dkl*(((A*FA-etha*FB)*dij)/(A*A) +
					   2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPQy-rPy) +
					   2*(FB*(rPQx-rPx) + FA*(rPx-IIx))*(rPy-JJy) -
					   (etha*(((A*FB-etha*FC)*dij)/(A*A) +
						  2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPQy-rPy) +
						  2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPy-JJy)))/B))/(2*B) +
				     (djl*((FC*dik)/(B+A) +
					   2*(rQy-KKy)*(FC*(rPQx-rPx) + FB*(rPx-IIx)) +
					   2*(rPQy-rQy)*(FD*(rPQx-rPx) + FC*(rPx-IIx))) +
				      dil*((FC*dik)/(B+A) +
					   2*(rQy-KKy)*(FC*(rPQy-rPy) + FB*(rPy-JJy)) +
					   2*(rPQy-rQy)*(FD*(rPQy-rPy) + FC*(rPy-JJy))))/(2*(B+A)) +
				     (rQz-LLz)*((rQy-KKy)*(((A*FA-etha*FB)*dij)/(A*A) +
							   2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPQy-rPy) +
							   2*(FB*(rPQx-rPx) + FA*(rPx-IIx))*(rPy-JJy)) +
						(rPQy-rQy)*(((A*FB-etha*FC)*dij)/(A*A) +
							    2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPQy-rPy) +
							    2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPy-JJy)) +
						(FC*(djk*(rPQx-rPx) + dik*(rPQy-rPy)) +
						 FB*(djk*(rPx-IIx) + dik*(rPy-JJy)))/(B+A)) +
				     (rPQz-rQz)*((rQy-KKy)*(((A*FB-etha*FC)*dij)/(A*A) +
							    2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPQy-rPy) +
							    2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPy-JJy)) +
						 (rPQy-rQy)*(((A*FC-etha*FD)*dij)/(A*A) +
							     2*(FE*(rPQx-rPx) + FD*(rPx-IIx))*(rPQy-rPy) +
							     2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPy-JJy)) +
						 (FD*(djk*(rPQx-rPx) + dik*(rPQy-rPy)) +
						  FC*(djk*(rPx-IIx) + dik*(rPy-JJy)))/(B+A)));
	      break;
	    case 111:  // Integral (px,py|pz,pz)
	      preIntegral = prefact*((dkl*(((A*FA-etha*FB)*dij)/(A*A) +
					   2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPQy-rPy) +
					   2*(FB*(rPQx-rPx) + FA*(rPx-IIx))*(rPy-JJy) -
					   (etha*(((A*FB-etha*FC)*dij)/(A*A) +
						  2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPQy-rPy) +
						  2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPy-JJy)))/B))/(2*B) +
				     (djl*((FC*dik)/(B+A) +
					   2*(rQz-KKz)*(FC*(rPQx-rPx) + FB*(rPx-IIx)) +
					   2*(rPQz-rQz)*(FD*(rPQx-rPx) + FC*(rPx-IIx))) +
				      dil*((FC*dik)/(B+A) +
					   2*(rQz-KKz)*(FC*(rPQy-rPy) + FB*(rPy-JJy)) +
					   2*(rPQz-rQz)*(FD*(rPQy-rPy) + FC*(rPy-JJy))))/(2*(B+A)) +
				     (rQz-LLz)*((rQz-KKz)*(((A*FA-etha*FB)*dij)/(A*A) +
							   2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPQy-rPy) +
							   2*(FB*(rPQx-rPx) + FA*(rPx-IIx))*(rPy-JJy)) +
						(rPQz-rQz)*(((A*FB-etha*FC)*dij)/(A*A) +
							    2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPQy-rPy) +
							    2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPy-JJy)) +
						(FC*(djk*(rPQx-rPx) + dik*(rPQy-rPy)) +
						 FB*(djk*(rPx-IIx) + dik*(rPy-JJy)))/(B+A)) +
				     (rPQz-rQz)*((rQz-KKz)*(((A*FB-etha*FC)*dij)/(A*A) +
							    2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPQy-rPy) +
							    2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPy-JJy)) +
						 (rPQz-rQz)*(((A*FC-etha*FD)*dij)/(A*A) +
							     2*(FE*(rPQx-rPx) + FD*(rPx-IIx))*(rPQy-rPy) +
							     2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPy-JJy)) +
						 (FD*(djk*(rPQx-rPx) + dik*(rPQy-rPy)) +
						  FC*(djk*(rPx-IIx) + dik*(rPy-JJy)))/(B+A)));
	      break;
	    case 119:  // Integral (px,pz|px,pz)
	      preIntegral = prefact*((dkl*(((A*FA-etha*FB)*dij)/(A*A) +
					   2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPQz-rPz) +
					   2*(FB*(rPQx-rPx) + FA*(rPx-IIx))*(rPz-JJz) -
					   (etha*(((A*FB-etha*FC)*dij)/(A*A) +
						  2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPQz-rPz) +
						  2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPz-JJz)))/B))/(2*B) +
				     (djl*((FC*dik)/(B+A) +
					   2*(rQx-KKx)*(FC*(rPQx-rPx) + FB*(rPx-IIx)) +
					   2*(rPQx-rQx)*(FD*(rPQx-rPx) + FC*(rPx-IIx))) +
				      dil*((FC*dik)/(B+A) +
					   2*(rQx-KKx)*(FC*(rPQz-rPz) + FB*(rPz-JJz)) +
					   2*(rPQx-rQx)*(FD*(rPQz-rPz) + FC*(rPz-JJz))))/(2*(B+A)) +
				     (rQz-LLz)*((rQx-KKx)*(((A*FA-etha*FB)*dij)/(A*A) +
							   2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPQz-rPz) +
							   2*(FB*(rPQx-rPx) + FA*(rPx-IIx))*(rPz-JJz)) +
						(rPQx-rQx)*(((A*FB-etha*FC)*dij)/(A*A) +
							    2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPQz-rPz) +
							    2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPz-JJz)) +
						(FC*(djk*(rPQx-rPx) + dik*(rPQz-rPz)) +
						 FB*(djk*(rPx-IIx) + dik*(rPz-JJz)))/(B+A)) +
				     (rPQz-rQz)*((rQx-KKx)*(((A*FB-etha*FC)*dij)/(A*A) +
							    2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPQz-rPz) +
							    2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPz-JJz)) +
						 (rPQx-rQx)*(((A*FC-etha*FD)*dij)/(A*A) +
							     2*(FE*(rPQx-rPx) + FD*(rPx-IIx))*(rPQz-rPz) +
							     2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPz-JJz)) +
						 (FD*(djk*(rPQx-rPx) + dik*(rPQz-rPz)) +
						  FC*(djk*(rPx-IIx) + dik*(rPz-JJz)))/(B+A)));
	      break;
	    case 122:  // Integral (px,pz|py,py)
	      preIntegral = prefact*((dkl*(((A*FA-etha*FB)*dij)/(A*A) +
					   2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPQz-rPz) +
					   2*(FB*(rPQx-rPx) + FA*(rPx-IIx))*(rPz-JJz) -
					   (etha*(((A*FB-etha*FC)*dij)/(A*A) +
						  2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPQz-rPz) +
						  2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPz-JJz)))/B))/(2*B) +
				     (djl*((FC*dik)/(B+A) +
					   2*(rQy-KKy)*(FC*(rPQx-rPx) + FB*(rPx-IIx)) +
					   2*(rPQy-rQy)*(FD*(rPQx-rPx) + FC*(rPx-IIx))) +
				      dil*((FC*dik)/(B+A) +
					   2*(rQy-KKy)*(FC*(rPQz-rPz) + FB*(rPz-JJz)) +
					   2*(rPQy-rQy)*(FD*(rPQz-rPz) + FC*(rPz-JJz))))/(2*(B+A)) +
				     (rQy-LLy)*((rQy-KKy)*(((A*FA-etha*FB)*dij)/(A*A) +
							   2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPQz-rPz) +
							   2*(FB*(rPQx-rPx) + FA*(rPx-IIx))*(rPz-JJz)) +
						(rPQy-rQy)*(((A*FB-etha*FC)*dij)/(A*A) +
							    2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPQz-rPz) +
							    2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPz-JJz)) +
						(FC*(djk*(rPQx-rPx) + dik*(rPQz-rPz)) +
						 FB*(djk*(rPx-IIx) + dik*(rPz-JJz)))/(B+A)) +
				     (rPQy-rQy)*((rQy-KKy)*(((A*FB-etha*FC)*dij)/(A*A) +
							    2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPQz-rPz) +
							    2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPz-JJz)) +
						 (rPQy-rQy)*(((A*FC-etha*FD)*dij)/(A*A) +
							     2*(FE*(rPQx-rPx) + FD*(rPx-IIx))*(rPQz-rPz) +
							     2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPz-JJz)) +
						 (FD*(djk*(rPQx-rPx) + dik*(rPQz-rPz)) +
						  FC*(djk*(rPx-IIx) + dik*(rPz-JJz)))/(B+A)));
	      break;
	    case 123:  // Integral (px,pz|py,pz)
	      preIntegral = prefact*((dkl*(((A*FA-etha*FB)*dij)/(A*A) +
					   2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPQz-rPz) +
					   2*(FB*(rPQx-rPx) + FA*(rPx-IIx))*(rPz-JJz) -
					   (etha*(((A*FB-etha*FC)*dij)/(A*A) +
						  2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPQz-rPz) +
						  2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPz-JJz)))/B))/(2*B) +
				     (djl*((FC*dik)/(B+A) +
					   2*(rQy-KKy)*(FC*(rPQx-rPx) + FB*(rPx-IIx)) +
					   2*(rPQy-rQy)*(FD*(rPQx-rPx) + FC*(rPx-IIx))) +
				      dil*((FC*dik)/(B+A) +
					   2*(rQy-KKy)*(FC*(rPQz-rPz) + FB*(rPz-JJz)) +
					   2*(rPQy-rQy)*(FD*(rPQz-rPz) + FC*(rPz-JJz))))/(2*(B+A)) +
				     (rQz-LLz)*((rQy-KKy)*(((A*FA-etha*FB)*dij)/(A*A) +
							   2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPQz-rPz) +
							   2*(FB*(rPQx-rPx) + FA*(rPx-IIx))*(rPz-JJz)) +
						(rPQy-rQy)*(((A*FB-etha*FC)*dij)/(A*A) +
							    2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPQz-rPz) +
							    2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPz-JJz)) +
						(FC*(djk*(rPQx-rPx) + dik*(rPQz-rPz)) +
						 FB*(djk*(rPx-IIx) + dik*(rPz-JJz)))/(B+A)) +
				     (rPQz-rQz)*((rQy-KKy)*(((A*FB-etha*FC)*dij)/(A*A) +
							    2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPQz-rPz) +
							    2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPz-JJz)) +
						 (rPQy-rQy)*(((A*FC-etha*FD)*dij)/(A*A) +
							     2*(FE*(rPQx-rPx) + FD*(rPx-IIx))*(rPQz-rPz) +
							     2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPz-JJz)) +
						 (FD*(djk*(rPQx-rPx) + dik*(rPQz-rPz)) +
						  FC*(djk*(rPx-IIx) + dik*(rPz-JJz)))/(B+A)));
	      break;
	    case 127:  // Integral (px,pz|pz,pz)
	      preIntegral = prefact*((dkl*(((A*FA-etha*FB)*dij)/(A*A) +
					   2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPQz-rPz) +
					   2*(FB*(rPQx-rPx) + FA*(rPx-IIx))*(rPz-JJz) -
					   (etha*(((A*FB-etha*FC)*dij)/(A*A) +
						  2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPQz-rPz) +
						  2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPz-JJz)))/B))/(2*B) +
				     (djl*((FC*dik)/(B+A) +
					   2*(rQz-KKz)*(FC*(rPQx-rPx) + FB*(rPx-IIx)) +
					   2*(rPQz-rQz)*(FD*(rPQx-rPx) + FC*(rPx-IIx))) +
				      dil*((FC*dik)/(B+A) +
					   2*(rQz-KKz)*(FC*(rPQz-rPz) + FB*(rPz-JJz)) +
					   2*(rPQz-rQz)*(FD*(rPQz-rPz) + FC*(rPz-JJz))))/(2*(B+A)) +
				     (rQz-LLz)*((rQz-KKz)*(((A*FA-etha*FB)*dij)/(A*A) +
							   2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPQz-rPz) +
							   2*(FB*(rPQx-rPx) + FA*(rPx-IIx))*(rPz-JJz)) +
						(rPQz-rQz)*(((A*FB-etha*FC)*dij)/(A*A) +
							    2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPQz-rPz) +
							    2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPz-JJz)) +
						(FC*(djk*(rPQx-rPx) + dik*(rPQz-rPz)) +
						 FB*(djk*(rPx-IIx) + dik*(rPz-JJz)))/(B+A)) +
				     (rPQz-rQz)*((rQz-KKz)*(((A*FB-etha*FC)*dij)/(A*A) +
							    2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPQz-rPz) +
							    2*(FC*(rPQx-rPx) + FB*(rPx-IIx))*(rPz-JJz)) +
						 (rPQz-rQz)*(((A*FC-etha*FD)*dij)/(A*A) +
							     2*(FE*(rPQx-rPx) + FD*(rPx-IIx))*(rPQz-rPz) +
							     2*(FD*(rPQx-rPx) + FC*(rPx-IIx))*(rPz-JJz)) +
						 (FD*(djk*(rPQx-rPx) + dik*(rPQz-rPz)) +
						  FC*(djk*(rPx-IIx) + dik*(rPz-JJz)))/(B+A)));
	      break;
	    case 170:  // Integral (py,py|py,py)
	      preIntegral = prefact*((dkl*(((A*FA-etha*FB)*dij)/(A*A) +
					   2*(FC*(rPQy-rPy) + FB*(rPy-IIy))*(rPQy-rPy) +
					   2*(FB*(rPQy-rPy) + FA*(rPy-IIy))*(rPy-JJy) -
					   (etha*(((A*FB-etha*FC)*dij)/(A*A) +
						  2*(FD*(rPQy-rPy) + FC*(rPy-IIy))*(rPQy-rPy) +
						  2*(FC*(rPQy-rPy) + FB*(rPy-IIy))*(rPy-JJy)))/B))/(2*B) +
				     (djl*((FC*dik)/(B+A) +
					   2*(rQy-KKy)*(FC*(rPQy-rPy) + FB*(rPy-IIy)) +
					   2*(rPQy-rQy)*(FD*(rPQy-rPy) + FC*(rPy-IIy))) +
				      dil*((FC*dik)/(B+A) +
					   2*(rQy-KKy)*(FC*(rPQy-rPy) + FB*(rPy-JJy)) +
					   2*(rPQy-rQy)*(FD*(rPQy-rPy) + FC*(rPy-JJy))))/(2*(B+A)) +
				     (rQy-LLy)*((rQy-KKy)*(((A*FA-etha*FB)*dij)/(A*A) +
							   2*(FC*(rPQy-rPy) + FB*(rPy-IIy))*(rPQy-rPy) +
							   2*(FB*(rPQy-rPy) + FA*(rPy-IIy))*(rPy-JJy)) +
						(rPQy-rQy)*(((A*FB-etha*FC)*dij)/(A*A) +
							    2*(FD*(rPQy-rPy) + FC*(rPy-IIy))*(rPQy-rPy) +
							    2*(FC*(rPQy-rPy) + FB*(rPy-IIy))*(rPy-JJy)) +
						(FC*(djk*(rPQy-rPy) + dik*(rPQy-rPy)) +
						 FB*(djk*(rPy-IIy) + dik*(rPy-JJy)))/(B+A)) +
				     (rPQy-rQy)*((rQy-KKy)*(((A*FB-etha*FC)*dij)/(A*A) +
							    2*(FD*(rPQy-rPy) + FC*(rPy-IIy))*(rPQy-rPy) +
							    2*(FC*(rPQy-rPy) + FB*(rPy-IIy))*(rPy-JJy)) +
						 (rPQy-rQy)*(((A*FC-etha*FD)*dij)/(A*A) +
							     2*(FE*(rPQy-rPy) + FD*(rPy-IIy))*(rPQy-rPy) +
							     2*(FD*(rPQy-rPy) + FC*(rPy-IIy))*(rPy-JJy)) +
						 (FD*(djk*(rPQy-rPy) + dik*(rPQy-rPy)) +
						  FC*(djk*(rPy-IIy) + dik*(rPy-JJy)))/(B+A)));
	      break;
	    case 171:  // Integral (py,py|py,pz)
	      preIntegral = prefact*((dkl*(((A*FA-etha*FB)*dij)/(A*A) +
					   2*(FC*(rPQy-rPy) + FB*(rPy-IIy))*(rPQy-rPy) +
					   2*(FB*(rPQy-rPy) + FA*(rPy-IIy))*(rPy-JJy) -
					   (etha*(((A*FB-etha*FC)*dij)/(A*A) +
						  2*(FD*(rPQy-rPy) + FC*(rPy-IIy))*(rPQy-rPy) +
						  2*(FC*(rPQy-rPy) + FB*(rPy-IIy))*(rPy-JJy)))/B))/(2*B) +
				     (djl*((FC*dik)/(B+A) +
					   2*(rQy-KKy)*(FC*(rPQy-rPy) + FB*(rPy-IIy)) +
					   2*(rPQy-rQy)*(FD*(rPQy-rPy) + FC*(rPy-IIy))) +
				      dil*((FC*dik)/(B+A) +
					   2*(rQy-KKy)*(FC*(rPQy-rPy) + FB*(rPy-JJy)) +
					   2*(rPQy-rQy)*(FD*(rPQy-rPy) + FC*(rPy-JJy))))/(2*(B+A)) +
				     (rQz-LLz)*((rQy-KKy)*(((A*FA-etha*FB)*dij)/(A*A) +
							   2*(FC*(rPQy-rPy) + FB*(rPy-IIy))*(rPQy-rPy) +
							   2*(FB*(rPQy-rPy) + FA*(rPy-IIy))*(rPy-JJy)) +
						(rPQy-rQy)*(((A*FB-etha*FC)*dij)/(A*A) +
							    2*(FD*(rPQy-rPy) + FC*(rPy-IIy))*(rPQy-rPy) +
							    2*(FC*(rPQy-rPy) + FB*(rPy-IIy))*(rPy-JJy)) +
						(FC*(djk*(rPQy-rPy) + dik*(rPQy-rPy)) +
						 FB*(djk*(rPy-IIy) + dik*(rPy-JJy)))/(B+A)) +
				     (rPQz-rQz)*((rQy-KKy)*(((A*FB-etha*FC)*dij)/(A*A) +
							    2*(FD*(rPQy-rPy) + FC*(rPy-IIy))*(rPQy-rPy) +
							    2*(FC*(rPQy-rPy) + FB*(rPy-IIy))*(rPy-JJy)) +
						 (rPQy-rQy)*(((A*FC-etha*FD)*dij)/(A*A) +
							     2*(FE*(rPQy-rPy) + FD*(rPy-IIy))*(rPQy-rPy) +
							     2*(FD*(rPQy-rPy) + FC*(rPy-IIy))*(rPy-JJy)) +
						 (FD*(djk*(rPQy-rPy) + dik*(rPQy-rPy)) +
						  FC*(djk*(rPy-IIy) + dik*(rPy-JJy)))/(B+A)));
	      break;
	    case 175:  // Integral (py,py|pz,pz)
	      preIntegral = prefact*((dkl*(((A*FA-etha*FB)*dij)/(A*A) +
					   2*(FC*(rPQy-rPy) + FB*(rPy-IIy))*(rPQy-rPy) +
					   2*(FB*(rPQy-rPy) + FA*(rPy-IIy))*(rPy-JJy) -
					   (etha*(((A*FB-etha*FC)*dij)/(A*A) +
						  2*(FD*(rPQy-rPy) + FC*(rPy-IIy))*(rPQy-rPy) +
						  2*(FC*(rPQy-rPy) + FB*(rPy-IIy))*(rPy-JJy)))/B))/(2*B) +
				     (djl*((FC*dik)/(B+A) +
					   2*(rQz-KKz)*(FC*(rPQy-rPy) + FB*(rPy-IIy)) +
					   2*(rPQz-rQz)*(FD*(rPQy-rPy) + FC*(rPy-IIy))) +
				      dil*((FC*dik)/(B+A) +
					   2*(rQz-KKz)*(FC*(rPQy-rPy) + FB*(rPy-JJy)) +
					   2*(rPQz-rQz)*(FD*(rPQy-rPy) + FC*(rPy-JJy))))/(2*(B+A)) +
				     (rQz-LLz)*((rQz-KKz)*(((A*FA-etha*FB)*dij)/(A*A) +
							   2*(FC*(rPQy-rPy) + FB*(rPy-IIy))*(rPQy-rPy) +
							   2*(FB*(rPQy-rPy) + FA*(rPy-IIy))*(rPy-JJy)) +
						(rPQz-rQz)*(((A*FB-etha*FC)*dij)/(A*A) +
							    2*(FD*(rPQy-rPy) + FC*(rPy-IIy))*(rPQy-rPy) +
							    2*(FC*(rPQy-rPy) + FB*(rPy-IIy))*(rPy-JJy)) +
						(FC*(djk*(rPQy-rPy) + dik*(rPQy-rPy)) +
						 FB*(djk*(rPy-IIy) + dik*(rPy-JJy)))/(B+A)) +
				     (rPQz-rQz)*((rQz-KKz)*(((A*FB-etha*FC)*dij)/(A*A) +
							    2*(FD*(rPQy-rPy) + FC*(rPy-IIy))*(rPQy-rPy) +
							    2*(FC*(rPQy-rPy) + FB*(rPy-IIy))*(rPy-JJy)) +
						 (rPQz-rQz)*(((A*FC-etha*FD)*dij)/(A*A) +
							     2*(FE*(rPQy-rPy) + FD*(rPy-IIy))*(rPQy-rPy) +
							     2*(FD*(rPQy-rPy) + FC*(rPy-IIy))*(rPy-JJy)) +
						 (FD*(djk*(rPQy-rPy) + dik*(rPQy-rPy)) +
						  FC*(djk*(rPy-IIy) + dik*(rPy-JJy)))/(B+A)));
	      break;
	    case 187:  // Integral (py,pz|py,pz)
	      preIntegral = prefact*((dkl*(((A*FA-etha*FB)*dij)/(A*A) +
					   2*(FC*(rPQy-rPy) + FB*(rPy-IIy))*(rPQz-rPz) +
					   2*(FB*(rPQy-rPy) + FA*(rPy-IIy))*(rPz-JJz) -
					   (etha*(((A*FB-etha*FC)*dij)/(A*A) +
						  2*(FD*(rPQy-rPy) + FC*(rPy-IIy))*(rPQz-rPz) +
						  2*(FC*(rPQy-rPy) + FB*(rPy-IIy))*(rPz-JJz)))/B))/(2*B) +
				     (djl*((FC*dik)/(B+A) +
					   2*(rQy-KKy)*(FC*(rPQy-rPy) + FB*(rPy-IIy)) +
					   2*(rPQy-rQy)*(FD*(rPQy-rPy) + FC*(rPy-IIy))) +
				      dil*((FC*dik)/(B+A) +
					   2*(rQy-KKy)*(FC*(rPQz-rPz) + FB*(rPz-JJz)) +
					   2*(rPQy-rQy)*(FD*(rPQz-rPz) + FC*(rPz-JJz))))/(2*(B+A)) +
				     (rQz-LLz)*((rQy-KKy)*(((A*FA-etha*FB)*dij)/(A*A) +
							   2*(FC*(rPQy-rPy) + FB*(rPy-IIy))*(rPQz-rPz) +
							   2*(FB*(rPQy-rPy) + FA*(rPy-IIy))*(rPz-JJz)) +
						(rPQy-rQy)*(((A*FB-etha*FC)*dij)/(A*A) +
							    2*(FD*(rPQy-rPy) + FC*(rPy-IIy))*(rPQz-rPz) +
							    2*(FC*(rPQy-rPy) + FB*(rPy-IIy))*(rPz-JJz)) +
						(FC*(djk*(rPQy-rPy) + dik*(rPQz-rPz)) +
						 FB*(djk*(rPy-IIy) + dik*(rPz-JJz)))/(B+A)) +
				     (rPQz-rQz)*((rQy-KKy)*(((A*FB-etha*FC)*dij)/(A*A) +
							    2*(FD*(rPQy-rPy) + FC*(rPy-IIy))*(rPQz-rPz) +
							    2*(FC*(rPQy-rPy) + FB*(rPy-IIy))*(rPz-JJz)) +
						 (rPQy-rQy)*(((A*FC-etha*FD)*dij)/(A*A) +
							     2*(FE*(rPQy-rPy) + FD*(rPy-IIy))*(rPQz-rPz) +
							     2*(FD*(rPQy-rPy) + FC*(rPy-IIy))*(rPz-JJz)) +
						 (FD*(djk*(rPQy-rPy) + dik*(rPQz-rPz)) +
						  FC*(djk*(rPy-IIy) + dik*(rPz-JJz)))/(B+A)));
	      break;
	    case 191:  // Integral (py,pz|pz,pz)
	      preIntegral = prefact*((dkl*(((A*FA-etha*FB)*dij)/(A*A) +
					   2*(FC*(rPQy-rPy) + FB*(rPy-IIy))*(rPQz-rPz) +
					   2*(FB*(rPQy-rPy) + FA*(rPy-IIy))*(rPz-JJz) -
					   (etha*(((A*FB-etha*FC)*dij)/(A*A) +
						  2*(FD*(rPQy-rPy) + FC*(rPy-IIy))*(rPQz-rPz) +
						  2*(FC*(rPQy-rPy) + FB*(rPy-IIy))*(rPz-JJz)))/B))/(2*B) +
				     (djl*((FC*dik)/(B+A) +
					   2*(rQz-KKz)*(FC*(rPQy-rPy) + FB*(rPy-IIy)) +
					   2*(rPQz-rQz)*(FD*(rPQy-rPy) + FC*(rPy-IIy))) +
				      dil*((FC*dik)/(B+A) +
					   2*(rQz-KKz)*(FC*(rPQz-rPz) + FB*(rPz-JJz)) +
					   2*(rPQz-rQz)*(FD*(rPQz-rPz) + FC*(rPz-JJz))))/(2*(B+A)) +
				     (rQz-LLz)*((rQz-KKz)*(((A*FA-etha*FB)*dij)/(A*A) +
							   2*(FC*(rPQy-rPy) + FB*(rPy-IIy))*(rPQz-rPz) +
							   2*(FB*(rPQy-rPy) + FA*(rPy-IIy))*(rPz-JJz)) +
						(rPQz-rQz)*(((A*FB-etha*FC)*dij)/(A*A) +
							    2*(FD*(rPQy-rPy) + FC*(rPy-IIy))*(rPQz-rPz) +
							    2*(FC*(rPQy-rPy) + FB*(rPy-IIy))*(rPz-JJz)) +
						(FC*(djk*(rPQy-rPy) + dik*(rPQz-rPz)) +
						 FB*(djk*(rPy-IIy) + dik*(rPz-JJz)))/(B+A)) +
				     (rPQz-rQz)*((rQz-KKz)*(((A*FB-etha*FC)*dij)/(A*A) +
							    2*(FD*(rPQy-rPy) + FC*(rPy-IIy))*(rPQz-rPz) +
							    2*(FC*(rPQy-rPy) + FB*(rPy-IIy))*(rPz-JJz)) +
						 (rPQz-rQz)*(((A*FC-etha*FD)*dij)/(A*A) +
							     2*(FE*(rPQy-rPy) + FD*(rPy-IIy))*(rPQz-rPz) +
							     2*(FD*(rPQy-rPy) + FC*(rPy-IIy))*(rPz-JJz)) +
						 (FD*(djk*(rPQy-rPy) + dik*(rPQz-rPz)) +
						  FC*(djk*(rPy-IIy) + dik*(rPz-JJz)))/(B+A)));
	      break;
	    case 255:  // Integral (pz,pz|pz,pz)
	      preIntegral = prefact*((dkl*(((A*FA-etha*FB)*dij)/(A*A) +
					   2*(FC*(rPQz-rPz) + FB*(rPz-IIz))*(rPQz-rPz) +
					   2*(FB*(rPQz-rPz) + FA*(rPz-IIz))*(rPz-JJz) -
					   (etha*(((A*FB-etha*FC)*dij)/(A*A) +
						  2*(FD*(rPQz-rPz) + FC*(rPz-IIz))*(rPQz-rPz) +
						  2*(FC*(rPQz-rPz) + FB*(rPz-IIz))*(rPz-JJz)))/B))/(2*B) +
				     (djl*((FC*dik)/(B+A) +
					   2*(rQz-KKz)*(FC*(rPQz-rPz) + FB*(rPz-IIz)) +
					   2*(rPQz-rQz)*(FD*(rPQz-rPz) + FC*(rPz-IIz))) +
				      dil*((FC*dik)/(B+A) +
					   2*(rQz-KKz)*(FC*(rPQz-rPz) + FB*(rPz-JJz)) +
					   2*(rPQz-rQz)*(FD*(rPQz-rPz) + FC*(rPz-JJz))))/(2*(B+A)) +
				     (rQz-LLz)*((rQz-KKz)*(((A*FA-etha*FB)*dij)/(A*A) +
							   2*(FC*(rPQz-rPz) + FB*(rPz-IIz))*(rPQz-rPz) +
							   2*(FB*(rPQz-rPz) + FA*(rPz-IIz))*(rPz-JJz)) +
						(rPQz-rQz)*(((A*FB-etha*FC)*dij)/(A*A) +
							    2*(FD*(rPQz-rPz) + FC*(rPz-IIz))*(rPQz-rPz) +
							    2*(FC*(rPQz-rPz) + FB*(rPz-IIz))*(rPz-JJz)) +
						(FC*(djk*(rPQz-rPz) + dik*(rPQz-rPz)) +
						 FB*(djk*(rPz-IIz) + dik*(rPz-JJz)))/(B+A)) +
				     (rPQz-rQz)*((rQz-KKz)*(((A*FB-etha*FC)*dij)/(A*A) +
							    2*(FD*(rPQz-rPz) + FC*(rPz-IIz))*(rPQz-rPz) +
							    2*(FC*(rPQz-rPz) + FB*(rPz-IIz))*(rPz-JJz)) +
						 (rPQz-rQz)*(((A*FC-etha*FD)*dij)/(A*A) +
							     2*(FE*(rPQz-rPz) + FD*(rPz-IIz))*(rPQz-rPz) +
							     2*(FD*(rPQz-rPz) + FC*(rPz-IIz))*(rPz-JJz)) +
						 (FD*(djk*(rPQz-rPz) + dik*(rPQz-rPz)) +
						  FC*(djk*(rPz-IIz) + dik*(rPz-JJz)))/(B+A)));
	      break;	      
	    }
	}
      // if(aa == 1 && bb == 1 && rr == 1 && ss == 2)
      // 	{
      // 	  printf("Sin Norm:  %f %f | %f %f\n",
      // 		 preIntegral, prefact, FA, rKL);
      // 	}
      normIntegral = primNormII*primNormJJ*primNormKK*primNormLL*preIntegral;
      integralValues_d[global1] = coefficientsII*coefficientsJJ*coefficientsKK*coefficientsLL*normIntegral;
    }
}

extern "C" void cuda_int_intraspecies_(int *numberOfContractions,
				       int *totalContIntegrals,
				       int *totalPrimitives,
                                       int *maxNumCartesianOrbital,
				       int *primNormalizationSize,
                                       int *contractionId,
                                       int *contractionLength,
                                       int *contractionAngularMoment,
                                       int *contractionNumCartesianOrbital,
                                       int *contractionOwner,
                                       double *contractionOrigin,
                                       double *contractionOrbitalExponents,
                                       double *contractionCoefficients,
                                       double *contractionContNormalization,
                                       double *contractionPrimNormalization,
				       double *contractionIntegrals,
				       int *contractionIndices, 
				       int *primitiveIndices,
				       int *numberOfPPUC,
				       int *labelsOfContractions)
{
  int N;
  double *integralValues, *integralValues_d;
  int a, b, r, s;
  int i,j;
  int m;
  int *contLength;
  int totalPrim;
  int contractionsMem, unicintegrals, unicintegralsMem, exponentSize;
  int *contIndices, *primIndices, *contCounter;
  double *exponents, *primNormalization, *coefficients, *origin, *contNormalization, *contractedIntegrals, *integralValuesTotal;
  int *angularMoments;
  int *numCartesianOrbitals, *labelsForContractions;
  int *auxNumberOfPPUC, contractionsMemDoub, unicintegralsMemDoub;
  int auxCounter, originSize;

  //Cuda Arrays
  int *contIndices_d, *primIndices_d, *contLength_d, *contCounter_d, *angularMoments_d;
  double *exponents_d, *primNormalization_d, *coefficients_d, *origin_d;

  // unicintegrals = ((*numberOfContractions*(*numberOfContractions+1)/2)+1)*(*numberOfContractions*(*numberOfContractions+1)/2)/2;
  unicintegrals = *totalContIntegrals;
  totalPrim = *totalPrimitives;

  //////////////////////////////////////////////////////////////////////
  /// Memory size
  contractionsMem = *numberOfContractions*sizeof(int);
  contractionsMemDoub = *numberOfContractions*sizeof(double);
  unicintegralsMem = unicintegrals*sizeof(int);
  unicintegralsMemDoub = unicintegrals*sizeof(double);
  exponentSize = *primNormalizationSize*sizeof(double);
  originSize = *numberOfContractions*3*sizeof(double);
  //////////////////////////////////////////////////////////////////////

  //////////////////////////////////////////////////////////////////////
  /// Malloc
  //contLength = Contraction size
  contLength = (int *)malloc(contractionsMem);
  // Counter for contractions
  contCounter = (int *)malloc(contractionsMem);
  //numberOfPPC = Number of Primitives per Unic Integral Contraction
  auxNumberOfPPUC = (int *)malloc(3*unicintegralsMem);
  //Unic Integral Contraction Indices
  contIndices = (int *)malloc(4*unicintegralsMem); 
  //Exponents of contractions
  exponents = (double *)malloc(exponentSize);
  //Primitive normalization constants
  primNormalization = (double *)malloc(exponentSize);
  //Coefficients of contractions
  coefficients = (double *)malloc(exponentSize);
  // Origins of contractions
  origin = (double *)malloc(originSize);
  // Contracted Integrals
  contractedIntegrals = (double *)malloc(unicintegralsMemDoub);
  // Normalization constants of contractions
  contNormalization = (double *)malloc(contractionsMemDoub);
  // Angular moments of contractions
  angularMoments = (int *)malloc(contractionsMem);
  // Number of cartesian orbitals
  numCartesianOrbitals = (int *)malloc(contractionsMem);
  // Labels of cartesian orbitals
  labelsForContractions = (int *)malloc(contractionsMem);
  //////////////////////////////////////////////////////////////////////

  auxCounter = 0;
  for(i=0; i<*numberOfContractions;i++)
    {
      contNormalization[i] = *(contractionContNormalization+i);
      angularMoments[i] = *(contractionAngularMoment+i);
      numCartesianOrbitals[i] = *(contractionNumCartesianOrbital+i);
      labelsForContractions[i] = *(labelsOfContractions+i);
      // printf("Angular moments: %d\n", angularMoments[i]);
      for(j=0; j<3; j++)
	{
	  origin[j+i*3] = *(contractionOrigin+(j+i*3));
             printf("Origin from inter %f \n",*(contractionOrigin+(j+i*3)), origin[j+i*3]);
	}
      contLength[i] = *(contractionLength+i);
      contCounter[i] = auxCounter; 
      // printf("Contraction length: %d %d\n", contLength[i], contCounter[i]);
      printf("Origins: (%f, %f, %f)\n", origin[i*3], origin[i*3+1], origin[i*3+2]);
      auxCounter += contLength[i];
    }

  // printf("Exponents, coefficients and Primitive Normalization constants:\n");
  for(i=0; i<*primNormalizationSize;i++)
      {
	exponents[i] = *(contractionOrbitalExponents+i);
	primNormalization[i] = *(contractionPrimNormalization+i);
	coefficients[i] = *(contractionCoefficients+i);
	// printf(" (%d) %f %f %f\n", i, exponents[i], coefficients[i], primNormalization[i]);
      }

  m=0;
  for( i=0; i<unicintegrals; i++ )
    {
      contIndices[i*4] = *(contractionIndices+(i*4));
      contIndices[i*4+1] = *(contractionIndices+(i*4+1));
      contIndices[i*4+2] = *(contractionIndices+(i*4+2));
      contIndices[i*4+3] = *(contractionIndices+(i*4+3));
      auxNumberOfPPUC[i*3] = *(numberOfPPUC+(i*3));
      auxNumberOfPPUC[i*3+1] = *(numberOfPPUC+(i*3+1));
      auxNumberOfPPUC[i*3+2] = *(numberOfPPUC+(i*3+2));
      /* printf("Contraction num: %d (%d,%d|%d,%d)\n", i, contIndices[i*4], contIndices[i*4+1], contIndices[i*4+2], contIndices[i*4+3]); */
    }

  primIndices = (int *)malloc(totalPrim*9*sizeof(int));
  for( i=0; i<totalPrim; i++)
    {
      primIndices[i*9] = *(primitiveIndices+(i*9));
      primIndices[i*9+1] = *(primitiveIndices+(i*9+1));
      primIndices[i*9+2] = *(primitiveIndices+(i*9+2));
      primIndices[i*9+3] = *(primitiveIndices+(i*9+3));
      primIndices[i*9+4] = *(primitiveIndices+(i*9+4));
      primIndices[i*9+5] = *(primitiveIndices+(i*9+5));
      primIndices[i*9+6] = *(primitiveIndices+(i*9+6));
      primIndices[i*9+7] = *(primitiveIndices+(i*9+7));
      primIndices[i*9+8] = *(primitiveIndices+(i*9+8));
    }

  N=totalPrim;	  
  integralValuesTotal = (double *)malloc(N*sizeof(double));
  ////////////////////////////////////////////////////////////////////
  /// Total threads in GPUs
  // printf("     *** GPU Especifications ***\n");
  int gpu, count;
  hipDeviceProp_t prop;
  hipGetDeviceCount(&count);
  int totalThreads=0;
  for (gpu = 0; gpu < count; gpu++) {
    hipGetDeviceProperties(&prop,gpu);
    totalThreads+=prop.multiProcessorCount*prop.maxThreadsPerMultiProcessor;
  }
  ////////////////////////////////////////////////////////////////////   
  int numberOfBlocks = totalThreads/numberOfThreads;
  dim3 blockSize(16,16,1);
  dim3 gridSize(numberOfBlocks,1,1);

  ////////////////////////////////////////////////////////////////////////////
  /// CUDA Malloc
  hipMalloc((void **)&primIndices_d, totalPrim*9*sizeof(int));
  hipMalloc((void **)&contIndices_d, 4*unicintegralsMem);
  hipMalloc((void **)&exponents_d, exponentSize);
  hipMalloc((void **)&primNormalization_d, exponentSize);
  hipMalloc((void **)&coefficients_d, exponentSize);
  hipMalloc((void **)&contCounter_d, contractionsMem);
  hipMalloc((void **)&angularMoments_d, contractionsMem);
  hipMalloc((void **)&contLength_d, contractionsMem);
  hipMalloc((void **)&origin_d, originSize);
  ///////////////////////////////////////////////////////////////////////////

  ///////////////////////////////////////////////////////////////////////////
  ///CUDA copy
  hipMemcpy(primIndices_d, primIndices, totalPrim*9*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(contIndices_d, contIndices, 4*unicintegralsMem, hipMemcpyHostToDevice);
  hipMemcpy(exponents_d, exponents, exponentSize, hipMemcpyHostToDevice);
  hipMemcpy(primNormalization_d, primNormalization, exponentSize, hipMemcpyHostToDevice);
  hipMemcpy(coefficients_d, coefficients, exponentSize, hipMemcpyHostToDevice);
  hipMemcpy(contCounter_d, contCounter, contractionsMem, hipMemcpyHostToDevice);
  hipMemcpy(angularMoments_d, angularMoments, contractionsMem, hipMemcpyHostToDevice);
  hipMemcpy(contLength_d, contLength, contractionsMem, hipMemcpyHostToDevice);
  hipMemcpy(origin_d, origin, originSize, hipMemcpyHostToDevice);
  //////////////////////////////////////////////////////////////////////////

  ////////////////////////////////////////////////////////////////////////
  ///Number of Calls to kernel

  int numberCallkernel = 0;
  
  i=0;
  int kernelIter = 0;
  int control2=0;
  while(control2<=totalPrim-1)
    {
      int control = 0;
      kernelIter = control2;
      while(control+auxNumberOfPPUC[i*3]<=totalThreads && i < unicintegrals)
	{
	  control += auxNumberOfPPUC[i*3];
          control2 += auxNumberOfPPUC[i*3];
	  i++;
	  // printf("Control: %d %d\n",i, control);
	}
      numberCallkernel++;
      integralValues = (double *)malloc(control*sizeof(double));
      hipMalloc((void **)&integralValues_d, control*sizeof(double));

      // printf("Control2: %d %d\n", numberCallkernel, control2);

           // printf("Kernel Call Number: %d\n", numberCallkernel );
      analyticInts<<<gridSize,blockSize>>>(N, primIndices_d, contIndices_d, exponents_d, primNormalization_d, coefficients_d, contCounter_d, contLength_d, origin_d, angularMoments_d, integralValues_d, control, kernelIter);

      hipMemcpy(integralValues, integralValues_d, control*sizeof(double),hipMemcpyDeviceToHost);

       for(j=kernelIter;j<control2;j++)
	{
	  integralValuesTotal[j] = integralValues[j-kernelIter];    
	  // if(numberCallkernel==3)
	     // printf("Integral post Kernel: %d, %d -> %f\n", j, j-kernelIter, integralValuesTotal[j]);
	}

      hipFree(integralValues_d);
      free(integralValues);
    }

  m=0;
  // printf("Unic Integrals Cuda:%d\n", unicintegrals);
  for(i=0; i<unicintegrals;i++)
    {
      contractedIntegrals[i] = 0.0;
      a = contIndices[i*4];
      b = contIndices[i*4+1];
      r = contIndices[i*4+2];
      s = contIndices[i*4+3];
      for(j=0; j<auxNumberOfPPUC[i*3];j++)
	{
	  contractedIntegrals[i] += contNormalization[a-1]*contNormalization[b-1]*contNormalization[r-1]*contNormalization[s-1]*integralValuesTotal[m];
	  *(contractionIntegrals+i) = contractedIntegrals[i];
	  m++;
	}
      // printf("%d %f %f %f %f\n", i, contNormalization[a],contNormalization[b],contNormalization[r],contNormalization[s]);
      printf("Contraida numero: %3d (%d,%d|%d,%d) | %15.12f \n", i,a,b,r,s,contractedIntegrals[i]);
    }

  hipFree(primIndices_d);
  hipFree(contIndices_d);
  hipFree(exponents_d);
  hipFree(primNormalization_d);
  hipFree(coefficients_d);
  hipFree(contCounter_d);
  hipFree(contLength_d);
  hipFree(origin_d);
  free(integralValuesTotal);
  free(contLength);
  free(contCounter);
  free(auxNumberOfPPUC);
  free(contIndices);
  free(exponents);
  free(primNormalization);
  free(coefficients);
  free(origin);
  free(contractedIntegrals);
  free(contNormalization);

  return;
}

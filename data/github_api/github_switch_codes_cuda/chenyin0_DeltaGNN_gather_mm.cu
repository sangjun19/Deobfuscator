#include "hip/hip_runtime.h"
// Repository: chenyin0/DeltaGNN
// File: src/array/cuda/gather_mm.cu

/*!
 *  Copyright (c) 2020 by Contributors
 * \file array/cuda/gather_mm.cu
 * \brief GatherMM C APIs and definitions.
 */
#include <dgl/array.h>
#include <algorithm>  // std::swap
#include "./utils.h"
#include "./functor.cuh"
#include "./atomic.cuh"

namespace dgl {
using namespace cuda;
namespace aten {

namespace {

/*! \brief Call cuBLAS geam API for transpose operation for float and double. */
template <typename DType>
hipblasStatus_t Xgeam(hipblasHandle_t handle, hipblasOperation_t transa,
    hipblasOperation_t transb, int m, int n,
    const DType* alpha, const DType* A, int lda,
    const DType* beta, const DType* B, int ldb,
    DType* C, int ldc) {
  LOG(INFO) << "Not supported dtype";
  return HIPBLAS_STATUS_EXECUTION_FAILED;
}

template <>
hipblasStatus_t Xgeam<float>(hipblasHandle_t handle, hipblasOperation_t transa,
    hipblasOperation_t transb, int m, int n,
    const float* alpha, const float* A, int lda,
    const float* beta, const float* B, int ldb,
    float* C, int ldc) {
  return hipblasSgeam(handle, transa, transb, m, n, alpha, A, lda,
      beta, B, ldb, C, ldc);
}

template <>
hipblasStatus_t Xgeam<double>(hipblasHandle_t handle, hipblasOperation_t transa,
    hipblasOperation_t transb, int m, int n,
    const double* alpha, const double* A, int lda,
    const double* beta, const double* B, int ldb,
    double* C, int ldc) {
  return hipblasDgeam(handle, transa, transb, m, n, alpha, A, lda,
      beta, B, ldb, C, ldc);
}

/*! \brief Call cuBLAS GEMM API for dense matmul operation for float and double. */
template <typename DType>
hipblasStatus_t cublasGemm(hipblasHandle_t handle, hipblasOperation_t transa,
    hipblasOperation_t transb, int m, int n, int k,
    const DType* alpha, const DType* A, int lda,
    const DType* B, int ldb, const DType* beta,
    DType* C, int ldc) {
  LOG(INFO) << "Not supported dtype";
  return HIPBLAS_STATUS_EXECUTION_FAILED;
}

template <>
hipblasStatus_t cublasGemm<float>(hipblasHandle_t handle, hipblasOperation_t transa,
    hipblasOperation_t transb, int m, int n, int k,
    const float* alpha, const float* A, int lda,
    const float* B, int ldb, const float* beta,
    float* C, int ldc) {
  return hipblasSgemm(handle, transa, transb, m, n, k, alpha, A, lda,
      B, ldb, beta, C, ldc);
}

template <>
hipblasStatus_t cublasGemm<double>(hipblasHandle_t handle, hipblasOperation_t transa,
    hipblasOperation_t transb, int m, int n, int k,
    const double* alpha, const double* A, int lda,
    const double* B, int ldb, const double* beta,
    double* C, int ldc) {
  return hipblasDgemm(handle, transa, transb, m, n, k, alpha, A, lda,
      B, ldb, beta, C, ldc);
}

/*
 * \brief Tranpose the input matrix.
 * \param row number of rows of input matrix.
 * \param col number of columns of input matrix.
 */
template <typename DType>
void _Transpose(hipblasHandle_t handle,
                const DType* in, DType* out,
                int row, int col) {
  DType alpha = 1., beta = 0.;
  CUBLAS_CALL(Xgeam<DType>(
      handle,
      HIPBLAS_OP_T,
      HIPBLAS_OP_N,
      row, col,
      &alpha, in, col,
      &beta, nullptr, row,
      out, row));
}

}  // namespace

namespace cuda {

/* \Note Each row of A multiplies a segment of matrix of B of dimension in_len * outlen.
  One warp is assigned to process one row of A. Each WARP sequentially multiplies
  one element of A and a row of B to compute partial result of the output. A
  is loaded in shared memory in a coalesced way. Output matrix is loaded in
  registers. B should get benefit from L2 cache.
*/
template <typename Idx, typename DType>
__global__ void gatherMMKernel(
    const DType* __restrict__ A,
    const DType* __restrict__ B,
    DType* __restrict__ C,
    const Idx* __restrict__ idx_a,
    const Idx* __restrict__ idx_b,
    int64_t num_rows,
    int64_t in_len, int64_t out_len) {
    unsigned int tId = threadIdx.x;
    unsigned int laneId = tId & 31;
    unsigned int gId = (blockIdx.x * blockDim.x + threadIdx.x);
    unsigned int warpId = gId >> 5;
    unsigned int row = warpId;
    if (row < num_rows) {
        unsigned int local_row = row & 3;  // hardcoded for TB size 128 (4 warps)
        Idx cur_rowA = (idx_a) ? idx_a[row] : row;
        Idx cur_rowB = (idx_b) ? idx_b[row] : row / in_len;
        Idx B_offset = cur_rowB * in_len * out_len;
        const int sh_a_tile = 64;
        __shared__ DType sh_A[4 * sh_a_tile];
        int a_tile = sh_a_tile;
        for (unsigned int k_start = 0; k_start < in_len; k_start += 64) {
            if ((in_len - k_start) < a_tile) a_tile = in_len - k_start;
            /* Load A in shared mem in a coalesced way */
            for (unsigned int l = laneId; l < a_tile; l += 32)
                sh_A[local_row * sh_a_tile + l] = A[cur_rowA * in_len + (k_start + l)];
            __syncwarp();

            for (unsigned int outloop = 0; outloop < out_len; outloop +=32) {
                DType out_reg = 0;  // thread private
                const unsigned int l = laneId;
                if (l < out_len) {
                    /* iterate over elements of a row of A */
                    for (unsigned int i = 0; i < a_tile; i++) {
                        const DType a_val =  sh_A[local_row * sh_a_tile + i];
                        /* iterate over elements of a row of B in parallel */
                        out_reg += a_val * B[B_offset + ((i + k_start) * out_len + (outloop + l))];
                    }
                    C[row * out_len + (outloop + l)] += out_reg;
                }
            }
        }
    }
}

/* \Note Output matrix is accumulated via atomic operations. Rest of the strategies
  are similar to gatherMMKernel. One warp is assigned to process one row of A. Each
  WARP sequentially multiplies one element of A and a row of B to compute partial
  result of the output. A is loaded in shared memory in a coalesced way. B should
  get benefit from L2 cache.
*/
template <typename Idx, typename DType>
__global__ void gatherMMScatterKernel(
    const DType* __restrict__ A,
    const DType* __restrict__ B,
    DType* __restrict__ C,
    const Idx* __restrict__ idx_a,
    const Idx* __restrict__ idx_b,
    const Idx* __restrict__ idx_c,
    int64_t num_rows,
    int64_t in_len, int64_t out_len) {
    unsigned int tId = threadIdx.x;
    unsigned int laneId = tId & 31;
    unsigned int gId = (blockIdx.x * blockDim.x + threadIdx.x);
    unsigned int warpId = gId >> 5;
    unsigned int row = warpId;
    if (row < num_rows) {
        unsigned int local_row = row & 3;  // hardcoded for TB size 128 (4 warps)
        unsigned int row_a = (idx_a) ? idx_a[row] : row;
        unsigned int row_b = (idx_b) ? idx_b[row] : row;
        Idx C_offset = (idx_c) ? idx_c[row] * in_len * out_len : 0;
        const int sh_a_tile = 64;
        __shared__ DType sh_A[4 * sh_a_tile];
        int a_tile = sh_a_tile;
        for (unsigned int k_start = 0; k_start < in_len; k_start += 64) {
            if ((in_len - k_start) < a_tile) a_tile = in_len - k_start;
            /* Load A in shared mem in a coalesced way */
            for (unsigned int l = laneId; l < a_tile; l += 32)
                sh_A[local_row * sh_a_tile + l] = A[row_a * in_len + (k_start + l)];
            __syncwarp();

            for (unsigned int outloop = 0; outloop < out_len; outloop +=32) {
                DType out_reg = 0;  // thread private
                const unsigned int l = laneId;
                if (l < out_len) {
                    const DType b_val = B[row_b * out_len + (outloop + l)];
                    /* iterate over elements of a row of A */
                    for (unsigned int i = 0; i < a_tile; i++) {
                        const DType a_val = sh_A[local_row * sh_a_tile + i];
                        const Idx C_idx = C_offset + ((i + k_start) * out_len + (outloop + l));
                        atomicAdd(reinterpret_cast<float*>(&C[C_idx]),
                            static_cast<float>(a_val * b_val));
                    }
                }
            }
        }
    }
}


/* \brief Implementation of GatherMM operator. The indices of A (or B)
 * are looked up from idx_a (or idx_b) when defined.
 */
template <int XPU, typename IdType, int bits>
void gatherMM(const NDArray A,
              const NDArray B,
              NDArray C,
              const NDArray idx_a,
              const NDArray idx_b,
              int64_t num_rel) {
    SWITCH_BITS(bits, DType, {
        auto device = runtime::DeviceAPI::Get(A->ctx);
        auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
        const DType *A_data = A.Ptr<DType>();
        const DType *B_data = B.Ptr<DType>();
        int64_t out_len = B->shape[1];  // cols of B
        int64_t in_len = A->shape[1];  // cols of A
        if (!thr_entry->cublas_handle)
            CUBLAS_CALL(hipblasCreate(&(thr_entry->cublas_handle)));
        CUBLAS_CALL(hipblasSetStream(thr_entry->cublas_handle,
            thr_entry->stream));
        int64_t tot_num_rows = A->shape[0];
        const int ntx = 128;
        const int warp_size = 32;
        const int nbx =  ((tot_num_rows * warp_size + ntx - 1) / ntx);
        const dim3 nblks(nbx);
        const dim3 nthrs(ntx);
        CUDA_KERNEL_CALL((gatherMMKernel<IdType, DType>),
            nblks, nthrs, 0, thr_entry->stream,
            static_cast<DType*>(A->data),
            static_cast<DType*>(B->data),
            static_cast<DType*>(C->data),
            static_cast<IdType*>(idx_a->data),
            static_cast<IdType*>(idx_b->data),
            tot_num_rows,
            in_len, out_len);
    });
}

/* \brief Implementation of GatherMM operator. The indices of A (or B or C)
 * are looked up from idx_a (or idx_b or idx_c) when defined.
 */
template <int XPU, typename IdType, int bits>
void gatherMM_scatter(const NDArray A,
              const NDArray B,
              NDArray C,
              const NDArray idx_a,
              const NDArray idx_b,
              const NDArray idx_c,
              int num_rel, bool a_trans, bool b_trans) {
    SWITCH_BITS(bits, DType, {
        auto device = runtime::DeviceAPI::Get(A->ctx);
        auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
        const IdType *idx_c_data = idx_c.Ptr<IdType>();
        int64_t out_len = B->shape[1];  // cols of B
        int64_t in_len = A->shape[1];  // cols of A
        if (!thr_entry->cublas_handle)
            CUBLAS_CALL(hipblasCreate(&(thr_entry->cublas_handle)));
        CUBLAS_CALL(hipblasSetStream(thr_entry->cublas_handle,
            thr_entry->stream));
        DType* B_trans_data = nullptr;
        if (b_trans) {
            int64_t B_offset = 0;
            const DType *B_data = B.Ptr<DType>();
            in_len = B->shape[0]/num_rel;
            B_trans_data = static_cast<DType*>(device->AllocWorkspace \
                (B->ctx, B->shape[0] * B->shape[1] * sizeof(DType)));
            // tranpose B per relation
            for (int rel = 0; rel < num_rel; ++rel) {
                _Transpose(thr_entry->cublas_handle, B_data + B_offset,
                    B_trans_data + B_offset, in_len, out_len);
                B_offset += in_len * out_len;
            }
            std::swap(in_len, out_len);
        }
        int64_t tot_num_rows = A->shape[0];
        const int ntx = 128;
        const int warp_size = 32;
        const int nbx =  ((tot_num_rows * warp_size + ntx - 1) / ntx);
        const dim3 nblks(nbx);
        const dim3 nthrs(ntx);

        if (idx_c_data) {
            // Custom kernel for W_grad[idx_c[i]] = H^T[i] * C.grad[i]
            // This kernel accesses rows of A in a transposed way w/o explicitly converting A
            CUDA_KERNEL_CALL((gatherMMScatterKernel<IdType, DType>),
                nblks, nthrs, 0, thr_entry->stream,
                static_cast<DType*>(A->data),
                static_cast<DType*>(B->data),
                static_cast<DType*>(C->data),
                static_cast<IdType*>(idx_a->data),
                static_cast<IdType*>(idx_b->data),
                static_cast<IdType*>(idx_c->data),
                tot_num_rows,
                in_len, out_len);
        } else {  // use generic gather_mm
                CUDA_KERNEL_CALL((gatherMMKernel<IdType, DType>),
                    nblks, nthrs, 0, thr_entry->stream,
                    static_cast<DType*>(A->data),
                    (b_trans) ? B_trans_data : static_cast<DType*>(B->data),
                    static_cast<DType*>(C->data),
                    static_cast<IdType*>(idx_a->data),
                    static_cast<IdType*>(idx_b->data),
                    tot_num_rows,
                    in_len, out_len);
        }
        if (b_trans)
            device->FreeWorkspace(B->ctx, B_trans_data);
    });
}

}  // namespace cuda

/* \brief Implementation of SegmentMM operator. Each segment calls cuBLAS
 * GEMM operator to multiply segment of A and B. When A or B needs to be
 * tranposed, cuBLAS GEMM switches it's transpose parameter (HIPBLAS_OP_T).
 */
template <int XPU, typename IdType, int bits>
void segment_mm(const NDArray A,
              const NDArray B,
              NDArray C,
              const NDArray seglen_A,
              bool a_trans, bool b_trans) {
    SWITCH_BITS(bits, DType, {
        auto device = runtime::DeviceAPI::Get(A->ctx);
        const DType *A_data = A.Ptr<DType>();
        const DType *B_data = B.Ptr<DType>();
        const IdType* seglen_A_data = seglen_A.Ptr<IdType>();
        DType *C_data = C.Ptr<DType>();
        int64_t A_offset = 0, B_offset = 0, C_offset = 0;
        int64_t m, n, k;
        int64_t num_rel = seglen_A.NumElements();
        DType alpha = 1., beta = 0.;

        auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
        if (!thr_entry->cublas_handle)
            CUBLAS_CALL(hipblasCreate(&(thr_entry->cublas_handle)));
        CUBLAS_CALL(hipblasSetStream(thr_entry->cublas_handle,
            thr_entry->stream));

        for (int etype = 0; etype < num_rel; ++etype) {
            IdType B_dim1 = B->shape[0] / num_rel;
            assert((a_trans) ? seglen_A_data[etype] : A->shape[1] ==  \
                (b_trans) ? B->shape[1] : B_dim1);
            m = seglen_A_data[etype];  // rows of A
            n = B->shape[1];  // cols of B
            k = A->shape[1];  // cols of A == rows of B
            int ldb = n, lda = k, ldc = n;
            hipblasOperation_t transB = HIPBLAS_OP_N;
            hipblasOperation_t transA = HIPBLAS_OP_N;
            if (a_trans) {
                transA = HIPBLAS_OP_T;
                ldb = n, lda = k, ldc = n;
                std::swap(m, k);
            }
            if (b_trans) {
                transB = HIPBLAS_OP_T;
                k = B_dim1;
                ldb = n, lda = n, ldc = k;
                std::swap(n, k);
            }
            CUBLAS_CALL(cublasGemm<DType>(
                thr_entry->cublas_handle,
                transB,
                transA,
                n, m, k,
                &alpha,
                B_data + B_offset, ldb,
                A_data + A_offset, lda,
                &beta,
                C_data + C_offset, ldc));
            A_offset += m * k;
            B_offset += k * n;
            C_offset += m * n;
        }
    });
}

/*!
 * \brief Implementation of Gather_mm operator. The input matrix A is
 *        expected to be sorted according to relation type.
 * \param A The input dense matrix of dimension m x k
 * \param B The input dense matrix of dimension k x n
 * \param C The output dense matrix of dimension m x n
 * \param seglen_A The input vector of size R. Each element
 *        is the length of segments of input ``A``
 * \param a_trans Matrix A to be transposed
 * \param b_trans Matrix B to be transposed
 */
template <int XPU, typename IdType, int bits>
void segmentMM(const NDArray A,
          const NDArray B,
          NDArray C,
          const NDArray seglen_A,
          bool a_trans, bool b_trans) {
    segment_mm<XPU, IdType, bits>(A, B, C, seglen_A, a_trans, b_trans);
}

/*!
 * \brief Implementation of Gather_mm operator. The input matrix A is
 *        expected to be sorted according to relation type.
 * \param A The input dense matrix of dimension m x k
 * \param B The input dense matrix of dimension k x n
 * \param C The output dense matrix of dimension m x n
 * \param idx_a The input vector to gather left hand operand on
 * \param idx_b The input vector to gather right hand operand on
 * \param num_rel The number of idx types in idx_b
 */
template <int XPU, typename IdType, int bits>
void gatherMM(const NDArray A,
          const NDArray B,
          NDArray C,
          const NDArray idx_a,
          const NDArray idx_b,
          const int num_rel) {
    cuda::gatherMM<XPU, IdType, bits>(A, B, C, idx_a, idx_b, num_rel);
}

/*!
 * \brief Implementation of Gather_mm operator. The input matrix A is
 *        expected to be sorted according to relation type.
 * \param A The input dense matrix of dimension m x k
 * \param B The input dense matrix of dimension k x n
 * \param C The output dense matrix of dimension m x n
 * \param idx_a The input vector to gather left hand operand on
 * \param idx_b The input vector to gather right hand operand on
 * \param idx_c The input vector to gather output operand on
 * \param num_rel The number of idx types in idx_b
 * \param a_trans Matrix A to be transposed
 * \param b_trans Matrix B to be transposed
 */
template <int XPU, typename IdType, int bits>
void gatherMM_scatter(const NDArray A,
          const NDArray B,
          NDArray C,
          const NDArray idx_a,
          const NDArray idx_b,
          const NDArray idx_c,
          const int num_rel,
          bool a_trans, bool b_trans) {
    cuda::gatherMM_scatter<XPU, IdType, bits>(A, B, C, idx_a, idx_b, idx_c,
        num_rel, a_trans, b_trans);
}


template void gatherMM<kDLGPU, int32_t, 16>(
    const NDArray A, const NDArray B, NDArray C,
    const NDArray idx_a, const NDArray idx_b, const int num_rel);
template void gatherMM<kDLGPU, int64_t, 16>(
    const NDArray A, const NDArray B, NDArray C,
    const NDArray idx_a, const NDArray idx_b, const int num_rel);
template void gatherMM<kDLGPU, int32_t, 32>(
    const NDArray A, const NDArray B, NDArray C,
    const NDArray idx_a, const NDArray idx_b, const int num_rel);
template void gatherMM<kDLGPU, int64_t, 32>(
    const NDArray A, const NDArray B, NDArray C,
    const NDArray idx_a, const NDArray idx_b, const int num_rel);
template void gatherMM<kDLGPU, int32_t, 64>(
    const NDArray A, const NDArray B, NDArray C,
    const NDArray idx_a, const NDArray idx_b, const int num_rel);
template void gatherMM<kDLGPU, int64_t, 64>(
    const NDArray A, const NDArray B, NDArray C,
    const NDArray idx_a, const NDArray idx_b, const int num_rel);

template void gatherMM_scatter<kDLGPU, int32_t, 16>(
    const NDArray A, const NDArray B, NDArray C,
    const NDArray idx_a, const NDArray idx_b, const NDArray idx_c,
    const int num_rel, bool a_trans, bool b_trans);
template void gatherMM_scatter<kDLGPU, int64_t, 16>(
    const NDArray A, const NDArray B, NDArray C,
    const NDArray idx_a, const NDArray idx_b, const NDArray idx_c,
    const int num_rel, bool a_trans, bool b_trans);
template void gatherMM_scatter<kDLGPU, int32_t, 32>(
    const NDArray A, const NDArray B, NDArray C,
    const NDArray idx_a, const NDArray idx_b, const NDArray idx_c,
    const int num_rel, bool a_trans, bool b_trans);
template void gatherMM_scatter<kDLGPU, int64_t, 32>(
    const NDArray A, const NDArray B, NDArray C,
    const NDArray idx_a, const NDArray idx_b, const NDArray idx_c,
    const int num_rel, bool a_trans, bool b_trans);
template void gatherMM_scatter<kDLGPU, int32_t, 64>(
    const NDArray A, const NDArray B, NDArray C,
    const NDArray idx_a, const NDArray idx_b, const NDArray idx_c,
    const int num_rel, bool a_trans, bool b_trans);
template void gatherMM_scatter<kDLGPU, int64_t, 64>(
    const NDArray A, const NDArray B, NDArray C,
    const NDArray idx_a, const NDArray idx_b, const NDArray idx_c,
    const int num_rel, bool a_trans, bool b_trans);

template void segmentMM<kDLGPU, int32_t, 16>(
    const NDArray A, const NDArray B, NDArray C,
    const NDArray seglen_A, bool a_trans, bool b_trans);
template void segmentMM<kDLGPU, int64_t, 16>(
    const NDArray A, const NDArray B, NDArray C,
    const NDArray seglen_A, bool a_trans, bool b_trans);
template void segmentMM<kDLGPU, int32_t, 32>(
    const NDArray A, const NDArray B, NDArray C,
    const NDArray seglen_A, bool a_trans, bool b_trans);
template void segmentMM<kDLGPU, int64_t, 32>(
    const NDArray A, const NDArray B, NDArray C,
    const NDArray seglen_A, bool a_trans, bool b_trans);
template void segmentMM<kDLGPU, int32_t, 64>(
    const NDArray A, const NDArray B, NDArray C,
    const NDArray seglen_A, bool a_trans, bool b_trans);
template void segmentMM<kDLGPU, int64_t, 64>(
    const NDArray A, const NDArray B, NDArray C,
    const NDArray seglen_A, bool a_trans, bool b_trans);

}  // namespace aten
}  // namespace dgl

#include "hip/hip_runtime.h"
// Repository: JeffOwOSun/gpu-bm3d
// File: bm3d.cu

#include "bm3d.h"

/*
 * Read-only variables for all cuda kernels. These variables
 * will be stored in the "constant" memory on GPU for fast read.
 */
__constant__ GlobalConstants cu_const_params;

#include "block_matching.cu_inl"
#include "aggregation.cu_inl"




////////////////////////////////////////////////////////////////////////////////////////
// Putting all the cuda kernels here
///////////////////////////////////////////////////////////////////////////////////////
__device__ float norm2(hipComplex & a) {
    return (a.x * a.x) + (a.y * a.y);
}

__global__ void real2complex(uchar* h_data, hipfftComplex *output) {
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    int j = threadIdx.y + blockIdx.y*blockDim.y;
    int index = j*cu_const_params.image_width + i;

    if (i<cu_const_params.image_width && j<cu_const_params.image_height) {
        output[index].x = h_data[index];
        output[index].y = 0.0f;
    }
}

__global__ void complex2real(hipfftComplex *data, float* output, int total_size, int trans_size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index >= total_size) {
        return;
    }
    output[index] = data[index].x / (float)(trans_size);
}

/*
 *  normalize cufft inverse result by dividing number of elements per batch
 */
__global__ void normalize(hipfftComplex *data, int size) {
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    int j = threadIdx.y + blockIdx.y*blockDim.y;
    int index = idx2(i, j, cu_const_params.image_width);
    data[index].x = data[index].x / (float)(size);
    data[index].y = data[index].y / (float)(size);
}

/*
 * taking d_rearrange_stacks and perform thresholding. Count number of non zeros
 * Also will normalize the 1D transform result.
 */
__global__ void hard_filter(hipfftComplex *d_transformed_stacks, float *d_weight) {
    int group_id = threadIdx.x + blockIdx.x * blockDim.x;
    if (group_id >= cu_const_params.total_ref_patches) {
        return;
    }
    int patch_size = cu_const_params.patch_size;
    int non_zero = 0;
    float threshold = cu_const_params.lambda_3d * cu_const_params.lambda_3d *
                      cu_const_params.sigma * cu_const_params.sigma * patch_size * patch_size * cu_const_params.max_group_size;
    // printf("Threshold %f\n", threshold);
    int offset = group_id*cu_const_params.max_group_size * patch_size * patch_size;

    float x, y, val;
    for (int i=0; i<patch_size*patch_size*cu_const_params.max_group_size;i++) {
        x = d_transformed_stacks[offset + i].x;
        y = d_transformed_stacks[offset + i].y;
        val = x*x + y*y;
        if (val < threshold) {
            // printf("below threshold\n");
            x = 0.0f;
            y = 0.0f;
        } else {
            ++non_zero;
        }
        d_transformed_stacks[offset + i].x = x;
        d_transformed_stacks[offset + i].y = y;
    }
    d_weight[group_id] = 1.0f / (float)non_zero;
}


__global__ void get_wiener_coef(hipfftComplex *d_transformed_stacks, float *d_wien_coef) {
    int group_id = threadIdx.x + blockIdx.x * blockDim.x;
    if (group_id >= cu_const_params.total_ref_patches) {
        return;
    }
    const int patch_size = cu_const_params.patch_size;
    const int sigma = cu_const_params.sigma;
    const int norm_fator = patch_size * patch_size * cu_const_params.max_group_size;
    int offset = group_id*cu_const_params.max_group_size * patch_size * patch_size;

    float val;
    for (int i=0; i<patch_size*patch_size*cu_const_params.max_group_size;i++) {
        val = norm2(d_transformed_stacks[offset + i]) / (float)norm_fator;
        d_wien_coef[offset + i] = val / (val + sigma * sigma);
    }
}

__global__ void apply_wiener_coef(hipfftComplex *d_transformed_stacks, float *d_wien_coef, float *d_wien_weight) {
    int group_id = threadIdx.x + blockIdx.x * blockDim.x;
    if (group_id >= cu_const_params.total_ref_patches) {
        return;
    }
    const int patch_size = cu_const_params.patch_size;
    int offset = group_id*cu_const_params.max_group_size * patch_size * patch_size;
    float wien_acc = 0.0f;
    for (int i=0; i<patch_size*patch_size*cu_const_params.max_group_size;i++) {
        float wien = d_wien_coef[offset+i];
        d_transformed_stacks[offset + i].x *= wien;
        d_transformed_stacks[offset + i].y *= wien;
        wien_acc += wien * wien;
    }
    d_wien_weight[group_id] = 1.0f / wien_acc;
}

/*
 *  Each thread maps to a group, d_transformed_stacks is organized as (w, h, patch in group)
 */
__global__ void fill_patch_major_from_source(Q* d_stacks, uint* d_num_patches_in_stack, uchar* input_data, hipfftComplex* d_transformed_stacks) {
    int group_id = threadIdx.x + blockIdx.x * blockDim.x;
    if (group_id >= cu_const_params.total_ref_patches) {
        return;
    }
    int width = cu_const_params.image_width;
    int patch_size = cu_const_params.patch_size;

    // start patch num
    int start = group_id*cu_const_params.max_group_size;
    int offset = start * patch_size * patch_size;

    for (int z=0;z<d_num_patches_in_stack[group_id];z++) {
        // fill in the actual data
        uint patch_x = d_stacks[z+start].position.x;
        uint patch_y = d_stacks[z+start].position.y;
        for (int k=0;k<patch_size*patch_size;k++) {
            int index = idx2(patch_x + (k%patch_size), patch_y + (k/patch_size), width);
            int output_index = idx2(k, z, patch_size*patch_size);
            d_transformed_stacks[output_index+offset].x = (float)(input_data[index]);
        }
    }
}

////////////////////////////////////////////////////////////////////////////////////////
// Class member functions
///////////////////////////////////////////////////////////////////////////////////////

/*
 * Initialize params struct
 */
Bm3d::Bm3d() {
    h_width = 0;
    h_height = 0;
    h_channels = 0;
    d_noisy_image = NULL;
    d_denoised_image = NULL;

    d_stacks = NULL;
    d_num_patches_in_stack = NULL;
    d_weight = NULL;
    d_wien_coef = NULL;
    d_kaiser_window = NULL;
}

Bm3d::~Bm3d() {

}

/*
 * Set first step params
 */
void Bm3d::set_fst_step_param() {

}

/*
 * Set second step params
 */
void Bm3d::set_2nd_step_param() {

}

/*
 * Set device params and allocate device memories
 */
void Bm3d::set_device_param() {
    total_patches = (h_width - h_fst_step_params.patch_size + 1) * (h_height - h_fst_step_params.patch_size + 1);
    total_ref_patches = ((h_width - h_fst_step_params.patch_size) / h_fst_step_params.stripe + 1) * ((h_height - h_fst_step_params.patch_size) / h_fst_step_params.stripe + 1);
    // copy original image to cuda
    const uint size = h_width * h_height;
    hipMalloc(&d_noisy_image, sizeof(uchar) * h_channels * size);

    hipMalloc(&d_stacks, sizeof(Q) * total_ref_patches * h_fst_step_params.max_group_size);
    hipMalloc(&d_num_patches_in_stack, sizeof(uint) * total_ref_patches);
    hipMalloc(&d_transformed_stacks, sizeof(hipfftComplex) * h_fst_step_params.patch_size * h_fst_step_params.patch_size * h_fst_step_params.max_group_size * total_ref_patches);

    hipMalloc(&d_numerator, sizeof(float) * size);
    hipMalloc(&d_denominator, sizeof(float) * size);
    hipMalloc(&d_weight, sizeof(float) * total_ref_patches);
    hipMalloc(&d_wien_coef, sizeof(float) * h_fst_step_params.patch_size * h_fst_step_params.patch_size * h_fst_step_params.max_group_size * total_ref_patches);
    hipMalloc(&d_wien_weight, sizeof(float) * total_ref_patches);

    hipMalloc(&d_denoised_image, sizeof(uchar) * size);

    // Only use the generic params for now
    GlobalConstants params;
    params.image_width = h_width;
    params.image_height = h_height;
    params.image_channels = h_channels;

    params.patch_size = h_fst_step_params.patch_size;
    params.searching_window_size = h_fst_step_params.searching_window_size;
    params.stripe = h_fst_step_params.stripe;
    params.max_group_size = h_fst_step_params.max_group_size;
    params.distance_threshold_1 = h_fst_step_params.distance_threshold_1;
    params.distance_threshold_2 = h_fst_step_params.distance_threshold_2;
    params.sigma = h_fst_step_params.sigma;
    params.lambda_3d = h_fst_step_params.lambda_3d;
    params.beta = h_fst_step_params.beta;
    params.total_ref_patches = total_ref_patches;

    hipMemcpyToSymbol(HIP_SYMBOL(cu_const_params), &params, sizeof(GlobalConstants));
    int dim3D[3] = {h_fst_step_params.patch_size, h_fst_step_params.patch_size, h_fst_step_params.max_group_size};
    int size_3d = h_fst_step_params.patch_size * h_fst_step_params.patch_size * h_fst_step_params.max_group_size;
    if(hipfftPlanMany(&plan3D, 3, dim3D,
                     NULL, 1, size_3d,
                     NULL, 1, size_3d,
                     HIPFFT_C2C, total_ref_patches) != HIPFFT_SUCCESS) {
        fprintf(stderr, "CUFFT Plan error: Plan failed");
        return;
    }
}

/*
 * Initialize image stats and allocate memory
 */
void Bm3d::copy_image_to_device(uchar *src_image) {
    // set width and height
    hipMemcpy(d_noisy_image, src_image, sizeof(uchar) * h_channels * h_height * h_width, hipMemcpyHostToDevice);
}

void Bm3d::free_device_params() {
    if (d_noisy_image) {
        hipFree(d_noisy_image);
    }
}

void Bm3d::clean_up_buffer() {
    // clean up buffer
    hipMemset(d_numerator, 0, sizeof(float)*h_width*h_height);
    hipMemset(d_denominator, 0, sizeof(float)*h_width*h_height);
    // hipMemset(d_stacks, 0, sizeof(Q) * total_ref_patches * h_fst_step_params.max_group_size);
    // hipMemset(d_num_patches_in_stack, 0, sizeof(uint) * total_ref_patches);
    hipMemset(d_transformed_stacks, 0, sizeof(hipfftComplex) * h_fst_step_params.patch_size * h_fst_step_params.patch_size * h_fst_step_params.max_group_size * total_ref_patches);
    
    hipMemset(d_weight, 0, sizeof(float) * total_ref_patches);
    hipMemset(d_wien_coef, 0, sizeof(float) * h_fst_step_params.patch_size * h_fst_step_params.patch_size * h_fst_step_params.max_group_size * total_ref_patches);
    hipMemset(d_wien_weight, 0, sizeof(float) * total_ref_patches);

    hipMemset(d_denoised_image, 0, sizeof(uchar) * h_width*h_height);
}

void Bm3d::set_up_realtime(int width, int height, int channels) {
    h_width = width;
    h_height = height;
    h_channels = channels;
    set_device_param();
}

/*
 * need to call set_up_realtime first
 */
void Bm3d::realtime_denoise(uchar *src_image,
                            uchar *dst_image
                            ) {
    copy_image_to_device(src_image);
    clean_up_buffer();
    denoise_fst_step();
    hipMemset(d_transformed_stacks, 0, sizeof(hipfftComplex) * h_fst_step_params.patch_size * h_fst_step_params.patch_size * h_fst_step_params.max_group_size * total_ref_patches);
    denoise_2nd_step();
    hipMemcpy(dst_image, d_denoised_image, sizeof(uchar) * h_width * h_height, hipMemcpyDeviceToHost);
}

/*
 * Take an image and run the algorithm to denoise.
 */
void Bm3d::denoise(uchar *src_image,
                   uchar *dst_image,
                   int width,
                   int height,
                   int sigma,
                   int channels,
                   int step,
                   int verbose = 1) {
    Stopwatch init_time;
    Stopwatch first_step;
    Stopwatch sed_step;

    h_width = width;
    h_height = height;
    h_channels = channels;

    init_time.start();
    set_device_param();
    init_time.stop();

    copy_image_to_device(src_image);

    first_step.start();
    denoise_fst_step();
    first_step.stop();


    sed_step.start();
    if (step == 2) {
        denoise_2nd_step();
    }
    sed_step.stop();

    // copy image from device to host
    printf("<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<\n");
    printf("Init takes %f\n", init_time.getSeconds());
    printf("First step takes %f\n", first_step.getSeconds());
    printf("Second step takes %f\n", sed_step.getSeconds());

    const uint num_pixels = h_width * h_height;
    hipMemcpy(dst_image, d_denoised_image, sizeof(uchar) * num_pixels, hipMemcpyDeviceToHost);
}

/*
 * Perform the first step denoise
 */
void Bm3d::denoise_fst_step() {
    //Block matching, each thread maps to a ref patch
    do_block_matching(d_noisy_image, h_fst_step_params.distance_threshold_1);

    //gather patches
    arrange_block(d_noisy_image);

    // perform 3D dct transform;

    if (hipfftExecC2C(plan3D, d_transformed_stacks, d_transformed_stacks, HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
        fprintf(stderr, "CUFFT error: 3D Forward failed");
        return;
    }

    // hard thresholding and normalize
    hard_threshold();

    // perform inverse 3D dct transform;
    if (hipfftExecC2C(plan3D, d_transformed_stacks, d_transformed_stacks, HIPFFT_BACKWARD) != HIPFFT_SUCCESS) {
        fprintf(stderr, "CUFFT error: 3D inverse failed");
        return;
    }
    // Need to normalize 3D inverse result by dividing patch_size * patch_size
    // aggregate to single image by writing into buffer
    do_aggregation(d_weight);
}

/*
 * Perform the second step denoise
 */
void Bm3d::denoise_2nd_step() {
    //Block matching estimate image, each thread maps to a ref patch
    do_block_matching(d_denoised_image, h_fst_step_params.distance_threshold_2);
    //gather patches for estimate image
    arrange_block(d_denoised_image);
    // perform 3d transform for estimate groups
    if (hipfftExecC2C(plan3D, d_transformed_stacks, d_transformed_stacks, HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
        fprintf(stderr, "CUFFT error: 3D Forward failed");
        return;
    }
    // calculate Wiener coefficient for each estimate group
    cal_wiener_coef();
    // gather noisy image patches according to estimate block matching result
    arrange_block(d_noisy_image);
    // perform 3d transform on original image
    if (hipfftExecC2C(plan3D, d_transformed_stacks, d_transformed_stacks, HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
        fprintf(stderr, "CUFFT error: 3D Forward failed");
        return;
    }
    // apply wiener coefficient to each group of transformed noisy data
    apply_wien_filter();
    // inverse 3d transform
    if (hipfftExecC2C(plan3D, d_transformed_stacks, d_transformed_stacks, HIPFFT_BACKWARD) != HIPFFT_SUCCESS) {
        fprintf(stderr, "CUFFT error: 3D Forward failed");
        return;
    }
    // aggregate to single image by writing into buffer
    do_aggregation(d_wien_weight);
    hipError_t code = hipGetLastError();
    if (code != hipSuccess) {
        fprintf(stderr, "Cuda error: %s\n", hipGetErrorString(code));
        return;
    }
}

void Bm3d::test_block_matching(uchar *input_image, int width, int height) {
    // generate a dummy image
    printf("testing block_matching\n");
    if (!input_image) {
        const int img_width = 40; // a 40 by 40 checkerboard of 8x8 patch
        const int patch_width = 8;
        uchar *dummy_image = (uchar *)malloc(img_width * img_width * sizeof(uchar));
        bool isWhite = false;
        for (int y = 0; y < img_width; y += patch_width) {
            for (int x = 0; x < img_width; x += patch_width) {
                // (x, y) is the top-left corner coordinate
                for (int j = 0; j < patch_width; ++j) {
                    for (int i = 0; i < patch_width; ++i) {
                        // (x + i, y + j) is the pixel coordinate
                        int idx = idx2(x+i, y+j, img_width);
                        input_image[idx] = isWhite ? 255 : 0;
                    }
                }
                isWhite = !isWhite;
            }
        }

        // set up the parameters and consts
        input_image = dummy_image;
    }
    h_width = width;
    h_height = height;
    h_channels = 1;
    set_device_param();
    copy_image_to_device(input_image);

    printf("width, height: %d %d\n", width, height);

    // determine how many threads we need to spawn
    const int num_ref_patches_x = (h_width - h_fst_step_params.patch_size) / h_fst_step_params.stripe + 1;

    // printf("total_ref_patches %d\n", total_ref_patches);
    // const int total_num_threads = total_ref_patches;
    // const int threads_per_block = 256;
    // const int num_blocks = (total_num_threads + threads_per_block - 1) / threads_per_block;
    // printf("total_num_threads %d num_block %d\n", total_ref_patches, num_blocks);

    // // hipError_t code = hipGetLastError();
    // // if (code != hipSuccess) {
    // //     fprintf(stderr, "Cuda error: %s\n", hipGetErrorString(code));
    // //     return;
    // // }
    // // call our block matching magic
    // block_matching<<<num_blocks, threads_per_block>>>(d_stacks, d_num_patches_in_stack);

    do_block_matching(input_image, h_fst_step_params.distance_threshold_1);

    Q *h_stacks = (Q *)malloc(sizeof(Q) * total_ref_patches * h_fst_step_params.max_group_size);
    hipMemcpy(h_stacks, d_stacks, sizeof(Q) * total_ref_patches * h_fst_step_params.max_group_size, hipMemcpyDeviceToHost);
    uint *h_num_patches_in_stack = (uint *)malloc(sizeof(uint) * total_ref_patches);
    hipMemcpy(h_num_patches_in_stack, d_num_patches_in_stack, sizeof(uint) * total_ref_patches, hipMemcpyDeviceToHost);

    // print the first stack
    const int which_stack = 13970;
    const int stack_x = which_stack % num_ref_patches_x;
    const int stack_y = which_stack / num_ref_patches_x;

    h_stacks = &h_stacks[which_stack * h_fst_step_params.max_group_size];



    printf("number of patches in stack %d: %d\n", which_stack, h_num_patches_in_stack[which_stack]);
    for (int i = 0; i < h_num_patches_in_stack[which_stack]; ++i) {
        const uint start_x = h_stacks[i].position.x;
        const uint start_y = h_stacks[i].position.y;
        printf("distance %d, x %d y %d\n", h_stacks[i].distance, start_x, start_y);
        for (int y = 0; y < h_fst_step_params.patch_size; ++y) {
            for (int x = 0; x < h_fst_step_params.patch_size; ++x) {
                const int idx = idx2( start_x + x, start_y + y, width);
                input_image[idx] = 255;
            }
        }
    }

    // set the original ref patch to 0
    for (int y = 0; y < h_fst_step_params.patch_size; ++y) {
        for (int x = 0; x < h_fst_step_params.patch_size; ++x) {
            const int idx = idx2(
                stack_x * h_fst_step_params.stripe + x,
                stack_y * h_fst_step_params.stripe + y,
                width);
            input_image[idx] = 0;
        }
    }

    // for (int y = 0; y < img_width; y += 1) {
    //     for (int x = 0; x < img_width; x += 1) {
    //         int idx = idx2(x, y, img_width);
    //         switch(input_image[idx]) {
    //             case 255:
    //                 printf("x");
    //                 break;
    //             case 127:
    //                 printf("o");
    //                 break;
    //             case 110:
    //                 printf("*");
    //                 break;
    //             default:
    //                 printf(" ");
    //         }
    //     }
    //     printf("\n");
    // }

    free_device_params();
}

/*
 *  arrange_block - according to the stacked patch indices, fetching data from the transformed
 *                  data array for 2D DCT. Input is an array of uint2, every N uint2
 *                  is a group. This kernel will put each group into an continuous array
 *                  of hipfftComplex num with x component to be the value, y component to be 0.f
 */
void Bm3d::arrange_block(uchar* input_data) {
    // input: Q* each struct is a patch with top left index
    // output: d_transformed_stacks, each patch got patch*patch size continuous chunk
    // each group will be assigned a thread
    Stopwatch arrange;
    arrange.start();
    int thread_per_block = 512;
    int num_blocks = (total_ref_patches + thread_per_block - 1) / thread_per_block;
    fill_patch_major_from_source<<<num_blocks, thread_per_block>>>(d_stacks, d_num_patches_in_stack, input_data, d_transformed_stacks);
    hipDeviceSynchronize();
    arrange.stop();
    // printf("Arrange block takes %f\n", arrange.getSeconds());
}

void Bm3d::test_arrange_block(uchar *input_data) {
    int size = h_fst_step_params.patch_size * h_fst_step_params.patch_size * h_fst_step_params.max_group_size * total_ref_patches;

    Q* test_q = (Q*)malloc(sizeof(Q)*total_ref_patches * h_fst_step_params.max_group_size);
    for (int i=0;i<2*h_fst_step_params.max_group_size; i++) {
        test_q[i].position.x = i;
        test_q[i].position.y = 0;
    }
    float* h_data = (float*)malloc(sizeof(float) * size);
    float* d_data;
    hipMalloc(&d_data, sizeof(float) * size);
    hipMemcpy(d_stacks, test_q, sizeof(Q) * total_ref_patches * h_fst_step_params.max_group_size, hipMemcpyHostToDevice);
    uint* h_num_patches = (uint*)calloc(total_ref_patches, sizeof(uint));
    h_num_patches[0] = h_fst_step_params.max_group_size;
    h_num_patches[1] = h_fst_step_params.max_group_size - 2;
    hipMemcpy(d_num_patches_in_stack, h_num_patches, sizeof(uint)*total_ref_patches, hipMemcpyHostToDevice);
    arrange_block(d_noisy_image);

    if (hipfftExecC2C(plan, d_transformed_stacks, d_transformed_stacks, HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
        fprintf(stderr, "CUFFT error: ExecR2C Forward failed");
        return;
    }

    if (hipfftExecC2C(plan, d_transformed_stacks, d_transformed_stacks, HIPFFT_BACKWARD) != HIPFFT_SUCCESS) {
        fprintf(stderr, "CUFFT error: ExecR2C Forward failed");
        return;
    }
    int threads_per_block = 512;
    int num_blocks = (size + threads_per_block - 1) / threads_per_block;
    complex2real<<<num_blocks, threads_per_block>>>(d_transformed_stacks, d_data, size, h_fst_step_params.patch_size*h_fst_step_params.patch_size);

    hipMemcpy(h_data, d_data, size * sizeof(float), hipMemcpyDeviceToHost);
    if (hipGetLastError() != hipSuccess) {
        fprintf(stderr, "Cuda error: Failed results copy\n");
        return;
    }
    for (int i=0;i<2*h_fst_step_params.patch_size*h_fst_step_params.patch_size*h_fst_step_params.max_group_size;i++) {
        int x = i/(h_fst_step_params.patch_size*h_fst_step_params.patch_size);
        int y = 0;
        if (i % (h_fst_step_params.patch_size*h_fst_step_params.patch_size) == 0) {
            printf("Patch (%d, %d)\n", x, 0);
        }
        int z = i - x*(h_fst_step_params.patch_size*h_fst_step_params.patch_size);
        int index = idx2(x+(z%h_fst_step_params.patch_size), y+(z/h_fst_step_params.patch_size), h_width);
        printf("Transform: %.3f vs Original: %d\n",
            h_data[i],
            input_data[index]
            );
    }
}

void Bm3d::test_aggregation(
    uchar *src_image,
    uint width,
    uint height,
    uchar *dst_image) {

    // set environmental variables
    h_width = width;
    h_height = height;
    h_channels = 1;
    set_device_param();
    copy_image_to_device(src_image);

    // step 0: block matching
    do_block_matching(src_image, h_fst_step_params.distance_threshold_1);

    // step 1: arrange the data into stacks of pixels
    arrange_block(d_noisy_image);

    // step 2: fill the weights with dummy values;
    float *weights = (float*)malloc(total_ref_patches * sizeof(float));
    for (int i = 0; i < total_ref_patches; ++i) {
        weights[i] = i % 32 + 1;
    }
    hipMemcpy(d_weight, weights, sizeof(float) * total_ref_patches, hipMemcpyHostToDevice);

    // step 3: do aggregation
    do_aggregation(d_weight);

    const uint num_pixels = h_width * h_height;
    hipMemcpy(dst_image, d_denoised_image, sizeof(uchar) * num_pixels, hipMemcpyDeviceToHost);
}

/*
 * do_block_matching - launch kernel to run block matching
 */
void Bm3d::do_block_matching(uchar* input_image, const uint distance_threshold) {
    // determine how many threads we need to spawn
    Stopwatch bm_time;
    bm_time.start();
    const int total_num_threads = total_ref_patches;
    const int threads_per_block = 512;
    const int num_blocks = (total_num_threads + threads_per_block - 1) / threads_per_block;
    block_matching<<<num_blocks, threads_per_block>>>(d_stacks, d_num_patches_in_stack, input_image, distance_threshold);
    hipDeviceSynchronize();
    bm_time.stop();
    // printf("Block Matching: %f\n", bm_time.getSeconds());
}

void Bm3d::hard_threshold() {
    Stopwatch hard_threshold;
    hard_threshold.start();
    int thread_per_block = 512;
    int num_blocks = (total_ref_patches + thread_per_block - 1) / thread_per_block;
    hard_filter<<<num_blocks, thread_per_block>>>(d_transformed_stacks, d_weight);
    hipDeviceSynchronize();
    hard_threshold.stop();
    // printf("Hard threshold takes %.5f\n", hard_threshold.getSeconds());
}

void Bm3d::cal_wiener_coef() {
    Stopwatch wiener_coef;
    wiener_coef.start();
    int thread_per_block = 512;
    int num_blocks = (total_ref_patches + thread_per_block - 1) / thread_per_block;
    get_wiener_coef<<<num_blocks, thread_per_block>>>(d_transformed_stacks, d_wien_coef);
    hipDeviceSynchronize();
    wiener_coef.stop();
    // printf("Get wiener takes %.5f\n", wiener_coef.getSeconds());
}

void Bm3d::apply_wien_filter() {
    Stopwatch apply_wiener;
    apply_wiener.start();
    int thread_per_block = 512;
    int num_blocks = (total_ref_patches + thread_per_block - 1) / thread_per_block;
    apply_wiener_coef<<<num_blocks, thread_per_block>>>(d_transformed_stacks, d_wien_coef, d_wien_weight);
    hipDeviceSynchronize();
    apply_wiener.stop();
    // printf("Apply wiener takes %.5f\n", apply_wiener.getSeconds());
}

void Bm3d::do_aggregation(float* weight) {
    Stopwatch ag_time;
    ag_time.start();
    const uint num_threads_per_block = 512;
    // step 1: do aggregation, one thread per stack
    uint num_blocks = (total_ref_patches + num_threads_per_block - 1) / num_threads_per_block;
    aggregate<<<num_blocks, num_threads_per_block>>>(
        d_stacks,
        d_num_patches_in_stack,
        weight,
        d_transformed_stacks,
        d_numerator,
        d_denominator
    );
    // step 2: reduction. calculate how many pixels
    const uint num_pixels = h_width * h_height;
    num_blocks = (num_pixels + num_threads_per_block - 1) / num_threads_per_block;
    reduction<<<num_blocks, num_threads_per_block>>>(d_numerator, d_denominator, d_denoised_image, num_pixels);
    hipDeviceSynchronize();
    ag_time.stop();
    // printf("Aggregation: %f\n", ag_time.getSeconds());
}
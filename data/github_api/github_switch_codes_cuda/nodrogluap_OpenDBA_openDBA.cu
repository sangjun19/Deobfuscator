#include "hip/hip_runtime.h"
// Repository: nodrogluap/OpenDBA
// File: openDBA.cu


/*******************************************************************************
 * (c) 2019 Paul Gordon's parallel (CUDA) NVIDIA GPU implementation of the Dynamic Time 
 * Warp Barycenter Averaging algorithm as conceived (without parallel compuation conception) by Francois Petitjean 
 ******************************************************************************/

#include <string>
#include <vector>
#include "openDBA.cuh"

__host__
int main(int argc, char **argv){
	
	int norm_sequences = 1; // signal range
	int prefix_to_skip = 0; // where do we start looking for a prefix when in open_prefix mode?
	int prefix_length = 0; // if non-zero, look only at the first N segments after prefix_to_skip for alignment
	
	char c;
	while( ( c = getopt (argc, argv, "n") ) != -1 ) {
		switch(c) {
			case 'n':
				norm_sequences = 0;
				break;
			default:
				/* You won't actually get here. */
				break;
		}
	}

	if(argc < 9){
		std::cout << "Usage: " << argv[0] << " <binary|text|tsv";
#if SLOW5_SUPPORTED == 1
		std::cout << "|slow5";
#endif	
#if HDF5_SUPPORTED == 1
		std::cout << "|fast5";
#endif
		std::cout << "> ";
#if DOUBLE_UNSUPPORTED == 1
		std::cout << "<short|int|uint|ulong|float> " <<
#else
		std::cout << "<short|int|uint|ulong|float|double> " <<
#endif
		          "<global|open_start|open_end|open> <output files prefix> <minimum unimodal segment length for clustering[,for consensus generation]> <prefix sequence to remove|/dev/null> <clustering threshold> <series.tsv|<series1> <series2> [series3...]>\n";
		exit(1);
     	}

	int num_series = argc-8;
	char *min_segment_length = argv[5]; // reasonable settings for nanopore RNA dwell time distributions would be 4 (lower to 2 for DNA)
	int read_mode = TEXT_READ_MODE;
	if(!strcmp(argv[1],"binary")){
		read_mode = BINARY_READ_MODE;
	}
#if SLOW5_SUPPORTED == 1
	else if(!strcmp(argv[1],"slow5")){
		read_mode = SLOW5_READ_MODE;
	}
#endif	
#if HDF5_SUPPORTED == 1
	else if(!strcmp(argv[1],"fast5")){
		read_mode = FAST5_READ_MODE;
	}
#endif
	else if(!strcmp(argv[1],"tsv")){
		read_mode = TSV_READ_MODE;
	}
	else if(strcmp(argv[1],"text")){
		std::cerr << "First argument (" << argv[1] << ") is neither 'binary' nor 'text'" << std::endl;
		exit(1);
	}

	int use_open_start = 0;
	int use_open_end = 0;
	if(!strcmp(argv[3],"global")){
        }
        else if(!strcmp(argv[3],"open_start")){
		use_open_start = 1;
        }
        else if(!strcmp(argv[3],"open_end")){
		use_open_end = 1;
	}
	// In format open_prefix_#_# where the numbers are the start and end of the segmented sequence positions to inspect
	else if(!strncmp(argv[3],"open_prefix", 11)){
	       	use_open_start = 0;
	       	use_open_end = 1;
		norm_sequences = 1; // TODO: quantile norm when set to 2? 
		std::stringstream ss(std::string(argv[3]+12));
		std::vector <std::string> fields;
		std::string tmp;
		while(std::getline(ss, tmp, '_')){
    			fields.push_back(tmp);
		}
		if(fields.size() != 2){
			std::cerr << "Unexpected alignment type specified, expected open_prefix_##_## but did not find a second underscore in " << argv[3] << std::endl;
			exit(1);
		}
		prefix_to_skip = std::stoi(fields[0]);
		prefix_length = std::stoi(fields[1]);
		std::cerr << "Aligning only the first " << prefix_length << " elements of each sequence" << std::endl;
        }
	else if(!strcmp(argv[3],"open")){
		use_open_start = 1;
		use_open_end = 1;
        }
	else{
		std::cerr << "Third argument (" << argv[3] << ") is not one of the accept values 'global', 'open_start', 'open_end' or 'open'" << std::endl;
                exit(1);
	}

	char *output_prefix = argv[4];

	char *seqprefix_filename = 0;
	if(strcmp(argv[6], "/dev/null")){
		seqprefix_filename = argv[6];
	}

	double cdist = (double) atof(argv[7]);

	int argind = 8; // Where the file names start
	// The following are all the data types supported by CUDA's atomicAdd() operation, so we support them too for best value precision maintenance.
	if(!strcmp(argv[2],"int")){
		setupAndRun<int>(seqprefix_filename, &argv[argind], num_series, output_prefix, read_mode, use_open_start, use_open_end, min_segment_length, norm_sequences, cdist, prefix_to_skip, prefix_length);
	}
	else if(!strcmp(argv[2],"uint")){
		setupAndRun<unsigned int>(seqprefix_filename, &argv[argind], num_series, output_prefix, read_mode, use_open_start, use_open_end, min_segment_length, norm_sequences, cdist, prefix_to_skip, prefix_length);
	}
	else if(!strcmp(argv[2],"ulong")){
		setupAndRun<unsigned long long>(seqprefix_filename, &argv[argind], num_series, output_prefix, read_mode, use_open_start, use_open_end, min_segment_length, norm_sequences, cdist, prefix_to_skip, prefix_length);
	}
	else if(!strcmp(argv[2],"float")){
		setupAndRun<float>(seqprefix_filename, &argv[argind], num_series, output_prefix, read_mode, use_open_start, use_open_end, min_segment_length, norm_sequences, cdist, prefix_to_skip, prefix_length);
	}
	// Only since CUDA 6.1 (Pascal and later architectures) is atomicAdd(double *...) supported.  Remove if you want to compile for earlier graphics cards.
#if DOUBLE_UNSUPPORTED == 1
#else
	else if(!strcmp(argv[2],"double")){
		setupAndRun<double>(seqprefix_filename, &argv[argind], num_series, output_prefix, read_mode, use_open_start, use_open_end, min_segment_length, norm_sequences, cdist, prefix_to_skip, prefix_length);
	}
#endif
	else if(!strcmp(argv[2], "short")){
		// Short is not properly supported in the hardware nor by z-normalization, we will convert to float  (last arg=1)
		setupAndRun<float>(seqprefix_filename, &argv[argind], num_series, output_prefix, read_mode, use_open_start, use_open_end, min_segment_length, norm_sequences, cdist, prefix_to_skip, prefix_length, 1);
	}
	else{
		std::cerr << "Second argument (" << argv[2] << ") was not one of the accepted numerical representations: 'int', 'uint', 'ulong', 'float' or 'double'" << std::endl;
		exit(1);
	}

	// Following needed to allow cuda-memcheck to detect memory leaks
	int deviceCount;
        hipGetDeviceCount(&deviceCount); CUERR("Getting GPU device count in teardown/cleanup");
	for(int i = 0; i < deviceCount; i++){
                hipSetDevice(i);
		hipDeviceReset(); CUERR("Resetting GPU device");
	}

	return 0;
}

#include "hip/hip_runtime.h"
// Repository: imogenproject/Imogen
// File: gravity/utils/cudaMHDKernels.cu

#include <stdio.h>
#include <string.h>
#include <stdarg.h>
#ifdef UNIX
#include <stdint.h>
#include <unistd.h>
#endif
#include "mex.h"

// CUDA
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "GPUmat.hh"

// static paramaters
static int init = 0;
static GPUmat *gm;

#include "cudaCommon.h"

//double **getGPUSourcePointers(const mxArray *prhs[], int num, int *retNumel);
//double **makeGPUDestinationArrays(GPUtype src, mxArray *retArray[], int howmany);
//double *makeDestinationArray(GPUtype src, mxArray *retArray[]);

#define OP_SOUNDSPEED 1
#define OP_GASPRESSURE 2
#define OP_TOTALPRESSURE 3
#define OP_MAGPRESSURE 4
#define OP_TOTALANDSND 5
#define OP_WARRAYS 6
#define OP_RELAXINGFLUX 7
#define OP_SEPERATELRFLUX 8
__global__ void cukern_Soundspeed(double *rho, double *E, double *px, double *py, double *pz, double *bx, double *by, double *bz, double *dout, double gam, int n);
__global__ void cukern_GasPressure(double *rho, double *E, double *px, double *py, double *pz, double *bx, double *by, double *bz, double *dout, double gam, int n);
__global__ void cukern_TotalPressure(double *rho, double *E, double *px, double *py, double *pz, double *bx, double *by, double *bz, double *dout, double gam, int n);
__global__ void cukern_MagneticPressure(double *bx, double *by, double *bz, double *dout, int n);
__global__ void cukern_TotalAndSound(double *rho, double *E, double *px, double *py, double *pz, double *bx, double *by, double *bz, double *total, double *sound, double gam, int n);
__global__ void cukern_CalcWArrays(double *rho, double *E, double *px, double *py, double *pz, double *bx, double *by, double *bz, double *P, double *Cfreeze, double *rhoW, double *enerW, double *pxW, double *pyW, double *pzW, int dir, int n);

__global__ void cukern_SeperateLRFlux(double *arr, double *wArr, double *left, double *right, int n);
__global__ void cukern_PerformFlux(double *array0, double *Cfreeze, double *fluxRa, double *fluxRb, double *fluxLa, double *fluxLb, double *out, double lambda, int n);

#define BLOCKWIDTH 256
#define THREADLOOPS 1


void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
  if (init == 0) {
    // Initialize function
    // mexLock();
    // load GPUmat
    gm = gmGetGPUmat();
    init = 1;
  }

  // Determine appropriate number of arguments for RHS
  if (nrhs < 2) mexErrMsgTxt("Require at least (computation type, input argument)");
  int operation = (int)*mxGetPr(prhs[0]);

  dim3 blocksize; blocksize.x = BLOCKWIDTH; blocksize.y = blocksize.z = 1;
  int numel; dim3 gridsize;

  // Select the appropriate kernel to invoke
  if((operation == OP_SOUNDSPEED) || (operation == OP_GASPRESSURE) || (operation == OP_TOTALPRESSURE)) {
    if( (nlhs != 1) || (nrhs != 10)) { mexErrMsgTxt("Soundspeed operator is Cs = cudaMHDKernels(1, rho, E, px, py, pz, bx, by, bz, gamma)"); }
    double gam = *mxGetPr(prhs[9]);
    int arrdim[3];
    double **srcs = getGPUSourcePointers(prhs, arrdim, 1, 8);
    numel = arrdim[0]*arrdim[1]*arrdim[2];

    gridsize.x = numel / (BLOCKWIDTH*THREADLOOPS); if(gridsize.x * (BLOCKWIDTH*THREADLOOPS) < numel) gridsize.x++;
    gridsize.y = gridsize.z =1;
    double **destPtr = makeGPUDestinationArrays(gm->gputype.getGPUtype(prhs[1]), plhs, 1, gm);
//printf("%i %i %i %i %i %i\n", blocksize.x, blocksize.y, blocksize.z, gridsize.x, gridsize.y, gridsize.z);
    switch(operation) {
      case OP_SOUNDSPEED:       cukern_Soundspeed<<<gridsize, blocksize>>>(srcs[0], srcs[1], srcs[2], srcs[3], srcs[4], srcs[5], srcs[6], srcs[7], destPtr[0], gam, numel); break;
      case OP_GASPRESSURE:     cukern_GasPressure<<<gridsize, blocksize>>>(srcs[0], srcs[1], srcs[2], srcs[3], srcs[4], srcs[5], srcs[6], srcs[7], destPtr[0], gam, numel); break;
      case OP_TOTALPRESSURE: cukern_TotalPressure<<<gridsize, blocksize>>>(srcs[0], srcs[1], srcs[2], srcs[3], srcs[4], srcs[5], srcs[6], srcs[7], destPtr[0], gam, numel); break;
    }
    free(destPtr);

  } else if((operation == OP_MAGPRESSURE)) {
    if( (nlhs != 1) || (nrhs != 4)) { mexErrMsgTxt("Magnetic pressure operator is Pm = cudaMHDKernels(4, bx, by, bz)"); }

    double **srcs = getGPUSourcePointers(prhs, 3, &numel, 1, gm);
    gridsize.x = numel / (BLOCKWIDTH*THREADLOOPS); if(gridsize.x * (BLOCKWIDTH*THREADLOOPS) < numel) gridsize.x++;
    gridsize.y = gridsize.z =1;
    double **destPtr = makeGPUDestinationArrays(gm->gputype.getGPUtype(prhs[1]), plhs, 1, gm);

    cukern_MagneticPressure<<<gridsize, blocksize>>>(srcs[0], srcs[1], srcs[2], destPtr[0], numel);
    free(destPtr); free(srcs);

  } else if((operation == OP_TOTALANDSND)) {
    if( (nlhs != 2) || (nrhs != 10)) { mexErrMsgTxt("Soundspeed operator is [Ptot Cs] = cudaMHDKernels(5, rho, E, px, py, pz, bx, by, bz, gamma)"); }
    double gam = *mxGetPr(prhs[9]);
    double **srcs = getGPUSourcePointers(prhs, 8, &numel, 1, gm);
    gridsize.x = numel / (BLOCKWIDTH*THREADLOOPS); if(gridsize.x * (BLOCKWIDTH*THREADLOOPS) < numel) gridsize.x++;
    gridsize.y = gridsize.z =1;
    double **destPtr = makeGPUDestinationArrays(gm->gputype.getGPUtype(prhs[1]), plhs, 2, gm);

    cukern_TotalAndSound<<<gridsize, blocksize>>>(srcs[0], srcs[1], srcs[2], srcs[3], srcs[4], srcs[5], srcs[6], srcs[7], destPtr[0], destPtr[1], gam, numel);
    free(destPtr); free(srcs);
  } else if ((operation == OP_WARRAYS)) {
    if( (nlhs != 5) || (nrhs != 12)) { mexErrMsgTxt("solving W operator is [rhoW enerW pxW pyW pzW] = cudaMHDKernels(6, rho, E, px, py, pz, bx, by, bz, P, cFreeze, direction)"); }
    int dir = (int)*mxGetPr(prhs[11]);
    double **srcs = getGPUSourcePointers(prhs, 10, &numel, 1, gm);
    gridsize.x = numel / (BLOCKWIDTH*THREADLOOPS); if(gridsize.x * (BLOCKWIDTH*THREADLOOPS) < numel) gridsize.x++;
    gridsize.y = gridsize.z =1;
    double **destPtr = makeGPUDestinationArrays(gm->gputype.getGPUtype(prhs[1]), plhs, 5, gm);

    cukern_CalcWArrays<<<gridsize, blocksize>>>(srcs[0], srcs[1], srcs[2], srcs[3], srcs[4], srcs[5], srcs[6], srcs[7], srcs[8], srcs[9], destPtr[0], destPtr[1], destPtr[2], destPtr[3], destPtr[4], dir, numel);
    free(destPtr); free(srcs);
  } else if ((operation == OP_RELAXINGFLUX)) {
    if( (nlhs != 1) || (nrhs != 8)) { mexErrMsgTxt("relaxing flux operator is fluxed = cudaMHDKernels(7, old, tempfreeze, right, right_shifted, left, left_shifted, lambda)"); }
    double lambda = *mxGetPr(prhs[7]);
    double **srcs = getGPUSourcePointers(prhs, 6, &numel, 1, gm);
    gridsize.x = numel / (BLOCKWIDTH*THREADLOOPS); if(gridsize.x * (BLOCKWIDTH*THREADLOOPS) < numel) gridsize.x++;
    gridsize.y = gridsize.z =1;
    double **destPtr = makeGPUDestinationArrays(gm->gputype.getGPUtype(prhs[1]), plhs, 1, gm);

    cukern_PerformFlux<<<gridsize, blocksize>>>(srcs[0], srcs[1], srcs[2], srcs[3], srcs[4], srcs[5], destPtr[0], lambda, numel);
    free(destPtr); free(srcs);
  } else if ((operation == OP_SEPERATELRFLUX)) {
    if ((nlhs != 2) || (nrhs != 3)) { mexErrMsgTxt("flux seperation operator is [Fl Fr] = cudaMHDKernels(8, array, wArray)"); }
    double **srcs = getGPUSourcePointers(prhs, 2, &numel, 1, gm);
    gridsize.x = numel / (BLOCKWIDTH*THREADLOOPS); if(gridsize.x * (BLOCKWIDTH*THREADLOOPS) < numel) gridsize.x++;
    gridsize.y = gridsize.z =1;
    double **destPtr = makeGPUDestinationArrays(gm->gputype.getGPUtype(prhs[1]), plhs, 2, gm);

    cukern_SeperateLRFlux<<<gridsize, blocksize>>>(srcs[0], srcs[1], destPtr[0], destPtr[1], numel);
    free(destPtr); free(srcs);
  }

}

//#define KERNEL_PREAMBLE int x = THREADLOOPS*(threadIdx.x + blockDim.x*blockIdx.x); if (x >= n) {return;} int imax; ((x+THREADLOOPS) > n) ? imax = n : imax = x + THREADLOOPS; for(; x < imax; x++)
#define KERNEL_PREAMBLE int x = threadIdx.x + blockDim.x*blockIdx.x; if (x >= n) { return; }

// THIS KERNEL CALCULATES SOUNDSPEED 
__global__ void cukern_Soundspeed(double *rho, double *E, double *px, double *py, double *pz, double *bx, double *by, double *bz, double *dout, double gam, int n)
{
double gg1 = gam*(gam-1.0);

KERNEL_PREAMBLE
dout[x] = sqrt(abs( (gg1*(E[x] - .5*(px[x]*px[x] + py[x]*py[x] + pz[x]*pz[x])/rho[x]) + (2.0 -.5*gg1)*(bx[x]*bx[x] + by[x]*by[x] + bz[x]*bz[x]))/rho[x] ));
}

// THIS KERNEL CALCULATES GAS PRESSURE
__global__ void cukern_GasPressure(double *rho, double *E, double *px, double *py, double *pz, double *bx, double *by, double *bz, double *dout, double gam, int n)
{
KERNEL_PREAMBLE
dout[x] = (gam-1.0)*(E[x] - .5*((px[x]*px[x]+py[x]*py[x]+pz[x]*pz[x])/rho[x] + bx[x]*bx[x]+by[x]*by[x]+bz[x]*bz[x]));
}

// THIS KERNEL CALCULATES TOTAL PRESSURE
__global__ void cukern_TotalPressure(double *rho, double *E, double *px, double *py, double *pz, double *bx, double *by, double *bz, double *dout, double gam, int n)
{
KERNEL_PREAMBLE
dout[x] = (gam-1.0)*(E[x] - .5*((px[x]*px[x]+py[x]*py[x]+pz[x]*pz[x])/rho[x])) + .5*(2.0-gam)*(bx[x]*bx[x]+by[x]*by[x]+bz[x]*bz[x]);
}

// THIS KERNEL CALCULATES MAGNETIC PRESSURE
__global__ void cukern_MagneticPressure(double *bx, double *by, double *bz, double *dout, int n)
{
KERNEL_PREAMBLE
dout[x] = .5*(bx[x]*bx[x]+by[x]*by[x]+bz[x]*bz[x]);
}

__global__ void cukern_TotalAndSound(double *rho, double *E, double *px, double *py, double *pz, double *bx, double *by, double *bz, double *total, double *sound, double gam, int n)
{
double gg1 = gam*(gam-1.0);
double psqhf, bsqhf;

KERNEL_PREAMBLE {
	psqhf = .5*(px[x]*px[x]+py[x]*py[x]+pz[x]*pz[x]);
	bsqhf = .5*(bx[x]*bx[x]+by[x]*by[x]+bz[x]*bz[x]);
	
	total[x] = (gam-1.0)*(E[x] - psqhf/rho[x]) + (2.0-gam)*bsqhf;
	sound[x]   = sqrt(abs( (gg1*(E[x] - psqhf/rho[x]) + (4.0 - gg1)*bsqhf)/rho[x] ));
	}
}

__global__ void cukern_CalcWArrays(double *rho, double *E, double *px, double *py, double *pz, double *bx, double *by, double *bz, double *P, double *Cfreeze, double *rhoW, double *enerW, double *pxW, double *pyW, double *pzW, int dir, int n)
{
double Cinv, rhoinv;

KERNEL_PREAMBLE {

Cinv = 1.0/Cfreeze[x];
rhoinv = 1.0/rho[x];

switch(dir) {
  case 1:
    rhoW[x]  = px[x] * Cinv;
    enerW[x] = (px[x] * (E[x] + P[x]) - bx[x]*(px[x]*bx[x]+py[x]*by[x]+pz[x]*bz[x]) ) * (rhoinv*Cinv);
    pxW[x]   = (px[x]*px[x]*rhoinv + P[x] - bx[x]*bx[x])*Cinv;
    pyW[x]   = (px[x]*py[x]*rhoinv        - bx[x]*by[x])*Cinv;
    pzW[x]   = (px[x]*pz[x]*rhoinv        - bx[x]*bz[x])*Cinv;
    break;
  case 2:
    rhoW[x]  = py[x] * Cinv;
    enerW[x] = (py[x] * (E[x] + P[x]) - by[x]*(px[x]*bx[x]+py[x]*by[x]+pz[x]*bz[x]) ) * (rhoinv*Cinv);
    pxW[x]   = (py[x]*px[x]*rhoinv        - by[x]*bx[x])*Cinv;
    pyW[x]   = (py[x]*py[x]*rhoinv + P[x] - by[x]*by[x])*Cinv;
    pzW[x]   = (py[x]*pz[x]*rhoinv        - by[x]*bz[x])*Cinv;
    break;
  case 3:
    rhoW[x]  = pz[x] * Cinv;
    enerW[x] = (pz[x] * (E[x] + P[x]) - bz[x]*(px[x]*bx[x]+py[x]*by[x]+pz[x]*bz[x]) ) * (rhoinv*Cinv);
    pxW[x]   = (pz[x]*px[x]*rhoinv        - bz[x]*bx[x])*Cinv;
    pyW[x]   = (pz[x]*py[x]*rhoinv        - bz[x]*by[x])*Cinv;
    pzW[x]   = (pz[x]*pz[x]*rhoinv + P[x] - bz[x]*bz[x])*Cinv;
    break;
  }

}
/*mass.wArray    = mom(X).array ./ freezeSpd.array;

    %--- ENERGY DENSITY ---%
    ener.wArray    = velocity .* (ener.array + press) - mag(X).cellMag.array .* ...
                        ( mag(1).cellMag.array .* mom(1).array ...
                        + mag(2).cellMag.array .* mom(2).array ...
                        + mag(3).cellMag.array .* mom(3).array) ./ mass.array;
    ener.wArray    = ener.wArray ./ freezeSpd.array;

    %--- MOMENTUM DENSITY ---%
    for i=1:3
        mom(i).wArray    = (velocity .* mom(i).array + press*dirVec(i)...
                             - mag(X).cellMag.array .* mag(i).cellMag.array) ./ freezeSpd.array;
    end*/

}

__global__ void cukern_PerformFlux(double *array0, double *Cfreeze, double *fluxRa, double *fluxRb, double *fluxLa, double *fluxLb, double *out, double lambda, int n)
{
KERNEL_PREAMBLE 
out[x] = array0[x] - lambda*Cfreeze[x]*(fluxRa[x] - fluxRb[x] + fluxLa[x] - fluxLb[x]);

//v(i).store.array = v(i).array - 0.5*fluxFactor .* tempFreeze .* ...
//                        ( v(i).store.fluxR.array - v(i).store.fluxR.shift(X,-1) ...
//                        + v(i).store.fluxL.array - v(i).store.fluxL.shift(X,1) );
}

__global__ void cukern_SeperateLRFlux(double *arr, double *wArr, double *left, double *right, int n)
{
KERNEL_PREAMBLE {
	left[x]  = .5*(arr[x] - wArr[x]);
	right[x] = .5*(arr[x] + wArr[x]);
	}

}



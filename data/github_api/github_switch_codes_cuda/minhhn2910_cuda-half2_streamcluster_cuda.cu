#include "hip/hip_runtime.h"
// Repository: minhhn2910/cuda-half2
// File: benchmarks/rodinia_3.1/cuda/streamcluster_half2/streamcluster_cuda.cu

/***********************************************
	streamcluster_cuda.cu
	: parallelized code of streamcluster

	- original code from PARSEC Benchmark Suite
	- parallelization with CUDA API has been applied by

	Shawn Sang-Ha Lee - sl4ge@virginia.edu
	University of Virginia
	Department of Electrical and Computer Engineering
	Department of Computer Science

***********************************************/
#include "streamcluster_header.cu"
#include "newhalf.hpp"
#include <hip/hip_fp16.h>

#include "half2_operator_overload.cuh"
#include "half_operator_overload.cuh"
typedef struct {
  half2 weight;
  float *coord;
  long2 assign;  /* number of point where this one is assigned */
  half2 cost;  /* cost of that assignment, weight*distance */
} Point_dev_half2;

int counting = 0;
using namespace std;

// AUTO-ERROR CHECK FOR ALL CUDA FUNCTIONS
#define CUDA_SAFE_CALL( call) do {										\
   hipError_t err = call;												\
   if( hipSuccess != err) {											\
       fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",	\
               __FILE__, __LINE__, hipGetErrorString( err) );			\
   exit(EXIT_FAILURE);													\
   } } while (0)

#define THREADS_PER_BLOCK 512
#define MAXBLOCKS 65536
#define CUDATIME

// host memory
float *work_mem_h;
float *coord_h;

half_float::half *work_mem_half;
half_float::half *coord_half;

// device memory
half2 *work_mem_d;
half2 *coord_d;
int   *center_table_d;
bool  *switch_membership_d;
Point_dev_half2 *p;

static int iter = 0;		// counter for total# of iteration


//=======================================
// Euclidean Distance
//=======================================
__device__ __half2
d_dist(int p1, int p2, int num, int dim, __half2 *coord_d)
{
	__half2 retval = __float2half2_rn(0.0);
	for(int i = 0; i < dim; i++){
		//~ __half2 tmp = coord_d[(i*num/2)+p1] - coord_d[(i*num/2)+p2/2];
		__half2 tmp = coord_d[(i*num/2)+p1] - __half2half2(((half*)coord_d)[(i*num)+p2]);
		retval += tmp * tmp;
	}
	return retval;
}

//=======================================
// Kernel - Compute Cost
//=======================================
__global__ void
kernel_compute_cost(int num, int dim, long x, Point_dev_half2 *p, int K, int stride,
					__half2 *coord_d, __half2 *work_mem_d, int *center_table_d, bool *switch_membership_d)
{
	// block ID and global thread ID
	const int bid  = blockIdx.x + gridDim.x * blockIdx.y;
	const int tid = blockDim.x * bid + threadIdx.x;

	if(tid < num/2)
	{
		__half *lower1 = &((__half*)work_mem_d)[tid*2*stride]; //half array
		__half *lower2 = &((__half*)work_mem_d)[(tid*2+1)*stride]; //half array
		

		// cost between this point and point[x]: euclidean distance multiplied by weight
		__half2 x_cost = d_dist(tid, x, num, dim, coord_d) * p[tid].weight;

		//~ work_mem_d[tid] = x_cost;
		
		// if computed cost is less then original (it saves), mark it as to reassign
    float2 x_cost_temp = __half22float2(x_cost);
    float2 p_cost_temp = __half22float2(p[tid].cost);
//test
  /*   lower[K] += __float2half(1);
		lower[center_table_d[p[tid].assign.y]] = __float2half(1);
		lower[center_table_d[p[tid].assign.x]] = __float2half(1);
		lower[center_table_d[p[tid].assign.y]] = __float2half(1);
		lower[center_table_d[p[tid].assign.x]] = __float2half(1);
switch_membership_d[2*tid] = 1;
switch_membership_d[2*tid+1] = 1;
*/
		if ( x_cost_temp.x < p_cost_temp.x)
		{
			switch_membership_d[2*tid] = 1;
			lower1[K] += __float2half(x_cost_temp.x - p_cost_temp.x);
		}
		// if computed cost is larger, save the difference
		else
		{
			lower1[center_table_d[p[tid].assign.x]] += __float2half(p_cost_temp.x - x_cost_temp.x);
		}
		if ( x_cost_temp.y < p_cost_temp.y)
		{
			switch_membership_d[2*tid+1] = 1;
			lower2[K] += __float2half(x_cost_temp.y - p_cost_temp.y);
		}
		// if computed cost is larger, save the difference
		else
		{
			lower2[center_table_d[p[tid].assign.y]] += __float2half(p_cost_temp.y - x_cost_temp.y);
		}

  /*
		if ( x_cost < p[tid].cost )
		{
			switch_membership_d[tid] = 1;
			lower[K] += x_cost - p[tid].cost;
		}
		// if computed cost is larger, save the difference
		else
		{
			lower[center_table_d[p[tid].assign]] += p[tid].cost - x_cost;
		}*/

	}
}

//=======================================
// Allocate Device Memory
//=======================================
void allocDevMem(int num, int dim)
{
	CUDA_SAFE_CALL( hipMalloc((void**) &center_table_d,	  num * sizeof(int))   );
	CUDA_SAFE_CALL( hipMalloc((void**) &switch_membership_d, num * sizeof(bool))  );
	CUDA_SAFE_CALL( hipMalloc((void**) &p,					  num/2 * sizeof(Point_half2)) );
	CUDA_SAFE_CALL( hipMalloc((void**) &coord_d,		num * dim * sizeof(__half)) );
}

//=======================================
// Allocate Host Memory
//=======================================
void allocHostMem(int num, int dim)
{
	coord_h	= (float*) malloc( num * dim * sizeof(float) );
  coord_half	= (half_float::half*) malloc( num * dim * sizeof(half) );
}

//=======================================
// Free Device Memory
//=======================================
void freeDevMem()
{
	CUDA_SAFE_CALL( hipFree(center_table_d)	  );
	CUDA_SAFE_CALL( hipFree(switch_membership_d) );
	CUDA_SAFE_CALL( hipFree(p)					  );
	CUDA_SAFE_CALL( hipFree(coord_d)			  );
}

//=======================================
// Free Host Memory
//=======================================
void freeHostMem()
{
	free(coord_h);
  free(coord_half);

}

//=======================================
// pgain Entry - CUDA SETUP + CUDA CALL
//=======================================
float pgain( long x, Points *points, float z, long int *numcenters, int kmax, bool *is_center, int *center_table, bool *switch_membership, bool isCoordChanged,
							double *serial_t, double *cpu_to_gpu_t, double *gpu_to_cpu_t, double *alloc_t, double *kernel_t, double *free_t)
{
#ifdef CUDATIME
	float tmp_t;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, 0);
#endif

	hipError_t error;

	int stride	= *numcenters + 1;			// size of each work_mem segment
	int K		= *numcenters ;				// number of centers
	int num		=  points->num;				// number of points
	int dim		=  points->dim;				// number of dimension
	int nThread =  num;						// number of threads == number of data points

	//=========================================
	// ALLOCATE HOST MEMORY + DATA PREPARATION
	//=========================================
	work_mem_h = (float*) malloc(stride * (nThread + 1) * sizeof(float) );

  //printf ("num : %d\n",num);

  work_mem_half = (half_float::half*) malloc(stride * (nThread + 1) * sizeof(half) );

	// Only on the first iteration
	if(iter == 0)
	{
		allocHostMem(num, dim);
	}

	// build center-index table
	int count = 0;
	for( int i=0; i<num; i++)
	{
		if( is_center[i] )
		{
			center_table[i] = count++;
		}
	}

	// Extract 'coord'
	// Only if first iteration OR coord has changed
	if(isCoordChanged || iter == 0)
	{
		for(int i=0; i<dim; i++)
		{
			for(int j=0; j<num; j++)
			{
				coord_h[ (num*i)+j ] = points->p[j].coord[i];
        coord_half[ (num*i)+j ] = half_float::half(coord_h[ (num*i)+j ]);
			}
		}
	}
/*typedef struct {
  half_float::half weight;
  half_float::half *coord;
  long assign;
  half_float::half cost;
} Point_half;*/
  //copy points->p
  //points.num = chunksize;
  //points.p = (Point *)malloc(chunksize*sizeof(Point))
  Point_half2* point_half =(Point_half2*) malloc(points->num/2*sizeof(Point_half2));
  for(int i = 0;i <points->num/2; i++){
      point_half[i].weight = floats2half2(points->p[2*i+1].weight,points->p[2*i].weight);

      point_half[i].assign = make_long2(points->p[2*i].assign, points->p[2*i+1].assign); /// in reverse order

      point_half[i].cost = floats2half2(points->p[2*i+1].cost,points->p[2*i].cost);

  }
  //~ //printf("%ld %ld ",point_half[0].assign.x, point_half[0].assign.y);

#ifdef CUDATIME
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&tmp_t, start, stop);
	*serial_t += (double) tmp_t;

	hipEventRecord(start,0);
#endif

	//=======================================
	// ALLOCATE GPU MEMORY
	//=======================================
	CUDA_SAFE_CALL( hipMalloc((void**) &work_mem_d,  stride * (nThread + 1) * sizeof(half)) );
	// Only on the first iteration
	if( iter == 0 )
	{
		allocDevMem(num, dim);
	}

#ifdef CUDATIME
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&tmp_t, start, stop);
	*alloc_t += (double) tmp_t;

	hipEventRecord(start,0);
#endif

	//=======================================
	// CPU-TO-GPU MEMORY COPY
	//=======================================
	// Only if first iteration OR coord has changed
	if(isCoordChanged || iter == 0)
	{
		CUDA_SAFE_CALL( hipMemcpy(coord_d,  coord_half,	 num * dim * sizeof(half), hipMemcpyHostToDevice) );
	}
	CUDA_SAFE_CALL( hipMemcpy(center_table_d,  center_table,  num * sizeof(int),   hipMemcpyHostToDevice) );
	CUDA_SAFE_CALL( hipMemcpy(p,  point_half,				   num/2 * sizeof(Point_half2), hipMemcpyHostToDevice) );

	CUDA_SAFE_CALL( hipMemset((void*) switch_membership_d, 0,			num * sizeof(bool))  );
	CUDA_SAFE_CALL( hipMemset((void*) work_mem_d,  		0, stride * (nThread + 1) * sizeof(half)) );

#ifdef CUDATIME
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&tmp_t, start, stop);
	*cpu_to_gpu_t += (double) tmp_t;

	hipEventRecord(start,0);
#endif
	printf("call me %d  num %d\n", counting++, num);
	//=======================================
	// KERNEL: CALCULATE COST
	//=======================================
	// Determine the number of thread blocks in the x- and y-dimension
	int num_blocks 	 = (int) ((float) (num + THREADS_PER_BLOCK - 1) / (float) THREADS_PER_BLOCK);
	int num_blocks_y = (int) ((float) (num_blocks + MAXBLOCKS - 1)  / (float) MAXBLOCKS);
	int num_blocks_x = (int) ((float) (num_blocks+num_blocks_y - 1) / (float) num_blocks_y);
	dim3 grid_size(num_blocks_x, num_blocks_y, 1);
	//~ printf("%d %d %d stride %d k %d \n",num_blocks,num_blocks_y,num_blocks_x, stride, K);  

	kernel_compute_cost<<<grid_size, THREADS_PER_BLOCK/2>>>(
															num,					// in:	# of data
															dim,					// in:	dimension of point coordinates
															x,						// in:	point to open a center at
															p,						// in:	data point array
															K,						// in:	number of centers
															stride,					// in:  size of each work_mem segment
															coord_d,				// in:	array of point coordinates
															work_mem_d,				// out:	cost and lower field array
															center_table_d,			// in:	center index table
															switch_membership_d		// out:  changes in membership
															);
	hipDeviceSynchronize();

	// error check
	error = hipGetLastError();
	if (error != hipSuccess)
	{
		printf("kernel error: %s\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

#ifdef CUDATIME
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&tmp_t, start, stop);
	*kernel_t += (double) tmp_t;

	hipEventRecord(start,0);
#endif

	//=======================================
	// GPU-TO-CPU MEMORY COPY
	//=======================================
	CUDA_SAFE_CALL( hipMemcpy(work_mem_half, 		  work_mem_d, 	stride * (nThread + 1) * sizeof(half), hipMemcpyDeviceToHost) );

	CUDA_SAFE_CALL( hipMemcpy(switch_membership, switch_membership_d,	 num * sizeof(bool),  hipMemcpyDeviceToHost) );

#ifdef CUDATIME
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&tmp_t, start, stop);
	*gpu_to_cpu_t += (double) tmp_t;

	hipEventRecord(start,0);
#endif

  for(int i = 0 ; i<stride * (nThread + 1) ; i++){
    work_mem_h[i] = float(work_mem_half[i]);
    //~ printf("%f, ",work_mem_h[i]);
  }
   //~ printf ("\n");
   
     //~ for (int i = 0 ; i < num; i++){
	  //~ printf("%d, ", switch_membership[i]);
	  //~ }
	//~ printf ("\n"); 
	//=======================================
	// CPU (SERIAL) WORK
	//=======================================
	int number_of_centers_to_close = 0;
	float gl_cost_of_opening_x = z;
	float *gl_lower = &work_mem_h[stride * nThread];
	// compute the number of centers to close if we are to open i
	for(int i=0; i < num; i++)
	{
		if( is_center[i] )
		{
			float low = z;
		    for( int j = 0; j < num; j++ )
			{
				low += work_mem_h[ j*stride + center_table[i] ];
			}

		    gl_lower[center_table[i]] = low;

		    if ( low > 0 )
			{
				++number_of_centers_to_close;
				work_mem_h[i*stride+K] -= low;
		    }
		}
		gl_cost_of_opening_x += work_mem_h[i*stride+K];
	}

	//if opening a center at x saves cost (i.e. cost is negative) do so; otherwise, do nothing
	if ( gl_cost_of_opening_x < 0 )
	{
		for(int i = 0; i < num; i++)
		{
			bool close_center = gl_lower[center_table[points->p[i].assign]] > 0 ;
			if ( switch_membership[i] || close_center )
			{
				points->p[i].cost = dist(points->p[i], points->p[x], dim) * points->p[i].weight;
				points->p[i].assign = x;
			}
		}

		for(int i = 0; i < num; i++)
		{
			if( is_center[i] && gl_lower[center_table[i]] > 0 )
			{
				is_center[i] = false;
			}
		}

		if( x >= 0 && x < num)
		{
			is_center[x] = true;
		}
		*numcenters = *numcenters + 1 - number_of_centers_to_close;
	}
	else
	{
		gl_cost_of_opening_x = 0;
	}

	//=======================================
	// DEALLOCATE HOST MEMORY
	//=======================================
	free(work_mem_h);
  free(work_mem_half);
  free(point_half);

#ifdef CUDATIME
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&tmp_t, start, stop);
	*serial_t += (double) tmp_t;

	hipEventRecord(start,0);
#endif

	//=======================================
	// DEALLOCATE GPU MEMORY
	//=======================================
	CUDA_SAFE_CALL( hipFree(work_mem_d) );


#ifdef CUDATIME
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&tmp_t, start, stop);
	*free_t += (double) tmp_t;
#endif
	iter++;
	return -gl_cost_of_opening_x;
}

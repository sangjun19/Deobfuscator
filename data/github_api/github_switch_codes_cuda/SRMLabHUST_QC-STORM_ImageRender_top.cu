#include "hip/hip_runtime.h"
// Repository: SRMLabHUST/QC-STORM
// File: MLEbfgs/bfgs_MLE_dll x64/bfgsMLE_dll/bfgsMLE_dll/ImgRender/ImageRender_top.cu

/*
This program is free software: you can redistribute it and/or modify
it under the terms of the GNU LESSER GENERAL PUBLIC LICENSE as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU LESSER GENERAL PUBLIC LICENSE for more details.

You should have received a copy of the GNU LESSER GENERAL PUBLIC LICENSE
along with this program.  If not, see <https://www.gnu.org/licenses/>.
*/

#include "ImageRender_2d.h"
#include "ImageRender_3d.h"
#include "ImageRender_top.h"
#include <malloc.h>


/*
RGBImageEncodeMode 0 : peak photon as weight for each molecule, rendered by localization precision calculated by CRLB
RGBImageEncodeMode 1 : 1 as weight for each molecule, rendered by localization precision calculated by CRLB
RGBImageEncodeMode 2 : 1 as weight for each molecule, rendered by fixed localization precision
*/


void ImageRenderData_TypeDef::FluoRenderTop(float *h_LocArry, LocalizationPara & LocPara, int RenderingMode, float FixedlocPrec, int FluoNum, hipStream_t cstream)
{
	if (FluoNum <= 0)return;

	// for get display max intensity
	tRendFluoNum += FluoNum;

	if (tRendFluoNum > 20 * MaxPointNum)
	{
		tRendFluoNum = 0;
		ResetFillMaxVal(1);
	}

	hipMemcpyAsync(d_LocArry, h_LocArry, FluoNum*OutParaNumGS2D*sizeof(float), hipMemcpyHostToDevice, cstream);

	switch (LocPara.LocType)
	{
	case LocType_GS2D:
		// for 2d localization
		ImgRender_FluoRenderWithPrec_2D(h_LocArry, this, LocPara, RenderingMode, FixedlocPrec, FluoNum, cstream);

		break;

	case LocType_AS3D:
	case LocType_DH3D:
		// for 3d localization

		ImgRender_FluoRender_3D(h_LocArry, this, LocPara, RenderingMode, FixedlocPrec, FluoNum, cstream);

		break;

	default:
		break;
	}


}


void ImageRenderData_TypeDef::ResetFillMaxVal(int Mode)
{
	hipStream_t cstream;
	hipStreamCreate(&cstream);

	if (Mode == 0)
	{
		hipMemsetAsync(d_MaxImageVal, 0, sizeof(int), cstream);
		hipMemsetAsync(d_HistMaxDat, 0, ImgRend_MaxDatHistLen*sizeof(int), cstream);
		hipStreamSynchronize(cstream);

	}
	else
	{
		hipMemcpyAsync(h_HistMaxDat, d_HistMaxDat, ImgRend_MaxDatHistLen * sizeof(int), hipMemcpyDeviceToHost, cstream);
		hipStreamSynchronize(cstream);

		int NData = h_HistMaxDat[0];
		if (NData <= 1)NData = 1;

		for (int cnt = 0; cnt < ImgRend_MaxDatHistLen; cnt++)
		{
			h_HistMaxDat[cnt] = h_HistMaxDat[cnt] * 200 / NData;
		}

		hipMemcpyAsync(d_HistMaxDat, h_HistMaxDat, ImgRend_MaxDatHistLen * sizeof(int), hipMemcpyHostToDevice, cstream);
		hipStreamSynchronize(cstream);

	}

	hipStreamSynchronize(cstream);
	hipStreamDestroy(cstream);

}


void ImageRenderData_TypeDef::GetDispImgTop(LocalizationPara & LocPara, float BrightRatio, int oImgWidth, int oImgHigh, int cposX, int cposY, float DispZoom, hipStream_t cstream)
{
	
	if (RendType_Is2D(LocPara.LocType))
	{
		//2d image 
		ImgRender_GetDispImg_2D(this, BrightRatio, oImgWidth, oImgHigh, cposX, cposY, DispZoom, LocPara.PixelZoom, LocPara.SRImageWidth, LocPara.SRImageHigh, cstream);

	}
	else
	{
		// astigmatism 3d image
		ImgRender_GetDispImg_3D(this, LocPara.MinZDepth, LocPara.MaxZDepth, LocPara.ColorMode_3D, BrightRatio, oImgWidth, oImgHigh, cposX, cposY, DispZoom, LocPara.PixelZoom, LocPara.SRImageWidth, LocPara.SRImageHigh, cstream);

	}
	hipStreamSynchronize(cstream);

}

void ImageRenderData_TypeDef::GetSaveImgTop(LocalizationPara & LocPara, float BrightRatio, int RGBImageEncodeMode, hipStream_t cstream)
{
	if (RendType_Is2D(LocPara.LocType))
	{
		//2d image 
		ImgRender_GetSaveImg_2D(this, BrightRatio, RGBImageEncodeMode, LocPara.SRImageWidth, LocPara.SRImageHigh, cstream);

	}
	else
	{
		// astigmatism 3d image
		ImgRender_GetSaveImg_3D(this, LocPara, BrightRatio, RGBImageEncodeMode, cstream);

	}
	hipStreamSynchronize(cstream);

}

int ImageRenderData_TypeDef::GetDispMaxVal()
{
	hipStream_t cstream;
	hipStreamCreate(&cstream);

	hipMemcpyAsync(h_MaxImageVal, d_MaxImageVal, sizeof(int), hipMemcpyDeviceToHost, cstream);
	hipStreamSynchronize(cstream);

	int MaxImageVal = *h_MaxImageVal;

	float LightRatio;
	LightRatio = GetLightRatio(h_HistMaxDat, d_HistMaxDat);
	
	MaxImageVal = MaxImageVal*LightRatio;

	if (MaxImageVal > 20000000)
	{
		ResetFillMaxVal(1);
	}


	hipStreamSynchronize(cstream);
	hipStreamDestroy(cstream);

	return MaxImageVal;
}

void ImageRenderData_TypeDef::ResetFillImgTop(LocalizationPara & LocPara)
{
	if (RendType_Is2D(LocPara.LocType))
	{
		//2d image 
		ImgRender_ResetFillImg_2D(this, LocPara.SRImageWidth, LocPara.SRImageHigh);

	}
	else
	{
		// color encoded depth 3d image
		ImgRender_ResetFillImg_3D(this, LocPara.SRImageWidth, LocPara.SRImageHigh);

	}
}


void ImageRenderData_TypeDef::GetMaxImgSizeFromLocArry(float *h_LocArry, float *d_LocArry, int *MaxImgWidth, int *MaxImgHigh, int FluoNum, hipStream_t cstream)
{
	
	int *d_MaxImgWidth;
	int *d_MaxImgHigh;


	hipMalloc((void **)&d_MaxImgWidth, sizeof(int));
	hipMalloc((void **)&d_MaxImgHigh, sizeof(int));
	hipMalloc((void **)&d_LocArry, PointNumTh * 2 * OutParaNumGS2D*sizeof(float));


	hipMemsetAsync(d_MaxImgWidth, 0, sizeof(int), cstream);
	hipMemsetAsync(d_MaxImgHigh, 0, sizeof(int), cstream);


	hipMemcpyAsync(d_LocArry, h_LocArry, FluoNum*OutParaNumGS2D*sizeof(float), hipMemcpyHostToDevice, cstream);

	int BlockDim = ThreadsPerBlock;
	int BlockNum = ((FluoNum + ThreadsPerBlock - 1) / ThreadsPerBlock);

	FindMaxImgSize << <BlockNum, BlockDim, 0, cstream >> >(d_LocArry, d_MaxImgWidth, d_MaxImgHigh, FluoNum, OutParaNumGS2D);


	hipMemcpyAsync(MaxImgWidth, d_MaxImgWidth, sizeof(int), hipMemcpyDeviceToHost, cstream);
	hipMemcpyAsync(MaxImgHigh, d_MaxImgHigh, sizeof(int), hipMemcpyDeviceToHost, cstream);
	hipStreamSynchronize(cstream);


	*MaxImgWidth = (*MaxImgWidth) / 4 * 4;
	*MaxImgHigh = (*MaxImgHigh) / 4 * 4;


	hipFree(d_MaxImgWidth);
	hipFree(d_MaxImgHigh);

}

void ImageRenderData_TypeDef::Init(LocalizationPara & LocPara, int MaxDispImgWidth, int MaxDispImgHigh)
{
	// host and gpu
	hipError_t err;

	
	hipHostMalloc((void **)&h_LocArry, MaxPointNum*OutParaNumGS2D*sizeof(float));
	hipMalloc((void **)&d_LocArry, MaxPointNum*OutParaNumGS2D*sizeof(float));

	err = hipMalloc((void **)&d_SRIntensityImg, LocPara.SRImageWidth*LocPara.SRImageHigh*sizeof(float));
	HandleErr(err, "hipMalloc d_SRIntensityImg");

	if (RendType_Is2D(LocPara.LocType))
	{
		//2d image 
		d_SRColorMapImg = NULL;
	}
	else
	{
		// color encoded depth 3d image
		err = hipMalloc((void **)&d_SRColorMapImg, LocPara.SRImageWidth*LocPara.SRImageHigh*sizeof(float));
		HandleErr(err, "hipMalloc d_SRColorMapImg");
	}


	hipHostMalloc((void **)&h_MaxImageVal, sizeof(int));
	hipMalloc((void **)&d_MaxImageVal, sizeof(int));

	hipHostMalloc((void **)&h_HistMaxDat, ImgRend_MaxDatHistLen * sizeof(int));
	hipMalloc((void **)&d_HistMaxDat, ImgRend_MaxDatHistLen * sizeof(int));


	err = hipMalloc((void **)&d_DispRendImg, MaxDispImgWidth*MaxDispImgHigh * 4 * sizeof(char));
	HandleErr(err, "hipMalloc d_DispRendImg");

	err = hipMalloc((void **)&d_SaveRendImg, LocPara.SRImageWidth*LocPara.SRImageHigh * 4);
	HandleErr(err, "hipMalloc d_SaveRendImg");

	/*
	err = hipHostMalloc((void **)&h_DispRendImg, MaxDispImgWidth*MaxDispImgHigh * 4 * sizeof(char));
	HandleErr(err, "hipMalloc h_DispRendImg");

	err = hipHostMalloc((void **)&h_SaveRendImg, RawImgWidth*RawImgHigh*PixelZoom*PixelZoom * 4);
	HandleErr(err, "hipMalloc h_SaveRendImg");

	*/

	h_DispRendImg = (char *)malloc(MaxDispImgWidth*MaxDispImgHigh * 4 * sizeof(char));
	if (h_DispRendImg == NULL)
	{
		printf("malloc h_DispRendImg error\n");

	}
	h_SaveRendImg = (char *)malloc(LocPara.SRImageWidth*LocPara.SRImageHigh * 4);
	if (h_SaveRendImg == NULL)
	{
		printf("malloc h_SaveRendImg error\n");

	}

	hipHostMalloc((void **)&h_SRImageHist, SRImageHist_DataLen * sizeof(float));
	hipMalloc((void **)&d_SRImageHist, SRImageHist_DataLen * sizeof(float));



	// initial some parameters
	ResetFillImgTop(LocPara);
	ResetFillMaxVal(0);
	tRendFluoNum = 0;
}

void ImageRenderData_TypeDef::Deinit(LocalizationPara & LocPara)
{
	hipError_t err;

	hipHostFree(h_LocArry);
	hipFree(d_LocArry);

	err = hipFree(d_SRIntensityImg);
	HandleErr(err, "hipFree d_SRIntensityImg");

	if (RendType_Is2D(LocPara.LocType))
	{
		//2d image 

	}
	else
	{
		if (d_SRColorMapImg != NULL)
		{
			// color encoded depth 3d image
			err = hipFree(d_SRColorMapImg);
			HandleErr(err, "hipFree d_SRColorMapImg");

		}
	}


	hipHostFree(h_MaxImageVal);
	hipFree(d_MaxImageVal);
	hipHostFree(h_HistMaxDat);
	hipFree(d_HistMaxDat);

	hipFree(d_DispRendImg);
	err = hipFree(d_SaveRendImg);

	HandleErr(err, "hipFree d_SaveRendImg");

//	hipHostFree(h_DispRendImg);
//	hipHostFree(h_SaveRendImg);

	free(h_DispRendImg);
	free(h_SaveRendImg);


	hipHostFree(h_SRImageHist);
	hipFree(d_SRImageHist);

}



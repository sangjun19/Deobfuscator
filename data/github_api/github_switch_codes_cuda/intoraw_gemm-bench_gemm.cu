// Repository: intoraw/gemm-bench
// File: gemm.cu

#include <unistd.h>
#include <iostream>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#define STRIP_FLAG_HELP 1
#include <gflags/gflags.h>

using namespace std;

DEFINE_int32(m, 1, "m");
DEFINE_int32(n, 1, "n");
DEFINE_int32(k, 1, "k");


const char* cublasGetErrorString(hipblasStatus_t status)
{
    switch(status)
    {
        case HIPBLAS_STATUS_SUCCESS: return "HIPBLAS_STATUS_SUCCESS";
        case HIPBLAS_STATUS_NOT_INITIALIZED: return "HIPBLAS_STATUS_NOT_INITIALIZED";
        case HIPBLAS_STATUS_ALLOC_FAILED: return "HIPBLAS_STATUS_ALLOC_FAILED";
        case HIPBLAS_STATUS_INVALID_VALUE: return "HIPBLAS_STATUS_INVALID_VALUE"; 
        case HIPBLAS_STATUS_ARCH_MISMATCH: return "HIPBLAS_STATUS_ARCH_MISMATCH"; 
        case HIPBLAS_STATUS_MAPPING_ERROR: return "HIPBLAS_STATUS_MAPPING_ERROR";
        case HIPBLAS_STATUS_EXECUTION_FAILED: return "HIPBLAS_STATUS_EXECUTION_FAILED"; 
        case HIPBLAS_STATUS_INTERNAL_ERROR: return "HIPBLAS_STATUS_INTERNAL_ERROR"; 
    }
    return "unknown error";
}

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}

inline
hipblasStatus_t checkCublas(hipblasStatus_t result)
{
  if (result != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cublasGetErrorString(result));
    assert(result == HIPBLAS_STATUS_SUCCESS);
  }
  return result;
}

void CPU_fill_rand(float*A, int nr_rows_A, int nr_cols_A){
  int a = 1;
  for (int i = 0;i < nr_rows_A * nr_cols_A; i ++){
    A[i] = (float)rand()/(float)(RAND_MAX/a);
  }
}


void bench(int m, int n, int k, int repeats){
  hipblasStatus_t stat;
  hipblasHandle_t handle;

  checkCublas(hipblasCreate(&handle));

  float *h_A = (float*)malloc(m * k * sizeof(float));
  float *h_B = (float*)malloc(k * n * sizeof(float));
  float *h_C = (float*)malloc(m * n * sizeof(float));
  
  CPU_fill_rand(h_A, m, k);
  CPU_fill_rand(h_B, k, n);
  CPU_fill_rand(h_C, m, n);

  float *d_A, *d_B, *d_C;
  checkCuda(hipMallocManaged(&d_A, m * k * sizeof(float)));
  checkCuda(hipMallocManaged(&d_B, k * n * sizeof(float)));
  checkCuda(hipMallocManaged(&d_C, m * n * sizeof(float)));

  checkCuda(hipMemcpy(d_A, h_A, m * k * sizeof(float), hipMemcpyHostToDevice));
  checkCuda(hipMemcpy(d_B, h_B, k * n * sizeof(float), hipMemcpyHostToDevice));
  checkCuda(hipMemcpy(d_C, h_C, m * n * sizeof(float), hipMemcpyHostToDevice));

  int lda, ldb, ldc;
  const float alf = 1.0f;
  const float bet = 0.0f;
  const float *alpha = &alf;
  const float *beta = &bet;


  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  float sum = 0.0;
  for(int rep = 0; rep < repeats; rep ++) {
    hipEventRecord(start, 0);
    lda = m; 
    ldb = k;
    ldc = m;

    stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, d_A, lda, d_B, ldb, beta, d_C, ldc);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    if(stat != HIPBLAS_STATUS_SUCCESS){
      cerr << " hipblasSgemm failed" << endl;
      exit(1);
    }

    assert(!hipGetLastError());
    float elapsed;
    hipEventElapsedTime(&elapsed, start, stop);
    cout << m << " " << n << " " << k << " time : " << elapsed << " ms" << endl;
    sum += elapsed;
  }

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  free(h_A);
  free(h_B);
  free(h_C);
  return ;
}



int main(int argc, char **argv){
  gflags::ParseCommandLineFlags(&argc, &argv, true);


  int m, k, n;

  m = FLAGS_m;
  n = FLAGS_n;
  k = FLAGS_k;

  bench(m, n, k, 20);

  return 0;

}

#include "hip/hip_runtime.h"
// Repository: cern-nextgen/wp1.7-soa-wrapper
// File: kernel.cu

#include "kernel.h"

#include <iostream>  // This include fixes segfault due to uninitialized std::cout, even if the latter is not used

#include "gpu.h"
#include "skeleton.h"  // Needed only for forward declarations
#include "wrapper.h"

// #include <cuda/std/span>  // Should work out of the box


namespace kernel {

void print_cuda_error(hipError_t err) {
    if (err != hipSuccess) std::cerr << hipGetErrorString(err) << std::endl;
}

int cuda_malloc_managed(void** data, std::size_t size) { return hipMallocManaged(data, size); }

int cuda_free(void* ptr) { return hipFree(ptr); }

int cuda_malloc(void** d_data, std::size_t size) { return hipMalloc(d_data, size); }

int cuda_memcpy(void* to, void* from, std::size_t size, cuda_memcpy_kind kind) {
    hipError_t err;
    switch (kind) {
        case cuda_memcpy_kind::hipMemcpyHostToDevice:
        err = hipMemcpy(to, from, size, hipMemcpyHostToDevice);
        break;
        case cuda_memcpy_kind::hipMemcpyDeviceToHost:
        err = hipMemcpy(to, from, size, hipMemcpyDeviceToHost);
        break;
        default:
        err = hipError_t(-1);
    }
    print_cuda_error(err);
    return err;
}

template <
    template <class> class F,
    template <template <class> class> class S,
    wrapper::layout L
>
__global__ void add(int N, wrapper::wrapper<F, S, L> w) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += stride) w[i].y = w[i].getX() + w[i].y;
}

template <
    template <class> class F,
    template <template <class> class> class S,
    wrapper::layout L
>
void apply(int N, wrapper::wrapper<F, S, L> w) {
    add<<<1, 1>>>(N, w);
    print_cuda_error(hipDeviceSynchronize());
}

// Explicit instatiations needed for unit tests (TODO: Get rid of this)
template void apply<span_type, S, wrapper::layout::aos>(int N, wrapper::wrapper<span_type, S, wrapper::layout::aos> w);
template void apply<span_type, S, wrapper::layout::soa>(int N, wrapper::wrapper<span_type, S, wrapper::layout::soa> w);
template void apply<pointer_type, S, wrapper::layout::aos>(int N, wrapper::wrapper<pointer_type, S, wrapper::layout::aos> w);
template void apply<pointer_type, S, wrapper::layout::soa>(int N, wrapper::wrapper<pointer_type, S, wrapper::layout::soa> w);

}  // namespace kernel

#include "hip/hip_runtime.h"
// Repository: jeng1220/cuGemmProf
// File: verify.cu

/* Copyright 2020 Jeng Bai-Cheng
*
* Permission is hereby granted, free of charge, to any person obtaining a copy of
* this software and associated documentation files (the "Software"), to deal in
* the Software without restriction, including without limitation the rights to
* use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies
* of the Software, and to permit persons to whom the Software is furnished to do
* so, subject to the following conditions:
*
* The above copyright notice and this permission notice shall be included in all
* copies or substantial portions of the Software.
*
* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
* IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS
*  FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR
* COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER
* IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN
* CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
*/

#include "verify.h"
#include <cassert>
#include <cstdlib>
#include <iostream>
#include <map>
#include <hip/hip_fp16.h>
#include <thrust/complex.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/inner_product.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/transform.h>
#include "helper.h"
#include "macro.h"

template <typename data_t>
__global__ void InitMatrixKernal(void* dev_ptr, int w, int h, int ld) 
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    auto ptr = reinterpret_cast<data_t*>(dev_ptr);
    if (x < ld && y < h) {
        ptr[y * ld + x] = (x < w) ? (threadIdx.y * blockDim.x + threadIdx.x) : 0;
    }
}

template <>
__global__ void InitMatrixKernal<half>(void* dev_ptr, int w, int h, int ld) 
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    auto ptr = reinterpret_cast<half*>(dev_ptr);
    int max = blockDim.x * blockDim.y;
    float v = static_cast<float>(threadIdx.y * blockDim.x + threadIdx.x) / max;
    if (x < ld && y < h) {
        ptr[y * ld + x] = __float2half((x < w) ? v : 0.f);
    }
}

void InitMatrix(void* ptr, int w, int h, int ld, hipDataType dtype) 
{
    dim3 block(8, 8);
    dim3 grid;
    grid.x = (ld + block.x - 1) / block.x;
    grid.y = ( h + block.y - 1) / block.y;

    if (dtype == HIP_C_8I || dtype == HIP_C_32F || dtype == HIP_C_64F) {
        grid.x = (2 * ld + block.x - 1) / block.x;
    }

    switch (dtype) {

        case HIP_R_8I:
            InitMatrixKernal<char><<<grid, block>>>(ptr, w, h, ld);
            break;
        case HIP_R_16F:
            InitMatrixKernal<half><<<grid, block>>>(ptr, w, h, ld);
            break;
        case HIP_R_32F:
            InitMatrixKernal<float><<<grid, block>>>(ptr, w, h, ld);
            break;
        case HIP_R_64F:
            InitMatrixKernal<double><<<grid, block>>>(ptr, w, h, ld);
        case HIP_C_8I:
            InitMatrixKernal<char><<<grid, block>>>(ptr, 2 * w, h, 2 * ld);
            break;
        case HIP_C_32F:
            InitMatrixKernal<float><<<grid, block>>>(ptr, 2 * w, h, 2 * ld);
            break;
        case HIP_C_64F:
            InitMatrixKernal<double><<<grid, block>>>(ptr, 2 * w, h, 2 * ld);
            break;
        default:
            assert(false);
    }
    CUDA_CHECK(hipStreamSynchronize(0));
}

template <typename data_t>
__global__ void NaiveMatrixTransposeKernel(
    int w, int h,
    const void* src_ptr, void* dst_ptr)
{
    auto src = reinterpret_cast<const data_t*>(src_ptr);
    auto dst = reinterpret_cast<data_t*>(dst_ptr);

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < w && y < h) {
        dst[ x * h + y ] = src[ y * w + x ];
    }
}

void NaiveMatrixTranspose(
    int w, int h,
    const void* src, void* dst,
    hipDataType dtype)
{

    dim3 block(8, 8);
    dim3 grid;
    grid.x = (w + block.x - 1) / block.x;
    grid.y = (h + block.y - 1) / block.y;

    switch (dtype) {
        case HIP_R_8I:
            NaiveMatrixTransposeKernel<char><<<grid, block>>>(w, h, src, dst);
            break;
        case HIP_R_16F:
        case HIP_C_8I:
            NaiveMatrixTransposeKernel<half><<<grid, block>>>(w, h, src, dst);
            break;
        case HIP_R_32I:
        case HIP_R_32F:
            NaiveMatrixTransposeKernel<int><<<grid, block>>>(w, h, src, dst);
            break;
        case HIP_R_64F:
        case HIP_C_32F:
            NaiveMatrixTransposeKernel<double><<<grid, block>>>(w, h, src, dst);
            break;
        case HIP_C_64F:
            NaiveMatrixTransposeKernel<double2><<<grid, block>>>(w, h, src, dst);
            break;
        default:
            assert(false);
    }
    CUDA_CHECK(hipStreamSynchronize(0));
}

template <typename acc_t, typename src_t, typename dst_t>
__global__ void NaiveGemmKernelNN(
    int m, int n, int k,
    const void* A_ptr, int lda,
    const void* B_ptr, int ldb,
    void* C_ptr, int ldc) 
{
    auto A = reinterpret_cast<const src_t*>(A_ptr);
    auto B = reinterpret_cast<const src_t*>(B_ptr);
    auto C = reinterpret_cast<dst_t*>(C_ptr);

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    acc_t sum = 0;

    if (x < m && y < n) {
        for (int i = 0; i < k; ++i) {
            sum += static_cast<acc_t>(A[i * lda + x]) * static_cast<acc_t>(B[y * ldb + i]);
        }
        C[y * ldc + x] = static_cast<dst_t>(sum);
    }
}

template <>
__global__ void NaiveGemmKernelNN<float, half, half>(
    int m, int n, int k,
    const void* A_ptr, int lda,
    const void* B_ptr, int ldb,
    void* C_ptr, int ldc) 
{
    auto A = reinterpret_cast<const half*>(A_ptr);
    auto B = reinterpret_cast<const half*>(B_ptr);
    auto C = reinterpret_cast<half*>(C_ptr);

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    float sum = 0;

    if (x < m && y < n) {
        for (int i = 0; i < k; ++i) {
            sum += __half2float(A[i * lda + x]) * __half2float(B[y * ldb + i]);
        }
        C[y * ldc + x] = __float2half(sum);
    }
}

template <>
__global__ void NaiveGemmKernelNN<float, half, float>(
    int m, int n, int k,
    const void* A_ptr, int lda,
    const void* B_ptr, int ldb,
    void* C_ptr, int ldc) 
{
    auto A = reinterpret_cast<const half*>(A_ptr);
    auto B = reinterpret_cast<const half*>(B_ptr);
    auto C = reinterpret_cast<float*>(C_ptr);

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    float sum = 0;

    if (x < m && y < n) {
        for (int i = 0; i < k; ++i) {
            sum += __half2float(A[i * lda + x]) * __half2float(B[y * ldb + i]);
        }
        C[y * ldc + x] = sum;
    }
}

void NaiveGemmNN(
    int m, int n, int k,
    const void* A, int lda,
    const void* B, int ldb,
    void* C, int ldc,
    int gemm_type) 
{

    dim3 block(8, 8);
    dim3 grid;
    grid.x = (m + block.x - 1) / block.x;
    grid.y = (n + block.y - 1) / block.y;
    switch (gemm_type) {
        case 0: // HIP_R_16F, HIP_R_16F, HIP_R_16F, HIP_R_16F
            NaiveGemmKernelNN<float, half, half><<<grid, block>>>(m, n, k,
                A, lda, B, ldb, C, ldc);
            break;
        case 1: // HIP_R_32I, HIP_R_8I,  HIP_R_8I,  HIP_R_32I
            NaiveGemmKernelNN<int, char, int><<<grid, block>>>(m, n, k,
                A, lda, B, ldb, C, ldc);
            break;
        case 2: // HIP_R_32F, HIP_R_16F, HIP_R_16F, HIP_R_16F
            NaiveGemmKernelNN<float, half, half><<<grid, block>>>(m, n, k,
                A, lda, B, ldb, C, ldc);
            break;
        case 3: // HIP_R_32F, HIP_R_8I,  HIP_R_8I,  HIP_R_32F
            NaiveGemmKernelNN<float, char, float><<<grid, block>>>(m, n, k,
                A, lda, B, ldb, C, ldc);
            break;
        case 4: // HIP_R_32F, HIP_R_16F, HIP_R_16F, HIP_R_32F
            NaiveGemmKernelNN<float, half, float><<<grid, block>>>(m, n, k,
                A, lda, B, ldb, C, ldc);
            break;
        case 5: // HIP_R_32F, HIP_R_32F, HIP_R_32F, HIP_R_32F
            NaiveGemmKernelNN<float, float, float><<<grid, block>>>(m, n, k,
                A, lda, B, ldb, C, ldc);
            break;
        case 6: // HIP_R_64F, HIP_R_64F, HIP_R_64F, HIP_R_64F
            NaiveGemmKernelNN<double, double, double><<<grid, block>>>(m, n, k,
                A, lda, B, ldb, C, ldc);
            break;
        case 7: // HIP_C_32F, HIP_C_8I,  HIP_C_8I,  HIP_C_32F
            NaiveGemmKernelNN< thrust::complex<float>, thrust::complex<char>, thrust::complex<float> ><<<grid, block>>>(m, n, k,
                A, lda, B, ldb, C, ldc);
            break;
        case 8: // HIP_C_32F, HIP_C_32F, HIP_C_32F, HIP_C_32F
            NaiveGemmKernelNN< thrust::complex<float>, thrust::complex<float>, thrust::complex<float> ><<<grid, block>>>(m, n, k,
                A, lda, B, ldb, C, ldc);
            break;
        case 9: // HIP_C_64F, HIP_C_64F, HIP_C_64F, HIP_C_64F
            NaiveGemmKernelNN< thrust::complex<double>, thrust::complex<double>, thrust::complex<double> ><<<grid, block>>>(m, n, k,
                A, lda, B, ldb, C, ldc);
            break;
        default:
            assert(false);
    }
    CUDA_CHECK(hipStreamSynchronize(0));
}

int GetGemmTypeId(hipDataType compute_type,
    hipDataType src_type, hipDataType dst_type) 
{
    switch (compute_type) {
        case HIP_R_16F: return 0;
        case HIP_R_32I: return 1;
        case HIP_R_32F:
            switch (src_type) {
                case HIP_R_16F: return (dst_type == HIP_R_16F) ? 2 : 4;
                case HIP_R_8I: return 3;
                case HIP_R_32F: return 5;
                default: assert(false);
            }
        case HIP_R_64F: return 6;
        case HIP_C_32F: return (src_type == HIP_C_8I) ? 7 : 8;
        case HIP_C_64F: return 9;
        default: assert(false);
    }
    return -1;
}

void NaiveGemm(
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m, int n, int k,
    const void* A, hipDataType a_type, int lda,
    const void* B, hipDataType b_type, int ldb,
    void* C, hipDataType c_type, int ldc,
    hipDataType compute_type) 
{
    int src_dtype_size = DtypeToSize(a_type);
    void* dev_A = (void*)A;
    int trans_lda = lda;
    if (transa == HIPBLAS_OP_T) {
        CUDA_CHECK(hipMalloc(&dev_A, m * lda * src_dtype_size));
        NaiveMatrixTranspose(lda, m, A, dev_A, a_type);
        trans_lda = m;
    }

    void* dev_B = (void*)B;
    int trans_ldb = ldb;
    if (transb == HIPBLAS_OP_T) {
        CUDA_CHECK(hipMalloc(&dev_B, k * ldb * src_dtype_size));
        NaiveMatrixTranspose(ldb, k, B, dev_B, b_type);
        trans_ldb = k;
    }

    auto gemm_type = GetGemmTypeId(compute_type, a_type, c_type);
    NaiveGemmNN(m, n, k, dev_A, trans_lda, dev_B, trans_ldb, C, ldc, gemm_type);
    if (dev_A != A) CUDA_CHECK(hipFree(dev_A));
    if (dev_B != B) CUDA_CHECK(hipFree(dev_B));
}

template<typename T>
struct AbsMinus {
    __thrust_exec_check_disable__
    __host__ __device__ T operator()(const T &lhs, const T &rhs) const {
        return (lhs > rhs) ? lhs - rhs : rhs - lhs;
    }
};

template <typename T>
double VerifyT(const void* x_ptr, const void* y_ptr, int count) {
    auto x = reinterpret_cast<const T*>(x_ptr);
    auto y = reinterpret_cast<const T*>(y_ptr);

    thrust::device_vector<T> diff(count);
    AbsMinus<T> abs_minus_functor;
    thrust::transform(thrust::device, x, x + count,
        y, diff.begin(), abs_minus_functor);

    auto first = thrust::make_zip_iterator(thrust::make_tuple(diff.begin(), y));
    auto last  = thrust::make_zip_iterator(thrust::make_tuple(diff.end(),   y + count));

    thrust::maximum< thrust::tuple<T, T> > max_functor;
    thrust::tuple<T, T> init(-1, -1);
    auto result = thrust::reduce(thrust::device, first, last, init, max_functor);
    auto max_diff = thrust::get<0>(result);
    auto max_value = thrust::get<1>(result);

    return static_cast<double>(max_diff) / max_value;
}

struct HalfToFloat : public thrust::unary_function<half, float> {
    __host__ __device__
    float operator()(half x) { return __half2float(x); }
};

template <>
double VerifyT<half>(const void* x_ptr, const void* y_ptr, int count) {
    auto x = reinterpret_cast<const half*>(x_ptr);
    auto y = reinterpret_cast<const half*>(y_ptr);

    thrust::device_vector<float> x_fp32(count);
    thrust::device_vector<float> y_fp32(count);
    thrust::device_vector<float> diff(count);
    HalfToFloat functor;
    thrust::transform(thrust::device, x, x + count, x_fp32.begin(), functor);
    thrust::transform(thrust::device, y, y + count, y_fp32.begin(), functor);

    AbsMinus<float> abs_minus_functor;
    thrust::transform(thrust::device, x_fp32.begin(), x_fp32.end(),
        y_fp32.begin(), diff.begin(), abs_minus_functor);

    auto first = thrust::make_zip_iterator(thrust::make_tuple(diff.begin(), y_fp32.begin()));
    auto last  = thrust::make_zip_iterator(thrust::make_tuple(diff.end(),   y_fp32.end()));

    thrust::maximum< thrust::tuple<float, float> > max_functor;
    thrust::tuple<float, float> init(-1.f, -1.f);
    auto result = thrust::reduce(first, last, init, max_functor);
    auto max_diff = thrust::get<0>(result);
    auto max_value = thrust::get<1>(result);

    return static_cast<double>(max_diff) / max_value;
}

std::ostream& operator<<(std::ostream& os, const half& x) {
    os << __half2float(x);
    return os;
}

double Verify(const void* x, const void* y, int count, hipDataType dtype) {
    switch (dtype) {
        case HIP_R_16F:
            return VerifyT<half>(x, y, count);
        case HIP_R_32I:
            return VerifyT<int>(x, y, count);
        case HIP_R_32F:
            return VerifyT<float>(x, y, count);
        case HIP_R_64F:
            return VerifyT<double>(x, y, count);
        case HIP_C_32F:
            return VerifyT<float>(x, y, 2 * count);
        case HIP_C_64F:
            return VerifyT<double>(x, y, 2 * count);
        default:
            assert(false);
    }
    return std::numeric_limits<double>::max();
}

template <typename data_t>
void PrintMatrixT(const void* ptr, int w, int h, int ld)
{
    auto dev_ptr = reinterpret_cast<const data_t*>(ptr);
    size_t size = ld * h * sizeof(data_t);
    data_t* host_ptr = (data_t*)malloc(size);
    CUDA_CHECK(hipMemcpy(host_ptr, dev_ptr, size, hipMemcpyDeviceToHost));

    for (int y = 0; y < h; ++y) {
        for (int x = 0; x < ld; ++x) {
            std::cout << +host_ptr[y * ld + x] << ", ";
        }
        std::cout << std::endl;
    }
    std::cout << "\n" << std::endl;
    free(host_ptr);
}

template <>
void PrintMatrixT<half>(const void* ptr, int w, int h, int ld)
{
    auto dev_ptr = reinterpret_cast<const half*>(ptr);
    size_t size = ld * h * sizeof(half);
    half* host_ptr = (half*)malloc(size);
    CUDA_CHECK(hipMemcpy(host_ptr, dev_ptr, size, hipMemcpyDeviceToHost));

    for (int y = 0; y < h; ++y) {
        for (int x = 0; x < ld; ++x) {
            std::cout << __half2float(host_ptr[y * ld + x]) << ", ";
        }
        std::cout << std::endl;
    }
    std::cout << "\n" << std::endl;
    free(host_ptr);
}

void PrintMatrix(const void* dev_ptr, int w, int h,
    int ld, hipDataType dtype)
{
    switch (dtype) {
        case HIP_R_8I:
            PrintMatrixT<char>(dev_ptr, w, h, ld);
            break;
        case HIP_R_16F:
            PrintMatrixT<half>(dev_ptr, w, h, ld);
            break;
        case HIP_R_32I:
            PrintMatrixT<int>(dev_ptr, w, h, ld);
            break;
        case HIP_R_32F:
            PrintMatrixT<float>(dev_ptr, w, h, ld);
            break;
        case HIP_R_64F:
            PrintMatrixT<double>(dev_ptr, w, h, ld);
            break;
        case HIP_C_8I:
            PrintMatrixT<char>(dev_ptr, 2 * w, h, 2 * ld);
            break;
        case HIP_C_32F:
            PrintMatrixT<float>(dev_ptr, 2 * w, h, 2 * ld);
            break;
        case HIP_C_64F:
            PrintMatrixT<double>(dev_ptr, 2 * w, h, 2 * ld);
            break;
        default:
            assert(false);
    }
}

#include "hip/hip_runtime.h"
// Repository: MikkelStorgaard/pmph_project
// File: Project/Lattice Problem/Code/cuda_version/colonies3D.cu

#include "colonies3D.hpp"
#include "colonies3D_kernels.cu.h"
#include <chrono>

#define GPU_NC true                 // Works
#define GPU_MAXOCCUPANCY true       // Works
#define GPU_BIRTH true              // Works
#define GPU_INFECTIONS true         // Works
#define GPU_NEWINFECTIONS true	    // Works
#define GPU_PHAGEDECAY true         // Works
#define GPU_MOVEMENT true           // Works
#define GPU_SWAPZERO true           // Works
#define GPU_UPDATEOCCUPANCY true    // Works
#define GPU_NUTRIENTDIFFUSION true  // Works
#define GPU_SWAPZERO2 true          // Works

#define GPU_KERNEL_TIMING true

// Different optimization tests
#define GPU_REDUCE_ARRAYS true
#define GPU_REDUCE_ARRAYS_EXPORT true

#define GPU_COPY_TO_SHARED false

#define OPTIMIZED_MAXOCCUPANCY true // Needs GPU_MAXOCCUPANCY to be true!



using namespace std;
using namespace std::chrono;


// Constructers /////////////////////////////////////////////////////////////////////////
// Direct constructer
Colonies3D::Colonies3D(numtype B_0, numtype P_0){

	// Store the initial densities
	this->B_0 = B_0;
	this->P_0 = P_0;

	// Set some default parameters (initlize some default objects)
	K                       = 1.0 / 5.0;//          Half-Speed constant
	n_0                     = 1e9;      // [1/ml]   Initial nutrient level (Carrying capacity per ml)

	L                       = 1e4;      // [µm]     Side-length of simulation array
	H                       = L;        // [µm]     Height of the simulation array
	nGridXY                 = 100000;       //          Number of gridpoints
	nGridZ                  = nGridXY;  //          Number of gridpoints
 	volume                  = nGridXY*nGridXY*nGridZ;

	nSamp                   = 1000;       //          Number of samples to save per simulation hour

	g                       = 2;        // [1/h]    Doubling rate for the cells

	alpha                   = 0.5;      //          Percentage of phages which reinfect the colony upon lysis
	beta                    = 100;      //          Multiplication factor phage
	eta                     = 1e4;      // [µm^3/h] Adsorption coefficient
	delta                   = 1.0/10.0; // [1/h]    Rate of phage decay
	r                       = 10.0/0.5; //          Constant used in the time-delay mechanism
	zeta                    = 1.0;      //          permeability of colony surface

	D_P                     = 1e4;      // [µm^2/h] Diffusion constant for the phage
	D_B                     = 0;//D_P/20;   // [µm^2/h] Diffusion constant for the cells
	D_n                     = 25e5;     // [µm^2/h] Diffusion constant for the nutrient

	T                       = 0;        // [h]      Current time
	dT                      = -1;       // [h]      Time-step size (-1 to compute based on fastest diffusion rate)
	T_end                   = 0;        // [h]      End time of simulation
	T_i                     = -1;       // [h]      Time when the phage infections begins (less than 0 disables phage infection)

	initialOccupancy        = 0;        // Number of gridpoints occupied initially;

	exit                    = false;    // Boolean to control early exit

	Warn_g                  = false;    //
	Warn_r                  = false;    //
	Warn_eta                = false;    // Booleans to keep track of warnings
	Warn_delta              = false;    //
	Warn_density            = false;    //
	Warn_fastGrowth         = false;    //

	experimentalConditions  = false;    // Booleans to control simulation type

	clustering              = true;     // When false, the ((B+I)/nC)^(1/3) factor is removed.
	shielding               = true;     // When true the simulation uses the shielding function (full model)
	reducedBeta             = false;    // When true the simulation modifies the burst size by the growthfactor

	reducedBoundary         = false;    // When true, bacteria are spawned at X = 0 and Y = 0. And phages are only spawned within nGrid boxes from (0,0,z).
	s                       = 1;

	fastExit                = false;     // Stop simulation when all cells are dead

	exportAll               = false;    // Boolean to export everything, not just populationsize

	rngSeed                 = -1;       // Random number seed  ( set to -1 if unused )
	errC                    = 10;



}

///////////////////////////////////////////////////////////////////////
// CPU Loop start
//////////////////////////////////////////////////////////////////////

inline int cpu_round(numtype x){
	#if NUMTYPE_IS_FLOAT
		return static_cast<int>(roundf(x));
	#else
		return static_cast<int>(round(x));
	#endif
}

inline numtype cpu_exp(numtype x){
	#if NUMTYPE_IS_FLOAT
		return expf(x);
	#else
		return exp(x);
	#endif
}


////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////
//  Just a separator between CPU and GPU to make it easier to spot when scrolling
///////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////



int Colonies3D::Run_LoopDistributed_GPU(numtype T_end) {
	std::string filename_suffix = "loopDistributedGPU";

	this->T_end = T_end;

	// Get start time
	time_t  tic;
	time(&tic);

	// Get start time
	high_resolution_clock::time_point kernel_start;
	high_resolution_clock::duration kernel_elapsed;


	// Generate a path
	path = GeneratePath();

	// Initilize th e simulation matrices
	Initialize();

	// Export data
	ExportData_arr(T,filename_suffix);

  	if (GPU_KERNEL_TIMING){
      std::string s = "kernel_timings_GPU_n" + std::to_string(nGridXY);
		OpenFileStream(f_kerneltimings, s);
    f_kerneltimings << "NC \t";
    f_kerneltimings << "MAXOCCUPANCY \t";
    f_kerneltimings << "BIRTH \t";
    f_kerneltimings << "INFECTIONS \t";
    f_kerneltimings << "NEWINFECTIONS \t";
    f_kerneltimings << "PHAGEDECAY \t";
    f_kerneltimings << "MOVEMENT \t";
    f_kerneltimings << "SWAPZERO \t";
    f_kerneltimings << "UPDATEOCCUPANCY \t";
    f_kerneltimings << "NUTRIENTDIFFUSION \t";
    f_kerneltimings << "SWAPZERO2";
		f_kerneltimings << "\n";
    }

	// Determine the number of samples to take
	int nSamplings = nSamp*T_end;

	/* Allocate arrays on the device */
	int totalElements = nGridXY * nGridXY * nGridZ;
	int totalMemSize = totalElements * sizeof(numtype);
	int blockSize = 256;
	int gridSize = (totalElements + blockSize - 1) / blockSize;
	//int gridSize = ceil((double)totalElements / (double)blockSize);
	hipError_t err = hipSuccess;

	// Allocate on GPU
	numtype *maxOccupancy = new numtype;
	numtype *d_arr_partialSum;

	err = hipMalloc((void**)&d_arr_nC , totalMemSize);
	if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to allocate arr_nC on the device! error = %s\n", hipGetErrorString(err)); errC--;}

	err = hipMalloc((void**)&d_arr_Occ, totalMemSize);
	if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to allocate arr_Occ on the device! error = %s\n", hipGetErrorString(err)); errC--;}

	err = hipMalloc((void**)&d_arr_IsActive, blockSize*gridSize*sizeof(bool));
	if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to allocate arr_IsActive on the device! error = %s\n", hipGetErrorString(err)); errC--;}

	err = hipMalloc((void**)&d_arr_partialSum, sizeof(numtype)*gridSize);
	if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to allocate arr_partialSum on the device! error = %s\n", hipGetErrorString(err)); errC--;}

	err = hipMalloc((void**)&d_rng_state, sizeof(hiprandState)*totalElements);
	if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to allocate d_rng_state on the device! error = %s\n", hipGetErrorString(err)); errC--;}

	err = hipMemcpy(d_rng_state, rng_state, sizeof(hiprandState)*totalElements, hipMemcpyHostToDevice);
	if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to copy rng_state to the device! error = %s\n", hipGetErrorString(err)); errC--;}

	err = hipMalloc((void**)&d_arr_B, totalMemSize);
	if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to allocate arr_B on the device! error = %s\n", hipGetErrorString(err)); errC--;}

	err = hipMalloc((void**)&d_arr_B_new, totalMemSize);
	if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to allocate arr_B_new on the device! error = %s\n", hipGetErrorString(err)); errC--;}

	err = hipMalloc((void**)&d_arr_P, totalMemSize);
	if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to allocate arr_P on the device! error = %s\n", hipGetErrorString(err)); errC--;}

	err = hipMalloc((void**)&d_arr_P_new, totalMemSize);
	if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to allocate arr_P_new on the device! error = %s\n", hipGetErrorString(err)); errC--;}

	err = hipMalloc((void**)&d_arr_I0, totalMemSize);
	if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to allocate arr_I0 on the device! error = %s\n", hipGetErrorString(err)); errC--;}

	err = hipMalloc((void**)&d_arr_I0_new, totalMemSize);
	if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to allocate arr_I0_new on the device! error = %s\n", hipGetErrorString(err)); errC--;}

	err = hipMalloc((void**)&d_arr_I1, totalMemSize);
	if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to allocate arr_I1 on the device! error = %s\n", hipGetErrorString(err)); errC--;}

	err = hipMalloc((void**)&d_arr_I1_new, totalMemSize);
	if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to allocate arr_I1_new on the device! error = %s\n", hipGetErrorString(err)); errC--;}

	err = hipMalloc((void**)&d_arr_I2, totalMemSize);
	if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to allocate arr_I2 on the device! error = %s\n", hipGetErrorString(err)); errC--;}

	err = hipMalloc((void**)&d_arr_I2_new, totalMemSize);
	if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to allocate arr_I2_new on the device! error = %s\n", hipGetErrorString(err)); errC--;}

	err = hipMalloc((void**)&d_arr_I3, totalMemSize);
	if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to allocate arr_I3 on the device! error = %s\n", hipGetErrorString(err)); errC--;}

	err = hipMalloc((void**)&d_arr_I3_new, totalMemSize);
	if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to allocate arr_I3_new on the device! error = %s\n", hipGetErrorString(err)); errC--;}

	err = hipMalloc((void**)&d_arr_I4, totalMemSize);
	if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to allocate arr_I4 on the device! error = %s\n", hipGetErrorString(err)); errC--;}

	err = hipMalloc((void**)&d_arr_I4_new, totalMemSize);
	if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to allocate arr_I4_new on the device! error = %s\n", hipGetErrorString(err)); errC--;}

	err = hipMalloc((void**)&d_arr_I5, totalMemSize);
	if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to allocate arr_I5 on the device! error = %s\n", hipGetErrorString(err)); errC--;}

	err = hipMalloc((void**)&d_arr_I5_new, totalMemSize);
	if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to allocate arr_I5_new on the device! error = %s\n", hipGetErrorString(err)); errC--;}

	err = hipMalloc((void**)&d_arr_I6, totalMemSize);
	if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to allocate arr_I6 on the device! error = %s\n", hipGetErrorString(err)); errC--;}

	err = hipMalloc((void**)&d_arr_I6_new, totalMemSize);
	if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to allocate arr_I6_new on the device! error = %s\n", hipGetErrorString(err)); errC--;}

	err = hipMalloc((void**)&d_arr_I7, totalMemSize);
	if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to allocate arr_I7 on the device! error = %s\n", hipGetErrorString(err)); errC--;}

	err = hipMalloc((void**)&d_arr_I7_new, totalMemSize);
	if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to allocate arr_I7_new on the device! error = %s\n", hipGetErrorString(err)); errC--;}

	err = hipMalloc((void**)&d_arr_I8, totalMemSize);
	if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to allocate arr_I8 on the device! error = %s\n", hipGetErrorString(err)); errC--;}

	err = hipMalloc((void**)&d_arr_I8_new, totalMemSize);
	if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to allocate arr_I8_new on the device! error = %s\n", hipGetErrorString(err)); errC--;}

	err = hipMalloc((void**)&d_arr_I9, totalMemSize);
	if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to allocate arr_I9 on the device! error = %s\n", hipGetErrorString(err)); errC--;}

	err = hipMalloc((void**)&d_arr_I9_new, totalMemSize);
	if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to allocate arr_I9_new on the device! error = %s\n", hipGetErrorString(err)); errC--;}

	err = hipMalloc((void**)&d_arr_M, totalMemSize);
	if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to allocate arr_M on the device! error = %s\n", hipGetErrorString(err)); errC--;}

	err = hipMalloc((void**)&d_arr_p, totalMemSize);
	if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to allocate arr_p to the device! error = %s\n", hipGetErrorString(err)); errC--;}

	err = hipMalloc((void**)&d_arr_nutrient, totalMemSize);
	if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to allocate arr_nutrient on the device! error = %s\n", hipGetErrorString(err)); errC--;}

	err = hipMalloc((void**)&d_arr_nutrient_new, totalMemSize);
	if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to allocate arr_nutrient_new on the device! error = %s\n", hipGetErrorString(err)); errC--;}

	err = hipMalloc((void**)&d_arr_GrowthModifier, totalMemSize);
	if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to allocate arr_GrowthModifier to the device! error = %s\n", hipGetErrorString(err)); errC--;}

	err = hipMalloc((void**)&d_arr_n_0, totalMemSize);
	if (err != hipSuccess && errC > 0) {fprintf(stderr, "Failed to allocate arr_n_0 to the device! error = %s\n", hipGetErrorString(err)); errC--;}

	err = hipMalloc((void**)&d_arr_n_u, totalMemSize);
	if (err != hipSuccess && errC > 0) {fprintf(stderr, "Failed to allocate arr_n_u to the device! error = %s\n", hipGetErrorString(err)); errC--;}

	err = hipMalloc((void**)&d_arr_n_d, totalMemSize);
	if (err != hipSuccess && errC > 0) {fprintf(stderr, "Failed to allocate arr_n_d to the device! error = %s\n", hipGetErrorString(err)); errC--;}

	err = hipMalloc((void**)&d_arr_n_l, totalMemSize);
	if (err != hipSuccess && errC > 0) {fprintf(stderr, "Failed to allocate arr_n_l to the device! error = %s\n", hipGetErrorString(err)); errC--;}

	err = hipMalloc((void**)&d_arr_n_r, totalMemSize);
	if (err != hipSuccess && errC > 0) {fprintf(stderr, "Failed to allocate arr_n_r to the device! error = %s\n", hipGetErrorString(err)); errC--;}

	err = hipMalloc((void**)&d_arr_n_f, totalMemSize);
	if (err != hipSuccess && errC > 0) {fprintf(stderr, "Failed to allocate arr_n_f to the device! error = %s\n", hipGetErrorString(err)); errC--;}

	err = hipMalloc((void**)&d_arr_n_b, totalMemSize);
	if (err != hipSuccess && errC > 0) {fprintf(stderr, "Failed to allocate arr_n_b to the device! error = %s\n", hipGetErrorString(err)); errC--;}

	err = hipMalloc((void**)&d_Warn_g, sizeof(bool));
	if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to allocate Warn_g on the device! error = %s\n", hipGetErrorString(err)); errC--;}

	err = hipMalloc((void**)&d_Warn_fastGrowth, sizeof(bool));
	if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to allocate d_Warn_fastGrowth on the device! error = %s\n", hipGetErrorString(err)); errC--;}

	err = hipMalloc((void**)&d_Warn_r, sizeof(bool));
	if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to allocate Warn_r on the device! error = %s\n", hipGetErrorString(err)); errC--;}

	err = hipMalloc((void**)&d_Warn_delta, sizeof(bool));
	if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to allocate Warn_delta on the device! error = %s\n", hipGetErrorString(err)); errC--;}

    err = hipMalloc((void**)&d_Warn_density, sizeof(bool));
	if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to allocate Warn_density on the device! error = %s\n", hipGetErrorString(err)); errC--;}

	initRNG<<<gridSize,blockSize>>>(d_rng_state, totalElements);


	// cudaMemCpy to device
	//CopyAllToDevice();

	// Loop over samplings
	for (int n = 0; n < nSamplings; n++) {
		if (exit) break;

		// Determine the number of timesteps between sampings
		int nStepsPerSample = static_cast<int>(cpu_round(1 / (nSamp *  dT)));

		for (int t = 0; t < nStepsPerSample; t++) {
			if (exit) break;

			// Increase time
			T += dT;

			// Spawn phages
			if ((T_i >= 0) and (abs(T - T_i) < dT / 2)) {
				spawnPhages();
				T_i = -1;
			}

			// Reset density counter
			numtype maxOccupancy = 0.0;

			// /////////////////////////////////////////////////////
			// // Main loop start //////////////////////////////////
			// /////////////////////////////////////////////////////

			/* Do all the allocations and other CUDA device stuff here
			 * remember to do them outside the nSamplings loop afterwards
			 */

      if (GPU_KERNEL_TIMING){
        kernel_start = high_resolution_clock::now();
      }

			if (GPU_NC){

				// Copy to the device
				if (((t == 0) && (n == 0)) || !GPU_SWAPZERO2) {
					CopyAllToDevice();
				}

				// Run first Kernel
				FirstKernel<<<gridSize, blockSize>>>(d_arr_Occ, d_arr_nC, totalElements);
        hipDeviceSynchronize();

				err = hipGetLastError();
				if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failure in FirstKernel! error = %s\n", hipGetErrorString(err)); errC--;}

				// Copy data back from device
				if(!GPU_MAXOCCUPANCY) {
					CopyAllToHost();
				}

			} else {
				for (int i = 0; i < nGridXY; i++) {
					if (exit) break;

					for (int j = 0; j < nGridXY; j++) {
						if (exit) break;

						for (int k = 0; k < nGridZ; k++) {
							if (exit) break;

							// Ensure nC is updated
							if (arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k] < arr_nC[i*nGridXY*nGridZ + j*nGridZ + k]){
								arr_nC[i*nGridXY*nGridZ + j*nGridZ + k] = arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k];
							}
						}
					}
				}
			}
      if (GPU_KERNEL_TIMING){
        kernel_elapsed = high_resolution_clock::now() - kernel_start;
        f_kerneltimings << duration_cast<microseconds>(kernel_elapsed).count() << "\t";
      }

			SetIsActive<<<gridSize, blockSize>>>(d_arr_Occ, d_arr_P, d_arr_IsActive, totalElements);

			err = hipGetLastError();
			if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failure in SetIsActive! error = %s\n", hipGetErrorString(err)); errC--;}


      if (GPU_KERNEL_TIMING){
        hipDeviceSynchronize();
        kernel_start = high_resolution_clock::now();
      }
			if (GPU_MAXOCCUPANCY) {

				// Copy to the device
				if(!GPU_NC) {
					CopyAllToDevice();
				}

				if(!OPTIMIZED_MAXOCCUPANCY){
				// Run second Kernel
				SecondKernel<<<gridSize, blockSize, blockSize*sizeof(numtype)>>>(d_arr_Occ, d_arr_nC, d_arr_partialSum, d_arr_IsActive, blockSize);
				}

				if( OPTIMIZED_MAXOCCUPANCY){
                numtype thr = L * L * H / (nGridXY * nGridXY * nGridZ);

                MaxOccupancyOpt<<<gridSize, blockSize>>>(d_arr_Occ, d_Warn_density, thr,d_arr_IsActive);
				}

				err = hipGetLastError();
				if (err != hipSuccess && errC > 0) {fprintf(stderr, "Failure in SecondKernel! error = %s\n", hipGetErrorString(err)); errC--;}

				// This places the maximum occupancy in d_arr_partialSum[0]

				if( !OPTIMIZED_MAXOCCUPANCY){
				SequentialReduceMax<<<1,1>>>(d_arr_partialSum, gridSize);
				err = hipGetLastError();
				if (err != hipSuccess && errC > 0) {fprintf(stderr, "Failure in SequentialReduceMax! error = %s\n", hipGetErrorString(err)); errC--;}
				}

				// Copy data back from device
				if(!GPU_BIRTH) {
					CopyAllToHost();
				}
				if( !OPTIMIZED_MAXOCCUPANCY){
				err = hipMemcpy(&maxOccupancy, d_arr_partialSum, sizeof(numtype), hipMemcpyDeviceToHost);
				if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to copy arr_partialSum to the host! error = %s\n", hipGetErrorString(err));
					errC--; }
				}

			} else {
				for (int i = 0; i < nGridXY; i++) {
					if (exit) break;

					for (int j = 0; j < nGridXY; j++) {
						if (exit) break;

						for (int k = 0; k < nGridZ; k++) {
							if (exit) break;

							// Skip empty sites
							if ((arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k] < 1) and (arr_P[i*nGridXY*nGridZ + j*nGridZ + k] < 1)) continue;

							// Record the maximum observed density
							if (arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k] > maxOccupancy) maxOccupancy = arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k];

						}
					}
				}
			}
      if (GPU_KERNEL_TIMING){
        kernel_elapsed = high_resolution_clock::now() - kernel_start;
        f_kerneltimings << duration_cast<microseconds>(kernel_elapsed).count() << "\t";
      }



			// Birth //////////////////////////////////////////////////////////////////////
      if (GPU_KERNEL_TIMING){
        kernel_start = high_resolution_clock::now();
      }

			if (GPU_BIRTH){

				// Copy to the device
				if(!GPU_MAXOCCUPANCY) {
					CopyAllToDevice();
				}

				ComputeBirthEvents<<<gridSize, blockSize>>>(d_arr_B, d_arr_B_new, d_arr_nutrient, d_arr_GrowthModifier, K, g, dT, d_Warn_g, d_Warn_fastGrowth, d_rng_state, d_arr_IsActive);

				err = hipGetLastError();
				if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failure in ComputeBirthEvents! error = %s\n", hipGetErrorString(err)); errC--;}

				// Copy data back from device
				if(!GPU_INFECTIONS) {
					CopyAllToHost();
				}


			} else {
				for (int i = 0; i < nGridXY; i++) {
					if (exit) break;

					for (int j = 0; j < nGridXY; j++) {
						if (exit) break;

						for (int k = 0; k < nGridZ; k++) {
							if (exit) break;

							// Skip empty sites
							if ((arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k] < 1) and (arr_P[i*nGridXY*nGridZ + j*nGridZ + k] < 1)) {
								skipArray[i*nGridXY*nGridZ + j*nGridZ + k] = true;
								continue;
							} else {
								skipArray[i*nGridXY*nGridZ + j*nGridZ + k] = false;
							}

							numtype p = 0; // privatize
							numtype N = 0; // privatize

							// Compute the growth modifier
							numtype growthModifier = arr_nutrient[i*nGridXY*nGridZ + j*nGridZ + k] / (arr_nutrient[i*nGridXY*nGridZ + j*nGridZ + k] + K);
							arr_GrowthModifier[i*nGridXY*nGridZ + j*nGridZ + k] = growthModifier;

							p = g * growthModifier*dT;
							if (arr_nutrient[i*nGridXY*nGridZ + j*nGridZ + k] < 1) {		//
								p = 0;
							}

							if ((p > 0.1) and (!Warn_g)) {
								cout << "\tWarning: Birth Probability Large!" << "\n";
								f_log  << "Warning: Birth Probability Large!" << "\n";
								Warn_g = true;
							}

							/* BEGIN anden Map-kernel */
							N = ComputeEvents(arr_B[i*nGridXY*nGridZ + j*nGridZ + k], p, 1, i, j, k);
							// Ensure there is enough nutrient
							if ( N > arr_nutrient[i*nGridXY*nGridZ + j*nGridZ + k] ) {
								if (!Warn_fastGrowth) {
									cout << "\tWarning: Colonies growing too fast!" << "\n";
									f_log  << "Warning: Colonies growing too fast!" << "\n";
									Warn_fastGrowth = true;
								}

								// DETERMINITIC CHANGE
                            	// N = round( arr_nutrient[i*nGridXY*nGridZ + j*nGridZ + k] );
                            	N = arr_nutrient[i*nGridXY*nGridZ + j*nGridZ + k];
							}

							// Update count
							arr_B_new[i*nGridXY*nGridZ + j*nGridZ + k] += N;
							arr_nutrient[i*nGridXY*nGridZ + j*nGridZ + k] = max(0.0, arr_nutrient[i*nGridXY*nGridZ + j*nGridZ + k] - N);
							/* END anden Map-kernel */
						}
					}
				}
			}

      if (GPU_KERNEL_TIMING){
        hipDeviceSynchronize();
        kernel_elapsed = high_resolution_clock::now() - kernel_start;
        f_kerneltimings << duration_cast<microseconds>(kernel_elapsed).count() << "\t";
      }

			if (GPU_BIRTH) {	// We still need to compute the skip array
				for (int i = 0; i < nGridXY; i++) {
					for (int j = 0; j < nGridXY; j++) {
						for (int k = 0; k < nGridZ; k++) {

							// Skip empty sites
							if ((arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k] < 1) and (arr_P[i*nGridXY*nGridZ + j*nGridZ + k] < 1)) {
								skipArray[i*nGridXY*nGridZ + j*nGridZ + k] = true;
								continue;
							} else {
								skipArray[i*nGridXY*nGridZ + j*nGridZ + k] = false;
							}
						}
					}
				}
			}

      if (GPU_KERNEL_TIMING){
        kernel_start = high_resolution_clock::now();
      }
			if (GPU_INFECTIONS){

				// Copy to the device
				if(!GPU_BIRTH) {
					CopyAllToDevice();
				}

				// Infections kernels
				BurstingEvents<<<gridSize, blockSize>>>(d_arr_I9, d_arr_P_new, d_arr_Occ, d_arr_GrowthModifier, d_arr_M, d_arr_p, alpha, beta, r, dT, reducedBeta, d_Warn_r, d_rng_state, d_arr_IsActive, totalElements);
				NonBurstingEvents<<<gridSize, blockSize>>>(d_arr_I8, d_arr_I9, d_arr_p, d_rng_state, d_arr_IsActive);
				NonBurstingEvents<<<gridSize, blockSize>>>(d_arr_I7, d_arr_I8, d_arr_p, d_rng_state, d_arr_IsActive);
				NonBurstingEvents<<<gridSize, blockSize>>>(d_arr_I6, d_arr_I7, d_arr_p, d_rng_state, d_arr_IsActive);
				NonBurstingEvents<<<gridSize, blockSize>>>(d_arr_I5, d_arr_I6, d_arr_p, d_rng_state, d_arr_IsActive);
				NonBurstingEvents<<<gridSize, blockSize>>>(d_arr_I4, d_arr_I5, d_arr_p, d_rng_state, d_arr_IsActive);
				NonBurstingEvents<<<gridSize, blockSize>>>(d_arr_I3, d_arr_I4, d_arr_p, d_rng_state, d_arr_IsActive);
				NonBurstingEvents<<<gridSize, blockSize>>>(d_arr_I2, d_arr_I3, d_arr_p, d_rng_state, d_arr_IsActive);
				NonBurstingEvents<<<gridSize, blockSize>>>(d_arr_I1, d_arr_I2, d_arr_p, d_rng_state, d_arr_IsActive);
				NonBurstingEvents<<<gridSize, blockSize>>>(d_arr_I0, d_arr_I1, d_arr_p, d_rng_state, d_arr_IsActive);
        hipDeviceSynchronize();

				err = hipGetLastError();
				if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failure in BurstingEvents or NonBurstingEvents! error = %s\n", hipGetErrorString(err)); errC--;}

				// Copy data back from device
				if(!GPU_NEWINFECTIONS) {
					CopyAllToHost();
				}

			} else {

				for (int i = 0; i < nGridXY; i++) {
					if (exit) break;

					for (int j = 0; j < nGridXY; j++) {
						if (exit) break;

						for (int k = 0; k < nGridZ; k++) {
							if (exit) break;

							// Skip empty sites
							if (skipArray[i*nGridXY*nGridZ + j*nGridZ + k]) continue;

							numtype p = 0; // privatize
							numtype N = 0; // privatize

							// Compute the growth modifier
							numtype growthModifier = arr_GrowthModifier[i*nGridXY*nGridZ + j*nGridZ + k];

							// Compute beta
							numtype Beta = beta;

							if (reducedBeta) {
								Beta *= growthModifier;
							}

							if (r > 0.0){

								p = r*growthModifier*dT;
								if ((p > 0.25) and (!Warn_r)) {
									cout << "\tWarning: Infection Increase Probability Large!" << "\n";
									f_log  << "Warning: Infection Increase Probability Large!" << "\n";
									Warn_r = true;
								}
								N = ComputeEvents(arr_I9[i*nGridXY*nGridZ + j*nGridZ + k], p, 2, i, j, k);  // Bursting events

								// Update count
								arr_I9[i*nGridXY*nGridZ + j*nGridZ + k]    = max(0.0, arr_I9[i*nGridXY*nGridZ + j*nGridZ + k] - N);
								arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k]   = max(0.0, arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k] - N);
								// DETERMINITIC CHANGE
								// arr_P_new[i*nGridXY*nGridZ + j*nGridZ + k] += round( (1 - alpha) * Beta * N);   // Phages which escape the colony
								// arr_M[i*nGridXY*nGridZ + j*nGridZ + k] = round(alpha * Beta * N);                        // Phages which reinfect the colony
								arr_P_new[i*nGridXY*nGridZ + j*nGridZ + k] += (1 - alpha) * Beta * N;   // Phages which escape the colony
								arr_M[i*nGridXY*nGridZ + j*nGridZ + k] = alpha * Beta * N;

								// Non-bursting events
								N = ComputeEvents(arr_I8[i*nGridXY*nGridZ + j*nGridZ + k], p, 2, i, j, k);
								arr_I8[i*nGridXY*nGridZ + j*nGridZ + k] = max(0.0, arr_I8[i*nGridXY*nGridZ + j*nGridZ + k] - N);
								arr_I9[i*nGridXY*nGridZ + j*nGridZ + k] += N;

								N = ComputeEvents(arr_I7[i*nGridXY*nGridZ + j*nGridZ + k], p, 2, i, j, k);
								arr_I7[i*nGridXY*nGridZ + j*nGridZ + k] = max(0.0, arr_I7[i*nGridXY*nGridZ + j*nGridZ + k] - N);
								arr_I8[i*nGridXY*nGridZ + j*nGridZ + k] += N;

								N = ComputeEvents(arr_I6[i*nGridXY*nGridZ + j*nGridZ + k], p, 2, i, j, k);
								arr_I6[i*nGridXY*nGridZ + j*nGridZ + k] = max(0.0, arr_I6[i*nGridXY*nGridZ + j*nGridZ + k] - N);
								arr_I7[i*nGridXY*nGridZ + j*nGridZ + k] += N;

								N = ComputeEvents(arr_I5[i*nGridXY*nGridZ + j*nGridZ + k], p, 2, i, j, k);
								arr_I5[i*nGridXY*nGridZ + j*nGridZ + k] = max(0.0, arr_I5[i*nGridXY*nGridZ + j*nGridZ + k] - N);
								arr_I6[i*nGridXY*nGridZ + j*nGridZ + k] += N;

								N = ComputeEvents(arr_I4[i*nGridXY*nGridZ + j*nGridZ + k], p, 2, i, j, k);
								arr_I4[i*nGridXY*nGridZ + j*nGridZ + k] = max(0.0, arr_I4[i*nGridXY*nGridZ + j*nGridZ + k] - N);
								arr_I5[i*nGridXY*nGridZ + j*nGridZ + k] += N;

								N = ComputeEvents(arr_I3[i*nGridXY*nGridZ + j*nGridZ + k], p, 2, i, j, k);
								arr_I3[i*nGridXY*nGridZ + j*nGridZ + k] = max(0.0, arr_I3[i*nGridXY*nGridZ + j*nGridZ + k] - N);
								arr_I4[i*nGridXY*nGridZ + j*nGridZ + k] += N;

								N = ComputeEvents(arr_I2[i*nGridXY*nGridZ + j*nGridZ + k], p, 2, i, j, k);
								arr_I2[i*nGridXY*nGridZ + j*nGridZ + k] = max(0.0, arr_I2[i*nGridXY*nGridZ + j*nGridZ + k] - N);
								arr_I3[i*nGridXY*nGridZ + j*nGridZ + k] += N;

								N = ComputeEvents(arr_I1[i*nGridXY*nGridZ + j*nGridZ + k], p, 2, i, j, k);
								arr_I1[i*nGridXY*nGridZ + j*nGridZ + k] = max(0.0, arr_I1[i*nGridXY*nGridZ + j*nGridZ + k] - N);
								arr_I2[i*nGridXY*nGridZ + j*nGridZ + k] += N;

								N = ComputeEvents(arr_I0[i*nGridXY*nGridZ + j*nGridZ + k], p, 2, i, j, k);
								arr_I0[i*nGridXY*nGridZ + j*nGridZ + k] = max(0.0, arr_I0[i*nGridXY*nGridZ + j*nGridZ + k] - N);
								arr_I1[i*nGridXY*nGridZ + j*nGridZ + k] += N;

								/* END tredje Map-kernel */

							}
						}
					}
				}
			}

      if (GPU_KERNEL_TIMING){
        kernel_elapsed = high_resolution_clock::now() - kernel_start;
        f_kerneltimings << duration_cast<microseconds>(kernel_elapsed).count() << "\t";
      }

      if (GPU_KERNEL_TIMING){
        kernel_start = high_resolution_clock::now();
      }
			if (GPU_NEWINFECTIONS) {

				// Copy to the device
				if (!GPU_INFECTIONS) {
					CopyAllToDevice();
				}

				NewInfectionsKernel<<<gridSize, blockSize>>>(d_arr_Occ, d_arr_nC, d_arr_P, d_arr_P_new,
															d_arr_GrowthModifier, d_arr_B,
															d_arr_M, d_arr_I0_new, d_arr_IsActive,
															reducedBeta, clustering, shielding,
                              K, alpha, beta, eta, zeta, dT,
                              r, d_rng_state, totalElements);
					hipDeviceSynchronize();

				// Copy data back from device
				if (!GPU_PHAGEDECAY) {
					CopyAllToHost();
				}

			} else {
				// Kernel 5: New infections ///////////////////////////////////////////////////////////////////
				for (int i = 0; i < nGridXY; i++) {
					if (exit) break;

					for (int j = 0; j < nGridXY; j++) {
						if (exit) break;

						for (int k = 0; k < nGridZ; k++) {
							if (exit) break;

							// Skip empty sites
							if (skipArray[i*nGridXY*nGridZ + j*nGridZ + k]) continue;


							numtype p = 0; // privatize
							numtype N = 0; // privatize
							// double M = 0; // privatize

							// Compute beta
							numtype Beta = beta;
							if (reducedBeta) {
								Beta *= arr_GrowthModifier[i*nGridXY*nGridZ + j*nGridZ + k];
							}

							// PRIVATIZE BOTH OF THESE
							// numtype s;   // The factor which modifies the adsorption rate
							// numtype n;   // The number of targets the phage has
							// Infectons


							// KERNEL THIS
							// if ((arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k] >= 1) and (arr_P[i*nGridXY*nGridZ + j*nGridZ + k] >= 1)) {
							// 	if (clustering) {   // Check if clustering is enabled
							// 		s = pow(arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k] / arr_nC[i*nGridXY*nGridZ + j*nGridZ + k], 1.0 / 3.0);
							// 		n = arr_nC[i*nGridXY*nGridZ + j*nGridZ + k];
							// 	} else {            // Else use mean field computation
							// 		s = 1.0;
							// 		n = arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k];
							// 	}
							// }

							if ((arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k] >= 1) and (arr_P[i*nGridXY*nGridZ + j*nGridZ + k] >= 1)) {
								// Compute the number of hits
								// if (eta * s * dT >= 1) { // In the diffusion limited case every phage hits a target
									N = arr_P[i*nGridXY*nGridZ + j*nGridZ + k];
								// } else {
									// p = 1 - pow(1 - eta * s * dT, n);        // Probability hitting any target
									// N = ComputeEvents(arr_P[i*nGridXY*nGridZ + j*nGridZ + k], p, 4, i, j, k);     // Number of targets hit
								// }


								// If bacteria were hit, update events
								// DETERMINITIC CHANGE
								// if (N + arr_M[i*nGridXY*nGridZ + j*nGridZ + k] >= 1) {

									arr_P[i*nGridXY*nGridZ + j*nGridZ + k] = max(0.0, arr_P[i*nGridXY*nGridZ + j*nGridZ + k] - N);     // Update count

									numtype S;
									if (shielding) {
										// Absorbing medium model
										numtype d = pow(arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k] / arr_nC[i*nGridXY*nGridZ + j*nGridZ + k], 1.0 / 3.0) -
											pow(arr_B[i*nGridXY*nGridZ + j*nGridZ + k] / arr_nC[i*nGridXY*nGridZ + j*nGridZ + k], 1.0 / 3.0);
										S = cpu_exp(-zeta * d); // Probability of hitting succebtible target

									} else {
										// Well mixed model
										S = arr_B[i*nGridXY*nGridZ + j*nGridZ + k] / arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k];
									}

									p = max(0.0, min(arr_B[i*nGridXY*nGridZ + j*nGridZ + k] / arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k],
																	 S)); // Probability of hitting succebtible target
									N = ComputeEvents(N + arr_M[i*nGridXY*nGridZ + j*nGridZ + k], p, 4, i, j, k);                  // Number of targets hit

									if (N > arr_B[i*nGridXY*nGridZ + j*nGridZ + k])
										N = arr_B[i*nGridXY*nGridZ + j*nGridZ + k];              // If more bacteria than present are set to be infeced, round down

									// Update the counts
									arr_B[i*nGridXY*nGridZ + j*nGridZ + k] = max(0.0, arr_B[i*nGridXY*nGridZ + j*nGridZ + k] - N);
									if (r > 0.0) {
										arr_I0_new[i*nGridXY*nGridZ + j*nGridZ + k] += N;
									} else {
										arr_P_new[i*nGridXY*nGridZ + j*nGridZ + k] += N * (1 - alpha) * Beta;
									}
								// }
							}
						}
					}
				}
        }

				if (GPU_KERNEL_TIMING){
					kernel_elapsed = high_resolution_clock::now() - kernel_start;
					f_kerneltimings << duration_cast<microseconds>(kernel_elapsed).count() << "\t";
				}

			// Phage decay ///////////////////////////////////////////////////////////////////
				if (GPU_KERNEL_TIMING){
					kernel_start = high_resolution_clock::now();
				}

			if (GPU_PHAGEDECAY) {

				// Copy to the device
                if(!GPU_NEWINFECTIONS){
                    CopyAllToDevice();
                }

				PhageDecay<<<gridSize, blockSize>>>(d_arr_P, delta*dT,
                                            d_Warn_delta, d_rng_state,
                                            d_arr_IsActive);
					hipDeviceSynchronize();

				// Copy data back from device
                if(!GPU_MOVEMENT){
                    CopyAllToHost();
				}

			} else {
				for (int i = 0; i < nGridXY; i++) {
					if (exit) break;

					for (int j = 0; j < nGridXY; j++) {
						if (exit) break;

						for (int k = 0; k < nGridZ; k++) {
							if (exit) break;

							// Skip empty sites
							if (skipArray[i*nGridXY*nGridZ + j*nGridZ + k]) continue;


							numtype p = 0; // privatize
							numtype N = 0; // privatize

							// KERNEL BEGIN
							p = delta*dT;

							if ((p > 0.1) and (!Warn_delta)) {
								cout << "\tWarning: Decay Probability Large!" << "\n";
								f_log  << "Warning: Decay Probability Large!" << "\n";
								Warn_delta = true;
							}


							N = ComputeEvents(arr_P[i*nGridXY*nGridZ + j*nGridZ + k], p, 5, i, j, k);

							// Update count
							arr_P[i*nGridXY*nGridZ + j*nGridZ + k]    = max(0.0, arr_P[i*nGridXY*nGridZ + j*nGridZ + k] - N);
							// KERNEL END

						}
					}
				}
			}
      if (GPU_KERNEL_TIMING){
        kernel_elapsed = high_resolution_clock::now() - kernel_start;
        f_kerneltimings << duration_cast<microseconds>(kernel_elapsed).count() << "\t";
      }



			// Movement ///////////////////////////////////////////////////////////////////
      if (GPU_KERNEL_TIMING){
        kernel_start = high_resolution_clock::now();
      }

			if (GPU_MOVEMENT) {
				// Copy to the device
                if(!GPU_PHAGEDECAY){
                    CopyAllToDevice();
                }


				ComputeDiffusionWeights<<<gridSize,blockSize>>>(d_rng_state, d_arr_B, lambdaB, d_arr_n_0, d_arr_n_u, d_arr_n_d, d_arr_n_l, d_arr_n_r, d_arr_n_f, d_arr_n_b, nGridXY, d_arr_IsActive, totalElements);
				hipDeviceSynchronize();
				ApplyMovement<<<gridSize,blockSize>>>(d_arr_B_new, lambdaB, d_arr_n_0, d_arr_n_u, d_arr_n_d, d_arr_n_l, d_arr_n_r, d_arr_n_f, d_arr_n_b, nGridZ, nGridXY, experimentalConditions, d_arr_IsActive, false,totalElements);
				hipDeviceSynchronize();

				if (r > 0) {
					ComputeDiffusionWeights<<<gridSize,blockSize>>>(d_rng_state, d_arr_I0, lambdaB, d_arr_n_0, d_arr_n_u, d_arr_n_d, d_arr_n_l, d_arr_n_r, d_arr_n_f, d_arr_n_b, nGridXY, d_arr_IsActive, totalElements);
					hipDeviceSynchronize();
					ApplyMovement<<<gridSize,blockSize>>>(d_arr_I0_new, lambdaB, d_arr_n_0, d_arr_n_u, d_arr_n_d, d_arr_n_l, d_arr_n_r, d_arr_n_f, d_arr_n_b, nGridZ, nGridXY, experimentalConditions, d_arr_IsActive, false, totalElements);
					hipDeviceSynchronize();

					ComputeDiffusionWeights<<<gridSize,blockSize>>>(d_rng_state, d_arr_I1, lambdaB, d_arr_n_0, d_arr_n_u, d_arr_n_d, d_arr_n_l, d_arr_n_r, d_arr_n_f, d_arr_n_b, nGridXY, d_arr_IsActive, totalElements);
					hipDeviceSynchronize();
					ApplyMovement<<<gridSize,blockSize>>>(d_arr_I1_new, lambdaB, d_arr_n_0, d_arr_n_u, d_arr_n_d, d_arr_n_l, d_arr_n_r, d_arr_n_f, d_arr_n_b, nGridZ, nGridXY, experimentalConditions, d_arr_IsActive, true,totalElements);
					hipDeviceSynchronize();

					ComputeDiffusionWeights<<<gridSize,blockSize>>>(d_rng_state, d_arr_I2, lambdaB, d_arr_n_0, d_arr_n_u, d_arr_n_d, d_arr_n_l, d_arr_n_r, d_arr_n_f, d_arr_n_b, nGridXY, d_arr_IsActive, totalElements);
					hipDeviceSynchronize();
					ApplyMovement<<<gridSize,blockSize>>>(d_arr_I2_new, lambdaB, d_arr_n_0, d_arr_n_u, d_arr_n_d, d_arr_n_l, d_arr_n_r, d_arr_n_f, d_arr_n_b, nGridZ, nGridXY, experimentalConditions, d_arr_IsActive, true,totalElements);
					hipDeviceSynchronize();

					ComputeDiffusionWeights<<<gridSize,blockSize>>>(d_rng_state, d_arr_I3, lambdaB, d_arr_n_0, d_arr_n_u, d_arr_n_d, d_arr_n_l, d_arr_n_r, d_arr_n_f, d_arr_n_b, nGridXY, d_arr_IsActive, totalElements);
					hipDeviceSynchronize();
					ApplyMovement<<<gridSize,blockSize>>>(d_arr_I3_new, lambdaB, d_arr_n_0, d_arr_n_u, d_arr_n_d, d_arr_n_l, d_arr_n_r, d_arr_n_f, d_arr_n_b, nGridZ, nGridXY, experimentalConditions, d_arr_IsActive, true,totalElements);
					hipDeviceSynchronize();

					ComputeDiffusionWeights<<<gridSize,blockSize>>>(d_rng_state, d_arr_I4, lambdaB, d_arr_n_0, d_arr_n_u, d_arr_n_d, d_arr_n_l, d_arr_n_r, d_arr_n_f, d_arr_n_b, nGridXY, d_arr_IsActive, totalElements);
					hipDeviceSynchronize();
					ApplyMovement<<<gridSize,blockSize>>>(d_arr_I4_new, lambdaB, d_arr_n_0, d_arr_n_u, d_arr_n_d, d_arr_n_l, d_arr_n_r, d_arr_n_f, d_arr_n_b, nGridZ, nGridXY, experimentalConditions, d_arr_IsActive, true,totalElements);
					hipDeviceSynchronize();

					ComputeDiffusionWeights<<<gridSize,blockSize>>>(d_rng_state, d_arr_I5, lambdaB, d_arr_n_0, d_arr_n_u, d_arr_n_d, d_arr_n_l, d_arr_n_r, d_arr_n_f, d_arr_n_b, nGridXY, d_arr_IsActive, totalElements);
					hipDeviceSynchronize();
					ApplyMovement<<<gridSize,blockSize>>>(d_arr_I5_new, lambdaB, d_arr_n_0, d_arr_n_u, d_arr_n_d, d_arr_n_l, d_arr_n_r, d_arr_n_f, d_arr_n_b, nGridZ, nGridXY, experimentalConditions, d_arr_IsActive, true,totalElements);
					hipDeviceSynchronize();

					ComputeDiffusionWeights<<<gridSize,blockSize>>>(d_rng_state, d_arr_I6, lambdaB, d_arr_n_0, d_arr_n_u, d_arr_n_d, d_arr_n_l, d_arr_n_r, d_arr_n_f, d_arr_n_b, nGridXY, d_arr_IsActive, totalElements);
					hipDeviceSynchronize();
					ApplyMovement<<<gridSize,blockSize>>>(d_arr_I6_new, lambdaB, d_arr_n_0, d_arr_n_u, d_arr_n_d, d_arr_n_l, d_arr_n_r, d_arr_n_f, d_arr_n_b, nGridZ, nGridXY, experimentalConditions, d_arr_IsActive, true,totalElements);
					hipDeviceSynchronize();

					ComputeDiffusionWeights<<<gridSize,blockSize>>>(d_rng_state, d_arr_I7, lambdaB, d_arr_n_0, d_arr_n_u, d_arr_n_d, d_arr_n_l, d_arr_n_r, d_arr_n_f, d_arr_n_b, nGridXY, d_arr_IsActive, totalElements);
					hipDeviceSynchronize();
					ApplyMovement<<<gridSize,blockSize>>>(d_arr_I7_new, lambdaB, d_arr_n_0, d_arr_n_u, d_arr_n_d, d_arr_n_l, d_arr_n_r, d_arr_n_f, d_arr_n_b, nGridZ, nGridXY, experimentalConditions, d_arr_IsActive, true,totalElements);
					hipDeviceSynchronize();

					ComputeDiffusionWeights<<<gridSize,blockSize>>>(d_rng_state, d_arr_I8, lambdaB, d_arr_n_0, d_arr_n_u, d_arr_n_d, d_arr_n_l, d_arr_n_r, d_arr_n_f, d_arr_n_b, nGridXY, d_arr_IsActive, totalElements);
					hipDeviceSynchronize();
					ApplyMovement<<<gridSize,blockSize>>>(d_arr_I8_new, lambdaB, d_arr_n_0, d_arr_n_u, d_arr_n_d, d_arr_n_l, d_arr_n_r, d_arr_n_f, d_arr_n_b, nGridZ, nGridXY, experimentalConditions, d_arr_IsActive, true,totalElements);
					hipDeviceSynchronize();

					ComputeDiffusionWeights<<<gridSize,blockSize>>>(d_rng_state, d_arr_I9, lambdaB, d_arr_n_0, d_arr_n_u, d_arr_n_d, d_arr_n_l, d_arr_n_r, d_arr_n_f, d_arr_n_b, nGridXY, d_arr_IsActive, totalElements);
					hipDeviceSynchronize();
					ApplyMovement<<<gridSize,blockSize>>>(d_arr_I9_new, lambdaB, d_arr_n_0, d_arr_n_u, d_arr_n_d, d_arr_n_l, d_arr_n_r, d_arr_n_f, d_arr_n_b, nGridZ, nGridXY, experimentalConditions, d_arr_IsActive, true,totalElements);
					hipDeviceSynchronize();
				}

				ComputeDiffusionWeights<<<gridSize,blockSize>>>(d_rng_state, d_arr_P, lambdaP, d_arr_n_0, d_arr_n_u, d_arr_n_d, d_arr_n_l, d_arr_n_r, d_arr_n_f, d_arr_n_b, nGridXY, d_arr_IsActive, totalElements);
				hipDeviceSynchronize();
				ApplyMovement<<<gridSize,blockSize>>>(d_arr_P_new, lambdaP, d_arr_n_0, d_arr_n_u, d_arr_n_d, d_arr_n_l, d_arr_n_r, d_arr_n_f, d_arr_n_b, nGridZ, nGridXY, experimentalConditions, d_arr_IsActive, false,totalElements);
				hipDeviceSynchronize();

            	// Copy data back from device
				if(!GPU_SWAPZERO) {
					CopyAllToHost();
				}

			} else {
				for (int i = 0; i < nGridXY; i++) {
					if (exit) break;

					for (int j = 0; j < nGridXY; j++) {
						if (exit) break;

						for (int k = 0; k < nGridZ; k++) {
							if (exit) break;

							// Skip empty sites
							if (skipArray[i*nGridXY*nGridZ + j*nGridZ + k]) continue;

							if (nGridXY > 1) {
								// KERNEL BEGIN
								// Update positions
								int ip, jp, kp, im, jm, km;

								if (i + 1 >= nGridXY) ip = i + 1 - nGridXY;
								else ip = i + 1;

								if (i == 0) im = nGridXY - 1;
								else im = i - 1;

								if (j + 1 >= nGridXY) jp = j + 1 - nGridXY;
								else jp = j + 1;

								if (j == 0) jm = nGridXY - 1;
								else jm = j - 1;

								if (not experimentalConditions) {   // Periodic boundaries in Z direction

									if (k + 1 >= nGridZ) kp = k + 1 - nGridZ;
									else kp = k + 1;

									if (k == 0) km = nGridZ - 1;
									else km = k - 1;

								} else {    // Reflective boundaries in Z direction

									if (k + 1 >= nGridZ) kp = k - 1;
									else kp = k + 1;

									if (k == 0) km = k + 1;
									else km = k - 1;

								}

								// Update counts
								numtype n_0; // No movement
								numtype n_u; // Up
								numtype n_d; // Down
								numtype n_l; // Left
								numtype n_r; // Right
								numtype n_f; // Front
								numtype n_b; // Back

								// CELLS
								ComputeDiffusion(arr_B[i*nGridXY*nGridZ + j*nGridZ + k], lambdaB, &n_0, &n_u, &n_d, &n_l, &n_r, &n_f, &n_b,1, i, j, k);
                                    arr_B_new[i*nGridXY*nGridZ + j*nGridZ + k] += n_0;
                                    arr_B_new[ip*nGridXY*nGridZ + j*nGridZ + k] += n_u;
                                    arr_B_new[im*nGridXY*nGridZ + j*nGridZ + k] += n_d;
                                    arr_B_new[i*nGridXY*nGridZ + jp*nGridZ + k] += n_r;
                                    arr_B_new[i*nGridXY*nGridZ + jm*nGridZ + k] += n_l;
                                    arr_B_new[i*nGridXY*nGridZ + j*nGridZ + kp] += n_f;
                                    arr_B_new[i*nGridXY*nGridZ + j*nGridZ + km] += n_b;

								if (r > 0.0) {
									ComputeDiffusion(arr_I0[i*nGridXY*nGridZ + j*nGridZ + k],  lambdaB, &n_0, &n_u, &n_d, &n_l, &n_r, &n_f, &n_b, 2, i, j, k);
                                        arr_I0_new[i*nGridXY*nGridZ + j*nGridZ + k] += n_0;
                                        arr_I0_new[ip*nGridXY*nGridZ + j*nGridZ + k] += n_u;
                                        arr_I0_new[im*nGridXY*nGridZ + j*nGridZ + k] += n_d;
                                        arr_I0_new[i*nGridXY*nGridZ + jp*nGridZ + k] += n_r;
                                        arr_I0_new[i*nGridXY*nGridZ + jm*nGridZ + k] += n_l;
                                        arr_I0_new[i*nGridXY*nGridZ + j*nGridZ + kp] += n_f;
                                        arr_I0_new[i*nGridXY*nGridZ + j*nGridZ + km] += n_b;

									ComputeDiffusion(arr_I1[i*nGridXY*nGridZ + j*nGridZ + k],  lambdaB, &n_0, &n_u, &n_d, &n_l, &n_r, &n_f, &n_b, 2, i, j, k);
									arr_I1_new[i*nGridXY*nGridZ + j*nGridZ + k] += n_0; arr_I1_new[ip*nGridXY*nGridZ + j*nGridZ + k] += n_u; arr_I1_new[im*nGridXY*nGridZ + j*nGridZ + k] += n_d; arr_I1_new[i*nGridXY*nGridZ + jp*nGridZ + k] += n_r; arr_I1_new[i*nGridXY*nGridZ + jm*nGridZ + k] += n_l; arr_I1_new[i*nGridXY*nGridZ + j*nGridZ + kp] += n_f; arr_I1_new[i*nGridXY*nGridZ + j*nGridZ + km] += n_b;

									ComputeDiffusion(arr_I2[i*nGridXY*nGridZ + j*nGridZ + k],  lambdaB, &n_0, &n_u, &n_d, &n_l, &n_r, &n_f, &n_b, 2, i, j, k);
									arr_I2_new[i*nGridXY*nGridZ + j*nGridZ + k] += n_0; arr_I2_new[ip*nGridXY*nGridZ + j*nGridZ + k] += n_u; arr_I2_new[im*nGridXY*nGridZ + j*nGridZ + k] += n_d; arr_I2_new[i*nGridXY*nGridZ + jp*nGridZ + k] += n_r; arr_I2_new[i*nGridXY*nGridZ + jm*nGridZ + k] += n_l; arr_I2_new[i*nGridXY*nGridZ + j*nGridZ + kp] += n_f; arr_I2_new[i*nGridXY*nGridZ + j*nGridZ + km] += n_b;

									ComputeDiffusion(arr_I3[i*nGridXY*nGridZ + j*nGridZ + k],  lambdaB, &n_0, &n_u, &n_d, &n_l, &n_r, &n_f, &n_b, 2, i, j, k);
									arr_I3_new[i*nGridXY*nGridZ + j*nGridZ + k] += n_0; arr_I3_new[ip*nGridXY*nGridZ + j*nGridZ + k] += n_u; arr_I3_new[im*nGridXY*nGridZ + j*nGridZ + k] += n_d; arr_I3_new[i*nGridXY*nGridZ + jp*nGridZ + k] += n_r; arr_I3_new[i*nGridXY*nGridZ + jm*nGridZ + k] += n_l; arr_I3_new[i*nGridXY*nGridZ + j*nGridZ + kp] += n_f; arr_I3_new[i*nGridXY*nGridZ + j*nGridZ + km] += n_b;

									ComputeDiffusion(arr_I4[i*nGridXY*nGridZ + j*nGridZ + k],  lambdaB, &n_0, &n_u, &n_d, &n_l, &n_r, &n_f, &n_b, 2, i, j, k);
									arr_I4_new[i*nGridXY*nGridZ + j*nGridZ + k] += n_0; arr_I4_new[ip*nGridXY*nGridZ + j*nGridZ + k] += n_u; arr_I4_new[im*nGridXY*nGridZ + j*nGridZ + k] += n_d; arr_I4_new[i*nGridXY*nGridZ + jp*nGridZ + k] += n_r; arr_I4_new[i*nGridXY*nGridZ + jm*nGridZ + k] += n_l; arr_I4_new[i*nGridXY*nGridZ + j*nGridZ + kp] += n_f; arr_I4_new[i*nGridXY*nGridZ + j*nGridZ + km] += n_b;

									ComputeDiffusion(arr_I5[i*nGridXY*nGridZ + j*nGridZ + k],  lambdaB, &n_0, &n_u, &n_d, &n_l, &n_r, &n_f, &n_b, 2, i, j, k);
									arr_I5_new[i*nGridXY*nGridZ + j*nGridZ + k] += n_0; arr_I5_new[ip*nGridXY*nGridZ + j*nGridZ + k] += n_u; arr_I5_new[im*nGridXY*nGridZ + j*nGridZ + k] += n_d; arr_I5_new[i*nGridXY*nGridZ + jp*nGridZ + k] += n_r; arr_I5_new[i*nGridXY*nGridZ + jm*nGridZ + k] += n_l; arr_I5_new[i*nGridXY*nGridZ + j*nGridZ + kp] += n_f; arr_I5_new[i*nGridXY*nGridZ + j*nGridZ + km] += n_b;

									ComputeDiffusion(arr_I6[i*nGridXY*nGridZ + j*nGridZ + k],  lambdaB, &n_0, &n_u, &n_d, &n_l, &n_r, &n_f, &n_b, 2, i, j, k);
									arr_I6_new[i*nGridXY*nGridZ + j*nGridZ + k] += n_0; arr_I6_new[ip*nGridXY*nGridZ + j*nGridZ + k] += n_u; arr_I6_new[im*nGridXY*nGridZ + j*nGridZ + k] += n_d; arr_I6_new[i*nGridXY*nGridZ + jp*nGridZ + k] += n_r; arr_I6_new[i*nGridXY*nGridZ + jm*nGridZ + k] += n_l; arr_I6_new[i*nGridXY*nGridZ + j*nGridZ + kp] += n_f; arr_I6_new[i*nGridXY*nGridZ + j*nGridZ + km] += n_b;

									ComputeDiffusion(arr_I7[i*nGridXY*nGridZ + j*nGridZ + k],  lambdaB, &n_0, &n_u, &n_d, &n_l, &n_r, &n_f, &n_b, 2, i, j, k);
									arr_I7_new[i*nGridXY*nGridZ + j*nGridZ + k] += n_0; arr_I7_new[ip*nGridXY*nGridZ + j*nGridZ + k] += n_u; arr_I7_new[im*nGridXY*nGridZ + j*nGridZ + k] += n_d; arr_I7_new[i*nGridXY*nGridZ + jp*nGridZ + k] += n_r; arr_I7_new[i*nGridXY*nGridZ + jm*nGridZ + k] += n_l; arr_I7_new[i*nGridXY*nGridZ + j*nGridZ + kp] += n_f; arr_I7_new[i*nGridXY*nGridZ + j*nGridZ + km] += n_b;

									ComputeDiffusion(arr_I8[i*nGridXY*nGridZ + j*nGridZ + k],  lambdaB, &n_0, &n_u, &n_d, &n_l, &n_r, &n_f, &n_b, 2, i, j, k);
									arr_I8_new[i*nGridXY*nGridZ + j*nGridZ + k] += n_0; arr_I8_new[ip*nGridXY*nGridZ + j*nGridZ + k] += n_u; arr_I8_new[im*nGridXY*nGridZ + j*nGridZ + k] += n_d; arr_I8_new[i*nGridXY*nGridZ + jp*nGridZ + k] += n_r; arr_I8_new[i*nGridXY*nGridZ + jm*nGridZ + k] += n_l; arr_I8_new[i*nGridXY*nGridZ + j*nGridZ + kp] += n_f; arr_I8_new[i*nGridXY*nGridZ + j*nGridZ + km] += n_b;

									ComputeDiffusion(arr_I9[i*nGridXY*nGridZ + j*nGridZ + k], lambdaB, &n_0, &n_u, &n_d, &n_l, &n_r, &n_f, &n_b, 2, i, j, k);
									arr_I9_new[i*nGridXY*nGridZ + j*nGridZ + k] += n_0; arr_I9_new[ip*nGridXY*nGridZ + j*nGridZ + k] += n_u; arr_I9_new[im*nGridXY*nGridZ + j*nGridZ + k] += n_d; arr_I9_new[i*nGridXY*nGridZ + jp*nGridZ + k] += n_r; arr_I9_new[i*nGridXY*nGridZ + jm*nGridZ + k] += n_l; arr_I9_new[i*nGridXY*nGridZ + j*nGridZ + kp] += n_f; arr_I9_new[i*nGridXY*nGridZ + j*nGridZ + km] += n_b;
								}

								// PHAGES
								ComputeDiffusion(arr_P[i*nGridXY*nGridZ + j*nGridZ + k], lambdaP, &n_0, &n_u, &n_d, &n_l, &n_r, &n_f, &n_b, 3, i, j, k);
								arr_P_new[i*nGridXY*nGridZ + j*nGridZ + k] += n_0; arr_P_new[ip*nGridXY*nGridZ + j*nGridZ + k] += n_u; arr_P_new[im*nGridXY*nGridZ + j*nGridZ + k] += n_d; arr_P_new[i*nGridXY*nGridZ + jp*nGridZ + k] += n_r; arr_P_new[i*nGridXY*nGridZ + jm*nGridZ + k] += n_l; arr_P_new[i*nGridXY*nGridZ + j*nGridZ + kp] += n_f; arr_P_new[i*nGridXY*nGridZ + j*nGridZ + km] += n_b;

								// KERNEL END



							} else {
								arr_B_new[i*nGridXY*nGridZ + j*nGridZ + k] += arr_B[i*nGridXY*nGridZ + j*nGridZ + k];

								if (r > 0.0) {
									arr_I0_new[i*nGridXY*nGridZ + j*nGridZ + k] += arr_I0[i*nGridXY*nGridZ + j*nGridZ + k];
									arr_I1_new[i*nGridXY*nGridZ + j*nGridZ + k] += arr_I1[i*nGridXY*nGridZ + j*nGridZ + k];
									arr_I2_new[i*nGridXY*nGridZ + j*nGridZ + k] += arr_I2[i*nGridXY*nGridZ + j*nGridZ + k];
									arr_I3_new[i*nGridXY*nGridZ + j*nGridZ + k] += arr_I3[i*nGridXY*nGridZ + j*nGridZ + k];
									arr_I4_new[i*nGridXY*nGridZ + j*nGridZ + k] += arr_I4[i*nGridXY*nGridZ + j*nGridZ + k];
									arr_I5_new[i*nGridXY*nGridZ + j*nGridZ + k] += arr_I5[i*nGridXY*nGridZ + j*nGridZ + k];
									arr_I6_new[i*nGridXY*nGridZ + j*nGridZ + k] += arr_I6[i*nGridXY*nGridZ + j*nGridZ + k];
									arr_I7_new[i*nGridXY*nGridZ + j*nGridZ + k] += arr_I7[i*nGridXY*nGridZ + j*nGridZ + k];
									arr_I8_new[i*nGridXY*nGridZ + j*nGridZ + k] += arr_I8[i*nGridXY*nGridZ + j*nGridZ + k];
									arr_I9_new[i*nGridXY*nGridZ + j*nGridZ + k] += arr_I9[i*nGridXY*nGridZ + j*nGridZ + k];
								}

								// PHAGES
								arr_P_new[i*nGridXY*nGridZ + j*nGridZ + k] += arr_P[i*nGridXY*nGridZ + j*nGridZ + k];
								// KERNEL END
							}
						}
					}
				}
			}

      if (GPU_KERNEL_TIMING){
        kernel_elapsed = high_resolution_clock::now() - kernel_start;
        f_kerneltimings << duration_cast<microseconds>(kernel_elapsed).count() << "\t";
      }
      /////////////////////////////////////
      // Simple end of loop kernels

      if (GPU_KERNEL_TIMING){
        kernel_start = high_resolution_clock::now();
      }
			if(GPU_SWAPZERO){

				// Copy to the device
				if(!GPU_MOVEMENT) {
					CopyAllToDevice();
				}

                std::swap(d_arr_B, d_arr_B_new);
                std::swap(d_arr_I0, d_arr_I0_new);
                std::swap(d_arr_I1, d_arr_I1_new);
                std::swap(d_arr_I2, d_arr_I2_new);
                std::swap(d_arr_I3, d_arr_I3_new);
                std::swap(d_arr_I4, d_arr_I4_new);
                std::swap(d_arr_I5, d_arr_I5_new);
                std::swap(d_arr_I6, d_arr_I6_new);
                std::swap(d_arr_I7, d_arr_I7_new);
                std::swap(d_arr_I8, d_arr_I8_new);
                std::swap(d_arr_I9, d_arr_I9_new);
                std::swap(d_arr_P, d_arr_P_new);
                hipDeviceSynchronize();
				/*
                ZeroArray<<<gridSize,blockSize>>>(d_arr_B_new, totalElements);
                ZeroArray<<<gridSize,blockSize>>>(d_arr_I0_new, totalElements);
                ZeroArray<<<gridSize,blockSize>>>(d_arr_I1_new, totalElements);
				ZeroArray<<<gridSize,blockSize>>>(d_arr_I2_new, totalElements);
                ZeroArray<<<gridSize,blockSize>>>(d_arr_I3_new, totalElements);
                ZeroArray<<<gridSize,blockSize>>>(d_arr_I4_new, totalElements);
                ZeroArray<<<gridSize,blockSize>>>(d_arr_I5_new, totalElements);
                ZeroArray<<<gridSize,blockSize>>>(d_arr_I6_new, totalElements);
                ZeroArray<<<gridSize,blockSize>>>(d_arr_I7_new, totalElements);
                ZeroArray<<<gridSize,blockSize>>>(d_arr_I8_new, totalElements);
                ZeroArray<<<gridSize,blockSize>>>(d_arr_I9_new, totalElements);
                ZeroArray<<<gridSize,blockSize>>>(d_arr_P_new, totalElements);
                hipDeviceSynchronize();
				*/

				// Copy data back from device
                if(!GPU_UPDATEOCCUPANCY) {
					CopyAllToHost();
				}

            } else {
				// Swap pointers
                std::swap(arr_B, arr_B_new);
                std::swap(arr_I0, arr_I0_new);
                std::swap(arr_I1, arr_I1_new);
                std::swap(arr_I2, arr_I2_new);
                std::swap(arr_I3, arr_I3_new);
                std::swap(arr_I4, arr_I4_new);
                std::swap(arr_I5, arr_I5_new);
                std::swap(arr_I6, arr_I6_new);
                std::swap(arr_I7, arr_I7_new);
                std::swap(arr_I8, arr_I8_new);
                std::swap(arr_I9, arr_I9_new);
                std::swap(arr_P, arr_P_new);

                // Zero the _new arrays
                for (int i = 0; i < nGridXY; i++) {
                    for (int j = 0; j < nGridXY; j++ ) {
                        for (int k = 0; k < nGridZ; k++ ) {
                            arr_B_new[i*nGridXY*nGridZ + j*nGridZ + k]  = 0.0;
                            arr_I0_new[i*nGridXY*nGridZ + j*nGridZ + k] = 0.0;
                            arr_I1_new[i*nGridXY*nGridZ + j*nGridZ + k] = 0.0;
                            arr_I2_new[i*nGridXY*nGridZ + j*nGridZ + k] = 0.0;
                            arr_I3_new[i*nGridXY*nGridZ + j*nGridZ + k] = 0.0;
                            arr_I4_new[i*nGridXY*nGridZ + j*nGridZ + k] = 0.0;
                            arr_I5_new[i*nGridXY*nGridZ + j*nGridZ + k] = 0.0;
                            arr_I6_new[i*nGridXY*nGridZ + j*nGridZ + k] = 0.0;
                            arr_I7_new[i*nGridXY*nGridZ + j*nGridZ + k] = 0.0;
                            arr_I8_new[i*nGridXY*nGridZ + j*nGridZ + k] = 0.0;
                            arr_I9_new[i*nGridXY*nGridZ + j*nGridZ + k] = 0.0;
                            arr_P_new[i*nGridXY*nGridZ + j*nGridZ + k]  = 0.0;
                        }
                    }
                }
            }

      if (GPU_KERNEL_TIMING){
        kernel_elapsed = high_resolution_clock::now() - kernel_start;
        f_kerneltimings << duration_cast<microseconds>(kernel_elapsed).count() << "\t";
      }


      if (GPU_KERNEL_TIMING){
        hipDeviceSynchronize();
        kernel_start = high_resolution_clock::now();
      }
      if(GPU_UPDATEOCCUPANCY){

				// Copy data back from device
        if(!GPU_SWAPZERO) {
					CopyAllToDevice();
				}


				UpdateOccupancy<<<gridSize, blockSize>>>(d_arr_Occ, d_arr_B, d_arr_I0, d_arr_I1, d_arr_I2, d_arr_I3, d_arr_I4, d_arr_I5, d_arr_I6, d_arr_I7, d_arr_I8, d_arr_I9, totalElements);
                  hipDeviceSynchronize();

				// Copy data back from device
                if(!GPU_NUTRIENTDIFFUSION) CopyAllToHost();

            } else {
				// Update occupancy
                for (int i = 0; i < nGridXY; i++) {
                    for (int j = 0; j < nGridXY; j++ ) {
                        for (int k = 0; k < nGridZ; k++ ) {
                            arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k] = arr_B[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I0[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I1[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I2[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I3[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I4[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I5[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I6[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I7[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I8[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I9[i*nGridXY*nGridZ + j*nGridZ + k];
                        }
                    }
                }
            }
      if (GPU_KERNEL_TIMING){
        kernel_elapsed = high_resolution_clock::now() - kernel_start;
        f_kerneltimings << duration_cast<microseconds>(kernel_elapsed).count() << "\t";
      }



			// NUTRIENT DIFFUSION
			numtype alphaXY = D_n * dT / pow(L / (numtype)nGridXY, 2);
			numtype alphaZ  = D_n * dT / pow(H / (numtype)nGridZ, 2);

      if (GPU_KERNEL_TIMING){
        kernel_start = high_resolution_clock::now();
      }
            if(GPU_NUTRIENTDIFFUSION){

				// Copy data back from device
                if(!GPU_UPDATEOCCUPANCY) {
					 CopyAllToDevice();
				}


      			int sharedMemSize = 5*(blockSize+2);
				NutrientDiffusion<<<gridSize,blockSize,sharedMemSize>>>(d_arr_nutrient, d_arr_nutrient_new, alphaXY, alphaZ, nGridXY, nGridZ, experimentalConditions, totalElements);
                  hipDeviceSynchronize();


				// Copy data back from device
                if(!GPU_SWAPZERO2) {
					CopyAllToHost();
				}

            } else {
                for (int i = 0; i < nGridXY; i++) {
                    for (int j = 0; j < nGridXY; j++ ) {
                        for (int k = 0; k < nGridZ; k++ ) {

                            // Update positions
                            int ip, jp, kp, im, jm, km;

                            if (i + 1 >= nGridXY) ip = i + 1 - nGridXY;
                            else ip = i + 1;

                            if (i == 0) im = nGridXY - 1;
                            else im = i - 1;

                            if (j + 1 >= nGridXY) jp = j + 1 - nGridXY;
                            else jp = j + 1;

                            if (j == 0) jm = nGridXY - 1;
                            else jm = j - 1;

                            if (not experimentalConditions) {   // Periodic boundaries in Z direction

                                if (k + 1 >= nGridZ) kp = k + 1 - nGridZ;
                                else kp = k + 1;

                                if (k == 0) km = nGridZ - 1;
                                else km = k - 1;

                            } else {    // Reflective boundaries in Z direction

                                if (k + 1 >= nGridZ) kp = k - 1;
                                else kp = k + 1;

                                if (k == 0) km = k + 1;
                                else km = k - 1;

                            }

                            numtype tmp = arr_nutrient[i*nGridXY*nGridZ + j*nGridZ + k];
                            arr_nutrient_new[i*nGridXY*nGridZ + j*nGridZ + k]  += tmp - (4 * alphaXY + 2 * alphaZ) * tmp;
                            arr_nutrient_new[ip*nGridXY*nGridZ + j*nGridZ + k] += alphaXY * tmp;
                            arr_nutrient_new[im*nGridXY*nGridZ + j*nGridZ + k] += alphaXY * tmp;
                            arr_nutrient_new[i*nGridXY*nGridZ + jp*nGridZ + k] += alphaXY * tmp;
                            arr_nutrient_new[i*nGridXY*nGridZ + jm*nGridZ + k] += alphaXY * tmp;
                            arr_nutrient_new[i*nGridXY*nGridZ + j*nGridZ + kp] += alphaZ  * tmp;
                            arr_nutrient_new[i*nGridXY*nGridZ + j*nGridZ + km] += alphaZ  * tmp;
                        }
                    }
                }
			}
            if (GPU_KERNEL_TIMING){
              kernel_elapsed = high_resolution_clock::now() - kernel_start;
              f_kerneltimings << duration_cast<microseconds>(kernel_elapsed).count() << "\t";
            }

                if (GPU_KERNEL_TIMING){
                  hipDeviceSynchronize();
                  kernel_start = high_resolution_clock::now();
				}
            if(GPU_SWAPZERO2){

				// Copy data back from device
                if(!GPU_NUTRIENTDIFFUSION) {
					CopyAllToDevice();
				}


                std::swap(d_arr_nutrient, d_arr_nutrient_new);
 				//ZeroArray<<<gridSize,blockSize>>>(d_arr_nutrient_new, totalElements);


				// Copy data back from device
				if(!GPU_NC) {
					CopyAllToHost();
				}

            } else {
                std::swap(arr_nutrient, arr_nutrient_new);

                // Zero the _new arrays
                for (int i = 0; i < nGridXY; i++) {
                    for (int j = 0; j < nGridXY; j++ ) {
                        for (int k = 0; k < nGridZ; k++ ) {
                            arr_nutrient_new[i*nGridXY*nGridZ + j*nGridZ + k]  = 0.0;
                        }
                    }
                }
            }
                if (GPU_KERNEL_TIMING){
                  hipDeviceSynchronize();
                  kernel_elapsed = high_resolution_clock::now() - kernel_start;
                  f_kerneltimings << duration_cast<microseconds>(kernel_elapsed).count();
                }



			f_kerneltimings << "\n";

			if (!OPTIMIZED_MAXOCCUPANCY){
				if ((maxOccupancy > L * L * H / (nGridXY * nGridXY * nGridZ)) and (!Warn_density)) {
					cout << "\tWarning: Maximum Density Large!" << "\n";
					f_log  << "Warning: Maximum Density Large!" << "\n";
					Warn_density = true;
				}
			}

		}

		hipDeviceSynchronize();

        /////////////////////////////
        //Sample loop ends...
        ////////////////////////////

		#if !GPU_REDUCE_ARRAYS
			CopyAllToHost();
		#endif


		// Fast exit conditions
		// 1) There are no more sucebtible cells
		// -> Convert all infected cells to phages and stop simulation
		numtype accuB = 0.0;
		#if GPU_REDUCE_ARRAYS

			PartialSum<<<gridSize, blockSize, blockSize*sizeof(numtype)>>>(d_arr_B, d_arr_partialSum, totalElements);
			err = hipGetLastError();
			if (err != hipSuccess && errC > 0) {fprintf(stderr, "Failure in PartialSum (B)! error = %s\n", hipGetErrorString(err)); errC--;}

			SequentialReduceSum<<<1,1>>>(d_arr_partialSum, gridSize);
			err = hipGetLastError();
			if (err != hipSuccess && errC > 0) {fprintf(stderr, "Failure in SequentialReduceSum (B)! error = %s\n", hipGetErrorString(err)); errC--;}

			err = hipMemcpy(&accuB, d_arr_partialSum, sizeof(numtype), hipMemcpyDeviceToHost);
			if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to copy arr_partialSum to the host! error = %s\n", hipGetErrorString(err));
				errC--; }

		#else

			for (int i = 0; i < nGridXY; i++) {
				for (int j = 0; j < nGridXY; j++ ) {
					for (int k = 0; k < nGridZ; k++ ) {
						accuB += arr_B[i*nGridXY*nGridZ + j*nGridZ + k];
					}
				}
			}

		#endif

		if ((fastExit) and (accuB < 1)) {
			// Update the P array
			for (int i = 0; i < nGridXY; i++) {
				for (int j = 0; j < nGridXY; j++ ) {
					for (int k = 0; k < nGridZ; k++ ) {
						arr_P[i*nGridXY*nGridZ + j*nGridZ + k] += (1-alpha)*beta * (arr_I0[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I1[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I2[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I3[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I4[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I5[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I6[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I7[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I8[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I9[i*nGridXY*nGridZ + j*nGridZ + k]);
					}
				}
			}

			// Zero the I arrays
			ZeroArray<<<gridSize,blockSize>>>(d_arr_I0, totalElements);
			ZeroArray<<<gridSize,blockSize>>>(d_arr_I1, totalElements);
			ZeroArray<<<gridSize,blockSize>>>(d_arr_I2, totalElements);
			ZeroArray<<<gridSize,blockSize>>>(d_arr_I3, totalElements);
			ZeroArray<<<gridSize,blockSize>>>(d_arr_I4, totalElements);
			ZeroArray<<<gridSize,blockSize>>>(d_arr_I5, totalElements);
			ZeroArray<<<gridSize,blockSize>>>(d_arr_I6, totalElements);
			ZeroArray<<<gridSize,blockSize>>>(d_arr_I7, totalElements);
			ZeroArray<<<gridSize,blockSize>>>(d_arr_I8, totalElements);
			ZeroArray<<<gridSize,blockSize>>>(d_arr_I9, totalElements);

			CopyAllToHost();
			exit = true;
		}

		// 2) There are no more alive cells
		// -> Stop simulation

		numtype accuOcc = 0.0;
		#if GPU_REDUCE_ARRAYS

			PartialSum<<<gridSize, blockSize, blockSize*sizeof(numtype)>>>(d_arr_Occ, d_arr_partialSum, totalElements);
			err = hipGetLastError();
			if (err != hipSuccess && errC > 0) {fprintf(stderr, "Failure in PartialSum (B)! error = %s\n", hipGetErrorString(err)); errC--;}

			SequentialReduceSum<<<1,1>>>(d_arr_partialSum, gridSize);
			err = hipGetLastError();
			if (err != hipSuccess && errC > 0) {fprintf(stderr, "Failure in SequentialReduceSum (B)! error = %s\n", hipGetErrorString(err)); errC--;}

			err = hipMemcpy(&accuOcc, d_arr_partialSum, sizeof(numtype), hipMemcpyDeviceToHost);
			if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to copy arr_partialSum to the host! error = %s\n", hipGetErrorString(err));
				errC--; }

		#else

			for (int i = 0; i < nGridXY; i++) {
				for (int j = 0; j < nGridXY; j++ ) {
					for (int k = 0; k < nGridZ; k++ ) {
						accuOcc += arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k];
					}
				}
			}

		#endif

		if ((fastExit) and (accuOcc < 1)) {
			exit = true;
		}

		// 3) The food is on average less than one per gridpoint
		// and the maximal nutrient at any point in space is less than 1

		numtype accuNutrient = 0.0;
		numtype maxNutrient  = 0.0;
		#if GPU_REDUCE_ARRAYS

			PartialSum<<<gridSize, blockSize, blockSize*sizeof(numtype)>>>(d_arr_nutrient, d_arr_partialSum, totalElements);
			err = hipGetLastError();
			if (err != hipSuccess && errC > 0) {fprintf(stderr, "Failure in PartialSum (B)! error = %s\n", hipGetErrorString(err)); errC--;}

			SequentialReduceSum<<<1,1>>>(d_arr_partialSum, gridSize);
			err = hipGetLastError();
			if (err != hipSuccess && errC > 0) {fprintf(stderr, "Failure in SequentialReduceSum (B)! error = %s\n", hipGetErrorString(err)); errC--;}

			err = hipMemcpy(&accuNutrient, d_arr_partialSum, sizeof(numtype), hipMemcpyDeviceToHost);
			if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to copy arr_partialSum to the host! error = %s\n", hipGetErrorString(err));
				errC--; }

			PartialMax<<<gridSize, blockSize, blockSize*sizeof(numtype)>>>(d_arr_nutrient, d_arr_partialSum, totalElements);
			err = hipGetLastError();
			if (err != hipSuccess && errC > 0) {fprintf(stderr, "Failure in PartialSum (B)! error = %s\n", hipGetErrorString(err)); errC--;}

			SequentialReduceMax<<<1,1>>>(d_arr_partialSum, gridSize);
			err = hipGetLastError();
			if (err != hipSuccess && errC > 0) {fprintf(stderr, "Failure in SequentialReduceSum (B)! error = %s\n", hipGetErrorString(err)); errC--;}

			err = hipMemcpy(&maxNutrient, d_arr_partialSum, sizeof(numtype), hipMemcpyDeviceToHost);
			if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to copy arr_partialSum to the host! error = %s\n", hipGetErrorString(err));
				errC--; }

		#else

			for (int i = 0; i < nGridXY; i++) {
				for (int j = 0; j < nGridXY; j++ ) {
					for (int k = 0; k < nGridZ; k++ ) {
						numtype tmpN = arr_nutrient[i*nGridXY*nGridZ + j*nGridZ + k];
						accuNutrient += tmpN;

						if (tmpN > maxNutrient) {
							maxNutrient = tmpN;
						}
					}
				}
			}

		#endif

		if (fastExit) {
			if  ((accuNutrient < nGridZ*pow(nGridXY,2)) && (maxNutrient < 0.5)) {
				exit = true;
			}
		}


		#if GPU_REDUCE_ARRAYS_EXPORT

		if (!exportAll) {

			numtype accuI0, accuI1, accuI2, accuI3, accuI4, accuI5, accuI6;
			numtype accuI7, accuI8, accuI9, accuP, accuClusters, nz;

			// Reduce arr_I0
			PartialSum<<<gridSize, blockSize, blockSize*sizeof(numtype)>>>(d_arr_I0, d_arr_partialSum, totalElements);
			err = hipGetLastError();
			if (err != hipSuccess && errC > 0) {fprintf(stderr, "Failure in PartialSum (I0)! error = %s\n", hipGetErrorString(err)); errC--;}

			SequentialReduceSum<<<1,1>>>(d_arr_partialSum, gridSize);
			err = hipGetLastError();
			if (err != hipSuccess && errC > 0) {fprintf(stderr, "Failure in SequentialReduceSum (I0)! error = %s\n", hipGetErrorString(err)); errC--;}

			err = hipMemcpy(&accuI0, d_arr_partialSum, sizeof(numtype), hipMemcpyDeviceToHost);
			if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to copy arr_partialSum to the host! error = %s\n", hipGetErrorString(err));
				errC--; }

			// Reduce arr_I1
			PartialSum<<<gridSize, blockSize, blockSize*sizeof(numtype)>>>(d_arr_I1, d_arr_partialSum, totalElements);
			err = hipGetLastError();
			if (err != hipSuccess && errC > 0) {fprintf(stderr, "Failure in PartialSum (I1)! error = %s\n", hipGetErrorString(err)); errC--;}

			SequentialReduceSum<<<1,1>>>(d_arr_partialSum, gridSize);
			err = hipGetLastError();
			if (err != hipSuccess && errC > 0) {fprintf(stderr, "Failure in SequentialReduceSum (I1)! error = %s\n", hipGetErrorString(err)); errC--;}

			err = hipMemcpy(&accuI1, d_arr_partialSum, sizeof(numtype), hipMemcpyDeviceToHost);
			if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to copy arr_partialSum to the host! error = %s\n", hipGetErrorString(err));
				errC--; }

			// Reduce arr_I2
			PartialSum<<<gridSize, blockSize, blockSize*sizeof(numtype)>>>(d_arr_I2, d_arr_partialSum, totalElements);
			err = hipGetLastError();
			if (err != hipSuccess && errC > 0) {fprintf(stderr, "Failure in PartialSum (I2)! error = %s\n", hipGetErrorString(err)); errC--;}

			SequentialReduceSum<<<1,1>>>(d_arr_partialSum, gridSize);
			err = hipGetLastError();
			if (err != hipSuccess && errC > 0) {fprintf(stderr, "Failure in SequentialReduceSum (I2)! error = %s\n", hipGetErrorString(err)); errC--;}

			err = hipMemcpy(&accuI2, d_arr_partialSum, sizeof(numtype), hipMemcpyDeviceToHost);
			if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to copy arr_partialSum to the host! error = %s\n", hipGetErrorString(err));
				errC--; }

			// Reduce arr_I3
			PartialSum<<<gridSize, blockSize, blockSize*sizeof(numtype)>>>(d_arr_I3, d_arr_partialSum, totalElements);
			err = hipGetLastError();
			if (err != hipSuccess && errC > 0) {fprintf(stderr, "Failure in PartialSum (I3)! error = %s\n", hipGetErrorString(err)); errC--;}

			SequentialReduceSum<<<1,1>>>(d_arr_partialSum, gridSize);
			err = hipGetLastError();
			if (err != hipSuccess && errC > 0) {fprintf(stderr, "Failure in SequentialReduceSum (I3)! error = %s\n", hipGetErrorString(err)); errC--;}

			err = hipMemcpy(&accuI3, d_arr_partialSum, sizeof(numtype), hipMemcpyDeviceToHost);
			if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to copy arr_partialSum to the host! error = %s\n", hipGetErrorString(err));
				errC--; }

			// Reduce arr_I4
			PartialSum<<<gridSize, blockSize, blockSize*sizeof(numtype)>>>(d_arr_I4, d_arr_partialSum, totalElements);
			err = hipGetLastError();
			if (err != hipSuccess && errC > 0) {fprintf(stderr, "Failure in PartialSum (I4)! error = %s\n", hipGetErrorString(err)); errC--;}

			SequentialReduceSum<<<1,1>>>(d_arr_partialSum, gridSize);
			err = hipGetLastError();
			if (err != hipSuccess && errC > 0) {fprintf(stderr, "Failure in SequentialReduceSum (I4)! error = %s\n", hipGetErrorString(err)); errC--;}

			err = hipMemcpy(&accuI4, d_arr_partialSum, sizeof(numtype), hipMemcpyDeviceToHost);
			if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to copy arr_partialSum to the host! error = %s\n", hipGetErrorString(err));
				errC--; }

			// Reduce arr_I5
			PartialSum<<<gridSize, blockSize, blockSize*sizeof(numtype)>>>(d_arr_I5, d_arr_partialSum, totalElements);
			err = hipGetLastError();
			if (err != hipSuccess && errC > 0) {fprintf(stderr, "Failure in PartialSum (I5)! error = %s\n", hipGetErrorString(err)); errC--;}

			SequentialReduceSum<<<1,1>>>(d_arr_partialSum, gridSize);
			err = hipGetLastError();
			if (err != hipSuccess && errC > 0) {fprintf(stderr, "Failure in SequentialReduceSum (I5)! error = %s\n", hipGetErrorString(err)); errC--;}

			err = hipMemcpy(&accuI5, d_arr_partialSum, sizeof(numtype), hipMemcpyDeviceToHost);
			if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to copy arr_partialSum to the host! error = %s\n", hipGetErrorString(err));
				errC--; }

			// Reduce arr_I6
			PartialSum<<<gridSize, blockSize, blockSize*sizeof(numtype)>>>(d_arr_I6, d_arr_partialSum, totalElements);
			err = hipGetLastError();
			if (err != hipSuccess && errC > 0) {fprintf(stderr, "Failure in PartialSum (I6)! error = %s\n", hipGetErrorString(err)); errC--;}

			SequentialReduceSum<<<1,1>>>(d_arr_partialSum, gridSize);
			err = hipGetLastError();
			if (err != hipSuccess && errC > 0) {fprintf(stderr, "Failure in SequentialReduceSum (I6)! error = %s\n", hipGetErrorString(err)); errC--;}

			err = hipMemcpy(&accuI6, d_arr_partialSum, sizeof(numtype), hipMemcpyDeviceToHost);
			if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to copy arr_partialSum to the host! error = %s\n", hipGetErrorString(err));
				errC--; }

			// Reduce arr_I7
			PartialSum<<<gridSize, blockSize, blockSize*sizeof(numtype)>>>(d_arr_I7, d_arr_partialSum, totalElements);
			err = hipGetLastError();
			if (err != hipSuccess && errC > 0) {fprintf(stderr, "Failure in PartialSum (I7)! error = %s\n", hipGetErrorString(err)); errC--;}

			SequentialReduceSum<<<1,1>>>(d_arr_partialSum, gridSize);
			err = hipGetLastError();
			if (err != hipSuccess && errC > 0) {fprintf(stderr, "Failure in SequentialReduceSum (I7)! error = %s\n", hipGetErrorString(err)); errC--;}

			err = hipMemcpy(&accuI7, d_arr_partialSum, sizeof(numtype), hipMemcpyDeviceToHost);
			if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to copy arr_partialSum to the host! error = %s\n", hipGetErrorString(err));
				errC--; }

			// Reduce arr_I8
			PartialSum<<<gridSize, blockSize, blockSize*sizeof(numtype)>>>(d_arr_I8, d_arr_partialSum, totalElements);
			err = hipGetLastError();
			if (err != hipSuccess && errC > 0) {fprintf(stderr, "Failure in PartialSum (I8)! error = %s\n", hipGetErrorString(err)); errC--;}

			SequentialReduceSum<<<1,1>>>(d_arr_partialSum, gridSize);
			err = hipGetLastError();
			if (err != hipSuccess && errC > 0) {fprintf(stderr, "Failure in SequentialReduceSum (I8)! error = %s\n", hipGetErrorString(err)); errC--;}

			err = hipMemcpy(&accuI8, d_arr_partialSum, sizeof(numtype), hipMemcpyDeviceToHost);
			if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to copy arr_partialSum to the host! error = %s\n", hipGetErrorString(err));
				errC--; }

			// Reduce arr_I9
			PartialSum<<<gridSize, blockSize, blockSize*sizeof(numtype)>>>(d_arr_I9, d_arr_partialSum, totalElements);
			err = hipGetLastError();
			if (err != hipSuccess && errC > 0) {fprintf(stderr, "Failure in PartialSum (I9)! error = %s\n", hipGetErrorString(err)); errC--;}

			SequentialReduceSum<<<1,1>>>(d_arr_partialSum, gridSize);
			err = hipGetLastError();
			if (err != hipSuccess && errC > 0) {fprintf(stderr, "Failure in SequentialReduceSum (I9)! error = %s\n", hipGetErrorString(err)); errC--;}

			err = hipMemcpy(&accuI9, d_arr_partialSum, sizeof(numtype), hipMemcpyDeviceToHost);
			if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to copy arr_partialSum to the host! error = %s\n", hipGetErrorString(err));
				errC--; }

			// Reduce arr_P
			PartialSum<<<gridSize, blockSize, blockSize*sizeof(numtype)>>>(d_arr_P, d_arr_partialSum, totalElements);
			err = hipGetLastError();
			if (err != hipSuccess && errC > 0) {fprintf(stderr, "Failure in PartialSum (P)! error = %s\n", hipGetErrorString(err)); errC--;}

			SequentialReduceSum<<<1,1>>>(d_arr_partialSum, gridSize);
			err = hipGetLastError();
			if (err != hipSuccess && errC > 0) {fprintf(stderr, "Failure in SequentialReduceSum (P)! error = %s\n", hipGetErrorString(err)); errC--;}

			err = hipMemcpy(&accuP, d_arr_partialSum, sizeof(numtype), hipMemcpyDeviceToHost);
			if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to copy arr_partialSum to the host! error = %s\n", hipGetErrorString(err));
				errC--; }

			// Reduce arr_nC
			PartialSum<<<gridSize, blockSize, blockSize*sizeof(numtype)>>>(d_arr_nC, d_arr_partialSum, totalElements);
			err = hipGetLastError();
			if (err != hipSuccess && errC > 0) {fprintf(stderr, "Failure in PartialSum (nutrient)! error = %s\n", hipGetErrorString(err)); errC--;}

			SequentialReduceSum<<<1,1>>>(d_arr_partialSum, gridSize);
			err = hipGetLastError();
			if (err != hipSuccess && errC > 0) {fprintf(stderr, "Failure in SequentialReduceSum (nutrient)! error = %s\n", hipGetErrorString(err)); errC--;}

			err = hipMemcpy(&accuClusters, d_arr_partialSum, sizeof(numtype), hipMemcpyDeviceToHost);
			if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to copy arr_partialSum to the host! error = %s\n", hipGetErrorString(err));
				errC--; }

			// Reduce nz
			PartialNonZero<<<gridSize, blockSize, blockSize*sizeof(numtype)>>>(d_arr_B, d_arr_partialSum, totalElements);
			err = hipGetLastError();
			if (err != hipSuccess && errC > 0) {fprintf(stderr, "Failure in PartialSum! error = %s\n", hipGetErrorString(err)); errC--;}

			SequentialReduceSum<<<1,1>>>(d_arr_partialSum, gridSize);
			err = hipGetLastError();
			if (err != hipSuccess && errC > 0) {fprintf(stderr, "Failure in SequentialReduceSum! error = %s\n", hipGetErrorString(err)); errC--;}

			err = hipMemcpy(&nz, d_arr_partialSum, sizeof(numtype), hipMemcpyDeviceToHost);
			if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to copy arr_partialSum to the host! error = %s\n", hipGetErrorString(err));
				errC--; }

			numtype accuI = accuI0 + accuI1 + accuI2 + accuI3 + accuI4 + accuI5 + accuI6 + accuI7 + accuI8 + accuI9;

			ExportData_arr_reduced(T, accuB, accuI, accuP, accuNutrient, accuClusters, nz, filename_suffix);

		} else {
	#endif
			// cudaMemCpy to host
			CopyAllToHost();

			// Store the state
			ExportData_arr(T,filename_suffix);
	#if GPU_REDUCE_ARRAYS_EXPORT
		}
	#endif

		// Check for nutrient stability
		assert(accuNutrient >= 0);
		assert(accuNutrient <= n_0 * L * L * H);
	}

	/////////////////////////////////////////////////////
	// Main loop end ////////////////////////////////////
	/////////////////////////////////////////////////////

	if(Warn_delta) {
	cout << "\tWarning: Decay Probability Large!" << "\n";
	f_log  << "Warning: Decay Probability Large!" << "\n";
	}
	if(Warn_g) {
		cout << "\tWarning: Birth Probability Large!" << "\n";
		f_log  << "Warning: Birth Probability Large!" << "\n";
	}
	if(Warn_fastGrowth){
		cout << "\tWarning: Colonies growing too fast!" << "\n";
		f_log  << "Warning: Colonies growing too fast!" << "\n";
	}

	if(Warn_r){
		cout << "\tWarning: Infection Increase Probability Large!" << "\n";
		f_log  << "Warning: Infection Increase Probability Large!" << "\n";
	}

	if (OPTIMIZED_MAXOCCUPANCY    ){

    if(Warn_density){
		cout << "\tWarning: Maximum Density Large!" << "\n";
		f_log  << "Warning: Maximum Density Large!" << "\n";
	}


	}
	// Get stop time
	time_t  toc;
	time(&toc);

	// Calculate time difference
	float seconds = difftime(toc, tic);
	float hours   = floor(seconds/3600);
	float minutes = floor(seconds/60);
	minutes -= hours*60;
	seconds -= minutes*60 + hours*3600;

	cout << "\n";
	cout << "\tSimulation complete after ";
	if (hours > 0.0)   cout << hours   << " hours and ";
	if (minutes > 0.0) cout << minutes << " minutes and ";
	cout  << seconds << " seconds." << "\n";

	std::ofstream f_out;
	f_out.open(GetPath() + "/Completed_LOOP_DISTRIBUTED.txt",fstream::trunc);
	f_out << "\tSimulation complete after ";
	if (hours > 0.0)   f_out << hours   << " hours and ";
	if (minutes > 0.0) f_out << minutes << " minutes and ";
	f_out  << seconds << " seconds." << "\n";
	f_out.flush();
	f_out.close();

	// Write sucess to log
	if (exit) {
		f_log << ">>Simulation completed with exit flag<<" << "\n";
	} else {
		f_log << ">>Simulation completed without exit flag<<" << "\n";
	}

	std::ofstream f_timing;
	f_timing << "\t"       << setw(3) << difftime(toc, tic) << " s of total time" << "\n";

	f_timing.flush();
	f_timing.close();

	CopyAllToHost();

	// hipFree here!!
	hipFree(d_arr_nC );
	hipFree(d_arr_Occ);
	hipFree(d_arr_IsActive);
	hipFree(d_arr_partialSum);
	hipFree(d_rng_state);
	hipFree(d_arr_B);
	hipFree(d_arr_B_new);
	hipFree(d_arr_P);
	hipFree(d_arr_P_new);
	hipFree(d_arr_I0);
	hipFree(d_arr_I0_new);
	hipFree(d_arr_I1);
	hipFree(d_arr_I1_new);
	hipFree(d_arr_I2);
	hipFree(d_arr_I2_new);
	hipFree(d_arr_I3);
	hipFree(d_arr_I3_new);
	hipFree(d_arr_I4);
	hipFree(d_arr_I4_new);
	hipFree(d_arr_I5);
	hipFree(d_arr_I5_new);
	hipFree(d_arr_I6);
	hipFree(d_arr_I6_new);
	hipFree(d_arr_I7);
	hipFree(d_arr_I7_new);
	hipFree(d_arr_I8);
	hipFree(d_arr_I8_new);
	hipFree(d_arr_I9);
	hipFree(d_arr_I9_new);

	hipFree(d_arr_M);
	hipFree(d_arr_p);
	hipFree(d_arr_nutrient);
	hipFree(d_arr_nutrient_new);
	hipFree(d_arr_GrowthModifier);
	hipFree(d_Warn_g);
	hipFree(d_Warn_fastGrowth);
	hipFree(d_Warn_r);
	hipFree(d_Warn_delta);


	numtype accuB = 0.0;
	numtype accuI = 0.0;
	numtype accuP = 0.0;
	numtype accuClusters = 0.0;
	for (int i = 0; i < nGridXY; i++) {
		for (int j = 0; j < nGridXY; j++ ) {
			for (int k = 0; k < nGridZ; k++ ) {
				accuB += arr_B[i*nGridXY*nGridZ + j*nGridZ + k];
				accuI += arr_I0[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I1[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I2[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I3[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I4[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I5[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I6[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I7[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I8[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I9[i*nGridXY*nGridZ + j*nGridZ + k];
				accuP += arr_P[i*nGridXY*nGridZ + j*nGridZ + k];
				accuClusters += arr_nC[i*nGridXY*nGridZ + j*nGridZ + k];
			}
		}
	}
	return (int)(accuB+accuI+accuI+accuClusters);
}

/////////////////////////////////////////////////////////////////////
// GPU loop end
/////////////////////////////////////////////////////////////////////


int Colonies3D::Run_LoopDistributed_CPU(numtype T_end) {


	this->T_end = T_end;

	// Get start time
	time_t  tic;
	time(&tic);

	// Generate a path
	path = GeneratePath();

	// Initilize the simulation matrices
	Initialize();

	high_resolution_clock::time_point kernel_start;
	high_resolution_clock::duration kernel_elapsed;

	std::string filename_suffix = "loopDistributedCPU";
  	if (GPU_KERNEL_TIMING){
      std::string s = "kernel_timings_CPU_n" + std::to_string(nGridXY);
		OpenFileStream(f_kerneltimings, s);
    f_kerneltimings << "NC \t";
    f_kerneltimings << "MAXOCCUPANCY \t";
    f_kerneltimings << "BIRTH \t";
    f_kerneltimings << "INFECTIONS \t";
    f_kerneltimings << "NEWINFECTIONS \t";
    f_kerneltimings << "PHAGEDECAY \t";
    f_kerneltimings << "MOVEMENT \t";
    f_kerneltimings << "SWAPZERO \t";
    f_kerneltimings << "UPDATEOCCUPANCY \t";
    f_kerneltimings << "NUTRIENTDIFFUSION \t";
    f_kerneltimings << "SWAPZERO2";
		f_kerneltimings << "\n";
    }

	// Export data
	ExportData_arr(T,filename_suffix);

	// Determine the number of samples to take
	int nSamplings = nSamp*T_end;

	// Loop over samplings
	for (int n = 0; n < nSamplings; n++) {
		if (exit) break;

		// Determine the number of timesteps between sampings

		int nStepsPerSample = static_cast<int>(cpu_round(1 / (nSamp *  dT)));

		for (int t = 0; t < nStepsPerSample; t++) {
			if (exit) break;

			// Increase time
			T += dT;

			// Spawn phages
			if ((T_i >= 0) and (abs(T - T_i) < dT / 2)) {
				spawnPhages();
				T_i = -1;
			}

			// Reset density counter
			numtype maxOccupancy = 0.0;

			/////////////////////////////////////////////////////
			// Main loop start //////////////////////////////////
			/////////////////////////////////////////////////////

			// Kernel 1-2: nC update and maxOccupancy //////////////////////////////////////////////////////////////////////
      if (GPU_KERNEL_TIMING){
        kernel_start = high_resolution_clock::now();
      }

			for (int i = 0; i < nGridXY; i++) {
				if (exit) break;

				for (int j = 0; j < nGridXY; j++) {
					if (exit) break;

					for (int k = 0; k < nGridZ; k++) {
						if (exit) break;

						// Ensure nC is updated
						if (arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k] < arr_nC[i*nGridXY*nGridZ + j*nGridZ + k]){
							arr_nC[i*nGridXY*nGridZ + j*nGridZ + k] = arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k];
						}

					}
				}
			}
      if (GPU_KERNEL_TIMING){
        kernel_elapsed = high_resolution_clock::now() - kernel_start;
        f_kerneltimings << duration_cast<microseconds>(kernel_elapsed).count() << "\t";
      }

      if (GPU_KERNEL_TIMING){
        kernel_start = high_resolution_clock::now();
      }
			for (int i = 0; i < nGridXY; i++) {
				if (exit) break;

				for (int j = 0; j < nGridXY; j++) {
					if (exit) break;

					for (int k = 0; k < nGridZ; k++) {
						if (exit) break;

						// Skip empty sites
						if ((arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k] < 1) and (arr_P[i*nGridXY*nGridZ + j*nGridZ + k] < 1)) continue;

						// Record the maximum observed density
						if (arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k] > maxOccupancy) maxOccupancy = arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k];

					}
				}
			}

      if (GPU_KERNEL_TIMING){
        kernel_elapsed = high_resolution_clock::now() - kernel_start;
        f_kerneltimings << duration_cast<microseconds>(kernel_elapsed).count() << "\t";
      }

			// Kernel 3: Birth //////////////////////////////////////////////////////////////////////
      if (GPU_KERNEL_TIMING){
        kernel_start = high_resolution_clock::now();
      }
			for (int i = 0; i < nGridXY; i++) {
				if (exit) break;

				for (int j = 0; j < nGridXY; j++) {
					if (exit) break;

					for (int k = 0; k < nGridZ; k++) {
						if (exit) break;

						// Skip empty sites
						if ((arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k] < 1) and (arr_P[i*nGridXY*nGridZ + j*nGridZ + k] < 1)) {
							skipArray[i*nGridXY*nGridZ + j*nGridZ + k] = true;
							continue;
						} else {
							skipArray[i*nGridXY*nGridZ + j*nGridZ + k] = false;
						}

						numtype p = 0; // privatize
						numtype N = 0; // privatize

						// Compute the growth modifier
						numtype growthModifier = arr_nutrient[i*nGridXY*nGridZ + j*nGridZ + k] / (arr_nutrient[i*nGridXY*nGridZ + j*nGridZ + k] + K);
						arr_GrowthModifier[i*nGridXY*nGridZ + j*nGridZ + k] = growthModifier;

						p = g * growthModifier*dT;
						if (arr_nutrient[i*nGridXY*nGridZ + j*nGridZ + k] < 1) {
							p = 0;
						}

						if ((p > 0.1) and (!Warn_g)) {
							cout << "\tWarning: Birth Probability Large!" << "\n";
							f_log  << "Warning: Birth Probability Large!" << "\n";
							Warn_g = true;
						}

						/* BEGIN anden Map-kernel */
						N = ComputeEvents(arr_B[i*nGridXY*nGridZ + j*nGridZ + k], p, 1, i, j, k);
						// Ensure there is enough nutrient
						if ( N > arr_nutrient[i*nGridXY*nGridZ + j*nGridZ + k] ) {
								if (!Warn_fastGrowth) {
									cout << "\tWarning: Colonies growing too fast!" << "\n";
									f_log  << "Warning: Colonies growing too fast!" << "\n";
									Warn_fastGrowth = true;
								}

								// DETERMINITIC CHANGE
                            	// N = round( arr_nutrient[i*nGridXY*nGridZ + j*nGridZ + k] );
								N = arr_nutrient[i*nGridXY*nGridZ + j*nGridZ + k];
						}

						// Update count
						arr_B_new[i*nGridXY*nGridZ + j*nGridZ + k] += N;
						arr_nutrient[i*nGridXY*nGridZ + j*nGridZ + k] = max(0.0, arr_nutrient[i*nGridXY*nGridZ + j*nGridZ + k] - N);
						/* END anden Map-kernel */
					}
				}
			}
      if (GPU_KERNEL_TIMING){
        kernel_elapsed = high_resolution_clock::now() - kernel_start;
        f_kerneltimings << duration_cast<microseconds>(kernel_elapsed).count() << "\t";
      }

			// Kernel 4: Increase Infections ////////////////////////////////////////////////////////
      if (GPU_KERNEL_TIMING){
        kernel_start = high_resolution_clock::now();
      }
			for (int i = 0; i < nGridXY; i++) {
				if (exit) break;

				for (int j = 0; j < nGridXY; j++) {
					if (exit) break;

					for (int k = 0; k < nGridZ; k++) {
						if (exit) break;

						// Skip empty sites
						if (skipArray[i*nGridXY*nGridZ + j*nGridZ + k]) continue;

						numtype p = 0; // privatize
						numtype N = 0; // privatize

						// Compute the growth modifier
						numtype growthModifier = arr_GrowthModifier[i*nGridXY*nGridZ + j*nGridZ + k];

						// Compute beta
						numtype Beta = beta;
						if (reducedBeta) {
							Beta *= growthModifier;
						}

				 		if (r > 0.0) {
							/* BEGIN tredje Map-kernel */

							p = r*growthModifier*dT;
							if ((p > 0.25) and (!Warn_r)) {
								cout << "\tWarning: Infection Increase Probability Large!" << "\n";
								f_log  << "Warning: Infection Increase Probability Large!" << "\n";
								Warn_r = true;
							}
							N = ComputeEvents(arr_I9[i*nGridXY*nGridZ + j*nGridZ + k], p, 2, i, j, k);  // Bursting events

							// Update count
							arr_I9[i*nGridXY*nGridZ + j*nGridZ + k]    = max(0.0, arr_I9[i*nGridXY*nGridZ + j*nGridZ + k] - N);
							arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k]   = max(0.0, arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k] - N);
							// DETERMINITIC CHANGE
                            // arr_P_new[i*nGridXY*nGridZ + j*nGridZ + k] += round( (1 - alpha) * Beta * N);   // Phages which escape the colony
							// arr_M[i*nGridXY*nGridZ + j*nGridZ + k] = round(alpha * Beta * N);                        // Phages which reinfect the colony
							arr_P_new[i*nGridXY*nGridZ + j*nGridZ + k] += (1 - alpha) * Beta * N;   // Phages which escape the colony
                            arr_M[i*nGridXY*nGridZ + j*nGridZ + k] = alpha * Beta * N;

							// Non-bursting events
							N = ComputeEvents(arr_I8[i*nGridXY*nGridZ + j*nGridZ + k], p, 2, i, j, k);
							arr_I8[i*nGridXY*nGridZ + j*nGridZ + k] = max(0.0, arr_I8[i*nGridXY*nGridZ + j*nGridZ + k] - N);
							arr_I9[i*nGridXY*nGridZ + j*nGridZ + k] += N;

							N = ComputeEvents(arr_I7[i*nGridXY*nGridZ + j*nGridZ + k], p, 2, i, j, k);
							arr_I7[i*nGridXY*nGridZ + j*nGridZ + k] = max(0.0, arr_I7[i*nGridXY*nGridZ + j*nGridZ + k] - N);
							arr_I8[i*nGridXY*nGridZ + j*nGridZ + k] += N;

							N = ComputeEvents(arr_I6[i*nGridXY*nGridZ + j*nGridZ + k], p, 2, i, j, k);
							arr_I6[i*nGridXY*nGridZ + j*nGridZ + k] = max(0.0, arr_I6[i*nGridXY*nGridZ + j*nGridZ + k] - N);
							arr_I7[i*nGridXY*nGridZ + j*nGridZ + k] += N;

							N = ComputeEvents(arr_I5[i*nGridXY*nGridZ + j*nGridZ + k], p, 2, i, j, k);
							arr_I5[i*nGridXY*nGridZ + j*nGridZ + k] = max(0.0, arr_I5[i*nGridXY*nGridZ + j*nGridZ + k] - N);
							arr_I6[i*nGridXY*nGridZ + j*nGridZ + k] += N;

							N = ComputeEvents(arr_I4[i*nGridXY*nGridZ + j*nGridZ + k], p, 2, i, j, k);
							arr_I4[i*nGridXY*nGridZ + j*nGridZ + k] = max(0.0, arr_I4[i*nGridXY*nGridZ + j*nGridZ + k] - N);
							arr_I5[i*nGridXY*nGridZ + j*nGridZ + k] += N;

							N = ComputeEvents(arr_I3[i*nGridXY*nGridZ + j*nGridZ + k], p, 2, i, j, k);
							arr_I3[i*nGridXY*nGridZ + j*nGridZ + k] = max(0.0, arr_I3[i*nGridXY*nGridZ + j*nGridZ + k] - N);
							arr_I4[i*nGridXY*nGridZ + j*nGridZ + k] += N;

							N = ComputeEvents(arr_I2[i*nGridXY*nGridZ + j*nGridZ + k], p, 2, i, j, k);
							arr_I2[i*nGridXY*nGridZ + j*nGridZ + k] = max(0.0, arr_I2[i*nGridXY*nGridZ + j*nGridZ + k] - N);
							arr_I3[i*nGridXY*nGridZ + j*nGridZ + k] += N;

							N = ComputeEvents(arr_I1[i*nGridXY*nGridZ + j*nGridZ + k], p, 2, i, j, k);
							arr_I1[i*nGridXY*nGridZ + j*nGridZ + k] = max(0.0, arr_I1[i*nGridXY*nGridZ + j*nGridZ + k] - N);
							arr_I2[i*nGridXY*nGridZ + j*nGridZ + k] += N;

							N = ComputeEvents(arr_I0[i*nGridXY*nGridZ + j*nGridZ + k], p, 2, i, j, k);
							arr_I0[i*nGridXY*nGridZ + j*nGridZ + k] = max(0.0, arr_I0[i*nGridXY*nGridZ + j*nGridZ + k] - N);
							arr_I1[i*nGridXY*nGridZ + j*nGridZ + k] += N;

							/* END tredje Map-kernel */
						}
					}
				}
			}
      if (GPU_KERNEL_TIMING){
        kernel_elapsed = high_resolution_clock::now() - kernel_start;
        f_kerneltimings << duration_cast<microseconds>(kernel_elapsed).count() << "\t";
      }

			// Kernel 5: New infections ///////////////////////////////////////////////////////////////////
      if (GPU_KERNEL_TIMING){
        kernel_start = high_resolution_clock::now();
      }
			for (int i = 0; i < nGridXY; i++) {
				if (exit) break;

				for (int j = 0; j < nGridXY; j++) {
					if (exit) break;

					for (int k = 0; k < nGridZ; k++) {
						if (exit) break;

						// Skip empty sites
						if (skipArray[i*nGridXY*nGridZ + j*nGridZ + k]) continue;


						numtype p = 0; // privatize
						numtype N = 0; // privatize
												// numtype M = 0; // privatize

						// Compute beta
						numtype Beta = beta;
						if (reducedBeta) {
							Beta *= arr_GrowthModifier[i*nGridXY*nGridZ + j*nGridZ + k];
						}

						// PRIVATIZE BOTH OF THESE
						// numtype s;   // The factor which modifies the adsorption rate
						// numtype n;   // The number of targets the phage has
												// Infectons


												// KERNEL THIS
						// if ((arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k] >= 1) and (arr_P[i*nGridXY*nGridZ + j*nGridZ + k] >= 1)) {
						// 	if (clustering) {   // Check if clustering is enabled
						// 		s = pow(arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k] / arr_nC[i*nGridXY*nGridZ + j*nGridZ + k], 1.0 / 3.0);
						// 		n = arr_nC[i*nGridXY*nGridZ + j*nGridZ + k];
						// 	} else {            // Else use mean field computation
						// 		s = 1.0;
						// 		n = arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k];
						// 	}
						// }

						if ((arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k] >= 1) and (arr_P[i*nGridXY*nGridZ + j*nGridZ + k] >= 1)) {
							// Compute the number of hits
							// if (eta * s * dT >= 1) { // In the diffusion limited case every phage hits a target
								N = arr_P[i*nGridXY*nGridZ + j*nGridZ + k];
							// } else {
							// 	p = 1 - pow(1 - eta * s * dT, n);        // Probability hitting any target
							// 	N = ComputeEvents(arr_P[i*nGridXY*nGridZ + j*nGridZ + k], p, 4, i, j, k);     // Number of targets hit
							// }

							// If bacteria were hit, update events
							// DETERMINITIC CHANGE
							// if (N + arr_M[i*nGridXY*nGridZ + j*nGridZ + k] >= 1) {

								arr_P[i*nGridXY*nGridZ + j*nGridZ + k] = max(0.0, arr_P[i*nGridXY*nGridZ + j*nGridZ + k] - N);     // Update count

								numtype S;
								if (shielding) {
									// Absorbing medium model
									numtype d = pow(arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k] / arr_nC[i*nGridXY*nGridZ + j*nGridZ + k], 1.0 / 3.0) -
										pow(arr_B[i*nGridXY*nGridZ + j*nGridZ + k] / arr_nC[i*nGridXY*nGridZ + j*nGridZ + k], 1.0 / 3.0);
									S = cpu_exp(-zeta * d); // Probability of hitting succebtible target

								} else {
									// Well mixed model
									S = arr_B[i*nGridXY*nGridZ + j*nGridZ + k] / arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k];
								}

								p = max(0.0, min(arr_B[i*nGridXY*nGridZ + j*nGridZ + k] / arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k],
																	S)); // Probability of hitting succebtible target
								N = ComputeEvents(N + arr_M[i*nGridXY*nGridZ + j*nGridZ + k], p, 4, i, j, k);                  // Number of targets hit

								if (N > arr_B[i*nGridXY*nGridZ + j*nGridZ + k])
									N = arr_B[i*nGridXY*nGridZ + j*nGridZ + k];              // If more bacteria than present are set to be infeced, round down

								// Update the counts
								arr_B[i*nGridXY*nGridZ + j*nGridZ + k] = max(0.0, arr_B[i*nGridXY*nGridZ + j*nGridZ + k] - N);
								if (r > 0.0) {
									arr_I0_new[i*nGridXY*nGridZ + j*nGridZ + k] += N;
								} else {
									arr_P_new[i*nGridXY*nGridZ + j*nGridZ + k] += N * (1 - alpha) * Beta;
								}
							// }
						}
 					}
				}
			}
      if (GPU_KERNEL_TIMING){
        kernel_elapsed = high_resolution_clock::now() - kernel_start;
        f_kerneltimings << duration_cast<microseconds>(kernel_elapsed).count() << "\t";
      }

			// Kernel 6: Phage decay ///////////////////////////////////////////////////////////////////
      if (GPU_KERNEL_TIMING){
        kernel_start = high_resolution_clock::now();
      }
			for (int i = 0; i < nGridXY; i++) {
				if (exit) break;

				for (int j = 0; j < nGridXY; j++) {
					if (exit) break;

					for (int k = 0; k < nGridZ; k++) {
						if (exit) break;

						// Skip empty sites
						if (skipArray[i*nGridXY*nGridZ + j*nGridZ + k]) continue;


						numtype p = 0; // privatize
						numtype N = 0; // privatize

						// KERNEL BEGIN
						p = delta*dT;
						if ((p > 0.1) and (!Warn_delta)) {
								cout << "\tWarning: Decay Probability Large!" << "\n";
								f_log  << "Warning: Decay Probability Large!" << "\n";
								Warn_delta = true;
						}
						N = ComputeEvents(arr_P[i*nGridXY*nGridZ + j*nGridZ + k], p, 5, i, j, k);

						// Update count
						arr_P[i*nGridXY*nGridZ + j*nGridZ + k]    = max(0.0, arr_P[i*nGridXY*nGridZ + j*nGridZ + k] - N);
						// KERNEL END

					}
				}
			}
      if (GPU_KERNEL_TIMING){
        kernel_elapsed = high_resolution_clock::now() - kernel_start;
        f_kerneltimings << duration_cast<microseconds>(kernel_elapsed).count() << "\t";
      }


			// Movement ///////////////////////////////////////////////////////////////////
      if (GPU_KERNEL_TIMING){
        kernel_start = high_resolution_clock::now();
      }
			for (int i = 0; i < nGridXY; i++) {
				if (exit) break;

				for (int j = 0; j < nGridXY; j++) {
					if (exit) break;

					for (int k = 0; k < nGridZ; k++) {
						if (exit) break;

						// Skip empty sites
						if (skipArray[i*nGridXY*nGridZ + j*nGridZ + k]) continue;


						if (nGridXY > 1) {
							// Update positions

							// Skip empty sites
							if (skipArray[i*nGridXY*nGridZ + j*nGridZ + k]) continue;

							int ip, jp, kp, im, jm, km;

							if (i + 1 >= nGridXY) ip = i + 1 - nGridXY;
							else ip = i + 1;

							if (i == 0) im = nGridXY - 1;
							else im = i - 1;

							if (j + 1 >= nGridXY) jp = j + 1 - nGridXY;
							else jp = j + 1;

							if (j == 0) jm = nGridXY - 1;
							else jm = j - 1;

							if (not experimentalConditions) {   // Periodic boundaries in Z direction

								if (k + 1 >= nGridZ) kp = k + 1 - nGridZ;
								else kp = k + 1;

								if (k == 0) km = nGridZ - 1;
								else km = k - 1;

							} else {    // Reflective boundaries in Z direction

								if (k + 1 >= nGridZ) kp = k - 1;
								else kp = k + 1;

								if (k == 0) km = k + 1;
								else km = k - 1;

							}

							// Update counts
							numtype n_0; // No movement
							numtype n_u; // Up
							numtype n_d; // Down
							numtype n_l; // Left
							numtype n_r; // Right
							numtype n_f; // Front
							numtype n_b; // Back

							// CELLS
							ComputeDiffusion(arr_B[i*nGridXY*nGridZ + j*nGridZ + k], lambdaB, &n_0, &n_u, &n_d, &n_l, &n_r, &n_f, &n_b, 1, i, j, k);
							arr_B_new[i*nGridXY*nGridZ + j*nGridZ + k] += n_0;
							if (lambdaB > 0) {
								arr_B_new[ip*nGridXY*nGridZ + j*nGridZ + k] += n_u; arr_B_new[im*nGridXY*nGridZ + j*nGridZ + k] += n_d; arr_B_new[i*nGridXY*nGridZ + jp*nGridZ + k] += n_r; arr_B_new[i*nGridXY*nGridZ + jm*nGridZ + k] += n_l; arr_B_new[i*nGridXY*nGridZ + j*nGridZ + kp] += n_f; arr_B_new[i*nGridXY*nGridZ + j*nGridZ + km] += n_b;
							}

							if (r > 0.0) {
								ComputeDiffusion(arr_I0[i*nGridXY*nGridZ + j*nGridZ + k],  lambdaB, &n_0, &n_u, &n_d, &n_l, &n_r, &n_f, &n_b, 2, i, j, k);
								arr_I0_new[i*nGridXY*nGridZ + j*nGridZ + k] += n_0;
								if (lambdaB > 0) {
									arr_I0_new[ip*nGridXY*nGridZ + j*nGridZ + k] += n_u; arr_I0_new[im*nGridXY*nGridZ + j*nGridZ + k] += n_d; arr_I0_new[i*nGridXY*nGridZ + jp*nGridZ + k] += n_r; arr_I0_new[i*nGridXY*nGridZ + jm*nGridZ + k] += n_l; arr_I0_new[i*nGridXY*nGridZ + j*nGridZ + kp] += n_f; arr_I0_new[i*nGridXY*nGridZ + j*nGridZ + km] += n_b;
								}

								ComputeDiffusion(arr_I1[i*nGridXY*nGridZ + j*nGridZ + k],  lambdaB, &n_0, &n_u, &n_d, &n_l, &n_r, &n_f, &n_b, 2, i, j, k);
								arr_I1_new[i*nGridXY*nGridZ + j*nGridZ + k] += n_0;
								if (lambdaB > 0) {
									arr_I1_new[ip*nGridXY*nGridZ + j*nGridZ + k] += n_u; arr_I1_new[im*nGridXY*nGridZ + j*nGridZ + k] += n_d; arr_I1_new[i*nGridXY*nGridZ + jp*nGridZ + k] += n_r; arr_I1_new[i*nGridXY*nGridZ + jm*nGridZ + k] += n_l; arr_I1_new[i*nGridXY*nGridZ + j*nGridZ + kp] += n_f; arr_I1_new[i*nGridXY*nGridZ + j*nGridZ + km] += n_b;
								}

								ComputeDiffusion(arr_I2[i*nGridXY*nGridZ + j*nGridZ + k],  lambdaB, &n_0, &n_u, &n_d, &n_l, &n_r, &n_f, &n_b, 2, i, j, k);
								arr_I2_new[i*nGridXY*nGridZ + j*nGridZ + k] += n_0;
								if (lambdaB > 0) {
									arr_I2_new[ip*nGridXY*nGridZ + j*nGridZ + k] += n_u; arr_I2_new[im*nGridXY*nGridZ + j*nGridZ + k] += n_d; arr_I2_new[i*nGridXY*nGridZ + jp*nGridZ + k] += n_r; arr_I2_new[i*nGridXY*nGridZ + jm*nGridZ + k] += n_l; arr_I2_new[i*nGridXY*nGridZ + j*nGridZ + kp] += n_f; arr_I2_new[i*nGridXY*nGridZ + j*nGridZ + km] += n_b;
								}

								ComputeDiffusion(arr_I3[i*nGridXY*nGridZ + j*nGridZ + k],  lambdaB, &n_0, &n_u, &n_d, &n_l, &n_r, &n_f, &n_b, 2, i, j, k);
								arr_I3_new[i*nGridXY*nGridZ + j*nGridZ + k] += n_0;
								if (lambdaB > 0) {
									arr_I3_new[ip*nGridXY*nGridZ + j*nGridZ + k] += n_u; arr_I3_new[im*nGridXY*nGridZ + j*nGridZ + k] += n_d; arr_I3_new[i*nGridXY*nGridZ + jp*nGridZ + k] += n_r; arr_I3_new[i*nGridXY*nGridZ + jm*nGridZ + k] += n_l; arr_I3_new[i*nGridXY*nGridZ + j*nGridZ + kp] += n_f; arr_I3_new[i*nGridXY*nGridZ + j*nGridZ + km] += n_b;
								}

								ComputeDiffusion(arr_I4[i*nGridXY*nGridZ + j*nGridZ + k],  lambdaB, &n_0, &n_u, &n_d, &n_l, &n_r, &n_f, &n_b, 2, i, j, k);
								arr_I4_new[i*nGridXY*nGridZ + j*nGridZ + k] += n_0;
								if (lambdaB > 0) {
									arr_I4_new[ip*nGridXY*nGridZ + j*nGridZ + k] += n_u; arr_I4_new[im*nGridXY*nGridZ + j*nGridZ + k] += n_d; arr_I4_new[i*nGridXY*nGridZ + jp*nGridZ + k] += n_r; arr_I4_new[i*nGridXY*nGridZ + jm*nGridZ + k] += n_l; arr_I4_new[i*nGridXY*nGridZ + j*nGridZ + kp] += n_f; arr_I4_new[i*nGridXY*nGridZ + j*nGridZ + km] += n_b;
								}

								ComputeDiffusion(arr_I5[i*nGridXY*nGridZ + j*nGridZ + k],  lambdaB, &n_0, &n_u, &n_d, &n_l, &n_r, &n_f, &n_b, 2, i, j, k);
								arr_I5_new[i*nGridXY*nGridZ + j*nGridZ + k] += n_0;
								if (lambdaB > 0) {
									arr_I5_new[ip*nGridXY*nGridZ + j*nGridZ + k] += n_u; arr_I5_new[im*nGridXY*nGridZ + j*nGridZ + k] += n_d; arr_I5_new[i*nGridXY*nGridZ + jp*nGridZ + k] += n_r; arr_I5_new[i*nGridXY*nGridZ + jm*nGridZ + k] += n_l; arr_I5_new[i*nGridXY*nGridZ + j*nGridZ + kp] += n_f; arr_I5_new[i*nGridXY*nGridZ + j*nGridZ + km] += n_b;
								}

								ComputeDiffusion(arr_I6[i*nGridXY*nGridZ + j*nGridZ + k],  lambdaB, &n_0, &n_u, &n_d, &n_l, &n_r, &n_f, &n_b, 2, i, j, k);
								arr_I6_new[i*nGridXY*nGridZ + j*nGridZ + k] += n_0;
								if (lambdaB > 0) {
									arr_I6_new[ip*nGridXY*nGridZ + j*nGridZ + k] += n_u; arr_I6_new[im*nGridXY*nGridZ + j*nGridZ + k] += n_d; arr_I6_new[i*nGridXY*nGridZ + jp*nGridZ + k] += n_r; arr_I6_new[i*nGridXY*nGridZ + jm*nGridZ + k] += n_l; arr_I6_new[i*nGridXY*nGridZ + j*nGridZ + kp] += n_f; arr_I6_new[i*nGridXY*nGridZ + j*nGridZ + km] += n_b;
								}

								ComputeDiffusion(arr_I7[i*nGridXY*nGridZ + j*nGridZ + k],  lambdaB, &n_0, &n_u, &n_d, &n_l, &n_r, &n_f, &n_b, 2, i, j, k);
								arr_I7_new[i*nGridXY*nGridZ + j*nGridZ + k] += n_0;
								if (lambdaB > 0) {
									arr_I7_new[ip*nGridXY*nGridZ + j*nGridZ + k] += n_u; arr_I7_new[im*nGridXY*nGridZ + j*nGridZ + k] += n_d; arr_I7_new[i*nGridXY*nGridZ + jp*nGridZ + k] += n_r; arr_I7_new[i*nGridXY*nGridZ + jm*nGridZ + k] += n_l; arr_I7_new[i*nGridXY*nGridZ + j*nGridZ + kp] += n_f; arr_I7_new[i*nGridXY*nGridZ + j*nGridZ + km] += n_b;
								}

								ComputeDiffusion(arr_I8[i*nGridXY*nGridZ + j*nGridZ + k],  lambdaB, &n_0, &n_u, &n_d, &n_l, &n_r, &n_f, &n_b, 2, i, j, k);
								arr_I8_new[i*nGridXY*nGridZ + j*nGridZ + k] += n_0;
								if (lambdaB > 0) {
									arr_I8_new[ip*nGridXY*nGridZ + j*nGridZ + k] += n_u; arr_I8_new[im*nGridXY*nGridZ + j*nGridZ + k] += n_d; arr_I8_new[i*nGridXY*nGridZ + jp*nGridZ + k] += n_r; arr_I8_new[i*nGridXY*nGridZ + jm*nGridZ + k] += n_l; arr_I8_new[i*nGridXY*nGridZ + j*nGridZ + kp] += n_f; arr_I8_new[i*nGridXY*nGridZ + j*nGridZ + km] += n_b;
								}

								ComputeDiffusion(arr_I9[i*nGridXY*nGridZ + j*nGridZ + k], lambdaB, &n_0, &n_u, &n_d, &n_l, &n_r, &n_f, &n_b, 2, i, j, k);
								arr_I9_new[i*nGridXY*nGridZ + j*nGridZ + k] += n_0;
								if (lambdaB > 0) {
									arr_I9_new[ip*nGridXY*nGridZ + j*nGridZ + k] += n_u; arr_I9_new[im*nGridXY*nGridZ + j*nGridZ + k] += n_d; arr_I9_new[i*nGridXY*nGridZ + jp*nGridZ + k] += n_r; arr_I9_new[i*nGridXY*nGridZ + jm*nGridZ + k] += n_l; arr_I9_new[i*nGridXY*nGridZ + j*nGridZ + kp] += n_f; arr_I9_new[i*nGridXY*nGridZ + j*nGridZ + km] += n_b;
								}
							}

							// PHAGES
							ComputeDiffusion(arr_P[i*nGridXY*nGridZ + j*nGridZ + k], lambdaP, &n_0, &n_u, &n_d, &n_l, &n_r, &n_f, &n_b, 3, i, j, k);
							arr_P_new[i*nGridXY*nGridZ + j*nGridZ + k] += n_0;
							if (lambdaP > 0) {
								arr_P_new[ip*nGridXY*nGridZ + j*nGridZ + k] += n_u; arr_P_new[im*nGridXY*nGridZ + j*nGridZ + k] += n_d; arr_P_new[i*nGridXY*nGridZ + jp*nGridZ + k] += n_r; arr_P_new[i*nGridXY*nGridZ + jm*nGridZ + k] += n_l; arr_P_new[i*nGridXY*nGridZ + j*nGridZ + kp] += n_f; arr_P_new[i*nGridXY*nGridZ + j*nGridZ + km] += n_b;
							}

							// KERNEL END



						} else {
							// KERNEL BEGIN
							// CELLS
							// Skip empty sites
							if ((arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k] < 1) and (arr_P[i*nGridXY*nGridZ + j*nGridZ + k] < 1)) continue;

							arr_B_new[i*nGridXY*nGridZ + j*nGridZ + k] += arr_B[i*nGridXY*nGridZ + j*nGridZ + k];

							if (r > 0.0) {
								arr_I0_new[i*nGridXY*nGridZ + j*nGridZ + k] += arr_I0[i*nGridXY*nGridZ + j*nGridZ + k];
								arr_I1_new[i*nGridXY*nGridZ + j*nGridZ + k] += arr_I1[i*nGridXY*nGridZ + j*nGridZ + k];
								arr_I2_new[i*nGridXY*nGridZ + j*nGridZ + k] += arr_I2[i*nGridXY*nGridZ + j*nGridZ + k];
								arr_I3_new[i*nGridXY*nGridZ + j*nGridZ + k] += arr_I3[i*nGridXY*nGridZ + j*nGridZ + k];
								arr_I4_new[i*nGridXY*nGridZ + j*nGridZ + k] += arr_I4[i*nGridXY*nGridZ + j*nGridZ + k];
								arr_I5_new[i*nGridXY*nGridZ + j*nGridZ + k] += arr_I5[i*nGridXY*nGridZ + j*nGridZ + k];
								arr_I6_new[i*nGridXY*nGridZ + j*nGridZ + k] += arr_I6[i*nGridXY*nGridZ + j*nGridZ + k];
								arr_I7_new[i*nGridXY*nGridZ + j*nGridZ + k] += arr_I7[i*nGridXY*nGridZ + j*nGridZ + k];
								arr_I8_new[i*nGridXY*nGridZ + j*nGridZ + k] += arr_I8[i*nGridXY*nGridZ + j*nGridZ + k];
								arr_I9_new[i*nGridXY*nGridZ + j*nGridZ + k] += arr_I9[i*nGridXY*nGridZ + j*nGridZ + k];
							}

							// PHAGES
							arr_P_new[i*nGridXY*nGridZ + j*nGridZ + k] += arr_P[i*nGridXY*nGridZ + j*nGridZ + k];
						}
					}
				}
			}
      if (GPU_KERNEL_TIMING){
        kernel_elapsed = high_resolution_clock::now() - kernel_start;
        f_kerneltimings << duration_cast<microseconds>(kernel_elapsed).count() << "\t";
      }

      if (GPU_KERNEL_TIMING){
        kernel_start = high_resolution_clock::now();
      }
			// Swap pointers
			std::swap(arr_B, arr_B_new);
			std::swap(arr_I0, arr_I0_new);
			std::swap(arr_I1, arr_I1_new);
			std::swap(arr_I2, arr_I2_new);
			std::swap(arr_I3, arr_I3_new);
			std::swap(arr_I4, arr_I4_new);
			std::swap(arr_I5, arr_I5_new);
			std::swap(arr_I6, arr_I6_new);
			std::swap(arr_I7, arr_I7_new);
			std::swap(arr_I8, arr_I8_new);
			std::swap(arr_I9, arr_I9_new);
			std::swap(arr_P, arr_P_new);

			// Zero the _new arrays
			for (int i = 0; i < nGridXY; i++) {
				for (int j = 0; j < nGridXY; j++ ) {
					for (int k = 0; k < nGridZ; k++ ) {
						arr_B_new[i*nGridXY*nGridZ + j*nGridZ + k]  = 0.0;
						arr_I0_new[i*nGridXY*nGridZ + j*nGridZ + k] = 0.0;
						arr_I1_new[i*nGridXY*nGridZ + j*nGridZ + k] = 0.0;
						arr_I2_new[i*nGridXY*nGridZ + j*nGridZ + k] = 0.0;
						arr_I3_new[i*nGridXY*nGridZ + j*nGridZ + k] = 0.0;
						arr_I4_new[i*nGridXY*nGridZ + j*nGridZ + k] = 0.0;
						arr_I5_new[i*nGridXY*nGridZ + j*nGridZ + k] = 0.0;
						arr_I6_new[i*nGridXY*nGridZ + j*nGridZ + k] = 0.0;
						arr_I7_new[i*nGridXY*nGridZ + j*nGridZ + k] = 0.0;
						arr_I8_new[i*nGridXY*nGridZ + j*nGridZ + k] = 0.0;
						arr_I9_new[i*nGridXY*nGridZ + j*nGridZ + k] = 0.0;
						arr_P_new[i*nGridXY*nGridZ + j*nGridZ + k]  = 0.0;
					}
				}
			}
      if (GPU_KERNEL_TIMING){
        kernel_elapsed = high_resolution_clock::now() - kernel_start;
        f_kerneltimings << duration_cast<microseconds>(kernel_elapsed).count() << "\t";
      }


      if (GPU_KERNEL_TIMING){
        hipDeviceSynchronize();
        kernel_start = high_resolution_clock::now();
      }
				// Update occupancy
				for (int i = 0; i < nGridXY; i++) {
					for (int j = 0; j < nGridXY; j++ ) {
						for (int k = 0; k < nGridZ; k++ ) {
							arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k] = arr_B[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I0[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I1[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I2[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I3[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I4[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I5[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I6[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I7[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I8[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I9[i*nGridXY*nGridZ + j*nGridZ + k];
						}
					}
				}
      if (GPU_KERNEL_TIMING){
        kernel_elapsed = high_resolution_clock::now() - kernel_start;
        f_kerneltimings << duration_cast<microseconds>(kernel_elapsed).count() << "\t";
      }


      if (GPU_KERNEL_TIMING){
        hipDeviceSynchronize();
        kernel_start = high_resolution_clock::now();
      }
				// NUTRIENT DIFFUSION
				numtype alphaXY = D_n * dT / pow(L / (numtype)nGridXY, 2);
				numtype alphaZ  = D_n * dT / pow(H / (numtype)nGridZ, 2);

				for (int i = 0; i < nGridXY; i++) {
					for (int j = 0; j < nGridXY; j++ ) {
						for (int k = 0; k < nGridZ; k++ ) {

							// Update positions
							int ip, jp, kp, im, jm, km;

							if (i + 1 >= nGridXY) ip = i + 1 - nGridXY;
							else ip = i + 1;

							if (i == 0) im = nGridXY - 1;
							else im = i - 1;

							if (j + 1 >= nGridXY) jp = j + 1 - nGridXY;
							else jp = j + 1;

							if (j == 0) jm = nGridXY - 1;
							else jm = j - 1;

							if (not experimentalConditions) {   // Periodic boundaries in Z direction

								if (k + 1 >= nGridZ) kp = k + 1 - nGridZ;
								else kp = k + 1;

								if (k == 0) km = nGridZ - 1;
								else km = k - 1;

							} else {    // Reflective boundaries in Z direction

								if (k + 1 >= nGridZ) kp = k - 1;
								else kp = k + 1;

								if (k == 0) km = k + 1;
								else km = k - 1;

							}

							numtype tmp = arr_nutrient[i*nGridXY*nGridZ + j*nGridZ + k];
							arr_nutrient_new[i*nGridXY*nGridZ + j*nGridZ + k]  += tmp - (4 * alphaXY + 2 * alphaZ) * tmp;
							arr_nutrient_new[ip*nGridXY*nGridZ + j*nGridZ + k] += alphaXY * tmp;
							arr_nutrient_new[im*nGridXY*nGridZ + j*nGridZ + k] += alphaXY * tmp;
							arr_nutrient_new[i*nGridXY*nGridZ + jp*nGridZ + k] += alphaXY * tmp;
							arr_nutrient_new[i*nGridXY*nGridZ + jm*nGridZ + k] += alphaXY * tmp;
							arr_nutrient_new[i*nGridXY*nGridZ + j*nGridZ + kp] += alphaZ  * tmp;
							arr_nutrient_new[i*nGridXY*nGridZ + j*nGridZ + km] += alphaZ  * tmp;
						}
					}
				}
      if (GPU_KERNEL_TIMING){
        kernel_elapsed = high_resolution_clock::now() - kernel_start;
        f_kerneltimings << duration_cast<microseconds>(kernel_elapsed).count() << "\t";
      }

      if (GPU_KERNEL_TIMING){
        kernel_start = high_resolution_clock::now();
      }
				std::swap(arr_nutrient, arr_nutrient_new);
      if (GPU_KERNEL_TIMING){
        kernel_elapsed = high_resolution_clock::now() - kernel_start;
        f_kerneltimings << duration_cast<microseconds>(kernel_elapsed).count() << "\n";
      }

				// Zero the _new arrays
				for (int i = 0; i < nGridXY; i++) {
					for (int j = 0; j < nGridXY; j++ ) {
						for (int k = 0; k < nGridZ; k++ ) {
								arr_nutrient_new[i*nGridXY*nGridZ + j*nGridZ + k]  = 0.0;
						}
					}
				}

				if ((maxOccupancy > L * L * H / (nGridXY * nGridXY * nGridZ)) and (!Warn_density)) {
					cout << "\tWarning: Maximum Density Large!" << "\n";
					f_log  << "Warning: Maximum Density Large!" << "\n";
					Warn_density = true;
				}
		}

		// Fast exit conditions
		// 1) There are no more sucebtible cells
		// -> Convert all infected cells to phages and stop simulation
		numtype accuB = 0.0;
		for (int i = 0; i < nGridXY; i++) {
			for (int j = 0; j < nGridXY; j++ ) {
				for (int k = 0; k < nGridZ; k++ ) {
					accuB += arr_B[i*nGridXY*nGridZ + j*nGridZ + k];
				}
			}
		}
		if ((fastExit) and (accuB < 1)) {
			// Update the P array
			for (int i = 0; i < nGridXY; i++) {
				for (int j = 0; j < nGridXY; j++ ) {
					for (int k = 0; k < nGridZ; k++ ) {
						arr_P[i*nGridXY*nGridZ + j*nGridZ + k] += (1-alpha)*beta * (arr_I0[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I1[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I2[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I3[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I4[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I5[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I6[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I7[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I8[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I9[i*nGridXY*nGridZ + j*nGridZ + k]);
					}
				}
			}


			// Zero the I arrays
			for (int i = 0; i < nGridXY; i++) {
				for (int j = 0; j < nGridXY; j++ ) {
					for (int k = 0; k < nGridZ; k++ ) {
						arr_I0[i*nGridXY*nGridZ + j*nGridZ + k] = 0.0;
						arr_I1[i*nGridXY*nGridZ + j*nGridZ + k] = 0.0;
						arr_I2[i*nGridXY*nGridZ + j*nGridZ + k] = 0.0;
						arr_I3[i*nGridXY*nGridZ + j*nGridZ + k] = 0.0;
						arr_I4[i*nGridXY*nGridZ + j*nGridZ + k] = 0.0;
						arr_I5[i*nGridXY*nGridZ + j*nGridZ + k] = 0.0;
						arr_I6[i*nGridXY*nGridZ + j*nGridZ + k] = 0.0;
						arr_I7[i*nGridXY*nGridZ + j*nGridZ + k] = 0.0;
						arr_I8[i*nGridXY*nGridZ + j*nGridZ + k] = 0.0;
						arr_I9[i*nGridXY*nGridZ + j*nGridZ + k] = 0.0;
					}
				}
			}
			exit = true;
		}

		// 2) There are no more alive cells
		// -> Stop simulation

		numtype accuOcc = 0.0;
		for (int i = 0; i < nGridXY; i++) {
			for (int j = 0; j < nGridXY; j++ ) {
				for (int k = 0; k < nGridZ; k++ ) {
					accuOcc += arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k];
				}
			}
		}

		if ((fastExit) and (accuOcc < 1)) {
				exit = true;
		}

		// 3) The food is on average less than one per gridpoint
		// and the maximal nutrient at any point in space is less than 1

		numtype accuNutrient = 0.0;
		numtype maxNutrient  = 0.0;
		for (int i = 0; i < nGridXY; i++) {
			for (int j = 0; j < nGridXY; j++ ) {
				for (int k = 0; k < nGridZ; k++ ) {
					numtype tmpN = arr_nutrient[i*nGridXY*nGridZ + j*nGridZ + k];
					accuNutrient += tmpN;

					if (tmpN > maxNutrient) {
						maxNutrient = tmpN;
					}
				}
			}
		}

		if (fastExit) {
			if  ((accuNutrient < nGridZ*pow(nGridXY,2)) && (maxNutrient < 0.5)) {
				exit = true;
			}
		}

		// Store the state
		ExportData_arr(T,filename_suffix);

		// Check for nutrient stability
		assert(accuNutrient >= 0);
		assert(accuNutrient <= n_0 * L * L * H);
	}

	/////////////////////////////////////////////////////
	// Main loop end ////////////////////////////////////
	/////////////////////////////////////////////////////


	// Get stop time
	time_t  toc;
	time(&toc);

	// Calculate time difference
	float seconds = difftime(toc, tic);
	float hours   = floor(seconds/3600);
	float minutes = floor(seconds/60);
	minutes -= hours*60;
	seconds -= minutes*60 + hours*3600;

	cout << "\n";
	cout << "\tSimulation complete after ";
	if (hours > 0.0)   cout << hours   << " hours and ";
	if (minutes > 0.0) cout << minutes << " minutes and ";
	cout  << seconds << " seconds." << "\n";

	std::ofstream f_out;
	f_out.open(GetPath() + "/Completed_LOOP_DISTRIBUTED.txt",fstream::trunc);
	f_out << "\tSimulation complete after ";
	if (hours > 0.0)   f_out << hours   << " hours and ";
	if (minutes > 0.0) f_out << minutes << " minutes and ";
	f_out  << seconds << " seconds." << "\n";
	f_out.flush();
	f_out.close();

	// Write sucess to log
	if (exit) {
			f_log << ">>Simulation completed with exit flag<<" << "\n";
	} else {
			f_log << ">>Simulation completed without exit flag<<" << "\n";
	}

	std::ofstream f_timing;
	f_timing << "\t"       << setw(3) << difftime(toc, tic) << " s of total time" << "\n";

	f_timing.flush();
	f_timing.close();


	numtype accuB = 0.0;
	numtype accuI = 0.0;
	numtype accuP = 0.0;
	numtype accuClusters = 0.0;
	for (int i = 0; i < nGridXY; i++) {
		for (int j = 0; j < nGridXY; j++ ) {
			for (int k = 0; k < nGridZ; k++ ) {
				accuB += arr_B[i*nGridXY*nGridZ + j*nGridZ + k];
				accuI += arr_I0[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I1[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I2[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I3[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I4[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I5[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I6[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I7[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I8[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I9[i*nGridXY*nGridZ + j*nGridZ + k];
				accuP += arr_P[i*nGridXY*nGridZ + j*nGridZ + k];
				accuClusters += arr_nC[i*nGridXY*nGridZ + j*nGridZ + k];
			}
		}
	}
	return (int)(accuB+accuI+accuI+accuClusters);

}

int Colonies3D::Run_LoopDistributed_CPU_cuRand(numtype T_end) {
	std::string filename_suffix = "loopDistributedCPU_cuRand";

	this->T_end = T_end;

	// Get start time
	time_t  tic;
	time(&tic);

	// Generate a path
	path = GeneratePath();

	// Initilize the simulation matrices
	Initialize();

	// Export data
	ExportData_arr(T,filename_suffix);

	// Determine the number of samples to take
	int nSamplings = nSamp*T_end;

	/* Allocate arrays on the device */
	int totalElements = nGridXY * nGridXY * nGridZ;
	int blockSize = 256;
	int gridSize = (totalElements + blockSize - 1) / blockSize;

	hipError_t err = hipSuccess;

	err = hipMalloc((void**)&d_rng_state, sizeof(hiprandState)*totalElements);
	if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to allocate d_rng_state on the device! error = %s\n", hipGetErrorString(err)); errC--;}

	err = hipMemcpy(d_rng_state, rng_state, sizeof(hiprandState)*totalElements, hipMemcpyHostToDevice);
	if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to copy rng_state to the device! error = %s\n", hipGetErrorString(err)); errC--;}

	numtype *d_N;
	err = hipMalloc((void**)&d_N,sizeof(numtype));
	if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to allocate d_N on the device! error = %s\n", hipGetErrorString(err)); errC--;}

	initRNG<<<gridSize,blockSize>>>(d_rng_state, totalElements);



	// Loop over samplings
	for (int n = 0; n < nSamplings; n++) {
		if (exit) break;

		// Determine the number of timesteps between sampings
		int nStepsPerSample = static_cast<int>(cpu_round(1 / (nSamp *  dT)));

		for (int t = 0; t < nStepsPerSample; t++) {
			if (exit) break;

			// Increase time
			T += dT;

			// Spawn phages
			if ((T_i >= 0) and (abs(T - T_i) < dT / 2)) {
				spawnPhages();
				T_i = -1;
			}

			// Reset density counter
			numtype maxOccupancy = 0.0;

			/////////////////////////////////////////////////////
			// Main loop start //////////////////////////////////
			/////////////////////////////////////////////////////

			// Kernel 1-2: nC update and maxOccupancy //////////////////////////////////////////////////////////////////////
			for (int i = 0; i < nGridXY; i++) {
				if (exit) break;

				for (int j = 0; j < nGridXY; j++) {
					if (exit) break;

					for (int k = 0; k < nGridZ; k++) {
						if (exit) break;

						// Ensure nC is updated
						if (arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k] < arr_nC[i*nGridXY*nGridZ + j*nGridZ + k]){
								arr_nC[i*nGridXY*nGridZ + j*nGridZ + k] = arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k];
						}
					}
				}
			}

			for (int i = 0; i < nGridXY; i++) {
				if (exit) break;

				for (int j = 0; j < nGridXY; j++) {
					if (exit) break;

					for (int k = 0; k < nGridZ; k++) {
						if (exit) break;

						// Skip empty sites
						if ((arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k] < 1) and (arr_P[i*nGridXY*nGridZ + j*nGridZ + k] < 1)) continue;

						// Record the maximum observed density
						if (arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k] > maxOccupancy) maxOccupancy = arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k];

					}
				}
			}
			// Kernel 3: Birth //////////////////////////////////////////////////////////////////////
			for (int i = 0; i < nGridXY; i++) {
				if (exit) break;

				for (int j = 0; j < nGridXY; j++) {
					if (exit) break;

					for (int k = 0; k < nGridZ; k++) {
						if (exit) break;

						// Skip empty sites
						if ((arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k] < 1) and (arr_P[i*nGridXY*nGridZ + j*nGridZ + k] < 1)) continue;

						numtype p = 0; // privatize
						numtype N = 0; // privatize

						// Compute the growth modifier
						numtype growthModifier = arr_nutrient[i*nGridXY*nGridZ + j*nGridZ + k] / (arr_nutrient[i*nGridXY*nGridZ + j*nGridZ + k] + K);
						arr_GrowthModifier[i*nGridXY*nGridZ + j*nGridZ + k] = growthModifier;

						p = g * growthModifier*dT;
						if (arr_nutrient[i*nGridXY*nGridZ + j*nGridZ + k] < 1) {		//
							p = 0;
						}

						if ((p > 0.1) and (!Warn_g)) {
							cout << "\tWarning: Birth Probability Large!" << "\n";
							f_log  << "Warning: Birth Probability Large!" << "\n";
							Warn_g = true;
						}

						/* BEGIN anden Map-kernel */
						if (GPU_BIRTH) {
							numtype *tmp = new numtype;
							int index = i*nGridXY*nGridZ + j*nGridZ + k;
							ComputeEvents_seq<<<gridSize,blockSize>>>(d_N, arr_B[index], p, d_rng_state, index);
							err = hipGetLastError();
							if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failure in cuRandKernel! error = %s\n", hipGetErrorString(err)); errC--;}
							hipMemcpy(tmp, d_N, sizeof(numtype),hipMemcpyDeviceToHost);
							assert(*tmp != -1);
							N = *tmp;
							delete tmp;
						} else {
							N = ComputeEvents(arr_B[i*nGridXY*nGridZ + j*nGridZ + k], p, 2, i, j, k);
						}

						// Ensure there is enough nutrient
						if ( N > arr_nutrient[i*nGridXY*nGridZ + j*nGridZ + k] ) {
							if (!Warn_fastGrowth) {
								cout << "\tWarning: Colonies growing too fast!" << "\n";
								f_log  << "Warning: Colonies growing too fast!" << "\n";
								Warn_fastGrowth = true;
							}

							// DETERMINITIC CHANGE
							N = cpu_round( arr_nutrient[i*nGridXY*nGridZ + j*nGridZ + k] );
							// N = arr_nutrient[i*nGridXY*nGridZ + j*nGridZ + k];
						}

						// Update count
						arr_B_new[i*nGridXY*nGridZ + j*nGridZ + k] += N;
						arr_nutrient[i*nGridXY*nGridZ + j*nGridZ + k] = max(0.0, arr_nutrient[i*nGridXY*nGridZ + j*nGridZ + k] - N);
						/* END anden Map-kernel */
					}
				}
			}

			// Kernel 4: Increase Infections ////////////////////////////////////////////////////////
			for (int i = 0; i < nGridXY; i++) {
				if (exit) break;

				for (int j = 0; j < nGridXY; j++) {
					if (exit) break;

					for (int k = 0; k < nGridZ; k++) {
						if (exit) break;

						// Skip empty sites
						if ((arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k] < 1) and (arr_P[i*nGridXY*nGridZ + j*nGridZ + k] < 1)) continue;

						numtype p = 0; // privatize
						numtype N = 0; // privatize

						// Compute the growth modifier
						numtype growthModifier = arr_GrowthModifier[i*nGridXY*nGridZ + j*nGridZ + k];

						// Compute beta
						numtype Beta = beta;
						if (reducedBeta) {
							Beta *= growthModifier;
						}

				 		if (r > 0.0) {
							/* BEGIN tredje Map-kernel */

							p = r*growthModifier*dT;
							if ((p > 0.25) and (!Warn_r)) {
								cout << "\tWarning: Infection Increase Probability Large!" << "\n";
								f_log  << "Warning: Infection Increase Probability Large!" << "\n";
								Warn_r = true;
							}

							if (GPU_INFECTIONS) {
								numtype *tmp = new numtype;
								int index = i*nGridXY*nGridZ + j*nGridZ + k;
								ComputeEvents_seq<<<gridSize,blockSize>>>(d_N, arr_I9[index], p, d_rng_state, index);
								err = hipGetLastError();
								if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failure in cuRandKernel! error = %s\n", hipGetErrorString(err)); errC--;}
								hipMemcpy(tmp, d_N, sizeof(numtype),hipMemcpyDeviceToHost);
								assert(*tmp != -1);
								N = *tmp;
								delete tmp;
							} else {
								N = ComputeEvents(arr_I9[i*nGridXY*nGridZ + j*nGridZ + k], p, 2, i, j, k);
							}

							// Update count
							arr_I9[i*nGridXY*nGridZ + j*nGridZ + k]    = max(0.0, arr_I9[i*nGridXY*nGridZ + j*nGridZ + k] - N);
							arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k]   = max(0.0, arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k] - N);
                            // DETERMINITIC CHANGE
                            arr_P_new[i*nGridXY*nGridZ + j*nGridZ + k] += cpu_round( (1 - alpha) * Beta * N);   // Phages which escape the colony
                            arr_M[i*nGridXY*nGridZ + j*nGridZ + k] = cpu_round(alpha * Beta * N);                        // Phages which reinfect the colony
                            // arr_P_new[i*nGridXY*nGridZ + j*nGridZ + k] += (1 - alpha) * Beta * N;   // Phages which escape the colony
                            // arr_M[i*nGridXY*nGridZ + j*nGridZ + k] = alpha * Beta * N;

							// Non-bursting events
							if (GPU_INFECTIONS) {
								N = -1;
								numtype *tmp = new numtype;
								int index = i*nGridXY*nGridZ + j*nGridZ + k;
								ComputeEvents_seq<<<gridSize,blockSize>>>(d_N, arr_I8[index], p, d_rng_state, index);
								err = hipGetLastError();
								if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failure in cuRandKernel! error = %s\n", hipGetErrorString(err)); errC--;}
								hipMemcpy(tmp, d_N, sizeof(numtype),hipMemcpyDeviceToHost);
								assert(*tmp != -1);
								N = *tmp;
								delete tmp;
							} else {
								N = ComputeEvents(arr_I8[i*nGridXY*nGridZ + j*nGridZ + k], p, 2, i, j, k);
							}
							arr_I8[i*nGridXY*nGridZ + j*nGridZ + k] = max(0.0, arr_I8[i*nGridXY*nGridZ + j*nGridZ + k] - N);
							arr_I9[i*nGridXY*nGridZ + j*nGridZ + k] += N;

							if (GPU_INFECTIONS) {
								N = -1;
								numtype *tmp = new numtype;
								int index = i*nGridXY*nGridZ + j*nGridZ + k;
								ComputeEvents_seq<<<gridSize,blockSize>>>(d_N, arr_I7[index], p, d_rng_state, index);
								err = hipGetLastError();
								if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failure in cuRandKernel! error = %s\n", hipGetErrorString(err)); errC--;}
								hipMemcpy(tmp, d_N, sizeof(numtype),hipMemcpyDeviceToHost);
								assert(*tmp != -1);
								N = *tmp;
								delete tmp;
							} else {
								N = ComputeEvents(arr_I7[i*nGridXY*nGridZ + j*nGridZ + k], p, 2, i, j, k);
							}
							arr_I7[i*nGridXY*nGridZ + j*nGridZ + k] = max(0.0, arr_I7[i*nGridXY*nGridZ + j*nGridZ + k] - N);
							arr_I8[i*nGridXY*nGridZ + j*nGridZ + k] += N;

							if (GPU_INFECTIONS) {
								N = -1;
								numtype *tmp = new numtype;
								int index = i*nGridXY*nGridZ + j*nGridZ + k;
								ComputeEvents_seq<<<gridSize,blockSize>>>(d_N, arr_I6[index], p, d_rng_state, index);
								err = hipGetLastError();
								if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failure in cuRandKernel! error = %s\n", hipGetErrorString(err)); errC--;}
								hipMemcpy(tmp, d_N, sizeof(numtype),hipMemcpyDeviceToHost);
								assert(*tmp != -1);
								N = *tmp;
								delete tmp;
							} else {
								N = ComputeEvents(arr_I6[i*nGridXY*nGridZ + j*nGridZ + k], p, 2, i, j, k);
							}
							arr_I6[i*nGridXY*nGridZ + j*nGridZ + k] = max(0.0, arr_I6[i*nGridXY*nGridZ + j*nGridZ + k] - N);
							arr_I7[i*nGridXY*nGridZ + j*nGridZ + k] += N;

							if (GPU_INFECTIONS) {
								N = -1;
								numtype *tmp = new numtype;
								int index = i*nGridXY*nGridZ + j*nGridZ + k;
								ComputeEvents_seq<<<gridSize,blockSize>>>(d_N, arr_I5[index], p, d_rng_state, index);
								err = hipGetLastError();
								if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failure in cuRandKernel! error = %s\n", hipGetErrorString(err)); errC--;}
								hipMemcpy(tmp, d_N, sizeof(numtype),hipMemcpyDeviceToHost);
								assert(*tmp != -1);
								N = *tmp;
								delete tmp;
							} else {
								N = ComputeEvents(arr_I5[i*nGridXY*nGridZ + j*nGridZ + k], p, 2, i, j, k);
							}
							arr_I5[i*nGridXY*nGridZ + j*nGridZ + k] = max(0.0, arr_I5[i*nGridXY*nGridZ + j*nGridZ + k] - N);
							arr_I6[i*nGridXY*nGridZ + j*nGridZ + k] += N;

							if (GPU_INFECTIONS) {
								N = -1;
								numtype *tmp = new numtype;
								int index = i*nGridXY*nGridZ + j*nGridZ + k;
								ComputeEvents_seq<<<gridSize,blockSize>>>(d_N, arr_I4[index], p, d_rng_state, index);
								err = hipGetLastError();
								if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failure in cuRandKernel! error = %s\n", hipGetErrorString(err)); errC--;}
								hipMemcpy(tmp, d_N, sizeof(numtype),hipMemcpyDeviceToHost);
								assert(*tmp != -1);
								N = *tmp;
								delete tmp;
							} else {
								N = ComputeEvents(arr_I4[i*nGridXY*nGridZ + j*nGridZ + k], p, 2, i, j, k);
							}
							arr_I4[i*nGridXY*nGridZ + j*nGridZ + k] = max(0.0, arr_I4[i*nGridXY*nGridZ + j*nGridZ + k] - N);
							arr_I5[i*nGridXY*nGridZ + j*nGridZ + k] += N;

							if (GPU_INFECTIONS) {
								N = -1;
								numtype *tmp = new numtype;
								int index = i*nGridXY*nGridZ + j*nGridZ + k;
								ComputeEvents_seq<<<gridSize,blockSize>>>(d_N, arr_I3[index], p, d_rng_state, index);
								err = hipGetLastError();
								if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failure in cuRandKernel! error = %s\n", hipGetErrorString(err)); errC--;}
								hipMemcpy(tmp, d_N, sizeof(numtype),hipMemcpyDeviceToHost);
								assert(*tmp != -1);
								N = *tmp;
								delete tmp;
							} else {
								N = ComputeEvents(arr_I3[i*nGridXY*nGridZ + j*nGridZ + k], p, 2, i, j, k);
							}
							arr_I3[i*nGridXY*nGridZ + j*nGridZ + k] = max(0.0, arr_I3[i*nGridXY*nGridZ + j*nGridZ + k] - N);
							arr_I4[i*nGridXY*nGridZ + j*nGridZ + k] += N;

							if (GPU_INFECTIONS) {
								N = -1;
								numtype *tmp = new numtype;
								int index = i*nGridXY*nGridZ + j*nGridZ + k;
								ComputeEvents_seq<<<gridSize,blockSize>>>(d_N, arr_I2[index], p, d_rng_state, index);
								err = hipGetLastError();
								if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failure in cuRandKernel! error = %s\n", hipGetErrorString(err)); errC--;}
								hipMemcpy(tmp, d_N, sizeof(numtype),hipMemcpyDeviceToHost);
								assert(*tmp != -1);
								N = *tmp;
								delete tmp;
							} else {
								N = ComputeEvents(arr_I2[i*nGridXY*nGridZ + j*nGridZ + k], p, 2, i, j, k);
							}
							arr_I2[i*nGridXY*nGridZ + j*nGridZ + k] = max(0.0, arr_I2[i*nGridXY*nGridZ + j*nGridZ + k] - N);
							arr_I3[i*nGridXY*nGridZ + j*nGridZ + k] += N;

							if (GPU_INFECTIONS) {
								N = -1;
								numtype *tmp = new numtype;
								int index = i*nGridXY*nGridZ + j*nGridZ + k;
								ComputeEvents_seq<<<gridSize,blockSize>>>(d_N, arr_I1[index], p, d_rng_state, index);
								err = hipGetLastError();
								if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failure in cuRandKernel! error = %s\n", hipGetErrorString(err)); errC--;}
								hipMemcpy(tmp, d_N, sizeof(numtype),hipMemcpyDeviceToHost);
								assert(*tmp != -1);
								N = *tmp;
								delete tmp;
							} else {
								N = ComputeEvents(arr_I1[i*nGridXY*nGridZ + j*nGridZ + k], p, 2, i, j, k);
							}
							arr_I1[i*nGridXY*nGridZ + j*nGridZ + k] = max(0.0, arr_I1[i*nGridXY*nGridZ + j*nGridZ + k] - N);
							arr_I2[i*nGridXY*nGridZ + j*nGridZ + k] += N;

							if (GPU_INFECTIONS) {
								N = -1;
								numtype *tmp = new numtype;
								int index = i*nGridXY*nGridZ + j*nGridZ + k;
								ComputeEvents_seq<<<gridSize,blockSize>>>(d_N, arr_I0[index], p, d_rng_state, index);
								err = hipGetLastError();
								if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failure in cuRandKernel! error = %s\n", hipGetErrorString(err)); errC--;}
								hipMemcpy(tmp, d_N, sizeof(numtype),hipMemcpyDeviceToHost);
								assert(*tmp != -1);
								N = *tmp;
								delete tmp;
							} else {
								N = ComputeEvents(arr_I0[i*nGridXY*nGridZ + j*nGridZ + k], p, 2, i, j, k);
							}
							arr_I0[i*nGridXY*nGridZ + j*nGridZ + k] = max(0.0, arr_I0[i*nGridXY*nGridZ + j*nGridZ + k] - N);
							arr_I1[i*nGridXY*nGridZ + j*nGridZ + k] += N;

							/* END tredje Map-kernel */
						}
					}
				}
			}

			// Kernel 5: New infections ///////////////////////////////////////////////////////////////////
			for (int i = 0; i < nGridXY; i++) {
				if (exit) break;

				for (int j = 0; j < nGridXY; j++) {
					if (exit) break;

					for (int k = 0; k < nGridZ; k++) {
						if (exit) break;

						// Skip empty sites
						if ((arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k] < 1) and (arr_P[i*nGridXY*nGridZ + j*nGridZ + k] < 1)) continue;


						numtype p = 0; // privatize
						numtype N = 0; // privatize
												// numtype M = 0; // privatize

						// Compute beta
						numtype Beta = beta;
						if (reducedBeta) {
							Beta *= arr_GrowthModifier[i*nGridXY*nGridZ + j*nGridZ + k];
						}

						// PRIVATIZE BOTH OF THESE
						// numtype s;   // The factor which modifies the adsorption rate
						// numtype n;   // The number of targets the phage has
												// Infectons


												// KERNEL THIS
						// if ((arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k] >= 1) and (arr_P[i*nGridXY*nGridZ + j*nGridZ + k] >= 1)) {
						// 	if (clustering) {   // Check if clustering is enabled
						// 		s = pow(arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k] / arr_nC[i*nGridXY*nGridZ + j*nGridZ + k], 1.0 / 3.0);
						// 		n = arr_nC[i*nGridXY*nGridZ + j*nGridZ + k];
						// 	} else {            // Else use mean field computation
						// 		s = 1.0;
						// 		n = arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k];
						// 	}
						// }

						if ((arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k] >= 1) and (arr_P[i*nGridXY*nGridZ + j*nGridZ + k] >= 1)) {
							// Compute the number of hits
							if (eta * s * dT >= 1) { // In the diffusion limited case every phage hits a target
								N = arr_P[i*nGridXY*nGridZ + j*nGridZ + k];
							} else {
								p = 1 - pow(1 - eta * s * dT, n);        // Probability hitting any target

								if (GPU_NEWINFECTIONS) {
									N = -1;
									numtype *tmp = new numtype;
									int index = i*nGridXY*nGridZ + j*nGridZ + k;
									ComputeEvents_seq<<<gridSize,blockSize>>>(d_N, arr_P[index], p, d_rng_state, index);
									err = hipGetLastError();
									if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failure in cuRandKernel! error = %s\n", hipGetErrorString(err)); errC--;}
									hipMemcpy(tmp, d_N, sizeof(numtype),hipMemcpyDeviceToHost);
									assert(*tmp != -1);
									N = *tmp;
									delete tmp;
								} else {
									N = ComputeEvents(arr_P[i*nGridXY*nGridZ + j*nGridZ + k], p, 2, i, j, k);
								}
							}

							// DETERMINITIC CHANGE
							if (N + arr_M[i*nGridXY*nGridZ + j*nGridZ + k] >= 1) {
								// If bacteria were hit, update events
								arr_P[i*nGridXY*nGridZ + j*nGridZ + k] = max(0.0, arr_P[i*nGridXY*nGridZ + j*nGridZ + k] - N);     // Update count

								numtype S;
								if (shielding) {
									// Absorbing medium model
									numtype d = pow(arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k] / arr_nC[i*nGridXY*nGridZ + j*nGridZ + k], 1.0 / 3.0) -
									pow(arr_B[i*nGridXY*nGridZ + j*nGridZ + k] / arr_nC[i*nGridXY*nGridZ + j*nGridZ + k], 1.0 / 3.0);
									S = cpu_exp(-zeta * d); // Probability of hitting succebtible target

								} else {
									// Well mixed model
									S = arr_B[i*nGridXY*nGridZ + j*nGridZ + k] / arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k];
								}

								p = max(0.0, min(arr_B[i*nGridXY*nGridZ + j*nGridZ + k] / arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k],S)); // Probability of hitting succebtible target

								if (GPU_NEWINFECTIONS) {
									numtype *tmp = new numtype;
									int index = i*nGridXY*nGridZ + j*nGridZ + k;
									ComputeEvents_seq<<<gridSize,blockSize>>>(d_N, N + arr_M[index], p, d_rng_state, index);
									err = hipGetLastError();
									if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failure in cuRandKernel! error = %s\n", hipGetErrorString(err)); errC--;}
									hipMemcpy(tmp, d_N, sizeof(numtype),hipMemcpyDeviceToHost);
									assert(*tmp != -1);
									N = *tmp;
									delete tmp;
								} else {
									N = ComputeEvents(N + arr_M[i*nGridXY*nGridZ + j*nGridZ + k], p, 2, i, j, k);
								}

								if (N > arr_B[i*nGridXY*nGridZ + j*nGridZ + k])
									N = arr_B[i*nGridXY*nGridZ + j*nGridZ + k];              // If more bacteria than present are set to be infeced, round down

								// Update the counts
								arr_B[i*nGridXY*nGridZ + j*nGridZ + k] = max(0.0, arr_B[i*nGridXY*nGridZ + j*nGridZ + k] - N);
								if (r > 0.0) {
									arr_I0_new[i*nGridXY*nGridZ + j*nGridZ + k] += N;
								} else {
									arr_P_new[i*nGridXY*nGridZ + j*nGridZ + k] += N * (1 - alpha) * Beta;
								}
							}
						}
 					}
				}
			}

			// Kernel 6: Phage decay ///////////////////////////////////////////////////////////////////
			for (int i = 0; i < nGridXY; i++) {
				if (exit) break;

				for (int j = 0; j < nGridXY; j++) {
					if (exit) break;

					for (int k = 0; k < nGridZ; k++) {
						if (exit) break;

						// Skip empty sites
						if ((arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k] < 1) and (arr_P[i*nGridXY*nGridZ + j*nGridZ + k] < 1)) continue;


						numtype p = 0; // privatize
						numtype N = 0; // privatize

						// KERNEL BEGIN
						p = delta*dT;
						if ((p > 0.1) and (!Warn_delta)) {
								cout << "\tWarning: Decay Probability Large!" << "\n";
								f_log  << "Warning: Decay Probability Large!" << "\n";
								Warn_delta = true;
						}

						if (GPU_PHAGEDECAY) {
							N = -1;
							numtype *tmp = new numtype;
							int index = i*nGridXY*nGridZ + j*nGridZ + k;
							ComputeEvents_seq<<<gridSize,blockSize>>>(d_N, arr_P[index], p, d_rng_state, index);
							err = hipGetLastError();
							if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failure in cuRandKernel! error = %s\n", hipGetErrorString(err)); errC--;}
							hipMemcpy(tmp, d_N, sizeof(numtype),hipMemcpyDeviceToHost);
							assert(*tmp != -1);
							N = *tmp;
							delete tmp;
						} else {
							N = ComputeEvents(arr_P[i*nGridXY*nGridZ + j*nGridZ + k], p, 2, i, j, k);
						}



						// Update count
						arr_P[i*nGridXY*nGridZ + j*nGridZ + k]    = max(0.0, arr_P[i*nGridXY*nGridZ + j*nGridZ + k] - N);
						// KERNEL END

					}
				}
			}


			// Movement ///////////////////////////////////////////////////////////////////
			for (int i = 0; i < nGridXY; i++) {
				if (exit) break;

				for (int j = 0; j < nGridXY; j++) {
					if (exit) break;

					for (int k = 0; k < nGridZ; k++) {
						if (exit) break;

						// Skip empty sites
						if ((arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k] < 1) and (arr_P[i*nGridXY*nGridZ + j*nGridZ + k] < 1)) continue;


						if (nGridXY > 1) {
							// Update positions

							// Skip empty sites
							if ((arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k] < 1) and (arr_P[i*nGridXY*nGridZ + j*nGridZ + k] < 1)) continue;

							int ip, jp, kp, im, jm, km;

							if (i + 1 >= nGridXY) ip = i + 1 - nGridXY;
							else ip = i + 1;

							if (i == 0) im = nGridXY - 1;
							else im = i - 1;

							if (j + 1 >= nGridXY) jp = j + 1 - nGridXY;
							else jp = j + 1;

							if (j == 0) jm = nGridXY - 1;
							else jm = j - 1;

							if (not experimentalConditions) {   // Periodic boundaries in Z direction

								if (k + 1 >= nGridZ) kp = k + 1 - nGridZ;
								else kp = k + 1;

								if (k == 0) km = nGridZ - 1;
								else km = k - 1;

							} else {    // Reflective boundaries in Z direction

								if (k + 1 >= nGridZ) kp = k - 1;
								else kp = k + 1;

								if (k == 0) km = k + 1;
								else km = k - 1;

							}

							// Update counts
							numtype n_0; // No movement
							numtype n_u; // Up
							numtype n_d; // Down
							numtype n_l; // Left
							numtype n_r; // Right
							numtype n_f; // Front
							numtype n_b; // Back

							// CELLS
							ComputeDiffusion(arr_B[i*nGridXY*nGridZ + j*nGridZ + k], lambdaB, &n_0, &n_u, &n_d, &n_l, &n_r, &n_f, &n_b, 1, i, j, k);
							arr_B_new[i*nGridXY*nGridZ + j*nGridZ + k] += n_0; arr_B_new[ip*nGridXY*nGridZ + j*nGridZ + k] += n_u; arr_B_new[im*nGridXY*nGridZ + j*nGridZ + k] += n_d; arr_B_new[i*nGridXY*nGridZ + jp*nGridZ + k] += n_r; arr_B_new[i*nGridXY*nGridZ + jm*nGridZ + k] += n_l; arr_B_new[i*nGridXY*nGridZ + j*nGridZ + kp] += n_f; arr_B_new[i*nGridXY*nGridZ + j*nGridZ + km] += n_b;

							if (r > 0.0) {
								ComputeDiffusion(arr_I0[i*nGridXY*nGridZ + j*nGridZ + k],  lambdaB, &n_0, &n_u, &n_d, &n_l, &n_r, &n_f, &n_b, 2, i, j, k);
								arr_I0_new[i*nGridXY*nGridZ + j*nGridZ + k] += n_0; arr_I0_new[ip*nGridXY*nGridZ + j*nGridZ + k] += n_u; arr_I0_new[im*nGridXY*nGridZ + j*nGridZ + k] += n_d; arr_I0_new[i*nGridXY*nGridZ + jp*nGridZ + k] += n_r; arr_I0_new[i*nGridXY*nGridZ + jm*nGridZ + k] += n_l; arr_I0_new[i*nGridXY*nGridZ + j*nGridZ + kp] += n_f; arr_I0_new[i*nGridXY*nGridZ + j*nGridZ + km] += n_b;

								ComputeDiffusion(arr_I1[i*nGridXY*nGridZ + j*nGridZ + k],  lambdaB, &n_0, &n_u, &n_d, &n_l, &n_r, &n_f, &n_b, 2, i, j, k);
								arr_I1_new[i*nGridXY*nGridZ + j*nGridZ + k] += n_0; arr_I1_new[ip*nGridXY*nGridZ + j*nGridZ + k] += n_u; arr_I1_new[im*nGridXY*nGridZ + j*nGridZ + k] += n_d; arr_I1_new[i*nGridXY*nGridZ + jp*nGridZ + k] += n_r; arr_I1_new[i*nGridXY*nGridZ + jm*nGridZ + k] += n_l; arr_I1_new[i*nGridXY*nGridZ + j*nGridZ + kp] += n_f; arr_I1_new[i*nGridXY*nGridZ + j*nGridZ + km] += n_b;

								ComputeDiffusion(arr_I2[i*nGridXY*nGridZ + j*nGridZ + k],  lambdaB, &n_0, &n_u, &n_d, &n_l, &n_r, &n_f, &n_b, 2, i, j, k);
								arr_I2_new[i*nGridXY*nGridZ + j*nGridZ + k] += n_0; arr_I2_new[ip*nGridXY*nGridZ + j*nGridZ + k] += n_u; arr_I2_new[im*nGridXY*nGridZ + j*nGridZ + k] += n_d; arr_I2_new[i*nGridXY*nGridZ + jp*nGridZ + k] += n_r; arr_I2_new[i*nGridXY*nGridZ + jm*nGridZ + k] += n_l; arr_I2_new[i*nGridXY*nGridZ + j*nGridZ + kp] += n_f; arr_I2_new[i*nGridXY*nGridZ + j*nGridZ + km] += n_b;

								ComputeDiffusion(arr_I3[i*nGridXY*nGridZ + j*nGridZ + k],  lambdaB, &n_0, &n_u, &n_d, &n_l, &n_r, &n_f, &n_b, 2, i, j, k);
								arr_I3_new[i*nGridXY*nGridZ + j*nGridZ + k] += n_0; arr_I3_new[ip*nGridXY*nGridZ + j*nGridZ + k] += n_u; arr_I3_new[im*nGridXY*nGridZ + j*nGridZ + k] += n_d; arr_I3_new[i*nGridXY*nGridZ + jp*nGridZ + k] += n_r; arr_I3_new[i*nGridXY*nGridZ + jm*nGridZ + k] += n_l; arr_I3_new[i*nGridXY*nGridZ + j*nGridZ + kp] += n_f; arr_I3_new[i*nGridXY*nGridZ + j*nGridZ + km] += n_b;

								ComputeDiffusion(arr_I4[i*nGridXY*nGridZ + j*nGridZ + k],  lambdaB, &n_0, &n_u, &n_d, &n_l, &n_r, &n_f, &n_b, 2, i, j, k);
								arr_I4_new[i*nGridXY*nGridZ + j*nGridZ + k] += n_0; arr_I4_new[ip*nGridXY*nGridZ + j*nGridZ + k] += n_u; arr_I4_new[im*nGridXY*nGridZ + j*nGridZ + k] += n_d; arr_I4_new[i*nGridXY*nGridZ + jp*nGridZ + k] += n_r; arr_I4_new[i*nGridXY*nGridZ + jm*nGridZ + k] += n_l; arr_I4_new[i*nGridXY*nGridZ + j*nGridZ + kp] += n_f; arr_I4_new[i*nGridXY*nGridZ + j*nGridZ + km] += n_b;

								ComputeDiffusion(arr_I5[i*nGridXY*nGridZ + j*nGridZ + k],  lambdaB, &n_0, &n_u, &n_d, &n_l, &n_r, &n_f, &n_b, 2, i, j, k);
								arr_I5_new[i*nGridXY*nGridZ + j*nGridZ + k] += n_0; arr_I5_new[ip*nGridXY*nGridZ + j*nGridZ + k] += n_u; arr_I5_new[im*nGridXY*nGridZ + j*nGridZ + k] += n_d; arr_I5_new[i*nGridXY*nGridZ + jp*nGridZ + k] += n_r; arr_I5_new[i*nGridXY*nGridZ + jm*nGridZ + k] += n_l; arr_I5_new[i*nGridXY*nGridZ + j*nGridZ + kp] += n_f; arr_I5_new[i*nGridXY*nGridZ + j*nGridZ + km] += n_b;

								ComputeDiffusion(arr_I6[i*nGridXY*nGridZ + j*nGridZ + k],  lambdaB, &n_0, &n_u, &n_d, &n_l, &n_r, &n_f, &n_b, 2, i, j, k);
								arr_I6_new[i*nGridXY*nGridZ + j*nGridZ + k] += n_0; arr_I6_new[ip*nGridXY*nGridZ + j*nGridZ + k] += n_u; arr_I6_new[im*nGridXY*nGridZ + j*nGridZ + k] += n_d; arr_I6_new[i*nGridXY*nGridZ + jp*nGridZ + k] += n_r; arr_I6_new[i*nGridXY*nGridZ + jm*nGridZ + k] += n_l; arr_I6_new[i*nGridXY*nGridZ + j*nGridZ + kp] += n_f; arr_I6_new[i*nGridXY*nGridZ + j*nGridZ + km] += n_b;

								ComputeDiffusion(arr_I7[i*nGridXY*nGridZ + j*nGridZ + k],  lambdaB, &n_0, &n_u, &n_d, &n_l, &n_r, &n_f, &n_b, 2, i, j, k);
								arr_I7_new[i*nGridXY*nGridZ + j*nGridZ + k] += n_0; arr_I7_new[ip*nGridXY*nGridZ + j*nGridZ + k] += n_u; arr_I7_new[im*nGridXY*nGridZ + j*nGridZ + k] += n_d; arr_I7_new[i*nGridXY*nGridZ + jp*nGridZ + k] += n_r; arr_I7_new[i*nGridXY*nGridZ + jm*nGridZ + k] += n_l; arr_I7_new[i*nGridXY*nGridZ + j*nGridZ + kp] += n_f; arr_I7_new[i*nGridXY*nGridZ + j*nGridZ + km] += n_b;

								ComputeDiffusion(arr_I8[i*nGridXY*nGridZ + j*nGridZ + k],  lambdaB, &n_0, &n_u, &n_d, &n_l, &n_r, &n_f, &n_b, 2, i, j, k);
								arr_I8_new[i*nGridXY*nGridZ + j*nGridZ + k] += n_0; arr_I8_new[ip*nGridXY*nGridZ + j*nGridZ + k] += n_u; arr_I8_new[im*nGridXY*nGridZ + j*nGridZ + k] += n_d; arr_I8_new[i*nGridXY*nGridZ + jp*nGridZ + k] += n_r; arr_I8_new[i*nGridXY*nGridZ + jm*nGridZ + k] += n_l; arr_I8_new[i*nGridXY*nGridZ + j*nGridZ + kp] += n_f; arr_I8_new[i*nGridXY*nGridZ + j*nGridZ + km] += n_b;

								ComputeDiffusion(arr_I9[i*nGridXY*nGridZ + j*nGridZ + k], lambdaB, &n_0, &n_u, &n_d, &n_l, &n_r, &n_f, &n_b, 2, i, j, k);
								arr_I9_new[i*nGridXY*nGridZ + j*nGridZ + k] += n_0; arr_I9_new[ip*nGridXY*nGridZ + j*nGridZ + k] += n_u; arr_I9_new[im*nGridXY*nGridZ + j*nGridZ + k] += n_d; arr_I9_new[i*nGridXY*nGridZ + jp*nGridZ + k] += n_r; arr_I9_new[i*nGridXY*nGridZ + jm*nGridZ + k] += n_l; arr_I9_new[i*nGridXY*nGridZ + j*nGridZ + kp] += n_f; arr_I9_new[i*nGridXY*nGridZ + j*nGridZ + km] += n_b;
							}

							// PHAGES
							ComputeDiffusion(arr_P[i*nGridXY*nGridZ + j*nGridZ + k], lambdaP, &n_0, &n_u, &n_d, &n_l, &n_r, &n_f, &n_b, 3, i, j, k);
							arr_P_new[i*nGridXY*nGridZ + j*nGridZ + k] += n_0; arr_P_new[ip*nGridXY*nGridZ + j*nGridZ + k] += n_u; arr_P_new[im*nGridXY*nGridZ + j*nGridZ + k] += n_d; arr_P_new[i*nGridXY*nGridZ + jp*nGridZ + k] += n_r; arr_P_new[i*nGridXY*nGridZ + jm*nGridZ + k] += n_l; arr_P_new[i*nGridXY*nGridZ + j*nGridZ + kp] += n_f; arr_P_new[i*nGridXY*nGridZ + j*nGridZ + km] += n_b;

							// KERNEL END



						} else {
							// KERNEL BEGIN
							// CELLS
							// Skip empty sites
							if ((arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k] < 1) and (arr_P[i*nGridXY*nGridZ + j*nGridZ + k] < 1)) continue;

							arr_B_new[i*nGridXY*nGridZ + j*nGridZ + k] += arr_B[i*nGridXY*nGridZ + j*nGridZ + k];

							if (r > 0.0) {
								arr_I0_new[i*nGridXY*nGridZ + j*nGridZ + k] += arr_I0[i*nGridXY*nGridZ + j*nGridZ + k];
								arr_I1_new[i*nGridXY*nGridZ + j*nGridZ + k] += arr_I1[i*nGridXY*nGridZ + j*nGridZ + k];
								arr_I2_new[i*nGridXY*nGridZ + j*nGridZ + k] += arr_I2[i*nGridXY*nGridZ + j*nGridZ + k];
								arr_I3_new[i*nGridXY*nGridZ + j*nGridZ + k] += arr_I3[i*nGridXY*nGridZ + j*nGridZ + k];
								arr_I4_new[i*nGridXY*nGridZ + j*nGridZ + k] += arr_I4[i*nGridXY*nGridZ + j*nGridZ + k];
								arr_I5_new[i*nGridXY*nGridZ + j*nGridZ + k] += arr_I5[i*nGridXY*nGridZ + j*nGridZ + k];
								arr_I6_new[i*nGridXY*nGridZ + j*nGridZ + k] += arr_I6[i*nGridXY*nGridZ + j*nGridZ + k];
								arr_I7_new[i*nGridXY*nGridZ + j*nGridZ + k] += arr_I7[i*nGridXY*nGridZ + j*nGridZ + k];
								arr_I8_new[i*nGridXY*nGridZ + j*nGridZ + k] += arr_I8[i*nGridXY*nGridZ + j*nGridZ + k];
								arr_I9_new[i*nGridXY*nGridZ + j*nGridZ + k] += arr_I9[i*nGridXY*nGridZ + j*nGridZ + k];
							}

							// PHAGES
							arr_P_new[i*nGridXY*nGridZ + j*nGridZ + k] += arr_P[i*nGridXY*nGridZ + j*nGridZ + k];
					}
				}
			}
		}

			// Swap pointers
			std::swap(arr_B, arr_B_new);
			std::swap(arr_I0, arr_I0_new);
			std::swap(arr_I1, arr_I1_new);
			std::swap(arr_I2, arr_I2_new);
			std::swap(arr_I3, arr_I3_new);
			std::swap(arr_I4, arr_I4_new);
			std::swap(arr_I5, arr_I5_new);
			std::swap(arr_I6, arr_I6_new);
			std::swap(arr_I7, arr_I7_new);
			std::swap(arr_I8, arr_I8_new);
			std::swap(arr_I9, arr_I9_new);
			std::swap(arr_P, arr_P_new);

			// Zero the _new arrays
			for (int i = 0; i < nGridXY; i++) {
				for (int j = 0; j < nGridXY; j++ ) {
					for (int k = 0; k < nGridZ; k++ ) {
						arr_B_new[i*nGridXY*nGridZ + j*nGridZ + k]  = 0.0;
						arr_I0_new[i*nGridXY*nGridZ + j*nGridZ + k] = 0.0;
						arr_I1_new[i*nGridXY*nGridZ + j*nGridZ + k] = 0.0;
						arr_I2_new[i*nGridXY*nGridZ + j*nGridZ + k] = 0.0;
						arr_I3_new[i*nGridXY*nGridZ + j*nGridZ + k] = 0.0;
						arr_I4_new[i*nGridXY*nGridZ + j*nGridZ + k] = 0.0;
						arr_I5_new[i*nGridXY*nGridZ + j*nGridZ + k] = 0.0;
						arr_I6_new[i*nGridXY*nGridZ + j*nGridZ + k] = 0.0;
						arr_I7_new[i*nGridXY*nGridZ + j*nGridZ + k] = 0.0;
						arr_I8_new[i*nGridXY*nGridZ + j*nGridZ + k] = 0.0;
						arr_I9_new[i*nGridXY*nGridZ + j*nGridZ + k] = 0.0;
						arr_P_new[i*nGridXY*nGridZ + j*nGridZ + k]  = 0.0;
					}
				}
			}


				// Update occupancy
				for (int i = 0; i < nGridXY; i++) {
					for (int j = 0; j < nGridXY; j++ ) {
						for (int k = 0; k < nGridZ; k++ ) {
							arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k] = arr_B[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I0[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I1[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I2[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I3[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I4[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I5[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I6[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I7[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I8[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I9[i*nGridXY*nGridZ + j*nGridZ + k];
						}
					}
				}


				// NUTRIENT DIFFUSION
				numtype alphaXY = D_n * dT / pow(L / (numtype)nGridXY, 2);
				numtype alphaZ  = D_n * dT / pow(H / (numtype)nGridZ, 2);

				for (int i = 0; i < nGridXY; i++) {
					for (int j = 0; j < nGridXY; j++ ) {
						for (int k = 0; k < nGridZ; k++ ) {

							// Update positions
							int ip, jp, kp, im, jm, km;

							if (i + 1 >= nGridXY) ip = i + 1 - nGridXY;
							else ip = i + 1;

							if (i == 0) im = nGridXY - 1;
							else im = i - 1;

							if (j + 1 >= nGridXY) jp = j + 1 - nGridXY;
							else jp = j + 1;

							if (j == 0) jm = nGridXY - 1;
							else jm = j - 1;

							if (not experimentalConditions) {   // Periodic boundaries in Z direction

								if (k + 1 >= nGridZ) kp = k + 1 - nGridZ;
								else kp = k + 1;

								if (k == 0) km = nGridZ - 1;
								else km = k - 1;

							} else {    // Reflective boundaries in Z direction

								if (k + 1 >= nGridZ) kp = k - 1;
								else kp = k + 1;

								if (k == 0) km = k + 1;
								else km = k - 1;

							}

							numtype tmp = arr_nutrient[i*nGridXY*nGridZ + j*nGridZ + k];
							arr_nutrient_new[i*nGridXY*nGridZ + j*nGridZ + k]  += tmp - (4 * alphaXY + 2 * alphaZ) * tmp;
							arr_nutrient_new[ip*nGridXY*nGridZ + j*nGridZ + k] += alphaXY * tmp;
							arr_nutrient_new[im*nGridXY*nGridZ + j*nGridZ + k] += alphaXY * tmp;
							arr_nutrient_new[i*nGridXY*nGridZ + jp*nGridZ + k] += alphaXY * tmp;
							arr_nutrient_new[i*nGridXY*nGridZ + jm*nGridZ + k] += alphaXY * tmp;
							arr_nutrient_new[i*nGridXY*nGridZ + j*nGridZ + kp] += alphaZ  * tmp;
							arr_nutrient_new[i*nGridXY*nGridZ + j*nGridZ + km] += alphaZ  * tmp;
						}
					}
				}

				std::swap(arr_nutrient, arr_nutrient_new);

				// Zero the _new arrays
				for (int i = 0; i < nGridXY; i++) {
					for (int j = 0; j < nGridXY; j++ ) {
						for (int k = 0; k < nGridZ; k++ ) {
								arr_nutrient_new[i*nGridXY*nGridZ + j*nGridZ + k]  = 0.0;
						}
					}
				}

				if ((maxOccupancy > L * L * H / (nGridXY * nGridXY * nGridZ)) and (!Warn_density)) {
					cout << "\tWarning: Maximum Density Large!" << "\n";
					f_log  << "Warning: Maximum Density Large!" << "\n";
					Warn_density = true;
				}
		}

		// Fast exit conditions
		// 1) There are no more sucebtible cells
		// -> Convert all infected cells to phages and stop simulation
		numtype accuB = 0.0;
		for (int i = 0; i < nGridXY; i++) {
			for (int j = 0; j < nGridXY; j++ ) {
				for (int k = 0; k < nGridZ; k++ ) {
					accuB += arr_B[i*nGridXY*nGridZ + j*nGridZ + k];
				}
			}
		}
		if ((fastExit) and (accuB < 1)) {
			// Update the P array
			for (int i = 0; i < nGridXY; i++) {
				for (int j = 0; j < nGridXY; j++ ) {
					for (int k = 0; k < nGridZ; k++ ) {
						arr_P[i*nGridXY*nGridZ + j*nGridZ + k] += (1-alpha)*beta * (arr_I0[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I1[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I2[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I3[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I4[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I5[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I6[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I7[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I8[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I9[i*nGridXY*nGridZ + j*nGridZ + k]);
					}
				}
			}


			// Zero the I arrays
			for (int i = 0; i < nGridXY; i++) {
				for (int j = 0; j < nGridXY; j++ ) {
					for (int k = 0; k < nGridZ; k++ ) {
						arr_I0[i*nGridXY*nGridZ + j*nGridZ + k] = 0.0;
						arr_I1[i*nGridXY*nGridZ + j*nGridZ + k] = 0.0;
						arr_I2[i*nGridXY*nGridZ + j*nGridZ + k] = 0.0;
						arr_I3[i*nGridXY*nGridZ + j*nGridZ + k] = 0.0;
						arr_I4[i*nGridXY*nGridZ + j*nGridZ + k] = 0.0;
						arr_I5[i*nGridXY*nGridZ + j*nGridZ + k] = 0.0;
						arr_I6[i*nGridXY*nGridZ + j*nGridZ + k] = 0.0;
						arr_I7[i*nGridXY*nGridZ + j*nGridZ + k] = 0.0;
						arr_I8[i*nGridXY*nGridZ + j*nGridZ + k] = 0.0;
						arr_I9[i*nGridXY*nGridZ + j*nGridZ + k] = 0.0;
					}
				}
			}
			exit = true;
		}

		// 2) There are no more alive cells
		// -> Stop simulation

		numtype accuOcc = 0.0;
		for (int i = 0; i < nGridXY; i++) {
			for (int j = 0; j < nGridXY; j++ ) {
				for (int k = 0; k < nGridZ; k++ ) {
					accuOcc += arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k];
				}
			}
		}

		if ((fastExit) and (accuOcc < 1)) {
				exit = true;
		}

		// 3) The food is on average less than one per gridpoint
		// and the maximal nutrient at any point in space is less than 1

		numtype accuNutrient = 0.0;
		numtype maxNutrient  = 0.0;
		for (int i = 0; i < nGridXY; i++) {
			for (int j = 0; j < nGridXY; j++ ) {
				for (int k = 0; k < nGridZ; k++ ) {
					numtype tmpN = arr_nutrient[i*nGridXY*nGridZ + j*nGridZ + k];
					accuNutrient += tmpN;

					if (tmpN > maxNutrient) {
						maxNutrient = tmpN;
					}
				}
			}
		}

		if (fastExit) {
			if  ((accuNutrient < nGridZ*pow(nGridXY,2)) && (maxNutrient < 0.5)) {
				exit = true;
			}
		}

		// Store the state
		ExportData_arr(T,filename_suffix);

		// Check for nutrient stability
		assert(accuNutrient >= 0);
		assert(accuNutrient <= n_0 * L * L * H);
	}

	/////////////////////////////////////////////////////
	// Main loop end ////////////////////////////////////
	/////////////////////////////////////////////////////

	// Get stop time
	time_t  toc;
	time(&toc);

	// Calculate time difference
	float seconds = difftime(toc, tic);
	float hours   = floor(seconds/3600);
	float minutes = floor(seconds/60);
	minutes -= hours*60;
	seconds -= minutes*60 + hours*3600;

	cout << "\n";
	cout << "\tSimulation complete after ";
	if (hours > 0.0)   cout << hours   << " hours and ";
	if (minutes > 0.0) cout << minutes << " minutes and ";
	cout  << seconds << " seconds." << "\n";

	std::ofstream f_out;
	f_out.open(GetPath() + "/Completed_LOOP_DISTRIBUTED.txt",fstream::trunc);
	f_out << "\tSimulation complete after ";
	if (hours > 0.0)   f_out << hours   << " hours and ";
	if (minutes > 0.0) f_out << minutes << " minutes and ";
	f_out  << seconds << " seconds." << "\n";
	f_out.flush();
	f_out.close();

	// Write sucess to log
	if (exit) {
			f_log << ">>Simulation completed with exit flag<<" << "\n";
	} else {
			f_log << ">>Simulation completed without exit flag<<" << "\n";
	}

	std::ofstream f_timing;
	f_timing << "\t"       << setw(3) << difftime(toc, tic) << " s of total time" << "\n";

	f_timing.flush();
	f_timing.close();

	if (exit) {
		return 1;
	} else {
		return 0;
	}
}


// GPU copy helper functions
void Colonies3D::CopyToHost(numtype* hostArray, numtype* deviceArray, int failCode, int gridsz){
	hipError_t err = hipSuccess;
	err = hipMemcpy(hostArray, deviceArray, sizeof(numtype)*gridsz, hipMemcpyDeviceToHost);
		if (err != hipSuccess && errC > 0)	{
			fprintf(stderr, "Failed to copy to the host! Code %d error = %s\n", failCode, hipGetErrorString(err));
			errC--;
		}
}
/*
void Colonies3D::CopyToHost(bool *hostElement, bool *deviceElement, int failCode){
	hipError_t err = hipSuccess;
	err = hipMemcpy(hostElement, deviceElement, sizeof(bool), hipMemcpyDeviceToHost);
		if (err != hipSuccess && errC > 0)	{
			fprintf(stderr, "Failed to copy to the host! Code %d error = %s\n", failCode, hipGetErrorString(err));
			errC--;
		}
}
*/
///////
void Colonies3D::CopyAllToHost(){

	CopyToHost(arr_nC, 				d_arr_nC, 				1,  nGridXY*nGridXY*nGridZ);
	CopyToHost(arr_Occ, 			d_arr_Occ, 				2,  nGridXY*nGridXY*nGridZ);
	CopyToHost(arr_B, 				d_arr_B, 				3,  nGridXY*nGridXY*nGridZ);
	CopyToHost(arr_B_new, 			d_arr_B_new, 			4,  nGridXY*nGridXY*nGridZ);
	CopyToHost(arr_P, 				d_arr_P, 				5,  nGridXY*nGridXY*nGridZ);
	CopyToHost(arr_P_new, 			d_arr_P_new, 			6,  nGridXY*nGridXY*nGridZ);
	CopyToHost(arr_I0, 				d_arr_I0,				7,  nGridXY*nGridXY*nGridZ);
	CopyToHost(arr_I0_new, 			d_arr_I0_new,			8,  nGridXY*nGridXY*nGridZ);
	CopyToHost(arr_I1, 				d_arr_I1, 				9,  nGridXY*nGridXY*nGridZ);
	CopyToHost(arr_I1_new, 			d_arr_I1_new,			10, nGridXY*nGridXY*nGridZ);
	CopyToHost(arr_I2, 				d_arr_I2, 				11, nGridXY*nGridXY*nGridZ);
	CopyToHost(arr_I2_new, 			d_arr_I2_new,			12, nGridXY*nGridXY*nGridZ);
	CopyToHost(arr_I3, 				d_arr_I3, 				13, nGridXY*nGridXY*nGridZ);
	CopyToHost(arr_I3_new, 			d_arr_I3_new,			14, nGridXY*nGridXY*nGridZ);
	CopyToHost(arr_I4, 				d_arr_I4, 				15, nGridXY*nGridXY*nGridZ);
	CopyToHost(arr_I4_new, 			d_arr_I4_new,			16, nGridXY*nGridXY*nGridZ);
	CopyToHost(arr_I5, 				d_arr_I5, 				17, nGridXY*nGridXY*nGridZ);
	CopyToHost(arr_I5_new, 			d_arr_I5_new,			18, nGridXY*nGridXY*nGridZ);
	CopyToHost(arr_I6, 				d_arr_I6, 				19, nGridXY*nGridXY*nGridZ);
	CopyToHost(arr_I6_new, 			d_arr_I6_new,			20, nGridXY*nGridXY*nGridZ);
	CopyToHost(arr_I7, 				d_arr_I7, 				21, nGridXY*nGridXY*nGridZ);
	CopyToHost(arr_I7_new, 			d_arr_I7_new,			22, nGridXY*nGridXY*nGridZ);
	CopyToHost(arr_I8, 				d_arr_I8, 				23, nGridXY*nGridXY*nGridZ);
	CopyToHost(arr_I8_new, 			d_arr_I8_new,			24, nGridXY*nGridXY*nGridZ);
	CopyToHost(arr_I9, 				d_arr_I9, 				25, nGridXY*nGridXY*nGridZ);
	CopyToHost(arr_I9_new, 			d_arr_I9_new,			26, nGridXY*nGridXY*nGridZ);

	CopyToHost(arr_M, 				d_arr_M, 				27, nGridXY*nGridXY*nGridZ);
	CopyToHost(arr_p, 				d_arr_p, 				28, nGridXY*nGridXY*nGridZ);
	CopyToHost(arr_nutrient, 		d_arr_nutrient, 		29, nGridXY*nGridXY*nGridZ);
	CopyToHost(arr_nutrient_new,  	d_arr_nutrient_new,		30, nGridXY*nGridXY*nGridZ);
	CopyToHost(arr_GrowthModifier, 	d_arr_GrowthModifier, 	31, nGridXY*nGridXY*nGridZ);

//	CopyToHost(&this->Warn_r, 		d_Warn_r, 				100);
//	CopyToHost(&this->Warn_delta, 	d_Warn_delta,			101);
	hipError_t err = hipSuccess;
	err = hipMemcpy(&this->Warn_r, d_Warn_r, sizeof(bool), hipMemcpyDeviceToHost);
		if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to copy Warn_r to the host! error = %s\n", hipGetErrorString(err)); errC--;}

	err = hipMemcpy(&this->Warn_delta, d_Warn_delta, sizeof(bool), hipMemcpyDeviceToHost);
		if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to copy Warn_delta to the host! error = %s\n", hipGetErrorString(err)); errC--;}

	err = hipMemcpy(&this->Warn_g, d_Warn_g, sizeof(bool), hipMemcpyDeviceToHost);
		if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to copy Warn_g to the host! error = %s\n", hipGetErrorString(err)); errC--;}

	err = hipMemcpy(&this->Warn_fastGrowth, d_Warn_fastGrowth, sizeof(bool), hipMemcpyDeviceToHost);
		if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to copy Warn_fastGrowth to the host! error = %s\n", hipGetErrorString(err)); errC--;}

}



////
void Colonies3D::CopyToDevice(numtype* hostArray, numtype* deviceArray, int failCode, int gridsz){
	hipError_t err = hipSuccess;
	err = hipMemcpy(deviceArray, hostArray, sizeof(numtype)*gridsz, hipMemcpyHostToDevice);
		if (err != hipSuccess && errC > 0) {
			fprintf(stderr, "Failed to copy to the device! Code %d error = %s\n", failCode, hipGetErrorString(err));
			errC--;
		}
}
/*
void Colonies3D::CopyToDevice(bool hostElement, bool deviceElement, int failCode){
	hipError_t err = hipSuccess;
	err = hipMemcpy(deviceElement, hostElement, sizeof(bool), hipMemcpyHostToDevice);
		if (err != hipSuccess && errC > 0) {
			fprintf(stderr, "Failed to copy to the device! Code %d error = %s\n", failCode, hipGetErrorString(err));
			errC--;
		}
}
*/
////
void Colonies3D::CopyAllToDevice(){

	CopyToDevice(arr_nC, 				d_arr_nC, 				1,  nGridXY*nGridXY*nGridZ);
	CopyToDevice(arr_Occ, 				d_arr_Occ, 				2,  nGridXY*nGridXY*nGridZ);
	CopyToDevice(arr_B, 				d_arr_B, 				3,  nGridXY*nGridXY*nGridZ);
	CopyToDevice(arr_B_new, 			d_arr_B_new, 			4,  nGridXY*nGridXY*nGridZ);
	CopyToDevice(arr_P, 				d_arr_P, 				5,  nGridXY*nGridXY*nGridZ);
	CopyToDevice(arr_P_new, 			d_arr_P_new, 			6,  nGridXY*nGridXY*nGridZ);
	CopyToDevice(arr_I0, 				d_arr_I0,				7,  nGridXY*nGridXY*nGridZ);
	CopyToDevice(arr_I0_new, 			d_arr_I0_new,			8,  nGridXY*nGridXY*nGridZ);
	CopyToDevice(arr_I1, 				d_arr_I1, 				9,  nGridXY*nGridXY*nGridZ);
	CopyToDevice(arr_I1_new, 			d_arr_I1_new,			10, nGridXY*nGridXY*nGridZ);
	CopyToDevice(arr_I2, 				d_arr_I2, 				11, nGridXY*nGridXY*nGridZ);
	CopyToDevice(arr_I2_new, 			d_arr_I2_new,			12, nGridXY*nGridXY*nGridZ);
	CopyToDevice(arr_I3, 				d_arr_I3, 				13, nGridXY*nGridXY*nGridZ);
	CopyToDevice(arr_I3_new, 			d_arr_I3_new,			14, nGridXY*nGridXY*nGridZ);
	CopyToDevice(arr_I4, 				d_arr_I4, 				15, nGridXY*nGridXY*nGridZ);
	CopyToDevice(arr_I4_new, 			d_arr_I4_new,			16, nGridXY*nGridXY*nGridZ);
	CopyToDevice(arr_I5, 				d_arr_I5, 				17, nGridXY*nGridXY*nGridZ);
	CopyToDevice(arr_I5_new, 			d_arr_I5_new,			18, nGridXY*nGridXY*nGridZ);
	CopyToDevice(arr_I6, 				d_arr_I6, 				19, nGridXY*nGridXY*nGridZ);
	CopyToDevice(arr_I6_new, 			d_arr_I6_new,			20, nGridXY*nGridXY*nGridZ);
	CopyToDevice(arr_I7, 				d_arr_I7, 				21, nGridXY*nGridXY*nGridZ);
	CopyToDevice(arr_I7_new, 			d_arr_I7_new,			22, nGridXY*nGridXY*nGridZ);
	CopyToDevice(arr_I8, 				d_arr_I8, 				23, nGridXY*nGridXY*nGridZ);
	CopyToDevice(arr_I8_new, 			d_arr_I8_new,			24, nGridXY*nGridXY*nGridZ);
	CopyToDevice(arr_I9, 				d_arr_I9, 				25, nGridXY*nGridXY*nGridZ);
	CopyToDevice(arr_I9_new, 			d_arr_I9_new,			26, nGridXY*nGridXY*nGridZ);

	CopyToDevice(arr_M, 				d_arr_M, 				27, nGridXY*nGridXY*nGridZ);
	CopyToDevice(arr_p, 				d_arr_p, 				28, nGridXY*nGridXY*nGridZ);
	CopyToDevice(arr_nutrient, 			d_arr_nutrient, 		29, nGridXY*nGridXY*nGridZ);
	CopyToDevice(arr_nutrient_new,  	d_arr_nutrient_new,		30, nGridXY*nGridXY*nGridZ);
	CopyToDevice(arr_GrowthModifier, 	d_arr_GrowthModifier, 	31, nGridXY*nGridXY*nGridZ);

//	CopyToDevice(&this->Warn_r, 		d_Warn_r, 				100);
//	CopyToDevice(&this->Warn_delta, 	d_Warn_delta,			101);
	hipError_t err = hipSuccess;
	err = hipMemcpy(d_Warn_r, &this->Warn_r, sizeof(bool), hipMemcpyHostToDevice);
		if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to copy Warn_r to the device! error = %s\n", hipGetErrorString(err)); errC--;}

	err = hipMemcpy(d_Warn_delta, &this->Warn_delta, sizeof(bool), hipMemcpyHostToDevice);
		if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to copy Warn_delta to the device! error = %s\n", hipGetErrorString(err)); errC--;}

	err = hipMemcpy(d_Warn_g, &this->Warn_g, sizeof(bool), hipMemcpyHostToDevice);
		if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to copy Warn_g to the device! error = %s\n", hipGetErrorString(err)); errC--;}

	err = hipMemcpy(d_Warn_fastGrowth, &this->Warn_fastGrowth, sizeof(bool), hipMemcpyHostToDevice);
		if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to copy Warn_fastGrowth to the device! error = %s\n", hipGetErrorString(err)); errC--;}

	err = hipMemcpy(d_Warn_density, &this->Warn_density, sizeof(bool), hipMemcpyHostToDevice);
		if (err != hipSuccess && errC > 0)	{fprintf(stderr, "Failed to copy Warn_density to the device! error = %s\n", hipGetErrorString(err)); errC--;}


};

// Initialize the simulation
void Colonies3D::Initialize() {

		// Set the random number generator seed
		if (rngSeed >= 0.0) {
			rng.seed( rngSeed );
		} else {
			static std::random_device rd;
			rng.seed(rd());
		}

		// Compute nGridZ
		if (L != H) {
			nGridZ = cpu_round(H / L * nGridXY);
			H = nGridZ * L / nGridXY;
		} else {
			nGridZ = nGridXY;
		}

		// Allocate the arrays
		// Compute the step size
		numtype dXY = L / nGridXY;
		numtype dZ  = H / nGridZ;
		numtype dV  = dXY * dXY * dZ;

		// Allocate arrays
		arr_B   = new numtype[nGridXY*nGridXY*nGridZ]();
		arr_I0  = new numtype[nGridXY*nGridXY*nGridZ]();
		arr_I1  = new numtype[nGridXY*nGridXY*nGridZ]();
		arr_I2  = new numtype[nGridXY*nGridXY*nGridZ]();
		arr_I3  = new numtype[nGridXY*nGridXY*nGridZ]();
		arr_I4  = new numtype[nGridXY*nGridXY*nGridZ]();
		arr_I5  = new numtype[nGridXY*nGridXY*nGridZ]();
		arr_I6  = new numtype[nGridXY*nGridXY*nGridZ]();
		arr_I7  = new numtype[nGridXY*nGridXY*nGridZ]();
		arr_I8  = new numtype[nGridXY*nGridXY*nGridZ]();
		arr_I9  = new numtype[nGridXY*nGridXY*nGridZ]();
		arr_P   = new numtype[nGridXY*nGridXY*nGridZ]();
		arr_nC  = new numtype[nGridXY*nGridXY*nGridZ]();

		arr_B_new   = new numtype[nGridXY*nGridXY*nGridZ]();
		arr_I0_new  = new numtype[nGridXY*nGridXY*nGridZ]();
		arr_I1_new  = new numtype[nGridXY*nGridXY*nGridZ]();
		arr_I2_new  = new numtype[nGridXY*nGridXY*nGridZ]();
		arr_I3_new  = new numtype[nGridXY*nGridXY*nGridZ]();
		arr_I4_new  = new numtype[nGridXY*nGridXY*nGridZ]();
		arr_I5_new  = new numtype[nGridXY*nGridXY*nGridZ]();
		arr_I6_new  = new numtype[nGridXY*nGridXY*nGridZ]();
		arr_I7_new  = new numtype[nGridXY*nGridXY*nGridZ]();
		arr_I8_new  = new numtype[nGridXY*nGridXY*nGridZ]();
		arr_I9_new  = new numtype[nGridXY*nGridXY*nGridZ]();
		arr_P_new   = new numtype[nGridXY*nGridXY*nGridZ]();

		arr_nutrient = new numtype[nGridXY*nGridXY*nGridZ]();
		arr_Occ      = new numtype[nGridXY*nGridXY*nGridZ]();
		arr_nutrient_new = new numtype[nGridXY*nGridXY*nGridZ]();

		arr_rng = new std::mt19937[nGridXY*nGridXY*nGridZ];

		rng_state = new hiprandState[nGridXY*nGridXY*nGridZ];

		arr_M = new numtype[nGridXY*nGridXY*nGridZ]();
		arr_GrowthModifier = new numtype[nGridXY*nGridXY*nGridZ]();
		arr_p = new numtype[nGridXY*nGridXY*nGridZ]();

		skipArray = new bool[nGridXY*nGridXY*nGridZ];

		// Initialize arrays
		for (int i = 0; i < nGridXY; i++) {
			for (int j = 0; j < nGridXY; j++) {
				for (int k = 0; k < nGridZ; k++) {
					arr_nutrient[i*nGridXY*nGridZ + j*nGridZ + k] = n_0 / 1e12 * dV;
					arr_rng[i*nGridXY*nGridZ + j*nGridZ + k].seed(i*nGridXY*nGridZ + j*nGridZ + k);
				}
			}
		}

		// Compute the size of the time step
		ComputeTimeStep();

		// Store the parameters
		WriteLog();

		// Convert parameters to match gridpoint ///////
		// Adjust eta to match volume
		// eta/V is the number of collisions per hour for a single target
		eta = eta / dV;    // Number of collisions per gridpoint per hour

		// Adjust carrying capacity
		K = K * n_0 / 1e12 * dV;   // Deterine the Monod growth factor used in n(i,j)/(n(i,j)+K)

		// Initialize the bacteria and phage populations
		spawnBacteria();

		if (T_i <= dT) {
				spawnPhages();
				T_i = -1;
		}
}


// Spawns the bacteria
void Colonies3D::spawnBacteria() {

		// Determine the number of cells to spawn
		numtype nBacteria = cpu_round(L * L * H * B_0 / 1e12);

		// Average bacteria per gridpoint
		numtype avgBacteria = nBacteria / (nGridXY * nGridXY * nGridZ);

		// Keep track of the number of cells spawned
		numtype numB = 0;

		// Initialize cell and phage populations
		if (nBacteria > (nGridXY * nGridXY * nGridZ)) {
				for (int k = 0; k < nGridZ; k++) {
						for (int j = 0; j < nGridXY; j++) {
								for (int i = 0; i < nGridXY; i++) {

										// Compute the number of bacteria to land in this gridpoint
										numtype BB = RandP(avgBacteria);
										if (BB < 1) continue;

										// Store the number of clusters in this gridpoint
										arr_nC[i*nGridXY*nGridZ + j*nGridZ + k] = BB;

										// Add the bacteria
										arr_B[i*nGridXY*nGridZ + j*nGridZ + k] = BB;
										numB += BB;
								}
						}
				}
		}

		// Correct for underspawning
		while (numB < nBacteria) {

				// Choose random point in space
				int i = RandI(nGridXY - 1);
				int j = RandI(nGridXY - 1);
				int k = RandI(nGridZ  - 1);

				if (reducedBoundary) {
						i = 0;
						j = 0;
				}

				// Add the bacteria
				arr_B[i*nGridXY*nGridZ + j*nGridZ + k]++;
				arr_nC[i*nGridXY*nGridZ + j*nGridZ + k]++;

				numB++;


		}

		// Correct for overspawning
		while (numB > nBacteria) {
				int i = RandI(nGridXY - 1);
				int j = RandI(nGridXY - 1);
				int k = RandI(nGridZ  - 1);

				if (arr_B[i*nGridXY*nGridZ + j*nGridZ + k] < 1) continue;

				arr_B[i*nGridXY*nGridZ + j*nGridZ + k]--;
				arr_nC[i*nGridXY*nGridZ + j*nGridZ + k]--;

				numB--;
		}

		// Count the initial occupancy
		for (int k = 0; k < nGridZ; k++ ) {
				for (int j = 0; j < nGridXY; j++ ) {
						for (int i = 0; i < nGridXY; i++) {
								if (arr_B[i*nGridXY*nGridZ + j*nGridZ + k] > 0.0) {
										initialOccupancy++;
								}
						}
				}
		}

		// Determine the occupancy
		for (int k = 0; k < nGridZ; k++ ) {
				for (int j = 0; j < nGridXY; j++ ) {
						for (int i = 0; i < nGridXY; i++) {
								arr_Occ[i*nGridXY*nGridZ + j*nGridZ + k] = arr_B[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I0[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I1[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I2[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I3[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I4[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I5[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I6[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I7[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I8[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I9[i*nGridXY*nGridZ + j*nGridZ + k];
						}
				}
		}
}


// Spawns the phages
void Colonies3D::spawnPhages() {

		 // Determine the number of phages to spawn
		numtype nPhages = (numtype) cpu_round(L * L * H * P_0 / 1e12);

		// Apply generic spawning
		if (not experimentalConditions) {

				numtype numP = 0;
				if (nPhages <= nGridXY * nGridXY * nGridZ ) {
						for (numtype n = 0; n < nPhages; n++) {
								int i = RandI(nGridXY - 1);
								int j = RandI(nGridXY - 1);
								int k = RandI(nGridZ  - 1);
								arr_P[i*nGridXY*nGridZ + j*nGridZ + k]++;
								numP++;
						}
				} else {
						for (int k = 0; k < nGridZ; k++ ) {
								for (int j = 0; j < nGridXY; j++ ) {
										for (int i = 0; i < nGridXY; i++) {
												numtype PP = RandP(nPhages / (numtype)(nGridXY * nGridXY * nGridZ));

												if (PP < 1) continue;
												arr_P[i*nGridXY*nGridZ + j*nGridZ + k] = PP;
												numP += PP;
										}
								}
						}
						// Correct for overspawning
						while (numP > nPhages) {
								int i = RandI(nGridXY - 1);
								int j = RandI(nGridXY - 1);
								int k = RandI(nGridZ - 1);

								if (arr_P[i*nGridXY*nGridZ + j*nGridZ + k] > 0) {
										arr_P[i*nGridXY*nGridZ + j*nGridZ + k]--;
										numP--;
								}
						}
						// Correct for underspawning
						while (numP < nPhages) {
								int i = RandI(nGridXY - 1);
								int j = RandI(nGridXY - 1);
								int k = RandI(nGridZ - 1);

								arr_P[i*nGridXY*nGridZ + j*nGridZ + k]++;
								numP++;
						}
				}

		} else { // Apply scenario specific settings

				// Determine the number of phages to spawn
				numtype nPhages = (numtype) cpu_round(L * L * H * P_0 / 1e12);
				numtype numP = 0;
				if (nPhages <= nGridXY * nGridXY) {
						for (numtype n = 0; n < nPhages; n++) {
								arr_P[RandI(nGridXY - 1)*nGridXY*nGridZ + RandI(nGridXY - 1)*nGridZ + nGridZ - 1]++;
								numP++;
						}
				} else {
						for (int j = 0; j < nGridXY; j++ ) {
								for (int i = 0; i < nGridXY; i++ ) {
										arr_P[i*nGridXY*nGridZ + j*nGridZ + nGridZ - 1] = RandP(nPhages / (numtype)(nGridXY * nGridXY * nGridZ));
										numP += arr_P[i*nGridXY*nGridZ + j*nGridZ + nGridZ - 1];
								}
						}
						// Correct for overspawning
						while (numP > nPhages) {
								int i = RandI(nGridXY - 1);
								int j = RandI(nGridXY - 1);

								if (arr_P[i*nGridXY*nGridZ + j*nGridZ + nGridZ - 1] > 0) {
										arr_P[i*nGridXY*nGridZ + j*nGridZ + nGridZ - 1]--;
										numP--;
								}
						}
						// Correct for underspawning
						while (numP < nPhages) {
								int i = RandI(nGridXY - 1);
								int j = RandI(nGridXY - 1);

								arr_P[i*nGridXY*nGridZ + j*nGridZ + nGridZ - 1]++;
								numP++;
						}
				}
		}
}

// Computes the size of the time-step needed
void Colonies3D::ComputeTimeStep() {

	if (this->dT > 0) return;

	// Compute the step size
	numtype dXY = L / (numtype)nGridXY;
	numtype dZ  = H / (numtype)nGridZ;
	assert(dXY == dZ);
	numtype dx  = dXY;

	// Compute the time-step size
	int limiter = 0;

	numtype dT = min(pow(10,-2), 1 / nSamp);
	numtype dt;

	// Compute time-step limit set by D_P (LambdaP < 0.1)
	if (D_P > 0) {
			dt = pow(dx, 2) * 0.1 / (2 * D_P);
			if (dt < dT) {
					dT = dt;
					limiter = 1;
			}
	}

	// Compute time-step limit set by D_B (LambdaP < 0.1)
	if (D_B > 0) {
			dt = pow(dx, 2) * 0.1 / (2 * D_B);
			if (dt < dT) {
					dT = dt;
					limiter = 2;
			}
	}

	// Compute time-step limit set by D_n (D_n *dT/pow(dx,2) < 1/8)
	dt = pow(dx, 2) / (8 * D_n);
	if (dt < dT) {

			dT = dt;
			limiter = 3;
	}

	// Compute time-step limit set by r (r*dT < 0.25)
	if (r > 0.0) {
			dt = 0.25 / r;
			if (dt < dT) {
					dT = dt;
					limiter = 4;
			}
	}

	// Compute time-step limit set by g (g*dT < 0.1)
	dt = 0.1 / g;
	if (dt < dT) {
			dT = dt;
			limiter = 5;
	}


	// Compute time-step limit set by delta (delta*dT < 0.1)
	dt = 0.1 / delta;
	if (dt < dT) {

			dT = dt;
			limiter = 6;
	}

	// Get the order of magnitude of the timestep
	numtype m = floor(log10(dT));

	// Round remainder to 1, 2 or 5
	numtype r = cpu_round(dT * pow(10, -m));
	if (r >= 5)      dT = 5*pow(10, m);
	else if (r >= 2) dT = 2*pow(10, m);
	else             dT =   pow(10, m);

	if (this->dT != dT) {
		this->dT = dT;

		switch(limiter){
			case 1:
				cout << "\tdT is Limited by D_P" << "\n";
				break;
			case 2:
				cout << "\tdT is Limited by D_B" << "\n";
				break;
			case 3:
				cout << "\tdT is Limited by D_n" << "\n";
				break;
			case 4:
				cout << "\tdT is Limited by r" << "\n";
				break;
			case 5:
				cout << "\tdT is Limited by g" << "\n";
				break;
			case 6:
				cout << "\tdT is Limited by delta" << "\n";
				break;
		}
	}

	// Compute the jumping probabilities
	lambdaB = 2 * D_B * dT / pow(dx, 2);
	if (lambdaB > 0.1) {
		cout << "lambdaB = " << lambdaB << "\n";
		assert(lambdaB <= 0.1);
	}

	lambdaP = 2 * D_P * dT / pow(dx, 2);
	if (lambdaP > 0.1) {
		cout << "lambdaP = " << lambdaP << "\n";
		assert(lambdaP <= 0.1);
	}

}

// Returns the number of events ocurring for given n and p
numtype Colonies3D::ComputeEvents(numtype n, numtype p, int flag, int i, int j, int k) {

	// Trivial cases
    // if (p >= 1) return n;
    // if (p == 0) return 0.0;

    // DETERMINITIC CHANGE
    // if (n < 1)  return 0.0;
    // double N = RandP(n*p, i, j, k);
    // return round(N);

    return n*min(1.0,p);
}

// Returns the number of events ocurring for given n and p, flat array
numtype Colonies3D::ComputeEvents(numtype n, numtype p, int flag, int i) {

		// Trivial cases
		if (p == 1) return n;
		if (p == 0) return 0.0;
		if (n < 1)  return 0.0;

		numtype N = RandP(n*p, i);

		return cpu_round(N);
}

// Computes how many particles has moved to neighbouing points
void Colonies3D::ComputeDiffusion(numtype n, numtype lambda, numtype* n_0, numtype* n_u, numtype* n_d, numtype* n_l, numtype* n_r, numtype* n_f, numtype* n_b, int flag, int i, int j, int k) {

		// Reset positions
		*n_0 = 0.0;
		*n_u = 0.0;
		*n_d = 0.0;
		*n_l = 0.0;
		*n_r = 0.0;
		*n_f = 0.0;
		*n_b = 0.0;

		// DETERMINITIC CHANGE
    	// if (n < 1) return;

		// Check if diffusion should occur
		if ((lambda == 0) or (nGridXY == 1)) {
			*n_0 = n;
			return;
		}

		// if (lambda*n < 5) {   // Compute all movement individually

		// 	for (int l = 0; l < cpu_round(n); l++) {

		// 		double r = Rand(arr_rng[i*nGridXY*nGridZ + j*nGridZ + k]);

		// 		if       (r <    lambda)                     (*n_u)++;  // Up movement
		// 		else if ((r >=   lambda) and (r < 2*lambda)) (*n_d)++;  // Down movement
		// 		else if ((r >= 2*lambda) and (r < 3*lambda)) (*n_l)++;  // Left movement
		// 		else if ((r >= 3*lambda) and (r < 4*lambda)) (*n_r)++;  // Right movement
		// 		else if ((r >= 4*lambda) and (r < 5*lambda)) (*n_f)++;  // Forward movement
		// 		else if ((r >= 5*lambda) and (r < 6*lambda)) (*n_b)++;  // Backward movement
		// 		else                                         (*n_0)++;  // No movement

		// 	}


		// } else {

		// 	// Compute the number of agents which move
		// 	double N = RandP(3*lambda*n, i, j, k); // Factor of 3 comes from 3D

		// 	*n_u = RandP(N/6, i, j, k);
		// 	*n_d = RandP(N/6, i, j, k);
		// 	*n_l = RandP(N/6, i, j, k);
		// 	*n_r = RandP(N/6, i, j, k);
		// 	*n_f = RandP(N/6, i, j, k);
		// 	*n_b = RandP(N/6, i, j, k);
		// 	*n_0 = n - (*n_u + *n_d + *n_l + *n_r + *n_f + *n_b);
		// }

		// *n_u = round(*n_u);
		// *n_d = round(*n_d);
		// *n_l = round(*n_l);
		// *n_r = round(*n_r);
		// *n_f = round(*n_f);
		// *n_b = round(*n_b);
		// *n_0 = n - (*n_u + *n_d + *n_l + *n_r + *n_f + *n_b);

		*n_u = 0.5*lambda*n;
		*n_d = 0.5*lambda*n;
		*n_l = 0.5*lambda*n;
		*n_r = 0.5*lambda*n;
		*n_f = 0.5*lambda*n;
		*n_b = 0.5*lambda*n;
		*n_0 = n - (*n_u + *n_d + *n_l + *n_r + *n_f + *n_b);

		// assert(*n_0 >= 0);
		// assert(*n_u >= 0);
		// assert(*n_d >= 0);
		// assert(*n_l >= 0);
		// assert(*n_r >= 0);
		// assert(*n_f >= 0);
		// assert(*n_b >= 0);
		// assert(fabs(n - (*n_0 + *n_u + *n_d + *n_l + *n_r + *n_f + *n_b)) < 1);

}


// Settings /////////////////////////////////////////////////////////////////////////////
void Colonies3D::SetLength(numtype L){this->L=L;}                                 // Set the side-length of the simulation
void Colonies3D::SetHeight(numtype H) {this->H=H;}                                // Set the height of the simulation}
void Colonies3D::SetGridSize(numtype nGrid){this->nGridXY=nGrid;}                 // Set the number of gridpoints
void Colonies3D::SetTimeStep(numtype dT){this->dT=dT;}                            // Set the time step size
void Colonies3D::SetSamples(int nSamp){this->nSamp=nSamp;}                       // Set the number of output samples

void Colonies3D::PhageInvasionStartTime(numtype T_i){this->T_i=T_i;}              // Sets the time when the phages should start infecting

void Colonies3D::CellGrowthRate(numtype g){this->g=g;}                            // Sets the maximum growthrate
void Colonies3D::CellCarryingCapacity(numtype K){this->K=K;}                      // Sets the carrying capacity
void Colonies3D::CellDiffusionConstant(numtype D_B){this->D_B=D_B;}               // Sets the diffusion constant of the phages

void Colonies3D::PhageBurstSize(int beta){this->beta=beta;}                      // Sets the size of the bursts
void Colonies3D::PhageAdsorptionRate(numtype eta){this->eta=eta;}                 // sets the adsorption parameter eta
void Colonies3D::PhageDecayRate(numtype delta){this->delta=delta;}                // Sets the decay rate of the phages
void Colonies3D::PhageInfectionRate(numtype r){this->r=r;}                        // Sets rate of the infection increaasing in stage
void Colonies3D::PhageDiffusionConstant(numtype D_P){this->D_P=D_P;}              // Sets the diffusion constant of the phages

// Sets latency time of the phage (r and tau are related by r = 10 / tau)
void Colonies3D::PhageLatencyTime(numtype tau) {
		if (tau > 0.0) r = 10 / tau;
		else r = 0.0;
}

void Colonies3D::SurfacePermeability(numtype zeta){this->zeta=zeta;}             // Sets the permeability of the surface

void Colonies3D::InitialNutrient(numtype n_0){this->n_0=n_0;}                    // Sets the amount of initial nutrient
void Colonies3D::NutrientDiffusionConstant(numtype D_n){this->D_n=D_n;}          // Sets the nutrient diffusion rate

void Colonies3D::SimulateExperimentalConditions(){experimentalConditions=true;} // Sets the simulation to spawn phages at top layer and only have x-y periodic boundaries

void Colonies3D::DisableShielding(){shielding=false;}                           // Sets shielding bool to false
void Colonies3D::DisablesClustering(){clustering=false;}                        // Sets clustering bool to false
void Colonies3D::ReducedBurstSize(){reducedBeta=true;}                          // Sets the simulation to limit beta as n -> 0

// Sets the reduced boundary bool to true and the value of s
void Colonies3D::ReducedBoundary(int s) {
		this->s = s;
		reducedBoundary = true;
}

void Colonies3D::SetAlpha(numtype alpha){this->alpha=alpha;}                     // Sets the value of alpha

// Helping functions ////////////////////////////////////////////////////////////////////
// Returns random integter between 0 and n
int Colonies3D::RandI(int n) {

		// Set limit on distribution
		uniform_int_distribution <int> distr(0, n);

		return distr(rng);
}

// Returns random numtype between 0 and 1
numtype Colonies3D::Rand(std::mt19937 rng) {

		// Set limit on distribution
		uniform_real_distribution <numtype> distr(0, 1);

		return distr(rng);
}

// Returns random normal dist. number with mean m and variance s^2
numtype Colonies3D::RandN(numtype m, numtype s) {

		// Set limit on distribution
		normal_distribution <numtype> distr(m, s);

		return distr(rng);
}

// Returns poisson dist. number with mean l
numtype Colonies3D::RandP(numtype l, int i, int j, int k) {

		// // Set limit on distribution
		// poisson_distribution <long long> distr(l);

		// return distr(arr_rng[i*nGridXY*nGridZ + j*nGridZ + k]);

		double L = exp(-l);
		double p = 1.0;
		double n = 0;
		while (p > L) {
			n++;
			double u = rand(arr_rng[i*nGridXY*nGridZ + j*nGridZ + k]);
			p *= u;
		}
		return n - 1;
}

// Returns poisson dist. number with mean l, flat array
numtype Colonies3D::RandP(numtype l, int i) {

		// Set limit on distribution
		poisson_distribution <long long> distr(l);

		return distr(arr_rng[i]);
}

// Returns poisson dist. number with mean l
numtype Colonies3D::RandP(numtype l) {

		// Set limit on distribution
		poisson_distribution <long long> distr(l);

		return distr(rng);
}

// Returns poisson dist. number with mean l
numtype Colonies3D::RandP_fast(numtype l) {

		numtype N;

		if (l < 60) {

				numtype L = cpu_exp(-l);
				numtype p = 1;
				N = 0;
				do {
						N++;
						p *= drand48();
				} while (p > L);
				N--;

		} else {

				numtype r;
				numtype x;
				numtype pi = 3.14159265358979;
				numtype sqrt_l = sqrt(l);
				numtype log_l = log(l);
				numtype g_x;
				numtype f_m;

				do {
						do {
								x = l + sqrt_l*tan(pi*(drand48()-1/2.0));
						} while (x < 0);

						g_x = sqrt_l/(pi*((x-l)*(x-l) + l));
						N = floor(x);

						numtype xx = N + 1;
						numtype pi = 3.14159265358979;
						numtype xx2 = xx*xx;
						numtype xx3 = xx2*xx;
						numtype xx5 = xx3*xx2;
						numtype xx7 = xx5*xx2;
						numtype xx9 = xx7*xx2;
						numtype xx11 = xx9*xx2;
						numtype lgxx = xx*log(xx) - xx - 0.5*log(xx/(2*pi)) +
						1/(12*xx) - 1/(360*xx3) + 1/(1260*xx5) - 1/(1680*xx7) +
						1/(1188*xx9) - 691/(360360*xx11);

						f_m = cpu_exp(N*log_l - l - lgxx);
						r = f_m / g_x / 2.4;
				} while (drand48() > r);
		}

		return cpu_round(N);

}

// Sets the seed of the random number generator
void Colonies3D::SetRngSeed(int n) {
		rngSeed = n;
}

// Write a log.txt file
void Colonies3D::WriteLog() {
		if ((not f_log.is_open()) and (not exit)) {

				// Open the file stream and write the command
				f_log.open(path + "/log.txt", fstream::trunc);

				// Store the initial densities
				f_log << "B_0 = " << fixed << setw(12)  << B_0      << "\n";    // Initial density of bacteria
				f_log << "P_0 = " << fixed << setw(12)  << P_0      << "\n";    // Intiial density of phages
				f_log << "n_0 = " << fixed << setw(12)  << n_0      << "\n";    // Intiial density of nutrient
				f_log << "K = "   << fixed << setw(12)  << K        << "\n";    // Carrying capacity
				f_log << "L = "                         << L        << "\n";    // Side-length of simulation array
				f_log << "H = "                         << H        << "\n";    // height of simulation array
				f_log << "nGridXY = "                   << nGridXY  << "\n";    // Number of gridpoints
				f_log << "nGridZ = "                    << nGridZ   << "\n";    // Number of gridpoints
				f_log << "nSamp = "                     << nSamp    << "\n";    // Number of samples to save per simulation hour
				f_log << "g = "                         << g        << "\n";    // Growth rate for the cells
				f_log << "alpha = "                     << alpha    << "\n";    // Reinfection Percentage
				f_log << "beta = "                      << beta     << "\n";    // Multiplication factor phage
				f_log << "eta = "                       << eta      << "\n";    // Adsorption coefficient
				f_log << "delta = "                     << delta    << "\n";    // Rate of phage decay
				f_log << "r = "                         << r        << "\n";    // Constant used in the time-delay mechanism
				f_log << "zeta = "                      << zeta     << "\n";    // Permeability of surface
				f_log << "D_B = "                       << D_B      << "\n";    // Diffusion constant for the cells
				f_log << "D_P = "                       << D_P      << "\n";    // Diffusion constant for the phage
				f_log << "D_n = "                       << D_n      << "\n";    // Diffusion constant for the nutrient
				f_log << "dT = "                        << dT       << "\n";    // Time-step size
				f_log << "T_end = "                     << T_end    << "\n";    // Time when the simulation stops

				f_log << "rngSeed = "                   << rngSeed  << "\n";    // Random number seed  ( set to -1 if unused )

				f_log << "s = "                         << s        << "\n";    // The reduction of the phage boundary                       = 1;

				f_log << "experimentalConditions = "    << experimentalConditions   << "\n";
				f_log << "clustering = "                << clustering               << "\n";
				f_log << "shielding = "                 << shielding                << "\n";
				f_log << "reducedBeta = "               << reducedBeta              << "\n";
				f_log << "reducedBoundary = "           << reducedBoundary          << endl;

		}
}

// File outputs /////////////////////////////////////////////////////////////////////////

// Stop simulation when all cells are dead
void Colonies3D::FastExit(){fastExit=true;}

// Sets the simulation to export everything
void Colonies3D::ExportAll(){exportAll=true;}

// Master function to export the data
void Colonies3D::ExportData_arr(numtype t, std::string filename_suffix){

	// Verify the file stream is open
	string fileName = "PopulationSize_"+filename_suffix;
	OpenFileStream(f_N, fileName);


	numtype accuB = 0.0;
	numtype accuI = 0.0;
	numtype accuP = 0.0;
	numtype accuNutrient = 0.0;
	numtype accuClusters = 0.0;
	numtype nz = 0.0;
	for (int i = 0; i < nGridXY; i++) {
		for (int j = 0; j < nGridXY; j++ ) {
			for (int k = 0; k < nGridZ; k++ ) {
				accuB += arr_B[i*nGridXY*nGridZ + j*nGridZ + k];
				accuI += arr_I0[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I1[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I2[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I3[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I4[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I5[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I6[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I7[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I8[i*nGridXY*nGridZ + j*nGridZ + k] + arr_I9[i*nGridXY*nGridZ + j*nGridZ + k];
				accuP += arr_P[i*nGridXY*nGridZ + j*nGridZ + k];
				accuNutrient += arr_nutrient[i*nGridXY*nGridZ + j*nGridZ + k];
				accuClusters += arr_nC[i*nGridXY*nGridZ + j*nGridZ + k];
				if (arr_B[i*nGridXY*nGridZ + j*nGridZ + k] > 0.0) {
					nz++;
				}
			}
		}
	}

	// Writes the time, number of cells, number of infected cells, number of phages
	f_N << fixed    << setprecision(2);
	f_N << setw(6)  << t       << "\t";
	f_N << setw(12) << cpu_round(accuB)    << "\t";
	f_N << setw(12) << cpu_round(accuI)    << "\t";
	f_N << setw(12) << cpu_round(accuP)    << "\t";

	f_N << setw(12) << nz / initialOccupancy << "\t";
	f_N << setw(12) << cpu_round(n_0 / 1e12 * pow(L, 2) * H - accuNutrient) << "\t";
	f_N << setw(12) << cpu_round(accuClusters) << endl;

	if (exportAll) {
		// Save the position data
		// Verify the file stream is open
		fileName = "CellDensity_"+filename_suffix;
		OpenFileStream(f_B, fileName);

		fileName = "InfectedDensity_"+filename_suffix;
		OpenFileStream(f_I, fileName);

		fileName = "PhageDensity_"+filename_suffix;
		OpenFileStream(f_P, fileName);

		fileName = "NutrientDensity_"+filename_suffix;
		OpenFileStream(f_n, fileName);

		// Write file as MATLAB would a 3D matrix!
		// row 1 is x_vector, for y_1 and z_1
		// row 2 is x_vector, for y_2 and z_1
		// row 3 is x_vector, for y_3 and z_1
		// ...
		// When y_vector for x_n has been printed, it goes:
		// row n+1 is x_vector, for y_1 and z_2
		// row n+2 is x_vector, for y_2 and z_2
		// row n+3 is x_vector, for y_3 and z_2
		// ... and so on

		// Loop over z
		for (int z = 0; z < nGridZ; z++) {

			// Loop over x
			for (int x = 0; x < nGridXY; x++) {

				// Loop over y
				for (int y = 0; y < nGridXY - 1; y++) {
					#define XYZ x*nGridXY*nGridZ+y*nGridZ+z

					f_B << setw(6) << arr_B[x*nGridXY*nGridZ + y*nGridZ + z] << "\t";
					f_P << setw(6) << arr_P[x*nGridXY*nGridZ + y*nGridZ + z] << "\t";
					numtype nI = cpu_round(arr_I0[x*nGridXY*nGridZ + y*nGridZ + z] + arr_I1[x*nGridXY*nGridZ + y*nGridZ + z] + arr_I2[x*nGridXY*nGridZ + y*nGridZ + z] + arr_I3[x*nGridXY*nGridZ + y*nGridZ + z] + arr_I4[x*nGridXY*nGridZ + y*nGridZ + z] + arr_I5[x*nGridXY*nGridZ + y*nGridZ + z] + arr_I6[x*nGridXY*nGridZ + y*nGridZ + z] + arr_I7[x*nGridXY*nGridZ + y*nGridZ + z] + arr_I8[x*nGridXY*nGridZ + y*nGridZ + z] + arr_I9[x*nGridXY*nGridZ + y*nGridZ + z]);
					f_I << setw(6) << nI       << "\t";
					f_n << setw(6) << arr_nutrient[x*nGridXY*nGridZ + y*nGridZ + z] << "\t";
				}

				#define XnGridXYZ x*nGridXY*nGridZ+(nGridXY-1)*nGridZ+z
				// Write last line ("\n" instead of tab)
				f_B << setw(6) << cpu_round(arr_B[x*nGridXY*nGridZ + (nGridXY - 1)*nGridZ + z]) << "\n";
				f_P << setw(6) << cpu_round(arr_P[x*nGridXY*nGridZ + (nGridXY - 1)*nGridZ + z]) << "\n";
				numtype nI = cpu_round(arr_I0[x*nGridXY*nGridZ + (nGridXY - 1)*nGridZ + z] + arr_I1[x*nGridXY*nGridZ + (nGridXY - 1)*nGridZ + z] + arr_I2[x*nGridXY*nGridZ + (nGridXY - 1)*nGridZ + z] + arr_I3[x*nGridXY*nGridZ + (nGridXY - 1)*nGridZ + z] + arr_I4[x*nGridXY*nGridZ + (nGridXY - 1)*nGridZ + z] + arr_I5[x*nGridXY*nGridZ + (nGridXY - 1)*nGridZ + z] + arr_I6[x*nGridXY*nGridZ + (nGridXY - 1)*nGridZ + z] + arr_I7[x*nGridXY*nGridZ + (nGridXY - 1)*nGridZ + z] + arr_I8[x*nGridXY*nGridZ + (nGridXY - 1)*nGridZ + z] + arr_I9[x*nGridXY*nGridZ + (nGridXY - 1)*nGridZ + z]);
				f_I << setw(6) << nI                        << "\n";
				f_n << setw(6) << cpu_round(arr_nutrient[x*nGridXY*nGridZ + (nGridXY - 1)*nGridZ + z]) << "\n";
			}
		}
	}
}

void Colonies3D::ExportData_arr_reduced(numtype t, numtype accuB, numtype accuI, numtype accuP, numtype accuNutrient, numtype accuClusters, numtype nz, std::string filename_suffix) {

	// Verify the file stream is open
	string fileName = "PopulationSize_"+filename_suffix;
	OpenFileStream(f_N, fileName);

	// Writes the time, number of cells, number of infected cells, number of phages
	f_N << fixed    << setprecision(2);
	f_N << setw(6)  << t       << "\t";
	f_N << setw(12) << cpu_round(accuB)    << "\t";
	f_N << setw(12) << cpu_round(accuI)    << "\t";
	f_N << setw(12) << cpu_round(accuP)    << "\t";

	f_N << setw(12) << nz / initialOccupancy << "\t";
	f_N << setw(12) << cpu_round(n_0 / 1e12 * pow(L, 2) * H - accuNutrient) << "\t";
	f_N << setw(12) << cpu_round(accuClusters) << endl;

}


// Open filstream if not allready opened
void Colonies3D::OpenFileStream(ofstream& stream, string& fileName) {

		// Check that if file stream is open.
		if ((not stream.is_open()) and (not exit)) {

				// Debug info
				cout << "\tSaving data to file: " << path << "/" << fileName << ".txt" << "\n";


				// Check if the output file exists
				time_t theTime = time(NULL);
				struct tm *aTime = localtime(&theTime);

				string streamPath;
				streamPath = path+"/"+fileName+"_"+std::to_string(aTime->tm_hour)+"_"+std::to_string(aTime->tm_min)+".txt";

				// Open the file stream
				stream.open(streamPath, fstream::trunc);

				// Check stream is open
				if ((not exit) and (not stream.is_open())) {
						cerr << "\t>>Could not open filestream \"" << streamPath << "\"! Exiting..<<" << "\n";
						f_log <<  ">>Could not open filestream \"" << streamPath << "\"! Exiting..<<" << "\n";
						exit = true;
				};

				// Write meta data to the data file
				stream << "Datatype: "  << fileName << "\n";
		}
}

// Generates a save path for datafiles
string Colonies3D::GeneratePath() {

			// Generate a directory path
		string prefix = "data";    // Data folder name

		// Create the path variable
		string path_s = prefix;

		// Check if user has specified numbered folder
		if (path.empty()) {

				// Get current date
				time_t t = time(0);                               // Get time now
				struct tm tstruct;                                // And format the date
				tstruct = *localtime(&t);                         // as "MNT_DD_YY" for folder name
				char buffer[80];                                  // Create a buffer to store the date
				strftime(buffer, sizeof(buffer), "%F", &tstruct); // Store the formated foldername in buffer
				string dateFolder(buffer);

				// Add datefolder to path
				path_s += "/";
				path_s += dateFolder;

				// Check if path exists
				struct stat info;
				if (not(stat(path_s.c_str(), &info) == 0 && S_ISDIR(info.st_mode))) {
						// Create path if it does not exist
						mkdir(path_s.c_str(), 0700);
				}

				// Loop over folders in date folder, to find current number
				int currentNumerateFolder = 1;
				DIR *dir;
				if ((dir = opendir (path_s.c_str())) != NULL) {
						struct dirent *ent;
						while ((ent = readdir (dir)) != NULL) {
								if (ent->d_type == DT_DIR) {
										// Skip . or ..
										if (ent->d_name[0] == '.') {continue;}
										currentNumerateFolder++;        // Increment folder number
								}
						}
						closedir (dir);
				}

				// Append numerate folder
				path_s += "/";
				path_s += to_string(currentNumerateFolder);

				// Check if path exists
				if (not(stat(path_s.c_str(), &info) == 0 && S_ISDIR(info.st_mode))) {
						// Create path if it does not exist
						mkdir(path_s.c_str(), 0700);
				}

		} else {    // User has specified a path

				// This path maybe more than one layer deep, so attempt to make it recursively
				int len = path.length();

				// Boolean to see name of first folder
				bool firstFolder = true;

				string folder = "";
				for (int i = 0; i < len; i++) {
						folder += path[i]; // Append char to folder name

						// If seperator is found or if end of path is reached, construct folder
						if ((path[i] == '/') or (i == len - 1)) {

								// If seperator is found, remove it:
								if (path[i] == '/') folder.pop_back();

								// Check if this is the first subfolder
								if (firstFolder) {
										firstFolder = false;

										// Check if first folder contains date format
										if (not ((folder.length() == 10) and(folder[4] == '-') and (folder[7] == '-'))) {

												// Get current date
												time_t t = time(0);                               // Get time now
												struct tm tstruct;                                // And format the date
												tstruct = *localtime(&t);                         // as "MNT_DD_YY" for folder name
												char buffer[80];                                  // Create a buffer to store the date
												strftime(buffer, sizeof(buffer), "%F", &tstruct); // Store the formated foldername in buffer
												string dateFolder(buffer);

												// Add datefolder to path
												path_s += "/";
												path_s += dateFolder;

												// Check if path exists
												struct stat info;
												if (not(stat(path_s.c_str(), &info) == 0 && S_ISDIR(info.st_mode))) {
														// Create path if it does not exist
														mkdir(path_s.c_str(), 0700);
												}
										}
								}

								// Append folder to path
								path_s += "/";
								path_s += folder;

								// Make folder
								struct stat info;
								if (not(stat(path_s.c_str(), &info) == 0 && S_ISDIR(info.st_mode)))
								{ // Create path if it does not exist
										mkdir(path_s.c_str(), 0700);
								}

								folder = ""; // Reset folder
						}
				}
		}

		return path_s;
}

// Sets the folder number (useful when running parralel code)
void Colonies3D::SetFolderNumber(int number) {path = to_string(number);}

// Sets the folder path (useful when running parralel code)
void Colonies3D::SetPath(std::string& path) {this->path = path;}

// Returns the save path
std::string Colonies3D::GetPath() {
		return path;
}


// Clean up /////////////////////////////////////////////////////////////////////////////

// Delete the data folder
void Colonies3D::DeleteFolder() {
		DeleteFolderTree(path.c_str());
}

// Delete folders recursively
void Colonies3D::DeleteFolderTree(const char* directory_name) {

		DIR*            dp;
		struct dirent*  ep;
		char            p_buf[512] = {0};


		dp = opendir(directory_name);

		while ((ep = readdir(dp)) != NULL) {
				// Skip self dir "."
				if (strcmp(ep->d_name, ".") == 0 || strcmp(ep->d_name, "..") == 0) continue;

				sprintf(p_buf, "%s/%s", directory_name, ep->d_name);

				// Is the path a folder?
				struct stat s_buf;
				int IsDirectory = -1;
				if (stat(p_buf, &s_buf)){
						IsDirectory = 0;
				} else {
						IsDirectory = S_ISDIR(s_buf.st_mode);
				}

				// If it is a folder, go recursively into
				if (IsDirectory) {
						DeleteFolderTree(p_buf);
				} else {    // Else delete the file
						unlink(p_buf);
				}
		}

		closedir(dp);
		rmdir(directory_name);
}

// Destructor
Colonies3D::~Colonies3D() {

		// Close filestreams
		if (f_B.is_open()) {
				f_B.flush();
				f_B.close();
		}
		if (f_I.is_open()) {
				f_I.flush();
				f_I.close();
		}
		if (f_P.is_open()) {
				f_P.flush();
				f_P.close();
		}
		if (f_N.is_open()) {
				f_N.flush();
				f_N.close();
		}
		if (f_log.is_open()) {
				f_log.flush();
				f_log.close();
		}
		if (f_kerneltimings.is_open()) {
				f_kerneltimings.flush();
				f_kerneltimings.close();
		}

		 // Delete arrays
		 delete[] arr_B;
		 delete[] arr_I0;
		 delete[] arr_I1;
		 delete[] arr_I2;
		 delete[] arr_I3;
		 delete[] arr_I4;
		 delete[] arr_I5;
		 delete[] arr_I6;
		 delete[] arr_I7;
		 delete[] arr_I8;
		 delete[] arr_I9;
		 delete[] arr_P;
		 delete[] arr_nC;

		 delete[] arr_B_new;
		 delete[] arr_I0_new;
		 delete[] arr_I1_new;
		 delete[] arr_I2_new;
		 delete[] arr_I3_new;
		 delete[] arr_I4_new;
		 delete[] arr_I5_new;
		 delete[] arr_I6_new;
		 delete[] arr_I7_new;
		 delete[] arr_I8_new;
		 delete[] arr_I9_new;
		 delete[] arr_P_new;

		 delete[] arr_nutrient;
		 delete[] arr_Occ;
		 delete[] arr_nutrient_new;

		 delete[] arr_rng;

		 delete[] arr_M;
		 delete[] arr_GrowthModifier;

		 delete[] skipArray;
}

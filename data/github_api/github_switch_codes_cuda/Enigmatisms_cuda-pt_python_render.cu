// Repository: Enigmatisms/cuda-pt
// File: src/pyrender/python_render.cu

/**
 * @file python_render.cpp
 * @author Qianyue He
 * @brief Renderer Nanobind bindings
 * @date 2025-01-10
 * @copyright Copyright (c) 2025
 */
#include "./python_render.cuh"
#include "core/stats.h"
#include "core/scene.cuh"
#include "core/serialize.h"
#include "renderer/bvh_cost.cuh"
#include "renderer/light_tracer.cuh"
#include "renderer/wf_path_tracer.cuh"
#include "renderer/volume_pt.cuh"

template <size_t Ndim>
static nb::ndarray<nb::pytorch, float> gpu_ndarray_deep_copy(const float* gpu_src_ptr, size_t width, size_t height, int dev_id = 0) {
    int num_elements = width * height * Ndim;

    float* gpu_dst_ptr;
    CUDA_CHECK_RETURN(hipMalloc((void**)&gpu_dst_ptr, num_elements * sizeof(float)));

    nb::capsule deleter(gpu_dst_ptr, [](void *p) noexcept {
        CUDA_CHECK_RETURN(hipFree(p));
    });

    CUDA_CHECK_RETURN(hipMemcpy(gpu_dst_ptr, gpu_src_ptr, num_elements * sizeof(float), hipMemcpyDeviceToDevice));
    return nb::ndarray<nb::pytorch, float>(gpu_dst_ptr, {height, width, Ndim}, deleter, {}, nb::dtype<float>(), nb::device::cuda::value, dev_id);
}

nb::ndarray<nb::pytorch, float> PythonRenderer::render() {
    TicTocLocal timer;
    const float* gpu_ptr = rdr->render_raw(scene->config.md, scene->config.gamma_correction);
    ftimer->record(timer.toc());
    return gpu_ndarray_deep_copy<4>(gpu_ptr, rdr->width(), rdr->height(), device_id);
}

nb::ndarray<nb::pytorch, float> PythonRenderer::variance() {
    const float* var_buffer = rdr->get_variance_buffer();
    if (var_buffer) {
        return gpu_ndarray_deep_copy<1>(var_buffer, rdr->width(), rdr->height(), device_id);
    }
    return {};
}

PythonRenderer::PythonRenderer(const nb::str& xml_path, int _device_id, int seed_offset): valid(true), device_id(_device_id) {
    CUDA_CHECK_RETURN(hipSetDevice(_device_id));
    CUDA_CHECK_RETURN(hipFree(nullptr));           // initialize CUDA

    std::string path = std::string(xml_path.c_str());
    ftimer   = std::make_unique<SlidingWindowAverage>(32);
    xyz_host = std::make_unique<ColorSpaceXYZ>();
    scene    = std::make_unique<Scene>(path);

    xyz_host->init();
    CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(c_material), scene->bsdfs, scene->num_bsdfs * sizeof(BSDF*)));
    CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(c_emitter), scene->emitters, (scene->num_emitters + 1) * sizeof(Emitter*)));

    std::cout << "[RENDERER] Path tracer loaded: ";
    switch (scene->rdr_type) {
        case RendererType::MegaKernelPT: {
            rdr = std::make_unique<PathTracer>(*scene); 
            rdr->initialize_var_buffer();
            std::cout << "\tMegakernel Path Tracing.\n";
            break;
        }
        case RendererType::WavefrontPT: {
            rdr = std::make_unique<WavefrontPathTracer>(*scene);
            rdr->initialize_var_buffer();
            std::cout << "\tWavefront Path Tracing..\n";
            break;
        }
        case RendererType::MegeKernelLT: {
            rdr = std::make_unique<LightTracer>(*scene, scene->config.spec_constraint,
                        scene->config.bidirectional, scene->config.caustic_scaling); 
            if (scene->config.bidirectional)
                std::cout << "\tNaive Bidirectional Megakernel Light Tracing.\n";
            else
                std::cout << "\tMegakernel Light Tracing.\n";
            break;
        } 
        case RendererType::MegaKernelVPT: {
            rdr = std::make_unique<VolumePathTracer>(*scene);
            std::cerr << "\tVolumetric Path Tracer\n";
            break;
        }
        case RendererType::VoxelSDFPT: {
            std::cerr << "\tVoxelSDFPT is not implemented yet. Stay tuned. Rendering exits.\n";
            exit(0);
        }
        case RendererType::DepthTracing: {
            rdr = std::make_unique<DepthTracer>(*scene);
            std::cerr << "\tDepth Tracing\n";
            break;
        }
        case RendererType::BVHCostViz: {
            rdr = std::make_unique<BVHCostVisualizer>(*scene);
            std::cerr << "\tBVH Cost Visualizer\n";
            break;
        }
        default: {
            throw std::runtime_error("Unsupported renderer type.");
        }
    }
    scene->free_resources();
    rdr->set_seed_offset(seed_offset);
    rdr->update_camera(scene->cam);
    rdr->initialize_output_buffer();
}

void PythonRenderer::release() {
    xyz_host->destroy();
    valid = false;
}

void PythonRenderer::info() const {
    scene->print();
}
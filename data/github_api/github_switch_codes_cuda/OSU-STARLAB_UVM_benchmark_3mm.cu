#include "hip/hip_runtime.h"
// Repository: OSU-STARLAB/UVM_benchmark
// File: UVM_benchmarks_oversub/polybench/3MM/3mm.cu

/**
 * 3mm.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <unistd.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#include "../../../common/polybenchUtilFuncts.h"

#define GPU_DEVICE 0

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

/* Problem size. */
# define NI 512 * 16
# define NJ 512* 16
# define NK 512* 16
# define NL 512* 16
# define NM 512* 16

/* Thread block dimensions */
#define DIM_THREAD_BLOCK_X 32
#define DIM_THREAD_BLOCK_Y 8

/* Can switch DATA_TYPE between float and double */
typedef float DATA_TYPE;



void init_array(DATA_TYPE* A, DATA_TYPE* B, DATA_TYPE* C, DATA_TYPE* D, DATA_TYPE* A_gpu, DATA_TYPE* B_gpu, DATA_TYPE* C_gpu, DATA_TYPE* D_gpu )
{
	int i, j;

	for (i = 0; i < NI; i++)
	{
		for (j = 0; j < NK; j++)
		{
			A[i*NK + j] = ((DATA_TYPE) i*j) / NI;
			A_gpu[i*NK + j] = ((DATA_TYPE) i*j) / NI;
		}
	}
  
	for (i = 0; i < NK; i++)
	{
		for (j = 0; j < NJ; j++)
		{
			B[i*NJ + j] = ((DATA_TYPE) i*(j+1)) / NJ;
			B_gpu[i*NJ + j] = ((DATA_TYPE) i*(j+1)) / NJ;
		}
	}
  
	for (i = 0; i < NJ; i++)
	{
		for (j = 0; j < NM; j++)
		{
			C[i*NM + j] = ((DATA_TYPE) i*(j+3)) / NL;
			C_gpu[i*NM + j] = ((DATA_TYPE) i*(j+3)) / NL;
		}
	}
  
	for (i = 0; i < NM; i++)
	{
		for (j = 0; j < NL; j++)
		{
			D[i*NL + j] = ((DATA_TYPE) i*(j+2)) / NK;
			D_gpu[i*NL + j] = ((DATA_TYPE) i*(j+2)) / NK;
		}
	}
}


void compareResults(DATA_TYPE *G, DATA_TYPE *G_outputFromGpu)
{
	int i,j,fail;
	fail = 0;

	for (i=0; i < NI; i++)
	{
		for (j=0; j < NL; j++)
		{
			if (percentDiff(G[i*NL + j], G_outputFromGpu[i*NL + j]) > PERCENT_DIFF_ERROR_THRESHOLD)
			{
				fail++;				
			}
		}
	}
	
	// print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}


void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
	printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
	hipSetDevice( GPU_DEVICE );
}

	
__global__ void mm3_kernel1(DATA_TYPE *A, DATA_TYPE *B, DATA_TYPE *E)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < NI) && (j < NJ))
	{
		int k;
		for(k=0; k < NK; k++)
		{
			E[i * NJ + j] += A[i * NK + k] * B[k * NJ + j];
		}
	}
}

	
__global__ void mm3_kernel2(DATA_TYPE *C, DATA_TYPE *D, DATA_TYPE *F)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < NJ) && (j < NL))
	{
		int k;
		for(k=0; k < NM; k++)
		{
			F[i * NL + j] += C[i * NM + k] * D[k * NL +j];
		}
	}
}

	
__global__ void mm3_kernel3(DATA_TYPE *E, DATA_TYPE *F, DATA_TYPE *G)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < NI) && (j < NL))
	{
		int k;
		for(k=0; k < NJ; k++)
		{
			G[i * NL + j] += E[i * NJ + k] * F[k * NL + j];
		}
	}
}


void mm3_cpu(DATA_TYPE *A, DATA_TYPE *B, DATA_TYPE *C, DATA_TYPE *D, DATA_TYPE *E, DATA_TYPE *F, DATA_TYPE *G)
{
	int i,j,k;
	
	/* E := A*B */
	for (i = 0; i < NI; i++)
	{
		for (j = 0; j < NJ; j++)
		{
			E[i*NJ + j] = 0;
			for (k = 0; k < NK; ++k)
			{
				E[i*NJ + j] += A[i*NK + k] * B[k*NJ + j];
			}
		}
	}
		
	/* F := C*D */
	for (i = 0; i < NJ; i++)
	{
		for (j = 0; j < NL; j++)
		{
			F[i*NL + j] = 0;
			for (k = 0; k < NM; ++k)
			{
				F[i*NL + j] += C[i*NM + k] * D[k*NL + j];
			}
		}
	}

  	/* G := E*F */
	for (i = 0; i < NI; i++)
	{
		for (j = 0; j < NL; j++)
		{
			G[i*NL + j] = 0;
			for (k = 0; k < NJ; ++k)
			{
				G[i*NL + j] += E[i*NJ + k] * F[k*NL + j];
			}
		}
	}
}


void mm3Cuda(DATA_TYPE* A_gpu, DATA_TYPE* B_gpu, DATA_TYPE* C_gpu, DATA_TYPE* D_gpu, DATA_TYPE* E_gpu, DATA_TYPE* F_gpu, 
		DATA_TYPE* G_gpu)
{
	double t_start, t_end;

	
	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 grid1((size_t)(ceil( ((float)NJ) / ((float)DIM_THREAD_BLOCK_X) )),(size_t)(ceil((float)NI/ ((float)DIM_THREAD_BLOCK_Y) )));
	dim3 grid2((size_t)(ceil( ((float)NL) / ((float)DIM_THREAD_BLOCK_X) )),(size_t)(ceil((float)NJ/ ((float)DIM_THREAD_BLOCK_Y) )));
	dim3 grid3((size_t)(ceil( ((float)NL) / ((float)DIM_THREAD_BLOCK_X) )),(size_t)(ceil((float)NI/ ((float)DIM_THREAD_BLOCK_Y) )));

	t_start = rtclock();
	mm3_kernel1<<<grid1,block>>>(A_gpu, B_gpu, E_gpu);
	hipDeviceSynchronize();
	mm3_kernel2<<<grid2,block>>>(C_gpu, D_gpu, F_gpu);
	hipDeviceSynchronize();
	mm3_kernel3<<<grid3,block>>>(E_gpu, F_gpu, G_gpu);
	hipDeviceSynchronize();
	t_end = rtclock();

	fprintf(stdout, "GPU Runtime: %0.6lfs\n", t_end - t_start);
}


int main(int argc, char** argv)
{
	double t_start, t_end;

	DATA_TYPE* A;
	DATA_TYPE* B;
	DATA_TYPE* C;
	DATA_TYPE* D;
	DATA_TYPE* E;
	DATA_TYPE* F;
	DATA_TYPE* G;

	DATA_TYPE *A_gpu;
	DATA_TYPE *B_gpu;
	DATA_TYPE *C_gpu;
	DATA_TYPE *D_gpu;
	DATA_TYPE *E_gpu;
	DATA_TYPE *F_gpu;
	DATA_TYPE *G_gpu;


	A = (DATA_TYPE*)malloc(NI*NK*sizeof(DATA_TYPE));
	B = (DATA_TYPE*)malloc(NK*NJ*sizeof(DATA_TYPE));
	C = (DATA_TYPE*)malloc(NJ*NM*sizeof(DATA_TYPE));
	D = (DATA_TYPE*)malloc(NM*NL*sizeof(DATA_TYPE));
	E = (DATA_TYPE*)malloc(NI*NJ*sizeof(DATA_TYPE));
	F = (DATA_TYPE*)malloc(NJ*NL*sizeof(DATA_TYPE));
	G = (DATA_TYPE*)malloc(NI*NL*sizeof(DATA_TYPE));


	hipMallocManaged(&A_gpu, sizeof(DATA_TYPE) * NI * NK);
	hipMallocManaged(&B_gpu, sizeof(DATA_TYPE) * NK * NJ);
	hipMallocManaged(&C_gpu, sizeof(DATA_TYPE) * NJ * NM);
	hipMallocManaged(&D_gpu, sizeof(DATA_TYPE) * NM * NL);
	hipMallocManaged(&E_gpu, sizeof(DATA_TYPE) * NI * NJ);
	hipMallocManaged(&F_gpu, sizeof(DATA_TYPE) * NJ * NL);
	hipMallocManaged(&G_gpu, sizeof(DATA_TYPE) * NI * NL);

	init_array(A, B, C, D, A_gpu, B_gpu, C_gpu, D_gpu);

	GPU_argv_init();

	mm3Cuda(A_gpu, B_gpu, C_gpu, D_gpu, E_gpu, F_gpu, G_gpu);

	t_start = rtclock();

	mm3_cpu(A, B, C, D, E, F, G);
	
	t_end = rtclock();

	fprintf(stdout, "CPU Runtime: %0.6lfs\n", t_end - t_start);

	compareResults(G, G_gpu);

	free(A);
	free(B);
	free(C);
	free(D);
	free(E);
	free(F);
	free(G);

	hipFree(A_gpu);
	hipFree(B_gpu);
	hipFree(C_gpu);
	hipFree(D_gpu);
	hipFree(E_gpu);
	hipFree(F_gpu);
	hipFree(G_gpu);

	return 0;
}


#include "hip/hip_runtime.h"
// Repository: YiZhangCUG/liblcg
// File: src/lib/solver_cuda.cu

/******************************************************
 * C++ Library of the Linear Conjugate Gradient Methods (LibLCG)
 * 
 * Copyright (C) 2022  Yi Zhang (yizhang-geo@zju.edu.cn)
 * 
 * LibLCG is distributed under a dual licensing scheme. You can
 * redistribute it and/or modify it under the terms of the GNU Lesser
 * General Public License (LGPL) as published by the Free Software Foundation,
 * either version 2 of the License, or (at your option) any later version. 
 * You should have received a copy of the GNU Lesser General Public 
 * License along with this program. If not, see <http://www.gnu.org/licenses/>. 
 * 
 * If the terms and conditions of the LGPL v.2. would prevent you from
 * using the LibLCG, please consider the option to obtain a commercial
 * license for a fee. These licenses are offered by the LibLCG developing 
 * team. As a rule, licenses are provided "as-is", unlimited in time for 
 * a one time fee. Please send corresponding requests to: yizhang-geo@zju.edu.cn. 
 * Please do not forget to include some description of your company and the 
 * realm of its activities. Also add information on how to contact you by 
 * electronic and paper mail.
 ******************************************************/

#include "solver_cuda.h"

#include "cmath"
#include "ctime"
#include "iostream"

LCG_CUDA_Solver::LCG_CUDA_Solver()
{
	param_ = lcg_default_parameters();
	inter_ = 1;
	silent_ = false;
}

int LCG_CUDA_Solver::Progress(const lcg_float* m, const lcg_float converge, 
	const lcg_para* param, const int n_size, const int nz_size, const int k)
{
	if (inter_ > 0 && k%inter_ == 0)
	{
		std::clog << "\rIteration-times: " << k << "\tconvergence: " << converge;
		return 0;
	}

	if (converge <= param->epsilon)
	{
		std::clog << "\rIteration-times: " << k << "\tconvergence: " << converge;
	}
	return 0;
}

void LCG_CUDA_Solver::silent()
{
	silent_ = true;
	return;
}

void LCG_CUDA_Solver::set_report_interval(unsigned int inter)
{
	inter_ = inter;
	return;
}

void LCG_CUDA_Solver::set_lcg_parameter(const lcg_para &in_param)
{
	param_ = in_param;
	return;
}

void LCG_CUDA_Solver::Minimize(hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle, lcg_float *x, lcg_float *b, 
	const int n_size, const int nz_size, lcg_solver_enum solver_id, bool verbose, bool er_throw)
{
	if (silent_)
	{
		int ret = lcg_solver_cuda(_AxProduct, nullptr, x, b, n_size, nz_size, &param_, this, cub_handle, cus_handle, solver_id);
		if (ret < 0) lcg_error_str(ret, true);
		return;
	}
	
	// 使用lcg求解 注意当我们使用函数指针来调用求解函数时默认参数不可以省略
	clock_t start = clock();
	int ret = lcg_solver_cuda(_AxProduct, _Progress, x, b, n_size, nz_size, &param_, this, cub_handle, cus_handle, solver_id);
	clock_t end = clock();

	lcg_float costime = 1000*(end-start)/(double)CLOCKS_PER_SEC;
	
	if (!er_throw)
	{
		std::clog << std::endl;
		switch (solver_id)
		{
			case LCG_CG:
				std::clog << "Solver: CG. Time cost: " << costime << " ms" << std::endl;
				break;
			case LCG_CGS:
				std::clog << "Solver: CGS. Time cost: " << costime << " ms" << std::endl;
				break;
			default:
				std::clog << "Solver: Unknown. Time cost: " << costime << " ms" << std::endl;
				break;
		}	
	}

	if (verbose) lcg_error_str(ret, er_throw);
	else if (ret < 0) lcg_error_str(ret, er_throw);
	return;
}

void LCG_CUDA_Solver::MinimizePreconditioned(hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle, lcg_float *x, lcg_float *b, 
    const int n_size, const int nz_size, lcg_solver_enum solver_id, bool verbose, bool er_throw)
{
	if (silent_)
	{
		int ret = lcg_solver_preconditioned_cuda(_AxProduct, _MxProduct, nullptr, x, b, n_size, nz_size, &param_, this, cub_handle, cus_handle, solver_id);
		if (ret < 0) lcg_error_str(ret, true);
		return;
	}
	
	// 使用lcg求解 注意当我们使用函数指针来调用求解函数时默认参数不可以省略
	clock_t start = clock();
	int ret = lcg_solver_preconditioned_cuda(_AxProduct, _MxProduct, _Progress, x, b, n_size, nz_size, &param_, this, cub_handle, cus_handle, solver_id);
	clock_t end = clock();

	lcg_float costime = 1000*(end-start)/(double)CLOCKS_PER_SEC;
	
	if (!er_throw)
	{
		std::clog << std::endl;
		switch (solver_id)
		{
			case LCG_PCG:
				std::clog << "Solver: PCG. Time cost: " << costime << " ms" << std::endl;
				break;
			default:
				std::clog << "Solver: Unknown. Time cost: " << costime << " ms" << std::endl;
				break;
		}	
	}

	if (verbose) lcg_error_str(ret, er_throw);
	else if (ret < 0) lcg_error_str(ret, er_throw);
	return;
}

void LCG_CUDA_Solver::MinimizeConstrained(hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle, lcg_float *x, const lcg_float *b, 
    const lcg_float* low, const lcg_float *hig, const int n_size, const int nz_size, lcg_solver_enum solver_id, 
    bool verbose, bool er_throw)
{
	if (silent_)
	{
		int ret = lcg_solver_constrained_cuda(_AxProduct, nullptr, x, b, low, hig, n_size, nz_size, &param_, this, cub_handle, cus_handle, solver_id);
		if (ret < 0) lcg_error_str(ret, true);
		return;
	}
	
	// 使用lcg求解 注意当我们使用函数指针来调用求解函数时默认参数不可以省略
	clock_t start = clock();
	int ret = lcg_solver_constrained_cuda(_AxProduct, _Progress, x, b, low, hig, n_size, nz_size, &param_, this, cub_handle, cus_handle, solver_id);
	clock_t end = clock();

	lcg_float costime = 1000*(end-start)/(double)CLOCKS_PER_SEC;
	
	if (!er_throw)
	{
		std::clog << std::endl;
		switch (solver_id)
		{
			case LCG_PG:
				std::clog << "Solver: PG. Time cost: " << costime << " ms" << std::endl;
				break;
			default:
				std::clog << "Solver: Unknown. Time cost: " << costime << " ms" << std::endl;
				break;
		}	
	}

	if (verbose) lcg_error_str(ret, er_throw);
	else if (ret < 0) lcg_error_str(ret, er_throw);
	return;
}


CLCG_CUDAF_Solver::CLCG_CUDAF_Solver()
{
	param_ = clcg_default_parameters();
	inter_ = 1;
	silent_ = false;
}

int CLCG_CUDAF_Solver::Progress(const hipComplex* m, const float converge, 
	const clcg_para* param, const int n_size, const int nz_size, const int k)
{
	if (inter_ > 0 && k%inter_ == 0)
	{
		std::clog << "\rIteration-times: " << k << "\tconvergence: " << converge;
		return 0;
	}

	if (converge <= param->epsilon)
	{
		std::clog << "\rIteration-times: " << k << "\tconvergence: " << converge;
	}
	return 0;
}

void CLCG_CUDAF_Solver::silent()
{
	silent_ = true;
	return;
}

void CLCG_CUDAF_Solver::set_report_interval(unsigned int inter)
{
	inter_ = inter;
	return;
}

void CLCG_CUDAF_Solver::set_clcg_parameter(const clcg_para &in_param)
{
	param_ = in_param;
	return;
}

void CLCG_CUDAF_Solver::Minimize(hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle, hipComplex *x, hipComplex *b, 
	const int n_size, const int nz_size, clcg_solver_enum solver_id, bool verbose, bool er_throw)
{
	if (silent_)
	{
		int ret = clcg_solver_cuda(_AxProduct, nullptr, x, b, n_size, nz_size, &param_, this, cub_handle, cus_handle, solver_id);
		if (ret < 0) lcg_error_str(ret, true);
		return;
	}
	
	// 使用lcg求解 注意当我们使用函数指针来调用求解函数时默认参数不可以省略
	clock_t start = clock();
	int ret = clcg_solver_cuda(_AxProduct, _Progress, x, b, n_size, nz_size, &param_, this, cub_handle, cus_handle, solver_id);
	clock_t end = clock();

	float costime = 1000*(end-start)/(double)CLOCKS_PER_SEC;
	
	if (!er_throw)
	{
		std::clog << std::endl;
		switch (solver_id)
		{
			case CLCG_BICG:
				std::clog << "Solver: BI-CG. Time cost: " << costime << " ms" << std::endl;
				break;
			case CLCG_BICG_SYM:
				std::clog << "Solver: BI-CG (symmetrically accelerated). Time cost: " << costime << " ms" << std::endl;
				break;
			default:
				std::clog << "Solver: Unknown. Time cost: " << costime << " ms" << std::endl;
				break;
		}	
	}

	if (verbose) lcg_error_str(ret, er_throw);
	else if (ret < 0) lcg_error_str(ret, er_throw);
	return;
}

void CLCG_CUDAF_Solver::MinimizePreconditioned(hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle, hipComplex *x, hipComplex *b, 
	const int n_size, const int nz_size, clcg_solver_enum solver_id, bool verbose, bool er_throw)
{
	if (silent_)
	{
		int ret = clcg_solver_preconditioned_cuda(_AxProduct, _MxProduct, nullptr, x, b, n_size, nz_size, &param_, this, cub_handle, cus_handle, solver_id);
		if (ret < 0) lcg_error_str(ret, true);
		return;
	}
	
	// 使用lcg求解 注意当我们使用函数指针来调用求解函数时默认参数不可以省略
	clock_t start = clock();
	int ret = clcg_solver_preconditioned_cuda(_AxProduct, _MxProduct, _Progress, x, b, n_size, nz_size, &param_, this, cub_handle, cus_handle, solver_id);
	clock_t end = clock();

	float costime = 1000*(end-start)/(double)CLOCKS_PER_SEC;
	
	if (!er_throw)
	{
		std::clog << std::endl;
		switch (solver_id)
		{
			case CLCG_PCG:
				std::clog << "Solver: PCG. Time cost: " << costime << " ms" << std::endl;
				break;
			default:
				std::clog << "Solver: Unknown. Time cost: " << costime << " ms" << std::endl;
				break;
		}	
	}

	if (verbose) lcg_error_str(ret, er_throw);
	else if (ret < 0) lcg_error_str(ret, er_throw);
	return;
}


CLCG_CUDA_Solver::CLCG_CUDA_Solver()
{
	param_ = clcg_default_parameters();
	inter_ = 1;
	silent_ = false;
}

int CLCG_CUDA_Solver::Progress(const hipDoubleComplex* m, const lcg_float converge, 
	const clcg_para* param, const int n_size, const int nz_size, const int k)
{
	if (inter_ > 0 && k%inter_ == 0)
	{
		std::clog << "\rIteration-times: " << k << "\tconvergence: " << converge;
		return 0;
	}

	if (converge <= param->epsilon)
	{
		std::clog << "\rIteration-times: " << k << "\tconvergence: " << converge;
	}
	return 0;
}

void CLCG_CUDA_Solver::silent()
{
	silent_ = true;
	return;
}

void CLCG_CUDA_Solver::set_report_interval(unsigned int inter)
{
	inter_ = inter;
	return;
}

void CLCG_CUDA_Solver::set_clcg_parameter(const clcg_para &in_param)
{
	param_ = in_param;
	return;
}

void CLCG_CUDA_Solver::Minimize(hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle, hipDoubleComplex *x, hipDoubleComplex *b, 
	const int n_size, const int nz_size, clcg_solver_enum solver_id, bool verbose, bool er_throw)
{
	if (silent_)
	{
		int ret = clcg_solver_cuda(_AxProduct, nullptr, x, b, n_size, nz_size, &param_, this, cub_handle, cus_handle, solver_id);
		if (ret < 0) lcg_error_str(ret, true);
		return;
	}
	
	// 使用lcg求解 注意当我们使用函数指针来调用求解函数时默认参数不可以省略
	clock_t start = clock();
	int ret = clcg_solver_cuda(_AxProduct, _Progress, x, b, n_size, nz_size, &param_, this, cub_handle, cus_handle, solver_id);
	clock_t end = clock();

	lcg_float costime = 1000*(end-start)/(double)CLOCKS_PER_SEC;
	
	if (!er_throw)
	{
		std::clog << std::endl;
		switch (solver_id)
		{
			case CLCG_BICG:
				std::clog << "Solver: BI-CG. Time cost: " << costime << " ms" << std::endl;
				break;
			case CLCG_BICG_SYM:
				std::clog << "Solver: BI-CG (symmetrically accelerated). Time cost: " << costime << " ms" << std::endl;
				break;
			default:
				std::clog << "Solver: Unknown. Time cost: " << costime << " ms" << std::endl;
				break;
		}	
	}

	if (verbose) lcg_error_str(ret, er_throw);
	else if (ret < 0) lcg_error_str(ret, er_throw);
	return;
}

void CLCG_CUDA_Solver::MinimizePreconditioned(hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle, hipDoubleComplex *x, hipDoubleComplex *b, 
	const int n_size, const int nz_size, clcg_solver_enum solver_id, bool verbose, bool er_throw)
{
	if (silent_)
	{
		int ret = clcg_solver_preconditioned_cuda(_AxProduct, _MxProduct, nullptr, x, b, n_size, nz_size, &param_, this, cub_handle, cus_handle, solver_id);
		if (ret < 0) lcg_error_str(ret, true);
		return;
	}
	
	// 使用lcg求解 注意当我们使用函数指针来调用求解函数时默认参数不可以省略
	clock_t start = clock();
	int ret = clcg_solver_preconditioned_cuda(_AxProduct, _MxProduct, _Progress, x, b, n_size, nz_size, &param_, this, cub_handle, cus_handle, solver_id);
	clock_t end = clock();

	lcg_float costime = 1000*(end-start)/(double)CLOCKS_PER_SEC;
	
	if (!er_throw)
	{
		std::clog << std::endl;
		switch (solver_id)
		{
			case CLCG_PCG:
				std::clog << "Solver: PCG. Time cost: " << costime << " ms" << std::endl;
				break;
			default:
				std::clog << "Solver: Unknown. Time cost: " << costime << " ms" << std::endl;
				break;
		}	
	}

	if (verbose) lcg_error_str(ret, er_throw);
	else if (ret < 0) lcg_error_str(ret, er_throw);
	return;
}
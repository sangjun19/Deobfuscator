// Repository: Marco-Christiani/zigrad
// File: src/cuda/cuda_helpers.cu

#ifndef __CUDA_HELPERS_ZIG__
#define __CUDA_HELPERS_ZIG__

#include <cutensor/types.h>
#include <stdio.h>
#include "/usr/local/cuda/include/hip/hip_runtime.h"
#include "/usr/local/cuda/include/hipblas.h"
#include "cuda_includes.cu"
#include "decls.h"

typedef unsigned char u8;
typedef float f32;
typedef double f64;
typedef int64_t i64;
typedef int32_t i32;
typedef uint64_t u64;
typedef uint32_t u32;

#define WARP_SIZE 32

inline hipStream_t get_stream(void* context) {
  return static_cast<hipStream_t>(context);
}

inline hipblasHandle_t get_handle(void* context) {
  return static_cast<hipblasHandle_t>(context);
}

#define CUDA_ASSERT(err) (HandleCudaError( err, __FILE__, __LINE__ ))
inline void HandleCudaError(hipError_t err, const char *file, int line)
{
  if (err != hipSuccess) {
    printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
    exit(EXIT_FAILURE);
  }
}

#define CUBLAS_ASSERT(err) (handleCublasError( err, __FILE__, __LINE__ ))
inline void handleCublasError(hipblasStatus_t err, const char *file, int line)
{
  // TODO: Report better cublas errors
  if (err != HIPBLAS_STATUS_SUCCESS) {
      printf("Cublas failure in %s at line %d\n", file, line);
    exit(EXIT_FAILURE);
  }
}

#define CURESULT_ASSERT(err) (handleCuresultError( err, __FILE__, __LINE__ ))
inline void handleCuresultError(hipError_t err, const char *file, int line)
{
  if (err != hipSuccess) {
    const char** msg = nullptr;

    hipDrvGetErrorString(err, msg);

    if (*msg) {
      printf("%s in %s at line %d\n", *msg, file, line);
    } else {
      printf("Unkown error in %s at line %d\n", file, line);
    }   
    exit(EXIT_FAILURE);
  }
}

#define CUDNN_ASSERT(err) (handleCudnnError( err, __FILE__, __LINE__ ))
inline void handleCudnnError(hipdnnStatus_t err, const char *file, int line)
{
  // TODO: Report better cublas errors
  if (err != HIPDNN_STATUS_SUCCESS) {
      printf("CUDNN failure in %s at line %d\n", file, line);
    exit(EXIT_FAILURE);
  }
}

#define CUTENSOR_ASSERT(err) (handleCutensorStatus(err, __FILE__, __LINE__ ))
inline void handleCutensorStatus(hiptensorStatus_t status, const char *file, int line)
{
  // TODO: Report better cublas errors
  if (status != HIPTENSOR_STATUS_SUCCESS) {
      printf("%s in %s at line %d\n", hiptensorGetErrorString(status), file, line);
    exit(EXIT_FAILURE);
  }
}

#define CHECK_INVARIANT(b, msg) (CheckInvariant(b, msg, __FILE__, __LINE__ ))
inline void CheckInvariant(bool check, const char* message, const char *file, int line)
{
  if (!(check)) {
    printf("%s in %s at line %d\n", (message), file, line);
    exit(EXIT_FAILURE);
  }
}

#define SYSTEM_EXIT(msg) (SystemExit(msg, __FILE__, __LINE__ ))
inline void SystemExit(const char* message, const char *file, int line)
{
  printf("%s in %s at line %d\n", (message), file, line);
  exit(EXIT_FAILURE);
}

template <typename T>
T* __alloc(hipStream_t stream, len_t n, const char* file, int line) {
  hipDeviceptr_t dptr;
  const hipStream_t _stream = static_cast<hipStream_t>(stream);
  handleCuresultError(hipMallocAsync(&dptr, n * sizeof(T), _stream), file, line);
  return reinterpret_cast<T*>(dptr);
}

template <typename T>
inline void __free(hipStream_t stream, T* s, const char* file, int line) {
  hipStream_t _stream = static_cast<hipStream_t>(stream);
  hipDeviceptr_t dptr = reinterpret_cast<hipDeviceptr_t>(s);
  handleCuresultError(hipFreeAsync(dptr, _stream), file, line);
}

template <typename T>
void __ensure_scratch(
  hipStream_t stream,
  len_t* mem,
  len_t* cap,
  len_t new_cap,
  const char* file,
  int line
) {
  if (new_cap <= *cap)
    return;

  if (*cap > 0) {
    __free(stream, reinterpret_cast<T*>(*mem), file, line);
  }
  *mem = reinterpret_cast<len_t>(__alloc<T>(stream, new_cap, file, line));
  *cap = new_cap;
}

void __ensure_scratch(
  dtype id,
  hipStream_t stream,
  len_t* mem,
  len_t* cap,
  len_t new_cap,
  const char* file,
  int line
) {
  switch (id) {
    case SINGLE: return __ensure_scratch<f32>(stream, mem, cap, new_cap, file, line);
    case DOUBLE: return __ensure_scratch<f64>(stream, mem, cap, new_cap, file, line);
    default: SYSTEM_EXIT("Unsupported datatype");
  }
}

#define ENSURE_SCRATCH(id, stream, mem, cap, new_cap)                    \
  (__ensure_scratch(id, stream, mem, cap, new_cap, __FILE__, __LINE__ )) \


inline hipStream_t __cublas_stream(void* handle) {
  hipStream_t stream;
  CUBLAS_ASSERT(hipblasGetStream(static_cast<hipblasHandle_t>(handle), &stream));
  return stream;
}

inline hipStream_t __cudnn_stream(void* handle) {
  hipStream_t stream;
  CUDNN_ASSERT(hipdnnGetStream(static_cast<hipdnnHandle_t>(handle), &stream));
  return stream;
}

inline len_t __product(const len_t* dims, len_t size) {
  len_t res = 1;
  for (len_t i = 0; i < size; ++i) {
    res *= dims[i];
  }
  return res;
}

#endif


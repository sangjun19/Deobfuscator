// Repository: mingwang-zhong/Calcium-Wave
// File: intact/Hill7_cstar1.1_ISO_NoATP/cell.cu

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
using namespace std;

/////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////
// #define Permeabilized		// Permeabalized cell. No sarcolemmal ion channels
// #define PermeabilizedB		// Permeabalized cell. Myocyte diffuses with a bath ( boundary condition )
// #define LQT2			// Long-QT 2 syndrome simulation. No I_Kr
#define ISO			// Isoproterenol, increases Uptake and I_Ca,L (and/or IKs)
// #define Vclamp		// step function voltage clamp
// #define APclamp		// action potential clamp

/////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////
#define DT 	0.02 //ms, time step
#define stoptime (25001.0) //ms
#define PCL 2000.0	//ms, pacing cycle length
#define stopbeat 10
#define time_before_beat 100 //ms

#define out_step	50 // number of steps to output data
// #define output_linescan // output linescan
// #define output_fluoxt // output fluorescence

#ifdef	output_fluoxt
	#include <fftw3.h>
#endif
//////////////////////////////////////////////////////////////////
////////////////// CUDA block size 

#define BLOCK_SIZE_X 8
#define BLOCK_SIZE_Y 7
#define BLOCK_SIZE_Z 4
#define Nx 	64	// Number of CRUs in the x direction
#define Ny 	28	// Number of CRUs in the y direction
#define Nz 	12	// Number of CRUs in the z direction
#define Nix	2	// number of lattices in x direction in a CRU
#define Niy	2	// number of lattices in y direction in a CRU
#define Niz	2	// number of lattices in z direction in a CRU
#define Nci	8 //(Nix*Niy*Niz) // number of lattices in a CRU
#define DX	(1.65/Nix) // um, size of each lattice in longitudinal direction
#define DY	(0.76/Niy) // um
#define DZ	(0.76/Niz) // um

//////////////////////////////////////////////////////////////////
////////////////// cell properties

#define Vp 0.00126 	//um^3, Volume of the proximal space
#define Vs 0.025 //um^3, Volume of the submembrane space
#define Vjsr 0.03	//um^3, Volume of the JSR space
#define Vi (0.76/Nci)	//um^3, Volume of the cytosolic space, for each compartment
#define Vnsr (0.025/Nci)	//um^3, Volume of the NSR space
#define taups 0.01	//ms, Diffusion time from the proximal to the submembrane
#define taupi 0.05 //ms, Diffusion time from the proximal to the cytosol
#define tausi 0.02		//ms, Diffusion time from the submembrane to the cytosolic
#define taust 1.5  //ms, diffusion time in submembrane along transverse direction
#define tautr 4.25	//ms, Diffusion time from NSR to JSR 
#define taunl 4.2		//ms, diffusion time of longitudinal NSR
#define taunt 1.3		//ms, diffusion time of transverse NSR
// #define tauil 2.7	//ms, diffusion time of longitudinal cytosolic
#define tauit 0.963 	//ms, diffusion time of transverse cytosolic
#define	Datp 	(5.8/2.7)	// diffusion time prefactor of ATP
#define Ddye 	(5.4/2.7)	// diffusion time prefactor of dye 0.09*765/1100
#define Degta 	(7.0/2.7)	// diffusion time prefactor of EGTA 

#define ci_basal (atof(argv[2]))
#define cjsr_basal (atof(argv[3]))
/////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////
//////////// ion channel parameters 

#define	gKs  (0.4)  // mS/uF, IKs conductance
#define gtof (0.1)  // mS/uF, Itof conductance
#define gtos (0.04)  // mS/uF, Itos conductance
#define gNaK (1.5)  // mS/uF, INaK conductance
#define gK1  (0.3)  // mS/uF, IK1 conductance
#define gNa  (12.0)  // mS/uF, INa conductance
#define gNaLeak (0.0015)  // mS/uF, INaLeak conductance
#define Vncx 	(25.0) //uM/ms, strength of NCX current
#define	Vleak (0.00212) // ms^{-1}, Shannon et al 2004, Eq. 107 (0.00212 = 5.348e-6*0.5/0.00126 )

#define FracSL 0.89 // distribution between submembrane and dayad: 0.89:0.11

#ifdef ISO
	#define Vup (200.0) // uM/ms, uptake strength
#else
	#define Vup (140.0) // uM/ms
#endif

#ifdef LQT2
	#define	gKr 0
#else
	#define	gKr 0.024  // mS/uF, IKs conductance
#endif

// ryr gating
#define nryr 	84		//Number of Ryr channels
// #define f_Jmax 	18.0	//Jmax prefactor
#define taub 1.0 	    // ms, transition rate from CSQN-unbound states to CSQN-bound states
#define taucu 2.5    // ms, transition rate from open-unbound state to closed-unbound state
#define taucb 2.5    // ms, transition rate from open-bound state to closed-bound state
#define corbularSR_percentage	0.05
#define Spark_Threshold 3000.0 // uM/ms, when RyR release flux is larger than this, it is a spark.
#define Sparks_Interval 100.0 // ms, minimum time interval between sparks

// luminal gating
#define nCa 31.0 // number of Ca2+ binding sites of each CSQN molecule
#define BCSQN	460.0 //uM, concentration of CSQN
#define Kc 600.0 //uM, Dissociation constant of CSQN

// LCC ica
#define Pca 11.9	// umol/C/ms, 11.9: Restrepo 2008
#define NLCC 5	 // number of LCC channels in each dyadic space
#define gammai 0.341 // Activity coefficient of Ca2+

// NCX
#define	NaO 	136.0 // mM, [Na+]o
#define Kmcai 	0.00359 // uM
#define Kmcao 	1.3 // mM
#define Kmnai 	12.3 // mM
#define Kmnao 	87.5 // mM
#define eta		0.35
#define ksat	0.27

// other
#define CaO 1.8		//mM, external Ca2+ concentration
#define KI 	140.0	//mM, internal K+ concentration
#define KO 	5.40	//mM, external K+ concentration
#define Ek  ( (1.00/FRT)*log(KO/KI) )	//mV
#define Cm 	45 // pF, Capacitance of the whole cell membrane
#define Faraday 96.485		//	C/mmol
#define RR	8.314			//	J/mol/K
#define Temperature	308		//	K
#define FRT (Faraday/RR/Temperature)
#define PI 	3.1415926

#define pos(x,y,z)		(Nx*Ny*(z)+Nx*(y)+(x))
#define posi(i,j,k)		(Nix*Niy*(k)+Nix*(j)+(i))	// position in a CRU
#define posall(i,j,k)	((k)*(Nx*Nix)*(Ny*Niy)+(j)*(Nx*Nix)+(i))	// whole cell position
#define posallf(i,j,k)	((k)*(Nx*Nix/2+1)*Ny*Niy+(j)*(Nx*Nix/2+1)+(i))// whole cell position in k space(FFT)

#define pow2(x) ((x)*(x))
#define pow3(x) ((x)*(x)*(x))
//////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////

#define ktson 	0.00254 // (uM*ms)^(-1), associate constant
#define ktsoff 	0.000033 // ms^(-1), dissociate constant
#define Bts 	134.0	//uM, Mg binding/unbinding does not occur very much. Bts = steady state free+cabound troponin.
#define ktfon 	0.0327
#define ktfoff 	0.0196 // Troponin fast
#define Btf 	70.0

#define kcalon 	0.0543 // Calmodulin
#define kcaloff 0.238
#define Bcal 	24.0

#define ksron 	0.1 // SR
#define ksroff 	0.06
#define Bsr 	47.0

#define ksaron 	0.1 // Sarcolemma
#define ksaroff 1.3
#define Bsar 	(42*(Vi*Nci/Vs)) // in Bers book, it is 42 uM/l cytosol

#define ksarhon 	0.1  // Membrane/High
#define ksarhoff 	0.03
#define Bsarh 		(15.0*(Vi*Nci/Vs))

#define Bmyo 	140.0 // Myosin
#define konmyomg 	0.0000157
#define koffmyomg 	0.000057
#define konmyoca 	0.0138
#define koffmyoca 	0.00046
#define Mgi 	1000.0 // Mg
#define Kmyomg 	(koffmyomg/konmyomg)
#define Kmyoca 	(koffmyoca/konmyoca)

#define katpon 	0.15 // ATP
#define katpoff 30.0
#define Batp 	0//727.0

#define kdyeon		0.08 // Dye
#define kdyeoff		0.09
#define Bdye		50.0

#define kegtaon 	0.0015 // EGTA
#define kegtaoff	0.0003
#define Begta		0//350.0

//////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////

// point spread function for fluoxt.txt
#define SX	(1.44/sqrt(8*log(2))) // sigma of the point spread function
#define SY	(0.47/sqrt(8*log(2)))
#define SZ	(0.47/sqrt(8*log(2)))
#define gx(i,ix,Ax) ( 1.0/sqrt(2*PI*SX*SX)*exp(-((i*Nix+ix)-(Ax))*((i*Nix+ix)-(Ax))*DX*DX/(2.0*SX*SX)) )
#define gy(j,jy,Ay)	( 1.0/sqrt(2*PI*SY*SY)*exp(-((j*Niy+jy)-(Ay))*((j*Niy+jy)-(Ay))*DY*DY/(2.0*SY*SY)) )
#define gz(k,kz,Az)	( 1.0/sqrt(2*PI*SZ*SZ)*exp(-((k*Niz+kz)-(Az))*((k*Niz+kz)-(Az))*DZ*DZ/(2.0*SZ*SZ))	)
#define G(i,j,k,ix,jy,kz,Ax,Ay,Az)	( gx(i,ix,Ax) * gy(j,jy,Ay) * gz(k,kz,Az) )

struct sl_bu{
	double casar; // uM, Ca bound Sarcolemma buffer concentration in submembrane space
	double casarh; // uM, Ca bound Membrane/High
	double caatp; // uM, Ca bound ATP
	double cadye; // uM, Ca bound Dye
	double caegta; // uM, Ca bound EGTA
	double caatpnext; // uM, Ca bound ATP
	double cadyenext; // uM, Ca bound Dye
	double caegtanext; // uM, Ca bound EGTA

	double casarj; // uM, Ca bound Sarcolemma in dyad
	double casarhj; // uM, Ca bound Membrane/High in dyad
	double caatpj; // uM, Ca bound ATP in dyad
	double cadyej; // uM, Ca bound Dye in dyad
	double caegtaj; // uM, Ca bound EGTA in dyad
	double caatpjnext; // uM, Ca bound ATP in dyad
	double cadyejnext; // uM, Ca bound Dye in dyad
	double caegtajnext; // uM, Ca bound EGTA in dyad
};

struct cyt_bu{ // cytosolic buffers
	double cacal;
	double catf;
	double cats;
	double casr;
	double camyo;
	double mgmyo;
	double caatp;
	double cadye;
	double caegta;
	double caatpnext;
	double cadyenext;
	double caegtanext;
};

struct cytosol{
	double Juptake; // uM/ms, SERCA uptake flux
	double ci; // uM
	double cinext;
	double cnsr;
	double cnsrnext;
};

struct cru{
	double JNCX; // uM/ms, NCX flux
	double JCa; // uM/ms, ICa flux
	double Jbg; // uM/ms, sarcolemma background current flux
	double cs;
	double csnext;
};

struct cru2{
	double cp;
	double cpnext;
	double cjsr;
	double Tcj; // uM, total Ca2+ in jSR

	int lcc[NLCC];
	int nLCC_open;

	double Jrel; // uM/ms, SR release flux via RyRs
	double Jleak; // uM/ms, leak flux from JSR to dyad
	int nou;
	int ncu;
	int nob;
	int ncb;

	double tauil;
	double tausl;

	double Ka;	// for Inaca

	double cjsrc; // corbular SR
	int nouc;
	int ncuc;
	int nobc;
	int ncbc;
	double Jrelc;
	double Tcjc;

	hiprandState state;
};

__global__ void	setup_kernel(unsigned long long seed,cru2 *CRU2);
__global__ void Initial( cru *CRU, cru2 *CRU2, cytosol *CYT, cyt_bu *CBU, sl_bu *SBU, double ci_b, double cj_b);
__global__ void Compute( cru *CRU, cru2 *CRU2, cytosol *CYT, cyt_bu *CBU, sl_bu *SBU, double v, int step, double nai, 
						double f_Jmax, double alpha, double beta, int Hill, double custar, double cbstar);
__global__ void Finish( cru *CRU, cru2 *CRU2, cytosol *CYT, cyt_bu *CBU, sl_bu *SBU );

__device__ int ryrgating (double cp, double cjsr, hiprandState *state, int *ncu, int *nou, int *ncb, int *nob, int i, int j, int k, int step,
							double f_Jmax, double alpha, double beta, int Hill, double custar, double cbstar);
__device__ int number_RyR_transit(hiprandState *state, int NN, double probability, int upBound);
__device__ int LCCgating(double v, double cp, hiprandState *state, int i );
__device__ double Single_LCC_Current(double v, double cp); // cp in mM
__device__ double ncx(double v, double cs, double nai, double *Ka);
__device__ double uptake(double ci, double cnsr);

double Ina( double v, double *hh, double *jj, double *mm, double nai );
double Ikr( double v, double *Xkr );
double Iks( double v, double *Xs1, double *Xs2, double *Qks, double cst, double nai );
double Ik1( double v );
double Itos(double v, double *Xtos, double *Ytos);
double Itof(double v, double *Xtof, double *Ytof);
double Inak( double v, double nai );
double sodium(double v, double nai, double I_Na, double I_NaK, double I_NCX);

#ifdef output_fluoxt
	int convolve(double*fluo, cyt_bu *CBU);
#endif

void matrix2file(cytosol *CYT, int step);

int main(int argc, char **argv)
{
	int CudaDevice = 0;	
	if( argc >= 1 ) 
		CudaDevice = atoi(argv[1]);
	hipSetDevice(CudaDevice);

	size_t ArraySize_cru = Nx*Ny*Nz*sizeof(cru);		// CRU
	size_t ArraySize_cru2= Nx*Ny*Nz*sizeof(cru2);
	size_t ArraySize_cyt = Nci*Nx*Ny*Nz*sizeof(cytosol);	// cytosol space, Nci=Nci
	size_t ArraySize_cbu = Nci*Nx*Ny*Nz*sizeof(cyt_bu);	// cytosol space for the buffers
	size_t ArraySize_sbu = Nx*Ny*Nz*sizeof(sl_bu);		// submembrane space for buffers
	size_t ArraySize_dos = Nx*Ny*Nz*sizeof(double);		// total size of submembrane lattices
	size_t ArraySize_dol = Nci*Nx*Ny*Nz*sizeof(double);	// total # of cytosol lattices

	// Allocate arrays memory in CPU 
	cru *h_CRU;
	cru2 *h_CRU2;
	cytosol *h_CYT;
	cyt_bu *h_CBU;
	sl_bu *h_SBU;
	double *spark_clock, *fluo;
	
	h_CRU = (cru*) malloc(ArraySize_cru);
	h_CRU2 = (cru2*) malloc(ArraySize_cru2);
	h_CYT = (cytosol*) malloc(ArraySize_cyt);
	h_CBU = (cyt_bu*) malloc(ArraySize_cbu);
	h_SBU = (sl_bu*) malloc(ArraySize_sbu);
	spark_clock = (double*) malloc(ArraySize_dos);
	fluo =		(double*) malloc(ArraySize_dol);
	
	//Allocate arrays in GPU
	cru *d_CRU;
	cru2 *d_CRU2;
	cytosol *d_CYT;
	cyt_bu *d_CBU;
	sl_bu *d_SBU;

	hipMalloc((void**)&d_CRU, ArraySize_cru);
	hipMalloc((void**)&d_CRU2,ArraySize_cru2);
	hipMalloc((void**)&d_CYT, ArraySize_cyt);
	hipMalloc((void**)&d_CBU, ArraySize_cbu);
	hipMalloc((void**)&d_SBU, ArraySize_sbu);

	/////////////////////////////////// variables /////////////////////////////////////////////////
	int step = 0;
	int i, j, k, ix; // i,j,k for CRU index; ix, jy, kz for lattices in each CRU
	double start_time = clock()/(1.0*CLOCKS_PER_SEC),    end_time;

	double nai = 10.0;//73.987/( 1.0 + 6.707*sqrt(PCL/1000.0) );
	double CaExt = 0, TotalCa = 0, TotalCa_before = 0;

	double v = -86.00;	// voltage
	double mm = 0.0010, hh = 1.00, jj = 1.00;	// INa 
	double Xkr = 0.0; // IKr
	double Xs1 = 0.084, Xs2 = Xs1, Qks = 0.2;	// IKs 
	double Xtos = 0.01, Ytos = 1.0;	// Itos
	double Xtof = 0.02, Ytof = 0.8;	// Itof
	double I_NaK = 0, I_Na = 0, I_Kr = 0, I_Ks = 0, I_K1 = 0, I_tos = 0, I_tof = 0, I_Ca = 0, I_NCX = 0, I_bg = 0;

	double cit, cpt, cst, cjsrt, cnsrt ;
	int Nxyz = (Nx-2)*(Ny-2)*(Nz-2), NcorSR = 0;
	
	/////// to calculate spark rate
	double num_spark = 0, spark_rate = 0, spark_rate_history[20] = {0.0};
	for (k=0;k<Nz;k++)
	{
		for (j=0;j<Ny;j++)
		{
			for (i=0;i<Nx;i++)
			{
				spark_clock[pos(i,j,k)] = Sparks_Interval;
			}
		}
	}

	///////////////////////////////////////////// files /////////////////////////////////////////////////

	FILE * wholecell_file = fopen("wholecell.txt","w");
	
	#ifdef output_linescan
		FILE * linescan_file = fopen("linescan.txt","w");
	#endif

	#ifdef output_fluoxt
		FILE * fluoxt_file = fopen("fluoxt.txt","w");
	#endif

	////////////////////////////////////////////////////////////////////////////////////////////////////	
	// Set paramaters for geometry of computation
	dim3 threadsPerBlock(BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z);
	dim3 numBlocks(Nx/threadsPerBlock.x, Ny/threadsPerBlock.y, Nz/threadsPerBlock.z);

	setup_kernel<<<numBlocks, threadsPerBlock>>>(98,d_CRU2);
	Initial<<<numBlocks, threadsPerBlock>>>(d_CRU, d_CRU2, d_CYT, d_CBU, d_SBU, ci_basal, cjsr_basal);
	hipMemcpy(h_CRU, d_CRU, ArraySize_cru, hipMemcpyDeviceToHost);
	hipMemcpy(h_CRU2, d_CRU2, ArraySize_cru2, hipMemcpyDeviceToHost);
	for (k = 1; k < Nz-1; k++)
	{
		for (j = 1; j < Ny-1; j++)
		{
			for (i = 1; i < Nx-1; i++) 
			{
				if (h_CRU2[pos(i,j,k)].cjsrc>0)
					++NcorSR;
			}
		}
	}
	printf("NcorSR = %d\n", NcorSR);

	while ( step*DT < stoptime )
	{
		hipMemcpy(h_CRU, d_CRU, ArraySize_cru, hipMemcpyDeviceToHost);

		//////////////////////////////// whole cell average: cst, Ica, Inaca, Ibcg ///////////////////////
		
		if ( step%out_step==1 )
		{
			CaExt=0;	// total Ca2+ exchange through the cell membrane
		}

		cst = 0;
		I_Ca = 0;
		I_bg = 0;
		I_NCX = 0;
		
		for (k = 1; k < Nz-1; k++)
		{
			for (j = 1; j < Ny-1; j++)
			{
				for (i = 1; i < Nx-1; i++) 
				{
					cst += h_CRU[pos(i,j,k)].cs;
					I_Ca += h_CRU[pos(i,j,k)].JCa;
					I_bg += h_CRU[pos(i,j,k)].Jbg;
					I_NCX += h_CRU[pos(i,j,k)].JNCX;

					CaExt = CaExt - h_CRU[pos(i,j,k)].JCa*Vp*DT/Nxyz 
							+ h_CRU[pos(i,j,k)].JNCX*Vs*DT/Nxyz;
				}
			}
		}
		cst=cst/Nxyz;

		// Firstly, I convert uM/ms to pA, then I make it divided by capacitance (Cm, pF)
		// So the unit of I_Ca is pA/pF, equal to mV/ms, which is the unit of dv/dt
		I_Ca = I_Ca*0.0965*Vp*2.0 / Cm;
		I_NCX = I_NCX*0.0965*Vs / Cm;
		I_bg = I_bg*0.0965*Vp*2.0 / Cm;

		//////////////////// other ion channels ///////////////////
		I_Na = Ina(v, &hh, &jj, &mm, nai);
		I_Kr = Ikr(v, &Xkr);
		I_Ks = Iks(v, &Xs1, &Xs2, &Qks, cst, nai );
		I_K1 = Ik1(v);
		I_tos = Itos(v, &Xtos, &Ytos);
		I_tof = Itof(v, &Xtof, &Ytof);
		I_NaK = Inak(v, nai);
		//nai = sodium(v, nai, I_Na, I_NaK, I_NCX);

		///////////////////////////	Action Potential ///////////////////////////////////
		double stim = 0;
		if( fmod(step*DT+PCL-time_before_beat,PCL) < 1.0 && step*DT > time_before_beat && step*DT < PCL*stopbeat )
			stim = 80.0;
		double dvh = -( I_Na + I_K1 + I_Kr + I_Ks + I_tos + I_tof + I_NCX + I_Ca + I_NaK + I_bg ) + stim; 
		v += dvh*DT;

		#ifdef Permeabilized
			v = -86;
		#endif
	
		#ifdef Vclamp
			v = -86;
			if( step*DT > time_before_beat && step*DT < time_before_beat+200 )
				v = 0;
		#endif

		#ifdef APclamp
			v = varray[((int)( t/DT+0.1 ))%((int)(PCL/DT+0.1))];
		#endif
		
		///////////////////////////////////////////////////////////////////////////////
		//////////////////////////////// output ///////////////////////////////////////
		///////////////////////////////////////////////////////////////////////////////
		if ( step%out_step==0 )
		{
			hipMemcpy(h_CRU2,d_CRU2,ArraySize_cru2,hipMemcpyDeviceToHost);
			hipMemcpy(h_CYT, d_CYT, ArraySize_cyt, hipMemcpyDeviceToHost);
			hipMemcpy(h_CBU, d_CBU, ArraySize_cbu, hipMemcpyDeviceToHost);
			hipMemcpy(h_SBU, d_SBU, ArraySize_sbu, hipMemcpyDeviceToHost);

		//	matrix2file(h_CYT, step);

			cit = 0;
			cpt = 0;
			cjsrt = 0;
			cnsrt = 0;
			TotalCa = 0;

			double catft=0, catst=0, casrt=0, camyot=0, mgmyot=0, cacalt=0, caatpt=0, cadyet=0, caegtat=0;
			double casart = 0, casarht = 0, caatpst = 0, cadyest = 0, caegtast =0;
			double casarjt = 0, casarhjt = 0, caatpjt = 0, cadyejt = 0, caegtajt =0;
			double Jleakt=0, Juptaket=0, Jrelt=0;
			int nout = 0, nobt = 0, ncut = 0, ncbt = 0;
			double icaflux = 0, ncxflux = 0, ncxfwd = 0, Kat = 0;
			double cjt=0;	// total Ca2+ in jSR
			double nlcc_open = 0, nlcc_flip = 0, nlcc_act = 0, nlcc_v = 0, nlcc_Ca = 0;

			int ps = 0;
			for (k = 1; k < Nz-1; k++)
			{
				for (j = 1; j < Ny-1; j++)
				{
					for (i = 1; i < Nx-1; i++) 
					{	
						ps=pos(i,j,k);

						if ( h_CRU[ps].JNCX < 0 )
							ncxfwd += h_CRU[ps].JNCX;

						icaflux += h_CRU[ps].JCa;
						ncxflux += h_CRU[ps].JNCX;
						Kat += h_CRU2[ps].Ka;
						cpt += h_CRU2[ps].cp;
						cjsrt += ( h_CRU2[ps].cjsr + ( (h_CRU2[ps].cjsrc>0)?(h_CRU2[ps].cjsrc):0 ) );
						cjt += h_CRU2[ps].Tcj + h_CRU2[ps].Tcjc;
						Jrelt += h_CRU2[ps].Jrel;
						Jleakt += h_CRU2[ps].Jleak;
						nout += h_CRU2[ps].nou;
						nobt += h_CRU2[ps].nob;
						ncut += h_CRU2[ps].ncu;
						ncbt += h_CRU2[ps].ncb;

						casart += h_SBU[ps].casar;
						casarht += h_SBU[ps].casarh;
						caatpst += h_SBU[ps].caatp;
						cadyest += h_SBU[ps].cadye;
						caegtast+= h_SBU[ps].caegta;
						casarjt += h_SBU[ps].casarj;
						casarhjt += h_SBU[ps].casarhj;
						caatpjt += h_SBU[ps].caatpj;
						cadyejt += h_SBU[ps].cadyej;
						caegtajt += h_SBU[ps].caegtaj;

						for ( ix = 0; ix < Nci; ++ix )
						{
							cit += h_CYT[ps*Nci+ix].ci/Nci;
							cnsrt += h_CYT[ps*Nci+ix].cnsr/Nci;
							catft += h_CBU[ps*Nci+ix].catf/Nci;
							catst += h_CBU[ps*Nci+ix].cats/Nci;
							casrt += h_CBU[ps*Nci+ix].casr/Nci;
							camyot += h_CBU[ps*Nci+ix].camyo/Nci;
							mgmyot += h_CBU[ps*Nci+ix].mgmyo/Nci;
							cacalt += h_CBU[ps*Nci+ix].cacal/Nci;
							caatpt += h_CBU[ps*Nci+ix].caatp/Nci;
							cadyet += h_CBU[ps*Nci+ix].cadye/Nci;
							caegtat+= h_CBU[ps*Nci+ix].caegta/Nci;
							Juptaket += h_CYT[ps*Nci+ix].Juptake/Nci;
							if( h_CYT[ps*Nci+ix].ci > 50.0 )
							{
								cout << step*DT << " " << i << " " << j << " " << k << " " 
									 << ix << " error! ci=" << h_CYT[ps*Nci+ix].ci << endl;
							}
						}

						nlcc_open += h_CRU2[ps].nLCC_open;
						for( int ll = 0; ll < NLCC; ll++ )
						{
							switch ( h_CRU2[ps].lcc[ll] )
							{
								case 1: ++nlcc_flip; break;
								case 2: ++nlcc_act; break;
								case 3: ++nlcc_flip; ++nlcc_act; break;
								case 4: ++nlcc_v; break;
								case 5: ++nlcc_flip; ++nlcc_v; break;
								case 6: ++nlcc_act; ++nlcc_v; break;
								case 7: ++nlcc_flip; ++nlcc_act; ++nlcc_v; break;
								case 8: ++nlcc_Ca; break;
								case 9: ++nlcc_flip; ++nlcc_Ca; break;
								case 10: ++nlcc_act; ++nlcc_Ca; break;
								case 11: ++nlcc_flip; ++nlcc_act; ++nlcc_Ca; break;
								case 12: ++nlcc_v; ++nlcc_Ca; break;
								case 13: ++nlcc_flip; ++nlcc_v; ++nlcc_Ca; break;
								case 14: ++nlcc_act; ++nlcc_v; ++nlcc_Ca; break;
								case 15: ++nlcc_flip; ++nlcc_act; ++nlcc_v; ++nlcc_Ca; break;
							}
						}
						
						
					}
				}
			}
			
			cit /= Nxyz;
			cpt /= Nxyz;
			cjsrt /= (Nxyz+NcorSR);
			cjt /= Nxyz;
			cnsrt /= Nxyz;
			catft /= Nxyz;
			catst /= Nxyz;
			casrt /= Nxyz;
			camyot /= Nxyz;
			mgmyot /= Nxyz;
			cacalt /= Nxyz;
			cadyet /= Nxyz;
			caatpt /= Nxyz;
			caegtat /= Nxyz;
			Jleakt /= Nxyz;
			Juptaket /= Nxyz;
			Jrelt /= Nxyz;
			ncxflux /= Nxyz;
			ncxfwd /= Nxyz;
			icaflux /= Nxyz;
			Kat /= Nxyz;
			casart /= Nxyz;
			casarht /= Nxyz;
			caatpst /= Nxyz;
			cadyest /= Nxyz;
			caegtast/= Nxyz;
			casarjt /= Nxyz;
			casarhjt /= Nxyz;
			caatpjt /= Nxyz;
			cadyejt /= Nxyz;
			caegtajt /= Nxyz;
			nlcc_open /= Nxyz;
			nlcc_flip /= Nxyz;
			nlcc_act /= Nxyz;
			nlcc_v /= Nxyz;
			nlcc_Ca /= Nxyz;
			
			TotalCa =	( cit+ catft + catst + casrt + camyot + cacalt + caatpt + cadyet + caegtat )*Vi*Nci 
						+ ( cst + casart + casarht + caatpst + cadyest + caegtast )*Vs 
						+ ( cpt + casarjt + casarhjt + caatpjt + cadyejt + caegtajt )*Vp 
						+ cjt*Vjsr
						+ cnsrt*Vnsr*Nci;

			//////////////////////////////////// spark rate /////////////////////////////////
			num_spark = 0.0;
			for (k=0;k<Nz;k++)
			{
				for (j=0;j<Ny;j++)
				{
					for (i=0;i<Nx;i++)
					{
						if (h_CRU2[pos(i,j,k)].Jrel>Spark_Threshold && spark_clock[pos(i,j,k)]>Sparks_Interval)
						{
							num_spark = num_spark + 1.0;
							spark_clock[pos(i,j,k)] = 0.0;
						}
						spark_clock[pos(i,j,k)] += out_step*DT;
					}
				}
			}
			

			// output spark rate for every 20 ms. Within 20 ms, all the spark rate values are equal to the average.
			// Time shifted: 20ms. Example: in range [20ms, 39ms], all values are equal to the average value in the ranage [0,19ms]
			spark_rate_history[(step/out_step)%20] = num_spark*200.0/1.8/((Nx-2)*(Ny-2)*(Nz-2))/(out_step*DT/1000.0);

			if ( step%(20*out_step)==0 )
			{
				spark_rate = 0;

				for (i=0;i<20;i++)
					spark_rate += spark_rate_history[i];

				spark_rate = spark_rate/20.0;
			}

			////////////////////////////// output to screen /////////////////////////////
			if ( step%(100*out_step)==0 )
			{
				end_time=clock()/(1.0*CLOCKS_PER_SEC);	
				printf(	"t=%g\t/ %g\t\ttime = %.1fs = %.1fh\t\tcit = %g\t\tcjsrt = %g\n",
						step*DT, stoptime, 
						end_time-start_time, (end_time-start_time)/3600.0, 
						cit, cjsrt
					  );
			}

			////////////////////////////// whole cell ////////////////////////////////// flag
			fprintf(wholecell_file,	"%g %g %g %g %g " "%g %g %g %g %g "
									"%g %g %g %g %g " "%g %g %g %g %g "
									"%g %g %g %g %g " "%g %g %g %g %g "
									"%g %g %g %g %g " "%g %g %g %g\n",

									step*DT, cit,
									cpt, cst, 
									cjsrt, cnsrt, 
									v, I_NCX,
									I_Ca, Juptaket, 

									nai, I_Ks,
									I_Kr, I_K1,
									I_NaK, I_tos,
									I_tof, I_Na,
									Jleakt, Jrelt, 

									nout/(1.0*Nxyz), nobt/(1.0*Nxyz),
									ncut/(1.0*Nxyz), ncbt/(1.0*Nxyz),
									ncxflux*(Vs/Vp), icaflux, 
									I_bg, Kat, 
									TotalCa - TotalCa_before, CaExt, 

									nlcc_open, nlcc_flip,
									nlcc_act, nlcc_v,
									nlcc_Ca, spark_rate,
									cadyet, caatpt,
									TotalCa
					);

			fflush( wholecell_file );
			TotalCa_before = TotalCa;

			////////////////////////////// Line Scan ////////////////////////////////////	flag
			#ifdef output_linescan
				// if ( step*DT > ( stopbeat - 2 )*PCL && step*DT < ( stopbeat + 2 )*PCL )
				// if ( step*DT>12000 && step*DT<25000 )
				{
					for (i =1; i < Nx-1; i++)
					{
						int k = 4, j = Ny/2;
						ps = pos(i,j,k);
						fprintf(linescan_file, 	"%g %g %g %g %g " "%g %g %g %g %g "
												"%g %g %i %i %i " "%i %i %g %g\n",
												
												step*DT,			(double)i,
												h_CYT[ps*Nci].ci, 	h_CRU2[ps].cp,
												h_CRU[ps].cs, 		h_CRU2[ps].cjsr,
												h_CYT[ps*Nci].cnsr,	h_CRU2[ps].Jrel,
												h_CYT[ps*Nci].Juptake,	h_CRU2[ps].Jleak,

												h_CRU[ps].JCa, 		h_CRU[ps].JNCX,
												h_CRU2[ps].nou, 	h_CRU2[ps].nob, 
												h_CRU2[ps].ncu, 	h_CRU2[ps].ncb, 
												h_CRU2[ps].nLCC_open,  h_CRU[ps].Jbg,
												h_CRU2[ps].cjsrc
											 
							);
					}
					fprintf(linescan_file, "\n");
					fflush(linescan_file);
				}
			#endif
			
			///////////////////////////////////////// fluoxt ////////////////////////////////////
			#ifdef output_fluoxt
				if ( step*DT>12000 && step*DT<25000 )
				{
					convolve(fluo, h_CBU);
					for (i=0;i<Nx;i++)
					{
						for ( ix=0;ix<Nix;ix++)
						{
							fprintf(fluoxt_file,"%g\t%g\t%g\t%g\t%g\n",
									
									step*DT,	(i*Nix+ix)*DX, 
									fluo[posall(i*Nix+ix, Ny/2*Niy+0, 4*Niz+0)], fluo[posall(i*Nix+ix,(Ny/2-3)*Niy+0,6*Niz+0)], 
									fluo[posall(i*Nix+ix,(Ny/2+5)*Niy+0,6*Niz+0)]
									
									);
						}
					}
				
					fprintf(fluoxt_file,"\n");
					fflush( fluoxt_file );
				}
			#endif
			
		}

		double f_Jmax = atof(argv[4]);
		int Hill = atoi(argv[5]);
		double alpha = atof(argv[6]);
		double beta = atof(argv[7]);
		double custar = atof(argv[8]);
		double cbstar = atof(argv[9]);

		Compute<<<numBlocks, threadsPerBlock>>>( d_CRU, d_CRU2, d_CYT, d_CBU, d_SBU, v, step, nai, f_Jmax, alpha, beta, Hill, custar, cbstar);
		Finish<<<numBlocks, threadsPerBlock>>>( d_CRU, d_CRU2, d_CYT, d_CBU, d_SBU);
		step++;
	}

	fclose(wholecell_file);
	
	#ifdef output_linescan
		fclose(linescan_file);
	#endif
	
	#ifdef output_fluoxt
		fclose(fluoxt_file);
	#endif


	hipFree(d_CYT);
	hipFree(d_CRU);
	hipFree(d_CRU2);
	hipFree(d_SBU);
	hipFree(d_CBU);
	
	free(h_CYT);
	free(h_CRU);
	free(h_CRU2);
	free(h_SBU);
	free(h_CBU);
	free(fluo);
	free(spark_clock);
	
	return EXIT_SUCCESS;
}

__global__ void Initial( cru *CRU, cru2 *CRU2, cytosol *CYT, cyt_bu *CBU, sl_bu *SBU, double ci_b, double cj_b)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int k = threadIdx.z + blockIdx.z * blockDim.z;
	int ps = pos(i,j,k);

	hiprandState localState;
	localState=CRU2[ps].state;
	
	for ( int ix = 0; ix < Nci; ++ix )
	{
		int psi = ps*Nci+ix;
		CYT[psi].ci = ci_b;
		CYT[psi].cnsr = cj_b;
		CYT[psi].cinext = ci_b;
		CYT[psi].cnsrnext = cj_b;

		CBU[psi].catf = ktfon*ci_b*Btf/(ktfon*ci_b+ktfoff);
		CBU[psi].cats = ktson*ci_b*Bts/(ktson*ci_b+ktsoff);
		CBU[psi].cacal= kcalon*ci_b*Bcal/(kcalon*ci_b+kcaloff);
		CBU[psi].casr = ksron*ci_b*Bsr/(ksron*ci_b+ksroff);
		CBU[psi].caatp = katpon*ci_b*Batp/(katpon*ci_b+katpoff);
		CBU[psi].caatpnext = katpon*ci_b*Batp/(katpon*ci_b+katpoff);
		CBU[psi].cadye = kdyeon*ci_b*Bdye/(kdyeon*ci_b+kdyeoff);
		CBU[psi].cadyenext = kdyeon*ci_b*Bdye/(kdyeon*ci_b+kdyeoff);
		CBU[psi].caegta = kegtaon*ci_b*Begta/(kegtaon*ci_b+kegtaoff);
		CBU[psi].caegtanext = kegtaon*ci_b*Begta/(kegtaon*ci_b+kegtaoff);
	
		double ratio = Mgi*Kmyoca/(ci_b*Kmyomg);
		CBU[psi].camyo = ci_b*Bmyo/(Kmyoca+ci_b*(ratio+1.0));
		CBU[psi].mgmyo = CBU[psi].camyo*ratio;
	}
	
	SBU[ps].casar = ksaron*ci_b*Bsar/(ksaron*ci_b+ksaroff);
	SBU[ps].casarh = ksarhon*ci_b*Bsarh/(ksarhon*ci_b+ksarhoff);
	SBU[ps].caatp = katpon*ci_b*Batp/(katpon*ci_b+katpoff);	
	SBU[ps].cadye = kdyeon*ci_b*Bdye/(kdyeon*ci_b+kdyeoff);	
	SBU[ps].caegta= kegtaon*ci_b*Begta/(kegtaon*ci_b+kegtaoff);
	SBU[ps].caatpnext = katpon*ci_b*Batp/(katpon*ci_b+katpoff);	
	SBU[ps].cadyenext = kdyeon*ci_b*Bdye/(kdyeon*ci_b+kdyeoff);	
	SBU[ps].caegtanext= kegtaon*ci_b*Begta/(kegtaon*ci_b+kegtaoff);

	SBU[ps].casarj = ksaron*ci_b*Bsar/(ksaron*ci_b+ksaroff);
	SBU[ps].casarhj= ksarhon*ci_b*Bsarh/(ksarhon*ci_b+ksarhoff);
	SBU[ps].caatpj = katpon*ci_b*Batp/(katpon*ci_b+katpoff);
	SBU[ps].cadyej = kdyeon*ci_b*Bdye/(kdyeon*ci_b+kdyeoff);
	SBU[ps].caegtaj= kegtaon*ci_b*Begta/(kegtaon*ci_b+kegtaoff);
	SBU[ps].caatpjnext = katpon*ci_b*Batp/(katpon*ci_b+katpoff);
	SBU[ps].cadyejnext = kdyeon*ci_b*Bdye/(kdyeon*ci_b+kdyeoff);
	SBU[ps].caegtajnext= kegtaon*ci_b*Begta/(kegtaon*ci_b+kegtaoff);

	CRU[ps].cs = ci_b;
	CRU[ps].csnext = ci_b;
	CRU2[ps].cp = ci_b;
	CRU2[ps].cpnext = ci_b;
	CRU2[ps].cjsr = cj_b;
	CRU2[ps].Tcj = cj_b + BCSQN*nCa*cj_b/( Kc+cj_b );

	CRU[ps].Jbg= 0;
	CRU[ps].JCa = 0;
	CRU[ps].JNCX = 0;
	CRU2[ps].Jrel = 0;

	for(int ll=0; ll<NLCC; ll++)
	{
		CRU2[ps].lcc[ll]=3;
	}

	double cb=BCSQN*nCa*CRU2[ps].cjsr/(Kc+CRU2[ps].cjsr);
	double ku2b = 1.0/( 1.0+pow(cb/BCSQN/13.3, 24) )/taub;
	double kb2u = 1.0/(  4000.0/( 1.0+pow(CRU2[ps].cjsr/670.0, 24) ) + 350.0  );
	double fracbound = 1/(1+kb2u/ku2b);

	double nryr0 = nryr + (hiprand_uniform_double(&localState)-0.5)*41;
	if (nryr0 < 1)
		nryr0 = 1;
	CRU2[ps].nLCC_open = 0;
	CRU2[ps].ncb = int(fracbound*nryr);
	CRU2[ps].ncu = nryr-int(fracbound*nryr);
	CRU2[ps].nob = 0;
	CRU2[ps].nou = 0;

	CRU2[ps].Ka = 0.025;

	if ( hiprand_uniform_double(&localState) < corbularSR_percentage )
		CRU2[ps].cjsrc = cj_b;
	else
		CRU2[ps].cjsrc = -1;

	CRU2[ps].ncbc = CRU2[ps].ncb;
	CRU2[ps].ncuc = CRU2[ps].ncu;
	CRU2[ps].nobc = CRU2[ps].nob;
	CRU2[ps].nouc = CRU2[ps].nou;

	if ( k>=3 && k<=7 && ( (j==8 && i>=35 && i<=60) || (j==16 && i>=5 && i<=30) || (j==24 && i>=20 && i<=45)  ) )
	{
		CRU2[ps].tauil = 2.269;
		CRU2[ps].tausl = 1000000.0;
	}
	else 
	{
		CRU2[ps].tauil = 2.269;
		CRU2[ps].tausl = 1000000.0; // remove diffusion
	}

	CRU2[ps].state = localState;
}


#define FINESTEP 10
#define DTF 	(DT/FINESTEP)

__global__ void Compute( cru *CRU, cru2 *CRU2, cytosol *CYT, cyt_bu *CBU, sl_bu *SBU, double v, int step, double nai,
						double f_Jmax, double alpha, double beta, int Hill, double custar, double cbstar)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int k = threadIdx.z + blockIdx.z * blockDim.z;
	int ps = pos(i,j,k);

	hiprandState localState;
	localState=CRU2[ps].state;

	if ((i*j*k)!=0 && i<Nx-1 && j<Ny-1 && k<Nz-1)
	{
		//////////////////////////////////////////////////////////////////////
		/////////////////////////////////// ICa //////////////////////////////
		//////////////////////////////////////////////////////////////////////
		#ifndef Permeabilized
		{
			CRU2[ps].nLCC_open = 0;
			for (int  LCC_ichannel=0; LCC_ichannel<NLCC; LCC_ichannel++ )
			{
				CRU2[ps].lcc[LCC_ichannel] = LCCgating(v, CRU2[ps].cp, &localState, CRU2[ps].lcc[LCC_ichannel]);
				if ( CRU2[ps].lcc[LCC_ichannel] == 0 )
				{
					CRU2[ps].nLCC_open++;
				}
			}

			double ica = Single_LCC_Current(v, CRU2[ps].cp/1000.0 );
			
			CRU[ps].JCa = CRU2[ps].nLCC_open * ica;
		}
		#endif

		//////////////////////////////////////////////////////////////////////
		/////////////////////////////// INCX ////////////////////////////////
		//////////////////////////////////////////////////////////////////////
		#ifndef Permeabilized
			CRU[ps].JNCX = ncx( v, CRU[ps].cs, nai, &CRU2[ps].Ka );
		#endif

		//////////////////////////////////////////////////////////////////////
		/////////////////////////////// RyR //////////////////////////////////
		//////////////////////////////////////////////////////////////////////
		int Nryr_Open = ryrgating( CRU2[ps].cp, CRU2[ps].cjsr, &localState, &CRU2[ps].ncu, &CRU2[ps].nou, 
								&CRU2[ps].ncb, &CRU2[ps].nob, i, j, k, step, f_Jmax, alpha, beta, Hill, custar, cbstar );
		CRU2[ps].Jrel = Nryr_Open * f_Jmax*0.00015 * (CRU2[ps].cjsr-CRU2[ps].cp)/Vp;
		
		if ( CRU2[ps].cjsrc > 0 )
		{
			int Nryr_Open_c = ryrgating( CYT[ps*Nci+1].ci, CRU2[ps].cjsrc, &localState, &CRU2[ps].ncuc, &CRU2[ps].nouc, 
									&CRU2[ps].ncbc, &CRU2[ps].nobc, i, j, k, step, f_Jmax, alpha, beta, Hill, custar, cbstar );
			CRU2[ps].Jrelc = Nryr_Open_c * f_Jmax*0.00015 * (CRU2[ps].cjsrc-CYT[ps*Nci+1].ci)/(Vi/Nci);
		}
		else
			CRU2[ps].Jrelc = 0;

		CRU2[ps].Jleak = Vleak * (CRU2[ps].cjsr - CRU2[ps].cp); // Shannon et al 2004, Eq.107
		//////////////////////////////////////////////////////////////////////
		/////////////////////// other currents ////////////////////////////
		//////////////////////////////////////////////////////////////////////

		CRU[ps].Jbg = 0; // 0.005*(v-log(CaO*1000/CRU[ps].cs)/2.0/FRT);

		//////////////////////////////////////////////////////////////////////
		//////////////////////////////////////////////////////////////////////		
		//////////////////////////////////////////////////////////////////////
		double diffjn0 = (CRU2[ps].cjsr-CYT[ps*Nci].cnsr)/(tautr*2.0);
		double diffjn1 = (CRU2[ps].cjsr-CYT[ps*Nci+4].cnsr)/(tautr*2.0);
		
		double diffjn1c = (CRU2[ps].cjsrc-CYT[ps*Nci+1].cnsr)/(tautr*2.0);
		double diffjn5c = (CRU2[ps].cjsrc-CYT[ps*Nci+5].cnsr)/(tautr*2.0);
		if ( CRU2[ps].cjsrc < 0 )
		{
			diffjn1c = 0;
			diffjn5c = 0;
		}

		double diffpi0 = (CRU2[ps].cp-CYT[ps*Nci].ci)/(taupi*2.0);
		double diffpi1 = (CRU2[ps].cp-CYT[ps*Nci+4].ci)/(taupi*2.0);
		double diffsi0 = (CRU[ps].cs-CYT[ps*Nci].ci)/(tausi*2.0);
		double diffsi1 = (CRU[ps].cs-CYT[ps*Nci+4].ci)/(tausi*2.0);

		double diffpiatp0 = (SBU[ps].caatpj-CBU[ps*Nci].caatp)/(Datp*taupi*2.0);
		double diffpiatp1 = (SBU[ps].caatpj-CBU[ps*Nci+4].caatp)/(Datp*taupi*2.0);
		double diffpidye0 = (SBU[ps].cadyej-CBU[ps*Nci].cadye)/(Ddye*taupi*2.0);
		double diffpidye1 = (SBU[ps].cadyej-CBU[ps*Nci+4].cadye)/(Ddye*taupi*2.0);
		double diffpiegta0 = (SBU[ps].caegtaj-CBU[ps*Nci].caegta)/(Degta*taupi*2.0);
		double diffpiegta1 = (SBU[ps].caegtaj-CBU[ps*Nci+4].caegta)/(Degta*taupi*2.0);

		double diffsiatp0 = (SBU[ps].caatp-CBU[ps*Nci].caatp)/(Datp*tausi*2.0);
		double diffsiatp1 = (SBU[ps].caatp-CBU[ps*Nci+4].caatp)/(Datp*tausi*2.0);
		double diffsidye0 = (SBU[ps].cadye-CBU[ps*Nci].cadye)/(Ddye*tausi*2.0);
		double diffsidye1 = (SBU[ps].cadye-CBU[ps*Nci+4].cadye)/(Ddye*tausi*2.0);
		double diffsiegta0 = (SBU[ps].caegta-CBU[ps*Nci].caegta)/(Degta*tausi*2.0);
		double diffsiegta1 = (SBU[ps].caegta-CBU[ps*Nci+4].caegta)/(Degta*tausi*2.0);
		

		/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		////////////////////////////////////////////////////// dotci ////////////////////////////////////////////////////////
		/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		for ( int kz=0; kz<Niz; kz++ )
		{
			for ( int jy=0; jy<Niy; jy++ )
			{
				for ( int ix=0; ix<Nix; ix++ )
				{
					int psi = ps*Nci + posi(ix,jy,kz);
					int crui = posi(ix,jy,kz);

					CYT[psi].Juptake = uptake(CYT[psi].ci, CYT[psi].cnsr);

					double bufftf = ktfon*CYT[psi].ci*(Btf-CBU[psi].catf) - ktfoff*CBU[psi].catf;
					double buffts = ktson*CYT[psi].ci*(Bts-CBU[psi].cats) - ktsoff*CBU[psi].cats;
					double buffcal = kcalon*CYT[psi].ci*(Bcal-CBU[psi].cacal) - kcaloff*CBU[psi].cacal;
					double buffsr = ksron*CYT[psi].ci*(Bsr-CBU[psi].casr) - ksroff*CBU[psi].casr;
					double buffmyo = konmyoca*CYT[psi].ci*(Bmyo-CBU[psi].camyo-CBU[psi].mgmyo)-koffmyoca*CBU[psi].camyo;
					double buffmyomg = konmyomg*Mgi*(Bmyo-CBU[psi].camyo-CBU[psi].mgmyo)-koffmyomg*CBU[psi].mgmyo;
					double buffdye = kdyeon*CYT[psi].ci*(Bdye-CBU[psi].cadye) - kdyeoff*CBU[psi].cadye;
					double buffegta = kegtaon*CYT[psi].ci*(Begta-CBU[psi].caegta) - kegtaoff*CBU[psi].caegta;

					int inext =	  (ix==Nix-1)?( pos(i+1,j,k)*Nci+posi(0,jy,kz) ):( ps*Nci+posi(ix+1,jy,kz) );
					int ibefore = (ix==0)?( pos(i-1,j,k)*Nci+posi(Nix-1,jy,kz) ):( ps*Nci+posi(ix-1,jy,kz) );
					int jnext =	  (jy==Niy-1)?( pos(i,j+1,k)*Nci+posi(ix,0,kz) ):( ps*Nci+posi(ix,jy+1,kz) );
					int jbefore = (jy==0)?( pos(i,j-1,k)*Nci+posi(ix,Niy-1,kz) ):( ps*Nci+posi(ix,jy-1,kz) );
					int knext =   (kz==Niz-1)?( pos(i,j,k+1)*Nci+posi(ix,jy,0) ):( ps*Nci+posi(ix,jy,kz+1) );
					int kbefore = (kz==0)?( pos(i,j,k-1)*Nci+posi(ix,jy,Niz-1) ):( ps*Nci+posi(ix,jy,kz-1) );

					double coupleci =  	(CYT[knext].ci-CYT[psi].ci)/(tauit) +
										(CYT[kbefore].ci-CYT[psi].ci)/(tauit) +
										(CYT[jnext].ci-CYT[psi].ci)/(tauit) +
										(CYT[jbefore].ci-CYT[psi].ci)/(tauit) +
										(CYT[inext].ci-CYT[psi].ci)/(CRU2[ps].tauil) +
										(CYT[ibefore].ci-CYT[psi].ci)/(CRU2[pos(i-1,j,k)].tauil);

					double couplecnsr = (CYT[knext].cnsr-CYT[psi].cnsr)/(taunt) +
										(CYT[kbefore].cnsr-CYT[psi].cnsr)/(taunt) +
										(CYT[jnext].cnsr-CYT[psi].cnsr)/(taunt) +
										(CYT[jbefore].cnsr-CYT[psi].cnsr)/(taunt) +
										(CYT[inext].cnsr-CYT[psi].cnsr)/(taunl) +
										(CYT[ibefore].cnsr-CYT[psi].cnsr)/(taunl) ;

					double coupleATP = 	(CBU[knext].caatp-CBU[psi].caatp)/(Datp*tauit) +
										(CBU[kbefore].caatp-CBU[psi].caatp)/(Datp*tauit) +
										(CBU[jnext].caatp-CBU[psi].caatp)/(Datp*tauit) +
										(CBU[jbefore].caatp-CBU[psi].caatp)/(Datp*tauit) +
										(CBU[inext].caatp-CBU[psi].caatp)/(Datp*CRU2[ps].tauil) +
										(CBU[ibefore].caatp-CBU[psi].caatp)/(Datp*CRU2[pos(i-1,j,k)].tauil);

					double coupledye = 	(CBU[knext].cadye-CBU[psi].cadye)/(Ddye*tauit) +
										(CBU[kbefore].cadye-CBU[psi].cadye)/(Ddye*tauit) +
										(CBU[jnext].cadye-CBU[psi].cadye)/(Ddye*tauit) +
										(CBU[jbefore].cadye-CBU[psi].cadye)/(Ddye*tauit) +
										(CBU[inext].cadye-CBU[psi].cadye)/(Ddye*CRU2[ps].tauil) +
										(CBU[ibefore].cadye-CBU[psi].cadye)/(Ddye*CRU2[pos(i-1,j,k)].tauil);

					double coupleegta = (CBU[knext].caegta-CBU[psi].caegta)/(Degta*tauit) +
										(CBU[kbefore].caegta-CBU[psi].caegta)/(Degta*tauit) +
										(CBU[jnext].caegta-CBU[psi].caegta)/(Degta*tauit) +
										(CBU[jbefore].caegta-CBU[psi].caegta)/(Degta*tauit) +
										(CBU[inext].caegta-CBU[psi].caegta)/(Degta*CRU2[ps].tauil) +
										(CBU[ibefore].caegta-CBU[psi].caegta)/(Degta*CRU2[pos(i-1,j,k)].tauil);

					CYT[psi].cinext = CYT[psi].ci 
									  +(	- CYT[psi].Juptake
											- bufftf - buffts - buffcal - buffsr - buffmyo - buffdye - buffegta
											+ coupleci + coupleATP
											+( (crui==0)?( (Vs/Vi)*(diffsi0+diffsiatp0)+(Vp/Vi)*(diffpi0+diffpiatp0) ):0 )
											+( (crui==4)?( (Vs/Vi)*(diffsi1+diffsiatp1)+(Vp/Vi)*(diffpi1+diffpiatp1) ):0 ) 
											+( (crui==1)?( CRU2[ps].Jrelc ):0 )
									  )*DT/( 1.0+katpoff/katpon*Batp/pow2(katpoff/katpon+CYT[psi].ci) );
					
					CYT[psi].cnsrnext = CYT[psi].cnsr 
										+(  CYT[psi].Juptake * Vi/Vnsr
											+ couplecnsr
											+( (crui==0)?( diffjn0*Vjsr/Vnsr ):0 	)
											+( (crui==4)?( diffjn1*Vjsr/Vnsr ):0 	)
											+( (crui==1)?( diffjn1c*Vjsr/Vnsr ):0 )
											+( (crui==5)?( diffjn5c*Vjsr/Vnsr ):0 )
										)*DT;
					
					
					CBU[psi].catf += bufftf*DT;
					CBU[psi].cats += buffts*DT;
					CBU[psi].cacal += buffcal*DT;
					CBU[psi].casr += buffsr*DT;
					CBU[psi].camyo += buffmyo*DT;
					CBU[psi].mgmyo += buffmyomg*DT;
					CBU[psi].caatpnext = Batp*CYT[psi].ci/(katpoff/katpon+CYT[psi].ci);
					CBU[psi].cadyenext += ( buffdye + coupledye
											+( (crui==0)?( (Vs/Vi)*(diffsidye0)+(Vp/Vi)*(diffpidye0) ):0 )
											+( (crui==4)?( (Vs/Vi)*(diffsidye1)+(Vp/Vi)*(diffpidye1) ):0 ) )*DT;
					CBU[psi].caegtanext += ( buffegta + coupleegta
											+( (crui==0)?( (Vs/Vi)*(diffsiegta0)+(Vp/Vi)*(diffpiegta0) ):0 )
											+( (crui==4)?( (Vs/Vi)*(diffsiegta1)+(Vp/Vi)*(diffpiegta1) ):0 ) )*DT;

					if (CYT[psi].cinext < 0 ) 			CYT[psi].cinext = 1e-6;
					if (CYT[psi].cnsrnext < 0 )			CYT[psi].cnsrnext = 1e-6;
					if( CBU[psi].catf < 0 )				CBU[psi].catf = 1e-6;
					if( CBU[psi].cats < 0 )				CBU[psi].cats = 1e-6;
					if( CBU[psi].cacal < 0 )			CBU[psi].cacal = 1e-6;
					if( CBU[psi].casr < 0 )				CBU[psi].casr = 1e-6;
					if( CBU[psi].camyo < 0 )			CBU[psi].camyo = 1e-6;
					if( CBU[psi].mgmyo < 0 )			CBU[psi].mgmyo = 1e-6;
					if( CBU[psi].caatpnext < 0 )		CBU[psi].caatpnext = 1e-6;
					if( CBU[psi].cadyenext < 0 )		CBU[psi].cadyenext = 1e-6;
					if( CBU[psi].caegtanext < 0 )		CBU[psi].caegtanext = 1e-6;
				}
			}
		}
			


		/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		for( int iii = 0; iii < FINESTEP; ++iii )
		{
			////////////////////// submembrane: dotcs ///////////////////////// 
			double csdiff = ( CRU[pos(i+1,j,k)].cs+CRU[pos(i-1,j,k)].cs-2*CRU[ps].cs )/CRU2[ps].tausl
							+ ( CRU[pos(i,j+1,k)].cs+CRU[pos(i,j-1,k)].cs-2*CRU[ps].cs )/taust
							+ ( CRU[pos(i,j,k+1)].cs+CRU[pos(i,j,k-1)].cs-2*CRU[ps].cs )/taust ;
			double csdiffatp = ( SBU[pos(i+1,j,k)].caatp+SBU[pos(i-1,j,k)].caatp-2*SBU[ps].caatp )/(Datp*CRU2[ps].tausl)
							 + ( SBU[pos(i,j+1,k)].caatp+SBU[pos(i,j-1,k)].caatp-2*SBU[ps].caatp )/(Datp*taust)
							 + ( SBU[pos(i,j,k+1)].caatp+SBU[pos(i,j,k-1)].caatp-2*SBU[ps].caatp )/(Datp*taust);
			double csdiffdye = ( SBU[pos(i+1,j,k)].cadye+SBU[pos(i-1,j,k)].cadye-2*SBU[ps].cadye )/(Ddye*CRU2[ps].tausl)
							 + ( SBU[pos(i,j+1,k)].cadye+SBU[pos(i,j-1,k)].cadye-2*SBU[ps].cadye )/(Ddye*taust)
							 + ( SBU[pos(i,j,k+1)].cadye+SBU[pos(i,j,k-1)].cadye-2*SBU[ps].cadye )/(Ddye*taust);
			double csdiffegta = ( SBU[pos(i+1,j,k)].caegta+SBU[pos(i-1,j,k)].caegta-2*SBU[ps].caegta )/(Degta*CRU2[ps].tausl)
							  + ( SBU[pos(i,j+1,k)].caegta+SBU[pos(i,j-1,k)].caegta-2*SBU[ps].caegta )/(Degta*taust)
							  + ( SBU[pos(i,j,k+1)].caegta+SBU[pos(i,j,k-1)].caegta-2*SBU[ps].caegta )/(Degta*taust);

			double diffps = (CRU2[ps].cpnext-CRU[ps].csnext)/taups;
			double diffpsatp = ( SBU[ps].caatpjnext - SBU[ps].caatpnext )/(taups*Datp);
			double diffpsdye = ( SBU[ps].cadyejnext - SBU[ps].cadyenext )/(taups*Ddye);
			double diffpsegta= ( SBU[ps].caegtajnext - SBU[ps].caegtanext )/(taups*Degta);

			double buffsar = ksaron*CRU[ps].csnext*(Bsar-SBU[ps].casar) - ksaroff*SBU[ps].casar;
			double buffsarh= ksarhon*CRU[ps].csnext*(Bsarh-SBU[ps].casarh) - ksarhoff*SBU[ps].casarh;
			double buffatp = katpon*CRU[ps].csnext*(Batp-SBU[ps].caatpnext) - katpoff*SBU[ps].caatpnext;
			double buffdye = kdyeon*CRU[ps].csnext*(Bdye-SBU[ps].cadyenext) - kdyeoff*SBU[ps].cadyenext;
			double buffegta = kegtaon*CRU[ps].csnext*(Begta-SBU[ps].caegtanext) - kegtaoff*SBU[ps].caegtanext;

			SBU[ps].casar += buffsar*DTF;
			SBU[ps].casarh += buffsarh*DTF;

			SBU[ps].caatpnext += DTF * ( buffatp + diffpsatp*Vp/Vs - diffsiatp0 - diffsiatp1 + csdiffatp );
			SBU[ps].cadyenext += DTF * ( buffdye + diffpsdye*Vp/Vs - diffsidye0 - diffsidye1 + csdiffdye );
			SBU[ps].caegtanext += DTF * ( buffegta + diffpsegta*Vp/Vs - diffsiegta0 - diffsiegta1 + csdiffegta );
			CRU[ps].csnext += DTF*( CRU[ps].JNCX - CRU[ps].Jbg *FracSL* Vp/Vs
									+ diffps*Vp/Vs - diffsi0 - diffsi1 + csdiff
									- buffsar - buffsarh - buffatp - buffdye - buffegta );
			

			////////////////////// proximal space: dotcp ////////////////////// 
			buffsar = ksaron*CRU2[ps].cpnext*(Bsar-SBU[ps].casarj) - ksaroff*SBU[ps].casarj;
			buffsarh = ksarhon*CRU2[ps].cpnext*(Bsarh-SBU[ps].casarhj) - ksarhoff*SBU[ps].casarhj;
			buffatp = katpon*CRU2[ps].cpnext*(Batp-SBU[ps].caatpjnext) - katpoff*SBU[ps].caatpjnext;
			buffdye = kdyeon*CRU2[ps].cpnext*(Bdye-SBU[ps].cadyejnext) - kdyeoff*SBU[ps].cadyejnext;
			buffegta = kegtaon*CRU2[ps].cpnext*(Begta-SBU[ps].caegtajnext) - kegtaoff*SBU[ps].caegtajnext;

			SBU[ps].casarj += buffsar*DTF;
			SBU[ps].casarhj += buffsarh*DTF;

			SBU[ps].caatpjnext += DTF*( buffatp - diffpsatp - diffpiatp0 - diffpiatp1 );
			SBU[ps].cadyejnext += DTF*( buffdye - diffpsdye - diffpidye0 - diffpidye1 );
			SBU[ps].caegtajnext += DTF*( buffegta - diffpsegta - diffpiegta0 - diffpiegta1 );
			CRU2[ps].cpnext += DTF*( CRU2[ps].Jrel + CRU2[ps].Jleak - CRU[ps].JCa - CRU[ps].Jbg*(1.0-FracSL)
									- diffps - diffpi0 - diffpi1 
									- buffsar - buffsarh - buffatp - buffdye - buffegta );
		
			if ( CRU[ps].csnext < 0)			CRU[ps].csnext = 1e-6;
			if ( SBU[ps].casar < 0 )			SBU[ps].casar = 1e-6;
			if ( SBU[ps].casarh < 0 )			SBU[ps].casarh = 1e-6;
			if ( SBU[ps].caatpnext < 0 )		SBU[ps].caatpnext = 1e-6;
			if ( SBU[ps].cadyenext < 0 )		SBU[ps].cadyenext = 1e-6;
			if ( SBU[ps].caegtanext < 0 )		SBU[ps].caegtanext = 1e-6;

			if ( CRU2[ps].cpnext < 0 ) 			CRU2[ps].cpnext = 1e-6;
			if ( SBU[ps].casarj < 0 )			SBU[ps].casarj = 1e-6;
			if ( SBU[ps].casarhj < 0 )			SBU[ps].casarhj = 1e-6;
			if ( SBU[ps].caatpjnext < 0 )		SBU[ps].caatpjnext = 1e-6;
			if ( SBU[ps].cadyejnext < 0 )		SBU[ps].cadyejnext = 1e-6;
			if ( SBU[ps].caegtajnext < 0 )		SBU[ps].caegtajnext = 1e-6;
		}

		// dotcjsr
		for( int iii = 0; iii < FINESTEP; ++iii )
		{
			double betaCSQN = 1.0/( 1.0 + BCSQN*Kc*nCa/pow2(Kc+CRU2[ps].cjsr) );
			CRU2[ps].cjsr += betaCSQN*( -diffjn0-diffjn1 - CRU2[ps].Jrel*Vp/Vjsr - CRU2[ps].Jleak*Vp/Vjsr )*DTF;
			CRU2[ps].Tcj = CRU2[ps].cjsr + BCSQN*nCa*CRU2[ps].cjsr/(Kc+CRU2[ps].cjsr);

			betaCSQN = 1.0/( 1.0 + BCSQN*Kc*nCa/pow2(Kc+CRU2[ps].cjsrc) );
			CRU2[ps].cjsrc += betaCSQN*( -diffjn1c-diffjn5c - CRU2[ps].Jrelc*Vi/Vjsr )*DTF;
			CRU2[ps].Tcjc = CRU2[ps].cjsrc + BCSQN*nCa*CRU2[ps].cjsrc/(Kc+CRU2[ps].cjsrc);

			if (CRU2[ps].cjsrc<0)
				CRU2[ps].Tcjc = 0;
		}

		CRU2[ps].state = localState;
	}
}


__global__ void Finish( cru *CRU, cru2 *CRU2, cytosol *CYT, cyt_bu *CBU, sl_bu *SBU )
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int k = threadIdx.z + blockIdx.z * blockDim.z;
	int ps = pos(i,j,k);
	int ix, jy, kz, psb;

	if((i*j*k)!=0 && i<Nx-1 && j<Ny-1 && k<Nz-1)
	{
		///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		/////////////////////////////////////////////////////// update ////////////////////////////////////////////////////////
		///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		// this update should not be in the function Compute because of synchronization.
		CRU[ps].cs = CRU[ps].csnext;
		SBU[ps].caatp = SBU[ps].caatpnext;
		SBU[ps].cadye = SBU[ps].cadyenext;
		SBU[ps].caegta= SBU[ps].caegtanext;

		CRU2[ps].cp = CRU2[ps].cpnext;
		SBU[ps].caatpj=SBU[ps].caatpjnext;
		SBU[ps].cadyej=SBU[ps].cadyejnext;
		SBU[ps].caegtaj=SBU[ps].caegtajnext;

		for( ix = 0; ix < Nci; ix++ )
		{
			CYT[ps*Nci+ix].ci = CYT[ps*Nci+ix].cinext;
			CYT[ps*Nci+ix].cnsr = CYT[ps*Nci+ix].cnsrnext;
			CBU[ps*Nci+ix].caatp = CBU[ps*Nci+ix].caatpnext;
			CBU[ps*Nci+ix].cadye = CBU[ps*Nci+ix].cadyenext;
			CBU[ps*Nci+ix].caegta = CBU[ps*Nci+ix].caegtanext;
			
		}
		///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		/////////////////////////////////////////////////////// Boundary //////////////////////////////////////////////////////
		///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		#ifndef PermeabilizedB

			if (i==1)
			{
				psb=pos(0,j,k);
				for (jy=0;jy<Niy;jy++)
				{
					for (kz=0;kz<Niz;kz++)
					{
						CYT[psb*Nci+posi(Nix-1,jy,kz)].cnsr = CYT[ps*Nci+posi(0,jy,kz)].cnsr;
						CYT[psb*Nci+posi(Nix-1,jy,kz)].ci  =  CYT[ps*Nci+posi(0,jy,kz)].ci;
						CBU[psb*Nci+posi(Nix-1,jy,kz)].caatp= CBU[ps*Nci+posi(0,jy,kz)].caatp;
						CBU[psb*Nci+posi(Nix-1,jy,kz)].cadye= CBU[ps*Nci+posi(0,jy,kz)].cadye;
						CBU[psb*Nci+posi(Nix-1,jy,kz)].caegta=CBU[ps*Nci+posi(0,jy,kz)].caegta;
					}
				}
			}

			if (i==Nx-2)
			{
				psb=pos(Nx-1,j,k);
				for (jy=0;jy<Niy;jy++)
				{
					for (kz=0;kz<Niz;kz++)
					{
						CYT[psb*Nci+posi(0,jy,kz)].cnsr = CYT[ps*Nci+posi(Nix-1,jy,kz)].cnsr;
						CYT[psb*Nci+posi(0,jy,kz)].ci  =  CYT[ps*Nci+posi(Nix-1,jy,kz)].ci;
						CBU[psb*Nci+posi(0,jy,kz)].caatp= CBU[ps*Nci+posi(Nix-1,jy,kz)].caatp;
						CBU[psb*Nci+posi(0,jy,kz)].cadye= CBU[ps*Nci+posi(Nix-1,jy,kz)].cadye;
						CBU[psb*Nci+posi(0,jy,kz)].caegta=CBU[ps*Nci+posi(Nix-1,jy,kz)].caegta;
					}
				}
			}

			if (j==1)
			{
				psb=pos(i,0,k);
				for (ix=0;ix<Nix;ix++)
				{
					for (kz=0;kz<Niz;kz++)
					{
						CYT[psb*Nci+posi(ix,Niy-1,kz)].cnsr = CYT[ps*Nci+posi(ix,0,kz)].cnsr;
						CYT[psb*Nci+posi(ix,Niy-1,kz)].ci  =  CYT[ps*Nci+posi(ix,0,kz)].ci;
						CBU[psb*Nci+posi(ix,Niy-1,kz)].caatp= CBU[ps*Nci+posi(ix,0,kz)].caatp;
						CBU[psb*Nci+posi(ix,Niy-1,kz)].cadye= CBU[ps*Nci+posi(ix,0,kz)].cadye;
						CBU[psb*Nci+posi(ix,Niy-1,kz)].caegta=CBU[ps*Nci+posi(ix,0,kz)].caegta;
					}
				}
				CRU[psb].cs = CRU[ps].cs;
				SBU[psb].caatp = SBU[ps].caatp;
				SBU[psb].cadye = SBU[ps].cadye;
				SBU[psb].caegta= SBU[ps].caegta;
			}

			if (j==Ny-2)
			{
				psb=pos(i,Ny-1,k);
				for (ix=0;ix<Nix;ix++)
				{
					for (kz=0;kz<Niz;kz++)
					{
						CYT[psb*Nci+posi(ix,0,kz)].cnsr = CYT[ps*Nci+posi(ix,Niy-1,kz)].cnsr;
						CYT[psb*Nci+posi(ix,0,kz)].ci  =  CYT[ps*Nci+posi(ix,Niy-1,kz)].ci;
						CBU[psb*Nci+posi(ix,0,kz)].caatp= CBU[ps*Nci+posi(ix,Niy-1,kz)].caatp;
						CBU[psb*Nci+posi(ix,0,kz)].cadye= CBU[ps*Nci+posi(ix,Niy-1,kz)].cadye;
						CBU[psb*Nci+posi(ix,0,kz)].caegta=CBU[ps*Nci+posi(ix,Niy-1,kz)].caegta;
					}
				}
				CRU[psb].cs = CRU[ps].cs;
				SBU[psb].caatp = SBU[ps].caatp;
				SBU[psb].cadye = SBU[ps].cadye;
				SBU[psb].caegta= SBU[ps].caegta;
			}

			if (k==1)
			{
				psb=pos(i,j,0);
				for (ix=0;ix<Nix;ix++)
				{
					for (jy=0;jy<Niy;jy++)
					{
						CYT[psb*Nci+posi(ix,jy,Niz-1)].cnsr = CYT[ps*Nci+posi(ix,jy,0)].cnsr;
						CYT[psb*Nci+posi(ix,jy,Niz-1)].ci  =  CYT[ps*Nci+posi(ix,jy,0)].ci;
						CBU[psb*Nci+posi(ix,jy,Niz-1)].caatp= CBU[ps*Nci+posi(ix,jy,0)].caatp;
						CBU[psb*Nci+posi(ix,jy,Niz-1)].cadye= CBU[ps*Nci+posi(ix,jy,0)].cadye;
						CBU[psb*Nci+posi(ix,jy,Niz-1)].caegta=CBU[ps*Nci+posi(ix,jy,0)].caegta;
					}
				}
				CRU[psb].cs = CRU[ps].cs;
				SBU[psb].caatp = SBU[ps].caatp;
				SBU[psb].cadye = SBU[ps].cadye;
				SBU[psb].caegta= SBU[ps].caegta;
			}

			if (k==Nz-2)
			{
				psb=pos(i,j,Nz-1);
				for (ix=0;ix<Nix;ix++)
				{
					for (jy=0;jy<Niy;jy++)
					{
						CYT[psb*Nci+posi(ix,jy,0)].cnsr = CYT[ps*Nci+posi(ix,jy,Niz-1)].cnsr;
						CYT[psb*Nci+posi(ix,jy,0)].ci  =  CYT[ps*Nci+posi(ix,jy,Niz-1)].ci;
						CBU[psb*Nci+posi(ix,jy,0)].caatp= CBU[ps*Nci+posi(ix,jy,Niz-1)].caatp;
						CBU[psb*Nci+posi(ix,jy,0)].cadye= CBU[ps*Nci+posi(ix,jy,Niz-1)].cadye;
						CBU[psb*Nci+posi(ix,jy,0)].caegta=CBU[ps*Nci+posi(ix,jy,Niz-1)].caegta;
					}
				}
				CRU[psb].cs = CRU[ps].cs;
				SBU[psb].caatp = SBU[ps].caatp;
				SBU[psb].cadye = SBU[ps].cadye;
				SBU[psb].caegta= SBU[ps].caegta;
			}

		#else // Permeabilized cell
			if (i==1)
			{
				psb=pos(0,j,k);
				for (jy=0;jy<Niy;jy++)
				{
					for (kz=0;kz<Niz;kz++)
					{
						CYT[psb*Nci+posi(Nix-1,jy,kz)].cnsr = CYT[ps*Nci+posi(0,jy,kz)].cnsr;
					}
				}
			}

			if (i==Nx-2)
			{
				psb=pos(Nx-1,j,k);
				for (jy=0;jy<Niy;jy++)
				{
					for (kz=0;kz<Niz;kz++)
					{
						CYT[psb*Nci+posi(0,jy,kz)].cnsr = CYT[ps*Nci+posi(Nix-1,jy,kz)].cnsr;
					}
				}
			}

			if (j==1)
			{
				psb=pos(i,0,k);
				for (ix=0;ix<Nix;ix++)
				{
					for (kz=0;kz<Niz;kz++)
					{
						CYT[psb*Nci+posi(ix,Niy-1,kz)].cnsr = CYT[ps*Nci+posi(ix,0,kz)].cnsr;
					}
				}
			}

			if (j==Ny-2)
			{
				psb=pos(i,Ny-1,k);
				for (ix=0;ix<Nix;ix++)
				{
					for (kz=0;kz<Niz;kz++)
					{
						CYT[psb*Nci+posi(ix,0,kz)].cnsr = CYT[ps*Nci+posi(ix,Niy-1,kz)].cnsr;
					}
				}
			}

			if (k==1)
			{
				psb=pos(i,j,0);
				for (ix=0;ix<Nix;ix++)
				{
					for (jy=0;jy<Niy;jy++)
					{
						CYT[psb*Nci+posi(ix,jy,Niz-1)].cnsr = CYT[ps*Nci+posi(ix,jy,0)].cnsr;
					}
				}
			}

			if (k==Nz-2)
			{
				psb=pos(i,j,Nz-1);
				for (ix=0;ix<Nix;ix++)
				{
					for (jy=0;jy<Niy;jy++)
					{
						CYT[psb*Nci+posi(ix,jy,0)].cnsr = CYT[ps*Nci+posi(ix,jy,Niz-1)].cnsr;
					}
				}
			}
		#endif
	}
}

__global__ void	setup_kernel(unsigned long long seed, cru2 *CRU2 )
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int k = threadIdx.z + blockIdx.z * blockDim.z;
	hiprand_init(seed, pos(i,j,k), 0, &(CRU2[pos(i,j,k)].state)	);
}


__device__ int ryrgating (double cp, double cjsr, hiprandState *state, int *ncu, int *nou, int *ncb, int *nob, int i, int j, int k, int step,
							double f_Jmax, double alpha, double beta, int Hill, double custar, double cbstar)
{
	hiprandState localState = *state;

	// should be satisfied: ku*DT < 1.0
	double ku = alpha/( 1.0 + pow(custar/cp,Hill) ) + 2e-6;
	double kb = beta/( 1.0 + pow(cbstar/cp,Hill) ) + 2e-9; // /( 1.0 + pow(500/cjsr,4) )
	double kuminus = 1.0/taucu;
	double kbminus = 1.0/taucb;
	
	double cb = BCSQN * nCa * cjsr / (Kc+cjsr); // bound Ca2+ concentration in JSR
	double ku2b = 1.0/( 1.0+pow(cb/BCSQN/13.3, 24) )/taub;
	double kb2u = 1.0/(  4000.0/( 1.0+pow(cjsr/670.0, 24) ) + 300.0  );
	
	double p_cu_ou = ku * DT;
	double p_cb_ob = kb * DT;
	double p_ou_cu = kuminus * DT;
	double p_ob_cb = kbminus * DT;
	double p_ou_ob = ku2b * DT;
	double p_cu_cb = ku2b * DT;
	double p_cb_cu = kb2u * DT;
	double p_ob_ou = kb2u * (ku/kb) * DT;

	if ( kb < 1e-16 )
	{
		p_ou_ob = 0;
		p_ob_ou = 0;
	}

	int n_cu_ou = number_RyR_transit( &localState, *ncu, p_cu_ou, *ncu );
	int n_cu_cb = number_RyR_transit( &localState, *ncu, p_cu_cb, *ncu - n_cu_ou );
	int n_ou_cu = number_RyR_transit( &localState, *nou, p_ou_cu, *nou );
	int n_ou_ob = number_RyR_transit( &localState, *nou, p_ou_ob, *nou - n_ou_cu );
	int n_cb_cu = number_RyR_transit( &localState, *ncb, p_cb_cu, *ncb );
	int n_cb_ob = number_RyR_transit( &localState, *ncb, p_cb_ob, *ncb - n_cb_cu );
	int n_ob_ou = number_RyR_transit( &localState, *nob, p_ob_ou, *nob );
	int n_ob_cb = number_RyR_transit( &localState, *nob, p_ob_cb, *nob - n_ob_ou );

	*nou += - n_ou_ob - n_ou_cu + n_ob_ou + n_cu_ou;
	*nob += - n_ob_ou - n_ob_cb + n_ou_ob + n_cb_ob;
	*ncu += - n_cu_ou - n_cu_cb + n_ou_cu + n_cb_cu;
	*ncb += - n_cb_cu - n_cb_ob + n_cu_cb + n_ob_cb;

	*state = localState;

	return ( *nou + *nob );
}

// NN: number of RyRs in the current state
// probability: the probability to transit to another state
// upBound: maximum number of RyRs to transit
__device__ int number_RyR_transit(hiprandState *state, int NN, double probability, int upBound)
{
	int Ntransit = 1001; // larger than nryr
	double mean = NN*probability;

	// If the condition is satisfied, Ntransit is a poisson distribution,
	// otherwise it is a gaussian distribution. They are the approximations 
	// of the binomial distribution.
	if ( probability < 0.26*exp(-NN/2.245) + 0.12*exp(-NN/35.17) + 0.11 )
		while ( Ntransit > upBound ) // Poisson random nmber
		{
			int k = 0;
			double p = 1.0;
			while ( p >= exp(-mean) )
			{
				k++;
				p = p * hiprand_uniform_double(state);
			}
			Ntransit = k - 1;
		}
	else
		while ( Ntransit < 0 || Ntransit > upBound )
			Ntransit = lrintf( mean + sqrt( mean * (1.0-probability) ) * hiprand_normal_double(state) );

	return Ntransit;
}


__device__ int LCCgating(double v, double cp, hiprandState *state, int i )
{	

	hiprandState localState=*state;

	double dv5 = 5;
	double dvk = 8;

	double fv5 = -22.8;	
	double fvk = 9.1;

	double alphac = 0.22;
	double betac = 4;

	#ifdef ISO
		betac = 2;
		dv5 = 0;
		fv5 = -28;
		fvk = 8.5;
	#endif

	double dinf = 1.0/(1.0+exp(-(v-dv5)/dvk));
	double taud_inverse = 1.0/((1.0-exp(-(v-dv5)/dvk))/(0.035*(v-dv5))*dinf);
	if( (v > -0.0001) && (v < 0.0001) )
		taud_inverse = 0.035*dvk/dinf;
	
	double finf = 1.0-1.0/(1.0+exp(-(v-fv5)/fvk))/(1.+exp((v-60)/12.));
	double tauf_inverse = (0.02-0.007*exp(-pow2(0.0337*(v+10.5))));
	

	double alphad = dinf * taud_inverse;
	double betad = (1.0-dinf) * taud_inverse;
	
	double alphaf = finf * tauf_inverse;
	double betaf = (1.0-finf) * tauf_inverse;
	
	double alphafca = 0.006;
	double betafca = 0.175/( 1 + pow2(60.0/cp) );

	double random = hiprand_uniform_double(&localState)/DT;
	*state=localState;
	

	if ( i%2 )
		if ( random < alphac )
			return i-1;
		else
			random -= alphac;
	else
		if ( random < betac )
			return i+1;
		else
			random -= betac;
	

	if ( (i/2)%2 )
		if ( random < alphad )
			return i-2;
		else
			random -= alphad;
	else
		if ( random < betad )
			return i+2;
		else
			random -= betad;
	
	
	if ( (i/4)%2 )
		if ( random < alphaf )
			return i-4;
		else
			random -= alphaf;
	else
		if ( random < betaf )
			return i+4;
		else
			random -= betaf;
	
	
	if ( (i/8)%2 )
		if ( random < alphafca )
			return i-8;
		else
			random -= alphafca;
	else
		if ( random < betafca )
			return i+8;
		else
			random -= betafca;

	return (i);
}

__device__ double Single_LCC_Current(double v, double cp) // cp in mM
{
	double ica = 0;
	double za = v*Faraday/RR/Temperature;
	if ( fabs(za)<0.001 ) 
		ica = 2.0*Pca*Faraday*gammai*(cp*exp(2.0*za)-CaO);
	else 
		ica = 4.0*Pca*za*Faraday*gammai*(cp*exp(2.0*za)-CaO)/(exp(2.0*za)-1.0);

	if (ica > 0.0)
		ica = 0.0;

	return ( ica );
}

double Ina( double v, double *hh, double *jj, double *mm, double nai )
{
	double Ena = 1.0/FRT*log(NaO/nai);
	double am = 0.32*(v+47.13)/(1.0-exp(-0.1*(v+47.13)));
	double bm = 0.08*exp(-v/11.0);

	double ah,bh,aj,bj;

	if(v < -40.0)
	{
		ah = 0.135 * exp( -(80.0+v)/6.8 );
		bh = 3.56 * exp(0.079*v) + 310000.0*exp(0.35*v);
		aj = (-127140.0*exp(0.2444*v)-0.00003474*exp(-0.04391*v)) * ( (v+37.78)/(1.0+exp(0.311*(v+79.23))) );
		bj = (0.1212*exp(-0.01052*v))/(1.0+exp(-0.1378*(v+40.14)));
		
	}
	else
	{
		ah = 0.0;
		bh = 1.0/( 0.130*(1.0+exp((v+10.66)/(-11.1))) );
		aj = 0.0;
		bj = ( 0.3*exp(-0.0000002535*v) )/( 1.0 + exp(-0.1*(v+32.0)) );
				
	}
			
	double tauh = 1.0/(ah+bh);
	double tauj = 1.0/(aj+bj);
	double taum = 1.0/(am+bm);

	*hh = ah/(ah+bh)-((ah/(ah+bh))-*hh)*exp(-DT/tauh);
	*jj = aj/(aj+bj)-((aj/(aj+bj))-*jj)*exp(-DT/tauj);
	*mm = am/(am+bm)-((am/(am+bm))-*mm)*exp(-DT/taum);

	double INa = gNa*(*hh)*(*jj)*(*mm)*(*mm)*(*mm)*(v-Ena) + gNaLeak*(v-Ena);

	return INa;
}


double Ikr( double v, double *Xkr )
{
	double krv1 = 0.00138*(v+7.0)/( 1.0-exp(-0.123*(v+7.0))  );
	double krv2 = 0.00061*(v+10.0)/(exp( 0.145*(v+10.0))-1.0);
	double taukr = 1.0/(krv1+krv2);
	double Xkr_inf= 1.0/(1.0+exp(-(v+50.0)/7.5));
	double Rkr = 1.0/(1.0+exp((v+33.0)/22.4));

	*Xkr = Xkr_inf - ( Xkr_inf - *Xkr ) * exp(-DT/taukr);
	
	double I_Kr = gKr * sqrt(KO/5.40) * (*Xkr) * Rkr * (v-Ek);

	return I_Kr;
}

double Iks( double v, double *Xs1, double *Xs2, double *Qks, double cst, double nai )
{
	double prnak = 0.01833;
	double Eks = (1.0/FRT)*log((KO+prnak*NaO)/(KI+prnak*nai));

	double qks_inf =  1.0 + 0.8/( 1.0 + pow((0.5/cst),3) ) ;
	double tauqks = 1000.0;

	double Xs1_inf= 1.0/(1.0+exp(-(v-1.5)/16.7));
	double tauxs = 1.0/( 0.0000719*(v+30.0)/(1.0-exp(-0.148*(v+30.0)))
							+ 0.000131*(v+30.0)/(exp(0.06870*(v+30.0))-1.0) );

	*Xs1 = Xs1_inf-(Xs1_inf-*Xs1)*exp(-DT/tauxs);
	*Xs2 = Xs1_inf-(Xs1_inf-*Xs2)*exp(-DT/tauxs/4.0);
	*Qks = *Qks + DT*( qks_inf-*Qks )/tauqks;

	double I_Ks = gKs*(*Qks)*(*Xs1)*(*Xs2)*(v-Eks);

	return I_Ks;
}

double Ik1( double v )
{
	double Aki = 1.02/(1.0+exp(0.2385*(v-Ek-59.215)));
	double Bki = (0.49124*exp(0.08032*(v-Ek+5.476))+exp(0.061750*(v-Ek-594.31)))/(1.0+exp(-0.5143*(v-Ek+4.753)));
	double I_K1 = gK1 * sqrt(KO/5.4) * Aki/(Aki+Bki) * (v-Ek);

	return I_K1;
}

double Itos(double v, double *Xtos, double *Ytos)
{
	double Xtos_inf = 1.0/( 1.0 + exp( -(v+3.0)/15.0) );
	double Ytos_inf = 1.0/( 1.0 + exp( (v+33.5)/10.0) );
	double Rs_inf = 1.0/( 1.0 + exp( (v+33.5)/10.0) );
	double txs = 9.0/( 1.0 + exp( (v+3.0)/15.0) ) + 0.5;
	double tys = 3000.0/(1.0+exp( (v+60.0)/10.0) ) + 30.0;

	*Xtos = Xtos_inf-(Xtos_inf-*Xtos)*exp(-DT/txs);
	*Ytos = Ytos_inf-(Ytos_inf-*Ytos)*exp(-DT/tys);

	double I_tos = gtos*(*Xtos)*(*Ytos+0.5*Rs_inf)*(v-Ek);

	return I_tos;
}

///////////////// Ito /////////////////

double Itof(double v, double *Xtof, double *Ytof)
{
	double Xtof_inf = 1.0/(1.0+exp( -(v+3.0)/15.0) );
	double Ytof_inf = 1.0/(1.0+exp( (v+33.5)/10.0) );
	double txf = 3.5 * exp( -(v/30.00)*(v/30.0) ) + 1.5;
	double tyf = 20.0/( 1.0+exp( (v+33.5)/10.0 ) )+20.0;

	*Xtof = Xtof_inf-(Xtof_inf-*Xtof)*exp(-DT/txf);
	*Ytof = Ytof_inf-(Ytof_inf-*Ytof)*exp(-DT/tyf);

	double I_tof = gtof*(*Xtof)*(*Ytof)*(v-Ek);

	return I_tof;
}


double Inak( double v, double nai )	 // Mahajan et al 2008
{
	double sigma = ( exp(NaO/67.3) - 1.0 )/7.0;
	double fNaK = 1.0/( 1.0 + 0.1245*exp(-0.1*v*FRT) + 0.0365*sigma*exp(-v*FRT) );
	double I_NaK = gNaK * fNaK * 1.0/( 1.0+pow(12.0/nai,1.0) ) * KO/(KO+1.5);

	return I_NaK;
}	

///////////////////////////	sodium dynamics /////////////////////////////////
double sodium(double v, double nai, double I_Na, double I_NaK, double I_NCX)
{
	// convert pA/pF to mM/ms. Mahajan et al 2008, Eq. 33
	double alpha = 1.0/(2.0*0.096485)/( Vi*Nci*(Nx-2)*(Ny-2)*(Nz-2) ) * Cm / 1000.0;
	double trick = 10.0; // just to speed up Nai dynamics

	double dnai = - trick * alpha * ( I_Na + 3.0*I_NaK + 3.0*I_NCX );

	return (nai + dnai*DT);
}

__device__ double ncx(double v, double cs, double nai, double *Ka)
{
	double csm = cs/1000.0;
	double za = v*Faraday/RR/Temperature;

	double t1 = Kmcai*pow3(NaO)*( 1.0+pow3(nai/Kmnai) );
	double t2 = pow3(Kmnao)*csm*(1.0+csm/Kmcai);
	double t3 = (Kmcao+CaO)*pow3(nai) + csm*pow3(NaO);

	*Ka = 1.0/(1.0+pow3(0.0007/csm));

	double Inaca = Vncx * (*Ka) * ( exp(eta*za)*pow3(nai)*CaO-exp((eta-1.0)*za)*pow3(NaO)*csm )
					/((t1+t2+t3)*(1.0+ksat*exp((eta-1.0)*za)));

	return Inaca;
}	

__device__ double uptake1(double ci, double cnsr)		//uptake
{
	double Ki = 0.2;
	double Knsr = 1700.0;
	double HH = 1.787;
	double Iuptake = Vup * (pow(ci/Ki,HH)-pow(cnsr/Knsr,HH)) / (1.0+pow(ci/Ki,HH)+pow(cnsr/Knsr,HH));
	return Iuptake;
}


__device__ double uptake(double ci, double cnsr)			//uptake
{
	double k1p = 25900/1000.0;
	double k2p = 2540/1000.0;
	double k3p = 20.5/1000.0;
	double k1m = 2*10/1000.0;
	double k2m = 67200*10/1000.0;
	double k3m = 149*10.0/1000.0;
	double kdci = 0.91*4.0;
	double kdcj = 2.24/1.0;//2.24;
	double kdh1 = 1.09e-5;
	double kdhi = 3.54e-3;
	double kdhj = 1.05e-8;
	double kdh = 7.24e-5;

	double MgATP = 4;
	double MgADP = 0.0363;
	double H = 6.3e-8; // pH = 7.2
	double Pii = 10.0;

	double T_MgATP = MgATP/k1m*k1p;
	double T_ci = ci/1000.0/kdci;
	double T_Hi = H/kdhi;
	double T_H1 = H/kdh1;
	double T_cj = cnsr/1000.0/kdcj;
	double T_Hj = H/kdhj;
	double T_H = H/kdh;

	double a1p = k2p * T_MgATP * pow2(T_ci) / ( T_MgATP * pow2(T_ci) + T_Hi*(1+T_MgATP*(1+T_H1+pow2(T_ci))) );
	double a2p = k3p * T_Hj / ( T_Hj * (1+T_H)+ T_H*(1+pow2(T_cj)) );
	double a1m = k2m * MgADP * pow2(T_cj)*T_H / ( T_Hj * (1+T_H) + T_H*(1+pow2(T_cj)) );
	double a2m = k3m * Pii * T_Hi / ( T_MgATP * pow2(T_ci) + T_Hi*( 1+T_MgATP*(1+T_H1 + pow2(T_ci)) ) );

	double Iup = Vup * (a1p*a2p-a1m*a2m) / (a1m+a2m+a1p+a2p);

	return(Iup);
}

#ifdef	output_fluoxt
int convolve(double*fluo, cyt_bu *CBU)
{
	int i,j,k,ix,jy,kz; // i,j,k for CRU index; ix, jy, kz for lattices in each CRU
	double *object,*kernel,*result;
	fftw_complex *hat_object,*hat_kernel,*hat_ko;
	fftw_plan object_forward,kernel_forward,ko_backward;
	
	object =	(double*) fftw_malloc(sizeof(double) * Nx*Nix * Ny*Niy *Nz*Niz);
	kernel =	(double*) fftw_malloc(sizeof(double) * Nx*Nix * Ny*Niy *Nz*Niz);
	result =	(double*) fftw_malloc(sizeof(double) * Nx*Nix * Ny*Niy *Nz*Niz);
	hat_object=	(fftw_complex*) fftw_malloc(sizeof(fftw_complex) * ((Nx*Nix)/2+1) * Ny*Niy * Nz*Niz);
	hat_kernel=	(fftw_complex*) fftw_malloc(sizeof(fftw_complex) * ((Nx*Nix)/2+1) * Ny*Niy * Nz*Niz);
	hat_ko=		(fftw_complex*) fftw_malloc(sizeof(fftw_complex) * ((Nx*Nix)/2+1) * Ny*Niy * Nz*Niz);
	object_forward = 	fftw_plan_dft_r2c_3d(Nz*Niz, Ny*Niy, Nx*Nix, object,	hat_object, FFTW_ESTIMATE);
	kernel_forward = 	fftw_plan_dft_r2c_3d(Nz*Niz, Ny*Niy, Nx*Nix, kernel,	hat_kernel, FFTW_ESTIMATE);
	ko_backward = 		fftw_plan_dft_c2r_3d(Nz*Niz, Ny*Niy, Nx*Nix, hat_ko,	result, FFTW_ESTIMATE);
	
	// initial condition	
	for (k=0; k<Nz; k++) 
	{
		for (kz=0; kz<Niz; kz++)
		{
			for (j=0; j<Ny; j++) 
			{
				for (jy=0; jy<Niy; jy++) 
				{
					for (i=0;i<Nx;i++)
					{
						for (ix=0;ix<Nix;ix++)
						{
							kernel[posall(i*Nix+ix,j*Niy+jy,k*Niz+kz)]=G(i,j,k,ix,jy,kz,Nx*Nix/2,Ny*Niy/2,Nz*Niz/2);
							object[posall(i*Nix+ix,j*Niy+jy,k*Niz+kz)]=CBU[pos(i,j,k)*Nci+posi(ix,jy,kz)].cadye;
						}
					}
				}
			}
		}
	}

	fftw_execute(object_forward);
	fftw_execute(kernel_forward); 
	
	for (k=0; k<Nz*Niz; k++) 
	{
		for (j=0; j<Ny*Niy; j++) 
		{
			for (i=0;i<Nx*Nix/2+1;i++)
			{
				hat_ko[posallf(i,j,k)][0]=hat_object[posallf(i,j,k)][0]*hat_kernel[posallf(i,j,k)][0]
										 -hat_object[posallf(i,j,k)][1]*hat_kernel[posallf(i,j,k)][1];
				hat_ko[posallf(i,j,k)][1]=hat_object[posallf(i,j,k)][0]*hat_kernel[posallf(i,j,k)][1]
										 +hat_object[posallf(i,j,k)][1]*hat_kernel[posallf(i,j,k)][0];
			}
		}
	}
	fftw_execute(ko_backward);

	//Normalization	
	for (k=0; k<Nz*Niz; k++) 
	{
		for (j=0; j<Ny*Niy; j++) 
		{
			for (i=0;i<Nx*Nix;i++)
			{
				result[posall(i,j,k)]=DX*DY*DZ*result[posall(i,j,k)]/((double)(Nx*Nix*Ny*Niy*Nz*Niz)); 
			}
		}
	}
	
	
	// update
	for (k=0; k<Nz*Niz; k++) 
	{
		for (j=0; j<Ny*Niy; j++) 
		{
			for (i=0; i<Nx*Nix; i++)
			{
				int xx = (i+Nx*Nix/2)*(i<Nx*Nix/2) + (i-Nx*Nix/2)*(i>=Nx*Nix/2);
				int yy = (j+Ny*Niy/2)*(j<Ny*Niy/2) + (j-Ny*Niy/2)*(j>=Ny*Niy/2);
				int zz = (k+Nz*Niz/2)*(k<Nz*Niz/2) + (k-Nz*Niz/2)*(k>=Nz*Niz/2);
				fluo[posall(i,j,k)] = result[posall(xx,yy,zz)];
			}
		}
	}
	
	fftw_destroy_plan(object_forward);
	fftw_destroy_plan(kernel_forward);
	fftw_destroy_plan(ko_backward);
	fftw_free(object);
	fftw_free(kernel);
	fftw_free(result);
	fftw_free(hat_object);
	fftw_free(hat_kernel);
	fftw_free(hat_ko);
	return 1;
}
#endif




void matrix2file(cytosol *CYT, int step)
{
	int i,j,k,ix,jy,kz; // i,j,k for CRU index; ix, jy, kz for lattices in each CRU
	double average=0;
	char FileName[50];
	sprintf(FileName,"%-s%d%s","step",step,".vtk");
	
	FILE * file_pointer;
	file_pointer=fopen(FileName,"w");
		fprintf(file_pointer, "# vtk DataFile Version 3.0\n");
		fprintf(file_pointer, "3d\n");
		fprintf(file_pointer, "ASCII\n");
		fprintf(file_pointer, "DATASET STRUCTURED_POINTS\n");
		fprintf(file_pointer, "DIMENSIONS %d %d %d\n",(Nx-2)*Nix,(Ny-2)*Niy,(Nz-2)*Niz);
		fprintf(file_pointer, "ASPECT_RATIO 1 1 1\n");
		fprintf(file_pointer, "ORIGIN 0 0 0\n");
		fprintf(file_pointer, "POINT_DATA %d\n",(Nx-2)*Nix*(Ny-2)*Niy*(Nz-2)*Niz);
		fprintf(file_pointer, "SCALARS ci double 1\n");
		fprintf(file_pointer, "LOOKUP_TABLE default\n\n");
		for (k=1;k<(Nz-1);k++)
		{
			for (kz=0;kz<Niz;kz++)
			{
				for (j=1;j<(Ny-1);j++)
				{
					for (jy=0;jy<Niy;jy++)
					{
						for (i=1;i<(Nx-1);i++)
						{
							for (ix=0;ix<Nix;ix++)
							{
								fprintf(file_pointer,"%g \t", CYT[pos(i,j,k)*Nci+posi(ix,jy,kz)].ci);
								average += CYT[pos(i,j,k)*Nci+posi(ix,jy,kz)].ci;
								if( CYT[pos(i,j,k)*Nci+posi(ix,jy,kz)].ci > 100.0 )
								{
									cout << step*DT <<" "<<i<<" "<<j<<" "<<k<<" "<<posi(ix,jy,kz)<<
									" error! ci="<< CYT[pos(i,j,k)*Nci+posi(ix,jy,kz)].ci << endl;
								}
							}
						}
						fprintf(file_pointer, "\n");
					}
				}
			}
		}
		average /= (1.0*(Nx-2)*(Ny-2)*(Nz-2)*Nix*Niy*Niz);
		printf("Average=%g\t",average);
	fclose(file_pointer);
}

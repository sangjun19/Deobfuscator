#include "hip/hip_runtime.h"
// Repository: dynexcoin/Dynex-Neuromorphic-Chip
// File: dynex.cu

// Copyright (c) 2021-2022, The DYNEX Project
// 
// All rights reserved.
// 
// Redistribution and use in source and binary forms, with or without modification, are
// permitted provided that the following conditions are met:
// 
// 1. Redistributions of source code must retain the above copyright notice, this list of
//    conditions and the following disclaimer.
// 
// 2. Redistributions in binary form must reproduce the above copyright notice, this list
//    of conditions and the following disclaimer in the documentation and/or other
//    materials provided with the distribution.
// 
// 3. Neither the name of the copyright holder nor the names of its contributors may be
//    used to endorse or promote products derived from this software without specific
//    prior written permission.
// 
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF
// MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL
// THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
// SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
// INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,
// STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF
// THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//
// REQUIREMENTS:
// brew install boost (macos)

// COMPILE w. CUDA:      nvcc dynex.cu -o dynex_gpu -std=c++17 -O4 

#include <memory.h>
#include <stdio.h>
#include <stdlib.h>
#include <signal.h>
#include <math.h>
#include <stdbool.h>
#include <locale.h>
#include <random>
#include <iostream>

/* BOOST */ /// 
#include <iostream>
#include <vector> //?
#include <boost/array.hpp>
#include <boost/numeric/odeint.hpp>
#include <boost/algorithm/clamp.hpp>
#include "boost/date_time/posix_time/posix_time.hpp"

void     INThandler(int);
bool     apply_restart = false;

#define         PARAM_COUNT     9       //number of parameters
#define         LBH     0               //lower bound hard
#define         HBH     1               //higher bound hard
#define         LBS     2               //lower bound soft
#define         HBS     3               //higher bound soft
#define         ALPHA   0
#define         BETA    1
#define         GAMMA   2
#define         DELTA   3
#define         EPSILON 4
#define         ZETA    5
#define         ERR1    6
#define         ERR2    7
#define         INITDT  8
#define         ODE_CONSTANT        1
#define         ODE_CUSTOM_ADAPTIVE 2
#define         ODE_RUNGEKUTTA      3
#define         ODE_IMPLICIT        4
#define         ODE_ONE_STEP        5
#define         ODE_LOGFILE         "log.csv"
#define         TUNE_LOGFILE        "tuninglog.csv"
#define         PARTABLE_FILE       "partable.txt"
#define         SOLUTION_FILE       "solution.txt"
#define         FLOWVECTOR_FILE     "flowvector.csv"

int THREAD_COUNT = 8;

//-------------------------------------------------------------------------------------------------------------------
// precision (attention: change from double needs to update AtomicAddd, too!)
typedef double TFloat; //precision of cuda vars
//typedef float TFloat; //precision of cuda vars

typedef TFloat value_type;
typedef std::vector< value_type > state_type; //we use a vector which has dynamic size

//--------------------------------------------------------------------------------------------------------------
/* SETTINGS & PARAMETERS */
bool   quiet          = false; //no screen output
bool   flowvector_log = false; //output entire FLOWVECTOR_FILE (only thread 0)
bool   load_partable  = true;  //load partable.txt
bool   writelogfile   = false; //write logfile ODE_LOGFILE
bool   coupledsystem  = false; //create a coupled circuit  !!! TODO: NEEDS TO BE FIXED AND CHECKED
bool   loadsolution   = false; 

// Equations - constants:
__device__ TFloat vmin = -1.0;  // voltage lower bound - solved by ./dmm_gpu -i red_8bit_10rds_cut3.cnf -s 0.025 -c 1.25 -k 0.0125 (sometimes)
__device__ TFloat vmax =  1.0;  // voltage upper bound

double dmm_alpha      = 5.0;    // -c  growth rate for long term memory Xl
double dmm_beta       = 20.0;   // -b  growth rate for short term memory Xs
double dmm_gamma      = 0.25;   // -n  restriction for Cm in short term memory Xs
double dmm_delta      = 0.05;   // -h  restriction for Cm in long term memory Xl
double dmm_epsilon    = 0.1;    // -j  remove spurious solution X,s,m = 0
double dmm_zeta       = 0.1;    // -k  reduction factor of rigidity G (learning rate) 10^-3; for ratio>=6: 10^-1 (0.1)
int    seed           = 1;      // -l  random seed value (for initial assignment)
int    xl_max         = 10000;  // -m  10^4 M (x count clauses will be applied automatically - ODE should NEVER reach this value)

// ODE settings:
int INTEGRATION_MODE = ODE_CONSTANT;

// Constant integration params:
double stepsize             = 0.15; //15; // 0.15; //0.015; //0.15; // 0.0078125;        //2^-7
double timeout              = INT_MAX; //max simulated time; stops at reaching it
double walltime_timeout     = INT_MAX; 
double walltime_abs_timeout = INT_MAX;

// Runge-Kutta Adaptive params:
double rk_errorrate_1       = 0.52; // 1.0e-5; //both 0.1 for CBS_k3_n100_m403_b10_3.cnf 
double rk_errorrate_2       = 0.10;
double init_dt              = 0.0078125;; //2^-7
double maxsteps             = INT_MAX; 

// tuneing options:
bool   tune                 = false;
double switchfraction       = 0.0001; 
int    tune_mode            = 0; // 0 = always from -1 assignments; 1 = continous 
int    tune_mode_params     = 2; // 0 = alpha..zeta, 1=ODE params, 2=all params
int    tune_global;

// custom adaptive (experiemental) params:
double adaptive_min         = 0.0078125; //2^-7   0.0078125
double adaptive_max         = 1000; //1000;        //10^3
double step_error           = 5.5;//5.5; //0.001;

// Additional ODE heuristics:
bool         heuristics     = true;
double       alpha_increase = 1.1; //1.1; //increase factor for dmm_alpha_m
double       alpha_decrease = 0.9; //0.9; //decrease factor for dmm_alpha_m
int          alpha_correction = 10000; //10000; //reallocation of dmm_alpha_m every alpha_correction steps
int          alpha_resetvalue = 5.00; //1.00; //after maximum reached, this will be the new alpha

//-------------------------------------------------------------------------------------------------------------------
/* VARIABLES */
bool            debug = false;
void            INThandler(int);       // ctrl-c handler
#define         max_lits_system        10 //10   //3;   //max k-SAT

int             * cls;                 //stores the clauses (max_lits_system columns)
int             * occurrence;          //occurences of each litint             * occurenceCounter;    //counter for allocation
int             * numOccurrenceT;      //number of occurence of a lit
int             * clauseSizes;         //number of literals of a clause
int             * occurenceCounter; 

int             maxNumOccurences = 0;  //max #occurence in clauses of a var
int             n;                     //number of variables
int             m;                     //number of clauses
int             solved;                //is formula solved? UNSAT = 0; SAT = 1;
int             global;                //current lowest loc over all threads
int             global_all_runs;       //best global over all runs (while tuning or restarting f.e.)
TFloat          * v_best;              //G_field assignment of global local minima for each thread
char            input_filename[256];

/* thread specific vars */
int             * loc_thread;           //current local minima of thread (currently)
int             * global_thread;        //global local minima of thread
int             * global_all_runs_thread; //global local minimum of thread over all runs
double          * time_thread;          //simtime of thread (better loc)
double          * time_thread_actual;   //simtime of thread (currently)
double          * walltime_thread;      //walltime of thread (better loc)
TFloat          * initial_assignments;  //initial assignment for each thread
double          * thread_params;        //parameters for each thread
double          * t_begin_thread;       //starting time of thread
double          * t_end_thread;         //current time of thread
int             global_best_thread;     //thread# which has currently best global;

double          * partable;             //if partable.txt is provided, here are the bounds
double          * defaults;             //if partable.txt is provided, here are the default values (max 128 threads)
bool            partable_loaded = false; 

int             stepcounter;

struct node {
    int id;                 //thread-id
    int *model;             //current assignment
    int *temporal;          //temp assignment for oracle (not used in production)
    int *optimal;           //best assignment and solution afterwards
};

double t_begin;
double t_end; 
double t_abs_begin;
double t_abs_end;

TFloat t_rem;
//-------------------------------------------------------------------------------------------------------------------
/* CUDA vars */
TFloat * d_x;// state variables (V,Xs,Xl) 
TFloat *h_v; 
TFloat * d_x_tmp; // temporary state variables (V,Xs,Xl) for adaptive step
TFloat * d_dxdt; // derivative
int * d_cls;
int * d_clauseSizes;
int * d_loc; 
int h_loc[1];
int * d_varchanges;
int h_varchanges[1];
float * d_energy; 
float h_energy[1];
//-------------------------------------------------------------------------------------------------------------------

/* color definitions */
#define TEXT_DEFAULT  "\033[0m"
#define TEXT_YELLOW   "\033[1;33m"
#define TEXT_GREEN    "\033[1;32m"
#define TEXT_RED      "\033[1;31m"
#define TEXT_BLUE     "\033[1;34m"
#define TEXT_CYAN     "\033[1;36m"
#define TEXT_WHITE    "\033[1;37m"
#define TEXT_SILVER   "\033[1;315m" 


//-------------------------------------------------------------------------------------------------------------------
//-------------------------------------------------------------------------------------------------------------------
//                                                    CUDA                                                         //
//-------------------------------------------------------------------------------------------------------------------
//-------------------------------------------------------------------------------------------------------------------

//
#define gpuErrorCheck(ans, abort) { gpuAssert((ans), __FILE__, __LINE__, abort); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
    if(code != hipSuccess) {
        fprintf(stderr,"assert: %s %s %d\n", hipGetErrorString(code), file, line);
        if(abort) {
        exit(code);
        }
    }
}
//
// GPU Helper function: atomicAdd for double:
__device__ double atomicAddd(double* address, double val)
{
    unsigned long long int* address_as_ull =
                                          (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, 
                        __double_as_longlong(val + 
                        __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}

//-------------------------------------------------------------------------------------------------------------------
// -o 1 forward Euler
// GPU kernel: x=>dxdt=>x 
__global__
void gpu_euler(TFloat * d_x_tmp, TFloat * d_x, TFloat * d_dxdt, int size, TFloat t, double h, int n, int m, int xl_max, int * d_varchanges) {
    
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
         i < size; 
         i += blockDim.x * gridDim.x) 
      {

    	// euler step:
        d_x_tmp[i] = d_x[i] + h * d_dxdt[i]; 
        
        // bounded variables:
        if (i<n) {
            if (d_x_tmp[i]<-1.0) d_x_tmp[i] = -1.0;
            if (d_x_tmp[i]> 1.0) d_x_tmp[i] =  1.0;
        }
        if (i>=n && i<n+m) {
            if (d_x_tmp[i]<0.0) d_x_tmp[i] = 0.0;
            if (d_x_tmp[i]>1.0) d_x_tmp[i] = 1.0;   
        }
        if (i>=n+m*2) {
            if (d_x_tmp[i]<1.0) d_x_tmp[i] = 1.0;
            if (d_x_tmp[i]>xl_max) d_x_tmp[i] = xl_max;   
        }
        
	// change of var > x? increase counter d_varchanges
        if (fabs(d_x[i]-d_x_tmp[i])>=1.0) {
      		atomicAdd(&d_varchanges[0], 1); 
      	}
    }
    
}
//-------------------------------------------------------------------------------------------------------------------
// GPU kernel: reset d_loc (set to m), d_energy and d_varchanges
__global__
void gpu_reset_vars(int * d_loc, float * d_energy, int * d_varchanges, int m) {
    d_loc[0] = m;
    d_energy[0] = 0.0;
    d_varchanges[0] = 0;
}
//-------------------------------------------------------------------------------------------------------------------
// GPU kernel: reset dxdf, set all values to 0.00
__global__
void gpu_reset_dxdt(TFloat * d_dxdt, int size) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
         i < size; 
         i += blockDim.x * gridDim.x) 
      {
        d_dxdt[i] = 0.0;
      }
}

//-------------------------------------------------------------------------------------------------------------------
// GPU kernel: reset x, set all values to 0.00
__global__
void gpu_reset_x(TFloat * d_x, int size) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
         i < size; 
         i += blockDim.x * gridDim.x) 
      {
        d_x[i] = 0.0;
      }
}

//-------------------------------------------------------------------------------------------------------------------
// GPU kernel: do a step => generate dxdt & update d_loc
__global__
void gpu_step(TFloat * d_x, TFloat * d_dxdt, int * d_cls, int * d_loc, float * d_energy, int m, int n, double xl_max, double m_alpha, double m_beta, double m_gamma, double m_delta, double m_epsilon, double m_zeta, int * d_clauseSizes) {
    for (int clause = blockIdx.x * blockDim.x + threadIdx.x; 
         clause < m; 
         clause += blockDim.x * gridDim.x) 
      {
        
        TFloat Qi = (d_cls[clause*max_lits_system+0]>0)? 1.0:-1.0; // +1 if literal is >0, otherwise -1
        TFloat Qj = (d_cls[clause*max_lits_system+1]>0)? 1.0:-1.0; // +1 if literal is >0, otherwise -1
        TFloat Qk = (d_cls[clause*max_lits_system+2]>0)? 1.0:-1.0; // +1 if literal is >0, otherwise -1
        TFloat C;
        TFloat Xs = d_x[clause+n]; if (Xs<0.0) Xs=0.0; if (Xs>1.0) Xs=1.0; //boundary for xs € [0,1]:
        TFloat Xl = d_x[clause+n+m]; if (Xl<1.0) Xl=1.0; if (Xl>xl_max) Xl=xl_max; //boundary for xl € [1,10⁴M]:

        //k-sat implementation:
        int k = d_clauseSizes[clause];
        
        if (k==3) {
            int liti = abs(d_cls[clause*max_lits_system+0]);
            int litj = abs(d_cls[clause*max_lits_system+1]);
            int litk = abs(d_cls[clause*max_lits_system+2]);
            TFloat Vi = d_x[liti-1]; if (Vi<vmin) Vi=vmin; if (Vi>vmax) Vi=vmax; //boundary for v € [-1,1]:
            TFloat Vj = d_x[litj-1]; if (Vj<vmin) Vj=vmin; if (Vj>vmax) Vj=vmax; //boundary for v € [-1,1]:
            TFloat Vk = d_x[litk-1]; if (Vk<vmin) Vk=vmin; if (Vk>vmax) Vk=vmax; //boundary for v € [-1,1]:
            TFloat i = 1.0-Qi*Vi;
            TFloat j = 1.0-Qj*Vj;
            TFloat k = 1.0-Qk*Vk;
            C = fmin(i, fmin(j, k));
            C = C / 2.0 ;
            if (C<0.0) C=0.0;
            if (C>1.0) C=1.0;
            //voltages:
            TFloat Gi = Qi * fmin(j,k) / 2.0;
            TFloat Gj = Qj * fmin(i,k) / 2.0;
            TFloat Gk = Qk * fmin(i,j) / 2.0;
            TFloat Ri, Rj, Rk;
            if (C != i/2.0 ) {Ri = 0.0;} else {Ri = (Qi - Vi) / 2.0;}
            if (C != j/2.0 ) {Rj = 0.0;} else {Rj = (Qj - Vj) / 2.0;}
            if (C != k/2.0 ) {Rk = 0.0;} else {Rk = (Qk - Vk) / 2.0;}
            atomicAddd(&d_dxdt[liti-1], (Xl * Xs * Gi + (1.0 + m_zeta * Xl) * (1.0 - Xs) * Ri) );
            atomicAddd(&d_dxdt[litj-1], (Xl * Xs * Gj + (1.0 + m_zeta * Xl) * (1.0 - Xs) * Rj) );
            atomicAddd(&d_dxdt[litk-1], (Xl * Xs * Gk + (1.0 + m_zeta * Xl) * (1.0 - Xs) * Rk) );

        }
        if (k==2) {
            int liti = abs(d_cls[clause*max_lits_system+0]);
            int litj = abs(d_cls[clause*max_lits_system+1]);
            TFloat Vi = d_x[liti-1]; if (Vi<vmin) Vi=vmin; if (Vi>vmax) Vi=vmax; //boundary for v € [-1,1]:
            TFloat Vj = d_x[litj-1]; if (Vj<vmin) Vj=vmin; if (Vj>vmax) Vj=vmax; //boundary for v € [-1,1]:
            TFloat i = 1.0-Qi*Vi;
            TFloat j = 1.0-Qj*Vj;
            C = fmin(i, j);
            C = C / 2.0;
            if (C<0.0) C=0.0;
            if (C>1.0) C=1.0;
            //voltages:
            TFloat Gi = Qi * j / 2.0;
            TFloat Gj = Qj * i / 2.0;
            TFloat Ri, Rj;
            if (C != i/ 2.0 ) {Ri = 0.0;} else {Ri = (Qi - Vi) / 2.0;}
            if (C != j/ 2.0 ) {Rj = 0.0;} else {Rj = (Qj - Vj) / 2.0;}
            atomicAddd(&d_dxdt[liti-1], (Xl * Xs * Gi + (1.0 + m_zeta * Xl) * (1.0 - Xs) * Ri) );
            atomicAddd(&d_dxdt[litj-1], (Xl * Xs * Gj + (1.0 + m_zeta * Xl) * (1.0 - Xs) * Rj) );
        }
        if (k!=3 && k!=2) {
            int lit[max_lits_system];
            TFloat Q[max_lits_system], V[max_lits_system], _i[max_lits_system], R[max_lits_system], G[max_lits_system];
            TFloat c_min=INT_MAX;
            for (int i=0; i<k; i++) {
                Q[i] = (d_cls[clause*max_lits_system+i]>0)? 1.0:-1.0; // +1 if literal is >0, otherwise -1
                lit[i] = abs(d_cls[clause*max_lits_system+i]);
                V[i] = d_x[lit[i]-1]; if (V[i]<vmin) V[i]=vmin; if (V[i]>vmax) V[i]=vmax; //boundary for v € [-1,1]:
                _i[i] = 1.0-Q[i]*V[i];
                // find min:
                if (_i[i]<c_min) c_min = _i[i]; 
            }
            C = c_min / 2.0;
            //voltages:            
            for (int i=0; i<k; i++) {
                //find min of others:
                TFloat g_min = INT_MAX;
                for (int x=0; x<k; x++) {if (x!=i && _i[x]<g_min) g_min = _i[x];}
                G[i] = Q[i] * g_min / 2.0;
                TFloat comp = (1.0-Q[i]*V[i])/2.0;
                if (C != comp) {R[i] = 0.0;} else {R[i] = (Q[i] - V[i]) / 2.0;}
                atomicAddd(&d_dxdt[lit[i]-1], (Xl * Xs * G[i] + (1.0 + m_zeta * Xl) * (1.0 - Xs) * R[i]) );
            }    
        }

        //update #satsified? 
        if (C<0.5) atomicAdd(&d_loc[0], -1); //this clause is sat
        //update energy:
        atomicAdd(&d_energy[0], C);
        // Calculate Xs:
        d_dxdt[n+clause] = m_beta * (Xs + m_epsilon) * (C - m_gamma);
        
        // Calculate Xl:
        d_dxdt[n+m+clause] = m_alpha * (C - m_delta);

    }
    
}

//-------------------------------------------------------------------------------------------------------------------
// this functions solves the CNF with GPU
int solveGPU() {

    printf("c [GPU] STARTING INTEGRATION AT t=%.5f\n",t_rem);
    int threadsPerBlock    = 256;
    int blocksPerGrid      =  ( (n+m*2) + threadsPerBlock - 1 ) / threadsPerBlock;// + 1;
    int threadsPerBlock_m  = 256;
    int blocksPerGrid_m    = ( m + threadsPerBlock_m - 1 ) / threadsPerBlock_m;// + 1;
    int threadsPerBlock_n  = 256;
    int blocksPerGrid_n    = ( n + threadsPerBlock_n - 1 ) / threadsPerBlock_n;// + 1;

    // initiate d_x;
    gpuErrorCheck(hipMemcpy(d_x, initial_assignments, (n+m*2)*sizeof(TFloat), hipMemcpyHostToDevice),true); //only if seed is not 0; seed 0 sets everything to zero
    printf("c [GPU] INITIAL ASSIGNMENTS: %.5f, %.5f, %.5f, %.5f, %.5f\n",initial_assignments[0], initial_assignments[1], initial_assignments[2], initial_assignments[3], initial_assignments[4]);
     
    int integration_steps;
    int no_improvement_since;
    TFloat t;
    double t_begin, t_end;
    double global_energy = m;

    printf("c [GPU] STARTING INTEGRATION...\n");    
    printf("c [GPU] PARAMETERS: α=%.15f β=%.15f γ=%.15f ε=%.15f δ=%.15f ζ=%.15f\n",dmm_alpha, dmm_beta, dmm_gamma, dmm_delta, dmm_epsilon, dmm_zeta);
    printf("c [GPU] INTEGRATION MODE: %d\n",INTEGRATION_MODE);
    printf("c [GPU] STARTING STEPSIZE=%.15f\n",stepsize);

    solved = 0;
    global = m;
    t = t_rem;

    integration_steps = 0;
    no_improvement_since = 0;

    t_begin = clock(); 
    t_abs_begin = t_begin;

    int corr_best = 0;
    
    gpu_reset_vars<<<1,1>>>(d_loc, d_energy, d_varchanges, m); // init loc = m:
    
    /// main integration routine: ///
    while (solved == 0) {
        
        // do one step (updates d_dxdt, d_loc[0]):
        gpu_reset_dxdt<<<blocksPerGrid,threadsPerBlock>>>(d_dxdt, n+m*2); // kernel reset dxdf = all zero:
        gpu_reset_vars<<<1,1>>>(d_loc, d_energy, d_varchanges, m); // init loc = m:
        gpu_step<<<blocksPerGrid_m,threadsPerBlock_m>>>(d_x, d_dxdt, d_cls, d_loc, d_energy, m, n, xl_max, dmm_alpha, dmm_beta, dmm_gamma, dmm_delta, dmm_epsilon, dmm_zeta, d_clauseSizes);
        
        // only periodically update screen etc:
        if (integration_steps % 1 == 0)  {
		
		// update time:
		t_end = clock(); 
		double time_spent = (double)(t_end - t_begin)/CLOCKS_PER_SEC;//1000;// / 
		t_abs_end = clock();
		double time_abs_spent = (double)(t_abs_end - t_abs_begin)/CLOCKS_PER_SEC;//1000;// / 

		//get loc:
		hipMemcpy(h_loc, d_loc, sizeof(int), hipMemcpyDeviceToHost);

		//get energy:
		hipMemcpy(h_energy, d_energy, sizeof(float), hipMemcpyDeviceToHost);
		
		//screen output:
		if (tune) {
		    printf("\rc [GPU] %.2fs \tt=%.5f \tglobal=%6d (%6d) [T:%6d] steps=%9d (+%.5f) α=%.5f β=%.5f γ=%.5f ε=%.5f δ=%.5f ζ=%.5f E=%.5f ", time_spent,t,global,h_loc[0],tune_global,integration_steps,stepsize,dmm_alpha, dmm_beta, dmm_gamma, dmm_delta, dmm_epsilon, dmm_zeta, h_energy[0]);    
		} else {
		   printf("\rc [GPU] %.2fs \tt=%.5f \tglobal=%6d (%6d) steps=%9d (+%.5f) α=%.5f β=%.5f γ=%.5f ε=%.5f δ=%.5f ζ=%.5f E=%.5f (%.5f) ", time_spent,t,global,h_loc[0],integration_steps,stepsize,dmm_alpha, dmm_beta, dmm_gamma, dmm_delta, dmm_epsilon, dmm_zeta, h_energy[0], global_energy);    
		}
		
		// solved?
		if (h_loc[0]==0) {
		    solved = 1;
		    break;
		}
		// better loc?
		if (h_loc[0]<global) {
		    global = h_loc[0];
		    if (!tune) printf("\n");
		    if (global<2000 & (!tune || (tune && global<tune_global))) {
		        //write solution to file:
		        hipMemcpy(h_v, d_x, n*sizeof(TFloat), hipMemcpyDeviceToHost);
		        FILE *fs = fopen(SOLUTION_FILE, "w");
		        for (int i=0; i<n; i++) fprintf(fs,"%.32f, ",h_v[i]); // current G_field = solution
		        fclose(fs);       
		    }
		    if (tune && tune_mode==1) {
		        hipMemcpy(initial_assignments, d_x, n*sizeof(TFloat), hipMemcpyDeviceToHost);
		    }                
		    // restart counters reset after better loc found:
		    no_improvement_since = 0;
		    if (!tune) t_begin = clock(); 
		} else {
			// better energy?
			if (h_energy[0] < global_energy) {
				global_energy = h_energy[0];
				if (!tune) printf("\n");
			}
	    	}

		// exit on: walltime_timeout, maxsteps (no improvement since)
		if (time_spent>walltime_timeout || no_improvement_since>maxsteps || (tune && time_abs_spent>walltime_abs_timeout)){
		    printf(TEXT_SILVER);
		    if (no_improvement_since>maxsteps) printf("\nc [GPU] BREAK ON NO IMPROVEMENTS SINCE LIMIT %d\n",no_improvement_since);
		    if (time_spent>walltime_timeout) printf("\nc [GPU] BREAK ON WALLTIME LIMIT %.0fs\n",walltime_timeout);
		    if (time_abs_spent>walltime_abs_timeout) printf("\nc [GPU] BREAK ON WALLTIME ABS LIMIT %.0fs\n",walltime_abs_timeout);
		    printf(TEXT_DEFAULT);
		    std::cout << "c integration steps=" << integration_steps << std::endl;
		    return global;
		}
	}

        /// apply rhs: ------------------------------------------------------------------------------------------------------------------------------
        t = t + stepsize;

        /// CUSTOM FORWARD ADAPTIVE EULER: ////////////////////////////////////////
        bool adaptive_accepted = false;
        TFloat h_min = 0.0078125;
        TFloat h_max = 1.0;
        stepsize = 0.125;
        while (!adaptive_accepted) {
	        gpu_reset_vars<<<1,1>>>(d_loc, d_energy, d_varchanges, m); // reset d_varchanges -> WE SHOULD HAVE ITS OWN ROUTINE HERE!
        	gpu_euler<<<blocksPerGrid,threadsPerBlock>>>(d_x_tmp, d_x, d_dxdt, n+m*2, t, stepsize, n, m , xl_max, d_varchanges); // MOVE ENTIRELY TO CUDA
        	//get #changed vars:
		hipMemcpy(h_varchanges, d_varchanges, sizeof(int), hipMemcpyDeviceToHost);
		if (h_varchanges[0]==0) adaptive_accepted=true;
		stepsize = stepsize * 1/2;
		if (stepsize<=h_min) {
			stepsize = h_min;
			adaptive_accepted=true;
		}
        }
        /// --- CUSTOM FORWARD //////////////////////////////////////////////////////

        // do the final step:
        gpu_euler<<<blocksPerGrid,threadsPerBlock>>>(d_x, d_x, d_dxdt, n+m*2, t, stepsize, n, m , xl_max, d_varchanges); //we can optimize -> reuse from adaptive
        integration_steps++;
        no_improvement_since++;
    	}
    /// --- 
    std::cout << "c integration steps=" << integration_steps << std::endl;
    
    // output solution:
    hipMemcpy(h_v, d_x, n*sizeof(TFloat), hipMemcpyDeviceToHost);

    printf(TEXT_YELLOW); printf("v [GPU] ");
    for (int i=0; i<n; i++) {
        if (h_v[i]>0) printf("%d ",i+1);
        if (h_v[i]<0) printf("%d ",(i+1)*-1);
        if (h_v[i]==0) printf("%d ",(i+1));
    }
    printf("\n"); printf(TEXT_DEFAULT);
    
    // verify solution:
    printf("\nc [GPU] VERIFYING...\n"); printf(TEXT_DEFAULT);
    bool sat = true; bool clausesat;
    for (int i=0; i<m; i++) {
        for (int j=0; j<clauseSizes[i]; j++) {
            clausesat = false;
            int lit = abs(cls[i*max_lits_system+j]);
            if ( (h_v[lit-1]>0 && cls[i*max_lits_system+j]>0) || (h_v[lit-1]<0 && cls[i*max_lits_system+j]<0) || (h_v[lit-1]==0 && cls[i*max_lits_system+j]>0) ) {
                clausesat = true;
                break;
            }
        }
        if (!clausesat) {
            sat = false;
            //output wrong assignment:
            printf("CLAUSE %d [",i);
            for (int j=0; j<clauseSizes[i]; j++) printf("%d ",cls[i*max_lits_system+j]);
            printf("] -> ");
        	for (int j=0; j<clauseSizes[i]; j++) printf("%.2f ",h_v[abs(cls[i*max_lits_system+j])]);
        	printf(" IS NOT SAT.\n");
            
            break;
        }
    }
    if (sat)  {
        printf(TEXT_YELLOW); printf("c [GPU] SAT (VERIFIED)\n"); solved = 1;
        //write solution to file:
        FILE *fs = fopen(SOLUTION_FILE, "w");
        for (int i=0; i<n; i++) fprintf(fs,"%.15f, ",h_v[i]); // current G_field = solution
        fclose(fs);
    }
    if (!sat) {printf(TEXT_RED); printf("c [GPU] UNSAT (VERIFIED)\n");}
    printf(TEXT_DEFAULT);

    return 0;
}
//-------------------------------------------------------------------------------------------------------------------
// run on GPU and tune on the go:
void tuneGPU() {
    tune_global = m;
    int tune_loc = m;
    int PC = 7;
    TFloat rem_tune_param[PC];
    int    rem_tune_param_idx[PC];
    int tune_iteration = 0;
    int tune_no_improvement_since = 0;
    bool tune_change_params = false;

    /* RNG std::uniform_real_distribution<double> */
    std::mt19937 generator(seed);
    std::uniform_int_distribution<int>      rand_param(0,PC-1); //which params to change TEST: DONT CHANGE STEPSIZE - BREAKS THE FLOW FIELD?
    std::lognormal_distribution<double>     rand_num_vars(0.0,0.5);
    std::uniform_real_distribution<double>      rand_alpha(1e-16, 1e2);
    std::uniform_real_distribution<double>      rand_beta(1e-16, 1e2);
    std::uniform_real_distribution<double>      rand_gamma(1e-16, 0.5);
    std::uniform_real_distribution<double>      rand_delta(1e-16, 1.0);
    std::uniform_real_distribution<double>      rand_epsilon(1e-16, 0.5);
    std::uniform_real_distribution<double>      rand_zeta(1e-16, 1.0);
    std::uniform_real_distribution<double>      rand_stepsize(1e-8, 0.15);
    
    std::uniform_int_distribution<int>  rand_v(-1.0, 1.0);
    std::uniform_real_distribution<double>  rand_Xs(0.0, 1.0);
    std::uniform_real_distribution<double>  rand_Xl(1.0, 10.0); //xl_max); //10.00

    std::uniform_int_distribution<int>      rand_choosevar(0,n); //+m*2); //variable choosen from switchfraction
    std::uniform_int_distribution<int>      rand_choosstrategy(0,1); //tuning strategy
    std::uniform_real_distribution<double>  rand_smallstep(0.985111111, 1.015111111);

    while (solved==0) {
        printf("c ------------------------------------------------------------------------------------\n");
        printf("c ITERATION %d (NO IMPROVEMENT SINCE %d)\n",tune_iteration, tune_no_improvement_since);

        // move params => rem_tune_param:
        rem_tune_param[0] = dmm_alpha;
        rem_tune_param[1] = dmm_beta;
        rem_tune_param[2] = dmm_gamma;
        rem_tune_param[3] = dmm_delta;
        rem_tune_param[4] = dmm_epsilon;
        rem_tune_param[5] = dmm_zeta;
        rem_tune_param[6] = stepsize;
        //empty rem_tune_param_idx:
        for (int i=0; i<PC; i++) rem_tune_param_idx[i] = 0;

        if (tune_change_params) {
            // how many parameters do we change?
            double _num_change_vars = rand_num_vars(generator);
            int num_change_vars = (int)_num_change_vars; 
            if (num_change_vars<1) num_change_vars = 1;
            if (num_change_vars>PC) num_change_vars = PC;

            for (int p=0; p<num_change_vars; p++) {
                int tune_param = rand_param(generator);
                // mark the tuned param:
                rem_tune_param_idx[tune_param] = 1;
                // choose tuning strategy:
                int strategy = rand_choosstrategy(generator);
                if (strategy>0) {
                    // small move:
                    double smallstep = rand_smallstep(generator);
                    switch (tune_param) {
                        case 0: dmm_alpha =     dmm_alpha*smallstep; break;
                        case 1: dmm_beta =      dmm_beta*smallstep; break;
                        case 2: dmm_gamma =     dmm_gamma*smallstep; break;
                        case 3: dmm_delta =     dmm_delta*smallstep; break;
                        case 4: dmm_epsilon =   dmm_epsilon*smallstep; if (dmm_epsilon>=dmm_gamma) dmm_epsilon = dmm_gamma - 1e-8; break;
                        case 5: dmm_zeta =      dmm_zeta*smallstep; break;
                        case 6: stepsize =      stepsize*smallstep; break;
                    }    
                }
                if (strategy==0) {
                    // random point
                    switch (tune_param) {
                        case 0: dmm_alpha =     rand_alpha(generator); break;
                        case 1: dmm_beta =      rand_beta(generator); break;
                        case 2: dmm_gamma =     rand_gamma(generator); break;
                        case 3: dmm_delta =     rand_delta(generator); break;
                        case 4: dmm_epsilon =   rand_epsilon(generator); if (dmm_epsilon>=dmm_gamma) dmm_epsilon = dmm_gamma - 1e-8; break;
                        case 5: dmm_zeta =      rand_zeta(generator); break;
                        case 6: stepsize =      rand_stepsize(generator); break;
                    }    
                }
                
                printf(TEXT_SILVER);
                switch (tune_param) {
                    case 0: printf("c STRATEGY %d CHANGED α TO %.15f \n", strategy, dmm_alpha); break;
                    case 1: printf("c STRATEGY %d CHANGED β TO %.15f \n", strategy, dmm_beta); break;
                    case 2: printf("c STRATEGY %d CHANGED γ TO %.15f \n", strategy, dmm_gamma); break;
                    case 3: printf("c STRATEGY %d CHANGED ε TO %.15f \n", strategy, dmm_delta); break;
                    case 4: printf("c STRATEGY %d CHANGED δ TO %.15f \n", strategy, dmm_epsilon); break;
                    case 5: printf("c STRATEGY %d CHANGED ζ TO %.15f \n", strategy, dmm_zeta); break;
                    case 6: printf("c STRATEGY %d CHANGED stepsize TO %.15f \n", strategy, stepsize); break;
                }
                printf(TEXT_DEFAULT);
            }    
        }
        
        // run GPU integration:
        tune_loc = solveGPU();
        
        tune_change_params = true;

        // did we improve? (we also accept similar loc as this deviates the parameter space a bit)
        if (tune_loc<tune_global) {
            if (tune_loc<tune_global) tune_no_improvement_since = 0;
            tune_global = tune_loc;
            printf(TEXT_YELLOW);
            printf("c LOC=%d ACCEPTED.\n",tune_loc);
            printf(TEXT_DEFAULT);
            //tune_change_params = false;
        } else {
            printf(TEXT_SILVER);
            printf("c LOC=%d (GLOBAL=%d) - REJECTED\n",tune_loc, tune_global);
            printf(TEXT_DEFAULT);
            // reset parameters:
            for (int p=0; p<PC; p++) {
                if (rem_tune_param_idx[p]==1) {
                    switch (p) {
                        case 0: dmm_alpha = rem_tune_param[0]; break;
                        case 1: dmm_beta = rem_tune_param[1]; break;
                        case 2: dmm_gamma = rem_tune_param[2]; break;
                        case 3: dmm_delta = rem_tune_param[3]; break;
                        case 4: dmm_epsilon = rem_tune_param[4]; break;
                        case 5: dmm_zeta = rem_tune_param[5]; break;
                        case 6: stepsize = rem_tune_param[6]; break;
                    }
                }
            }
        }
        
        // adjust walltime_timeout? only if not continous...
        if (tune_no_improvement_since>=50) {//25) {
            tune_no_improvement_since = 0;
            walltime_timeout += 1.0;
            walltime_abs_timeout += 1.0;
            tune_change_params = false; // we increased the time, so we want to see if it gets better with the current (best) params...
            printf(TEXT_CYAN);
            printf("c UPDATED walltime_timout TO %.2fs, walltime_abs_timeout TO %.2fs\n",walltime_timeout,walltime_abs_timeout);
            printf(TEXT_DEFAULT);
        }

        tune_iteration++;
        tune_no_improvement_since++;

        // switchfraction:
        double _switchvars = n * switchfraction;
        int switchvars = abs(_switchvars);
        for (int i=0; i<switchvars; i++) {
            int _var = rand_choosevar(generator);
            if (_var<n) initial_assignments[_var] = rand_v(generator);
            if (_var>=n && _var < n+m) initial_assignments[_var] = rand_Xs(generator);
            if (_var>=n+m) initial_assignments[_var] = rand_Xl(generator);
        }
        printf("c SWITCHFRACTION CHANGED %d VARS\n",switchvars);

    }
    printf("c FINISHED. SOLUTION FOUND.\n");
    
}

//-------------------------------------------------------------------------------------------------------------------
// keyboard runtime menu
void  INThandler(int sig)
{
    signal(SIGINT, INThandler);

}


//-------------------------------------------------------------------------------------------------------------------
//parse command line options
int scan_opt(int argc, char **argv, const char *opt) {
    char c;
    while ((c = getopt (argc, argv, opt)) != -1)
        switch (c) {
            case 't': tune=atoi(optarg); break;
            case 'w': THREAD_COUNT=atoi(optarg); break;
            case 'q': quiet=atoi(optarg); break;
            case 'o': INTEGRATION_MODE=atoi(optarg); break;
            case 'i': strcpy(input_filename, optarg); break;
            case 'd': init_dt=atof(optarg); break;
            case 'x': rk_errorrate_1=atof(optarg); break;
            case 'y': rk_errorrate_2=atof(optarg); break;
            case 'z': maxsteps=atof(optarg); break;
            case 's': stepsize=atof(optarg); break;
            case 'a': timeout   =atof(optarg); break;
            
            case 'e': load_partable=atoi(optarg); break;
            case 'f': adaptive_max   =atof(optarg); break;
            case 'g': tune_mode_params   =atoi(optarg); break;

            case 'b': dmm_beta=atof(optarg); break;
            case 'c': dmm_alpha=atof(optarg); break;
            case 'n': dmm_gamma=atof(optarg); break;
            case 'h': dmm_delta=atof(optarg); break;
            case 'j': dmm_epsilon=atof(optarg); break;
            case 'k': dmm_zeta=atof(optarg); break;
            case 'l': seed=atoi(optarg); break;
            case 'm': xl_max=atoi(optarg); break;

            case 'p': heuristics=atoi(optarg); break;
            case 'u': walltime_timeout   =atof(optarg); break;

            case 'v': switchfraction   =atof(optarg); break;
            case 'r': tune_mode   =atoi(optarg); break;
            
            default: return(-1);
        }
    return(0);
}

//-------------------------------------------------------------------------------------------------------------------
void printDevProp(hipDeviceProp_t devProp)
{
    printf(TEXT_GREEN);
    printf("c [GPU] Major revision number:         %d\n",  devProp.major);
    printf("c [GPU] Minor revision number:         %d\n",  devProp.minor);
    printf("c [GPU] Name:                          %s\n",  devProp.name);
    printf("c [GPU] Total global memory:           %u\n",  devProp.totalGlobalMem);
    printf("c [GPU] Total shared memory per block: %u\n",  devProp.sharedMemPerBlock);
    printf("c [GPU] Total registers per block:     %d\n",  devProp.regsPerBlock);
    printf("c [GPU] Warp size:                     %d\n",  devProp.warpSize);
    printf("c [GPU] Maximum memory pitch:          %u\n",  devProp.memPitch);
    printf("c [GPU] Maximum threads per block:     %d\n",  devProp.maxThreadsPerBlock);
    for (int i = 0; i < 3; ++i)
    printf("c [GPU] Maximum dimension %d of block:  %d\n", i, devProp.maxThreadsDim[i]);
    for (int i = 0; i < 3; ++i)
    printf("c [GPU] Maximum dimension %d of grid:   %d\n", i, devProp.maxGridSize[i]);
    printf("c [GPU] Clock rate:                    %d\n",  devProp.clockRate);
    printf("c [GPU] Total constant memory:         %u\n",  devProp.totalConstMem);
    printf("c [GPU] Texture alignment:             %u\n",  devProp.textureAlignment);
    printf("c [GPU] Concurrent copy and execution: %s\n",  (devProp.deviceOverlap ? "Yes" : "No"));
    printf("c [GPU] Number of multiprocessors:     %d\n",  devProp.multiProcessorCount);
    printf("c [GPU] Kernel execution timeout:      %s\n",  (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));
    printf(TEXT_DEFAULT);
    return;
}

//-------------------------------------------------------------------------------------------------------------------
int main(int argc, char **argv) {
    int i, j;
    char buffer[32];
    global_best_thread = -1;

    char *syntax =
    "c          GENERAL OPTIONS:\n"
    "c          -i file      : INPUT FILE (CNF)\n"
    "c          -o [1;2;3;4] : ODE INTEGRATION: 1=CONSTANT, 2=CUSTOM ADAPTIVE, 3=RUNGE KUTTA, 4=IMPLICIT (DEFAULT:3)\n"
    "c          -q [0;1]     : QUIET MODE; 0=OFF 1=ON (DEFAULT:0)\n"
    "c          -u [double]  : WALL TIME TIMEOUT AFTER x s\n"
    "c          -w [int]     : NUMBER OF PARALLEL THREADS\n"
    "c          -e [0;1]     : LOAD partable.txt\n"
    "c\n"
    "c          -c [double]  : ALPHA (GROWTH RATE FOR LONG TERM MEMORY Xl)\n"
    "c          -b [double]  : BETA (GROWTH RATE FOR SHORT TERM MEMORY Xs)\n"
    "c          -n [double]  : GAMMA (RESTRICTION FOR CM IN Xs)\n"
    "c          -h [double]  : DELTA (GROWTH RATE FOR CM IN Xl)\n"
    "c          -j [double]  : EPSILON (REMOVE SPURIOUS SOLUTION Xm=0)\n"
    "c          -k [double]  : ZETA (REDUCTION FACTOR OF RIGIDITY G)\n"
    "c          -l [int]     : RANDOM SEED\n"
    "c          -m [int]     : MAX VALUE FOR Xl\n"
    "c\n"
    "c          -p [0;1]     : ALPHA HEURISTICS; 0=OFF 1=ON (DEFAULT:1)\n"
    "c\n"
    "c          TUNING OPTIONS:\n"
    "c          -t [0;1]     : TUNE CIRCUIT; 0=OFF 1=ON (DEFAULT:0)\n"
    "c          -g [0;1;2]   : PARAMS TO TUNE: 0:α..δ 1:ODE 2:ALL\n"
    "c          -r [0;1]     : TUNING MODE: 0=AWAYS FROM START; 1=CONTINOUS FROM BEST; 2=CONTINOUS OVERALL BEST\n"
    "c          -v [double]  : SWITCH FRACTION (DEFAULT 0.0001)\n"
    "c\n"
    "c          ODE - CONSTANT OPTIONS:\n"
    "c          -s [double]  : STEP SIZE\n"
    "c          -a [double]  : TIME OUT (INTEGRATION TIME)\n"
    "c\n"
    "c          ODE - RUNGE KUTTA OPTIONS:\n"
    "c          -x [double]  : ERROR RATE 1\n"
    "c          -y [double]  : ERROR RATE 2\n"
    "c          -d [double]  : INITIAL dt VALUE\n"
    "c          -z [double]  : MAXIMUM INTEGRATION STEPS\n"
    ;
    goto on_continue;

on_break:
    printf("c Syntax: %s <... Args ...>\n", argv[0]);
    printf("c Args:\n");
    printf("%s", syntax);
    printf("\n");
    return EXIT_FAILURE;
    
on_continue:
    if(scan_opt(argc, argv, "r:v:w:u:q:p:b:c:n:h:j:k:l:m:e:f:g:s:a:z:x:y:d:o:t:i:")) goto on_break;

    if (!quiet) printf("c ------------------------------------------------------------------------------------\n");
    if (!quiet) printf("c TURINGX SAT-SOLVER GPU VERSION                                               (C)2022\n");
    if (!quiet) printf("c ------------------------------------------------------------------------------------\n");
    if (!quiet) printf("c INSTANCE  : %s\n", input_filename);

    /// load CNF header:
    FILE *file = fopen(input_filename, "r");
    if (strcmp(buffer, "c") == 0) {
        while (strcmp(buffer, "\n") != 0) {
            fscanf(file, "%s", buffer);
        }
    }
    while (strcmp(buffer, "p") != 0) {
        fscanf(file, "%s", buffer);
    }
    fscanf(file, " cnf %i %i", &n, &m);

    if (coupledsystem) m = (int)(m * 2); 
    
    if (!quiet) printf("c VARIABLES : %'d\n", n);
    if (!quiet) printf("c CLAUSES   : %'d\n", m);
    if (!quiet) printf("c RATIO     : %lf\n", (double) m / n);

    xl_max = xl_max * m;
    if (xl_max<=0) xl_max = INT_MAX;

    /// reserve  memory - needs to be done before anything else:
    cls = (int *) calloc((size_t) m*max_lits_system, sizeof(int));
    for (int i=0; i<m*max_lits_system; i++) cls[i] = 0; 
    numOccurrenceT = (int *) calloc((size_t) n+1, sizeof(int));
    clauseSizes = (int *) calloc((size_t) m, sizeof(int));
        
    /// read CNF: /////////////////////////////////////////
    int lit; int lit_coupled;
    for (i = 0; i < m; i++) {
        j = 0; 
        do {
            fscanf(file, "%s", buffer);
            if (strcmp(buffer, "c") == 0) {
                continue;
            }
            lit = atoi(buffer);
            if (lit!=0) cls[i*max_lits_system+j] = lit;

            if (coupledsystem) {
                if (lit!=0) cls[((int)(m/2)+i)*max_lits_system+j] = lit;    
            }

            // increase number of Occurence of the variable, max number of occurences
            if (lit!=0) {
                numOccurrenceT[abs(lit)]++;
                if (numOccurrenceT[abs(lit)]>maxNumOccurences) {maxNumOccurences=numOccurrenceT[abs(lit)];}
                clauseSizes[i] = j+1;
            }

            if (coupledsystem) {
                if (lit!=0) {
                    numOccurrenceT[abs(lit)]++;
                    clauseSizes[(int)(m/2)+i] = j+1;
                }    
            }

            j++;
        } while (strcmp(buffer, "0") != 0);
        j--;
        if (j > max_lits_system) {
            printf("c ERROR: CLAUSE %d HAS MORE THAN %d LITERALS.\n",i,max_lits_system);
            return EXIT_FAILURE;
        }
    }
    
    if (!quiet) printf("c MAX VARIABLE OCCURENCE: %'d\n", maxNumOccurences);

    if (!quiet) printf("c FIRST 10 CLAUSES:\n");
    for (i = 0; i < 11; i++) {
        if (!quiet) printf("c CLAUSE %i: ",i);
        for (j = 0; j < clauseSizes[i]; j++) {if (!quiet) printf(" %d",cls[i*max_lits_system+j]);}
        if (!quiet) printf(" (%d)",clauseSizes[i]);
        if (!quiet) printf("\n");
    }

    //build occurence array: [var][cls...] 
    occurrence = (int *) calloc((size_t) (n+1)*maxNumOccurences, sizeof(int));
    occurenceCounter = (int *) calloc((size_t) n+1, sizeof(int));
    
    for (i=0; i<m; i++) {
        for (j = 0; j < clauseSizes[i]; j++) {
            lit = abs(cls[i*max_lits_system+j]);
            occurrence[lit*maxNumOccurences+occurenceCounter[lit]] = i;
            occurenceCounter[lit]++;
        }
    }

    /// initialize arrays:
    v_best = (TFloat *) calloc((size_t) THREAD_COUNT*(n+m*2), sizeof(TFloat));
    loc_thread = (int *) calloc((size_t) THREAD_COUNT, sizeof(int));
    global_thread = (int *) calloc((size_t) THREAD_COUNT, sizeof(int));
    global_all_runs_thread = (int *) calloc((size_t) THREAD_COUNT, sizeof(int));
    time_thread = (double *) calloc((size_t) THREAD_COUNT, sizeof(double));
    time_thread_actual = (double *) calloc((size_t) THREAD_COUNT, sizeof(double));
    walltime_thread = (double *) calloc((size_t) THREAD_COUNT, sizeof(double));
    t_begin_thread = (double *) calloc((size_t) THREAD_COUNT, sizeof(double));
    t_end_thread = (double *) calloc((size_t) THREAD_COUNT, sizeof(double));
    initial_assignments = (TFloat *) calloc((size_t) THREAD_COUNT*(n+m*2), sizeof(TFloat));
    thread_params = (double *) calloc((size_t) THREAD_COUNT*PARAM_COUNT, sizeof(double));
    partable = (double *) calloc((size_t) PARAM_COUNT*4, sizeof(double));
    defaults = (double *) calloc((size_t) PARAM_COUNT*128, sizeof(double));

    h_v = (TFloat *) calloc((size_t) n, sizeof(TFloat));

    gpuErrorCheck(hipMalloc(&d_x, (n+m*2)*sizeof(TFloat) ),true);
    gpuErrorCheck(hipMalloc(&d_x_tmp, (n+m*2)*sizeof(TFloat) ),true);
    gpuErrorCheck(hipMalloc(&d_dxdt, (n+m*2)*sizeof(TFloat) ),true); 
    gpuErrorCheck(hipMalloc(&d_cls, m*max_lits_system*sizeof(int) ),true);
    gpuErrorCheck(hipMemcpy(d_cls, cls, m*max_lits_system*sizeof(int), hipMemcpyHostToDevice),true);
    gpuErrorCheck(hipMalloc(&d_clauseSizes, m*sizeof(int) ),true);
    gpuErrorCheck(hipMemcpy(d_clauseSizes, clauseSizes, m*sizeof(int), hipMemcpyHostToDevice),true);
    hipMalloc(&d_loc, sizeof(int));
    hipMalloc(&d_varchanges, sizeof(int));
    hipMalloc(&d_energy, sizeof(float));

    if (!quiet) printf("c ARRAYS INITIALISED\n");

    /// set _all_runs vars: ---------------------------------------------------------------------
    global_all_runs = m;
    for (int i=0; i<THREAD_COUNT; i++) global_all_runs_thread[i] = m;

    /// load solution: --------------------------------------------------------------------------
    if (loadsolution) {
        FILE *fs;
        fs = fopen(SOLUTION_FILE, "r");
        if (fs == NULL) {
            fprintf(stdout, "c solution.txt NOT PROVIDED. USING RANDOM ASSIGNMENTS.\n");
        } else {
            double ivar;
            for (int i=0; i<n;i++) {
                fscanf(fs,"%lf,",&ivar);
                v_best[i] = ivar;
                //if (i<5) printf("SOLUTION_FILE p%d=%.5f, ivar=%.5f ",i,v_best[i],ivar);
            }
            fclose(fs);
            printf("c SOLUTION solution.txt LOADED.\n");
        }
    } else {
        printf("c SOLUTION solution.txt NOT LOADED (SEED!=0)\n");
    }

    /// load partable: --------------------------------------------------------------------------
    if (load_partable) {
        FILE *fp;
        fp = fopen(PARTABLE_FILE, "r");
        if (fp == NULL) {
            fprintf(stdout, "c partable.txt NOT PROVIDED. USING DEFAULT SETTINGS.\n");
        } else {
            // first line: number of default values:
            int num_defaults;
            fscanf(fp,"%d",&num_defaults);
            // num_defaults x params:
            double ipar;
            for (int j=0; j<num_defaults; j++) {
                for (int i=0; i<PARAM_COUNT; i++) {
                    fscanf(fp,"%lf;",&ipar);
                    defaults[j*PARAM_COUNT+i] = ipar;
                    // the #0 default will be set here, too:
                    if (j==0) {
                        switch (i) {
                            case 0: dmm_alpha = ipar; break;
                            case 1: dmm_beta = ipar; break;
                            case 2: dmm_gamma = ipar; break;
                            case 3: dmm_delta = ipar; break;
                            case 4: dmm_epsilon = ipar; break;
                            case 5: dmm_zeta = ipar; break;
                            case 6: rk_errorrate_1 = ipar; break;
                            case 7: rk_errorrate_2 = ipar; break;
                            case 8: init_dt = ipar; break;
                        }
                    }
                }
            }
            //now bounds for the vars:
            for (int i=0; i<PARAM_COUNT; i++) {
                double softlb; fscanf(fp,"%lf;",&softlb); partable[i*4+0] = softlb;
                double softub; fscanf(fp,"%lf;",&softub); partable[i*4+1] = softub;
                double hardlb; fscanf(fp,"%lf;",&hardlb); partable[i*4+2] = hardlb;
                double hardub; fscanf(fp,"%lf;",&hardub); partable[i*4+3] = hardub;
                //printf("c P%d: soft: %.10f-%.10f hard: %.10f-%.10f\n",i,softlb,softub,hardlb,hardub);
            }
            fclose(fp);
            printf("c PARTABLE LOADED %d PARAMS AND %d DEFAULTS.\n",PARAM_COUNT,num_defaults);
            for (int i=0; i<num_defaults; i++) {
                printf("c DEFAULTS SET %d: ",i);
                for (int j=0; j<PARAM_COUNT; j++) printf("%.32f ",defaults[i*PARAM_COUNT+j]);
                printf("\n");
            }
            partable_loaded = true;
        }
    } else {
        fprintf(stdout, "c partable.txt NOT LOADED.\n");
    }

    /// OUPUT SETTINGS: ---------------------------------------------------------------------------
    if (!quiet) printf(TEXT_CYAN);
    if (!quiet) printf("c SETTINGS:\n");
    if (!quiet) printf("c #THREADS        : %d\n",THREAD_COUNT);
    if (!quiet) printf("c SWITCHFRACTION  : %.17f\n",switchfraction);
    if (INTEGRATION_MODE==ODE_RUNGEKUTTA) {
        if (!quiet) printf("c INTGRATION MODE : ADAPTIVE RUNGE KUTTA\n");
        if (!quiet) printf("c ERROR RATE 1    : %.17f\n",rk_errorrate_1);
        if (!quiet) printf("c ERROR RATE 2    : %.17f\n",rk_errorrate_2);
        if (!quiet) printf("c INITAL dt       : %.17f\n",init_dt);
        if (!quiet) printf("c MAX STEPS       : %.0f\n",maxsteps);
    }
    if (INTEGRATION_MODE==ODE_CUSTOM_ADAPTIVE) {
        if (!quiet) printf("c INTGRATION MODE : CUSTOM ADAPTIVE\n");
        if (!quiet) printf("c MIN STEPSIZE    : %.10f\n",adaptive_min);
        if (!quiet) printf("c MAX STEPSIZE    : %.2f\n",adaptive_max);
        if (!quiet) printf("c MAX ERROR       : %.2f\n",step_error);
        if (!quiet) printf("c TIMEOUT         : %.0f\n",timeout);
    }
    if (INTEGRATION_MODE==ODE_CONSTANT) {
        if (!quiet) printf("c INTGRATION MODE : CONSTANT\n");
        if (!quiet) printf("c STEPSIZE        : %.15f\n",stepsize);
        if (!quiet) printf("c TIMEOUT         : %.0f\n",timeout);
    }
    if (!quiet) printf("c TUNE CIRCUIT    : %d\n",tune);
    if (!quiet) printf("c ALPHA HEURISTICS: %d\n",heuristics);
    if (!quiet) printf("c ALPHA           : %.17f\n",dmm_alpha);
    if (!quiet) printf("c BETA            : %.17f\n",dmm_beta);
    if (!quiet) printf("c GAMMA           : %.17f\n",dmm_gamma);
    if (!quiet) printf("c DELTA           : %.17f\n",dmm_delta);
    if (!quiet) printf("c EPSILON         : %.17f\n",dmm_epsilon);
    if (!quiet) printf("c ZETA            : %.17f\n",dmm_zeta);
    if (!quiet) printf("c SEED            : %d\n",seed);
    if (!quiet) printf("c XL_MAX          : %.d\n",xl_max);

    if (!quiet) printf(TEXT_DEFAULT);

    /// init thread specific parameters //////////////////////
    
    /* RNG */
    std::mt19937 generator(seed);
    std::uniform_int_distribution<int>  rand_v(-1.0, 1.0);
    std::uniform_real_distribution<double>  rand_Xs(0.0, 1.0);
    std::uniform_real_distribution<double>  rand_Xl(1.0, 10.0); //xl_max); //10.00
    
    for(int i = 0; i < THREAD_COUNT; i++ ) {
        
        //params:
        thread_params[i*PARAM_COUNT+0]    = dmm_alpha;
        thread_params[i*PARAM_COUNT+1]    = dmm_beta;
        thread_params[i*PARAM_COUNT+2]    = dmm_gamma;
        thread_params[i*PARAM_COUNT+3]    = dmm_delta;
        thread_params[i*PARAM_COUNT+4]    = dmm_epsilon;
        thread_params[i*PARAM_COUNT+5]    = dmm_zeta;
        thread_params[i*PARAM_COUNT+6]    = rk_errorrate_1;
        thread_params[i*PARAM_COUNT+7]    = rk_errorrate_2;
        thread_params[i*PARAM_COUNT+8]    = init_dt;
        
        //initial assignments:
        int set_seed = seed + (i*4096);
        for (int j=0; j<n; j++) initial_assignments[i*(n+m*2)+j] = rand_v(generator);
        for (int j=n; j<n+m; j++) initial_assignments[i*(n+m*2)+j] = 0.0; //rand_Xs(generator);
        for (int j=n+m; j<n+m*2; j++) initial_assignments[i*(n+m*2)+j] = 1.0;//rand_Xl(generator);
        //special case: seed 0: all V zeros:
        if (seed==0 && i==0) for (int j=0; j<n+m*2; j++) initial_assignments[i*(n+m*2)+j] = 0.0; //v_best[i];// -1.00;
        //solution loaded?
        if (loadsolution) {
            for (int j=0; j<n; j++) {
                initial_assignments[i*(n+m*2)+j] = v_best[j];
            }
        }
    }

    /// start solver (GPU) ///////////////////////////////////////////////
    printf("c SOLVING WITH GPU....\n");
    // Number of CUDA devices
    int devCount;
    hipGetDeviceCount(&devCount);
    printf("c [GPU] There are %d CUDA devices.\n", devCount);

    // Iterate through devices:
    for (int i = 0; i < devCount; ++i)
    {
        // Get device properties
        printf("c [GPU] CUDA Device #%d:\n", i);
        hipDeviceProp_t devProp;
        hipGetDeviceProperties(&devProp, i);
        printDevProp(devProp);
    }

    // start ODE with GPU:
    t_rem = 0.0; // in tuning mode, we continue from last lwoest loc and also at that time (t_rem)
    if (tune) {
        tuneGPU();  
    } else {
        solveGPU();
    }
    
    // free memory:
    free(cls);
    free(v_best);
    free(occurrence);          //occurences of each lit
    free(occurenceCounter);    //counter for allocation
    free(numOccurrenceT);      //number of occurence of a lit
    free(clauseSizes); 
    free(thread_params);
    free(initial_assignments);
    free(loc_thread);
    free(global_thread);
    free(time_thread);
    free(time_thread_actual);

    // free device memory:
    hipFree(d_x);
    hipFree(d_x_tmp);
    hipFree(d_dxdt);
    hipFree(d_cls);
    hipFree(d_clauseSizes);
    hipFree(d_loc);
    hipFree(d_energy);
    
    return EXIT_SUCCESS;
}



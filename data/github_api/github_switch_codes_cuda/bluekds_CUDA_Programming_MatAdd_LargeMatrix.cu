#include "hip/hip_runtime.h"
// Repository: bluekds/CUDA_Programming
// File: Book_BJ/Chap5/5_3_LargeMatrix_Add_skeleton/MatAdd_LargeMatrix.cu

﻿#include "MatAdd_LargeMatrix.h"

/******************************************************************
* Complete following three kernels
******************************************************************/

__global__ void MatAdd_G2D_B2D
(float* MatA, float* MatB, float* MatC, int nRow, int nCol)
{
	// Write your 2D_2D kernel here
}

__global__ void MatAdd_G1D_B1D
(float* MatA, float* MatB, float* MatC, int nRow, int nCol)
{
	// Write your 1D_1D kernel here
}

__global__ void MatAdd_G2D_B1D
(float* MatA, float* MatB, float* MatC, int nRow, int nCol)
{
	// Write your 2D_1D kernel here
}




bool kernelCall(float* _MatA, float* _MatB, float* _MatC, int _nRow, int _nCol
	, int _layout, dim3 _gridDim, dim3 _blockDim)
{
	switch (_layout)
	{
	case ThreadLayout::G1D_B1D:
		MatAdd_G1D_B1D <<<_gridDim, _blockDim >>> (_MatA, _MatB, _MatC, _nRow, _nCol);
		break;
	case ThreadLayout::G2D_B1D:
		MatAdd_G2D_B1D <<<_gridDim, _blockDim >>> (_MatA, _MatB, _MatC, _nRow, _nCol);
		break;
	case ThreadLayout::G2D_B2D:
		MatAdd_G2D_B2D <<<_gridDim, _blockDim >>> (_MatA, _MatB, _MatC, _nRow, _nCol);
		break;
	default:
		printf("Not supported layout\n");
		return false;
	}
	return true;
}
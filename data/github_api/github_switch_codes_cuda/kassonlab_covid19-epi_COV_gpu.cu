#include "hip/hip_runtime.h"
// Repository: kassonlab/covid19-epi
// File: COV_gpu.cu

#include <hip/hip_runtime.h>
#include <thrust/copy.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "locale.h"

/* Earth flattening and radius according to GRS80 taken from https://en.wikipedia.org/wiki/Geodetic_Reference_System_1980 */
#define FLATTENING 0.003352810681183637418
#define Earth_Radius_GRS80 6378.137

/* Earth Mean Radius according to IUGG, taken from https://en.wikipedia.org/wiki/Earth_radius */
#define Earth_Radius_Mean 6371.0087714

#define x(radius, lat, lon) ((radius)*sin(deg2rad(90.0-(lat)))*cos(deg2rad((lon))))
#define y(radius, lat, lon) ((radius)*sin(deg2rad(90.0-(lat)))*sin(deg2rad((lon))))
#define z(radius, lat) ((radius)*cos(deg2rad(90.0-(lat))))

#undef locale_distance
#if defined(USE_LAMBERT)
#define locale_distance d_locale_distance_Lambert
#else
#define locale_distance d_locale_distance_GCD_1
#endif

/* Distance using Lambert formula, taken from https://en.wikipedia.org/wiki/Geographical_distance */
static __device__ double d_locale_distance_Lambert(struct locale l1, struct locale l2)
{
    double d, ca, sca, cca, p, q, x, y, sp, sq, cp, cq;

    if (l1.lat == l2.lat && l1.lon == l2.lon) {
        return 0.0;
    }
    ca = acos(sin(l1.rlat) * sin(l2.rlat) + cos(l1.rlat) * cos(l2.rlat) * cos(l1.rlon - l2.rlon));
    sca = sin(ca/2);
    cca = cos(ca/2);
    p = (l1.rbeta+l2.rbeta)/2;
    sp = sin(p);
    cp = cos(p);
    q = (l2.rbeta-l1.rbeta)/2;
    sq = sin(q);
    cq = cos(q);
    x = (ca-sin(ca))*(sp*sp*cq*cq/(cca*cca));
    y = (ca+sin(ca))*(cp*cp*sq*sq/(sca*sca));
    d = Earth_Radius_GRS80*(ca - FLATTENING/2*(x+y));
    return d;
}

/* Great Circle Distance */
static __device__ double d_locale_distance_GCD_1(struct locale l1, struct locale l2)
{
    double d;
    if (l1.lat == l2.lat && l1.lon == l2.lon) {
        return 0.0;
    }
    d = (l1.x * l2.x + l1.y * l2.y + l1.z * l2.z) / (Earth_Radius_Mean*Earth_Radius_Mean);
    return acos(d)*Earth_Radius_Mean;
}

#define deg2rad(deg) ((deg) * M_PI / 180)
#define rad2deg(rad) ((rad) * 180 / M_PI)

static __device__ double calc_community_infect(double kappa, double omega, int severe, double d, double betac_scale) {

	/* need to work on this.  Perhaps we take a random distance for each two people based on population density, number of people in county, county area, etc. */
	double zeta[]={0.1, 0.25, 0.5, 0.75, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 0.75, 0.50, 0.25, 0.25, 0.25} ; //   # Travel related parameter for community transmission. Ferguson Nature 2006
	double fd;
	double betac=0.103 ; // Scaled from betac=0.075 in influenza pandemic with R0=1.6, COVID-19 R0=2.2 (Ferguson 2020)

	fd=1/(1+pow((d/4), 3)); //kernel density function as parameterized for GB.
	return (betac_scale*betac*kappa*fd*(1+severe*(omega-1)));
}

static __device__ double d_distance(double lat1, double lon1, double lat2, double lon2, char unit) {
    double theta, dist;
    if ((lat1 == lat2) && (lon1 == lon2)) {
      return 0;
    }
    else {
      theta = lon1 - lon2;
  //    dist = sin(deg2rad(lat1)) * sin(deg2rad(lat2)) + cos(deg2rad(lat1)) * cos(deg2rad(lat2)) * cos(deg2rad(theta));
      double ang1,ang2;
      ang1 = deg2rad(lat1);
      ang2 = deg2rad(lat2);
      dist = cos(ang1) * cos(ang2) * ( 1.0 + cos(deg2rad(theta)) ) - cos(ang1 + ang2);
      dist = acos(dist);
      dist = rad2deg(dist);
      dist = dist * 60 * 1.1515;
      switch(unit) {
        case 'M':
          break;
        case 'K':
          dist = dist * 1.609344;
          break;
        case 'N':
          dist = dist * 0.8684;
          break;
      }
      return (dist);
    }
}

struct LoopInvariantData {
    thrust::device_vector<int> infectious;
    thrust::device_vector<double> infect_kappa;
    thrust::device_vector<int> intervene;
    thrust::device_vector<double> tau;
    thrust::device_vector<double> tauI;
    thrust::device_vector<double> interIc;
    thrust::device_vector<int> hosp_pop;
    thrust::device_vector<int> icu_pop;
    thrust::device_vector<double> lat_locale;
    thrust::device_vector<double> lon_locale;
    thrust::device_vector<int> locale_HH;
    thrust::device_vector<int> HH;
    thrust::device_vector<struct locale> locale_list;
    thrust::device_vector<int> severe;
};


static __global__ void locale_infectious_step_kernel(
    int j,
    int num_infectious,
    double Ic,
    double t,
    double dt,
    double omega,
    double betac_scale,

    int const* infectious,
    double const* infect_kappa,
    int const* intervene,
    double const* tau,
    double const* tauI,
    double const* interIc,
    int const* hosp_pop,
    int const* icu_pop,
    double const* lat_locale,
    double const* lon_locale,
    int const* locale_HH,
    int const* HH,
    struct locale const* locale_list,
    int const* severe,

    double* tmp_comm_inf_arr)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_infectious) {
        double tmp_comm_inf = 0.0;
        int infec_person; //Counter for infected person.
        double kappa; // #Infectiousness
        double tIc;
        infec_person = infectious[i];
        tIc = Ic;
        if ( intervene[infec_person] > 0 && t>tau[infec_person]+tauI[intervene[infec_person]]) {
            tIc = interIc[intervene[infec_person]];
        }
        kappa = infect_kappa[i];
    
        if (hosp_pop[infec_person]==0) {
            double d; //distance between people.
            // Community transmission //
#if !defined(USE_LOCALE_DISTANCE)
            d = d_distance(lat_locale[j], lon_locale[j], lat_locale[locale_HH[HH[infec_person]]], lon_locale[locale_HH[HH[infec_person]]], 'K');
#else
            d = locale_distance(locale_list[j], locale_list[locale_HH[HH[infec_person]]]);
#endif
            tmp_comm_inf += tIc * calc_community_infect( kappa, omega, severe[infec_person], d, betac_scale);
        }
    
        tmp_comm_inf_arr[i] = tmp_comm_inf;
    }
}

void locale_infectious_step(LoopInvariantData const& lid, int population, int j, int num_households, int num_infectious, double Ic, double t, double dt, double omega, double& out_tmp_comm_inf, double betac_scale) {
    if (num_infectious == 0) {
        out_tmp_comm_inf = 0.0;
        return;
    }
    
    thrust::device_vector<double> d_tmp_comm_inf_arr(num_infectious, 0.0);

    // Run kernel
    size_t const THREAD_COUNT = 512;
    size_t const BLOCK_COUNT = (num_infectious + THREAD_COUNT - 1) / THREAD_COUNT;
    locale_infectious_step_kernel<<<BLOCK_COUNT, THREAD_COUNT>>>(
        j,
        num_infectious,
        Ic,
        t,
        dt,
        omega,
        betac_scale,
        
        lid.infectious.data().get(),
        lid.infect_kappa.data().get(),
        lid.intervene.data().get(),
        lid.tau.data().get(),
        lid.tauI.data().get(),
        lid.interIc.data().get(),
        lid.hosp_pop.data().get(),
        lid.icu_pop.data().get(),
        lid.lat_locale.data().get(),
        lid.lon_locale.data().get(),
        lid.locale_HH.data().get(),
        lid.HH.data().get(),
        lid.locale_list.data().get(),
        lid.severe.data().get(),

        d_tmp_comm_inf_arr.data().get());

    out_tmp_comm_inf = thrust::reduce(d_tmp_comm_inf_arr.begin(), d_tmp_comm_inf_arr.end(), 0.0, thrust::plus<double>());
}

extern "C" void locale_infectious_loop(int num_locale, int population, int num_households, int num_infectious, int* infectious, double const* infect_kappa, double Ic, int* intervene, double t, double* tau, double* tauI, double* interIc, double dt, int* hosp_pop, int* icu_pop, double* lat_locale, double* lon_locale, int* locale_HH, int* HH, struct locale* locale_list, double omega, int* severe, double betac_scale, double* commun_nom1, double* fd_tot) {

    size_t const num_I = 10;

    // Allocate device and host arrays
    LoopInvariantData lid;
    lid.infectious.assign(infectious, infectious + num_infectious);
    lid.infect_kappa.assign(infect_kappa, infect_kappa + num_infectious);
    lid.intervene.assign(intervene, intervene + population);
    lid.tau.assign(tau, tau + population);
    lid.tauI.assign(tauI, tauI + num_I);
    lid.interIc.assign(interIc, interIc + num_I);
    lid.hosp_pop.assign(hosp_pop, hosp_pop + population);
    lid.icu_pop.assign(icu_pop, icu_pop + population);
    lid.lat_locale.assign(lat_locale, lat_locale + num_locale);
    lid.lon_locale.assign(lon_locale, lon_locale + num_locale);
    lid.locale_HH.assign(locale_HH, locale_HH + num_households);
    lid.HH.assign(HH, HH + population);
    lid.locale_list.assign(locale_list, locale_list + num_locale);
    lid.severe.assign(severe, severe + population);

	for (int j=0; j<num_locale; j++) {
		double tmp_comm_inf = 0.0;

		locale_infectious_step(lid, population, j, num_households, num_infectious, Ic, t, dt, omega, tmp_comm_inf, betac_scale);

		commun_nom1[j] = tmp_comm_inf / fd_tot[j];
	}
}

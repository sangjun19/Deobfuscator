#include "hip/hip_runtime.h"
// Repository: bogdanadnan/multiminer
// File: algo/argon2/argon2d/argon2-gpu/lib/argon2-cuda/kernels.cu

/* For IDE: */
#ifndef __HIPCC__
#define __HIPCC__
#endif
#include <thread>
#include <chrono>

#ifdef _WIN64
#include <windows.h>
#endif

#include "argon2-cuda/kernels.h"
#include "argon2-gpu-common/argon2-common.h"

#include "argon2-cuda/cudaexception.h"

#include <stdexcept>
#ifndef NDEBUG
#include <iostream>

#endif

#define ARGON2_D  0
#define ARGON2_I  1
#define ARGON2_ID 2

#define ARGON2_VERSION_10 0x10
#define ARGON2_VERSION_13 0x13

#define ARGON2_BLOCK_SIZE 1024
#define ARGON2_DWORDS_IN_BLOCK (ARGON2_BLOCK_SIZE / 4)
#define ARGON2_QWORDS_IN_BLOCK (ARGON2_BLOCK_SIZE / 8)
#define ARGON2_SYNC_POINTS 4

#define THREADS_PER_LANE 32
#define QWORDS_PER_THREAD (ARGON2_QWORDS_IN_BLOCK / 32)

#define BLOCK_BYTES	32
#define OUT_BYTES	16
#define ARGON2_PREHASH_DIGEST_LENGTH	16
#define ARGON2_PREHASH_SEED_LENGTH		18

#define G(m, r, i, a, b, c, d) \
do { \
	a = a + b + m[blake2b_sigma[r][2 * i + 0]]; \
	d = rotr64(d ^ a, 32); \
	c = c + d; \
	b = rotr64(b ^ c, 24); \
	a = a + b + m[blake2b_sigma[r][2 * i + 1]]; \
	d = rotr64(d ^ a, 16); \
	c = c + d; \
	b = rotr64(b ^ c, 63); \
} while ((void)0, 0)

#define ROUND(m, v, r) \
do { \
	G(m, r, 0, v[0], v[4], v[ 8], v[12]); \
	G(m, r, 1, v[1], v[5], v[ 9], v[13]); \
	G(m, r, 2, v[2], v[6], v[10], v[14]); \
	G(m, r, 3, v[3], v[7], v[11], v[15]); \
	G(m, r, 4, v[0], v[5], v[10], v[15]); \
	G(m, r, 5, v[1], v[6], v[11], v[12]); \
	G(m, r, 6, v[2], v[7], v[ 8], v[13]); \
	G(m, r, 7, v[3], v[4], v[ 9], v[14]); \
} while ((void)0, 0)

namespace argon2 {
namespace cuda {

typedef struct blake2b_state_ {
	uint64_t h[8];
	uint64_t t[2];
	uint32_t buf[BLOCK_BYTES];
	uint32_t bufLen;
} blake2b_state;

__device__ uint64_t rotr64(uint64_t x, uint32_t n)
{
	return (x >> n) | (x << (64 - n));
}

__device__ void blake2b_init(blake2b_state *state, uint32_t outlen)
{
	uint64_t blake2b_IV[8] = {
			0x6A09E667F3BCC908, 0xBB67AE8584CAA73B,
			0x3C6EF372FE94F82B, 0xA54FF53A5F1D36F1,
			0x510E527FADE682D1, 0x9B05688C2B3E6C1F,
			0x1F83D9ABFB41BD6B, 0x5BE0CD19137E2179
	};

	state->t[1] = state->t[0] = 0;
	state->bufLen = 0;

	for(int i=0;i<8;i++) {
		state->h[i] = blake2b_IV[i];
	}

	state->h[0] ^= ((outlen * 4) | (1 << 16) | (1 << 24));
}

__device__ void blake2b_compress(blake2b_state *state, uint64_t *m, uint64_t f0)
{
	uint64_t v[16];

	uint64_t blake2b_IV[8] = {
			0x6A09E667F3BCC908, 0xBB67AE8584CAA73B,
			0x3C6EF372FE94F82B, 0xA54FF53A5F1D36F1,
			0x510E527FADE682D1, 0x9B05688C2B3E6C1F,
			0x1F83D9ABFB41BD6B, 0x5BE0CD19137E2179
	};
	unsigned int blake2b_sigma[12][16] = {
			{0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15},
			{14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3},
			{11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4},
			{7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8},
			{9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13},
			{2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9},
			{12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11},
			{13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10},
			{6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5},
			{10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13, 0},
			{0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15},
			{14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3},
	};

	v[ 0] = state->h[0];
	v[ 1] = state->h[1];
	v[ 2] = state->h[2];
	v[ 3] = state->h[3];
	v[ 4] = state->h[4];
	v[ 5] = state->h[5];
	v[ 6] = state->h[6];
	v[ 7] = state->h[7];
	v[ 8] = blake2b_IV[0];
	v[ 9] = blake2b_IV[1];
	v[10] = blake2b_IV[2];
	v[11] = blake2b_IV[3];
	v[12] = blake2b_IV[4] ^ state->t[0];
	v[13] = blake2b_IV[5] ^ state->t[1];
	v[14] = blake2b_IV[6] ^ f0;
	v[15] = blake2b_IV[7];

	ROUND(m, v, 0);
	ROUND(m, v, 1);
	ROUND(m, v, 2);
	ROUND(m, v, 3);
	ROUND(m, v, 4);
	ROUND(m, v, 5);
	ROUND(m, v, 6);
	ROUND(m, v, 7);
	ROUND(m, v, 8);
	ROUND(m, v, 9);
	ROUND(m, v, 10);
	ROUND(m, v, 11);

	state->h[0] ^= v[0] ^ v[ 8];
	state->h[1] ^= v[1] ^ v[ 9];
	state->h[2] ^= v[2] ^ v[10];
	state->h[3] ^= v[3] ^ v[11];
	state->h[4] ^= v[4] ^ v[12];
	state->h[5] ^= v[5] ^ v[13];
	state->h[6] ^= v[6] ^ v[14];
	state->h[7] ^= v[7] ^ v[15];
}

__device__ void blake2b_incrementCounter(blake2b_state *state, uint32_t inc)
{
	state->t[0] += (inc * 4);
	state->t[1] += (state->t[0] < (inc * 4));
}

__device__ void blake2b_update(blake2b_state *state, uint32_t *in, int32_t inLen)
{
	if (state->bufLen + inLen > BLOCK_BYTES) {
		uint32_t temp[BLOCK_BYTES];
		uint32_t have = state->bufLen;
		uint32_t left = BLOCK_BYTES - have;

		for(int i=0;i<left;i++) {
			*(state->buf +  have + i) = in[i];
		}

		blake2b_incrementCounter(state, BLOCK_BYTES);
		blake2b_compress(state, (uint64_t*)state->buf, 0);

		state->bufLen = 0;
		inLen -= left;
		in += left;

		while (inLen > BLOCK_BYTES) {
			blake2b_incrementCounter(state, BLOCK_BYTES);

			for(int i=0;i<BLOCK_BYTES;i++) {
				temp[i] = in[i];
			}

			blake2b_compress(state, (uint64_t *)temp, 0);
			inLen -= BLOCK_BYTES;
			in += BLOCK_BYTES;
		}
	}
	for(int i=0;i<inLen;i++) {
		*(state->buf +  state->bufLen + i) = *(in + i);
	}
	state->bufLen += inLen;
}

__device__ void blake2b_final(blake2b_state *state, uint32_t *out, uint32_t outLen)
{
	blake2b_incrementCounter(state, state->bufLen);
	for(int i=0;i<BLOCK_BYTES - state->bufLen;i++) {
		*(state->buf + state->bufLen + i) = 0;
	}
	blake2b_compress(state, (uint64_t*)state->buf, 0xFFFFFFFFFFFFFFFF);
	for(int i=0;i<outLen;i++) {
		*(out + i) = *((uint32_t*)state->h + i);
	}
}

__device__ void blake2b_digestLong(uint32_t *out, uint32_t outLen,
								   uint32_t *in, uint32_t inLen)
{
	blake2b_state blake;

	if (outLen <= OUT_BYTES) {
		blake2b_init(&blake, outLen);

		blake.buf[0] = (outLen * 4);
		blake.bufLen = 1;

		blake2b_update(&blake, in, inLen);
		blake2b_final(&blake, out, outLen);
	} else {
		uint32_t out_buffer[OUT_BYTES];

		blake2b_init(&blake, OUT_BYTES);

		blake.buf[0] = (outLen * 4);
		blake.bufLen = 1;

		blake2b_update(&blake, in, inLen);
		blake2b_final(&blake, out_buffer, OUT_BYTES);

		for(int i=0;i<OUT_BYTES / 2;i++) {
			*(out + i) = *(out_buffer + i);
		}
		out += OUT_BYTES / 2;

		uint32_t toProduce = outLen - OUT_BYTES / 2;
		while (toProduce > OUT_BYTES) {
			blake2b_init(&blake, OUT_BYTES);
			blake2b_update(&blake, out_buffer, OUT_BYTES);
			blake2b_final(&blake, out_buffer, OUT_BYTES);

			for(int i=0;i<OUT_BYTES / 2;i++) {
				*(out + i) = *(out_buffer + i);
			}
			out += OUT_BYTES / 2;
			toProduce -= OUT_BYTES / 2;
		}

		blake2b_init(&blake, toProduce);
		blake2b_update(&blake, out_buffer, OUT_BYTES);
		blake2b_final(&blake, out, toProduce);
	}
}

__device__ uint64_t u64_build(uint32_t hi, uint32_t lo)
{
    return ((uint64_t)hi << 32) | (uint64_t)lo;
}

__device__ uint32_t u64_lo(uint64_t x)
{
    return (uint32_t)x;
}

__device__ uint32_t u64_hi(uint64_t x)
{
    return (uint32_t)(x >> 32);
}

struct u64_shuffle_buf {
    uint32_t lo[THREADS_PER_LANE];
    uint32_t hi[THREADS_PER_LANE];
};

__device__ uint64_t u64_shuffle(uint64_t v, uint32_t thread_src,
                                uint32_t thread, struct u64_shuffle_buf *buf)
{
    uint32_t lo = u64_lo(v);
    uint32_t hi = u64_hi(v);

    buf->lo[thread] = lo;
    buf->hi[thread] = hi;

    __syncthreads();

    lo = buf->lo[thread_src];
    hi = buf->hi[thread_src];

    return u64_build(hi, lo);
}

struct block_g {
    uint64_t data[ARGON2_QWORDS_IN_BLOCK];
};

struct block_th {
    uint64_t a, b, c, d;
};

__device__ uint64_t cmpeq_mask(uint32_t test, uint32_t ref)
{
    uint32_t x = -(uint32_t)(test == ref);
    return u64_build(x, x);
}

__device__ uint64_t block_th_get(const struct block_th *b, uint32_t idx)
{
    uint64_t res = 0;
    res ^= cmpeq_mask(idx, 0) & b->a;
    res ^= cmpeq_mask(idx, 1) & b->b;
    res ^= cmpeq_mask(idx, 2) & b->c;
    res ^= cmpeq_mask(idx, 3) & b->d;
    return res;
}

__device__ void block_th_set(struct block_th *b, uint32_t idx, uint64_t v)
{
    b->a ^= cmpeq_mask(idx, 0) & (v ^ b->a);
    b->b ^= cmpeq_mask(idx, 1) & (v ^ b->b);
    b->c ^= cmpeq_mask(idx, 2) & (v ^ b->c);
    b->d ^= cmpeq_mask(idx, 3) & (v ^ b->d);
}

__device__ void move_block(struct block_th *dst, const struct block_th *src)
{
    *dst = *src;
}

__device__ void xor_block(struct block_th *dst, const struct block_th *src)
{
    dst->a ^= src->a;
    dst->b ^= src->b;
    dst->c ^= src->c;
    dst->d ^= src->d;
}

__device__ void load_block(struct block_th *dst, const struct block_g *src,
                           uint32_t thread)
{
    dst->a = src->data[0 * THREADS_PER_LANE + thread];
    dst->b = src->data[1 * THREADS_PER_LANE + thread];
    dst->c = src->data[2 * THREADS_PER_LANE + thread];
    dst->d = src->data[3 * THREADS_PER_LANE + thread];
}

__device__ void load_block_xor(struct block_th *dst, const struct block_g *src,
                               uint32_t thread)
{
    dst->a ^= src->data[0 * THREADS_PER_LANE + thread];
    dst->b ^= src->data[1 * THREADS_PER_LANE + thread];
    dst->c ^= src->data[2 * THREADS_PER_LANE + thread];
    dst->d ^= src->data[3 * THREADS_PER_LANE + thread];
}

__device__ void store_block(struct block_g *dst, const struct block_th *src,
                            uint32_t thread)
{
    dst->data[0 * THREADS_PER_LANE + thread] = src->a;
    dst->data[1 * THREADS_PER_LANE + thread] = src->b;
    dst->data[2 * THREADS_PER_LANE + thread] = src->c;
    dst->data[3 * THREADS_PER_LANE + thread] = src->d;
}

__device__ uint64_t f(uint64_t x, uint64_t y)
{
    uint32_t xlo = u64_lo(x);
    uint32_t ylo = u64_lo(y);
    return x + y + 2 * u64_build(__umulhi(xlo, ylo), xlo * ylo);
}

__device__ void g(struct block_th *block)
{

	asm ("{"
	     ".reg .u64 d1, d2, a, b, c, d;\n\t"
		 ".reg .u32 s1, s2, s3, s4;\n\t"
		 "add.u64 d1, %0, %1;\n\t"
		 "cvt.u32.u64 s1, %0;\n\t"
		 "cvt.u32.u64 s2, %1;\n\t"
		 "mul.lo.u32 s3, s1, s2;\n\t"
		 "mul.hi.u32 s4, s1, s2;\n\t"
		 "mov.b64 a, {s3, s4};\n\t"
		 "shl.b64 d2, a, 1;\n\t"
		 "add.u64 a, d1, d2;\n\t"
		 "xor.b64 d1, %3, a;\n\t"
		 "mov.b64 {s1, s2}, d1;\n\t"
		 "mov.b64 d, {s2, s1};\n\t"
		 "add.u64 d1, %2, d;\n\t"
		 "cvt.u32.u64 s1, %2;\n\t"
		 "mul.lo.u32 s3, s1, s2;\n\t"
		 "mul.hi.u32 s4, s1, s2;\n\t"
		 "mov.b64 c, {s3, s4};\n\t"
		 "shl.b64 d2, c, 1;\n\t"
		 "add.u64 c, d1, d2;\n\t"
		 "xor.b64 d1, %1, c;\n\t"
		 "mov.b64 {s3, s4}, d1;\n\t"
		 "prmt.b32 s2, s3, s4, 0x6543;\n\t"
		 "prmt.b32 s1, s3, s4, 0x2107;\n\t"
		 "mov.b64 b, {s2, s1};\n\t"
		 "add.u64 d1, a, b;\n\t"
		 "cvt.u32.u64 s1, a;\n\t"
		 "mul.lo.u32 s3, s1, s2;\n\t"
		 "mul.hi.u32 s4, s1, s2;\n\t"
		 "mov.b64 a, {s3, s4};\n\t"
		 "shl.b64 d2, a, 1;\n\t"
		 "add.u64 %0, d1, d2;\n\t"
		 "xor.b64 d1, d, %0;\n\t"
		 "mov.b64 {s3, s4}, d1;\n\t"
		 "prmt.b32 s2, s3, s4, 0x5432;\n\t"
		 "prmt.b32 s1, s3, s4, 0x1076;\n\t"
		 "mov.b64 %3, {s2, s1};\n\t"
		 "add.u64 d1, c, %3;\n\t"
		 "cvt.u32.u64 s1, c;\n\t"
		 "mul.lo.u32 s3, s1, s2;\n\t"
		 "mul.hi.u32 s4, s1, s2;\n\t"
		 "mov.b64 c, {s3, s4};\n\t"
		 "shl.b64 d2, c, 1;\n\t"
		 "add.u64 %2, d1, d2;\n\t"
		 "xor.b64 d1, b, %2;\n\t"
		 "shl.b64 a, d1, 1;\n\t"
		 "shr.b64 b, d1, 63;\n\t"
		 "add.u64 %1, a, b;\n\t"
	"}" : "+l"(block->a), "+l"(block->b), "+l"(block->c), "+l"(block->d));

/*
    uint64_t a, b, c, d;
    a = block->a;
    b = block->b;
    c = block->c;
    d = block->d;

 	a = f(a, b);
    d = rotr64(d ^ a, 32);
    c = f(c, d);
    b = rotr64(b ^ c, 24);
    a = f(a, b);
    d = rotr64(d ^ a, 16);
    c = f(c, d);
    b = rotr64(b ^ c, 63);

    block->a = a;
    block->b = b;
    block->c = c;
    block->d = d; */
}

template<class shuffle>
__device__ void apply_shuffle(struct block_th *block, uint32_t thread,
                              struct u64_shuffle_buf *buf)
{
    for (uint32_t i = 0; i < QWORDS_PER_THREAD; i++) {
        uint32_t src_thr = shuffle::apply(thread, i);

        uint64_t v = block_th_get(block, i);
        v = u64_shuffle(v, src_thr, thread, buf);
        block_th_set(block, i, v);
    }
}

__device__ void transpose(struct block_th *block, uint32_t thread,
                          struct u64_shuffle_buf *buf)
{
    uint32_t thread_group = (thread & 0x0C) >> 2;
    for (uint32_t i = 1; i < QWORDS_PER_THREAD; i++) {
        uint32_t thr = (i << 2) ^ thread;
        uint32_t idx = thread_group ^ i;

        uint64_t v = block_th_get(block, idx);
        v = u64_shuffle(v, thr, thread, buf);
        block_th_set(block, idx, v);
    }
}

struct identity_shuffle {
    __device__ static uint32_t apply(uint32_t thread, uint32_t idx)
    {
        return thread;
    }
};

struct shift1_shuffle {
    __device__ static uint32_t apply(uint32_t thread, uint32_t idx)
    {
        return (thread & 0x1c) | ((thread + idx) & 0x3);
    }
};

struct unshift1_shuffle {
    __device__ static uint32_t apply(uint32_t thread, uint32_t idx)
    {
        idx = (QWORDS_PER_THREAD - idx) % QWORDS_PER_THREAD;

        return (thread & 0x1c) | ((thread + idx) & 0x3);
    }
};

struct shift2_shuffle {
    __device__ static uint32_t apply(uint32_t thread, uint32_t idx)
    {
        uint32_t lo = (thread & 0x1) | ((thread & 0x10) >> 3);
        lo = (lo + idx) & 0x3;
        return ((lo & 0x2) << 3) | (thread & 0xe) | (lo & 0x1);
    }
};

struct unshift2_shuffle {
    __device__ static uint32_t apply(uint32_t thread, uint32_t idx)
    {
        idx = (QWORDS_PER_THREAD - idx) % QWORDS_PER_THREAD;

        uint32_t lo = (thread & 0x1) | ((thread & 0x10) >> 3);
        lo = (lo + idx) & 0x3;
        return ((lo & 0x2) << 3) | (thread & 0xe) | (lo & 0x1);
    }
};

__device__ void shuffle_block(struct block_th *block, uint32_t thread,
                              struct u64_shuffle_buf *buf)
{
    transpose(block, thread, buf);

    g(block);

    apply_shuffle<shift1_shuffle>(block, thread, buf);

    g(block);

    apply_shuffle<unshift1_shuffle>(block, thread, buf);
    transpose(block, thread, buf);

    g(block);

    apply_shuffle<shift2_shuffle>(block, thread, buf);

    g(block);

    apply_shuffle<unshift2_shuffle>(block, thread, buf);
}

__device__ void next_addresses(struct block_th *addr, struct block_th *tmp,
                               uint32_t thread_input, uint32_t thread,
                               struct u64_shuffle_buf *buf)
{
    addr->a = u64_build(0, thread_input);
    addr->b = 0;
    addr->c = 0;
    addr->d = 0;

    shuffle_block(addr, thread, buf);

    addr->a ^= u64_build(0, thread_input);
    move_block(tmp, addr);

    shuffle_block(addr, thread, buf);

    xor_block(addr, tmp);
}

__device__ void compute_ref_pos(
        uint32_t lanes, uint32_t segment_blocks,
        uint32_t pass, uint32_t lane, uint32_t slice, uint32_t offset,
        uint32_t *ref_lane, uint32_t *ref_index)
{
    uint32_t lane_blocks = ARGON2_SYNC_POINTS * segment_blocks;

    *ref_lane = *ref_lane % lanes;

    uint32_t base;
    if (pass != 0) {
        base = lane_blocks - segment_blocks;
    } else {
        if (slice == 0) {
            *ref_lane = lane;
        }
        base = slice * segment_blocks;
    }

    uint32_t ref_area_size = base + offset - 1;
    if (*ref_lane != lane) {
        ref_area_size = min(ref_area_size, base);
    }

    *ref_index = __umulhi(*ref_index, *ref_index);
    *ref_index = ref_area_size - 1 - __umulhi(ref_area_size, *ref_index);

    if (pass != 0 && slice != ARGON2_SYNC_POINTS - 1) {
        *ref_index += (slice + 1) * segment_blocks;
        if (*ref_index >= lane_blocks) {
            *ref_index -= lane_blocks;
        }
    }
}

struct ref {
    uint32_t ref_lane;
    uint32_t ref_index;
};

/*
 * Refs hierarchy:
 * lanes -> passes -> slices -> blocks
 */
template<uint32_t type>
__global__ void argon2_precompute_kernel(
        struct ref *refs, uint32_t passes, uint32_t lanes,
        uint32_t segment_blocks)
{
    uint32_t block_id = blockIdx.y * blockDim.y + threadIdx.y;
    uint32_t warp = threadIdx.y;
    uint32_t thread = threadIdx.x;

    extern __shared__ struct u64_shuffle_buf shuffle_bufs[];
    struct u64_shuffle_buf *shuffle_buf = &shuffle_bufs[warp];

    uint32_t segment_addr_blocks = (segment_blocks + ARGON2_QWORDS_IN_BLOCK - 1)
            / ARGON2_QWORDS_IN_BLOCK;
    uint32_t block = block_id % segment_addr_blocks;
    uint32_t segment = block_id / segment_addr_blocks;

    uint32_t slice, pass, pass_id, lane;
    if (type == ARGON2_ID) {
        slice = segment % (ARGON2_SYNC_POINTS / 2);
        lane = segment / (ARGON2_SYNC_POINTS / 2);
        pass_id = pass = 0;
    } else {
        slice = segment % ARGON2_SYNC_POINTS;
        pass_id = segment / ARGON2_SYNC_POINTS;

        pass = pass_id % passes;
        lane = pass_id / passes;
    }

    struct block_th addr, tmp;

    uint32_t thread_input;
    switch (thread) {
    case 0:
        thread_input = pass;
        break;
    case 1:
        thread_input = lane;
        break;
    case 2:
        thread_input = slice;
        break;
    case 3:
        thread_input = lanes * segment_blocks * ARGON2_SYNC_POINTS;
        break;
    case 4:
        thread_input = passes;
        break;
    case 5:
        thread_input = type;
        break;
    case 6:
        thread_input = block + 1;
        break;
    default:
        thread_input = 0;
        break;
    }

    next_addresses(&addr, &tmp, thread_input, thread, shuffle_buf);

    refs += segment * segment_blocks;

    for (uint32_t i = 0; i < QWORDS_PER_THREAD; i++) {
        uint32_t pos = i * THREADS_PER_LANE + thread;
        uint32_t offset = block * ARGON2_QWORDS_IN_BLOCK + pos;
        if (offset < segment_blocks) {
            uint64_t v = block_th_get(&addr, i);
            uint32_t ref_index = u64_lo(v);
            uint32_t ref_lane  = u64_hi(v);

            compute_ref_pos(lanes, segment_blocks, pass, lane, slice, offset,
                            &ref_lane, &ref_index);

            refs[offset].ref_index = ref_index;
            refs[offset].ref_lane  = ref_lane;
        }
    }
}

template<uint32_t version>
__device__ void argon2_core(
        struct block_g *memory, struct block_g *mem_curr,
        struct block_th *prev, struct block_th *tmp,
        struct u64_shuffle_buf *shuffle_buf, uint32_t lanes,
        uint32_t thread, uint32_t pass, uint32_t ref_index, uint32_t ref_lane)
{
    struct block_g *mem_ref = memory + ref_index * lanes + ref_lane;

    if (version != ARGON2_VERSION_10 && pass != 0) {
        load_block(tmp, mem_curr, thread);
        load_block_xor(prev, mem_ref, thread);
        xor_block(tmp, prev);
    } else {
        load_block_xor(prev, mem_ref, thread);
        move_block(tmp, prev);
    }

    shuffle_block(prev, thread, shuffle_buf);

    xor_block(prev, tmp);

    store_block(mem_curr, prev, thread);
}

template<uint32_t type, uint32_t version>
__device__ void argon2_step_precompute(
        struct block_g *memory, struct block_g *mem_curr,
        struct block_th *prev, struct block_th *tmp,
        struct u64_shuffle_buf *shuffle_buf, const struct ref **refs,
        uint32_t lanes, uint32_t segment_blocks, uint32_t thread,
        uint32_t lane, uint32_t pass, uint32_t slice, uint32_t offset)
{
    uint32_t ref_index, ref_lane;
    if (type == ARGON2_I || (type == ARGON2_ID && pass == 0 &&
            slice < ARGON2_SYNC_POINTS / 2)) {
        ref_index = (*refs)->ref_index;
        ref_lane = (*refs)->ref_lane;
        (*refs)++;
    } else {
        uint64_t v = u64_shuffle(prev->a, 0, thread, shuffle_buf);
        ref_index = u64_lo(v);
        ref_lane  = u64_hi(v);

        compute_ref_pos(lanes, segment_blocks, pass, lane, slice, offset,
                        &ref_lane, &ref_index);
    }

    argon2_core<version>(memory, mem_curr, prev, tmp, shuffle_buf, lanes,
                         thread, pass, ref_index, ref_lane);
}

template<uint32_t type, uint32_t version>
__global__ void argon2_kernel_segment_precompute(
        struct block_g *memory, const struct ref *refs,
        uint32_t passes, uint32_t lanes, uint32_t segment_blocks,
        uint32_t pass, uint32_t slice)
{
    extern __shared__ struct u64_shuffle_buf shuffle_bufs[];
    struct u64_shuffle_buf *shuffle_buf =
            &shuffle_bufs[blockDim.y * threadIdx.z + threadIdx.y];

    uint32_t job_id = blockIdx.z * blockDim.z + threadIdx.z;
    uint32_t lane   = blockIdx.y * blockDim.y + threadIdx.y;
    uint32_t thread = threadIdx.x;

    uint32_t lane_blocks = ARGON2_SYNC_POINTS * segment_blocks;

    /* select job's memory region: */
    memory += (size_t)job_id * lanes * lane_blocks;

    struct block_th prev, tmp;

    struct block_g *mem_segment =
            memory + slice * segment_blocks * lanes + lane;
    struct block_g *mem_prev, *mem_curr;
    uint32_t start_offset = 0;
    if (pass == 0) {
        if (slice == 0) {
            mem_prev = mem_segment + 1 * lanes;
            mem_curr = mem_segment + 2 * lanes;
            start_offset = 2;
        } else {
            mem_prev = mem_segment - lanes;
            mem_curr = mem_segment;
        }
    } else {
        mem_prev = mem_segment + (slice == 0 ? lane_blocks * lanes : 0) - lanes;
        mem_curr = mem_segment;
    }

    load_block(&prev, mem_prev, thread);

    if (type == ARGON2_ID) {
        if (pass == 0 && slice < ARGON2_SYNC_POINTS / 2) {
            refs += lane * (lane_blocks / 2) + slice * segment_blocks;
            refs += start_offset;
        }
    } else {
        refs += (lane * passes + pass) * lane_blocks + slice * segment_blocks;
        refs += start_offset;
    }

    for (uint32_t offset = start_offset; offset < segment_blocks; ++offset) {
        argon2_step_precompute<type, version>(
                    memory, mem_curr, &prev, &tmp, shuffle_buf, &refs, lanes,
                    segment_blocks, thread, lane, pass, slice, offset);

        mem_curr += lanes;
    }
}

template<uint32_t type, uint32_t version>
__global__ void argon2_kernel_oneshot_precompute(
        struct block_g *memory, const struct ref *refs, uint32_t passes,
        uint32_t lanes, uint32_t segment_blocks)
{
    extern __shared__ struct u64_shuffle_buf shuffle_bufs[];
    struct u64_shuffle_buf *shuffle_buf =
            &shuffle_bufs[lanes * threadIdx.z + threadIdx.y];

    uint32_t job_id = blockIdx.z * blockDim.z + threadIdx.z;
    uint32_t lane   = threadIdx.y;
    uint32_t thread = threadIdx.x;

    uint32_t lane_blocks = ARGON2_SYNC_POINTS * segment_blocks;

    /* select job's memory region: */
    memory += (size_t)job_id * lanes * lane_blocks;

    struct block_th prev, tmp;

    struct block_g *mem_lane = memory + lane;
    struct block_g *mem_prev = mem_lane + 1 * lanes;
    struct block_g *mem_curr = mem_lane + 2 * lanes;

    load_block(&prev, mem_prev, thread);

    if (type == ARGON2_ID) {
        refs += lane * (lane_blocks / 2) + 2;
    } else {
        refs += lane * passes * lane_blocks + 2;
    }

    uint32_t skip = 2;
    for (uint32_t pass = 0; pass < passes; ++pass) {
        for (uint32_t slice = 0; slice < ARGON2_SYNC_POINTS; ++slice) {
            for (uint32_t offset = 0; offset < segment_blocks; ++offset) {
                if (skip > 0) {
                    --skip;
                    continue;
                }

                argon2_step_precompute<type, version>(
                            memory, mem_curr, &prev, &tmp, shuffle_buf, &refs,
                            lanes, segment_blocks, thread,
                            lane, pass, slice, offset);

                mem_curr += lanes;
            }

            __syncthreads();
        }

        mem_curr = mem_lane;
    }
}

template<uint32_t type, uint32_t version>
__device__ void argon2_step(
        struct block_g *memory, struct block_g *mem_curr,
        struct block_th *prev, struct block_th *tmp, struct block_th *addr,
        struct u64_shuffle_buf *shuffle_buf, uint32_t lanes,
        uint32_t segment_blocks, uint32_t thread, uint32_t *thread_input,
        uint32_t lane, uint32_t pass, uint32_t slice, uint32_t offset)
{
    uint32_t ref_index, ref_lane;

    if (type == ARGON2_I || (type == ARGON2_ID && pass == 0 &&
            slice < ARGON2_SYNC_POINTS / 2)) {
        uint32_t addr_index = offset % ARGON2_QWORDS_IN_BLOCK;
        if (addr_index == 0) {
            if (thread == 6) {
                ++*thread_input;
            }
            next_addresses(addr, tmp, *thread_input, thread, shuffle_buf);
        }

        uint32_t thr = addr_index % THREADS_PER_LANE;
        uint32_t idx = addr_index / THREADS_PER_LANE;

        uint64_t v = block_th_get(addr, idx);
        v = u64_shuffle(v, thr, thread, shuffle_buf);
        ref_index = u64_lo(v);
        ref_lane  = u64_hi(v);
    } else {
        uint64_t v = u64_shuffle(prev->a, 0, thread, shuffle_buf);
        ref_index = u64_lo(v);
        ref_lane  = u64_hi(v);
    }

    compute_ref_pos(lanes, segment_blocks, pass, lane, slice, offset,
                    &ref_lane, &ref_index);

    argon2_core<version>(memory, mem_curr, prev, tmp, shuffle_buf, lanes,
                         thread, pass, ref_index, ref_lane);
}

template<uint32_t type, uint32_t version>
__global__ void argon2_kernel_segment(
        struct block_g *memory, uint32_t passes, uint32_t lanes,
        uint32_t segment_blocks, uint32_t pass, uint32_t slice)
{
    extern __shared__ struct u64_shuffle_buf shuffle_bufs[];
    struct u64_shuffle_buf *shuffle_buf =
            &shuffle_bufs[blockDim.y * threadIdx.z + threadIdx.y];

    uint32_t job_id = blockIdx.z * blockDim.z + threadIdx.z;
    uint32_t lane   = blockIdx.y * blockDim.y + threadIdx.y;
    uint32_t thread = threadIdx.x;

    uint32_t lane_blocks = ARGON2_SYNC_POINTS * segment_blocks;

    /* select job's memory region: */
    memory += (size_t)job_id * lanes * lane_blocks;

    struct block_th prev, addr, tmp;
    uint32_t thread_input;

    if (type == ARGON2_I || type == ARGON2_ID) {
        switch (thread) {
        case 0:
            thread_input = pass;
            break;
        case 1:
            thread_input = lane;
            break;
        case 2:
            thread_input = slice;
            break;
        case 3:
            thread_input = lanes * lane_blocks;
            break;
        case 4:
            thread_input = passes;
            break;
        case 5:
            thread_input = type;
            break;
        default:
            thread_input = 0;
            break;
        }

        if (pass == 0 && slice == 0 && segment_blocks > 2) {
            if (thread == 6) {
                ++thread_input;
            }
            next_addresses(&addr, &tmp, thread_input, thread, shuffle_buf);
        }
    }

    struct block_g *mem_segment =
            memory + slice * segment_blocks * lanes + lane;
    struct block_g *mem_prev, *mem_curr;
    uint32_t start_offset = 0;
    if (pass == 0) {
        if (slice == 0) {
            mem_prev = mem_segment + 1 * lanes;
            mem_curr = mem_segment + 2 * lanes;
            start_offset = 2;
        } else {
            mem_prev = mem_segment - lanes;
            mem_curr = mem_segment;
        }
    } else {
        mem_prev = mem_segment + (slice == 0 ? lane_blocks * lanes : 0) - lanes;
        mem_curr = mem_segment;
    }

    load_block(&prev, mem_prev, thread);

    for (uint32_t offset = start_offset; offset < segment_blocks; ++offset) {
        argon2_step<type, version>(
                    memory, mem_curr, &prev, &tmp, &addr, shuffle_buf,
                    lanes, segment_blocks, thread, &thread_input,
                    lane, pass, slice, offset);

        mem_curr += lanes;
    }
}

template<uint32_t type, uint32_t version>
__global__ void argon2_kernel_oneshot(
        struct block_g *memory, uint32_t passes, uint32_t lanes,
        uint32_t segment_blocks)
{
    extern __shared__ struct u64_shuffle_buf shuffle_bufs[];
    struct u64_shuffle_buf *shuffle_buf =
            &shuffle_bufs[lanes * threadIdx.z + threadIdx.y];

    uint32_t job_id = blockIdx.z * blockDim.z + threadIdx.z;
    uint32_t lane   = threadIdx.y;
    uint32_t thread = threadIdx.x;

    uint32_t lane_blocks = ARGON2_SYNC_POINTS * segment_blocks;

    /* select job's memory region: */
    memory += (size_t)job_id * lanes * lane_blocks;

    struct block_th prev, addr, tmp;
    uint32_t thread_input;

    if (type == ARGON2_I || type == ARGON2_ID) {
        switch (thread) {
        case 1:
            thread_input = lane;
            break;
        case 3:
            thread_input = lanes * lane_blocks;
            break;
        case 4:
            thread_input = passes;
            break;
        case 5:
            thread_input = type;
            break;
        default:
            thread_input = 0;
            break;
        }

        if (segment_blocks > 2) {
            if (thread == 6) {
                ++thread_input;
            }
            next_addresses(&addr, &tmp, thread_input, thread, shuffle_buf);
        }
    }

    struct block_g *mem_lane = memory + lane;
    struct block_g *mem_prev = mem_lane + 1 * lanes;
    struct block_g *mem_curr = mem_lane + 2 * lanes;

    load_block(&prev, mem_prev, thread);

    uint32_t skip = 2;
    for (uint32_t pass = 0; pass < passes; ++pass) {
        for (uint32_t slice = 0; slice < ARGON2_SYNC_POINTS; ++slice) {
            for (uint32_t offset = 0; offset < segment_blocks; ++offset) {
                if (skip > 0) {
                    --skip;
                    continue;
                }

                argon2_step<type, version>(
                            memory, mem_curr, &prev, &tmp, &addr, shuffle_buf,
                            lanes, segment_blocks, thread, &thread_input,
                            lane, pass, slice, offset);

                mem_curr += lanes;
            }

            __syncthreads();

            if (type == ARGON2_I || type == ARGON2_ID) {
                if (thread == 2) {
                    ++thread_input;
                }
                if (thread == 6) {
                    thread_input = 0;
                }
            }
        }
        if (type == ARGON2_I) {
            if (thread == 0) {
                ++thread_input;
            }
            if (thread == 2) {
                thread_input = 0;
            }
        }
        mem_curr = mem_lane;
    }
}

__global__ void argon2_kernel_preseed(
		struct block_g *memory, uint32_t *seed, uint32_t lanes, uint32_t segment_blocks) {
	int job_id = blockIdx.x;
	int lane = threadIdx.x % lanes;
	int idx = threadIdx.x / lanes;

	/* select job's memory region: */
	memory += job_id * lanes * ARGON2_SYNC_POINTS * segment_blocks;
	seed += job_id * ARGON2_PREHASH_DIGEST_LENGTH;

	uint32_t initHash[ARGON2_PREHASH_SEED_LENGTH];
	for(int i=0;i<ARGON2_PREHASH_DIGEST_LENGTH;i++) {
		initHash[i] = seed[i];
	}

	initHash[ARGON2_PREHASH_DIGEST_LENGTH] = idx;
	initHash[ARGON2_PREHASH_DIGEST_LENGTH + 1] = lane;
	blake2b_digestLong((uint32_t*)(memory + lane + idx * lanes)->data, ARGON2_DWORDS_IN_BLOCK, initHash, ARGON2_PREHASH_SEED_LENGTH);
}

__global__ void argon2_kernel_finalize(
		struct block_g *memory, uint32_t *out, uint32_t outLen, uint32_t lanes, uint32_t segment_blocks) {
	int job_id = blockIdx.x;
	int thread = threadIdx.x;

	int lane_blocks = ARGON2_SYNC_POINTS * segment_blocks;
	/* select job's memory region: */
	memory += ((job_id + 1) * lanes * lane_blocks - lanes);
	out += job_id * outLen;
	struct block_g *dst = memory;

	for(int l=1;l<lanes;l++) {
		memory += 1;
		for (int i = 0; i < 4/*ARGON2_QWORDS_IN_BLOCK*/; i++) {
			dst->data[thread * 4 + i] ^= memory->data[thread * 4 + i];
		}
	}

	if(thread == 0) {
		blake2b_digestLong(out, outLen, (uint32_t *) dst, ARGON2_DWORDS_IN_BLOCK);
	}
}

KernelRunner::KernelRunner(uint32_t type, uint32_t version, uint32_t passes,
                           uint32_t lanes, uint32_t segmentBlocks,
                           size_t batchSize, size_t outLen, int32_t deviceIndex, bool bySegment, bool precompute)
    : type(type), version(version), passes(passes), lanes(lanes),
      segmentBlocks(segmentBlocks), batchSize(batchSize), outLen(outLen), bySegment(bySegment),
      precompute(precompute), stream(nullptr), memory(nullptr), seed(nullptr), seed_host(nullptr),
      out(nullptr), out_host(nullptr), refs(nullptr)
{

    int currentIndex = -1;
    CudaException::check(hipGetDevice(&currentIndex));
    if (currentIndex != deviceIndex) {
        CudaException::check(hipSetDevice(deviceIndex));
    }

    // FIXME: check overflow:
    size_t memorySize = batchSize * lanes * segmentBlocks
            * ARGON2_SYNC_POINTS * ARGON2_BLOCK_SIZE;

#ifndef NDEBUG
        std::cerr << "[INFO] Allocating " << memorySize << " bytes for memory..."
                  << std::endl;
#endif

    CudaException::check(hipMalloc(&memory, memorySize));
    CudaException::check(hipMalloc(&seed, batchSize * ARGON2_PREHASH_DIGEST_LENGTH * sizeof(uint32_t)));
    CudaException::check(hipMalloc(&out, batchSize * outLen));
    CudaException::check(hipHostMalloc(&seed_host, batchSize * lanes * 2 * ARGON2_BLOCK_SIZE));
    CudaException::check(hipHostMalloc(&out_host, batchSize * outLen));

    CudaException::check(hipStreamCreate(&stream));

    if ((type == ARGON2_I || type == ARGON2_ID) && precompute) {
        uint32_t segments =
                type == ARGON2_ID
                ? lanes * (ARGON2_SYNC_POINTS / 2)
                : passes * lanes * ARGON2_SYNC_POINTS;

        size_t refsSize = segments * segmentBlocks * sizeof(struct ref);

#ifndef NDEBUG
        std::cerr << "[INFO] Allocating " << refsSize << " bytes for refs..."
                  << std::endl;
#endif

        CudaException::check(hipMalloc(&refs, refsSize));

        precomputeRefs();
        synchronize();
    }
}

void KernelRunner::precomputeRefs()
{
    struct ref *refs = (struct ref *)this->refs;

    uint32_t segmentAddrBlocks = (segmentBlocks + ARGON2_QWORDS_IN_BLOCK - 1)
            / ARGON2_QWORDS_IN_BLOCK;
    uint32_t segments =
            type == ARGON2_ID
            ? lanes * (ARGON2_SYNC_POINTS / 2)
            : passes * lanes * ARGON2_SYNC_POINTS;

    dim3 blocks = dim3(1, segments * segmentAddrBlocks);
    dim3 threads = dim3(THREADS_PER_LANE);

    size_t shmemSize = sizeof(struct u64_shuffle_buf);
    if (type == ARGON2_I) {
        argon2_precompute_kernel<ARGON2_I>
            <<<blocks, threads, shmemSize, stream>>>(
                refs, passes, lanes, segmentBlocks);
    } else {
        argon2_precompute_kernel<ARGON2_ID>
            <<<blocks, threads, shmemSize, stream>>>(
                refs, passes, lanes, segmentBlocks);
    }
}

KernelRunner::~KernelRunner()
{
    if (stream != nullptr) {
        hipStreamDestroy(stream);
    }
    if (memory != nullptr) {
        hipFree(memory);
    }
    if (refs != nullptr) {
        hipFree(refs);
    }
    if (seed != nullptr) {
        hipFree(seed);
    }
    if (out != nullptr) {
        hipFree(out);
    }
    if (seed_host != nullptr) {
        hipHostFree(seed_host);
    }
    if (out_host != nullptr) {
        hipHostFree(out_host);
    }
}

void *KernelRunner::getSeedBuffer(int index) {
    return &((uint8_t*)seed_host)[index * ARGON2_PREHASH_DIGEST_LENGTH * sizeof(uint32_t)];
}

void *KernelRunner::getOutBuffer(int index) {
    return &((uint8_t*)out_host)[index * outLen];
}

void KernelRunner::writeInputMemory()
{
    CudaException::check(hipMemcpyAsync(seed, seed_host, batchSize * ARGON2_PREHASH_DIGEST_LENGTH * sizeof(uint32_t),
                                         hipMemcpyHostToDevice, stream));
}

void KernelRunner::readOutputMemory()
{
    CudaException::check(hipMemcpyAsync(out_host, out, batchSize * outLen,
                                         hipMemcpyDeviceToHost, stream));
}

void KernelRunner::runKernelSegment(uint32_t lanesPerBlock,
                                    uint32_t jobsPerBlock,
                                    uint32_t pass, uint32_t slice)
{
    if (lanesPerBlock > lanes || lanes % lanesPerBlock != 0) {
        throw std::logic_error("Invalid lanesPerBlock!");
    }

    if (jobsPerBlock > batchSize || batchSize % jobsPerBlock != 0) {
        throw std::logic_error("Invalid jobsPerBlock!");
    }

    struct block_g *memory_blocks = (struct block_g *)memory;
    dim3 blocks = dim3(1, lanes / lanesPerBlock, batchSize / jobsPerBlock);
    dim3 threads = dim3(THREADS_PER_LANE, lanesPerBlock, jobsPerBlock);
    uint32_t blockSize = lanesPerBlock * jobsPerBlock;
    uint32_t shared_size = blockSize * sizeof(struct u64_shuffle_buf);
    if (type == ARGON2_I) {
        if (precompute) {
            struct ref *refs = (struct ref *)this->refs;
            if (version == ARGON2_VERSION_10) {
                argon2_kernel_segment_precompute<ARGON2_I, ARGON2_VERSION_10>
                        <<<blocks, threads, shared_size, stream>>>(
                            memory_blocks, refs, passes, lanes, segmentBlocks,
                            pass, slice);
            } else {
                argon2_kernel_segment_precompute<ARGON2_I, ARGON2_VERSION_13>
                        <<<blocks, threads, shared_size, stream>>>(
                            memory_blocks, refs, passes, lanes, segmentBlocks,
                            pass, slice);
            }
        } else {
            if (version == ARGON2_VERSION_10) {
                argon2_kernel_segment<ARGON2_I, ARGON2_VERSION_10>
                        <<<blocks, threads, shared_size, stream>>>(
                            memory_blocks, passes, lanes, segmentBlocks,
                            pass, slice);
            } else {
                argon2_kernel_segment<ARGON2_I, ARGON2_VERSION_13>
                        <<<blocks, threads, shared_size, stream>>>(
                            memory_blocks, passes, lanes, segmentBlocks,
                            pass, slice);
            }
        }
    } else if (type == ARGON2_ID) {
        if (precompute) {
            struct ref *refs = (struct ref *)this->refs;
            if (version == ARGON2_VERSION_10) {
                argon2_kernel_segment_precompute<ARGON2_ID, ARGON2_VERSION_10>
                        <<<blocks, threads, shared_size, stream>>>(
                            memory_blocks, refs, passes, lanes, segmentBlocks,
                            pass, slice);
            } else {
                argon2_kernel_segment_precompute<ARGON2_ID, ARGON2_VERSION_13>
                        <<<blocks, threads, shared_size, stream>>>(
                            memory_blocks, refs, passes, lanes, segmentBlocks,
                            pass, slice);
            }
        } else {
            if (version == ARGON2_VERSION_10) {
                argon2_kernel_segment<ARGON2_ID, ARGON2_VERSION_10>
                        <<<blocks, threads, shared_size, stream>>>(
                            memory_blocks, passes, lanes, segmentBlocks,
                            pass, slice);
            } else {
                argon2_kernel_segment<ARGON2_ID, ARGON2_VERSION_13>
                        <<<blocks, threads, shared_size, stream>>>(
                            memory_blocks, passes, lanes, segmentBlocks,
                            pass, slice);
            }
        }
    } else {
        if (version == ARGON2_VERSION_10) {
            argon2_kernel_segment<ARGON2_D, ARGON2_VERSION_10>
                    <<<blocks, threads, shared_size, stream>>>(
                        memory_blocks, passes, lanes, segmentBlocks,
                        pass, slice);
        } else {
            argon2_kernel_segment<ARGON2_D, ARGON2_VERSION_13>
                    <<<blocks, threads, shared_size, stream>>>(
                        memory_blocks, passes, lanes, segmentBlocks,
                        pass, slice);
        }
    }
}

void KernelRunner::runKernelOneshot(uint32_t lanesPerBlock,
                                    uint32_t jobsPerBlock)
{
    if (lanesPerBlock != lanes) {
        throw std::logic_error("Invalid lanesPerBlock!");
    }

    if (jobsPerBlock > batchSize || batchSize % jobsPerBlock != 0) {
        throw std::logic_error("Invalid jobsPerBlock!");
    }

    struct block_g *memory_blocks = (struct block_g *)memory;
    dim3 blocks = dim3(1, 1, batchSize / jobsPerBlock);
    dim3 threads = dim3(THREADS_PER_LANE, lanes, jobsPerBlock);
    uint32_t blockSize = lanesPerBlock * jobsPerBlock;
    uint32_t shared_size = blockSize * sizeof(struct u64_shuffle_buf);
    if (type == ARGON2_I) {
        if (precompute) {
            struct ref *refs = (struct ref *)this->refs;
            if (version == ARGON2_VERSION_10) {
                argon2_kernel_oneshot_precompute<ARGON2_I, ARGON2_VERSION_10>
                        <<<blocks, threads, shared_size, stream>>>(
                            memory_blocks, refs, passes, lanes, segmentBlocks);
            } else {
                argon2_kernel_oneshot_precompute<ARGON2_I, ARGON2_VERSION_13>
                        <<<blocks, threads, shared_size, stream>>>(
                            memory_blocks, refs, passes, lanes, segmentBlocks);
            }
        } else {
            if (version == ARGON2_VERSION_10) {
                argon2_kernel_oneshot<ARGON2_I, ARGON2_VERSION_10>
                        <<<blocks, threads, shared_size, stream>>>(
                            memory_blocks, passes, lanes, segmentBlocks);
            } else {
                argon2_kernel_oneshot<ARGON2_I, ARGON2_VERSION_13>
                        <<<blocks, threads, shared_size, stream>>>(
                            memory_blocks, passes, lanes, segmentBlocks);
            }
        }
    } else if (type == ARGON2_ID) {
        if (precompute) {
            struct ref *refs = (struct ref *)this->refs;
            if (version == ARGON2_VERSION_10) {
                argon2_kernel_oneshot_precompute<ARGON2_ID, ARGON2_VERSION_10>
                        <<<blocks, threads, shared_size, stream>>>(
                            memory_blocks, refs, passes, lanes, segmentBlocks);
            } else {
                argon2_kernel_oneshot_precompute<ARGON2_ID, ARGON2_VERSION_13>
                        <<<blocks, threads, shared_size, stream>>>(
                            memory_blocks, refs, passes, lanes, segmentBlocks);
            }
        } else {
            if (version == ARGON2_VERSION_10) {
                argon2_kernel_oneshot<ARGON2_ID, ARGON2_VERSION_10>
                        <<<blocks, threads, shared_size, stream>>>(
                            memory_blocks, passes, lanes, segmentBlocks);
            } else {
                argon2_kernel_oneshot<ARGON2_ID, ARGON2_VERSION_13>
                        <<<blocks, threads, shared_size, stream>>>(
                            memory_blocks, passes, lanes, segmentBlocks);
            }
        }
    } else {
        if (version == ARGON2_VERSION_10) {
            argon2_kernel_oneshot<ARGON2_D, ARGON2_VERSION_10>
                    <<<blocks, threads, shared_size, stream>>>(
                        memory_blocks, passes, lanes, segmentBlocks);
        } else {
            argon2_kernel_oneshot<ARGON2_D, ARGON2_VERSION_13>
                    <<<blocks, threads, shared_size, stream>>>(
                        memory_blocks, passes, lanes, segmentBlocks);
        }
    }
}

void KernelRunner::runKernelPreseed() {
	struct block_g *memory_blocks = (struct block_g *)memory;
	argon2_kernel_preseed<<<batchSize, lanes * 2>>>(memory_blocks, (uint32_t *)seed, lanes, segmentBlocks);
}

void KernelRunner::runKernelFinalize() {
	struct block_g *memory_blocks = (struct block_g *)memory;
	argon2_kernel_finalize<<<batchSize, 32>>>(memory_blocks, (uint32_t *)out, outLen / 4, lanes, segmentBlocks);
}

void KernelRunner::run(uint32_t lanesPerBlock, uint32_t jobsPerBlock)
{
    timer = get_time();
    runKernelPreseed();
    if (bySegment) {
        for (uint32_t pass = 0; pass < passes; pass++) {
            for (uint32_t slice = 0; slice < ARGON2_SYNC_POINTS; slice++) {
                runKernelSegment(lanesPerBlock, jobsPerBlock, pass, slice);
            }
        }
    } else {
        runKernelOneshot(lanesPerBlock, jobsPerBlock);
    }
	runKernelFinalize();
}

uint64_t KernelRunner::finish()
{
	synchronize();
	return get_time() - timer;
}

void KernelRunner::synchronize() {
	while(hipStreamQuery(stream) != hipSuccess) {
		std::this_thread::sleep_for(std::chrono::milliseconds(1));
	};

	CudaException::check(hipGetLastError());
}

uint64_t KernelRunner::get_time() {
#ifdef _WIN64
	FILETIME ft;
	uint64_t tmpres = 0;
	static int tzflag;

	GetSystemTimeAsFileTime(&ft);

	tmpres |= ft.dwHighDateTime;
	tmpres <<= 32;
	tmpres |= ft.dwLowDateTime;

	return tmpres;
#else
    timespec t;
    clock_gettime(CLOCK_MONOTONIC, &t);
    return t.tv_sec * 1000000000 + t.tv_nsec;
#endif
}

} // cuda
} // argon2

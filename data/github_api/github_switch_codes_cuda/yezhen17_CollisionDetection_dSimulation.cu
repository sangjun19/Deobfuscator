#include "hip/hip_runtime.h"
// Repository: yezhen17/CollisionDetection
// File: CollisionDetection/CollisionDetection/dSimulation.cu

/*
 * The implementation of GPU basic functions and simulation functions
 * reference: CUDA 10.1 samples (particles)
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <thrust/device_ptr.h>
#include <thrust/sort.h>

#include "dSimulationKernel.cuh"

typedef unsigned int uint;

extern "C" {
    void cudaInit(int argc, char **argv) {
        int devID;

        // use command-line specified CUDA device, otherwise use device with highest Gflops/s
        devID = findCudaDevice(argc, (const char **)argv);

        if (devID < 0) {
            printf("No CUDA Capable devices found, please switch to CPU mode, exiting...\n");
            exit(EXIT_SUCCESS);
        }
    }

    void allocateArray(void **dev_ptr, uint size) {
        checkCudaErrors(hipMalloc(dev_ptr, size));
    }

	void zeroizeArray(void *dev_ptr, uint size) {
		checkCudaErrors(hipMemset(dev_ptr, 0x0, size));
	}

    void freeArray(void *dev_ptr) {
        checkCudaErrors(hipFree(dev_ptr));
    }

    void copyHost2Device(void *device, const void *host, int offset, int size) {
        checkCudaErrors(hipMemcpy((char *) device + offset, host, size, hipMemcpyHostToDevice));
    }

	void copyDevice2Host(void *host, const void *device, int size) {
		checkCudaErrors(hipMemcpy(host, device, size, hipMemcpyDeviceToHost));
	}

    void dSetupSimulation(SimulationEnv *h_env,  SimulationSphereProto *h_protos) {
        // copy parameters to constant memory
        checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_env), h_env, sizeof(SimulationEnv)));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_protos), h_protos, sizeof(SimulationSphereProto)));
    }

	void dSimulateFast(float * pos_s, float * velo_s, float * velo_delta_s, uint * types, 
		uint * hashes, uint * indices, uint * cell_start, uint * cell_end, 
		float elapse, uint sphere_num, uint max_hash_value) {
		uint num_threads, num_blocks;
		num_threads = min(256, sphere_num);
		num_blocks = (sphere_num + num_threads - 1) / num_threads;

		// first calculate the hash value of every sphere
		hashifyKernel <<< num_blocks, num_threads >>> (
			hashes,
			indices,
			(float3 *)pos_s);

		getLastCudaError("HashifyKernel execution failed.");

		// use thrust radix sort to sort the hashes
		// and we get the index mapping
		thrust::sort_by_key(
			thrust::device_ptr<uint>(hashes),
			thrust::device_ptr<uint>(hashes + sphere_num),
			thrust::device_ptr<uint>(indices));

		// set all cells to empty
		checkCudaErrors(hipMemset(cell_start, 0xffffffff, max_hash_value * sizeof(uint)));

		// find out all the locations where a cell starts or ends
		collectCellsKernel <<< num_blocks, num_threads >>> (
			cell_start,
			cell_end,
			hashes);

		getLastCudaError("CollectCellsKernel execution failed.");

		// process collision by parallelly traversing every cell
		collisionKernel <<< num_blocks, num_threads >>> (
			(float3 *)velo_delta_s,
			(float3 *)pos_s,
			(float3 *)velo_s,
			types,
			indices,
			cell_start,
			cell_end);

		getLastCudaError("CollisionKernel execution failed.");

		// update the position and velocity of each sphere
		updateDynamicsKernel <<<  num_blocks, num_threads >>> (
			(float3 *)pos_s,
			(float3 *)velo_s,
			(float3 *)velo_delta_s,
			types,
			elapse);

		getLastCudaError("UpdateDynamicsKernel execution failed.");
	}
}

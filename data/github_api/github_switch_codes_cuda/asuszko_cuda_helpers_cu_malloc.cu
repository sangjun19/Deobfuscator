#include "hip/hip_runtime.h"
// Repository: asuszko/cuda_helpers
// File: src/cu_malloc.cu

#include <hip/hip_runtime.h>
#include "cu_errchk.h"
#include "cu_malloc.h"


/**
*  Allocate memory on the device.
*  @param size - [size_t] : Size to allocate in bytes.
*/
void *cu_malloc(size_t size)
{
    void *d_arr;
    gpuErrchk(hipMalloc((void **)&d_arr, size));
    return d_arr;
}

/**
*  Allocate memory on the device.
*  @param int - [n] : Batch count or number of pointers to allocate.
*/
void **cu_malloc_dblptr(void *A_dflat, unsigned long long N, int batch_size, int dtype)
{
    void **A_d;
    gpuErrchk(hipMalloc((void**)&A_d,batch_size*sizeof(size_t)));

    switch(dtype) {
        case 0: 
        {
            float **A = (float **)malloc(batch_size*sizeof(float*));
            A[0] = static_cast<float*>(A_dflat);
            for (int i = 1; i < batch_size; i++) {
                A[i] = A[i-1]+N;
            }
            gpuErrchk(hipMemcpy(A_d,A,batch_size*sizeof(float*),hipMemcpyHostToDevice));
            break;
        }
        case 1: 
        {
            double **A = (double **)malloc(batch_size*sizeof(double*));
            A[0] = static_cast<double*>(A_dflat);
            for (int i = 1; i < batch_size; i++) {
                A[i] = A[i-1]+N;
            }
            gpuErrchk(hipMemcpy(A_d,A,batch_size*sizeof(double*),hipMemcpyHostToDevice));
            break;
        }
        case 2: 
        {
            float2 **A = (float2 **)malloc(batch_size*sizeof(float2*));
            A[0] = static_cast<float2*>(A_dflat);
            for (int i = 1; i < batch_size; i++) {
                A[i] = A[i-1]+N;
            }
            gpuErrchk(hipMemcpy(A_d,A,batch_size*sizeof(float2*),hipMemcpyHostToDevice));
            break;
        }
        case 3: 
        {
            double2 **A = (double2 **)malloc(batch_size*sizeof(double2*));
            A[0] = static_cast<double2*>(A_dflat);
            for (int i = 1; i < batch_size; i++) {
                A[i] = A[i-1]+N;
            }
            gpuErrchk(hipMemcpy(A_d,A,batch_size*sizeof(double2*),hipMemcpyHostToDevice));
            break;
        }
    }
     
    return A_d;
}


/**
*  Allocate mananged memory on the host and device. CUDA will link
*  host and device memory to the same pointer. Thus, this pointer can
*  be accessed from either. Updating values within the array on either
*  the host or device will result in an automatic update of the other.
*  Using managed memory removes the need to explicity call h2d or d2h
*  memory transfers, albeit at the cost of performance.
*  @param size - [size_t] : Size to allocate in bytes.
*/
void *cu_malloc_managed(size_t size)
{
    void *arr;
    gpuErrchk(hipMallocManaged(&arr, size));
    return arr;
}

/**
*  Allocate memory (hipArray) on the device.
*  @param channel - [hipChannelFormatDesc] : hipChannelFormatDesc object
*  @param extent - [dim3] : Dimensions of the hipArray [x,y,z].
*  @param layered - [bool] : hipArray treated as layered.
*/
hipArray *cu_malloc_3d(hipChannelFormatDesc *channel,
                        dim3 extent,
                        bool layered)
{
    hipArray *cu_array;
    if (layered) {
        gpuErrchk(hipMalloc3DArray(&cu_array,
                                    channel,
                                    make_hipExtent(extent.x, extent.y, extent.z),
                                    hipArrayLayered));
    }
    else {
        gpuErrchk(hipMalloc3DArray(&cu_array,
                                    channel,
                                    make_hipExtent(extent.x, extent.y, extent.z)));
    }
    return cu_array;
}



void cu_free(void *d_arr)
{
    gpuErrchk(hipFree(d_arr));
    return;
}

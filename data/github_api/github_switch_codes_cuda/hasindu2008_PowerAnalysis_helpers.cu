// Repository: hasindu2008/PowerAnalysis
// File: other_testbeds/Breaking_Speck/6.postanalysis/corr vs time/keyA-8bit - selection/cuda/helpers.cu

#include <stdio.h>
#include "helpers.cuh"


//Check whether RAM is full
void isMemoryFull(void *ptr){
	if (ptr==NULL){
		fprintf(stderr, "Memory Full.\nYour array is too large. Please try a smaller array.\n");
		exit(EXIT_FAILURE);
	}
}

//check whether file access is ok
void isFileOK(FILE *fp){
	if (fp==NULL){
		perror("A file access error occurred\n");
		exit(EXIT_FAILURE);
	}
}

static const char *getcudaError(hipError_t error){

     switch (error){
        case hipSuccess:
            return "hipSuccess";

        case hipErrorMissingConfiguration:
            return "hipErrorMissingConfiguration";

        case hipErrorOutOfMemory:
            return "hipErrorOutOfMemory";

        case hipErrorNotInitialized:
            return "hipErrorNotInitialized";

        case hipErrorLaunchFailure:
            return "hipErrorLaunchFailure";

        case hipErrorPriorLaunchFailure:
            return "hipErrorPriorLaunchFailure";

        case hipErrorLaunchTimeOut:
            return "hipErrorLaunchTimeOut";

        case hipErrorLaunchOutOfResources:
            return "hipErrorLaunchOutOfResources";

        case hipErrorInvalidDeviceFunction:
            return "hipErrorInvalidDeviceFunction";

        case hipErrorInvalidConfiguration:
            return "hipErrorInvalidConfiguration";

        case hipErrorInvalidDevice:
            return "hipErrorInvalidDevice";

        case hipErrorInvalidValue:
            return "hipErrorInvalidValue";

        case hipErrorInvalidPitchValue:
            return "hipErrorInvalidPitchValue";

        case hipErrorInvalidSymbol:
            return "hipErrorInvalidSymbol";

        case hipErrorMapFailed:
            return "hipErrorMapFailed";

        case hipErrorUnmapFailed:
            return "hipErrorUnmapFailed";

        case cudaErrorInvalidHostPointer:
            return "cudaErrorInvalidHostPointer";

        case hipErrorInvalidDevicePointer:
            return "hipErrorInvalidDevicePointer";

        case hipErrorInvalidTexture:
            return "hipErrorInvalidTexture";

        case cudaErrorInvalidTextureBinding:
            return "cudaErrorInvalidTextureBinding";

        case hipErrorInvalidChannelDescriptor:
            return "hipErrorInvalidChannelDescriptor";

        case hipErrorInvalidMemcpyDirection:
            return "hipErrorInvalidMemcpyDirection";

        case cudaErrorAddressOfConstant:
            return "cudaErrorAddressOfConstant";

        case cudaErrorTextureFetchFailed:
            return "cudaErrorTextureFetchFailed";

        case cudaErrorTextureNotBound:
            return "cudaErrorTextureNotBound";

        case cudaErrorSynchronizationError:
            return "cudaErrorSynchronizationError";

        case cudaErrorInvalidFilterSetting:
            return "cudaErrorInvalidFilterSetting";

        case cudaErrorInvalidNormSetting:
            return "cudaErrorInvalidNormSetting";

        case cudaErrorMixedDeviceExecution:
            return "cudaErrorMixedDeviceExecution";

        case hipErrorDeinitialized:
            return "hipErrorDeinitialized";

        case hipErrorUnknown:
            return "hipErrorUnknown";

        case cudaErrorNotYetImplemented:
            return "cudaErrorNotYetImplemented";

        case cudaErrorMemoryValueTooLarge:
            return "cudaErrorMemoryValueTooLarge";

        case hipErrorInvalidHandle:
            return "hipErrorInvalidHandle";

        case hipErrorNotReady:
            return "hipErrorNotReady";

        case hipErrorInsufficientDriver:
            return "hipErrorInsufficientDriver";

        case hipErrorSetOnActiveProcess:
            return "hipErrorSetOnActiveProcess";

        case cudaErrorInvalidSurface:
            return "cudaErrorInvalidSurface";

        case hipErrorNoDevice:
            return "hipErrorNoDevice";

        case hipErrorECCNotCorrectable:
            return "hipErrorECCNotCorrectable";

        case hipErrorSharedObjectSymbolNotFound:
            return "hipErrorSharedObjectSymbolNotFound";

        case hipErrorSharedObjectInitFailed:
            return "hipErrorSharedObjectInitFailed";

        case hipErrorUnsupportedLimit:
            return "hipErrorUnsupportedLimit";

        case cudaErrorDuplicateVariableName:
            return "cudaErrorDuplicateVariableName";

        case cudaErrorDuplicateTextureName:
            return "cudaErrorDuplicateTextureName";

        case cudaErrorDuplicateSurfaceName:
            return "cudaErrorDuplicateSurfaceName";

        case cudaErrorDevicesUnavailable:
            return "cudaErrorDevicesUnavailable";

        case hipErrorInvalidImage:
            return "hipErrorInvalidImage";

        case hipErrorNoBinaryForGpu:
            return "hipErrorNoBinaryForGpu";

        case cudaErrorIncompatibleDriverContext:
            return "cudaErrorIncompatibleDriverContext";

        case hipErrorPeerAccessAlreadyEnabled:
            return "hipErrorPeerAccessAlreadyEnabled";

        case hipErrorPeerAccessNotEnabled:
            return "hipErrorPeerAccessNotEnabled";

        case hipErrorContextAlreadyInUse:
            return "hipErrorContextAlreadyInUse";

        case hipErrorProfilerDisabled:
            return "hipErrorProfilerDisabled";

        case hipErrorProfilerNotInitialized:
            return "hipErrorProfilerNotInitialized";

        case hipErrorProfilerAlreadyStarted:
            return "hipErrorProfilerAlreadyStarted";

        case hipErrorProfilerAlreadyStopped:
            return "hipErrorProfilerAlreadyStopped";

        /* Since CUDA 4.0*/
        case hipErrorAssert:
            return "hipErrorAssert";

        case cudaErrorTooManyPeers:
            return "cudaErrorTooManyPeers";

        case hipErrorHostMemoryAlreadyRegistered:
            return "hipErrorHostMemoryAlreadyRegistered";

        case hipErrorHostMemoryNotRegistered:
            return "hipErrorHostMemoryNotRegistered";

        /* Since CUDA 5.0 */
        case hipErrorOperatingSystem:
            return "hipErrorOperatingSystem";

        case hipErrorPeerAccessUnsupported:
            return "hipErrorPeerAccessUnsupported";

        case cudaErrorLaunchMaxDepthExceeded:
            return "cudaErrorLaunchMaxDepthExceeded";

        case cudaErrorLaunchFileScopedTex:
            return "cudaErrorLaunchFileScopedTex";

        case cudaErrorLaunchFileScopedSurf:
            return "cudaErrorLaunchFileScopedSurf";

        case cudaErrorSyncDepthExceeded:
            return "cudaErrorSyncDepthExceeded";

        case cudaErrorLaunchPendingCountExceeded:
            return "cudaErrorLaunchPendingCountExceeded";

        case cudaErrorNotPermitted:
            return "cudaErrorNotPermitted";

        case hipErrorNotSupported:
            return "hipErrorNotSupported";

        /* Since CUDA 6.0 */
        case cudaErrorHardwareStackError:
            return "cudaErrorHardwareStackError";

        case cudaErrorIllegalInstruction:
            return "cudaErrorIllegalInstruction";

        case cudaErrorMisalignedAddress:
            return "cudaErrorMisalignedAddress";

        case cudaErrorInvalidAddressSpace:
            return "cudaErrorInvalidAddressSpace";

        case cudaErrorInvalidPc:
            return "cudaErrorInvalidPc";

        case hipErrorIllegalAddress:
            return "hipErrorIllegalAddress";

        /* Since CUDA 6.5*/
        case hipErrorInvalidKernelFile:
            return "hipErrorInvalidKernelFile";

        case hipErrorInvalidGraphicsContext:
            return "hipErrorInvalidGraphicsContext";

        case cudaErrorStartupFailure:
            return "cudaErrorStartupFailure";

        case cudaErrorApiFailureBase:
            return "cudaErrorApiFailureBase";
    }

    return "<unknown>";
}
		
		
//check whether cuda errors
void checkCudaError(hipError_t status){
	if (status!=hipSuccess){
		fprintf(stderr,"Some Error occured in CUDA.\n:%s \nError Code : %d\n",getcudaError(status),status);
		exit(EXIT_FAILURE);
	}
}
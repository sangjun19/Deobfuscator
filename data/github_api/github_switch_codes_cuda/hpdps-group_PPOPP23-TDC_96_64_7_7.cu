// Repository: hpdps-group/PPOPP23-TDC
// File: A100-evaluation/TDC-LAYERS-ORACLE/96_64_7_7.cu

#include <cudnn.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
#include <omp.h>
#define TH 4
#define TW 1
#define TC 8
#define C 96
#define N 64
#define H 7
#define W 7

#define TCS ((C-1)/TC + 1)
#define THS ((H-1)/TH + 1)
#define TWS ((W-1)/TW+1)
#define WPAD (TWS*TW + 2)
#define R 3
#define S 3


using namespace std;
#define checkCUDNN(expression)                               \
  {                                                          \
    cudnnStatus_t status = (expression);                     \
    if (status != CUDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << cudnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }
inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cerr << "ERROR!!!:" << hipGetErrorString(code) <<endl;
        exit(-1);
    }
}
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[1];
  __shared__ float pad_temp_shared[504];
  __shared__ float kernel_shared[576];
  float pad_temp_shared_local[24];
  float kernel_shared_local[24];
  compute_local[(0)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 4; ++rc_outer) {
    __syncthreads();
    pad_temp_shared[(((((int)threadIdx.z) * 63) + (((int)threadIdx.x) * 9)))] = ((((1 <= ((((((int)threadIdx.x) * 9) % 21) / 7) + ((int)blockIdx.y))) && (((((((int)threadIdx.x) * 9) % 21) / 7) + ((int)blockIdx.y)) < 8)) && (1 <= ((((int)threadIdx.x) * 9) % 7))) ? data[(((((((rc_outer * 1176) + (((int)threadIdx.z) * 147)) + (((((int)threadIdx.x) * 9) / 21) * 49)) + (((int)blockIdx.y) * 7)) + ((((int)threadIdx.x) * 9) % 21)) - 8))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 63) + (((int)threadIdx.x) * 9)) + 1))] = ((((1 <= (((((((int)threadIdx.x) * 9) + 1) % 21) / 7) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 9) + 1) % 21) / 7) + ((int)blockIdx.y)) < 8)) && (1 <= (((((int)threadIdx.x) * 9) + 1) % 7))) ? data[(((((((rc_outer * 1176) + (((int)threadIdx.z) * 147)) + ((((((int)threadIdx.x) * 9) + 1) / 21) * 49)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 9) + 1) % 21)) - 8))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 63) + (((int)threadIdx.x) * 9)) + 2))] = ((((1 <= (((((((int)threadIdx.x) * 9) + 2) % 21) / 7) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 9) + 2) % 21) / 7) + ((int)blockIdx.y)) < 8)) && (1 <= (((((int)threadIdx.x) * 9) + 2) % 7))) ? data[(((((((rc_outer * 1176) + (((int)threadIdx.z) * 147)) + ((((((int)threadIdx.x) * 9) + 2) / 21) * 49)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 9) + 2) % 21)) - 8))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 63) + (((int)threadIdx.x) * 9)) + 3))] = ((((1 <= (((((((int)threadIdx.x) * 9) + 3) % 21) / 7) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 9) + 3) % 21) / 7) + ((int)blockIdx.y)) < 8)) && (1 <= (((((int)threadIdx.x) * 9) + 3) % 7))) ? data[(((((((rc_outer * 1176) + (((int)threadIdx.z) * 147)) + ((((((int)threadIdx.x) * 9) + 3) / 21) * 49)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 9) + 3) % 21)) - 8))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 63) + (((int)threadIdx.x) * 9)) + 4))] = ((((1 <= (((((((int)threadIdx.x) * 9) + 4) % 21) / 7) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 9) + 4) % 21) / 7) + ((int)blockIdx.y)) < 8)) && (1 <= (((((int)threadIdx.x) * 9) + 4) % 7))) ? data[(((((((rc_outer * 1176) + (((int)threadIdx.z) * 147)) + ((((((int)threadIdx.x) * 9) + 4) / 21) * 49)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 9) + 4) % 21)) - 8))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 63) + (((int)threadIdx.x) * 9)) + 5))] = ((((1 <= (((((((int)threadIdx.x) * 9) + 5) % 21) / 7) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 9) + 5) % 21) / 7) + ((int)blockIdx.y)) < 8)) && (1 <= (((((int)threadIdx.x) * 9) + 5) % 7))) ? data[(((((((rc_outer * 1176) + (((int)threadIdx.z) * 147)) + ((((((int)threadIdx.x) * 9) + 5) / 21) * 49)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 9) + 5) % 21)) - 8))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 63) + (((int)threadIdx.x) * 9)) + 6))] = ((((1 <= (((((((int)threadIdx.x) * 9) + 6) % 21) / 7) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 9) + 6) % 21) / 7) + ((int)blockIdx.y)) < 8)) && (1 <= (((((int)threadIdx.x) * 9) + 6) % 7))) ? data[(((((((rc_outer * 1176) + (((int)threadIdx.z) * 147)) + ((((((int)threadIdx.x) * 9) + 6) / 21) * 49)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 9) + 6) % 21)) - 8))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 63) + (((int)threadIdx.x) * 9)) + 7))] = ((((1 <= (((((((int)threadIdx.x) * 9) + 7) % 21) / 7) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 9) + 7) % 21) / 7) + ((int)blockIdx.y)) < 8)) && (1 <= ((((int)threadIdx.x) * 9) % 7))) ? data[(((((((rc_outer * 1176) + (((int)threadIdx.z) * 147)) + ((((((int)threadIdx.x) * 9) + 7) / 21) * 49)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 9) + 7) % 21)) - 8))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 63) + (((int)threadIdx.x) * 9)) + 8))] = ((((1 <= (((((((int)threadIdx.x) * 9) + 8) % 21) / 7) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 9) + 8) % 21) / 7) + ((int)blockIdx.y)) < 8)) && (1 <= (((((int)threadIdx.x) * 9) + 1) % 7))) ? data[(((((((rc_outer * 1176) + (((int)threadIdx.z) * 147)) + ((((((int)threadIdx.x) * 9) + 8) / 21) * 49)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 9) + 8) % 21)) - 8))] : 0.000000e+00f);
    kernel_shared[(((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)))] = kernel[(((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 1))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 3))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 2))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 6))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 3))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 9))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 4))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 12))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 5))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 15))];
    if (((((((int)threadIdx.x) * 11) + 6) / 72) + ((int)threadIdx.z)) < 8) {
      if (((((int)threadIdx.z) * 24) + ((((int)threadIdx.x) * 11) / 3)) < 190) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 570) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 6))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 18))];
          }
        }
      }
    }
    if (((((((int)threadIdx.x) * 11) + 7) / 72) + ((int)threadIdx.z)) < 8) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.x) * 11) + 7) / 3)) < 192) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 569) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 7))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 21))];
          }
        }
      }
    }
    if (((((((int)threadIdx.x) * 11) + 8) / 72) + ((int)threadIdx.z)) < 8) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.x) * 11) + 8) / 3)) < 192) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 568) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 8))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 24))];
          }
        }
      }
    }
    if (((((((int)threadIdx.x) * 11) + 9) / 72) + ((int)threadIdx.z)) < 8) {
      if (((((int)threadIdx.z) * 24) + ((((int)threadIdx.x) * 11) / 3)) < 189) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 567) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 9))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 27))];
          }
        }
      }
    }
    if (((((((int)threadIdx.x) * 11) + 10) / 72) + ((int)threadIdx.z)) < 8) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.x) * 11) + 10) / 3)) < 192) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 566) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 10))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 30))];
          }
        }
      }
    }
    __syncthreads();
    pad_temp_shared_local[(0)] = pad_temp_shared[(((int)threadIdx.x))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 21))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 42))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 63))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 84))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 105))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 126))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 147))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 168))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 189))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 210))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 231))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 252))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 273))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 294))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 315))];
    pad_temp_shared_local[(16)] = pad_temp_shared[((((int)threadIdx.x) + 336))];
    pad_temp_shared_local[(17)] = pad_temp_shared[((((int)threadIdx.x) + 357))];
    pad_temp_shared_local[(18)] = pad_temp_shared[((((int)threadIdx.x) + 378))];
    pad_temp_shared_local[(19)] = pad_temp_shared[((((int)threadIdx.x) + 399))];
    pad_temp_shared_local[(20)] = pad_temp_shared[((((int)threadIdx.x) + 420))];
    pad_temp_shared_local[(21)] = pad_temp_shared[((((int)threadIdx.x) + 441))];
    pad_temp_shared_local[(22)] = pad_temp_shared[((((int)threadIdx.x) + 462))];
    pad_temp_shared_local[(23)] = pad_temp_shared[((((int)threadIdx.x) + 483))];
    kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 72))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 3))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 6))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 9))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 12))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 15))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 72) + 18))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 72) + 21))];
    kernel_shared_local[(8)] = kernel_shared[(((((int)threadIdx.z) * 72) + 24))];
    kernel_shared_local[(9)] = kernel_shared[(((((int)threadIdx.z) * 72) + 27))];
    kernel_shared_local[(10)] = kernel_shared[(((((int)threadIdx.z) * 72) + 30))];
    kernel_shared_local[(11)] = kernel_shared[(((((int)threadIdx.z) * 72) + 33))];
    kernel_shared_local[(12)] = kernel_shared[(((((int)threadIdx.z) * 72) + 36))];
    kernel_shared_local[(13)] = kernel_shared[(((((int)threadIdx.z) * 72) + 39))];
    kernel_shared_local[(14)] = kernel_shared[(((((int)threadIdx.z) * 72) + 42))];
    kernel_shared_local[(15)] = kernel_shared[(((((int)threadIdx.z) * 72) + 45))];
    kernel_shared_local[(16)] = kernel_shared[(((((int)threadIdx.z) * 72) + 48))];
    kernel_shared_local[(17)] = kernel_shared[(((((int)threadIdx.z) * 72) + 51))];
    kernel_shared_local[(18)] = kernel_shared[(((((int)threadIdx.z) * 72) + 54))];
    kernel_shared_local[(19)] = kernel_shared[(((((int)threadIdx.z) * 72) + 57))];
    kernel_shared_local[(20)] = kernel_shared[(((((int)threadIdx.z) * 72) + 60))];
    kernel_shared_local[(21)] = kernel_shared[(((((int)threadIdx.z) * 72) + 63))];
    kernel_shared_local[(22)] = kernel_shared[(((((int)threadIdx.z) * 72) + 66))];
    kernel_shared_local[(23)] = kernel_shared[(((((int)threadIdx.z) * 72) + 69))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(11)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(12)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(13)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(14)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(15)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(16)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(17)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(18)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(19)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(20)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(21)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(22)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(23)] * kernel_shared_local[(23)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 7))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 28))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 49))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 70))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 91))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 112))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 133))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 154))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 175))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 196))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 217))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 238))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 259))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 280))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 301))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 322))];
    pad_temp_shared_local[(16)] = pad_temp_shared[((((int)threadIdx.x) + 343))];
    pad_temp_shared_local[(17)] = pad_temp_shared[((((int)threadIdx.x) + 364))];
    pad_temp_shared_local[(18)] = pad_temp_shared[((((int)threadIdx.x) + 385))];
    pad_temp_shared_local[(19)] = pad_temp_shared[((((int)threadIdx.x) + 406))];
    pad_temp_shared_local[(20)] = pad_temp_shared[((((int)threadIdx.x) + 427))];
    pad_temp_shared_local[(21)] = pad_temp_shared[((((int)threadIdx.x) + 448))];
    pad_temp_shared_local[(22)] = pad_temp_shared[((((int)threadIdx.x) + 469))];
    pad_temp_shared_local[(23)] = pad_temp_shared[((((int)threadIdx.x) + 490))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 1))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 4))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 7))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 10))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 13))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 16))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 72) + 19))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 72) + 22))];
    kernel_shared_local[(8)] = kernel_shared[(((((int)threadIdx.z) * 72) + 25))];
    kernel_shared_local[(9)] = kernel_shared[(((((int)threadIdx.z) * 72) + 28))];
    kernel_shared_local[(10)] = kernel_shared[(((((int)threadIdx.z) * 72) + 31))];
    kernel_shared_local[(11)] = kernel_shared[(((((int)threadIdx.z) * 72) + 34))];
    kernel_shared_local[(12)] = kernel_shared[(((((int)threadIdx.z) * 72) + 37))];
    kernel_shared_local[(13)] = kernel_shared[(((((int)threadIdx.z) * 72) + 40))];
    kernel_shared_local[(14)] = kernel_shared[(((((int)threadIdx.z) * 72) + 43))];
    kernel_shared_local[(15)] = kernel_shared[(((((int)threadIdx.z) * 72) + 46))];
    kernel_shared_local[(16)] = kernel_shared[(((((int)threadIdx.z) * 72) + 49))];
    kernel_shared_local[(17)] = kernel_shared[(((((int)threadIdx.z) * 72) + 52))];
    kernel_shared_local[(18)] = kernel_shared[(((((int)threadIdx.z) * 72) + 55))];
    kernel_shared_local[(19)] = kernel_shared[(((((int)threadIdx.z) * 72) + 58))];
    kernel_shared_local[(20)] = kernel_shared[(((((int)threadIdx.z) * 72) + 61))];
    kernel_shared_local[(21)] = kernel_shared[(((((int)threadIdx.z) * 72) + 64))];
    kernel_shared_local[(22)] = kernel_shared[(((((int)threadIdx.z) * 72) + 67))];
    kernel_shared_local[(23)] = kernel_shared[(((((int)threadIdx.z) * 72) + 70))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(11)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(12)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(13)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(14)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(15)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(16)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(17)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(18)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(19)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(20)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(21)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(22)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(23)] * kernel_shared_local[(23)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 14))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 35))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 56))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 77))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 98))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 119))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 140))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 161))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 182))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 203))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 224))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 245))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 266))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 287))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 308))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 329))];
    pad_temp_shared_local[(16)] = pad_temp_shared[((((int)threadIdx.x) + 350))];
    pad_temp_shared_local[(17)] = pad_temp_shared[((((int)threadIdx.x) + 371))];
    pad_temp_shared_local[(18)] = pad_temp_shared[((((int)threadIdx.x) + 392))];
    pad_temp_shared_local[(19)] = pad_temp_shared[((((int)threadIdx.x) + 413))];
    pad_temp_shared_local[(20)] = pad_temp_shared[((((int)threadIdx.x) + 434))];
    pad_temp_shared_local[(21)] = pad_temp_shared[((((int)threadIdx.x) + 455))];
    pad_temp_shared_local[(22)] = pad_temp_shared[((((int)threadIdx.x) + 476))];
    pad_temp_shared_local[(23)] = pad_temp_shared[((((int)threadIdx.x) + 497))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 2))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 5))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 8))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 11))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 14))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 17))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 72) + 20))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 72) + 23))];
    kernel_shared_local[(8)] = kernel_shared[(((((int)threadIdx.z) * 72) + 26))];
    kernel_shared_local[(9)] = kernel_shared[(((((int)threadIdx.z) * 72) + 29))];
    kernel_shared_local[(10)] = kernel_shared[(((((int)threadIdx.z) * 72) + 32))];
    kernel_shared_local[(11)] = kernel_shared[(((((int)threadIdx.z) * 72) + 35))];
    kernel_shared_local[(12)] = kernel_shared[(((((int)threadIdx.z) * 72) + 38))];
    kernel_shared_local[(13)] = kernel_shared[(((((int)threadIdx.z) * 72) + 41))];
    kernel_shared_local[(14)] = kernel_shared[(((((int)threadIdx.z) * 72) + 44))];
    kernel_shared_local[(15)] = kernel_shared[(((((int)threadIdx.z) * 72) + 47))];
    kernel_shared_local[(16)] = kernel_shared[(((((int)threadIdx.z) * 72) + 50))];
    kernel_shared_local[(17)] = kernel_shared[(((((int)threadIdx.z) * 72) + 53))];
    kernel_shared_local[(18)] = kernel_shared[(((((int)threadIdx.z) * 72) + 56))];
    kernel_shared_local[(19)] = kernel_shared[(((((int)threadIdx.z) * 72) + 59))];
    kernel_shared_local[(20)] = kernel_shared[(((((int)threadIdx.z) * 72) + 62))];
    kernel_shared_local[(21)] = kernel_shared[(((((int)threadIdx.z) * 72) + 65))];
    kernel_shared_local[(22)] = kernel_shared[(((((int)threadIdx.z) * 72) + 68))];
    kernel_shared_local[(23)] = kernel_shared[(((((int)threadIdx.z) * 72) + 71))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(11)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(12)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(13)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(14)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(15)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(16)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(17)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(18)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(19)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(20)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(21)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(22)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(23)] * kernel_shared_local[(23)]));
    __syncthreads();
    pad_temp_shared[(((((int)threadIdx.z) * 63) + (((int)threadIdx.x) * 9)))] = (((1 <= ((((((int)threadIdx.x) * 9) % 21) / 7) + ((int)blockIdx.y))) && (((((((int)threadIdx.x) * 9) % 21) / 7) + ((int)blockIdx.y)) < 8)) ? data[(((((((rc_outer * 1176) + (((int)threadIdx.z) * 147)) + (((((int)threadIdx.x) * 9) / 21) * 49)) + (((int)blockIdx.y) * 7)) + ((((int)threadIdx.x) * 9) % 21)) - 7))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 63) + (((int)threadIdx.x) * 9)) + 1))] = (((1 <= (((((((int)threadIdx.x) * 9) + 1) % 21) / 7) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 9) + 1) % 21) / 7) + ((int)blockIdx.y)) < 8)) ? data[(((((((rc_outer * 1176) + (((int)threadIdx.z) * 147)) + ((((((int)threadIdx.x) * 9) + 1) / 21) * 49)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 9) + 1) % 21)) - 7))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 63) + (((int)threadIdx.x) * 9)) + 2))] = (((1 <= (((((((int)threadIdx.x) * 9) + 2) % 21) / 7) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 9) + 2) % 21) / 7) + ((int)blockIdx.y)) < 8)) ? data[(((((((rc_outer * 1176) + (((int)threadIdx.z) * 147)) + ((((((int)threadIdx.x) * 9) + 2) / 21) * 49)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 9) + 2) % 21)) - 7))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 63) + (((int)threadIdx.x) * 9)) + 3))] = (((1 <= (((((((int)threadIdx.x) * 9) + 3) % 21) / 7) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 9) + 3) % 21) / 7) + ((int)blockIdx.y)) < 8)) ? data[(((((((rc_outer * 1176) + (((int)threadIdx.z) * 147)) + ((((((int)threadIdx.x) * 9) + 3) / 21) * 49)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 9) + 3) % 21)) - 7))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 63) + (((int)threadIdx.x) * 9)) + 4))] = (((1 <= (((((((int)threadIdx.x) * 9) + 4) % 21) / 7) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 9) + 4) % 21) / 7) + ((int)blockIdx.y)) < 8)) ? data[(((((((rc_outer * 1176) + (((int)threadIdx.z) * 147)) + ((((((int)threadIdx.x) * 9) + 4) / 21) * 49)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 9) + 4) % 21)) - 7))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 63) + (((int)threadIdx.x) * 9)) + 5))] = (((1 <= (((((((int)threadIdx.x) * 9) + 5) % 21) / 7) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 9) + 5) % 21) / 7) + ((int)blockIdx.y)) < 8)) ? data[(((((((rc_outer * 1176) + (((int)threadIdx.z) * 147)) + ((((((int)threadIdx.x) * 9) + 5) / 21) * 49)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 9) + 5) % 21)) - 7))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 63) + (((int)threadIdx.x) * 9)) + 6))] = (((1 <= (((((((int)threadIdx.x) * 9) + 6) % 21) / 7) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 9) + 6) % 21) / 7) + ((int)blockIdx.y)) < 8)) ? data[(((((((rc_outer * 1176) + (((int)threadIdx.z) * 147)) + ((((((int)threadIdx.x) * 9) + 6) / 21) * 49)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 9) + 6) % 21)) - 7))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 63) + (((int)threadIdx.x) * 9)) + 7))] = (((1 <= (((((((int)threadIdx.x) * 9) + 7) % 21) / 7) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 9) + 7) % 21) / 7) + ((int)blockIdx.y)) < 8)) ? data[(((((((rc_outer * 1176) + (((int)threadIdx.z) * 147)) + ((((((int)threadIdx.x) * 9) + 7) / 21) * 49)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 9) + 7) % 21)) - 7))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 63) + (((int)threadIdx.x) * 9)) + 8))] = (((1 <= (((((((int)threadIdx.x) * 9) + 8) % 21) / 7) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 9) + 8) % 21) / 7) + ((int)blockIdx.y)) < 8)) ? data[(((((((rc_outer * 1176) + (((int)threadIdx.z) * 147)) + ((((((int)threadIdx.x) * 9) + 8) / 21) * 49)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 9) + 8) % 21)) - 7))] : 0.000000e+00f);
    kernel_shared[(((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 1))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 1))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 4))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 2))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 7))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 3))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 10))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 4))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 13))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 5))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 16))];
    if (((((((int)threadIdx.x) * 11) + 6) / 72) + ((int)threadIdx.z)) < 8) {
      if (((((int)threadIdx.z) * 24) + ((((int)threadIdx.x) * 11) / 3)) < 190) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 570) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 6))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 19))];
          }
        }
      }
    }
    if (((((((int)threadIdx.x) * 11) + 7) / 72) + ((int)threadIdx.z)) < 8) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.x) * 11) + 7) / 3)) < 192) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 569) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 7))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 22))];
          }
        }
      }
    }
    if (((((((int)threadIdx.x) * 11) + 8) / 72) + ((int)threadIdx.z)) < 8) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.x) * 11) + 8) / 3)) < 192) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 568) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 8))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 25))];
          }
        }
      }
    }
    if (((((((int)threadIdx.x) * 11) + 9) / 72) + ((int)threadIdx.z)) < 8) {
      if (((((int)threadIdx.z) * 24) + ((((int)threadIdx.x) * 11) / 3)) < 189) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 567) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 9))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 28))];
          }
        }
      }
    }
    if (((((((int)threadIdx.x) * 11) + 10) / 72) + ((int)threadIdx.z)) < 8) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.x) * 11) + 10) / 3)) < 192) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 566) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 10))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 31))];
          }
        }
      }
    }
    __syncthreads();
    pad_temp_shared_local[(0)] = pad_temp_shared[(((int)threadIdx.x))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 21))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 42))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 63))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 84))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 105))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 126))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 147))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 168))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 189))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 210))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 231))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 252))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 273))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 294))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 315))];
    pad_temp_shared_local[(16)] = pad_temp_shared[((((int)threadIdx.x) + 336))];
    pad_temp_shared_local[(17)] = pad_temp_shared[((((int)threadIdx.x) + 357))];
    pad_temp_shared_local[(18)] = pad_temp_shared[((((int)threadIdx.x) + 378))];
    pad_temp_shared_local[(19)] = pad_temp_shared[((((int)threadIdx.x) + 399))];
    pad_temp_shared_local[(20)] = pad_temp_shared[((((int)threadIdx.x) + 420))];
    pad_temp_shared_local[(21)] = pad_temp_shared[((((int)threadIdx.x) + 441))];
    pad_temp_shared_local[(22)] = pad_temp_shared[((((int)threadIdx.x) + 462))];
    pad_temp_shared_local[(23)] = pad_temp_shared[((((int)threadIdx.x) + 483))];
    kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 72))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 3))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 6))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 9))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 12))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 15))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 72) + 18))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 72) + 21))];
    kernel_shared_local[(8)] = kernel_shared[(((((int)threadIdx.z) * 72) + 24))];
    kernel_shared_local[(9)] = kernel_shared[(((((int)threadIdx.z) * 72) + 27))];
    kernel_shared_local[(10)] = kernel_shared[(((((int)threadIdx.z) * 72) + 30))];
    kernel_shared_local[(11)] = kernel_shared[(((((int)threadIdx.z) * 72) + 33))];
    kernel_shared_local[(12)] = kernel_shared[(((((int)threadIdx.z) * 72) + 36))];
    kernel_shared_local[(13)] = kernel_shared[(((((int)threadIdx.z) * 72) + 39))];
    kernel_shared_local[(14)] = kernel_shared[(((((int)threadIdx.z) * 72) + 42))];
    kernel_shared_local[(15)] = kernel_shared[(((((int)threadIdx.z) * 72) + 45))];
    kernel_shared_local[(16)] = kernel_shared[(((((int)threadIdx.z) * 72) + 48))];
    kernel_shared_local[(17)] = kernel_shared[(((((int)threadIdx.z) * 72) + 51))];
    kernel_shared_local[(18)] = kernel_shared[(((((int)threadIdx.z) * 72) + 54))];
    kernel_shared_local[(19)] = kernel_shared[(((((int)threadIdx.z) * 72) + 57))];
    kernel_shared_local[(20)] = kernel_shared[(((((int)threadIdx.z) * 72) + 60))];
    kernel_shared_local[(21)] = kernel_shared[(((((int)threadIdx.z) * 72) + 63))];
    kernel_shared_local[(22)] = kernel_shared[(((((int)threadIdx.z) * 72) + 66))];
    kernel_shared_local[(23)] = kernel_shared[(((((int)threadIdx.z) * 72) + 69))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(11)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(12)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(13)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(14)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(15)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(16)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(17)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(18)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(19)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(20)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(21)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(22)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(23)] * kernel_shared_local[(23)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 7))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 28))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 49))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 70))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 91))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 112))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 133))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 154))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 175))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 196))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 217))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 238))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 259))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 280))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 301))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 322))];
    pad_temp_shared_local[(16)] = pad_temp_shared[((((int)threadIdx.x) + 343))];
    pad_temp_shared_local[(17)] = pad_temp_shared[((((int)threadIdx.x) + 364))];
    pad_temp_shared_local[(18)] = pad_temp_shared[((((int)threadIdx.x) + 385))];
    pad_temp_shared_local[(19)] = pad_temp_shared[((((int)threadIdx.x) + 406))];
    pad_temp_shared_local[(20)] = pad_temp_shared[((((int)threadIdx.x) + 427))];
    pad_temp_shared_local[(21)] = pad_temp_shared[((((int)threadIdx.x) + 448))];
    pad_temp_shared_local[(22)] = pad_temp_shared[((((int)threadIdx.x) + 469))];
    pad_temp_shared_local[(23)] = pad_temp_shared[((((int)threadIdx.x) + 490))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 1))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 4))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 7))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 10))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 13))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 16))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 72) + 19))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 72) + 22))];
    kernel_shared_local[(8)] = kernel_shared[(((((int)threadIdx.z) * 72) + 25))];
    kernel_shared_local[(9)] = kernel_shared[(((((int)threadIdx.z) * 72) + 28))];
    kernel_shared_local[(10)] = kernel_shared[(((((int)threadIdx.z) * 72) + 31))];
    kernel_shared_local[(11)] = kernel_shared[(((((int)threadIdx.z) * 72) + 34))];
    kernel_shared_local[(12)] = kernel_shared[(((((int)threadIdx.z) * 72) + 37))];
    kernel_shared_local[(13)] = kernel_shared[(((((int)threadIdx.z) * 72) + 40))];
    kernel_shared_local[(14)] = kernel_shared[(((((int)threadIdx.z) * 72) + 43))];
    kernel_shared_local[(15)] = kernel_shared[(((((int)threadIdx.z) * 72) + 46))];
    kernel_shared_local[(16)] = kernel_shared[(((((int)threadIdx.z) * 72) + 49))];
    kernel_shared_local[(17)] = kernel_shared[(((((int)threadIdx.z) * 72) + 52))];
    kernel_shared_local[(18)] = kernel_shared[(((((int)threadIdx.z) * 72) + 55))];
    kernel_shared_local[(19)] = kernel_shared[(((((int)threadIdx.z) * 72) + 58))];
    kernel_shared_local[(20)] = kernel_shared[(((((int)threadIdx.z) * 72) + 61))];
    kernel_shared_local[(21)] = kernel_shared[(((((int)threadIdx.z) * 72) + 64))];
    kernel_shared_local[(22)] = kernel_shared[(((((int)threadIdx.z) * 72) + 67))];
    kernel_shared_local[(23)] = kernel_shared[(((((int)threadIdx.z) * 72) + 70))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(11)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(12)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(13)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(14)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(15)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(16)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(17)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(18)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(19)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(20)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(21)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(22)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(23)] * kernel_shared_local[(23)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 14))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 35))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 56))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 77))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 98))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 119))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 140))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 161))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 182))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 203))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 224))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 245))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 266))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 287))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 308))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 329))];
    pad_temp_shared_local[(16)] = pad_temp_shared[((((int)threadIdx.x) + 350))];
    pad_temp_shared_local[(17)] = pad_temp_shared[((((int)threadIdx.x) + 371))];
    pad_temp_shared_local[(18)] = pad_temp_shared[((((int)threadIdx.x) + 392))];
    pad_temp_shared_local[(19)] = pad_temp_shared[((((int)threadIdx.x) + 413))];
    pad_temp_shared_local[(20)] = pad_temp_shared[((((int)threadIdx.x) + 434))];
    pad_temp_shared_local[(21)] = pad_temp_shared[((((int)threadIdx.x) + 455))];
    pad_temp_shared_local[(22)] = pad_temp_shared[((((int)threadIdx.x) + 476))];
    pad_temp_shared_local[(23)] = pad_temp_shared[((((int)threadIdx.x) + 497))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 2))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 5))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 8))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 11))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 14))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 17))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 72) + 20))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 72) + 23))];
    kernel_shared_local[(8)] = kernel_shared[(((((int)threadIdx.z) * 72) + 26))];
    kernel_shared_local[(9)] = kernel_shared[(((((int)threadIdx.z) * 72) + 29))];
    kernel_shared_local[(10)] = kernel_shared[(((((int)threadIdx.z) * 72) + 32))];
    kernel_shared_local[(11)] = kernel_shared[(((((int)threadIdx.z) * 72) + 35))];
    kernel_shared_local[(12)] = kernel_shared[(((((int)threadIdx.z) * 72) + 38))];
    kernel_shared_local[(13)] = kernel_shared[(((((int)threadIdx.z) * 72) + 41))];
    kernel_shared_local[(14)] = kernel_shared[(((((int)threadIdx.z) * 72) + 44))];
    kernel_shared_local[(15)] = kernel_shared[(((((int)threadIdx.z) * 72) + 47))];
    kernel_shared_local[(16)] = kernel_shared[(((((int)threadIdx.z) * 72) + 50))];
    kernel_shared_local[(17)] = kernel_shared[(((((int)threadIdx.z) * 72) + 53))];
    kernel_shared_local[(18)] = kernel_shared[(((((int)threadIdx.z) * 72) + 56))];
    kernel_shared_local[(19)] = kernel_shared[(((((int)threadIdx.z) * 72) + 59))];
    kernel_shared_local[(20)] = kernel_shared[(((((int)threadIdx.z) * 72) + 62))];
    kernel_shared_local[(21)] = kernel_shared[(((((int)threadIdx.z) * 72) + 65))];
    kernel_shared_local[(22)] = kernel_shared[(((((int)threadIdx.z) * 72) + 68))];
    kernel_shared_local[(23)] = kernel_shared[(((((int)threadIdx.z) * 72) + 71))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(11)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(12)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(13)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(14)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(15)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(16)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(17)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(18)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(19)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(20)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(21)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(22)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(23)] * kernel_shared_local[(23)]));
    __syncthreads();
    pad_temp_shared[(((((int)threadIdx.z) * 63) + (((int)threadIdx.x) * 9)))] = ((((1 <= ((((((int)threadIdx.x) * 9) % 21) / 7) + ((int)blockIdx.y))) && (((((((int)threadIdx.x) * 9) % 21) / 7) + ((int)blockIdx.y)) < 8)) && (((((int)threadIdx.x) * 9) % 7) < 6)) ? data[(((((((rc_outer * 1176) + (((int)threadIdx.z) * 147)) + (((((int)threadIdx.x) * 9) / 21) * 49)) + (((int)blockIdx.y) * 7)) + ((((int)threadIdx.x) * 9) % 21)) - 6))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 63) + (((int)threadIdx.x) * 9)) + 1))] = ((((1 <= (((((((int)threadIdx.x) * 9) + 1) % 21) / 7) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 9) + 1) % 21) / 7) + ((int)blockIdx.y)) < 8)) && ((((((int)threadIdx.x) * 9) + 1) % 7) < 6)) ? data[(((((((rc_outer * 1176) + (((int)threadIdx.z) * 147)) + ((((((int)threadIdx.x) * 9) + 1) / 21) * 49)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 9) + 1) % 21)) - 6))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 63) + (((int)threadIdx.x) * 9)) + 2))] = ((((1 <= (((((((int)threadIdx.x) * 9) + 2) % 21) / 7) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 9) + 2) % 21) / 7) + ((int)blockIdx.y)) < 8)) && ((((((int)threadIdx.x) * 9) + 2) % 7) < 6)) ? data[(((((((rc_outer * 1176) + (((int)threadIdx.z) * 147)) + ((((((int)threadIdx.x) * 9) + 2) / 21) * 49)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 9) + 2) % 21)) - 6))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 63) + (((int)threadIdx.x) * 9)) + 3))] = ((((1 <= (((((((int)threadIdx.x) * 9) + 3) % 21) / 7) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 9) + 3) % 21) / 7) + ((int)blockIdx.y)) < 8)) && ((((((int)threadIdx.x) * 9) + 3) % 7) < 6)) ? data[(((((((rc_outer * 1176) + (((int)threadIdx.z) * 147)) + ((((((int)threadIdx.x) * 9) + 3) / 21) * 49)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 9) + 3) % 21)) - 6))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 63) + (((int)threadIdx.x) * 9)) + 4))] = ((((1 <= (((((((int)threadIdx.x) * 9) + 4) % 21) / 7) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 9) + 4) % 21) / 7) + ((int)blockIdx.y)) < 8)) && ((((((int)threadIdx.x) * 9) + 4) % 7) < 6)) ? data[(((((((rc_outer * 1176) + (((int)threadIdx.z) * 147)) + ((((((int)threadIdx.x) * 9) + 4) / 21) * 49)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 9) + 4) % 21)) - 6))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 63) + (((int)threadIdx.x) * 9)) + 5))] = ((((1 <= (((((((int)threadIdx.x) * 9) + 5) % 21) / 7) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 9) + 5) % 21) / 7) + ((int)blockIdx.y)) < 8)) && ((((((int)threadIdx.x) * 9) + 5) % 7) < 6)) ? data[(((((((rc_outer * 1176) + (((int)threadIdx.z) * 147)) + ((((((int)threadIdx.x) * 9) + 5) / 21) * 49)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 9) + 5) % 21)) - 6))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 63) + (((int)threadIdx.x) * 9)) + 6))] = ((((1 <= (((((((int)threadIdx.x) * 9) + 6) % 21) / 7) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 9) + 6) % 21) / 7) + ((int)blockIdx.y)) < 8)) && ((((((int)threadIdx.x) * 9) + 6) % 7) < 6)) ? data[(((((((rc_outer * 1176) + (((int)threadIdx.z) * 147)) + ((((((int)threadIdx.x) * 9) + 6) / 21) * 49)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 9) + 6) % 21)) - 6))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 63) + (((int)threadIdx.x) * 9)) + 7))] = ((((1 <= (((((((int)threadIdx.x) * 9) + 7) % 21) / 7) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 9) + 7) % 21) / 7) + ((int)blockIdx.y)) < 8)) && (((((int)threadIdx.x) * 9) % 7) < 6)) ? data[(((((((rc_outer * 1176) + (((int)threadIdx.z) * 147)) + ((((((int)threadIdx.x) * 9) + 7) / 21) * 49)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 9) + 7) % 21)) - 6))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 63) + (((int)threadIdx.x) * 9)) + 8))] = ((((1 <= (((((((int)threadIdx.x) * 9) + 8) % 21) / 7) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 9) + 8) % 21) / 7) + ((int)blockIdx.y)) < 8)) && ((((((int)threadIdx.x) * 9) + 1) % 7) < 6)) ? data[(((((((rc_outer * 1176) + (((int)threadIdx.z) * 147)) + ((((((int)threadIdx.x) * 9) + 8) / 21) * 49)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 9) + 8) % 21)) - 6))] : 0.000000e+00f);
    kernel_shared[(((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 2))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 1))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 5))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 2))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 8))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 3))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 11))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 4))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 14))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 5))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 17))];
    if (((((((int)threadIdx.x) * 11) + 6) / 72) + ((int)threadIdx.z)) < 8) {
      if (((((int)threadIdx.z) * 24) + ((((int)threadIdx.x) * 11) / 3)) < 190) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 570) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 6))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 20))];
          }
        }
      }
    }
    if (((((((int)threadIdx.x) * 11) + 7) / 72) + ((int)threadIdx.z)) < 8) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.x) * 11) + 7) / 3)) < 192) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 569) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 7))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 23))];
          }
        }
      }
    }
    if (((((((int)threadIdx.x) * 11) + 8) / 72) + ((int)threadIdx.z)) < 8) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.x) * 11) + 8) / 3)) < 192) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 568) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 8))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 26))];
          }
        }
      }
    }
    if (((((((int)threadIdx.x) * 11) + 9) / 72) + ((int)threadIdx.z)) < 8) {
      if (((((int)threadIdx.z) * 24) + ((((int)threadIdx.x) * 11) / 3)) < 189) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 567) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 9))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 29))];
          }
        }
      }
    }
    if (((((((int)threadIdx.x) * 11) + 10) / 72) + ((int)threadIdx.z)) < 8) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.x) * 11) + 10) / 3)) < 192) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 566) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 10))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 32))];
          }
        }
      }
    }
    __syncthreads();
    pad_temp_shared_local[(0)] = pad_temp_shared[(((int)threadIdx.x))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 21))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 42))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 63))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 84))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 105))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 126))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 147))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 168))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 189))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 210))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 231))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 252))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 273))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 294))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 315))];
    pad_temp_shared_local[(16)] = pad_temp_shared[((((int)threadIdx.x) + 336))];
    pad_temp_shared_local[(17)] = pad_temp_shared[((((int)threadIdx.x) + 357))];
    pad_temp_shared_local[(18)] = pad_temp_shared[((((int)threadIdx.x) + 378))];
    pad_temp_shared_local[(19)] = pad_temp_shared[((((int)threadIdx.x) + 399))];
    pad_temp_shared_local[(20)] = pad_temp_shared[((((int)threadIdx.x) + 420))];
    pad_temp_shared_local[(21)] = pad_temp_shared[((((int)threadIdx.x) + 441))];
    pad_temp_shared_local[(22)] = pad_temp_shared[((((int)threadIdx.x) + 462))];
    pad_temp_shared_local[(23)] = pad_temp_shared[((((int)threadIdx.x) + 483))];
    kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 72))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 3))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 6))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 9))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 12))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 15))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 72) + 18))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 72) + 21))];
    kernel_shared_local[(8)] = kernel_shared[(((((int)threadIdx.z) * 72) + 24))];
    kernel_shared_local[(9)] = kernel_shared[(((((int)threadIdx.z) * 72) + 27))];
    kernel_shared_local[(10)] = kernel_shared[(((((int)threadIdx.z) * 72) + 30))];
    kernel_shared_local[(11)] = kernel_shared[(((((int)threadIdx.z) * 72) + 33))];
    kernel_shared_local[(12)] = kernel_shared[(((((int)threadIdx.z) * 72) + 36))];
    kernel_shared_local[(13)] = kernel_shared[(((((int)threadIdx.z) * 72) + 39))];
    kernel_shared_local[(14)] = kernel_shared[(((((int)threadIdx.z) * 72) + 42))];
    kernel_shared_local[(15)] = kernel_shared[(((((int)threadIdx.z) * 72) + 45))];
    kernel_shared_local[(16)] = kernel_shared[(((((int)threadIdx.z) * 72) + 48))];
    kernel_shared_local[(17)] = kernel_shared[(((((int)threadIdx.z) * 72) + 51))];
    kernel_shared_local[(18)] = kernel_shared[(((((int)threadIdx.z) * 72) + 54))];
    kernel_shared_local[(19)] = kernel_shared[(((((int)threadIdx.z) * 72) + 57))];
    kernel_shared_local[(20)] = kernel_shared[(((((int)threadIdx.z) * 72) + 60))];
    kernel_shared_local[(21)] = kernel_shared[(((((int)threadIdx.z) * 72) + 63))];
    kernel_shared_local[(22)] = kernel_shared[(((((int)threadIdx.z) * 72) + 66))];
    kernel_shared_local[(23)] = kernel_shared[(((((int)threadIdx.z) * 72) + 69))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(11)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(12)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(13)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(14)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(15)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(16)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(17)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(18)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(19)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(20)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(21)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(22)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(23)] * kernel_shared_local[(23)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 7))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 28))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 49))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 70))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 91))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 112))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 133))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 154))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 175))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 196))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 217))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 238))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 259))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 280))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 301))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 322))];
    pad_temp_shared_local[(16)] = pad_temp_shared[((((int)threadIdx.x) + 343))];
    pad_temp_shared_local[(17)] = pad_temp_shared[((((int)threadIdx.x) + 364))];
    pad_temp_shared_local[(18)] = pad_temp_shared[((((int)threadIdx.x) + 385))];
    pad_temp_shared_local[(19)] = pad_temp_shared[((((int)threadIdx.x) + 406))];
    pad_temp_shared_local[(20)] = pad_temp_shared[((((int)threadIdx.x) + 427))];
    pad_temp_shared_local[(21)] = pad_temp_shared[((((int)threadIdx.x) + 448))];
    pad_temp_shared_local[(22)] = pad_temp_shared[((((int)threadIdx.x) + 469))];
    pad_temp_shared_local[(23)] = pad_temp_shared[((((int)threadIdx.x) + 490))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 1))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 4))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 7))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 10))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 13))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 16))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 72) + 19))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 72) + 22))];
    kernel_shared_local[(8)] = kernel_shared[(((((int)threadIdx.z) * 72) + 25))];
    kernel_shared_local[(9)] = kernel_shared[(((((int)threadIdx.z) * 72) + 28))];
    kernel_shared_local[(10)] = kernel_shared[(((((int)threadIdx.z) * 72) + 31))];
    kernel_shared_local[(11)] = kernel_shared[(((((int)threadIdx.z) * 72) + 34))];
    kernel_shared_local[(12)] = kernel_shared[(((((int)threadIdx.z) * 72) + 37))];
    kernel_shared_local[(13)] = kernel_shared[(((((int)threadIdx.z) * 72) + 40))];
    kernel_shared_local[(14)] = kernel_shared[(((((int)threadIdx.z) * 72) + 43))];
    kernel_shared_local[(15)] = kernel_shared[(((((int)threadIdx.z) * 72) + 46))];
    kernel_shared_local[(16)] = kernel_shared[(((((int)threadIdx.z) * 72) + 49))];
    kernel_shared_local[(17)] = kernel_shared[(((((int)threadIdx.z) * 72) + 52))];
    kernel_shared_local[(18)] = kernel_shared[(((((int)threadIdx.z) * 72) + 55))];
    kernel_shared_local[(19)] = kernel_shared[(((((int)threadIdx.z) * 72) + 58))];
    kernel_shared_local[(20)] = kernel_shared[(((((int)threadIdx.z) * 72) + 61))];
    kernel_shared_local[(21)] = kernel_shared[(((((int)threadIdx.z) * 72) + 64))];
    kernel_shared_local[(22)] = kernel_shared[(((((int)threadIdx.z) * 72) + 67))];
    kernel_shared_local[(23)] = kernel_shared[(((((int)threadIdx.z) * 72) + 70))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(11)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(12)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(13)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(14)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(15)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(16)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(17)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(18)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(19)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(20)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(21)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(22)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(23)] * kernel_shared_local[(23)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 14))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 35))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 56))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 77))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 98))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 119))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 140))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 161))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 182))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 203))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 224))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 245))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 266))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 287))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 308))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 329))];
    pad_temp_shared_local[(16)] = pad_temp_shared[((((int)threadIdx.x) + 350))];
    pad_temp_shared_local[(17)] = pad_temp_shared[((((int)threadIdx.x) + 371))];
    pad_temp_shared_local[(18)] = pad_temp_shared[((((int)threadIdx.x) + 392))];
    pad_temp_shared_local[(19)] = pad_temp_shared[((((int)threadIdx.x) + 413))];
    pad_temp_shared_local[(20)] = pad_temp_shared[((((int)threadIdx.x) + 434))];
    pad_temp_shared_local[(21)] = pad_temp_shared[((((int)threadIdx.x) + 455))];
    pad_temp_shared_local[(22)] = pad_temp_shared[((((int)threadIdx.x) + 476))];
    pad_temp_shared_local[(23)] = pad_temp_shared[((((int)threadIdx.x) + 497))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 2))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 5))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 8))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 11))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 14))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 17))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 72) + 20))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 72) + 23))];
    kernel_shared_local[(8)] = kernel_shared[(((((int)threadIdx.z) * 72) + 26))];
    kernel_shared_local[(9)] = kernel_shared[(((((int)threadIdx.z) * 72) + 29))];
    kernel_shared_local[(10)] = kernel_shared[(((((int)threadIdx.z) * 72) + 32))];
    kernel_shared_local[(11)] = kernel_shared[(((((int)threadIdx.z) * 72) + 35))];
    kernel_shared_local[(12)] = kernel_shared[(((((int)threadIdx.z) * 72) + 38))];
    kernel_shared_local[(13)] = kernel_shared[(((((int)threadIdx.z) * 72) + 41))];
    kernel_shared_local[(14)] = kernel_shared[(((((int)threadIdx.z) * 72) + 44))];
    kernel_shared_local[(15)] = kernel_shared[(((((int)threadIdx.z) * 72) + 47))];
    kernel_shared_local[(16)] = kernel_shared[(((((int)threadIdx.z) * 72) + 50))];
    kernel_shared_local[(17)] = kernel_shared[(((((int)threadIdx.z) * 72) + 53))];
    kernel_shared_local[(18)] = kernel_shared[(((((int)threadIdx.z) * 72) + 56))];
    kernel_shared_local[(19)] = kernel_shared[(((((int)threadIdx.z) * 72) + 59))];
    kernel_shared_local[(20)] = kernel_shared[(((((int)threadIdx.z) * 72) + 62))];
    kernel_shared_local[(21)] = kernel_shared[(((((int)threadIdx.z) * 72) + 65))];
    kernel_shared_local[(22)] = kernel_shared[(((((int)threadIdx.z) * 72) + 68))];
    kernel_shared_local[(23)] = kernel_shared[(((((int)threadIdx.z) * 72) + 71))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(11)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(12)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(13)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(14)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(15)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(16)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(17)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(18)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(19)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(20)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(21)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(22)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(23)] * kernel_shared_local[(23)]));
  }
  compute[(((((((int)blockIdx.z) * 392) + (((int)threadIdx.z) * 49)) + (((int)blockIdx.y) * 7)) + ((int)threadIdx.x)))] = compute_local[(0)];
}






class ConvGemm{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvGemm::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvGemm::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}

class ConvWinogradeNon{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvWinogradeNon::initialize(){
    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvWinogradeNon::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvFFT{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvFFT::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvFFT::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
__device__ void load_input_2_shared_memory(float *input, float *shared_input, unsigned int h_start,
                                           unsigned int h_end, unsigned int h_offset, unsigned int c_start,
                                           unsigned int warp_id, unsigned int lane_id, unsigned int warp_size){
    switch(h_offset){
        case 0:
            for(unsigned int c = warp_id; c<TC; c+=TWS){
                for(unsigned int i=lane_id; i<(h_end - h_start) * W; i+=warp_size){
                    unsigned int r = i/W;
                    unsigned int s = i%W;
                    shared_input[c*(TH + 2)*(WPAD) + r * WPAD + s + 1] = input[(c_start + c) * H * W + h_start * W + i];
                }
            }
            break;
        case 1:
            for(unsigned int c = warp_id; c<TC; c+=TWS){
                for(unsigned int i=lane_id; i<(h_end - h_start) * W; i+=warp_size){
                    unsigned int r = i/W;
                    unsigned int s = i%W;
                    shared_input[c*(TH + 2)*(WPAD) + (1 + r) * WPAD + s + 1] = input[(c_start + c) * H * W + h_start * W + i];
                }
            }
            break;
    }
}
__device__ __forceinline__ void switch_write_back(unsigned int write_h, unsigned int write_w, unsigned int h_out_start, unsigned int w_out_start, unsigned int n, float * outputs, float * temp_result){
	switch(write_h){
		case 1: 
 		switch(write_w){
			case 1:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 1; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
		} 
		break;
		case 2: 
 		switch(write_w){
			case 1:
 			#pragma unroll
			for (unsigned int th = 0; th < 2; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 1; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
		} 
		break;
		case 3: 
 		switch(write_w){
			case 1:
 			#pragma unroll
			for (unsigned int th = 0; th < 3; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 1; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
		} 
		break;
		case 4: 
 		switch(write_w){
			case 1:
 			#pragma unroll
			for (unsigned int th = 0; th < 4; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 1; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
		} 
		break;
	}
}
__global__ void conv2d(float * __restrict__ input,const float * __restrict__ kernel, float * __restrict__ outputs){
    extern __shared__ float shared_input[];
    const unsigned int tile_id = blockIdx.x;
    const unsigned int tc_id = tile_id / THS;
    const unsigned int th_id = tile_id % THS;
    const unsigned int tw_id = threadIdx.x / N;
    const int h_out_start = th_id * TH;
    const int w_out_start = tw_id * TW;
    const unsigned int warp_id = tw_id;
    const unsigned int lane_id = threadIdx.x % N;
    float data_array[9];
    float temp_result[TH*TW] = {0.0f};
    for(unsigned int i=threadIdx.x;i<TC*(TH+2)*WPAD;i+=blockDim.x){
        shared_input[i] = 0.0f;
    }
    unsigned int n = lane_id;
    unsigned int c_offset = tc_id * TC;
    int h_offset = (h_out_start == 0)?1:0;
    int h_padded_start = h_out_start;
    int h_padded_end = min(h_padded_start + TH + 2, H + 2);
    int h_non_padded_start = max(h_out_start - 1, 0);
    int h_non_padded_end = min(H, h_padded_end - 1);
    __syncthreads();
    load_input_2_shared_memory(input, shared_input, h_non_padded_start, h_non_padded_end, h_offset, c_offset, warp_id, lane_id, N);
    __syncthreads();
#pragma unroll
    for(unsigned int c=0;c<TC;c++){
#pragma unroll
        for(unsigned int r=0;r<R;++r){
#pragma unroll
            for(unsigned int s=0;s<S;++s){
                data_array[r*S+s] = kernel[(c + c_offset)*N*9+r*3*N+s*N+n];
            }
        }
        		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 0]*data_array[0];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 1]*data_array[1];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 2]*data_array[2];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 0]*data_array[0];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 0]*data_array[3];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[1];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[4];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[2];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[5];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 0]*data_array[0];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 0]*data_array[3];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 0]*data_array[6];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[1];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[4];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[7];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[2];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[5];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[8];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 0]*data_array[0];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 0]*data_array[3];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 0]*data_array[6];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 1]*data_array[1];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 1]*data_array[4];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 1]*data_array[7];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 2]*data_array[2];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 2]*data_array[5];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 2]*data_array[8];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 4 * WPAD + tw_id * TW + 0]*data_array[3];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 4 * WPAD + tw_id * TW + 0]*data_array[6];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 4 * WPAD + tw_id * TW + 1]*data_array[4];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 4 * WPAD + tw_id * TW + 1]*data_array[7];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 4 * WPAD + tw_id * TW + 2]*data_array[5];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 4 * WPAD + tw_id * TW + 2]*data_array[8];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 5 * WPAD + tw_id * TW + 0]*data_array[6];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 5 * WPAD + tw_id * TW + 1]*data_array[7];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 5 * WPAD + tw_id * TW + 2]*data_array[8];

    }
    switch_write_back(min(TH, H - h_out_start), min(TW, W - w_out_start), h_out_start, w_out_start, n, outputs, temp_result);
}

float check_diff(float *x, float *y, unsigned int size){
    float diff = 0.0f;
#pragma omp parallel for reduction(+ : diff)
    for(unsigned int i=0;i<size;++i){
        diff += abs(x[i] - y[i]);
    }
    return diff;
}
int main(void){
    float *input = new float[C*H*W];
    time_t t;
    float *matrix;
    hipMalloc(&matrix,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    hipMemset(matrix,0,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    srand((unsigned) time(&t));
    for(int i =0;i<C*H*W;++i){
        input[i] = rand() % 10;
    }
    float *device_input;
    hipMalloc(&device_input,C*H*W*sizeof(float));
    hipMemcpy(device_input,input,C*H*W*sizeof(float),hipMemcpyHostToDevice);
    float *K = new float[C*N*9];
    for(int i=0;i<C*N*9;++i){
        K[i] = 1.0f;
    }

    ConvGemm convGemm;
    convGemm.initialize();
    ConvWinogradeNon convWinogradeNon;
    convWinogradeNon.initialize();
    ConvFFT convFFT;
    convFFT.initialize();

    float *out_cudnn;
    float *out_cudnn_host = new float[N*H*W];
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    out_cudnn = convGemm.forward(device_input);
    hipMemcpy(out_cudnn_host,out_cudnn,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    out_cudnn = convFFT.forward(device_input);
    out_cudnn = convWinogradeNon.forward(device_input);

    float *device_K;
    float *device_out;
    hipMalloc(&device_out,H*W*N*sizeof(float));
    hipMemset(device_out,0,H*W*N*sizeof(float));
    hipMalloc(&device_K,C*N*9*sizeof(float));
    hipMemcpy(device_K,K,C*N*9*sizeof(float),hipMemcpyHostToDevice);

    hipEventRecord(event_start);
    convGemm.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnGemmTime;
    hipEventElapsedTime(&cudnnGemmTime, event_start, event_stop);


    hipEventRecord(event_start);
    convWinogradeNon.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnWinogradeTimeNon;
    hipEventElapsedTime(&cudnnWinogradeTimeNon, event_start, event_stop);

    hipEventRecord(event_start);
    convFFT.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnFFTTime;
    hipEventElapsedTime(&cudnnFFTTime, event_start, event_stop);


        dim3 grid(1,7,8);

                dim3 block(7,1,8);

    hipEventRecord(event_start);
    default_function_kernel0<<<grid, block>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tvm;
    hipEventElapsedTime(&time_tvm, event_start, event_stop);
    float *out_tvm = new float[N*H*W];
    hipMemcpy(out_tvm,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    hipMemset(device_out, 0, sizeof(float)*N*H*W);

    chkerr(hipFuncSetAttribute(reinterpret_cast<const void*>(conv2d),hipFuncAttributeMaxDynamicSharedMemorySize, TC*(TH+2)*(WPAD)*4));
    hipEventRecord(event_start);
    conv2d<<<TCS*THS, N * TWS, TC*(TH+2)*(WPAD)*4>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tdc;
    hipEventElapsedTime(&time_tdc, event_start, event_stop);
    float *out_tdc = new float[N*H*W];
    hipMemcpy(out_tdc,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);

    ofstream outfile;
    char buffer[1000];
    int ret = sprintf(buffer,"%d,%d,%d,%d,%f,%f,%f,%f,%f,%f,%f,%f,%f\n",N,C,H,W,
            cudnnFFTTime,cudnnWinogradeTimeNon,cudnnGemmTime,time_tvm,time_tdc,
            cudnnFFTTime/time_tdc,cudnnWinogradeTimeNon/time_tdc,cudnnGemmTime/time_tdc,time_tvm/time_tdc);
    outfile.open("../../evaluation_outcome/A100-layers-eval-oracle.csv", std::ios_base::app);
    outfile << buffer;


    float difference = check_diff(out_tvm, out_tdc, N*H*W);
    cout<<N<<","<<C<<","<<H<<","<<W<<","<<cudnnFFTTime<<","<<cudnnWinogradeTimeNon<<","<<cudnnGemmTime<<","<<
                                   time_tvm<<","<<time_tdc<<","<<cudnnFFTTime/time_tdc<<","<<cudnnWinogradeTimeNon/time_tdc<<","<<
                                   cudnnGemmTime/time_tdc<<","<<time_tvm/time_tdc<<","<<difference<<endl;
    return 0;
}


